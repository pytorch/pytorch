#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <assert.h>

using namespace at;

__device__ void test(){
  // test half construction and implicit conversions in device
  assert(Half(3) == Half(3.0f));
  assert(static_cast<Half>(3.0f) == Half(3.0f));
  // there is no float <=> __half implicit conversion
  assert(static_cast<Half>(3.0f) == 3.0f);

  __half a = __float2half(3.0f);
  __half b = __float2half(2.0f);
  __half c = Half(a) - Half(b);
  assert(static_cast<Half>(c) == Half(1.0));

  // asserting if the  functions used on
  // half types give almost equivalent results when using
  //  functions on double.
  // The purpose of these asserts are to test the device side
  // half API for the common mathematical functions.
  // Note: When calling std math functions from device, don't
  // use the std namespace, but just "::" so that the function
  // gets resolved from nvcc math_functions.hpp

  [[maybe_unused]] float threshold = 0.00001;
  assert(::abs(::lgamma(Half(10.0)) - ::lgamma(10.0f)) <= threshold);
  assert(::abs(::exp(Half(1.0)) - ::exp(1.0f)) <= threshold);
  assert(::abs(::log(Half(1.0)) - ::log(1.0f)) <= threshold);
  assert(::abs(::log10(Half(1000.0)) - ::log10(1000.0f)) <= threshold);
  assert(::abs(::log1p(Half(0.0)) - ::log1p(0.0f)) <= threshold);
  assert(::abs(::log2(Half(1000.0)) - ::log2(1000.0f)) <= threshold);
  assert(::abs(::expm1(Half(1.0)) - ::expm1(1.0f)) <= threshold);
  assert(::abs(::cos(Half(0.0)) - ::cos(0.0f)) <= threshold);
  assert(::abs(::sin(Half(0.0)) - ::sin(0.0f)) <= threshold);
  assert(::abs(::sqrt(Half(100.0)) - ::sqrt(100.0f)) <= threshold);
  assert(::abs(::ceil(Half(2.4)) - ::ceil(2.4f)) <= threshold);
  assert(::abs(::floor(Half(2.7)) - ::floor(2.7f)) <= threshold);
  assert(::abs(::trunc(Half(2.7)) - ::trunc(2.7f)) <= threshold);
  assert(::abs(::acos(Half(-1.0)) - ::acos(-1.0f)) <= threshold);
  assert(::abs(::cosh(Half(1.0)) - ::cosh(1.0f)) <= threshold);
  assert(::abs(::acosh(Half(1.0)) - ::acosh(1.0f)) <= threshold);
  assert(::abs(::acosh(Half(1.0)) - ::acosh(1.0f)) <= threshold);
  assert(::abs(::asinh(Half(1.0)) - ::asinh(1.0f)) <= threshold);
  // See note below about VC++ and isinf
#ifndef  _MSC_VER
  assert(::isinf(::atanh(Half(1.0))));
#endif
  assert(::abs(::atanh(Half(.5)) - ::atanh(.5f)) <= threshold);
  assert(::abs(::asin(Half(1.0)) - ::asin(1.0f)) <= threshold);
  assert(::abs(::sinh(Half(1.0)) - ::sinh(1.0f)) <= threshold);
  assert(::abs(::asinh(Half(1.0)) - ::asinh(1.0f)) <= threshold);
  assert(::abs(::tan(Half(0.0)) - ::tan(0.0f)) <= threshold);
  assert(::abs(::atan(Half(1.0)) - ::atan(1.0f)) <= threshold);
  assert(::abs(::tanh(Half(1.0)) - ::tanh(1.0f)) <= threshold);
  assert(::abs(::erf(Half(10.0)) - ::erf(10.0f)) <= threshold);
  assert(::abs(::erfc(Half(10.0)) - ::erfc(10.0f)) <= threshold);
  assert(::abs(::abs(Half(-3.0)) - ::abs(-3.0f)) <= threshold);
  assert(::abs(::round(Half(2.3)) - ::round(2.3f)) <= threshold);
  assert(::abs(::pow(Half(2.0), Half(10.0)) - ::pow(2.0f, 10.0f)) <= threshold);
  assert(
      ::abs(::atan2(Half(7.0), Half(0.0)) - ::atan2(7.0f, 0.0f)) <= threshold);
  // note: can't use  namespace on isnan and isinf in device code

  // Windows requires this explicit conversion. The reason is unclear
  // related issue with clang: https://reviews.llvm.org/D37906
#ifndef _MSC_VER
  assert(::abs(::isnan(Half(0.0)) - ::isnan(0.0f)) <= threshold);
  assert(::abs(::isinf(Half(0.0)) - ::isinf(0.0f)) <= threshold);
#endif

  // test complex<32>
  Half real = 3.0f;
  Half imag = -10.0f;
  auto complex = c10::complex<Half>(real, imag);
  assert(complex.real() == real);
  assert(complex.imag() == imag);
}

__global__ void kernel(){
  test();
}

void launch_function(){
  kernel<<<1, 1>>>();
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

// half common math functions tests in device
TEST(HalfCuda, HalfCuda) {
  if (!at::cuda::is_available()) return;
  launch_function();
  hipError_t err = hipDeviceSynchronize();
  bool isEQ = err == hipSuccess;
  ASSERT_TRUE(isEQ);
}
