#include "hip/hip_runtime.h"
#include <ATen/cuda/CUDABlas.h>
#include <c10/cuda/CUDAException.h>
#include <c10/test/util/complex_test_common.h>

__global__ void test_thrust_kernel() {
  // thrust conversion
  {
  [[maybe_unused]] constexpr float num1 = float(1.23);
  [[maybe_unused]] constexpr float num2 = float(4.56);
  assert(c10::complex<float>(thrust::complex<float>(num1, num2)).real() == num1);
  assert(c10::complex<float>(thrust::complex<float>(num1, num2)).imag() == num2);
  }
  {
  [[maybe_unused]] constexpr double num1 = double(1.23);
  [[maybe_unused]] constexpr double num2 = double(4.56);
  assert(c10::complex<double>(thrust::complex<double>(num1, num2)).real() == num1);
  assert(c10::complex<double>(thrust::complex<double>(num1, num2)).imag() == num2);
  }
  // thrust assignment
  auto tup = assignment::one_two_thrust();
  assert(std::get<c10::complex<double>>(tup).real() == double(1));
  assert(std::get<c10::complex<double>>(tup).imag() == double(2));
  assert(std::get<c10::complex<float>>(tup).real() == float(1));
  assert(std::get<c10::complex<float>>(tup).imag() == float(2));
}

__global__ void test_std_functions_kernel() {
  assert(std::abs(c10::complex<float>(3, 4)) == float(5));
  assert(std::abs(c10::complex<double>(3, 4)) == double(5));

  assert(std::abs(std::arg(c10::complex<float>(0, 1)) - PI / 2) < 1e-6);
  assert(std::abs(std::arg(c10::complex<double>(0, 1)) - PI / 2) < 1e-6);

  assert(std::abs(c10::polar(float(1), float(PI / 2)) - c10::complex<float>(0, 1)) < 1e-6);
  assert(std::abs(c10::polar(double(1), double(PI / 2)) - c10::complex<double>(0, 1)) < 1e-6);
}

__global__ void test_reinterpret_cast() {
  std::complex<float> z(1, 2);
  c10::complex<float> zz = *reinterpret_cast<c10::complex<float>*>(&z);
  assert(zz.real() == float(1));
  assert(zz.imag() == float(2));

  std::complex<double> zzz(1, 2);
  c10::complex<double> zzzz = *reinterpret_cast<c10::complex<double>*>(&zzz);
  assert(zzzz.real() == double(1));
  assert(zzzz.imag() == double(2));

  [[maybe_unused]] hipComplex cuComplex_zz = *reinterpret_cast<hipComplex*>(&zz);
  assert(cuComplex_zz.x == float(1));
  assert(cuComplex_zz.y == float(2));

  [[maybe_unused]] hipDoubleComplex cuDoubleComplex_zzzz = *reinterpret_cast<hipDoubleComplex*>(&zzzz);
  assert(cuDoubleComplex_zzzz.x == double(1));
  assert(cuDoubleComplex_zzzz.y == double(2));
}

int safeDeviceCount() {
  int count;
  hipError_t err = hipGetDeviceCount(&count);
  if (err == hipErrorInsufficientDriver || err == hipErrorNoDevice) {
    return 0;
  }
  return count;
}

#define SKIP_IF_NO_GPU()                    \
  do {                                      \
    if (safeDeviceCount() == 0) {           \
      return;                               \
    }                                       \
  } while(0)

TEST(DeviceTests, ThrustConversion) {
  SKIP_IF_NO_GPU();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  hipDeviceSynchronize();
  test_thrust_kernel<<<1, 1>>>();
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
}

TEST(DeviceTests, StdFunctions) {
  SKIP_IF_NO_GPU();
  hipDeviceSynchronize();
  test_std_functions_kernel<<<1, 1>>>();
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
}

TEST(DeviceTests, ReinterpretCast) {
  SKIP_IF_NO_GPU();
  hipDeviceSynchronize();
  test_reinterpret_cast<<<1, 1>>>();
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
}
