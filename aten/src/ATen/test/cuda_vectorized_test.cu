#include "hip/hip_runtime.h"
#include <array>
#include <gtest/gtest.h>
#include <ATen/ATen.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/cuda/HIPContext.h>

using namespace at::native;
using namespace at::native::memory;

constexpr int buffer_size = 1024;

__managed__ double4 buffer1[buffer_size];
__managed__ double4 buffer2[buffer_size];

void reset_buffers() {
  for (int i = 0; i < buffer_size; i++) {
    buffer1[i].x = i;
    buffer1[i].y = i + 0.1;
    buffer1[i].z = i + 0.2;
    buffer1[i].w = i + 0.3;

    buffer2[2].x = -i;
    buffer2[2].y = -(i + 0.1);
    buffer2[2].z = -(i + 0.2);
    buffer2[2].w = -(i + 0.3);
  }
}

#if defined(USE_ROCM) && !defined(_WIN32)
TEST(TestLoops, HasSameArgTypes) {
  // This is a compile-time unit test. If this file compiles without error,
  // then the test passes and during runtime, we just need to return.
  using namespace at::native::modern::detail;
  using func1_t = int (*)(float, float);
  using func2_t = int (*)(bool, float, float);
  using func3_t = int (*)(float);
  using func4_t = int (*)();
  static_assert(has_same_arg_types<func1_t>::value, "func1_t has the same argument types");
  static_assert(!has_same_arg_types<func2_t>::value, "func2_t does not have the same argument types");
  static_assert(has_same_arg_types<func3_t>::value, "func3_t has the same argument types");
  static_assert(has_same_arg_types<func4_t>::value, "func4_t has the same argument types");
  return;
}
#endif

TEST(TestVectorizedMemoryAccess, CanVectorizeUpTo) {
  char *ptr = reinterpret_cast<char *>(buffer1);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr), 8);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr), 8);
  ASSERT_EQ(memory::can_vectorize_up_to<int16_t>(ptr), 8);
  ASSERT_EQ(memory::can_vectorize_up_to<int>(ptr), 8);
  ASSERT_EQ(memory::can_vectorize_up_to<int64_t>(ptr), 8);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr + 1), 1);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr + 1), 1);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr + 2), 2);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr + 2), 2);
  ASSERT_EQ(memory::can_vectorize_up_to<int16_t>(ptr + 2), 1);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr + 4), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr + 4), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int16_t>(ptr + 4), 2);
  ASSERT_EQ(memory::can_vectorize_up_to<int>(ptr + 4), 1);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr + 8), 8);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr + 8), 8);
  ASSERT_EQ(memory::can_vectorize_up_to<int16_t>(ptr + 8), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int>(ptr + 8), 2);
  ASSERT_EQ(memory::can_vectorize_up_to<int64_t>(ptr + 8), 1);
}

// The following kernel copy values by using vectorized policies
// defined in `ATen/native/cuda/MemoryAccess.cuh`
template <typename scalar_t, int vec_size>
__global__ void vectorized_copy(scalar_t *dst, scalar_t *src) {
  static_assert(vec_size <= thread_work_size() && thread_work_size() % vec_size == 0, "Invalid vec_size");
  using array_t = std::array<char*, 2>;
  array_t data;
  data[0] = reinterpret_cast<char *>(dst);
  data[1] = reinterpret_cast<char *>(src);
  int idx = blockIdx.x;
  using vectorized = policies::vectorized<vec_size, array_t, thread_work_size()>;
  auto policy = vectorized(data);
  scalar_t buf[thread_work_size()];
#if !defined(USE_ROCM)
  // This fails only on CUDA 10.x, remove this after CUDA 10.x support is dropped
  scalar_t *buf_ = &buf[0];
  auto accessor = [&](int index) -> scalar_t & { return buf_[index]; };
#else
  auto accessor = [&](int index) -> scalar_t & { return buf[index]; };
#endif
  policy.load_single_arg(accessor, src + block_work_size() * blockIdx.x);
  policy.store(buf, idx);
}

TEST(TestVectorizedMemoryAccess, CopyKernel) {
  if (!at::cuda::is_available()) {
    return;
  }

  double *b1 = reinterpret_cast<double *>(buffer1);
  double *b2 = reinterpret_cast<double *>(buffer2);

  // vec4 copy
  reset_buffers();
  hipDeviceSynchronize();
  constexpr int total_work_size = buffer_size * 4;
  vectorized_copy<double, 4><<<total_work_size / block_work_size() , num_threads()>>>(b2, b1);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  ASSERT_EQ(hipSuccess, hipDeviceSynchronize());
  for (int i = 0; i < 1024; i++) {
    ASSERT_EQ(buffer1[i].x, buffer2[i].x);
    ASSERT_EQ(buffer1[i].y, buffer2[i].y);
    ASSERT_EQ(buffer1[i].z, buffer2[i].z);
    ASSERT_EQ(buffer1[i].w, buffer2[i].w);
  }

  // vec2 copy
  reset_buffers();
  hipDeviceSynchronize();
  vectorized_copy<double, 2><<<total_work_size / block_work_size() , num_threads()>>>(b2, b1);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  ASSERT_EQ(hipSuccess, hipDeviceSynchronize());
  for (int i = 0; i < 1024; i++) {
    ASSERT_EQ(buffer1[i].x, buffer2[i].x);
    ASSERT_EQ(buffer1[i].y, buffer2[i].y);
    ASSERT_EQ(buffer1[i].z, buffer2[i].z);
    ASSERT_EQ(buffer1[i].w, buffer2[i].w);
  }

  // vec1 copy
  reset_buffers();
  hipDeviceSynchronize();
  vectorized_copy<double, 1><<<total_work_size / block_work_size() , num_threads()>>>(b2, b1);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  ASSERT_EQ(hipSuccess, hipDeviceSynchronize());
  for (int i = 0; i < 1024; i++) {
    ASSERT_EQ(buffer1[i].x, buffer2[i].x);
    ASSERT_EQ(buffer1[i].y, buffer2[i].y);
    ASSERT_EQ(buffer1[i].z, buffer2[i].z);
    ASSERT_EQ(buffer1[i].w, buffer2[i].w);
  }

// Skipping this part until https://github.com/pytorch/pytorch/issues/51863 is resolved
#if 0
  // unaligned
  for (int i = 0; i < 16; i++) {
    for (int j = 0; j < 16; j++) {
      b1 = reinterpret_cast<double *>(reinterpret_cast<char *>(buffer1) + i);
      b2 = reinterpret_cast<double *>(reinterpret_cast<char *>(buffer2) + j);
      (void)hipGetLastError();
      hipDeviceSynchronize();
      vectorized_copy<double, 4><<<1, num_threads()>>>(b2, b1);
      hipDeviceSynchronize();
      auto err = hipGetLastError();
      if (i % 16 == 0 && j % 16 == 0) {
        ASSERT_EQ(err, hipSuccess);
      } else {
        ASSERT_EQ(err, cudaErrorMisalignedAddress);
      }
    }
  }
#endif
}
