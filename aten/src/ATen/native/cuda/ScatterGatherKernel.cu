#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/native/TensorAdvancedIndexing.h>
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>
#include <ATen/ceil_div.h>
#include <ATen/MemoryOverlap.h>

#include <ATen/native/ScatterGatherChecks.h>
#include <ATen/native/ReduceOpsUtils.h>
#include <ATen/native/cuda/IndexKernelUtils.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>

namespace at::native {

// Implement as functors since lambdas don't get optimized.
class ReduceMultiply {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
    (void)numel; // suppress unused warning
    gpuAtomicMul(self_data_start + index, *src_data);
  }
};
static ReduceMultiply reduce_multiply;

class ReduceAdd {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
#if (defined(__gfx940__) || defined(__gfx941__) || defined(__gfx942__) || defined(__gfx950__))
    opportunistic_fastAtomicAdd(self_data_start, index, numel, *src_data);
#else
    fastAtomicAdd(self_data_start, index, numel, *src_data, true);
#endif
  }
};
static ReduceAdd reduce_add;

class ReduceMean {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
    fastAtomicAdd(self_data_start, index, numel, *src_data, true);
  }
};
static ReduceMean reduce_mean;

class ReduceMinimum {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
    (void)numel; // suppress unused warning
    gpuAtomicMin(self_data_start + index, *src_data);
  }
};
static ReduceMinimum reduce_minimum;

class ReduceMaximum {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
    (void)numel; // suppress unused warning
    gpuAtomicMax(self_data_start + index, *src_data);
  }
};
static ReduceMaximum reduce_maximum;

class TensorAssign {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
    (void)numel; // suppress unused warning
    *(self_data_start + index) = *src_data;
  }
};
static TensorAssign tensor_assign;

// The kernels are implemented on an opaque,
// self-aligned type of the correct size,
// to avoid redundant kernels for different types
// of the same size.
template <int N> struct alignas(N) OpaqueType { char data[N]; };

// essentially rewritten related to legacy::launch_kernel parts
template <int nt, int vt, typename func_t>
C10_LAUNCH_BOUNDS_2(nt, vt)
__global__ void _scatter_gather_elementwise_kernel(int N, func_t f) {
  constexpr int nv = nt * vt;
  int idx = nv * blockIdx.x + threadIdx.x;

  #pragma unroll
  for (int i = 0; i < vt; ++i) {
    if (idx < N) {
      f(idx);
      idx += nt;
    }
  }
}

template <int nt, int vt, typename func_t>
static void _launch_scatter_gather_kernel(int64_t N, const func_t& f) {
  TORCH_INTERNAL_ASSERT(N >= 0 && N <= std::numeric_limits<int32_t>::max());
  if (N == 0) {
    return;
  }

  const dim3 block(nt);
  const dim3 grid((N + block.x * vt - 1) / (block.x * vt));
  const auto stream = at::cuda::getCurrentCUDAStream();
  _scatter_gather_elementwise_kernel<nt, vt, func_t><<<grid, block, 0, stream>>>(N, f);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <bool is_scatter_like, typename scalar_t, typename index_t>
struct _cuda_scatter_gather_internal_kernel {
  template <typename func_t>
  void operator() (
    TensorIterator& iter,
    int64_t index_size,
    int64_t index_stride,
    int64_t numel,  // Do not use `const` qualifier here as it may cause issue in cuda 11.6.x. See #75434, #75545
    const func_t& f
  ) {
    if (!iter.can_use_32bit_indexing()) {
      for (auto& sub_iter : iter.with_32bit_indexing()) {
        _cuda_scatter_gather_internal_kernel<is_scatter_like, scalar_t, index_t>()(
          sub_iter, index_size, index_stride, numel, f
        );
      }
      return;
    }

    char* self_ptr = (char*)iter.data_ptr(0);
    char* src_ptr = (char*)iter.data_ptr(1);
    char* index_ptr = (char*)iter.data_ptr(2);

    if constexpr (!is_scatter_like) {
      // we can go to faster path if we are indexing on the first dim
      // the dst and src are contiguous and all the dims and pts are multiple of 16
      constexpr size_t element_size = sizeof(scalar_t);
      constexpr size_t alignment = 16;
      if (at::native::fast_gather_kernel_eligible<alignment>(iter, self_ptr, src_ptr, index_stride * element_size, element_size)) {
        auto slice_size = iter.shape()[0] * element_size;
        auto num_ind = iter.shape()[1];
        auto ind_dim_size = index_size;
        auto inp_stride_bytes = index_stride * element_size;
        auto out_stride_bytes = iter.strides(0)[1];
        if (iter.numel() == 0) return;
        at::native::vectorized_gather_kernel_launch<alignment, index_t>(self_ptr, src_ptr, (index_t*)index_ptr, num_ind, slice_size, ind_dim_size, inp_stride_bytes, out_stride_bytes);
        return;
      }
    }
    auto offset_calc = make_offset_calculator<3>(iter);
    auto loop = [=]C10_DEVICE(int i) {
      auto offsets = offset_calc.get(i);

      int64_t idx_dim = *(index_t*)(index_ptr + offsets[2]);
      CUDA_KERNEL_ASSERT(idx_dim >= 0 && idx_dim < index_size
        && "scatter gather kernel index out of bounds");

      f(
        (scalar_t*)(self_ptr + offsets[0]),
        is_scatter_like ? idx_dim * index_stride : 0,
        numel,
        (scalar_t*)(src_ptr + offsets[1]) + (is_scatter_like ? 0 : idx_dim * index_stride)
      );
    };

    _launch_scatter_gather_kernel<num_threads(), thread_work_size()>(iter.numel(), loop);

  }
}; // struct _cuda_scatter_gather_internal_kernel

template <bool is_scatter_like = true, bool cast_to_opaque = true>
struct cuda_scatter_gather_base_kernel {
  void operator()(
    const Tensor& self, int64_t dim,
    const Tensor& index, const Tensor& src,
    const std::string& method_name,
    const ReduceAdd& f
  ) {
    at::assert_no_internal_overlap(self);

    auto index_sizes = ensure_nonempty_vec(index.sizes().vec());
    auto self_strides = ensure_nonempty_vec(self.strides().vec());
    auto src_strides = ensure_nonempty_vec(src.strides().vec());

    // restride self and src such that
    // self.shape = src.shape = index.shape
    //
    // restride stride[dim] such that
    // if (is_scatter_like) self.stride[dim] = 0
    // else src.stride[dim] = 0
    auto self_restrided = is_scatter_like ?
        restride_dim(self, dim, index_sizes)
      : self.as_strided(index_sizes, self_strides);
    auto src_restrided = is_scatter_like ?
        src.as_strided(index_sizes, src_strides)
      : restride_dim(src, dim, index_sizes);

    auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self_restrided)
      .add_const_input(src_restrided)
      .add_const_input(index)
      .build();

    auto self_dim_stride = ensure_nonempty_stride(self, dim);
    auto self_dim_size = ensure_nonempty_size(self, dim);

    auto src_dim_stride = ensure_nonempty_stride(src, dim);
    auto src_dim_size = ensure_nonempty_size(src, dim);

    auto index_size = is_scatter_like ? self_dim_size : src_dim_size;
    auto index_stride = is_scatter_like ? self_dim_stride : src_dim_stride;


    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
      iter.dtype(),
      "cuda_scatter_gather_base_kernel_func", [&] {
        using dtype = typename std::conditional<cast_to_opaque,
          OpaqueType<sizeof(scalar_t)>, scalar_t>::type;

        AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "cuda_scatter_gather_base_kernel_func", [&] () {
          _cuda_scatter_gather_internal_kernel<is_scatter_like, dtype, index_t>()(
            iter, index_size, index_stride, self.numel(), f
          );
        });
      }
    );
  }

  void operator()(
    const Tensor& self, int64_t dim,
    const Tensor& index, const Tensor& src,
    const std::string& method_name,
    const TensorAssign& f
  ) {
    at::assert_no_internal_overlap(self);

    auto index_sizes = ensure_nonempty_vec(index.sizes().vec());
    auto self_strides = ensure_nonempty_vec(self.strides().vec());
    auto src_strides = ensure_nonempty_vec(src.strides().vec());

    // restride self and src such that
    // self.shape = src.shape = index.shape
    //
    // restride stride[dim] such that
    // if (is_scatter_like) self.stride[dim] = 0
    // else src.stride[dim] = 0
    auto self_restrided = is_scatter_like ?
        restride_dim(self, dim, index_sizes)
      : self.as_strided(index_sizes, self_strides);
    auto src_restrided = is_scatter_like ?
        src.as_strided(index_sizes, src_strides)
      : restride_dim(src, dim, index_sizes);

    auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self_restrided)
      .add_const_input(src_restrided)
      .add_const_input(index)
      .build();

    auto self_dim_stride = ensure_nonempty_stride(self, dim);
    auto self_dim_size = ensure_nonempty_size(self, dim);

    auto src_dim_stride = ensure_nonempty_stride(src, dim);
    auto src_dim_size = ensure_nonempty_size(src, dim);

    auto index_size = is_scatter_like ? self_dim_size : src_dim_size;
    auto index_stride = is_scatter_like ? self_dim_stride : src_dim_stride;

    if (self.is_quantized()) {
      TORCH_CHECK(
          self.qscheme() == kPerTensorAffine,
          "Only per_tensor quantized quantized tensors are supported by gather.")
      AT_DISPATCH_QINT_TYPES(iter.dtype(), "gather_quant_cuda", [&] {
        using dtype = typename std::conditional<cast_to_opaque,
            OpaqueType<sizeof(scalar_t)>, scalar_t>::type;
        AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "cuda_scatter_gather_base_kernel_func", [&] () {
          _cuda_scatter_gather_internal_kernel<is_scatter_like, dtype, index_t>()(
            iter, index_size, index_stride, self.numel(), f
          );
        });
      });
    } else {
      AT_DISPATCH_V2(
          iter.dtype(),
          "gather_cuda",
          AT_WRAP([&] {
            using dtype = typename std::conditional<cast_to_opaque,
                OpaqueType<sizeof(scalar_t)>, scalar_t>::type;
            AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "cuda_scatter_gather_base_kernel_func", [&] () {
              _cuda_scatter_gather_internal_kernel<is_scatter_like, dtype, index_t>()(
                iter, index_size, index_stride, self.numel(), f
              );
            });
          }),
          AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX),
          AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES),
          AT_EXPAND(AT_FLOAT8_TYPES),
          kComplexHalf,
          kHalf,
          kBool,
          kBFloat16);
    }
  }

  template <typename func_t>
  void operator()(
    const Tensor& self, int64_t dim,
    const Tensor& index, const Tensor& src,
    const std::string& method_name,
    const func_t& f
  ) {
    at::assert_no_internal_overlap(self);

    auto index_sizes = ensure_nonempty_vec(index.sizes().vec());
    auto self_strides = ensure_nonempty_vec(self.strides().vec());
    auto src_strides = ensure_nonempty_vec(src.strides().vec());

    // restride self and src such that
    // self.shape = src.shape = index.shape
    //
    // restride stride[dim] such that
    // if (is_scatter_like) self.stride[dim] = 0
    // else src.stride[dim] = 0
    auto self_restrided = is_scatter_like ?
        restride_dim(self, dim, index_sizes)
      : self.as_strided(index_sizes, self_strides);
    auto src_restrided = is_scatter_like ?
        src.as_strided(index_sizes, src_strides)
      : restride_dim(src, dim, index_sizes);

    auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self_restrided)
      .add_const_input(src_restrided)
      .add_const_input(index)
      .build();

    auto self_dim_stride = ensure_nonempty_stride(self, dim);
    auto self_dim_size = ensure_nonempty_size(self, dim);

    auto src_dim_stride = ensure_nonempty_stride(src, dim);
    auto src_dim_size = ensure_nonempty_size(src, dim);

    auto index_size = is_scatter_like ? self_dim_size : src_dim_size;
    auto index_stride = is_scatter_like ? self_dim_stride : src_dim_stride;

    AT_DISPATCH_ALL_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      iter.dtype(),
      "cuda_scatter_gather_base_kernel_func", [&] {
        using dtype = typename std::conditional<cast_to_opaque,
          OpaqueType<sizeof(scalar_t)>, scalar_t>::type;

        AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "cuda_scatter_gather_base_kernel_func", [&] () {
          _cuda_scatter_gather_internal_kernel<is_scatter_like, dtype, index_t>()(
            iter, index_size, index_stride, self.numel(), f
          );
        });
      }
    );
  }
}; // struct cuda_scatter_gather_base_kernel

template <typename scalar_t, typename index_t>
struct _cuda_scatter_fill_internal_kernel {
  template <typename func_t>
  void operator()(
    TensorIterator& iter,
    scalar_t src_val,
    int64_t index_size,
    int64_t index_stride,
    int64_t numel,  // Do not use `const` qualifier here as it may cause issue in cuda 11.6.x. See #75434, #75545
    const func_t& f
  ) {
    if (!iter.can_use_32bit_indexing()) {
      for (auto& sub_iter : iter.with_32bit_indexing()) {
        _cuda_scatter_fill_internal_kernel<scalar_t, index_t>()(
          sub_iter, src_val, index_size, index_stride, numel, f
        );
      }
      return;
    }

    char* self_ptr = (char*)iter.data_ptr(0);
    char* index_ptr = (char*)iter.data_ptr(1);

    auto offset_calc = make_offset_calculator<2>(iter);
    auto loop = [=]C10_DEVICE(int i) {
      auto offsets = offset_calc.get(i);

      int64_t idx_dim = *(index_t*)(index_ptr + offsets[1]);
      CUDA_KERNEL_ASSERT(idx_dim >= 0 && idx_dim < index_size
        && "index out of bounds"
      );

      f(
        (scalar_t*)(self_ptr + offsets[0]),
        idx_dim * index_stride,
        numel,
        (scalar_t*)&src_val
      );
    };

    _launch_scatter_gather_kernel<num_threads(), thread_work_size()>(iter.numel(), loop);
  }
}; // struct _cuda_scatter_fill_internal_kernel

template <bool cast_to_opaque = true>
struct cuda_scatter_fill_base_kernel {
  template <typename func_t>
  void operator()(
    const Tensor& self, int64_t dim,
    const Tensor& index, Scalar src,
    const std::string& method_name,
    const func_t& f
  ) {
    at::assert_no_internal_overlap(self);

    auto index_sizes = ensure_nonempty_vec(index.sizes().vec());

    // restride self such that
    // self.shape = index.shape and
    // self.stride[dim] = 0
    auto self_restrided = restride_dim(self, dim, index_sizes);

    auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self_restrided)
      .add_const_input(index)
      .build();

    auto index_size = ensure_nonempty_size(self, dim);
    auto index_stride = ensure_nonempty_stride(self, dim);

    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
      iter.dtype(),
      "cuda_scatter_fill_base_kernel_func", [&] {
        using dtype = typename std::conditional<cast_to_opaque,
          OpaqueType<sizeof(scalar_t)>, scalar_t>::type;

        auto src_scalar_val = src.to<scalar_t>();
        auto src_val = *(dtype*)&src_scalar_val;

        AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "cuda_scatter_fill_base_kernel_func", [&] () {
          _cuda_scatter_fill_internal_kernel<dtype, index_t>()(
            iter, src_val, index_size, index_stride, self.numel(), f
          );
        });
      }
    );
  }

  void operator()(
    const Tensor& self, int64_t dim,
    const Tensor& index, Scalar src,
    const std::string& method_name,
    const ReduceMultiply& f
  ) {
    at::assert_no_internal_overlap(self);

    auto index_sizes = ensure_nonempty_vec(index.sizes().vec());

    // restride self such that
    // self.shape = index.shape and
    // self.stride[dim] = 0
    auto self_restrided = restride_dim(self, dim, index_sizes);

    auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self_restrided)
      .add_const_input(index)
      .build();

    auto index_size = ensure_nonempty_size(self, dim);
    auto index_stride = ensure_nonempty_stride(self, dim);

    AT_DISPATCH_ALL_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      iter.dtype(),
      "cuda_scatter_fill_base_kernel_reduce_multiply", [&] {
        using dtype = typename std::conditional<cast_to_opaque,
          OpaqueType<sizeof(scalar_t)>, scalar_t>::type;

        auto src_scalar_val = src.to<scalar_t>();
        auto src_val = *(dtype*)&src_scalar_val;

        AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "cuda_scatter_fill_base_kernel_reduce_multiply", [&] () {
          _cuda_scatter_fill_internal_kernel<dtype, index_t>()(
            iter, src_val, index_size, index_stride, self.numel(), f
          );
        });
      }
    );
  }
}; // struct cuda_scatter_fill_base_kernel

void gather_cuda_kernel(const Tensor& result, const Tensor& self, int64_t dim, const Tensor& index) {
  cuda_scatter_gather_base_kernel</*is_scatter_like=*/false>()(
    result, dim, index, self,
    "gather_out_cuda", tensor_assign);
}

void scatter_cuda_kernel(const Tensor& self, int64_t dim, const Tensor& index, const Tensor& src) {
  // When indices are not unique, the behavior is non-deterministic
  globalContext().alertNotDeterministic("scatter_cuda_");
  cuda_scatter_gather_base_kernel<>()(
    self, dim, index, src,
    "scatter_cuda_", tensor_assign);
}

void scatter_fill_cuda_kernel(const Tensor& self, int64_t dim, const Tensor& index, const Scalar& src) {
  cuda_scatter_fill_base_kernel<>()(
    self, dim, index, src,
    "scatter_fill_cuda_", tensor_assign);
}

void scatter_add_cuda_kernel(const Tensor& self, int64_t dim, const Tensor& index, const Tensor& src) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("scatter_add_cuda_kernel");
  cuda_scatter_gather_base_kernel</*is_scatter_like=*/true, /*cast_to_opaque=*/false>()(
    self, dim, index, src,
    "scatter_add_cuda_", reduce_add);
}

void scatter_reduce_cuda_kernel(const Tensor& self, const int64_t dim, const Tensor& index,
                               const Tensor& src, const ReductionType& reduce) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd/AtomicMul usage
  globalContext().alertNotDeterministic("scatter_reduce_cuda_kernel");
  switch (reduce) {
  case ReductionType::SUM :
    cuda_scatter_gather_base_kernel<true, false>()(self, dim, index, src,
                                       "scatter_reduce_cuda_add_", reduce_add);
    break;
  case ReductionType::PROD :
    cuda_scatter_gather_base_kernel<true, false>()(self, dim, index, src,
                                       "scatter_reduce_cuda_multiply_", reduce_multiply);
    break;
  default :
    break;
  }
}

void scatter_reduce_two_cuda_kernel(const Tensor& self, const int64_t dim, const Tensor& index,
                                    const Tensor& src, const ReductionType& reduce) {
  switch (reduce) {
  case ReductionType::SUM :
    globalContext().alertNotDeterministic("scatter_reduce_cuda_sum_");
    cuda_scatter_gather_base_kernel<true, false>()(self, dim, index, src,
            "scatter_reduce_cuda_sum_", reduce_add);
    break;
  case ReductionType::PROD :
    globalContext().alertNotDeterministic("scatter_reduce_cuda_prod_");
    cuda_scatter_gather_base_kernel<true, false>()(self, dim, index, src,
            "scatter_reduce_cuda_prod_", reduce_multiply);
    break;
  case ReductionType::MAX :
    cuda_scatter_gather_base_kernel<true, false>()(self, dim, index, src,
            "scatter_reduce_cuda_amax_", reduce_maximum);
    break;
  case ReductionType::MIN :
    cuda_scatter_gather_base_kernel<true, false>()(self, dim, index, src,
            "scatter_reduce_cuda_amin_", reduce_minimum);
    break;
  case ReductionType::MEAN :
    globalContext().alertNotDeterministic("scatter_reduce_cuda_mean_");
    cuda_scatter_gather_base_kernel<true, false>()(self, dim, index, src,
            "scatter_reduce_cuda_mean_", reduce_mean);
    break;
  }
}

void scatter_scalar_reduce_cuda_kernel(const Tensor& self, const int64_t dim, const Tensor& index,
                               const Scalar& value, const ReductionType& reduce) {
  switch (reduce) {
  case ReductionType::SUM :
    cuda_scatter_fill_base_kernel<false>()(self, dim, index, value,
                                      "scatter_fill_cuda_add_", reduce_add);
    break;
  case ReductionType::PROD :
    cuda_scatter_fill_base_kernel<false>()(self, dim, index, value,
                                      "scatter_fill_cuda_multiply_", reduce_multiply);
    break;
  default :
    break;
  }
}


REGISTER_DISPATCH(gather_stub, &gather_cuda_kernel)
REGISTER_DISPATCH(scatter_stub, &scatter_cuda_kernel)
REGISTER_DISPATCH(scatter_fill_stub, &scatter_fill_cuda_kernel)
REGISTER_DISPATCH(scatter_add_stub, &scatter_add_cuda_kernel)
REGISTER_DISPATCH(scatter_reduce_stub, &scatter_reduce_cuda_kernel)
REGISTER_DISPATCH(scatter_scalar_reduce_stub, &scatter_scalar_reduce_cuda_kernel)
REGISTER_DISPATCH(scatter_reduce_two_stub, &scatter_reduce_two_cuda_kernel)

} // namespace at::native
