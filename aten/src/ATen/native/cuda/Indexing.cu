#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/native/TensorAdvancedIndexing.h>
#include <ATen/native/IndexingUtils.h>
#include <ATen/native/quantized/IndexKernel.h>
#include <ATen/native/cuda/KernelUtils.cuh>

#include <ATen/core/Tensor.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/Dispatch_v2.h>
#include <ATen/ExpandUtils.h>
#include <ATen/MemoryOverlap.h>
#include <ATen/TensorOperators.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/Resize.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDAUtils.h>
#include <ATen/cuda/DeviceUtils.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_assert_async.h>
#include <ATen/ops/arange.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/zeros_like.h>
#include <ATen/ops/ones_like.h>
#include <ATen/ops/empty_quantized.h>
#include <ATen/ops/index_add_native.h>
#include <ATen/ops/index_reduce_native.h>
#include <ATen/ops/index_select_native.h>
#include <ATen/ops/masked_fill_native.h>
#include <ATen/ops/_sparse_coo_tensor_with_dims_and_tensors.h>
#endif

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/cub.h>
#include <c10/util/irange.h>
#include <c10/core/QScheme.h>
#include <ATen/native/quantized/AffineQuantizerBase.h>

#include <limits>

#include <c10/macros/Macros.h>

namespace {
constexpr uint64_t getDefaultMaxThreadsPerBlock() {
#ifndef USE_ROCM
  return 128;
#else
  // bigger default
  return 512;
#endif
}

#ifdef USE_ROCM
#define SKIP_SORTED_INDICES 32
template <typename scalar_t, int SZ>
__global__ void indexing_backward_kernel(
  const int64_t* sorted_indices, const int64_t* indices, const scalar_t* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim, bool accumulate) {
  using opmath_t = at::opmath_type<scalar_t>;

  extern __shared__ unsigned char smem[];
  auto smem_dups_cache = reinterpret_cast<int64_t*>(smem);

  int smem_offset = threadIdx.y * C10_WARP_SIZE;

  int laneIdx = threadIdx.x % C10_WARP_SIZE;
  int64_t grad_row = 0;

  for (int64_t z = blockIdx.z; z < outer_dim; z += gridDim.z) {
    // Init duplicates every time we compute a new set of entries:
    smem_dups_cache[smem_offset + laneIdx] = 0;
    WARP_SYNC();

    int64_t base_idx = blockIdx.x * blockDim.y * C10_WARP_SIZE + threadIdx.y * C10_WARP_SIZE;
    int64_t idx = base_idx + laneIdx;

    if (idx < numel) {
      int64_t crnt_sorted_idx = sorted_indices[idx];

      if (idx == 0 || crnt_sorted_idx != sorted_indices[idx - 1]) {
        // Determine the number of duplicates in advance:
        int64_t num_duplicates = 1;

        // Lookahead in case there is a large number of duplicates. Once that is done, handle the tail.
        while ((idx + num_duplicates + SKIP_SORTED_INDICES - 1) < numel) {
          if (sorted_indices[idx + num_duplicates + SKIP_SORTED_INDICES - 1] != crnt_sorted_idx) break;
            num_duplicates += SKIP_SORTED_INDICES;
        }
        while (((idx + num_duplicates) < numel) && (sorted_indices[idx + num_duplicates] == crnt_sorted_idx)) {
          num_duplicates++;
        }

        smem_dups_cache[smem_offset + laneIdx] = num_duplicates;
      }
    }

    WARP_SYNC();

    // All lanes in the warp are still active here. Use them all to reduce duplicates when
    // large number of duplicates are present:
    for (int subwarp = 0; subwarp < C10_WARP_SIZE; subwarp++) {
      // All lanes read the shared memory entry for number of duplicates
      int64_t new_num_duplicates = smem_dups_cache[smem_offset + subwarp];

      // Check if the original sub-warp had duplicates to eliminate, if not skip.
      if (new_num_duplicates == 0)
        continue;

      // There are duplicates that need eliminating:
      int64_t new_idx = base_idx + subwarp;
      int64_t new_crnt_sorted_idx = sorted_indices[new_idx];
      const int64_t new_weight_row = new_crnt_sorted_idx * stride + z * stride_before;

      if (!accumulate) {
        const int64_t grad_row = ((int64_t)indices[new_idx + new_num_duplicates - 1]) * stride + z * numel * stride;
        int64_t feature_dim = blockIdx.y * blockDim.x + threadIdx.x;
        while (feature_dim < stride) {
          grad_weight[new_weight_row + feature_dim] = grad_output[grad_row + feature_dim];
          feature_dim += gridDim.y * blockDim.x;
        }
        continue;
      }

      for (int dup = 0; dup < new_num_duplicates; dup++) {
        const int64_t grad_row = ((int64_t) indices[new_idx + dup]) * stride + z * numel * stride;

        // All lanes do the same thing up to here.
        int64_t feature_dim = blockIdx.y * blockDim.x + threadIdx.x;

        // Each lane has a different feature_dim.
        while (feature_dim < stride) {
          grad_weight[new_weight_row + feature_dim] += grad_output[grad_row + feature_dim];
          feature_dim += gridDim.y * blockDim.x;
        }
      }
    }
  }
}

template <typename scalar_t>
__global__ void indexing_backward_kernel_stride_1(
  const int64_t* sorted_indices, const int64_t* indices, const scalar_t* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim, bool accumulate) {
  using opmath_t = at::opmath_type<scalar_t>;

  int laneIdx = threadIdx.x % C10_WARP_SIZE;

  const opmath_t scale = (opmath_t)1.0;
  int64_t grad_row = 0;

  extern __shared__ unsigned char smem[];
  auto smem_dups_cache = reinterpret_cast<int64_t*>(smem);

  // Each warp gets a different section of the share memory allocation:
  int smem_offset = threadIdx.y * C10_WARP_SIZE;

  // Number of values processed by each thread (grain size)
  for (int64_t z = blockIdx.z; z < outer_dim; z += gridDim.z) {
    // Init duplicates every time we compute a new set of entries:
    smem_dups_cache[smem_offset + laneIdx] = 0;

    int64_t base_idx = blockIdx.x * blockDim.y * C10_WARP_SIZE + threadIdx.y * C10_WARP_SIZE;
    int64_t idx = base_idx + laneIdx;

    // Each lane calculates the number of duplicates:
    if (idx < numel) {
      int64_t crnt_sorted_idx = sorted_indices[idx];

      if (idx == 0 || crnt_sorted_idx != sorted_indices[idx - 1]) {
        // Determine the number of duplicates in advance:
        int64_t num_duplicates = 1;

        // Lookahead in case there is a large number of duplicates. Once that is done, handle the tail.
        while ((idx + num_duplicates + SKIP_SORTED_INDICES - 1) < numel) {
          if (sorted_indices[idx + num_duplicates + SKIP_SORTED_INDICES - 1] != crnt_sorted_idx) break;
            num_duplicates += SKIP_SORTED_INDICES;
        }
        while (((idx + num_duplicates) < numel) && (sorted_indices[idx + num_duplicates] == crnt_sorted_idx)) {
          num_duplicates++;
        }

        if (!accumulate) {
          const int64_t weight_row = crnt_sorted_idx * stride + z * stride_before;
          grad_row = ((int64_t)indices[idx + num_duplicates - 1]) * stride + z * numel * stride;
          grad_weight[weight_row] =
            static_cast<scalar_t>(static_cast<opmath_t>(grad_output[grad_row]) * scale);
          continue;
        }

        // Each lane sequentially handles the duplicate elimination:
        if (num_duplicates < C10_WARP_SIZE) {
          opmath_t gradient = (opmath_t)0.0;
          const int64_t weight_row = crnt_sorted_idx * stride + z * stride_before;
          for (int64_t i = 0; i < num_duplicates; ++i) {
            grad_row = ((int64_t) indices[idx + i]) * stride + z * numel * stride;
            gradient += static_cast<opmath_t>(grad_output[grad_row]) * scale;
          }

          grad_weight[weight_row] = static_cast<scalar_t>(static_cast<opmath_t>(grad_weight[weight_row]) + gradient);
        } else {
          // Add duplicate to the cache:
          smem_dups_cache[smem_offset + laneIdx] = num_duplicates;
        }
      }
    }

    WARP_SYNC();

    // All lanes in the warp are still active here. Use them all to reduce duplicates when
    // large number of duplicates are present:
    for (int subwarp = 0; subwarp < C10_WARP_SIZE; subwarp++) {
      // All lanes read the shared memory entry for number of duplicates
      int64_t new_num_duplicates = smem_dups_cache[smem_offset + subwarp];

      // Check if the original sub-warp had duplicates to eliminate, if not skip.
      if (new_num_duplicates == 0)
        continue;

      // There are duplicates that need eliminating:
      int64_t new_idx = base_idx + subwarp;
      int64_t new_crnt_sorted_idx = sorted_indices[new_idx];
      const int64_t new_weight_row = new_crnt_sorted_idx * stride + z * stride_before;

      // Result of the reduction will be in this variable:
      opmath_t gradient = (opmath_t)0.0;

      int64_t num_warp_passes = new_num_duplicates / C10_WARP_SIZE;
      // Parallel reduction across the array of duplicates using all the lanes in the warp:
      for (int64_t i = 0; i < num_warp_passes; ++i) {
        grad_row = ((int64_t) indices[new_idx + i * C10_WARP_SIZE + laneIdx]) * stride + z * numel * stride;
        gradient += static_cast<opmath_t>(grad_output[grad_row]) * scale;
      }

      // Reduce across the lanes of the warp:
      WARP_SYNC();
      for (int offset = C10_WARP_SIZE / 2; offset > 0; offset /= 2) {
        gradient += WARP_SHFL_DOWN(gradient, offset);
      }

      if (laneIdx == 0) {
        for (int64_t i = num_warp_passes * C10_WARP_SIZE; i < new_num_duplicates; ++i) {
          grad_row = ((int64_t) indices[new_idx + i]) * stride + z * numel * stride;
          gradient += static_cast<opmath_t>(grad_output[grad_row]) * scale;
        }

        grad_weight[new_weight_row] = static_cast<scalar_t>(static_cast<opmath_t>(grad_weight[new_weight_row]) + gradient);
      }
    }
  }
}
#else
template <typename scalar_t, int SZ>
__global__ void indexing_backward_kernel(
  const int64_t* sorted_indices, const int64_t* indices, const scalar_t* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim, bool accumulate) {
//numel is total number of flattened indices, not expanded to dimensions that are not indexed.
//stride is the cumulative size of the not-indexed last dimensions
//stride_before is the stride of the dimension immediately preceding first indexed dimension
//if indexing starts from the 0th dimension, stride_before does not matter because blockIdx.z will be 0 in this case
//outer_dim is number of elements in the first unindexed dimensions
  using opmath_t = at::opmath_type<scalar_t>;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceding input has the same destination index as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values processed by each thread (grain size)
  for (int64_t z = blockIdx.z; z < outer_dim; z += gridDim.z){
    int64_t idx = blockIdx.x * blockDim.y + threadIdx.y;
    if (idx < numel
        && (idx == 0 || sorted_indices[idx] != sorted_indices[idx - 1])){
      do {
        int64_t start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        // if not accumulate, we only keep the last duplicate index so skip those before it
        if (!accumulate && (idx < numel - 1) && sorted_indices[idx] == sorted_indices[idx + 1]) {
          idx++;
          continue;
        }
        const int64_t weight_row = ((int64_t) sorted_indices[idx]) * stride + z * stride_before;
        const int64_t grad_row = ((int64_t) indices[idx]) * stride + z * numel * stride;
        const opmath_t scale = (opmath_t)1.0;

        opmath_t gradient[SZ];
        opmath_t weight[SZ];

        while (start_feature < stride) {
          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
              gradient[ii] = static_cast<opmath_t>(grad_output[grad_row + feature_dim]);
              if (accumulate) {
                weight[ii] = static_cast<opmath_t>(grad_weight[weight_row + feature_dim]);
              }
            }
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            if (accumulate) {
              weight[ii] += gradient[ii] * scale;
            } else {
              weight[ii] = gradient[ii] * scale;
            }
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
                grad_weight[weight_row + feature_dim] = static_cast<scalar_t>(weight[ii]);
            }
          }
          start_feature += gridDim.y * blockDim.x * SZ;
        }

        idx++;
      } while (idx < numel && sorted_indices[idx] == sorted_indices[idx - 1]);
    }
  }
}

template <typename scalar_t>
__global__ void indexing_backward_kernel_stride_1(
  const int64_t* sorted_indices, const int64_t* indices, const scalar_t* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim, bool accumulate) {
  using opmath_t = at::opmath_type<scalar_t>;

  // Number of values processed by each thread (grain size)
  for (int64_t z = blockIdx.z; z < outer_dim; z += gridDim.z){
    int64_t idx = blockIdx.x * blockDim.y + threadIdx.y;
    int64_t crnt_sorted_idx = sorted_indices[idx];

    if ((idx < numel) &&
        (idx == 0 || crnt_sorted_idx != sorted_indices[idx - 1]))
    {
      // Determine the number of duplicates in advance
      int64_t num_duplicates = 1;
      while (((idx + num_duplicates) < numel) && (sorted_indices[idx + num_duplicates] == crnt_sorted_idx)) {
        num_duplicates++;
      }

      // Continue computing weights
      const int64_t weight_row = crnt_sorted_idx * stride + z * stride_before;
      int64_t grad_row = 0;
      const opmath_t scale = (opmath_t)1.0;

      if (!accumulate) {
        grad_row = ((int64_t)indices[idx + num_duplicates - 1]) * stride + z * numel * stride;
        grad_weight[weight_row] =
          static_cast<scalar_t>(static_cast<opmath_t>(grad_output[grad_row]) * scale);
      } else {
        opmath_t gradient = (opmath_t)0.0;

        int laneIdx = threadIdx.x % C10_WARP_SIZE;
        int64_t num_warp_passes = num_duplicates / C10_WARP_SIZE;
        for (int64_t i = 0; i < num_warp_passes; ++i) {
            grad_row = ((int64_t) indices[idx + i * C10_WARP_SIZE + laneIdx]) * stride + z * numel * stride;
            gradient += static_cast<opmath_t>(grad_output[grad_row]) * scale;
        }
        WARP_SYNC();
        for (int offset = C10_WARP_SIZE / 2; offset > 0; offset /= 2) {
          gradient += WARP_SHFL_DOWN(gradient, offset);
        }

        if (laneIdx == 0) {
          for (int64_t i = num_warp_passes * C10_WARP_SIZE; i < num_duplicates; ++i) {
            grad_row = ((int64_t) indices[idx + i]) * stride + z * numel * stride;
            gradient += static_cast<opmath_t>(grad_output[grad_row]) * scale;
          }

          grad_weight[weight_row] = static_cast<scalar_t>(static_cast<opmath_t>(grad_weight[weight_row]) + gradient);
        }
      }
    }
  }
}
#endif

template <typename scalar_t>
__global__ void indexing_backward_kernel_small_stride(
  const int64_t* sorted_indices, const int64_t* indices, const scalar_t* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim, bool accumulate) {
  using opmath_t = at::opmath_type<scalar_t>;

  // Number of values processed by each thread (grain size)
  for (int64_t z = blockIdx.z; z < outer_dim; z += gridDim.z){
    int64_t idx = blockIdx.x * blockDim.y + threadIdx.y;
    int64_t tidx = threadIdx.x;
    int64_t crnt_sorted_idx = sorted_indices[idx];

    if ((idx < numel) &&
        (tidx < stride) &&
        (idx == 0 || crnt_sorted_idx != sorted_indices[idx - 1]))
    {
      // Determine the number of duplicates in advance
      int64_t num_duplicates = 1;
      while (((idx + num_duplicates) < numel) && (sorted_indices[idx + num_duplicates] == crnt_sorted_idx)) {
        num_duplicates++;
      }

      // Continue computing weights
      const int64_t weight_row = crnt_sorted_idx * stride + z * stride_before;
      int64_t grad_row = 0;
      const opmath_t scale = (opmath_t)1.0;

      if (!accumulate) {
        grad_row = ((int64_t)indices[idx + num_duplicates - 1]) * stride + z * numel * stride;
        grad_weight[weight_row + tidx] =
          static_cast<scalar_t>(static_cast<opmath_t>(grad_output[grad_row + tidx]) * scale);
      } else {
        opmath_t gradient = (opmath_t)0.0;
        for (int64_t i = 0; i < num_duplicates; ++i) {
          grad_row = ((int64_t) indices[idx + i]) * stride + z * numel * stride;
          gradient += static_cast<opmath_t>(grad_output[grad_row + tidx]) * scale;
        }

        grad_weight[weight_row + tidx] = static_cast<scalar_t>(static_cast<opmath_t>(grad_weight[weight_row + tidx]) + gradient);
      }
    }
  }
}

template <typename scalar_t, int SZ>
__global__ void indexing_backward_kernel_quantized(
  const int64_t* sorted_indices, const int64_t* indices, const float* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim,
  float inv_scale, int zero_point, int64_t qmin, int64_t qmax) {

  // This implementation is adopted from indexing_backward_kernel above.
  using opmath_t = at::opmath_type<float>;
  for (int64_t z = blockIdx.z; z < outer_dim; z += gridDim.z){
    int64_t idx = blockIdx.x * blockDim.y + threadIdx.y;
    if (idx < numel
        && (idx == 0 || sorted_indices[idx] != sorted_indices[idx - 1])){
      do {
        int64_t start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        // we only keep the last duplicate index so skip those before it
        if ((idx < numel - 1) && sorted_indices[idx] == sorted_indices[idx + 1]) {
          idx++;
          continue;
        }
        const int64_t weight_row = ((int64_t) sorted_indices[idx]) * stride + z * stride_before;
        const int64_t grad_row = ((int64_t) indices[idx]) * stride + z * numel * stride;
        const opmath_t scale = (opmath_t)1.0;

        opmath_t gradient[SZ];
        opmath_t weight[SZ];

        while (start_feature < stride) {
          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
              gradient[ii] = static_cast<opmath_t>(grad_output[grad_row + feature_dim]);
            }
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            weight[ii] = gradient[ii] * scale;
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
                // we do quantization here
                int64_t qvalue = static_cast<int64_t>(zero_point + nearbyintf(weight[ii]* inv_scale));
                qvalue = min(max(qvalue, qmin), qmax);
                grad_weight[weight_row + feature_dim] = static_cast<scalar_t>(qvalue);
            }
          }
          start_feature += gridDim.y * blockDim.x * SZ;
        }

        idx++;
      } while (idx < numel && sorted_indices[idx] == sorted_indices[idx - 1]);
    }
  }
}


}


namespace at::native {

namespace {

class ReduceMultiply {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
    (void)numel; // suppress unused warning
    gpuAtomicMul(self_data_start + index, *src_data);
  }
};
static ReduceMultiply reduce_multiply;

class ReduceAdd {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
#if (defined(__gfx940__) || defined(__gfx941__) || defined(__gfx942__) || defined(__gfx950__))
    opportunistic_fastAtomicAdd(self_data_start, index, numel, *src_data);
#else
    fastAtomicAdd(self_data_start, index, numel, *src_data, true);
#endif
  }
};
static ReduceAdd reduce_add;

class ReduceMinimum {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
    (void)numel; // suppress unused warning
    gpuAtomicMin(self_data_start + index, *src_data);
  }
};
static ReduceMinimum reduce_minimum;

class ReduceMaximum {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t* self_data_start, int64_t index, int64_t numel, const scalar_t * src_data) const {
    (void)numel; // suppress unused warning
    gpuAtomicMax(self_data_start + index, *src_data);
  }
};
static ReduceMaximum reduce_maximum;

}

static Tensor wrapIndexOnce(const Tensor & index, int64_t dim, int64_t dim_size, bool check_range=true) {
//we don't need to check range in backward - if there were out of bounds indices forward should already have errored out
  if (index.numel() != 0 && check_range) {
    at::_assert_async(index.max() < dim_size);
    at::_assert_async(index.min() >= -dim_size);
  }
  return index.remainder(dim_size);
}

static std::vector<int64_t> computeLinearStride(const Tensor & tensor) {
  // computes the stride as if tensor were contiguous
  auto sizes = tensor.sizes();
  std::vector<int64_t> stride(tensor.dim());
  if (stride.empty()) {
    return stride;
  }
  stride[tensor.dim() - 1] = 1;
  std::partial_sum(sizes.rbegin(), sizes.rend() - 1, stride.rbegin() + 1, std::multiplies<int64_t>());
  return stride;
}

static std::tuple<Tensor, int64_t, int64_t, int64_t>
computeLinearIndex(const Tensor & src, TensorList indices, bool check_range) {
  auto strides = computeLinearStride(src);
  const auto& device = src.options().device();

  // Compute the linear index by multiplying the indexing tensors by the
  // stride and summing them. All the indexing tensors have the same shape at
  // this point. We also compute the number of dimensions before and after that
  // are not being index.
  Tensor linearIndex;
  int64_t nElemBefore = 1, nElemAfter = 1, strideBefore =0;
  for (const auto i: c10::irange(src.dim())) {
    if (indices[i].defined()) {
      // Cast index to the longType matching src's device
      // This allows us to support ie indexing a cuda tensor with a cpu tensor
      Tensor index = (wrapIndexOnce(indices[i], i, src.size(i), check_range) * strides[i]).to(device);
      if (linearIndex.defined()) {
        linearIndex += index;
      } else {
        linearIndex = index;
        if (i>0) {
           strideBefore = src.stride(i-1); // stride after undefined dimensions
        }
      }
    } else if (linearIndex.defined()) {
      nElemAfter *= src.size(i);
    } else {
      nElemBefore *= src.size(i);
    }
  }

  return std::make_tuple(std::move(linearIndex), nElemBefore, strideBefore, nElemAfter);
}


static std::tuple<Tensor, Tensor, int64_t, int64_t, int64_t, std::vector<int64_t>> makeLinearIndex(Tensor self, IOptTensorListRef orig, bool check_range) {
  checkIndexTensorTypes(orig, /*allow_int*/true);
  // first expand BoolTensor (masks) or ByteTensor (masks) into 1 or more LongTensors
  auto indices = expandTensors(self, orig);
  for (auto & i : indices) {
    if (i.defined() && i.dtype() == at::kInt) {
      i = i.to(at::kLong);
    }
  }
  // next broadcast all index tensors together
  indices = expand_outplace(indices);
  // add missing null Tensors so that it matches self.dim()
  while (indices.size() < (size_t)self.dim()) {
    indices.emplace_back();
  }
  // if the non-null indices are not all adjacent, transpose self and indices
  // together so that they're adjacent at the front
  std::vector<int64_t> inversePerm;
  if (!hasContiguousSubspace(indices)) {
    std::tie(self, indices, inversePerm) = transposeToFrontAndInvPerm(self, indices);
  }
  auto [linearIndex, nElemBefore, strideBefore, nElemAfter] = computeLinearIndex(self, indices, check_range);
  return std::make_tuple(linearIndex, self, nElemBefore, strideBefore, nElemAfter, inversePerm);
}


void index_put_with_sort_kernel_thrust_helper(Tensor &linearIndex, Tensor &orig_indices, Tensor &sorted_indices, int64_t num_indices);

namespace {

int64_t largestIndex(const Tensor &self) {
  int64_t result = 0;
  for (const auto i: c10::irange(self.dim())) {
    result += (self.sizes()[i] - 1) * self.strides()[i];
  }
  return result;
}

void index_put_with_sort_kernel(Tensor & self, const c10::List<std::optional<Tensor>>& indices, const Tensor & value, bool accumulate, bool unsafe) {
  TORCH_CHECK(!indices.empty() || is_expandable_to(value.sizes(), self.sizes()), "shape mismatch: value tensor of shape ", value.sizes(),
             " cannot be broadcast to indexing result of shape ", self.sizes());
  if (indices.size() > (size_t)self.dim()) {
    TORCH_CHECK_INDEX(false, "too many indices for tensor of dimension ", self.dim(), " (got ", indices.size(), ")");
  }
  bool self_contiguous = self.is_contiguous();
  auto self_ = self_contiguous ? self : self.contiguous();
  Tensor linearIndex, src, expandedValue = value;
  int64_t nElemBefore, strideBefore, sliceSize;
  std::vector<int64_t> inversePerm;
  std::tie(linearIndex, src, nElemBefore, strideBefore, sliceSize, inversePerm) = makeLinearIndex(self_, indices, !unsafe);
  int64_t num_indices = linearIndex.numel();

  if (expandedValue.numel() < num_indices * nElemBefore * sliceSize) {
    auto expanded_size = at::DimVector(expandedValue.sizes());
    auto size1 = expandedValue.sizes();
    auto size2 = linearIndex.sizes();
    if (are_expandable(size1, size2)) {
      expanded_size = infer_size_dimvector(size1, size2);
    }
    if (nElemBefore > 1) {
      expanded_size.insert(expanded_size.begin(), nElemBefore);
    }
    if (sliceSize > 1) {
      expanded_size.insert(expanded_size.end(), sliceSize);
    }
    expandedValue = expandedValue.expand(expanded_size);
  }
  expandedValue = expandedValue.contiguous();

  if (num_indices > 0 && sliceSize > 0) {
      const bool permuted = !src.is_contiguous();
      auto src_ = permuted ? src.contiguous() : src;
      linearIndex = linearIndex.reshape(-1);
      auto sorted_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      auto orig_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      const hipStream_t stream = at::cuda::getCurrentCUDAStream();

      linearIndex.divide_(sliceSize, "trunc");

      // cub on CUDA <= 11.2 have a bug that for small sizes
      // cub's sort can be much slower than thrust's merge sort
      // this bug is fixed in CUDA 11.3
#if (defined(CUDA_VERSION) && CUDA_VERSION < 11030) && !defined(USE_ROCM)
      if (num_indices < 50000) {
        index_put_with_sort_kernel_thrust_helper(linearIndex, orig_indices, sorted_indices, num_indices);
      } else
#endif
      {
      // Sort the inputs into sorted with the corresponding indices
      auto range = at::arange(num_indices, linearIndex.options());
      // linearIndex can not be negative, and we take advantage of this
      // fact to sort on less bits for better performance.
      int64_t nbits = cuda::hipcub::get_num_bits(largestIndex(self_) / sliceSize);
      cuda::hipcub::radix_sort_pairs(
        linearIndex.const_data_ptr<int64_t>(), sorted_indices.mutable_data_ptr<int64_t>(),
        range.const_data_ptr<int64_t>(), orig_indices.mutable_data_ptr<int64_t>(),
        num_indices, false, 0, nbits);
      }

      TORCH_INTERNAL_ASSERT(
          linearIndex.numel()*sliceSize*nElemBefore == expandedValue.numel(),
          "number of flattened indices did not match number of elements in the value tensor: ",
          linearIndex.numel()*sliceSize*nElemBefore, " vs ", expandedValue.numel());

      const int UNROLL = 4;
      const int indices_per_block = 4;
      const int warp_size = at::cuda::warp_size();
      dim3 grid(ceil_div(num_indices, (int64_t) indices_per_block),
           std::min<int>(at::cuda::getCurrentDeviceProperties()->maxGridSize[1], ceil_div(sliceSize, (int64_t) (warp_size*UNROLL))),
           std::min(std::max<int>(1,nElemBefore), at::cuda::getCurrentDeviceProperties()->maxGridSize[2]));
      dim3 block(warp_size, indices_per_block);

#ifdef USE_ROCM
      dim3 new_grid(ceil_div(num_indices, (int64_t) (indices_per_block * warp_size)), grid.y, grid.z);
      size_t smem_dups_size = indices_per_block * warp_size * sizeof(int64_t);
#define KERNEL_GRID new_grid
#define KERNEL_SMEM smem_dups_size
#else
#define KERNEL_GRID grid
#define KERNEL_SMEM 0
#endif

      if (sliceSize == 1) {
        // This implementation is faster with high amounts of duplicates but could overflow
        // if FP16 / BF16 is used
        AT_DISPATCH_V2(
          expandedValue.scalar_type(),
          "indexing_backward_kernel_stride_1",
          AT_WRAP([&] {
            indexing_backward_kernel_stride_1<scalar_t><<<KERNEL_GRID, block, KERNEL_SMEM, stream>>>
            (
              sorted_indices.const_data_ptr<int64_t>(),
              orig_indices.const_data_ptr<int64_t>(),
              expandedValue.const_data_ptr<scalar_t>(),
              src_.mutable_data_ptr<scalar_t>(),
              num_indices,
              sliceSize,
              strideBefore,
              nElemBefore,
              accumulate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          }),
          AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX),
          // AT_EXPAND(AT_FLOAT8_TYPES),
          // TODO(#113663): clean up accumulation behavior in float8 dtypes, accumulate=True
          // should not be supported here, then reenable AT_FLOAT8_DTYPES
          kFloat8_e4m3fn,
          kFloat8_e5m2,
          kFloat8_e4m3fnuz,
          kFloat8_e5m2fnuz,
          kComplexHalf,
          kHalf,
          kBool,
          kBFloat16);
      } else {
        if (sliceSize <= warp_size) {
          AT_DISPATCH_V2(
            expandedValue.scalar_type(),
            "indexing_backward_kernel_small_stride",
            AT_WRAP([&] {
              indexing_backward_kernel_small_stride<scalar_t><<<grid, block, 0, stream>>>(
                sorted_indices.const_data_ptr<int64_t>(),
                orig_indices.const_data_ptr<int64_t>(),
                expandedValue.const_data_ptr<scalar_t>(),
                src_.mutable_data_ptr<scalar_t>(),
                num_indices,
                sliceSize,
                strideBefore,
                nElemBefore,
                accumulate);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }),
            AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX),
            // AT_EXPAND(AT_FLOAT8_TYPES),
            // TODO(#113663): clean up accumulation behavior in float8 dtypes, accumulate=True
            // should not be supported here, then reenable AT_FLOAT8_DTYPES
            kFloat8_e4m3fn,
            kFloat8_e5m2,
            kFloat8_e4m3fnuz,
            kFloat8_e5m2fnuz,
            kComplexHalf,
            kHalf,
            kBool,
            kBFloat16);
        } else {
          AT_DISPATCH_V2(
            expandedValue.scalar_type(),
            "indexing_backward",
            AT_WRAP([&] {
              indexing_backward_kernel<scalar_t, UNROLL><<<KERNEL_GRID, block, KERNEL_SMEM, stream>>>(
                sorted_indices.const_data_ptr<int64_t>(),
                orig_indices.const_data_ptr<int64_t>(),
                expandedValue.const_data_ptr<scalar_t>(),
                src_.mutable_data_ptr<scalar_t>(),
                num_indices,
                sliceSize,
                strideBefore,
                nElemBefore,
                accumulate);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }),
            AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX),
            // AT_EXPAND(AT_FLOAT8_TYPES),
            // TODO(#113663): clean up accumulation behavior in float8 dtypes, accumulate=True
            // should not be supported here, then reenable AT_FLOAT8_DTYPES
            kFloat8_e4m3fn,
            kFloat8_e5m2,
            kFloat8_e4m3fnuz,
            kFloat8_e5m2fnuz,
            kComplexHalf,
            kHalf,
            kBool,
            kBFloat16);
        }
      }

#undef KERNEL_GRID
#undef KERNEL_SMEM

      if (permuted) {
        self.copy_(src_.permute(inversePerm));
      } else if (!self_contiguous) {
        self.copy_(self_);
      }
  }
}

REGISTER_CUDA_DISPATCH(index_put_with_sort_stub, &index_put_with_sort_kernel)

void index_put_with_sort_quantized(Tensor & self, const c10::List<std::optional<Tensor>>& indices, const Tensor & value, double scale, int zero_point, bool unsafe) {
  if (indices.size() > (size_t)self.dim()) {
    TORCH_CHECK_INDEX(false, "too many indices for tensor of dimension ", self.dim(), " (got ", indices.size(), ")");
  }
  bool self_contiguous = self.is_contiguous();
  auto self_ = self_contiguous ? self : self.contiguous();
  Tensor linearIndex, src, expandedValue = value;
  int64_t nElemBefore, strideBefore, sliceSize;
  std::vector<int64_t> inversePerm;
  std::tie(linearIndex, src, nElemBefore, strideBefore, sliceSize, inversePerm) = makeLinearIndex(self_, indices, !unsafe);
  int64_t num_indices = linearIndex.numel();

  if (expandedValue.numel() < num_indices * nElemBefore * sliceSize) {
    auto expanded_size = at::DimVector(expandedValue.sizes());
    auto size1 = expandedValue.sizes();
    auto size2 = linearIndex.sizes();
    if (are_expandable(size1, size2)) {
      expanded_size = infer_size_dimvector(size1, size2);
    }
    if (nElemBefore > 1) {
      expanded_size.insert(expanded_size.begin(), nElemBefore);
    }
    expandedValue = expandedValue.expand(expanded_size);
  }
  expandedValue = expandedValue.contiguous();

  if (num_indices > 0 && sliceSize > 0) {
      const bool permuted = !src.is_contiguous();
      auto src_ = permuted ? src.contiguous() : src;
      linearIndex = linearIndex.reshape(-1);
      auto sorted_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      auto orig_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      const hipStream_t stream = at::cuda::getCurrentCUDAStream();

      linearIndex.divide_(sliceSize, "trunc");

      // cub on CUDA <= 11.2 have a bug that for small sizes
      // cub's sort can be much slower than thrust's merge sort
      // this bug is fixed in CUDA 11.3
#if (defined(CUDA_VERSION) && CUDA_VERSION < 11030) && !defined(USE_ROCM)
      if (num_indices < 50000) {
        index_put_with_sort_kernel_thrust_helper(linearIndex, orig_indices, sorted_indices, num_indices);
      } else
#endif
      {
      // Sort the inputs into sorted with the corresponding indices
      auto range = at::arange(num_indices, linearIndex.options());
      // linearIndex can not be negative, and we take advantage of this
      // fact to sort on less bits for better performance.
      int64_t nbits = cuda::hipcub::get_num_bits(largestIndex(self_) / sliceSize);
      cuda::hipcub::radix_sort_pairs(
        linearIndex.const_data_ptr<int64_t>(), sorted_indices.mutable_data_ptr<int64_t>(),
        range.const_data_ptr<int64_t>(), orig_indices.mutable_data_ptr<int64_t>(),
        num_indices, false, 0, nbits);
      }

      TORCH_INTERNAL_ASSERT(
          linearIndex.numel()*sliceSize*nElemBefore == expandedValue.numel(),
          "number of flattened indices did not match number of elements in the value tensor: ",
          linearIndex.numel()*sliceSize*nElemBefore, " vs ", expandedValue.numel());
      const int UNROLL = 4;
      const int indices_per_block = 4;
      const int warp_size = at::cuda::warp_size();
      dim3 grid(ceil_div(num_indices, (int64_t) indices_per_block),
           std::min<int>(at::cuda::getCurrentDeviceProperties()->maxGridSize[1], ceil_div(sliceSize, (int64_t) (warp_size*UNROLL))),
           std::min(std::max<int>(1,nElemBefore), at::cuda::getCurrentDeviceProperties()->maxGridSize[2]));
      dim3 block(warp_size, indices_per_block);

      AT_DISPATCH_QINT_TYPES(
        src.scalar_type(), "indexing_backward_quantized", [&] {
        constexpr int64_t qmin = std::numeric_limits<typename scalar_t::underlying>::min();
        constexpr int64_t qmax = std::numeric_limits<typename scalar_t::underlying>::max();
        float inv_scale = 1.0f / static_cast<float>(scale);

        indexing_backward_kernel_quantized<scalar_t, UNROLL><<<grid, block, 0, stream>>>(
          sorted_indices.const_data_ptr<int64_t>(),
          orig_indices.const_data_ptr<int64_t>(),
          expandedValue.const_data_ptr<float>(),
          src_.mutable_data_ptr<scalar_t>(),
          num_indices,
          sliceSize,
          strideBefore,
          nElemBefore,
          inv_scale,
          zero_point,
          qmin,
          qmax);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });

      if (permuted) {
        self.copy_(src_.permute(inversePerm));
      } else if (!self_contiguous) {
        self.copy_(self_);
      }
  }
}

REGISTER_CUDA_DISPATCH(index_put_with_sort_quantized_stub, &index_put_with_sort_quantized)
} //anonymous


// Check tensor dimensions for index operations, and return the slice size.
static size_t getSliceSize(const Tensor & dst,
                              int dim,
                              const Tensor & index,
                              const Tensor & src)
{
  const auto dstDims = dst.dim();
  const auto srcDims = src.dim();

  TORCH_CHECK(index.dim() <= 1, "Index must be vector or scalar");

  size_t dstSliceSize = 1;
  TORCH_CHECK(dim >= 0 && dim < dstDims, "Indexing dim ", dim, " is out of bounds");
  for (const auto d: c10::irange(dstDims)) {
    if (d != dim) {
      dstSliceSize *= dst.size(d);
    }
  }

  TORCH_CHECK(dim < srcDims, "Indexing dim ", dim, " is out of bounds");
  TORCH_CHECK(index.numel() == src.size(dim),
             "length of src.size[dim] is not equal to length of indices");

  size_t srcSliceSize = 1;
  bool mismatch = false;

  if (dstDims != srcDims) mismatch = true;

  for (const auto d: c10::irange(srcDims)) {
    if (d != dim) {
      srcSliceSize *= src.size(d);
      if (!mismatch && dst.size(d) != src.size(d)) mismatch = true;
    }
  }

  TORCH_CHECK(dstSliceSize == srcSliceSize,
             "Source/destination tensor have different slice sizes (%ld vs %ld)",
             dstSliceSize, srcSliceSize);

  if (mismatch) {
    TORCH_WARN_ONCE(
        "Warning: source/destination slices have same size but different "
        "shape for an index operation.  This behavior is deprecated.\n");
  }

  return dstSliceSize;
}

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexFuncLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim,
          typename func_t>
__global__ void indexFuncSmallIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                    cuda::detail::TensorInfo<const T, IndexType> src,
                                    cuda::detail::TensorInfo<const IndicesType, IndexType> indices,
                                    int dstAddDim,
                                    int srcAddDim,
                                    IndexType innerSize,
                                    int64_t dstAddDimSize,
                                    int64_t dstNumel,
                                    const func_t& op,
                                    T alpha) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType srcIndex = 0; srcIndex < indices.sizes[0]; ++srcIndex) {
    // Lua indices begin at 1
    IndexType dstIndex =
        indices.data[cuda::detail::IndexToOffset<const IndicesType, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstAddDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
          cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex * dst.strides[dstAddDim];

      IndexType srcOffset =
          cuda::detail::IndexToOffset<const T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcAddDim];

      T val = src.data[srcOffset] * alpha;
      op(dst.data, dstOffset, dstNumel, &val);
    }

  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexFuncSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim,
          bool IndexIsMajor, typename func_t>
__global__ void indexFuncLargeIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                    cuda::detail::TensorInfo<const T, IndexType> src,
                                    cuda::detail::TensorInfo<const IndicesType, IndexType> indices,
                                    int dstAddDim,
                                    int srcAddDim,
                                    IndexType totalSize,
                                    IndexType innerSize,
                                    int64_t dstAddDimSize,
                                    int64_t dstNumel,
                                    const func_t& op,
                                    T alpha) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalSize;
       linearIndex += gridDim.x * blockDim.x) {
    IndexType srcIndex, elementInSlice;
    if (IndexIsMajor) {
      srcIndex = linearIndex / innerSize;
      elementInSlice = linearIndex % innerSize;
    }
    else {
      elementInSlice = linearIndex / innerSize;
      srcIndex = linearIndex % innerSize;
    }

    // Lua indices begin at 1
    IndexType dstIndex =
        indices.data[cuda::detail::IndexToOffset<const IndicesType, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstAddDimSize);

    IndexType dstOffset =
      cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstAddDim];

    IndexType srcOffset =
      cuda::detail::IndexToOffset<const T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcAddDim];

    T val = src.data[srcOffset] * alpha;
    op(dst.data, dstOffset, dstNumel, &val);
  }
}

// Compare the stride between adjacent slices (sliceStride) with strides in the
// other dimensions (i.e., strides *inside* each slice).
//
// - Returns true if some dimension inside the slice has lower stride than
//   sliceStride.  The simplest example is a 2-D contiguous tensor with sliceDim
//   == 0 (that is, each slice is a row).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "index-major order".  For example, if thread count equals slice size, then
//   all threads process slice #0 in lockstep, and then slice #1, and so on.
//
// - Otherwise (i.e., sliceStride has the lowest value), this function returns
//   false.  The simplest example is a 2-D contiguous tensor with sliceDim == 1
//   (each slice is a column).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "elementInSlice-major order".  For example, each thread can process element
//   #0 of every slice, and then element #1 of every slice, and so on.
template <typename scalar_t>
bool indexShouldBeMajor(cuda::detail::TensorInfo<scalar_t, unsigned int> &info,
                                    int sliceDim)
{
  // The stride between adjacent slices (e.g., between element #0 of slice #100
  // and element #0 of slice #101).
  unsigned int sliceStride = info.strides[sliceDim];

  for (const auto i: c10::irange(info.dims)) {
    if (i != sliceDim && info.sizes[i] > 1 && info.strides[i] < sliceStride) {
      return true;
    }
  }

  return false;
}

void index_add_cuda_impl(const Tensor& self, int64_t dim, const Tensor& index, const Tensor& source, const Scalar& alpha, const Tensor& result) {
  if (!result.is_same(self)) {
    result.copy_(self);
  }

  // Scalars are treated as 1-d tensor
  const Tensor self_ = (result.dim() == 0) ? result.view(1) : result;
  const Tensor source_ = (source.dim() == 0) ? source.view(1) : source;

  TORCH_CHECK(result.dim() <= MAX_TENSORINFO_DIMS, "tensor has too many (>", MAX_TENSORINFO_DIMS, ") dims");
  TORCH_CHECK(source.dim() <= MAX_TENSORINFO_DIMS, "tensor has too many (>", MAX_TENSORINFO_DIMS, ") dims" );
  TORCH_CHECK(index.dim() <= MAX_TENSORINFO_DIMS, "tensor has too many (>", MAX_TENSORINFO_DIMS, ") dims");

  if (globalContext().deterministicAlgorithms()){
    torch::List<std::optional<Tensor>> indices;
    indices.reserve(dim + 1);
    for (const auto i: c10::irange(dim)) {
      indices.emplace_back();
    }
    indices.emplace_back(index.to(at::kLong));
    result.index_put_(indices, source * alpha, true);
    return;
  }

  // The `source` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of index we are choosing, which is the total size
  // of the tensor `index`.
  const uint64_t sliceSize = getSliceSize(self_, dim, index, source_);
  const uint64_t sourceTotalSize = source.numel();
  const uint64_t selfAddDimSize = self_.size(dim);
  const uint64_t numIndex = index.numel();
  const uint64_t selfNumel = self_.numel();

  if (sliceSize == 0) {
    return;
  }
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  const bool indContig = index.is_contiguous();

  const int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM)     \
  indexFuncSmallIndex<TENSOR_TYPE, INDICES_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM>   \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(                                   \
      selfInfo, sourceInfo, indexInfo,                                                  \
      selfAddDim, sourceAddDim, sliceSize, selfAddDimSize,                              \
      selfNumel, reduce_add, alpha_value);                                              \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define LARGE_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE,                        \
                    SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR)            \
  indexFuncLargeIndex<TENSOR_TYPE, INDICES_TYPE, TYPE,                      \
                      SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR>          \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                       \
      selfInfo, sourceInfo, indexInfo,                                      \
      selfAddDim, sourceAddDim, sourceTotalSize,                            \
      (IDX_IS_MAJOR) ? sliceSize : numIndex,                                \
      selfAddDimSize, selfNumel, reduce_add, alpha_value);                  \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  uint64_t defaultMaxBlockThreads = getDefaultMaxThreadsPerBlock();
  const dim3 smallIndexGrid(std::min(ceil_div(sliceSize, (uint64_t)128), (uint64_t)(mpc * 8)));
  const dim3 smallIndexBlock(std::min(sliceSize, (uint64_t)128));

  const dim3 largeIndexGrid(std::min(ceil_div(sourceTotalSize, (uint64_t)128), (uint64_t)(mpc * 8)));
  //On ROCm, std::min -> ::min did not work as expected on when outTotalSize>=2147483648
  dim3 largeIndexBlock( (sourceTotalSize < defaultMaxBlockThreads) ? sourceTotalSize : defaultMaxBlockThreads );

  if (cuda::detail::canUse32BitIndexMath(result) &&
      cuda::detail::canUse32BitIndexMath(source) &&
      cuda::detail::canUse32BitIndexMath(index)) {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND4(at::ScalarType::Bool, at::ScalarType::Half, at::ScalarType::BFloat16, at::ScalarType::ComplexHalf, result.scalar_type(), "index_add", [&] {
      cuda::detail::TensorInfo<scalar_t, unsigned int> selfInfo =
          cuda::detail::getTensorInfo<scalar_t, unsigned int>(self_);
      const int selfAddDim = selfInfo.collapseDims(dim);
      selfInfo.reduceDim(selfAddDim);
      const auto alpha_value = alpha.to<scalar_t>();
      AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_add_cuda_", [&] () {
        auto sourceInfo =
          cuda::detail::getTensorInfo<const scalar_t, unsigned int>(source_);
        const int sourceAddDim = sourceInfo.collapseDims(dim);
        sourceInfo.reduceDim(sourceAddDim);

        auto indexInfo =
        cuda::detail::getTensorInfo<const index_t, unsigned int>(index);
        indexInfo.collapseDims();

        // A reasonable choice for when to have each thread iterate over
        // index to choose
        if (numIndex <= 16) {
          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2);
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2);
          } else {
            SMALL_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1);
          }
        } else {
          const bool indexIsMajor = indexShouldBeMajor(selfInfo, selfAddDim);

          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2, true);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, true);
            } else {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, false);
            }
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, true);
            } else {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, false);
            }
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1, true);
          }
        }
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(at::ScalarType::Bool, at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "index_add", [&] {
      cuda::detail::TensorInfo<scalar_t, uint64_t> selfInfo =
        cuda::detail::getTensorInfo<scalar_t, uint64_t>(self_);
      const int selfAddDim = selfInfo.collapseDims(dim);
      selfInfo.reduceDim(selfAddDim);
      const auto alpha_value = alpha.to<scalar_t>();

      cuda::detail::TensorInfo<const scalar_t, uint64_t> sourceInfo =
        cuda::detail::getTensorInfo<const scalar_t, uint64_t>(source_);
      const int sourceAddDim = sourceInfo.collapseDims(dim);
      sourceInfo.reduceDim(sourceAddDim);

      AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_add_cuda_", [&] () {
        cuda::detail::TensorInfo<const index_t, uint64_t> indexInfo =
          cuda::detail::getTensorInfo<const index_t, uint64_t>(index);
        indexInfo.collapseDims();

        LARGE_INDEX(scalar_t, index_t, uint64_t, -1, -1, -1, true);
      });
    });
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

template <typename func_t>
void index_reduce_func_cuda_impl(
  const Tensor& self,
  int64_t dim,
  const Tensor& index,
  const Tensor& source,
  bool include_self,
  const ReductionType& reduce,
  const func_t& reduce_func,
  const Tensor& result) {
  globalContext().alertNotDeterministic("index_reduce_cuda");

  if (!result.is_same(self)) result.copy_(self);

  // Scalars are treated as 1-d tensor
  Tensor self_ = (result.dim() == 0) ? result.view(1) : result;
  Tensor source_ = (source.dim() == 0) ? source.view(1) : source;

  TORCH_CHECK(result.dim() <= MAX_TENSORINFO_DIMS, "tensor has too many (>", MAX_TENSORINFO_DIMS, ") dims");
  TORCH_CHECK(source.dim() <= MAX_TENSORINFO_DIMS, "tensor has too many (>", MAX_TENSORINFO_DIMS, ") dims" );
  TORCH_CHECK(index.dim() <= MAX_TENSORINFO_DIMS, "tensor has too many (>", MAX_TENSORINFO_DIMS, ") dims");

  if (!include_self) {
    AT_DISPATCH_ALL_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      self.scalar_type(), "index_reduce_func_cuda_exclude_input_init", [&] {
      scalar_t init_val;
      switch (reduce) {
        case ReductionType::PROD:
          init_val = (scalar_t)1;
          break;
        case ReductionType::MAX:
          init_val = std::numeric_limits<scalar_t>::has_infinity ? -std::numeric_limits<scalar_t>::infinity()
                     : std::numeric_limits<scalar_t>::lowest();
          break;
        case ReductionType::MIN:
          init_val = std::numeric_limits<scalar_t>::has_infinity ? std::numeric_limits<scalar_t>::infinity()
                     : std::numeric_limits<scalar_t>::max();
          break;
        default:
          init_val = (scalar_t)0;
          break;
      }
      // index_fill_ requires index to be a LongTensor
      self_.index_fill_(dim, index.to(at::ScalarType::Long), init_val);
    });
  }

  // The `source` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of index we are choosing, which is the total size
  // of the tensor `index`.
  uint64_t sliceSize = getSliceSize(self_, dim, index, source_);
  uint64_t sourceTotalSize = source.numel();
  uint64_t selfReduceDimSize = self_.size(dim);
  uint64_t numIndex = index.numel();
  uint64_t selfNumel = self_.numel();

  if (sliceSize == 0) {
    return;
  }
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  bool indContig = index.is_contiguous();

  int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM)                  \
  indexFuncSmallIndex<TENSOR_TYPE, INDICES_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM>                \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(                                                \
      selfInfo, sourceInfo, indexInfo,                                                               \
      selfReduceDim, sourceReduceDim, sliceSize, selfReduceDimSize,                                  \
      selfNumel, reduce_func, alpha_value);                                                          \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define LARGE_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE,                                     \
                    SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR)                         \
  indexFuncLargeIndex<TENSOR_TYPE, INDICES_TYPE, TYPE,                                   \
                     SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR>                        \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                                    \
      selfInfo, sourceInfo, indexInfo,                                                   \
      selfReduceDim, sourceReduceDim, sourceTotalSize,                                   \
      (IDX_IS_MAJOR) ? sliceSize : numIndex,                                             \
      selfReduceDimSize, selfNumel, reduce_func, alpha_value);                           \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  uint64_t defaultMaxBlockThreads = getDefaultMaxThreadsPerBlock();
  dim3 smallIndexGrid(std::min(ceil_div(sliceSize, (uint64_t)128), (uint64_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (uint64_t)128));

  dim3 largeIndexGrid(std::min(ceil_div(sourceTotalSize, (uint64_t)128), (uint64_t)(mpc * 8)));
  //On ROCm, std::min -> ::min did not work as expected on when outTotalSize>=2147483648
  dim3 largeIndexBlock( (sourceTotalSize < defaultMaxBlockThreads) ? sourceTotalSize : defaultMaxBlockThreads );

  if (cuda::detail::canUse32BitIndexMath(result) &&
      cuda::detail::canUse32BitIndexMath(source) &&
      cuda::detail::canUse32BitIndexMath(index)) {
    AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, result.scalar_type(), "index_reduce", [&] {
      cuda::detail::TensorInfo<scalar_t, unsigned int> selfInfo =
          cuda::detail::getTensorInfo<scalar_t, unsigned int>(self_);
      int selfReduceDim = selfInfo.collapseDims(dim);
      selfInfo.reduceDim(selfReduceDim);
      auto alpha_value = (scalar_t) 1;
      AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_reduce_cuda", [&] () {
        auto sourceInfo =
          cuda::detail::getTensorInfo<const scalar_t, unsigned int>(source_);
        int sourceReduceDim = sourceInfo.collapseDims(dim);
        sourceInfo.reduceDim(sourceReduceDim);

        auto indexInfo =
        cuda::detail::getTensorInfo<const index_t, unsigned int>(index);
        indexInfo.collapseDims();

        // A reasonable choice for when to have each thread iterate over
        // index to choose
        if (numIndex <= 16) {
          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2);
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2);
          } else {
            SMALL_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1);
          }
        } else {
          bool indexIsMajor = indexShouldBeMajor(selfInfo, selfReduceDim);

          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2, true);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, true);
            } else {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, false);
            }
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, true);
            } else {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, false);
            }
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1, true);
          }
        }
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "index_reduce", [&] {
      cuda::detail::TensorInfo<scalar_t, uint64_t> selfInfo =
        cuda::detail::getTensorInfo<scalar_t, uint64_t>(self_);
      int selfReduceDim = selfInfo.collapseDims(dim);
      selfInfo.reduceDim(selfReduceDim);
      auto alpha_value = (scalar_t) 1;

      cuda::detail::TensorInfo<const scalar_t, uint64_t> sourceInfo =
        cuda::detail::getTensorInfo<const scalar_t, uint64_t>(source_);
      int sourceReduceDim = sourceInfo.collapseDims(dim);
      sourceInfo.reduceDim(sourceReduceDim);

      AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_reduce_cuda", [&] () {
        cuda::detail::TensorInfo<const index_t, uint64_t> indexInfo =
          cuda::detail::getTensorInfo<const index_t, uint64_t>(index);
        indexInfo.collapseDims();

        LARGE_INDEX(scalar_t, index_t, uint64_t, -1, -1, -1, true);
      });
    });
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

TORCH_IMPL_FUNC(index_add_cuda_out)
(const Tensor& self, int64_t dim, const Tensor& index, const Tensor& source, const Scalar& alpha, const Tensor& result) {
  index_add_cuda_impl(self, dim, index, source, alpha, result);
}

TORCH_IMPL_FUNC(index_reduce_cuda_out)
(const Tensor& self,
 int64_t dim,
 const Tensor& index,
 const Tensor& source,
 const std::string_view reduce,
 bool include_self,
 const Tensor& result) {
  TORCH_WARN_ONCE("index_reduce() is in beta and the API may change at any time.");

  if (reduce == "prod") {
    index_reduce_func_cuda_impl(self, dim, index, source, include_self, ReductionType::PROD, reduce_multiply, result);
  } else if (reduce == "mean") {
    index_reduce_func_cuda_impl(self, dim, index, source, include_self, ReductionType::MEAN, reduce_add, result);
    auto counts = include_self ? at::ones_like(result) : at::zeros_like(result);
    counts.index_add_(dim, index, at::ones_like(source));
    counts.masked_fill_(counts == 0, 1);
    if (result.is_floating_point() || result.is_complex()) {
      result.div_(counts);
    } else {
      result.div_(counts, "floor");
    }
  } else if (reduce == "amax") {
    index_reduce_func_cuda_impl(self, dim, index, source, include_self, ReductionType::MAX, reduce_maximum, result);
  } else if (reduce == "amin") {
    index_reduce_func_cuda_impl(self, dim, index, source, include_self, ReductionType::MIN, reduce_minimum, result);
  } else {
    TORCH_CHECK(false, "reduce argument must be either prod, mean, amax or amin, got ", reduce, ".");
  }
}

namespace {
// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexSelectLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexSelectSmallIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                      cuda::detail::TensorInfo<const T, IndexType> src,
                                      cuda::detail::TensorInfo<const IndicesType, IndexType> indices,
                                      int dstSelectDim,
                                      int srcSelectDim,
                                      IndexType innerSize,
                                      int64_t srcSelectDimSize) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType dstIndex = 0; dstIndex < indices.sizes[0]; ++dstIndex) {
    IndexType srcIndex =
      indices.data[cuda::detail::IndexToOffset<const IndicesType, IndexType, IdxDim>::get(dstIndex, indices)];
    CUDA_KERNEL_ASSERT(srcIndex < srcSelectDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
        cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex * dst.strides[dstSelectDim];

      IndexType srcOffset =
        cuda::detail::IndexToOffset<const T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcSelectDim];

      dst.data[dstOffset] = src.data[srcOffset];
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexSelectSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim,
          bool IndexIsMajor>
__global__ void indexSelectLargeIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                      cuda::detail::TensorInfo<const T, IndexType> src,
                                      cuda::detail::TensorInfo<const IndicesType, IndexType> indices,
                                      int dstSelectDim,
                                      int srcSelectDim,
                                      IndexType totalSize,
                                      IndexType innerSize,
                                      int64_t srcSelectDimSize) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  constexpr bool kPack4 = (sizeof(T) == 2);
  constexpr int kElemsPerVec = kPack4 ? 4 : 1;

  for (IndexType vecLinear = (blockIdx.x * blockDim.x + threadIdx.x) * kElemsPerVec;
       vecLinear < totalSize;
       vecLinear += gridDim.x * blockDim.x * kElemsPerVec) {

    IndexType linearIndex = vecLinear;
    IndexType dstIndex, elementInSlice;
    if constexpr (IndexIsMajor) {
      dstIndex = linearIndex / innerSize;
      elementInSlice = linearIndex % innerSize;
    }
    else {
      elementInSlice = linearIndex / innerSize;
      dstIndex = linearIndex % innerSize;
    }

    IndexType srcIndex =
      indices.data[cuda::detail::IndexToOffset<const IndicesType, IndexType, IdxDim>::get(dstIndex, indices)];
    CUDA_KERNEL_ASSERT(srcIndex < srcSelectDimSize);

    IndexType dstOffset =
      cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstSelectDim];

    IndexType srcOffset =
      cuda::detail::IndexToOffset<const T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcSelectDim];
    if constexpr (kPack4) {
      IndexType srcNextOffset = cuda::detail::IndexToOffset<const T, IndexType, SrcDim>::get(elementInSlice + 1, src) + srcIndex * src.strides[srcSelectDim];
      IndexType dstNextOffset = cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(elementInSlice + 1, dst) + dstIndex * dst.strides[dstSelectDim];

      bool inner_contiguous = (srcNextOffset - srcOffset == 1) && (dstNextOffset - dstOffset == 1);
      bool slic_has_4 = (elementInSlice + 3 < innerSize);
      bool aligned = (((uintptr_t)(dst.data + dstOffset) & 7)==0) && (((uintptr_t)(src.data + srcOffset) & 7)==0);

      bool can_vectorize = IndexIsMajor && inner_contiguous && slic_has_4 && aligned;
      #if defined(__CUDA_ARCH__)
        bool warp_fast = __all_sync(0xffffffffu, can_vectorize);
      #elif defined(__HIP_DEVICE_COMPILE__)
        unsigned long long mask = __ballot(can_vectorize);
        bool warp_fast = (mask == 0xffffffff);
      #else
        bool warp_fast = can_vectorize;
      #endif

      if (warp_fast) {
        uint64_t tmp;
        memcpy(&tmp, src.data + srcOffset, 8);
        memcpy(dst.data + dstOffset, &tmp, 8);
      } else {
      #pragma unroll
      for (int i = 0; i < kElemsPerVec; ++i) {
        IndexType li = linearIndex + i;
        if (li >= totalSize) break;

        IndexType dstIndex2, elem2;
        if constexpr (IndexIsMajor) {
          dstIndex2 = li / innerSize;
          elem2 = li % innerSize;
        } else {
          elem2 = li / innerSize;
          dstIndex2 = li % innerSize;
        }

        IndexType srcIndex2 = indices.data[cuda::detail::IndexToOffset<const IndicesType, IndexType, IdxDim>::get(dstIndex2, indices)];
        CUDA_KERNEL_ASSERT(srcIndex2 < srcSelectDimSize);

        IndexType dstOffset2 = cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(elem2, dst) + dstIndex2 * dst.strides[dstSelectDim];
        IndexType srcOffset2 = cuda::detail::IndexToOffset<const T, IndexType, SrcDim>::get(elem2, src) + srcIndex2 * src.strides[srcSelectDim];
        dst.data[dstOffset2] = src.data[srcOffset2];
        }
      }
    }
    else {
      dst.data[dstOffset] = src.data[srcOffset];
    }
  }
}

namespace {

// When using a 0-dim scalar tensor, we need the legacy (THC) semantics of
// TensorInfo: Pretend that the scalar tensor is in fact a one-element vector.
template <typename T, typename IndexType>
cuda::detail::TensorInfo<T, IndexType>
tensorInfoLegacyIfScalar(cuda::detail::TensorInfo<T, IndexType> ti) {
  if (ti.dims == 0) {
    ti.dims = 1;
    ti.sizes[0] = 1;
    ti.strides[0] = 1;
  }
  return ti;
}


}


template <typename scalar_t>
void index_select_out_cuda_impl(
    Tensor& out,
    const Tensor& self,
    int64_t dim,
    const Tensor& index) {
  uint64_t numIndices = index.numel();
  auto selfDims = self.dim() == 0 ? 1 : self.dim();

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(
      index.dim() <= 1, "Index is supposed to be an empty tensor or a vector");
  TORCH_CHECK(
      !(self.dim() == 0 && numIndices != 1), "index_select(): Index to scalar can have only 1 value, got ", numIndices, " value(s)");
  TORCH_CHECK(dim < selfDims, "Indexing dim is out of bounds");

  std::vector<int64_t> newSize = self.sizes().vec();
  if (self.dim() > 0) {
    newSize[dim] = numIndices;
  }

  if (self.is_quantized()){
      out = at::empty_quantized(newSize, out);
  } else {
    at::native::resize_output(out, newSize);
  }

  uint64_t outTotalSize = out.numel();
  if (outTotalSize == 0) {
    return;
  }

  bool indContig = index.is_contiguous();

  // The `self` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  uint64_t selfSelectDimSize = self.dim() == 0 ? 1 : self.size(dim);
  uint64_t sliceSize = outTotalSize / numIndices;

  int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)         \
  indexSelectSmallIndex<TENSOR_TYPE, INDICES_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>     \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(                                   \
      outInfo, selfInfo, indicesInfo,                                                   \
      outSelectDim, selfSelectDim, static_cast<TYPE>(sliceSize),                        \
      selfSelectDimSize);                                                               \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define LARGE_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE,                           \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)                   \
  indexSelectLargeIndex<TENSOR_TYPE, INDICES_TYPE, TYPE,                       \
                        DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR>               \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                          \
      outInfo, selfInfo, indicesInfo,                                          \
      outSelectDim, selfSelectDim, static_cast<TYPE>(outTotalSize),            \
      static_cast<TYPE>((IDX_IS_MAJOR) ? sliceSize : numIndices),              \
      selfSelectDimSize);                                                      \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  uint64_t defaultMaxBlockThreads = getDefaultMaxThreadsPerBlock();
  dim3 smallIndexGrid(std::min(ceil_div(sliceSize, defaultMaxBlockThreads), (uint64_t) (mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, defaultMaxBlockThreads));

  dim3 largeIndexGrid(std::min(ceil_div(outTotalSize, defaultMaxBlockThreads), (uint64_t) (mpc * 8)));
  // for issue https://github.com/pytorch/pytorch/issues/130806 there are two problems
  // 1: ptrdiff_t was used but it is signed int,  outTotalSize of 2147483648 can cause overflow
  // 2: On ROCm, std::min -> ::min did not work as expected on when outTotalSize>=2147483648
  dim3 largeIndexBlock( (outTotalSize < defaultMaxBlockThreads) ? outTotalSize : defaultMaxBlockThreads );

  if (cuda::detail::canUse32BitIndexMath(out) &&
      cuda::detail::canUse32BitIndexMath(self) &&
      cuda::detail::canUse32BitIndexMath(index)) {
    auto outInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, unsigned int>(out));
    int outSelectDim = outInfo.collapseDims(dim);
    outInfo.reduceDim(outSelectDim);

    auto  selfInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<const scalar_t, unsigned int>(self));
    int selfSelectDim = selfInfo.collapseDims(dim);
    selfInfo.reduceDim(selfSelectDim);

    AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_select_out_cuda_impl", [&] () {
      auto indicesInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<const index_t, unsigned int>(index));
      indicesInfo.collapseDims();

      // A reasonable choice for when to have each thread iterate over
      // indices to choose
      if (numIndices <= 16) {
        if (outInfo.dims == 1 && selfInfo.dims == 1 && indContig) {
          SMALL_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2);
        } else if (outInfo.dims == 2 && selfInfo.dims == 2 && indContig) {
          SMALL_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2);
        } else if (outInfo.dims == 3 && selfInfo.dims == 3 && indContig) {
          SMALL_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2);
        } else {
          SMALL_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1);
        }
      } else {
        bool indexIsMajor = indexShouldBeMajor(outInfo, outSelectDim);

        if (outInfo.dims == 1 && selfInfo.dims == 1 && indContig) {
          LARGE_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2, true);
        } else if (outInfo.dims == 2 && selfInfo.dims == 2 && indContig) {
          if (indexIsMajor) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, true);
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, false);
          }
        } else if (outInfo.dims == 3 && selfInfo.dims == 3 && indContig) {
          if (indexIsMajor) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, true);
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, false);
          }
        } else {
          if (indexIsMajor) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1, true);
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1, false);
          }

        }
      }
    });
  } else {
    auto outInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, uint64_t>(out));
    int outSelectDim = outInfo.collapseDims(dim);
    outInfo.reduceDim(outSelectDim);

    auto selfInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<const scalar_t, uint64_t>(self));
    int selfSelectDim = selfInfo.collapseDims(dim);
    selfInfo.reduceDim(selfSelectDim);
    AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_select_out_cuda_impl", [&] () {
      auto indicesInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<const index_t, uint64_t>(index));
      indicesInfo.collapseDims();

      LARGE_INDEX(scalar_t, index_t, uint64_t, -1, -1, -1, true);
    });
  }
#undef SMALL_INDEX
#undef LARGE_INDEX
}
} // anonymous namespace

Tensor& index_select_out_cuda(
    const Tensor& self,
    int64_t dim,
    const Tensor& index,
    Tensor& out) {
  static constexpr std::string_view DIM_WARNING =
      "Tensor too large or too many (> 25) dimensions";
  TORCH_CHECK(
      at::cuda::check_device({out, self, index}),
      "Input, output and indices must be on the current device");
  at::assert_no_internal_overlap(out);
  at::assert_no_overlap(out, self);
  at::assert_no_overlap(out, index);

  dim = at::maybe_wrap_dim(dim, self);
  TORCH_CHECK(self.dim() <= MAX_TENSORINFO_DIMS, DIM_WARNING);
  TORCH_CHECK(index.dim() <= MAX_TENSORINFO_DIMS, DIM_WARNING);
  if (self.is_quantized()) {
    TORCH_CHECK(
        self.qscheme() == kPerTensorAffine,
        "Only per_tensor quantized quantized tensors are supported by index_select.")
    AT_DISPATCH_QINT_TYPES(out.scalar_type(), "index_select_quant_cuda", [&] {
      index_select_out_cuda_impl<scalar_t>(out, self, dim, index);
    });
  } else {
    AT_DISPATCH_V2(
        out.scalar_type(),
        "index_select_cuda",
        AT_WRAP([&] {
          index_select_out_cuda_impl<scalar_t>(out, self, dim, index);
        }),
        AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX),
        AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES),
        AT_EXPAND(AT_FLOAT8_TYPES),
        kComplexHalf,
        kHalf,
        kBool,
        kBFloat16);
  }

  return out;
}

Tensor index_select_cuda(const Tensor& self, int64_t dim, const Tensor& index) {
  Tensor out = at::empty({0}, self.options());
  at::native::index_select_out_cuda(self, dim, index, out);
  return out;
}

Tensor index_select_quantized_cuda(const Tensor& self, int64_t dim, const Tensor& index) {
  TORCH_CHECK(
    self.qscheme() == kPerTensorAffine,
    "Only per_tensor quantized quantized tensors are supported by index_select.")
  Tensor out = at::empty_quantized({0}, self);
  at::native::index_select_out_cuda(self, dim, index, out);
  return out;
}

namespace {

void masked_fill_kernel(TensorIterator& iter, const Scalar& value) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND4(
      kBool, kHalf, kBFloat16, kComplexHalf, iter.common_dtype(), "masked_fill_", [&]() {
        const auto value_ = value.to<scalar_t>();
        gpu_kernel(
            iter, [value_] GPU_LAMBDA(scalar_t self, bool mask) -> scalar_t {
              if (mask) {
                return value_;
              }
              return self;
            });
      });
}

template <typename scalar_t>
void cuda_masked_fill_kernel_quantized(TensorIterator& iter, scalar_t quantized_val) {
    gpu_kernel(
        iter, [quantized_val] GPU_LAMBDA(scalar_t self, bool mask) -> scalar_t {
          if (mask) {
            return quantized_val;
          }
          return self;
    });
}

void masked_fill_kernel_quantized(TensorIterator& iter, const Scalar& value, double scale, int zero_point) {
  TORCH_CHECK(iter.input_dtype(1) == at::ScalarType::Bool, "masked_fill only supports boolean masks, ",
    "but got dtype ", iter.input_dtype(1));
  AT_DISPATCH_QINT_TYPES(
      iter.common_dtype(), "masked_fill_", [&]() {
        float float_val = value.to<float>();
        const auto quantized_val = quantize_val<scalar_t>(scale, zero_point, float_val);

        cuda_masked_fill_kernel_quantized<scalar_t>(iter, quantized_val);
    });
}

REGISTER_CUDA_DISPATCH(masked_fill_kernel_quantized_stub, &masked_fill_kernel_quantized)

} // anonymous namespace

Tensor & masked_fill__cuda(Tensor& self, const Tensor & mask, const Scalar& value) {
  TORCH_CHECK(self.device() == mask.device(), "expected self and mask to be on the same device, but got mask on ",
    mask.device(), " and self on ", self.device());
  TORCH_CHECK(mask.scalar_type() == kBool,
    "masked_fill only supports boolean masks, but got dtype ", mask.scalar_type());
  auto maybe_outnames = namedinference::broadcast_to_outnames(self, mask, "masked_fill_");
  if (at::has_internal_overlap(self) == MemOverlap::Yes) {
    TORCH_WARN(
      "Use of masked_fill_ on expanded tensors is deprecated. "
      "Please clone() the tensor before performing this operation. "
      "This also applies to advanced indexing e.g. tensor[mask] = scalar");
  }
  at::assert_no_partial_overlap(self, mask);

  c10::MaybeOwned<Tensor> b_mask = expand_inplace(self, mask, "masked_fill_");

  auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self)
      .add_const_input(self)
      .add_const_input(*b_mask)
      .build();

  masked_fill_kernel(iter, value);
  namedinference::propagate_names_if_nonempty(self, maybe_outnames);
  return self;
}

Tensor & masked_fill__cuda(Tensor& self, const Tensor & mask, const Tensor & value) {
  TORCH_CHECK(value.dim() == 0, "masked_fill_ only supports a 0-dimensional value tensor, but got tensor "
      "with ", value.dim(), " dimension(s).");
  // We hit this function if either of the input tensor lives on CUDA.
  // It is ok, if `value` is `CPU` tensor but we should not allow `self` or
  // `mask` to be CPU tensor. Check for `self` and `mask` being on same device
  // exists in `masked_fill__cuda` (Scalar version).
  TORCH_CHECK(!self.device().is_cpu(), "masked_fill_: Expected inputs to be on same device")
  return masked_fill__cuda(self, mask, value.item());
}

namespace {

// ForwardIt: only legacy random access iterator is supported.
template<class ForwardIt, class T, bool is_lower = true>
static __host__ __device__ __forceinline__
ForwardIt find_bound(ForwardIt first, ForwardIt last, const T& value) {
    ForwardIt it;
    typename std::iterator_traits<ForwardIt>::difference_type count, step;
    // NOTE: std::distance(first, last) compiles but produces wrong results here,
    // so only legacy random access iterators are safe in this code.
    count = last - first;

    while (count > 0) {
      it = first;
      step = count / 2;
      // avoiding std::advance(it, step),
      // although it does work unlike std::distance
      it += step;
      if (is_lower ? *it < value : value >= *it) {
        first = ++it;
        count -= step + 1;
      }
      else {
        count = step;
      }
    }
    return first;
}

}

Tensor index_select_sparse_cuda(const Tensor& self, int64_t dim, const Tensor& index) {
  const auto ndim = self.dim();
  TORCH_CHECK_INDEX(ndim, "index_select() cannot be applied to a 0-dim tensor.");
  TORCH_CHECK_INDEX(
      index.dim() == 1 && index.dtype() == at::kLong && index.options().layout() == at::kStrided,
      "index_select() argument index must be 1-D strided (non-sparse) long-tensor.");
  dim = maybe_wrap_dim(dim, ndim);
  const auto size = self.size(dim);
  const auto sparse_dim = self.sparse_dim();
  const auto dense_dim = self.dense_dim();
  const auto indices = self._indices();
  const auto values = self._values();
  const auto nnz = values.size(0);
  const auto index_len = index.size(0);
  auto res_sizes = self.sizes().vec();
  res_sizes[dim] = index_len;

  // If indexing into sparse dimensions
  if (dim < sparse_dim) {
    const auto make_output = [
      dim, sparse_dim, dense_dim, res_sizes, &self, &indices, &values
    ](
        const Tensor& selected_dim_indices,
        const Tensor& res_dim_indices
    ) -> Tensor {
      auto res_indices = indices.index_select(1, selected_dim_indices);
      res_indices[dim] = res_dim_indices;
      const auto res_values = values.index_select(0, selected_dim_indices);

      return at::_sparse_coo_tensor_with_dims_and_tensors(
          sparse_dim, dense_dim, res_sizes, res_indices, res_values, self.options());
    };

    // short-circuit if index is empty
    if (!index_len) {
      return make_output(index, index);
    }

    const auto nneg_index = [&index, size]() -> Tensor {
      auto nneg_index = at::empty_like(index, at::MemoryFormat::Contiguous);

      auto iter = TensorIteratorConfig()
        .add_output(nneg_index)
        .add_input(index)
        .build();

      AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_select_sparse_cuda", [&]() {
          gpu_kernel(iter, [size] GPU_LAMBDA (index_t idx) -> index_t {
              CUDA_KERNEL_ASSERT(idx >= -size && idx < size
                  && "index_select(): index out of bounds");
              return idx < 0 ? idx + size : idx;
          });
      });
      return nneg_index;
    }();

    const auto dim_indices = indices[dim].contiguous();
    const auto idx_nneg_index = at::arange(index_len, nneg_index.options());
    const auto idx_dim_indices = at::arange(nnz, dim_indices.options());

    Tensor sorted_dim_indices, argsort_dim_indices;
    std::tie(sorted_dim_indices, argsort_dim_indices) = [&]() -> std::tuple<Tensor, Tensor> {
      if (dim == 0 && self.is_coalesced()) {
        return std::make_tuple(dim_indices, idx_dim_indices);
      }
      else {
        return dim_indices.sort();
      }
    }();

    Tensor intrsc_counts_nneg_index;
    Tensor intrsc_first_match_nneg_index;
    std::tie(intrsc_counts_nneg_index, intrsc_first_match_nneg_index) = [&]() -> std::tuple<Tensor, Tensor> {
      auto intrsc_counts_nneg_index = at::zeros_like(nneg_index);
      auto intrsc_first_match_nneg_index = at::zeros_like(nneg_index);

      auto iter = TensorIteratorConfig()
        .add_output(intrsc_first_match_nneg_index)
        .add_input(nneg_index)
        .add_input(idx_nneg_index)
        .build();

      AT_DISPATCH_INDEX_TYPES(nneg_index.scalar_type(), "index_select_sparse_cuda", [&]() {
          index_t* ptr_intrsc_counts_nneg_index = intrsc_counts_nneg_index.mutable_data_ptr<index_t>();
          const index_t* ptr_sorted_dim_indices = sorted_dim_indices.const_data_ptr<index_t>();
          gpu_kernel(
              iter,
              [ptr_intrsc_counts_nneg_index, ptr_sorted_dim_indices, nnz] GPU_LAMBDA (
                index_t idx_val, index_t idx_idx
              ) -> index_t {
                auto* lb = find_bound<const index_t*, index_t, true>(
                  ptr_sorted_dim_indices,
                  ptr_sorted_dim_indices + nnz,
                  idx_val
                );
                auto* ub = find_bound<const index_t*, index_t, false>(
                  ptr_sorted_dim_indices,
                  ptr_sorted_dim_indices + nnz,
                  idx_val
                );
                const auto idx_count = ub - lb;
                ptr_intrsc_counts_nneg_index[idx_idx] = idx_count;

                return lb - ptr_sorted_dim_indices;
              }
          );
      });

      return std::make_tuple(intrsc_counts_nneg_index, intrsc_first_match_nneg_index);
    }();

    // Unavoidable sync since the shape of the result is not known in advance
    auto res_len = intrsc_counts_nneg_index.sum().item<int64_t>();
    // Short-circuit if empty intersection
    if (!res_len) {
      auto empty_idx = at::empty({0}, nneg_index.options());
      return make_output(empty_idx, empty_idx);
    }

    auto [selected_dim_indices, res_dim_indices] = [&]() -> std::tuple<Tensor, Tensor> {
      auto res_dim_indices = at::empty({res_len}, nneg_index.options());
      auto selected_dim_indices = at::empty_like(res_dim_indices);
      auto selected_dim_indices_offsets = intrsc_counts_nneg_index.cumsum(0)
        .sub_(intrsc_counts_nneg_index);

      // Need to have output as TensorIterator does not allow having void lambdas.
      auto dummy_output = at::empty({1}, dim_indices.options()).expand(IntArrayRef({index_len}));
      auto iter = TensorIteratorConfig()
        .add_output(dummy_output)
        // All iterations map to a single element in dummy_output by design,
        // hence removed output memory overlap check.
        .set_check_mem_overlap(false)
        .add_input(idx_nneg_index)
        .add_input(intrsc_counts_nneg_index)
        .add_input(selected_dim_indices_offsets)
        .add_input(intrsc_first_match_nneg_index)
        .build();

      AT_DISPATCH_INDEX_TYPES(nneg_index.scalar_type(), "index_select_sparse_cuda", [&]() {
          index_t* ptr_res_dim_indices = res_dim_indices.mutable_data_ptr<index_t>();
          index_t* ptr_selected_dim_indices = selected_dim_indices.mutable_data_ptr<index_t>();
          const index_t* ptr_argsort_dim_indices = argsort_dim_indices.const_data_ptr<index_t>();
          gpu_kernel(
              iter,
              [ptr_res_dim_indices, ptr_selected_dim_indices, ptr_argsort_dim_indices] GPU_LAMBDA (
                index_t idx_idx, index_t count, index_t offset, index_t first_match
              ) -> index_t {
                index_t* __restrict__ ptr_res_dim_indices_out = ptr_res_dim_indices + offset;
                const index_t* __restrict__ ptr_argsort_dim_indices_in = ptr_argsort_dim_indices + first_match;
                index_t* __restrict__ ptr_selected_dim_indices_out = ptr_selected_dim_indices + offset;
                for (index_t i = 0; i < count; ++i) {
                  *ptr_res_dim_indices_out++ = idx_idx;
                  *ptr_selected_dim_indices_out++ = *ptr_argsort_dim_indices_in++;
                }

                // A dummy return scalar for a dummy output
                return static_cast<index_t>(1);
              }
          );
      });

      return std::make_tuple(selected_dim_indices, res_dim_indices);
    }();

    return make_output(selected_dim_indices, res_dim_indices);
  }
  // If indexing into dense dimensions
  else {
    // It is sufficient to just perform `index_select` on values
    // if `dim` refers to dense dimensions.
    const auto res_values = values.index_select(dim - sparse_dim + 1, index);

    return _sparse_coo_tensor_with_dims_and_tensors(
        sparse_dim, dense_dim, res_sizes, indices, res_values, self.options());
  }
}


} // at::native
