#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/cuda/LaunchUtils.h>
#include <ATen/native/cuda/UpSample.cuh>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_upsample_nearest_exact2d_backward_native.h>
#include <ATen/ops/_upsample_nearest_exact2d_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/upsample_nearest2d_backward_native.h>
#include <ATen/ops/upsample_nearest2d_native.h>
#endif

namespace at::native {
namespace {

#define MAX_THREADS 512

// Define a typedef to dispatch to nearest_neighbor_compute_source_index or
// nearest_neighbor_exact_compute_source_index
typedef int (*nn_compute_source_index_fn_t)(const float, int, int);

// Define a typedef to dispatch to nearest_neighbor_bw_compute_source_index or
// nearest_neighbor_exact_bw_compute_source_index
typedef int (*nn_bw_compute_source_index_fn_t)(const float, int, int);

// see NOTE [ Nearest neighbor upsampling kernel implementation ]
template <typename scalar_t, nn_compute_source_index_fn_t nn_compute_source_index_fn>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest2d_out_frame(
    const scalar_t* idata,
    scalar_t* odata,
    const size_t nc,
    const size_t height1,
    const size_t width1,
    const size_t height2,
    const size_t width2,
    float height_scale,
    float width_scale) {
  size_t nc_iter = threadIdx.z + blockIdx.z * blockDim.z;
  int64_t w2 = ((int64_t) threadIdx.x) + blockIdx.x * blockDim.x;
  int64_t h2 = threadIdx.y + blockIdx.y * blockDim.y;

  if (w2 >= width2 || h2 >= height2) {
    return;
  }

  int64_t nc_stride = ((int64_t) blockDim.z) * gridDim.z;

  const size_t h1 = height1 == height2
      ? h2
      : nn_compute_source_index_fn(height_scale, h2, height1);
  const size_t w1 = width1 == width2
      ? w2
      : nn_compute_source_index_fn(width_scale, w2, width1);

  size_t src_index = (nc_iter * height1 + h1) * width1 + w1;
  size_t src_index_stride = nc_stride * width1 * height1;
  size_t dst_index = (nc_iter * height2 + h2) * width2 + w2;
  size_t dst_index_stride = nc_stride * width2 * height2;

  // iterating over
  while (nc_iter < nc) {
    odata[dst_index] = idata[src_index];
    dst_index += dst_index_stride;
    src_index += src_index_stride;
    nc_iter += nc_stride;
  }
}

template <typename scalar_t, nn_compute_source_index_fn_t nn_compute_source_index_fn>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest2d_nhwc_out_frame(
    const scalar_t* idata,
    scalar_t* odata,
    const size_t channels,
    const size_t height1,
    const size_t width1,
    const size_t height2,
    const size_t width2,
    float height_scale,
    float width_scale,
    const size_t out_numel) {

    const int64_t index = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;

    if (index < out_numel) {
    const auto c = index % channels;
    const auto w2 = (index / channels) % width2;
    const auto h2 = (index / channels / width2) % height2;
    const auto n = index / channels / width2 / height2;

    const size_t h1 = height1 == height2 ? h2 : nn_compute_source_index_fn(height_scale, h2, height1);
    const size_t w1 = width1 == width2 ? w2 : nn_compute_source_index_fn(width_scale, w2, width1);

    odata[index] = idata[idx_cl(n, h1, w1, c, height1, width1, channels)];
  }
}

// see NOTE [ Nearest neighbor upsampling kernel implementation ]
template <typename scalar_t, typename accscalar_t, nn_bw_compute_source_index_fn_t nn_bw_compute_source_index_fn>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest2d_backward_out_frame(
    const scalar_t* grad_o,
    size_t dim_b,
    size_t dim_c,
    size_t src_dim_h,
    size_t src_dim_w,
    size_t dst_dim_h,
    size_t dst_dim_w,
    scalar_t* grad_i,
    float height_scale,
    float width_scale) {
  int64_t dst_idx = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  if (dst_idx >= dim_c * dst_dim_h * dst_dim_w)
    return;

  int64_t dst_c_stride = dst_dim_h * dst_dim_w;
  int64_t src_c_stride = src_dim_h * src_dim_w;

  int c = (dst_idx / (dst_c_stride)) % dim_c;

  int dst_y = (dst_idx / dst_dim_w) % dst_dim_h;
  // note that we do not want to clamp src_y to src_dim_y, since we might
  // intentionally want to skip in case of scale_factor < 1.0
  int src_y =
      nn_bw_compute_source_index_fn(height_scale, dst_y, src_dim_h);
  int src_y_up = nn_bw_compute_source_index_fn(
      height_scale, dst_y + 1, src_dim_h);

  int dst_x = dst_idx % dst_dim_w;
  // note that we do not want to clamp src_x to src_dim_w, since we might
  // intentionally want to skip in case of scale_factor < 1.0
  int src_x =
      nn_bw_compute_source_index_fn(width_scale, dst_x, src_dim_w);
  int src_x_up = nn_bw_compute_source_index_fn(
      width_scale, dst_x + 1, src_dim_w);

  for (int b = 0; b < dim_b; b++) {
    accscalar_t grad = 0;
    for (int y = src_y; y < src_y_up; y++) {
      for (int x = src_x; x < src_x_up; x++) {
        int64_t src_idx =
            b * dim_c * src_c_stride + c * src_c_stride + y * src_dim_w + x;
        grad += grad_o[src_idx];
      }
    }
    grad_i[dst_idx] = grad;
    dst_idx += dim_c * dst_c_stride;
  }
}

template <typename scalar_t, typename accscalar_t, nn_bw_compute_source_index_fn_t nn_bw_compute_source_index_fn>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest2d_backward_nhwc_out_frame(
    const scalar_t* go,
    scalar_t* gi,
    const size_t height1,
    const size_t width1,
    const size_t height2,
    const size_t width2,
    const size_t channels,
    const float height_scale,
    const float width_scale,
    const size_t gi_numel) {

  // 1 is for grad_output (src)
  // 2 is for grad_input (dst)

  const int64_t index = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;

  if (index < gi_numel) {
    const int c = index % channels;
    const int w2 = (index / channels) % width2;
    const int h2 = (index / channels / width2) % height2;
    const int n = index / channels / width2 / height2;

    int h1 = nn_bw_compute_source_index_fn(height_scale, h2, height1);
    int h1_up = nn_bw_compute_source_index_fn(height_scale, h2 + 1, height1);

    int w1 = nn_bw_compute_source_index_fn(width_scale, w2, width1);
    int w1_up = nn_bw_compute_source_index_fn(width_scale, w2 + 1, width1);

    accscalar_t grad = 0;
    for (int ih = h1; ih < h1_up; ih++) {
      for (int iw = w1; iw < w1_up; iw++) {
        grad += go[idx_cl(n, ih, iw, c, height1, width1, channels)];
      }
    }
    gi[index] = static_cast<scalar_t>(grad);
  }
}

template<nn_compute_source_index_fn_t nn_compute_source_index_fn>
static void upsample_nearest2d_out_cuda_template(
    const Tensor& output,
    const Tensor& input_,
    IntArrayRef output_size,
    std::optional<double> scales_h,
    std::optional<double> scales_w) {
  TensorArg input_arg{input_, "input_", 1}, output_arg{output, "output", 2};
  checkAllSameGPU(__func__, {input_arg, output_arg});

  if (input_.numel() == 0) {
    return;
  }

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input_.size(0);
  int channels = input_.size(1);
  int input_height = input_.size(2);
  int input_width = input_.size(3);

  const float height_scale = compute_scales_value<float>(scales_h, input_height, output_height);
  const float width_scale = compute_scales_value<float>(scales_w, input_width, output_width);

  const auto memory_format = input_.suggest_memory_format();

  if (input_.sizes() == output.sizes()) {
    output.copy_(input_);
    return;
  }

  // heuristic: only use channels_last path when it's faster than the contiguous path
  if (memory_format == at::MemoryFormat::ChannelsLast && channels >= 4 && \
        output.is_contiguous(memory_format)) {
    at::Tensor input = input_.contiguous(at::MemoryFormat::ChannelsLast);

    TORCH_CHECK(input.numel() < std::numeric_limits<int64_t>::max(),
      "upsample_nearest_nhwc only supports input tensors with less than 2^63 - 1 elements, but got ", input.sizes());
    TORCH_CHECK(output.numel() < std::numeric_limits<int64_t>::max(),
      "upsample_nearest_nhwc only supports output tensors with less than 2^63 - 1 elements, but got ", output.sizes());

    const int64_t num_kernels = output.numel();
    const int64_t num_threads = std::min(at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);

    AT_DISPATCH_FLOATING_TYPES_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Byte, input.scalar_type(), "upsample_nearest2d_nhwc_out_frame", [&] {
      const scalar_t* idata = input.const_data_ptr<scalar_t>();
      scalar_t* odata = output.mutable_data_ptr<scalar_t>();
      upsample_nearest2d_nhwc_out_frame<scalar_t, nn_compute_source_index_fn>
        <<<ceil_div(num_kernels, num_threads), num_threads, 0, at::cuda::getCurrentCUDAStream()>>>(
          idata,
          odata,
          channels,
          input_height,
          input_width,
          output_height,
          output_width,
          height_scale,
          width_scale,
          output.numel()
      );
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  }
  else {
    // This is needed for non-contiguous tensors.
    Tensor output_c = output.is_contiguous() ? output : at::empty(output.sizes(), output.options());
    Tensor input = input_.contiguous();

    int64_t nc = nbatch * channels;

    const int max_threads = std::min<int>(
        at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS);

    int* maxThreadsDim = at::cuda::getCurrentDeviceProperties()->maxThreadsDim;
    int* maxGridSize = at::cuda::getCurrentDeviceProperties()->maxGridSize;

    // upsample_nearest2d meta call makes sure input/output tensor is not empty;
    int block_x = std::min<int>(
        maxThreadsDim[0], std::min<int>(lastPow2(output_width), max_threads));
    int block_y = std::min<int>(
        maxThreadsDim[1],
        std::min<int>(lastPow2(output_height), max_threads / block_x));
    int block_z = std::min<int>(
        maxThreadsDim[2], std::min<int>(nc, max_threads / block_x / block_y));
    const dim3 block(block_x, block_y, block_z);

    int grid_x = ceil_div(output_width, block_x);
    int grid_y = ceil_div(output_height, block_y);
    int grid_z = std::min<int>(
        maxGridSize[2], ceil_div(nc, (int64_t) block_z * 4));
    const dim3 grid(grid_x, grid_y, grid_z);
    // Error out on cases where grid_x & grid_y exceeds limit of launch config, as
    // the current kernel implementation doesn't loop over the two dimensions.
    // This is unlikely to happen.
    // TODO: kernel implementation could stride on spatial dimension. We probably
    //       need to overhaul the kernel.
    TORCH_CHECK(
        grid_x <= maxGridSize[0] && grid_y <= maxGridSize[1],
        "input tensor has spatial dimension larger than the kernel capacity");
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Byte, input.scalar_type(), "upsample_nearest2d_out_frame", [&] {
          using accscalar_t = at::acc_type<scalar_t, true>;

          auto idata = input.const_data_ptr<scalar_t>();
          auto odata = output_c.mutable_data_ptr<scalar_t>();

          upsample_nearest2d_out_frame<scalar_t, nn_compute_source_index_fn>
              <<<grid, block, 0, stream>>>(
                  idata,
                  odata,
                  nc,
                  input_height,
                  input_width,
                  output_height,
                  output_width,
                  height_scale,
                  width_scale);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });

    if (!output.is_contiguous()) {
        output.copy_(output_c);
    }
  }
}

template<nn_bw_compute_source_index_fn_t nn_bw_compute_source_index_fn>
static void upsample_nearest2d_backward_out_cuda_template(
    const Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    std::optional<double> scales_h,
    std::optional<double> scales_w) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(__func__, {grad_output_arg, grad_input_arg});

  if (grad_input.numel() == 0) {
    return;
  }

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_height = input_size[2];
  int input_width = input_size[3];

  const float height_scale = compute_scales_value_backwards<float>(scales_h, output_height, input_height);
  const float width_scale = compute_scales_value_backwards<float>(scales_w, output_width, input_width);

  auto memory_format = grad_output_.suggest_memory_format();

  if (grad_output_.sizes() == grad_input.sizes()) {
    grad_input.copy_(grad_output_);
    return;
  }

  if (memory_format == at::MemoryFormat::ChannelsLast && channels >= 4 && \
        grad_input.is_contiguous(memory_format)) {
    Tensor grad_output = grad_output_.contiguous(at::MemoryFormat::ChannelsLast);

    TORCH_CHECK(grad_input.numel() < std::numeric_limits<int>::max(),
      "upsample_nearest_nhwc only supports grad_input tensors with less than INT_MAX elements, but got ", grad_input.sizes());
    TORCH_CHECK(grad_output.numel() < std::numeric_limits<int>::max(),
      "upsample_nearest_nhwc only supports grad_output tensors with less than INT_MAX elements, but got ", grad_output.sizes());

    const int num_kernels = grad_input.numel();
    const int num_threads = std::min(at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);

    AT_DISPATCH_FLOATING_TYPES_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Byte, grad_output.scalar_type(), "upsample_nearest2d_backward_nhwc_out_frame", [&] {
      using accscalar_t = at::acc_type<scalar_t, true>;

      const scalar_t* go = grad_output.const_data_ptr<scalar_t>();
      scalar_t* gi = grad_input.mutable_data_ptr<scalar_t>();

      upsample_nearest2d_backward_nhwc_out_frame<scalar_t, accscalar_t, nn_bw_compute_source_index_fn>
        <<<ceil_div(num_kernels, num_threads), num_threads, 0, at::cuda::getCurrentCUDAStream()>>>(
          go,
          gi,
          output_height,
          output_width,
          input_height,
          input_width,
          channels,
          height_scale,
          width_scale,
          grad_input.numel()
      );
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  } else {
    // This is needed for non-contiguous tensors.
    Tensor grad_input_c = grad_input.is_contiguous() ? grad_input : at::empty(grad_input.sizes(), grad_input.options());
    Tensor grad_output = grad_output_.contiguous();

    // upsample_nearest2d meta call makes sure `nbatch != 0`
    size_t n = grad_input.numel() / nbatch;
    dim3 bdim{std::min<unsigned int>(
        at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS)};
    dim3 gdim{(unsigned int) ceil_div(n, (size_t) bdim.x)};
    // safe check for int64 indexing; implicitly restrict launch config for kernel
    TORCH_CHECK(grad_input.numel() <= std::numeric_limits<int64_t>::max(), "upsample2d grad_input.numel() <= std::numeric_limits<int64_t>::max(), but got ", grad_input.sizes());
    TORCH_CHECK(grad_output.numel() <= std::numeric_limits<int64_t>::max(), "upsample2d grad_output.numel() <= std::numeric_limits<int64_t>::max(), but got ", grad_output.sizes());

    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Byte, grad_output.scalar_type(), "upsample_nearest2d_backward_out_frame", [&] {
      using accscalar_t = at::acc_type<scalar_t, true>;

      auto idata = grad_input_c.mutable_data_ptr<scalar_t>();
      auto odata = grad_output.const_data_ptr<scalar_t>();


      upsample_nearest2d_backward_out_frame<scalar_t, accscalar_t, nn_bw_compute_source_index_fn>
          <<<gdim, bdim, 0, stream>>>(
              odata,
              nbatch,
              channels,
              output_height,
              output_width,
              input_height,
              input_width,
              idata,
              height_scale,
              width_scale);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });

    if (!grad_input.is_contiguous()) {
        grad_input.copy_(grad_input_c);
    }
  }
}

} // namespace

TORCH_IMPL_FUNC(upsample_nearest2d_out_cuda) (
    const Tensor& input,
    IntArrayRef output_size,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& output) {
  upsample_nearest2d_out_cuda_template<nearest_neighbor_compute_source_index>(
      output, input, output_size, scales_h, scales_w);
}

TORCH_IMPL_FUNC(_upsample_nearest_exact2d_out_cuda) (
    const Tensor& input,
    IntArrayRef output_size,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& output) {
  upsample_nearest2d_out_cuda_template<nearest_neighbor_exact_compute_source_index>(
      output, input, output_size, scales_h, scales_w);
}

TORCH_IMPL_FUNC(upsample_nearest2d_backward_out_cuda) (
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& grad_input) {
  upsample_nearest2d_backward_out_cuda_template<nearest_neighbor_bw_compute_source_index>(
      grad_input, grad_output, output_size, input_size, scales_h, scales_w);
}

TORCH_IMPL_FUNC(_upsample_nearest_exact2d_backward_out_cuda) (
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& grad_input) {
  upsample_nearest2d_backward_out_cuda_template<nearest_neighbor_exact_bw_compute_source_index>(
      grad_input, grad_output, output_size, input_size, scales_h, scales_w);
}

} // namespace at::native
