#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/Lerp.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/OpMathType.h>

namespace at::native {
namespace {

void lerp_scalar_kernel(
    at::TensorIteratorBase& iter,
    const c10::Scalar& weight);

constexpr char lerp_tensor_name[] = "lerp_tensor";
void lerp_tensor_kernel(at::TensorIteratorBase& iter) {
  auto dtype = iter.common_dtype();
  if(at::isComplexType(dtype)) {
#if AT_USE_JITERATOR()
  static const auto lerp_tensor_string = jiterator_stringify(
      template <typename T>
      T lerp_tensor(T self_val, T end_val, T weight_val) {
        return (std::abs(weight_val) < 0.5)
            ? self_val + weight_val * (end_val - self_val)
            : end_val -
                (end_val - self_val) * (static_cast<T>(1) - weight_val);
      }
  ); // lerp_tensor_string
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "lerp_cuda", [&] {
      if (iter.is_cpu_scalar(3)) {
        auto weight_val = iter.scalar_value<scalar_t>(3);
        iter.remove_operand(3);
        return lerp_scalar_kernel(iter, weight_val);
      }

      jitted_gpu_kernel<
        /*name=*/ lerp_tensor_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 3>(iter, lerp_tensor_string);
    });
#else
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "lerp_cuda", [&] {
      using opmath_t = at::opmath_type<scalar_t>;
      if (iter.is_cpu_scalar(3)) {
        auto weight_val = iter.scalar_value<scalar_t>(3);
        iter.remove_operand(3);
        return lerp_scalar_kernel(iter, weight_val);
      }

      at::native::gpu_kernel(
        iter,
        [] GPU_LAMBDA(
            scalar_t self_val,
            scalar_t end_val,
            scalar_t weight_val) -> scalar_t {
           opmath_t self_val_f = self_val;
           opmath_t end_val_f = end_val;
           opmath_t weight_val_f = weight_val;
          return lerp(self_val, end_val, weight_val);
        });
      });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      dtype, "lerp_cuda",
      [&] {
        if (iter.is_cpu_scalar(3)) {
          auto weight_val = iter.scalar_value<scalar_t>(3);
          iter.remove_operand(3);
          return lerp_scalar_kernel(iter, weight_val);
        }

        at::native::gpu_kernel(
          iter,
          [] GPU_LAMBDA(
              scalar_t self_val,
              scalar_t end_val,
              scalar_t weight_val) -> scalar_t {
            return lerp(self_val, end_val, weight_val);
          });
      });
  }
}

constexpr char lerp_scalar_name[] = "lerp_scalar";
void lerp_scalar_kernel(at::TensorIteratorBase& iter, const c10::Scalar& weight) {
  auto dtype = iter.common_dtype();
  if (at::isComplexType(dtype)) {
#if AT_USE_JITERATOR()
  static const auto lerp_scalar_string = jiterator_stringify(
      template <typename T>
      T lerp_scalar(T self_val, T end_val, T weight_val) {
        return (std::abs(weight_val) < 0.5)
            ? self_val + weight_val * (end_val - self_val)
            : end_val -
                (end_val - self_val) * (static_cast<T>(1) - weight_val);
      }
  ); // lerp_scalar_string
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "lerp_cuda", [&] {
      using opmath_t = at::opmath_type<scalar_t>;
      auto weight_val = weight.to<opmath_t>();
      jitted_gpu_kernel<
        /*name=*/ lerp_scalar_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 2>(
        iter,
        lerp_scalar_string,
        /*scalar_pos=*/ at::cuda::jit::BinaryFuncVariant::NoScalar,
        /*scalar_val=*/ 0,
        /*extra_args=*/ std::make_tuple(weight_val));
  });
#else
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "lerp_cuda", [&] {
    using opmath_t = at::opmath_type<scalar_t>;
    auto weight_val = weight.to<opmath_t>();
    at::native::gpu_kernel(
        iter,
        [=] GPU_LAMBDA(scalar_t self_val, scalar_t end_val) {
          opmath_t self_val_f = self_val;
          opmath_t end_val_f = end_val;
          return lerp(self_val, end_val, weight_val);
        });
  });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      dtype, "lerp_cuda",
      [&]{
        using opmath_t = at::opmath_type<scalar_t>;
        auto weight_val = weight.to<opmath_t>();
        at::native::gpu_kernel(
            iter, [=] GPU_LAMBDA(scalar_t self_val, scalar_t end_val) {
              return lerp(self_val, end_val, weight_val);
            });
      });
    }
}

} // anonymous namespace

REGISTER_DISPATCH(lerp_kernel_tensor_weight, &lerp_tensor_kernel)
REGISTER_DISPATCH(lerp_kernel_scalar_weight, &lerp_scalar_kernel)

} // namespace at::native
