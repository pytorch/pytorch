#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/core/TensorBase.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <ATen/cuda/ApplyGridUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/Loops.cuh>

namespace at::native {
namespace {

void hardswish_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_cuda", [&]() {
    using opmath_t = at::opmath_type<scalar_t>;
    const opmath_t zero(0.0f);
    const opmath_t one_sixth(1.0f / 6.0f);
    const opmath_t three(3.0f);
    const opmath_t six(6.0f);
    gpu_kernel(iter, [zero, one_sixth, three, six]GPU_LAMBDA(scalar_t self_val) -> scalar_t {
      opmath_t x = static_cast<opmath_t>(self_val);
      return x * std::min(std::max(x + three, zero), six) * one_sixth;
    });
  });
}

void hardswish_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_backward_cuda", [&]() {
    using opmath_t = at::opmath_type<scalar_t>;
    const opmath_t zero(0.0f);
    const opmath_t three(3.0f);
    const opmath_t neg_three(-3.0f);
    const opmath_t one_half(0.5f);
    gpu_kernel(
      iter,
      [zero, three, neg_three, one_half]GPU_LAMBDA(scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
        opmath_t grad_val = static_cast<opmath_t>(grad_val_);
        opmath_t self_val = static_cast<opmath_t>(self_val_);
        if (self_val <= neg_three) {
          return zero;
        } else if (self_val < three) {
          return grad_val * ((self_val / three) + one_half);
        } else {
          return grad_val;
        }
    });
  });
}
} // namespace

REGISTER_DISPATCH(hardswish_stub, &hardswish_kernel)
REGISTER_DISPATCH(hardswish_backward_stub, &hardswish_backward_kernel)

} // namespace at::native
