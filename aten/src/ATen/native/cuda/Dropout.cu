#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>
#include <ATen/Dispatch.h>
#include <ATen/Utils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <c10/macros/Macros.h>
#include <hiprand/hiprand_kernel.h>

#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_masked_scale_native.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/native_dropout_backward_native.h>
#include <ATen/ops/ones_like.h>
#include <ATen/ops/zeros_like.h>
#endif

namespace at::native {

namespace {

// philox generates 128 bits of randomness at a time. Kernel uses this explicitly by putting suitably transformed result into float4
// for all members of float4 to be consumed UNROLL has to be 4. Don't change!
// Note: VEC <= 4 (and in most real-world cases will be 4), so same logic applies.
const int UNROLL = 4;

template <
    typename scalar_t,
    typename accscalar_t,
    typename IndexType,
    int ADims,
    int VEC,
    typename mask_t>
#if __CUDA_ARCH__ >= 350 || defined(USE_ROCM)
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void
fused_dropout_kernel_vec(at::cuda::detail::TensorInfo<const scalar_t, IndexType> a,
                         at::cuda::detail::TensorInfo<scalar_t, IndexType> b,
                         at::cuda::detail::TensorInfo<mask_t, IndexType> c,
                         IndexType totalElements, accscalar_t p,
                         PhiloxCudaState philox_args) {
  using LoadT = memory::aligned_vector<scalar_t, VEC>;
  using MaskLoadT = memory::aligned_vector<mask_t, VEC>;

  auto [seed, offset] = at::cuda::philox::unpack(philox_args);
  IndexType idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);

  // Helps align the total number of times hiprand_uniform4 is called by each thread for the same totalElements
  // in the vec=2 and vec=4 cases.
  bool gridxvec_loop_state = 0;
  accscalar_t scale = 1.0 / p;

  constexpr int RAND_SIZE = (VEC + 4 - 1) / 4;
  float4 rand[RAND_SIZE];

  // Note: Vectorized loads means we'll stride each thread by an additional VEC factor, as we'll load VEC elements at a time
  for (IndexType linearIndex = idx * VEC;
      linearIndex < totalElements;
      linearIndex += gridDim.x * blockDim.x * VEC) {
    // local storage
    scalar_t src[VEC];
    // We'll use this to actually cause vectorized loads later
    LoadT *value = reinterpret_cast<LoadT*>(&src);

    //hiprand_uniform_double was pure evil anyway, not doing what it promises, and there's nothing for halfs, so generate float for everything
    // Note: need a new set of random values per 4 elements -- we'll handle VEC elements in this thread, so need ceil(VEC / 4)
    // sets of rand.
    if ((VEC >= 4) || (gridxvec_loop_state == 0)) {
      #pragma unroll
      for (int ii = 0; ii < RAND_SIZE; ii++) {
        rand[ii] = hiprand_uniform4(&state);
      }
    } else {
      // sets up the last two values we generated last iteration to be used this iteration.
      rand[0].x = rand[0].z;
      rand[0].y = rand[0].w;
      gridxvec_loop_state ^= 1;
    }

    rand[0].x = rand[0].x < p;
    rand[0].y = rand[0].y < p;
    if constexpr (VEC >= 4) {
      rand[0].z = rand[0].z < p;
      rand[0].w = rand[0].w < p;
    }

    #pragma unroll
    for (int ii = 1; ii < RAND_SIZE; ii++) {
      rand[ii].x = rand[ii].x < p;
      rand[ii].y = rand[ii].y < p;
      rand[ii].z = rand[ii].z < p;
      rand[ii].w = rand[ii].w < p;
    }

    // Note: We explicitly check for is_contiguous() before launching the vectorized kernel
    // and replace IndexToOffset call with linearIndex to allow vectorization of NHWC (or other)
    // ordering.
    // Single vectorized load
    *value = *reinterpret_cast<const LoadT*>(&a.data[linearIndex]);

    scalar_t r[VEC];
    mask_t mask[VEC];

    // Perform the actual computation
    #pragma unroll
    for (int jj = 0; jj < RAND_SIZE; jj++) {
      #pragma unroll
      for (int ii = 0; ii < std::min(VEC, 4); ii++) {
        r[jj * 4 + ii] = src[jj * 4 + ii]*(&rand[jj].x)[ii]*scale;
        mask[jj * 4 + ii] = (mask_t)(&rand[jj].x)[ii];
      }
    }

    // Vectorized writes for both mask & result
    *(reinterpret_cast<LoadT*>(&b.data[linearIndex])) = *reinterpret_cast<LoadT*>(&r[0]);
    *(reinterpret_cast<MaskLoadT*>(&c.data[linearIndex])) = *reinterpret_cast<MaskLoadT*>(&mask[0]);

    __syncthreads();
  }
}

template <
    typename scalar_t,
    typename accscalar_t,
    typename IndexType,
    int ADims,
    int BDims = ADims,
    typename mask_t>
#if __CUDA_ARCH__ >= 350 || defined(USE_ROCM)
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void
fused_dropout_kernel(cuda::detail::TensorInfo<const scalar_t, IndexType> a,
                     cuda::detail::TensorInfo<scalar_t, IndexType> b,
                     cuda::detail::TensorInfo<mask_t, IndexType> c,
                     IndexType totalElements, accscalar_t p,
                     PhiloxCudaState philox_args) {
  auto [seed, offset] = at::cuda::philox::unpack(philox_args);
  IndexType idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);
  accscalar_t scale = 1.0 / p;

  IndexType rounded_size = ((totalElements - 1)/(blockDim.x * gridDim.x * UNROLL)+1) *
        blockDim.x * gridDim.x * UNROLL;
  for (IndexType linearIndex = idx;
       linearIndex < rounded_size;
       linearIndex += gridDim.x * blockDim.x*UNROLL) {
//hiprand_uniform_double was pure evil anyway, not doing what it promises, and there's nothing for halfs, so generate float for everything
       float4 rand = hiprand_uniform4(&state);
       scalar_t src[UNROLL];
       rand.x = rand.x < p;
       rand.y = rand.y < p;
       rand.z = rand.z < p;
       rand.w = rand.w < p;
       for (int ii = 0; ii < UNROLL; ii++) {
           IndexType li = linearIndex + blockDim.x * gridDim.x * ii;
           if (li < totalElements) {
    // Convert `linearIndex` into an offset of `a`
               const IndexType aOffset =
                   cuda::detail::IndexToOffset<const scalar_t, IndexType, ADims>::get(li, a);
               src[ii] = a.data[aOffset];
           }
       }
       for (int ii = 0; ii < UNROLL; ii++) {
           IndexType li = linearIndex + blockDim.x * gridDim.x * ii;
           if (li < totalElements) {
    // Convert `linearIndex` into an offset of `b`
               const IndexType bOffset =
                   cuda::detail::IndexToOffset<scalar_t, IndexType, BDims>::get(li, b);
               b.data[bOffset] = src[ii]*(&rand.x)[ii]*scale;
               c.data[bOffset] = (mask_t)(&rand.x)[ii];
           }
       }
       __syncthreads();
  }
}

template<typename mask_t, typename scalar_t, typename accscalar_t>
void masked_scale_kernel(at::Tensor& ret, const at::Tensor& src, const at::Tensor& mask, accscalar_t scale){
   auto iter = at::TensorIteratorConfig()
     .check_all_same_dtype(false)
     .add_output(ret)
     .add_const_input(src)
     .add_const_input(mask)
     .build();

   at::native::gpu_kernel(
       iter,
       [=]GPU_LAMBDA(const scalar_t src_val, const mask_t mask_val) -> scalar_t {
          return (float)mask_val * src_val * scale;
       });
}

template <typename scalar_t>
int get_vector_size(at::Tensor self, at::Tensor ret, at::Tensor mask) {
  int vec_size = 4;
  // get the vector size
  if (!self.is_non_overlapping_and_dense() || !ret.is_non_overlapping_and_dense() || !mask.is_non_overlapping_and_dense()) {
    vec_size = 1;
  } else {
    vec_size = memory::can_vectorize_up_to<scalar_t>((const char*)self.const_data_ptr());
#ifdef USE_ROCM
    // make sure we don't break assumption that we can't have > 16 elements / thread
    TORCH_INTERNAL_ASSERT(vec_size <= 16, "Value of VEC must be in [2, 4, 8, 16]");
#else
    const int optimal_vec_size = 16 / static_cast<int>(sizeof(scalar_t));
    vec_size = std::min<int>(optimal_vec_size, vec_size);

    // make sure we don't break assumption that we can't have > 4 elements / thread
    TORCH_INTERNAL_ASSERT(vec_size <= 8, "Value of VEC must be in [2, 4, 8]");
#endif
  }

  // check that we'd have no remainders - prefer a smaller vector size with no remainders over a larger vector and remainder.
  bool can_vectorize = true;
  do {
    can_vectorize = self.numel() % vec_size == 0 && ret.numel() % vec_size == 0 && mask.numel() % vec_size == 0;
    if (!can_vectorize) vec_size /= 2;
  } while (vec_size > 1 && !can_vectorize);
  return can_vectorize ? vec_size : 1;
}

template <typename index_type, typename mask_t>
inline void launcher(
    const Tensor& self,
    Tensor& ret,
    Tensor& mask,
    double p,
    const int64_t nelem,
    const PhiloxCudaState rng_engine_inputs,
    dim3 grid,
    dim3 dim_block) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      self.scalar_type(),
      "fused_dropout",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        accscalar_t pa = (accscalar_t)(p);
        auto self_info =
            cuda::detail::getTensorInfo<const scalar_t, index_type>(self);
        auto ret_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(ret);
        auto mask_info =
            cuda::detail::getTensorInfo<mask_t, index_type>(mask);
        self_info.collapseDims();
        ret_info.collapseDims();
        mask_info.collapseDims(); // ret and mask are collapsed to 1d
                                  // contiguous tensor

        int vec_size = get_vector_size<scalar_t>(self, ret, mask);

        if (vec_size > 1) {
          switch (vec_size) {
            case 16:
              fused_dropout_kernel_vec<
                  scalar_t,
                  accscalar_t,
                  index_type,
                  1,
                  16>
                  <<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                      self_info,
                      ret_info,
                      mask_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
            case 8:
              fused_dropout_kernel_vec<
                  scalar_t,
                  accscalar_t,
                  index_type,
                  1,
                  8>
                  <<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                      self_info,
                      ret_info,
                      mask_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
            case 4:
              fused_dropout_kernel_vec<
                  scalar_t,
                  accscalar_t,
                  index_type,
                  1,
                  4>
                  <<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                      self_info,
                      ret_info,
                      mask_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
            case 2:
              fused_dropout_kernel_vec<
                  scalar_t,
                  accscalar_t,
                  index_type,
                  1,
                  2>
                  <<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                      self_info,
                      ret_info,
                      mask_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
            default:
              TORCH_INTERNAL_ASSERT(false, "Unexpected vectorization size");
          }
        } else {
          switch (self_info.dims) {
            case 1:
              fused_dropout_kernel<scalar_t, accscalar_t, index_type, 1>
                  <<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                      self_info,
                      ret_info,
                      mask_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
            default:
              if (!self.is_contiguous() && ret.is_contiguous() &&
                  mask.is_contiguous()) {
                fused_dropout_kernel<scalar_t, accscalar_t, index_type, -1, 1>
                    <<<grid,
                        dim_block,
                        0,
                        at::cuda::getCurrentCUDAStream()>>>(
                        self_info,
                        ret_info,
                        mask_info,
                        nelem,
                        pa,
                        rng_engine_inputs);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              } else {
                fused_dropout_kernel<scalar_t, accscalar_t, index_type, -1>
                    <<<grid,
                        dim_block,
                        0,
                        at::cuda::getCurrentCUDAStream()>>>(
                        self_info,
                        ret_info,
                        mask_info,
                        nelem,
                        pa,
                        rng_engine_inputs);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
          }
        }
      });
}

} //anonymous namespace

template <typename mask_t>
std::tuple<Tensor,Tensor>
dropout_cuda(CUDAGeneratorImpl* gen, const Tensor& self, double p){
  Tensor mask = at::empty_like(self, self.options().dtype(c10::CppTypeToScalarType<mask_t>::value));
  const int64_t nelem = self.numel();
  // empty tensors should not get here, but just in case, avoid FPE
  // non-training shot-cut
  if (nelem==0) return std::tuple<Tensor,Tensor>(self.clone(), mask);

  Tensor ret = at::empty_like(self);
  const int64_t block_size = 256;
  unsigned int blocks_per_sm = at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor/block_size;
  dim3 dim_block(block_size);
  dim3 grid((nelem + block_size -1)/block_size);
  grid.x = std::min((unsigned int)at::cuda::getCurrentDeviceProperties()->multiProcessorCount * blocks_per_sm, grid.x);
//number of times random will be generated per thread, to offset philox counter in thc random state
  int64_t counter_offset = ((nelem - 1)/(block_size*grid.x*UNROLL)+1)*UNROLL;
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(counter_offset);
  }
  if (cuda::detail::canUse32BitIndexMath(self)){
    launcher<unsigned int, mask_t>(
        self, ret, mask, p, nelem, rng_engine_inputs, grid, dim_block);
  } else {
    launcher<uint64_t, mask_t>(
        self, ret, mask, p, nelem, rng_engine_inputs, grid, dim_block);
  }
  return std::tuple<Tensor,Tensor>(ret, mask);
}

std::tuple<Tensor,Tensor>
native_dropout_cuda(const Tensor& self, double p, std::optional<bool> train){
  // short-cut for train == false
  if (train.has_value() && !train.value()) {
    return std::make_tuple(self.clone(), at::ones_like(self, self.options().dtype(c10::CppTypeToScalarType<bool>::value)));
  }
  // short-cut
  if (p == 1) {
    // native_dropout_cuda is in derivatives.yaml, so we don't need to add data
    // dependency from output to input for autograd
    auto ret = at::zeros_like(self);
    auto mask = at::zeros_like(self, self.options().dtype(c10::CppTypeToScalarType<bool>::value));
    return std::tuple<Tensor,Tensor>(ret, mask);
  }

  auto gen = get_generator_or_default<CUDAGeneratorImpl>(std::nullopt, cuda::detail::getDefaultCUDAGenerator());
  double p1m = 1. - p;
  return dropout_cuda<bool>(gen, self, p1m);
}

// TODO: _fused_dropout_cuda is to be removed, see PR #63937
std::tuple<Tensor,Tensor>
fused_dropout_cuda(const Tensor& self, double p, std::optional<Generator> gen_){
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  return dropout_cuda<uint8_t>(gen, self, p);
}

template <typename mask_t>
Tensor dropout_backward_cuda(const Tensor& grad, const Tensor& mask, double scale){
   Tensor ret = at::empty_like(grad, grad.suggest_memory_format());
   AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "masked_scale", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      masked_scale_kernel<mask_t, scalar_t>(ret, grad, mask, (accscalar_t)scale);
  });
  return ret;
}

Tensor native_dropout_backward_cuda(const Tensor& grad, const Tensor& mask, double scale){
   TORCH_CHECK(mask.scalar_type() == at::ScalarType::Bool, "Mask should be Bool Scalar Type", mask.scalar_type());
  return dropout_backward_cuda<bool>(grad, mask, scale);
}

// TODO: masked_scale_cuda is to be removed, see PR #63937
Tensor masked_scale_cuda(const Tensor& self, const Tensor& mask, double scale){
  TORCH_CHECK(mask.scalar_type() == at::ScalarType::Byte, "mask should be torch.uint8 dtype");
  return dropout_backward_cuda<uint8_t>(self, mask, scale);
}

} // namespace at::native
