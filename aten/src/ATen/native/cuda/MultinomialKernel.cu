#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/EmptyTensor.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/LaunchUtils.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/CUDAFunctions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty_native.h>
#include <ATen/ops/empty_like_native.h>
#include <ATen/ops/cumsum_cuda_dispatch.h>
#include <ATen/ops/uniform_native.h>
#endif

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <type_traits>

namespace at::native {

namespace {

template <
    typename T,
    typename = std::enable_if_t<
        std::is_floating_point_v<T> || std::is_convertible_v<T, float>>>
inline __device__ bool _isinf(T x) {
  if constexpr (std::is_floating_point_v<T>) {
    return ::isinf(x);
  } else {
    return ::isinf(static_cast<float>(x));
  }
}

#define MAX_NUM_BLOCKS 200

// Normalizes the L1 norm of every row to 1; used by multinomial
template <typename scalar_t>
C10_LAUNCH_BOUNDS_1(cuda::detail::CUDA_NUM_THREADS)
__global__ void renormRowsL1(scalar_t* dist, long rows, long cols) {
  extern __shared__  unsigned char my_smem[];
  scalar_t *smem = reinterpret_cast<scalar_t *>(my_smem);
  scalar_t zero = static_cast<scalar_t>(0);
  scalar_t val;
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    scalar_t sum = static_cast<scalar_t>(0);
    for (int64_t col = threadIdx.x; col < cols; col += blockDim.x) {
      val = dist[row * cols + col];
      CUDA_KERNEL_ASSERT(!(val < zero)); // ! < 0 for NaN handling
      sum = sum + val;
    }

    sum = cuda_utils::BlockReduceSum(sum, smem);
    if (threadIdx.x == 0) {
      CUDA_KERNEL_ASSERT(!(val < zero)); // ! < 0 for NaN handling
      smem[0] = sum;
    }
    __syncthreads();

    sum = smem[0];
    if (sum > zero) {
      for (int64_t col = threadIdx.x; col < cols; col += blockDim.x) {
        dist[row * cols + col] = dist[row * cols + col] / sum;
      }
    }
  }
}

void renormRows(Tensor& t) {
  TORCH_CHECK(t.dim() == 2);
  int64_t rows = t.size(0);
  int64_t cols = t.size(1);

  auto props = at::cuda::getCurrentDeviceProperties();
  TORCH_CHECK(props != nullptr);
  int numSM = props->multiProcessorCount;
  const int64_t maxThreads = std::min(
      props->maxThreadsPerBlock, cuda_utils::kCUDABlockReduceMaxThreads());

  int warp_size = at::cuda::warp_size();
  dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
  dim3 block(std::min(maxThreads, warp_size * ceil_div(cols, int64_t{warp_size})));

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, t.scalar_type(), "renormRows_cuda", [&] {
    renormRowsL1<scalar_t>
        <<<grid, block, (block.x / warp_size) * sizeof(scalar_t),
        at::cuda::getCurrentCUDAStream()>>>(t.mutable_data_ptr<scalar_t>(),
            rows, cols);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
}

template <typename scalar_t>
__device__ int binarySearchForMultinomial(const scalar_t* cumdist,
                                          const scalar_t* dist,
                                          int size,
                                          scalar_t val) {
  int start = 0;
  int end = size;
  // cumdist[size - 1] = 0 => all zero prob dist
  CUDA_KERNEL_ASSERT(cumdist[size - 1] > static_cast<scalar_t>(0));

  while (end - start > 0) {
    int mid = start + (end - start) / 2;

    scalar_t midVal = cumdist[mid];
    if (midVal < val) {
      start = mid + 1;
    } else {
      end = mid;
    }
  }

  if (start == size) {
    // No probability mass or precision problems; just return the
    // first non-zero element by setting start to size-1 here,
    // the code below will move it to the last non-zero probability
    // this actually can happen when the random number is 1
    // (github pytorch issue #4858).
    start = size - 1;
  }

  while(start >= 1 && dist[start] == 0) start--;

  return start;
}

template <typename scalar_t>
__global__ void
sampleMultinomialWithReplacement(PhiloxCudaState philox_args,
                                 int totalSamples,
                                 int64_t* dest,
                                 int64_t distributions,
                                 int categories,
                                 const scalar_t* normDistPrefixSum,
                                 const scalar_t* normDist) {
  // At the moment, each warp computes one sample value in the binary
  // search due to divergence. It seems possible to compute multiple
  // values and limit divergence though later on.

  auto seeds = at::cuda::philox::unpack(philox_args);

  // global index formula for 2D grid of 1D blocks
  int idx = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds),
              idx,
              std::get<1>(seeds),
              &state);

  // The block determines the distribution for which we generate a point
  for (int64_t curDist = blockIdx.y;
       curDist < distributions;
       curDist += gridDim.y) {
    for (int sample = blockIdx.x*blockDim.x + threadIdx.x;
         sample < totalSamples; sample += blockDim.x*gridDim.x) {

      //we are losing 3 out of 4 generated numbers but it's ok
      //this kernel is not very efficient anyway
      auto rand = hiprand_uniform4(&state);
      scalar_t r = static_cast<scalar_t>(rand.x);

      // Find the bucket that a uniform sample lies in
      int choice = binarySearchForMultinomial<scalar_t>(
          normDistPrefixSum + curDist * categories,
          normDist + curDist * categories,
          categories,
          r);

      dest[curDist * totalSamples + sample] = choice;

    }
  }
}

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(cuda::detail::CUDA_NUM_THREADS)
__global__ void sampleMultinomialOnce(
    int64_t* dest,
    int64_t distributions,
    int categories,
    const scalar_t* sampled,
    const scalar_t* dist,
    int stride_dist, // dist->stride(0)
    int stride_categories // dist->stride(1)
) {
  extern __shared__  unsigned char my_smem[];
  __shared__ bool found;
  __shared__ unsigned foundPos;

  accscalar_t *smem = reinterpret_cast<accscalar_t *>(my_smem);

  accscalar_t accZero = static_cast<accscalar_t>(0);
  scalar_t zero = static_cast<scalar_t>(0);

  for (int64_t curDist = blockIdx.x;
       curDist < distributions; curDist += gridDim.x) {
    // Each block handles one distribution
    // First pass, find the total sum of the distribution
    accscalar_t sum = accZero;
    scalar_t val;
    for (int cat = threadIdx.x; cat < categories; cat += blockDim.x) {
      val = dist[curDist * stride_dist + cat * stride_categories];
      CUDA_KERNEL_ASSERT(!at::_isnan(val));
      CUDA_KERNEL_ASSERT(!_isinf(val));
      CUDA_KERNEL_ASSERT(!(val < zero));
      sum = sum + static_cast<accscalar_t>(val);
    }

    // threadIdx.x == 0 has the sum value from this
    sum = cuda_utils::BlockReduceSum(sum, smem);

    // Broadcast sum and sample value
    if (threadIdx.x == 0) {
      // Make sure the sum of our distribution didn't overflow
      CUDA_KERNEL_ASSERT(!_isinf(val));
      CUDA_KERNEL_ASSERT(sum > accZero);

      foundPos = 0;
      smem[0] = sum;
      smem[1] = sampled[curDist];
    }
    __syncthreads();

    sum = smem[0];
    scalar_t sample = static_cast<scalar_t>(smem[1]);
    __syncthreads();

    if (sum == accZero) {
      // Choose the first element
      if (threadIdx.x == 0) {
        dest[curDist] = 0;
      }

      continue;
    }

    int chunks = (categories + (int)blockDim.x - 1) / blockDim.x;
    accscalar_t prevHighProb = accZero;
    found = false;

    for (int chunk = 0; chunk < chunks && !found; ++chunk) {
      // All threads in bounds load a value
      int cat = chunk * blockDim.x + threadIdx.x;

      accscalar_t dist_val = cat < categories ?
                             static_cast<accscalar_t>(dist[curDist * stride_dist + cat * stride_categories]) / sum :
                             accZero;

      smem[threadIdx.x] = dist_val;
      __syncthreads();

      // Perform an inclusive prefix sum of the shared memory contents
      for (int offset = 1; offset < blockDim.x; offset *= 2) {
        accscalar_t val = accZero;

        if (threadIdx.x >= offset) {
          val = smem[threadIdx.x - offset] + smem[threadIdx.x];
        }

        __syncthreads();
        if (threadIdx.x >= offset) {
          smem[threadIdx.x] = val;
        }
        __syncthreads();
      }

      // Each thread will check to see if the sample falls in its
      // bucket
      scalar_t curBucket =
          static_cast<scalar_t>(smem[threadIdx.x] + prevHighProb);
      scalar_t prevBucket = static_cast<scalar_t>(
          threadIdx.x == 0 ? prevHighProb
                          : smem[threadIdx.x - 1] + prevHighProb);
      bool inBucket =
          (cat < categories) &&
          (!(sample >= curBucket) &&
          (sample >= prevBucket) &&
          (dist_val > zero));

      if (inBucket) {
        // We're done; we have the sample
        // Torch indices are 1-based
        atomicMax(&foundPos, cat);
        found = true;
      }

      // Store the previous scan's high value for future use
      prevHighProb = prevHighProb + smem[blockDim.x - 1];

      __syncthreads();
    }

    if (threadIdx.x == 0) {
      if (found) {
          dest[curDist] = foundPos;
      } else {
        // This should address a rare bug where we don't select a valid index. This likely occurs when
        // due to floating point arithmetic rounding errors, our cumulative sum does not add up to 1, but
        // and our uniform sample is greater than this value. In this case we likely have uninitialized memory
        // in dest[curDist]. So basically we will loop through the distribution and pick the largest index
        // where the distribution is non-zero. This is obviously terribly inefficient, but due to the
        // rarity in which this occurs, this should not be an issue.
        for (int cat = categories - 1; cat >= 0; --cat) {
          if (dist[curDist * stride_dist + cat * stride_categories] > zero) {
            dest[curDist] = cat;
            break;
          }
        }
      }
    }
  }
}

void multinomial_with_replacement_kernel_impl(
    Tensor& result,
    const Tensor& self,
    const int64_t n_sample,
    std::optional<Generator> generator) {
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(generator, cuda::detail::getDefaultCUDAGenerator());

  int inputSize = self.dim();
  int64_t numDist =
      inputSize == 1 ? 1 : self.size(0);
  int numCategories =
      inputSize == 1 ? self.size(0) : self.size(1);

  // Restructure data for 2d
  auto self_v = inputSize == 1 ? self.view({numDist, numCategories}) : self;

  result.resize_({numDist, n_sample});

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, self_v.scalar_type(), "multinomial_kernel_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto props = at::cuda::getCurrentDeviceProperties();
    TORCH_CHECK(props != nullptr);
    int numSM = props->multiProcessorCount;
    int maxThreads = props->maxThreadsPerBlock;
    int maxShared = props->sharedMemPerBlock;

    int warp_size = at::cuda::warp_size();
    int requiredWarps = at::ceil_div(numCategories, warp_size);
    int requiredThreads = std::min(maxThreads, requiredWarps * warp_size);
    int requiredShared = requiredThreads * sizeof(accscalar_t);

    if (n_sample == 1 && maxShared >= requiredShared) {
      // Optimized allocation-free implementation
      // To exploit greater parallelism for the sampling, generate the
      // Uniform random samples in a separate kernel launch, into
      // temporarily allocated memory. The device RNG is thread-limited
      Tensor sampled = at::detail::empty_cuda({numDist, n_sample}, self_v.options());
      at::native::uniform_(sampled, 0.0, 1.0, generator);

      dim3 block(requiredThreads);
      dim3 grid(std::min(static_cast<int>(numDist), numSM * 4));

      sampleMultinomialOnce<scalar_t, accscalar_t>
          <<<grid, block,
          requiredShared,
          at::cuda::getCurrentCUDAStream()>>>(
              result.mutable_data_ptr<int64_t>(),
                  numDist,
                  numCategories,
                  sampled.const_data_ptr<scalar_t>(),
                  self_v.const_data_ptr<scalar_t>(),
                  self_v.stride(0),
                  self_v.stride(1)
          );
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      // Generic, slow implementation with memory allocations

      // For sampling without replacement, we modify the distribution
      // for subsequent samples in this space
      Tensor origDist = native::empty_like(
          self_v,
          std::nullopt /* dtype */,
          std::nullopt /* layout */,
          std::nullopt /* device */,
          std::nullopt /* pin_memory */,
          LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      origDist.copy_(self_v);

      Tensor normDist = native::empty_like(
          self_v,
          std::nullopt /* dtype */,
          std::nullopt /* layout */,
          std::nullopt /* device */,
          std::nullopt /* pin_memory */,
          LEGACY_CONTIGUOUS_MEMORY_FORMAT);

      Tensor prefixSum = native::empty_like(
          self_v,
          std::nullopt /* dtype */,
          std::nullopt /* layout */,
          std::nullopt /* device */,
          std::nullopt /* pin_memory */,
          LEGACY_CONTIGUOUS_MEMORY_FORMAT);

      // Renorm along rows
      normDist.copy_(origDist);
      renormRows(normDist);

      // Prefix sum along rows
      at::cuda::cumsum_out(prefixSum, normDist, 1);

      PhiloxCudaState rng_engine_inputs;

        // Binary search is warp divergent (so effectively we're running
        // with just a single thread), but for better utilization,
        // we need each block to have at least 4 warps.
        dim3 block(128);

        // Each block will generate a sample from one
        // distribution concurrently.
        int grid_y=std::min<int>(numDist, at::cuda::getCurrentDeviceProperties()->maxGridSize[1]);
        dim3 grid((n_sample-1)/block.x+1, grid_y);
        {
          // See Note [Acquire lock when using random generators]
          std::lock_guard<std::mutex> lock(gen->mutex_);

          // each thread generates a single sample for (numdist/numblocks.y) distributions, however, since we have to use
          // hiprand_uniform4 (See Note [Register spilling in hiprand call for CUDA < 10]),
          // offset is 4 times that.
          auto offset = ((numDist-1)/grid.y+1)*4;
          rng_engine_inputs = gen->philox_cuda_state(offset);
        }
        // Sample with replacement

        sampleMultinomialWithReplacement
            <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
                rng_engine_inputs,
                n_sample,
                result.mutable_data_ptr<int64_t>(),
                numDist, numCategories,
                prefixSum.const_data_ptr<scalar_t>(),
                normDist.const_data_ptr<scalar_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  });

  if (inputSize == 1) {
    result.resize_({n_sample});
  }
}
}

REGISTER_DISPATCH(
    multinomial_with_replacement_stub,
    &multinomial_with_replacement_kernel_impl);
} // namespace at::native
