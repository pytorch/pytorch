#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/ForeachUtils.h>
#include <ATen/native/cuda/ForeachFunctors.cuh>
#include <ATen/native/cuda/ForeachMinMaxFunctors.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_foreach_add_native.h>
#include <ATen/ops/_foreach_clamp_max_native.h>
#include <ATen/ops/_foreach_clamp_min_native.h>
#include <ATen/ops/_foreach_div_native.h>
#include <ATen/ops/_foreach_mul_native.h>
#include <ATen/ops/_foreach_pow_native.h>
#include <ATen/ops/_foreach_sub_native.h>

#include <ATen/ops/empty_like_native.h>
#endif

namespace at::native {

template <typename T, template <class> class Op>
std::vector<Tensor> foreach_binary_op(
    TensorList tensors,
    const Scalar& scalar) {
  std::vector<std::vector<at::Tensor>> tensor_lists;
  std::vector<at::Tensor> vec_res;
  vec_res.reserve(tensors.size());
  for (const auto& t : tensors) {
    vec_res.emplace_back(at::native::empty_like(t));
  }

  tensor_lists.emplace_back(tensors.vec());
  tensor_lists.emplace_back(std::move(vec_res));

  using opmath_t = at::opmath_type<T>;
  multi_tensor_apply<2>(
      tensor_lists,
      BinaryOpScalarFunctor<
          T,
          /* depth */ 2,
          /* r_args_depth */ 1,
          /* res_arg_index */ 1>(),
      Op<opmath_t>(),
      scalar.to<opmath_t>());
  return std::move(tensor_lists[1]);
}

template <typename T, template <class> class Op>
void foreach_binary_op_(TensorList tensors, const Scalar& scalar) {
  std::vector<std::vector<at::Tensor>> tensor_lists;
  tensor_lists.emplace_back(tensors.vec());

  using opmath_t = at::opmath_type<T>;
  multi_tensor_apply<1>(
      tensor_lists,
      BinaryOpScalarFunctor<
          T,
          /* depth */ 1,
          /* r_args_depth */ 1,
          /* res_arg_index */ 0>(),
      Op<opmath_t>(),
      scalar.to<opmath_t>());
  increment_version(tensors);
}

template <template <class> class Op>
std::vector<Tensor> all_types_complex_bool_half_bfloat16(
    TensorList tensors,
    const Scalar& scalar) {
  return AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      kBool,
      kHalf,
      kBFloat16,
      tensors[0].scalar_type(),
      "foreach_binary_op_scalar_cuda",
      [&]() { return foreach_binary_op<scalar_t, Op>(tensors, scalar); });
}

template <template <class> class Op>
void all_types_complex_bool_half_bfloat16_(
    TensorList tensors,
    const Scalar& scalar) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      kBool,
      kHalf,
      kBFloat16,
      tensors[0].scalar_type(),
      "foreach_binary_op_scalar_cuda_",
      [&]() { foreach_binary_op_<scalar_t, Op>(tensors, scalar); });
}

template <template <class> class Op>
std::vector<Tensor> all_types_half_bfloat16(
    TensorList tensors,
    const Scalar& scalar) {
  return AT_DISPATCH_ALL_TYPES_AND2(
      kHalf,
      kBFloat16,
      tensors[0].scalar_type(),
      "foreach_binary_op_scalar_cuda",
      [&]() { return foreach_binary_op<scalar_t, Op>(tensors, scalar); });
}

template <template <class> class Op>
void all_types_half_bfloat16_(TensorList tensors, const Scalar& scalar) {
  AT_DISPATCH_ALL_TYPES_AND2(
      kHalf,
      kBFloat16,
      tensors[0].scalar_type(),
      "foreach_binary_op_scalar_cuda_",
      [&]() { foreach_binary_op_<scalar_t, Op>(tensors, scalar); });
}

template <template <class> class Op>
std::vector<Tensor> all_types_complex_half_bfloat16(
    TensorList tensors,
    const Scalar& scalar) {
  return AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
      kHalf,
      kBFloat16,
      tensors[0].scalar_type(),
      "foreach_binary_op_scalar_cuda",
      [&]() { return foreach_binary_op<scalar_t, Op>(tensors, scalar); });
}

template <template <class> class Op>
void all_types_complex_half_bfloat16_(
    TensorList tensors,
    const Scalar& scalar) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
      kHalf,
      kBFloat16,
      tensors[0].scalar_type(),
      "foreach_binary_op_scalar_cuda_",
      [&]() { foreach_binary_op_<scalar_t, Op>(tensors, scalar); });
}

#define FOREACH_BINARY_OP_SCALAR(FUNCTION, NAME, OP, DIVISION_OP)     \
  void foreach_tensor_##NAME##_scalar_kernel_cuda_(                   \
      TensorList tensors, const Scalar& scalar) {                     \
    check_foreach_api_restrictions(tensors);                          \
    if (!can_use_fast_route(tensors, scalar, DIVISION_OP)) {          \
      return at::native::foreach_tensor_##NAME##_scalar_kernel_slow_( \
          tensors, scalar);                                           \
    }                                                                 \
                                                                      \
    FUNCTION##_<OP>(tensors, scalar);                                 \
  }                                                                   \
                                                                      \
  std::vector<Tensor> foreach_tensor_##NAME##_scalar_kernel_cuda(     \
      TensorList tensors, const Scalar& scalar) {                     \
    check_foreach_api_restrictions(tensors);                          \
    if (!can_use_fast_route(tensors, scalar, DIVISION_OP)) {          \
      return at::native::foreach_tensor_##NAME##_scalar_kernel_slow(  \
          tensors, scalar);                                           \
    }                                                                 \
                                                                      \
    return FUNCTION<OP>(tensors, scalar);                             \
  }

FOREACH_BINARY_OP_SCALAR(
    all_types_complex_bool_half_bfloat16,
    add,
    std::plus,
    /*div_op*/ false);
FOREACH_BINARY_OP_SCALAR(
    all_types_complex_bool_half_bfloat16,
    mul,
    std::multiplies,
    /*div_op*/ false);
// See [Why is foreach_pow's division_op=true?]
FOREACH_BINARY_OP_SCALAR(
    all_types_complex_half_bfloat16,
    pow,
    power_functor,
    /*div_op*/ true);
std::vector<Tensor> foreach_scalar_pow_list_kernel_cuda(
    const Scalar& scalar,
    TensorList exponent) {
  check_foreach_api_restrictions(exponent);
  if (!can_use_fast_route(exponent)) {
    return at::native::foreach_scalar_pow_list_kernel_slow(scalar, exponent);
  }
  return all_types_complex_half_bfloat16<reverse_power_functor>(
      exponent, scalar);
}

// In the case of division, integer inputs will result in float.
// Currently multi tensor apply can only return result of the same type as
// input.
//
// Implement via multiply with reciprocal as it's faster and makes it match
// the behavior of regular Tensor div by scalar.  Loses one bit of
// precision.
Scalar scalar_reciprocal(const Scalar& scalar) {
  if (scalar.isFloatingPoint()) {
    return Scalar(1. / scalar.toDouble());
  } else if (scalar.isIntegral(/*includeBool*/ true)) {
    return Scalar(1. / static_cast<double>(scalar.toLong()));
  } else if (scalar.isComplex()) {
    return Scalar(1. / scalar.toComplexDouble());
  }
  TORCH_INTERNAL_ASSERT(
      false, "divison with ", scalar.type(), " not supported");
}

void foreach_tensor_div_scalar_kernel_cuda_(
    TensorList tensors,
    const Scalar& scalar) {
  check_foreach_api_restrictions(tensors);
  if (!can_use_fast_route(tensors, scalar, true)) {
    return at::native::foreach_tensor_mul_scalar_kernel_slow_(
        tensors, scalar_reciprocal(scalar));
  }

  all_types_complex_bool_half_bfloat16_<std::multiplies>(
      tensors, scalar_reciprocal(scalar));
}

std::vector<Tensor> foreach_tensor_div_scalar_kernel_cuda(
    TensorList tensors,
    const Scalar& scalar) {
  check_foreach_api_restrictions(tensors);
  if (!can_use_fast_route(tensors, scalar, true)) {
    return at::native::foreach_tensor_mul_scalar_kernel_slow(
        tensors, scalar_reciprocal(scalar));
  }

  return all_types_complex_bool_half_bfloat16<std::multiplies>(
      tensors, scalar_reciprocal(scalar));
}

// In the case of subtraction, we dont allow scalar to be boolean following the
// torch.sub logic
void foreach_tensor_sub_scalar_kernel_cuda_(
    TensorList tensors,
    const Scalar& scalar) {
  check_foreach_api_restrictions(tensors);
  at::native::sub_check(tensors[0], scalar);

  if (!can_use_fast_route(tensors, scalar)) {
    return at::native::foreach_tensor_sub_scalar_kernel_slow_(tensors, scalar);
  }

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      kBool,
      kHalf,
      kBFloat16,
      tensors[0].scalar_type(),
      "foreach_binary_op_scalar_cuda_",
      [&]() { foreach_binary_op_<scalar_t, std::minus>(tensors, scalar); });
}

std::vector<Tensor> foreach_tensor_sub_scalar_kernel_cuda(
    TensorList tensors,
    const Scalar& scalar) {
  check_foreach_api_restrictions(tensors);
  at::native::sub_check(tensors[0], scalar);

  if (!can_use_fast_route(tensors, scalar)) {
    return at::native::foreach_tensor_sub_scalar_kernel_slow(tensors, scalar);
  }

  return AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      kBool,
      kHalf,
      kBFloat16,
      tensors[0].scalar_type(),
      "foreach_binary_op_scalar_cuda",
      [&]() {
        return foreach_binary_op<scalar_t, std::minus>(tensors, scalar);
      });
}

// NOTE(crcrpar): `all_types_half_bfloat16` does not cover bool, so temporarily
// set `division_op` to true.
FOREACH_BINARY_OP_SCALAR(all_types_half_bfloat16, clamp_max, minimum, true);
FOREACH_BINARY_OP_SCALAR(all_types_half_bfloat16, clamp_min, maximum, true);

} // namespace at::native
