#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/TensorOperators.h>
#include <ATen/WrapDimUtils.h>
#include <c10/macros/Macros.h>

#include <ATen/AccumulateType.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <type_traits>

#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/PersistentSoftmax.cuh>
#include <ATen/native/IndexingUtils.h>
#include <ATen/native/cuda/block_reduce.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_masked_softmax_native.h>
#include <ATen/ops/_log_softmax_native.h>
#include <ATen/ops/_log_softmax_backward_data_native.h>
#include <ATen/ops/_softmax_native.h>
#include <ATen/ops/_softmax_backward_data_native.h>
#include <ATen/ops/softmax.h>
#include <ATen/ops/_softmax_backward_data.h>
#endif

namespace at::native {

namespace {

constexpr int ALIGN_BYTES = 16;

template<typename T, typename AccumT, typename OutT>
struct LogSoftMaxForwardEpilogue {
  __device__ __forceinline__ LogSoftMaxForwardEpilogue(AccumT max_input, AccumT sum)
    : max_input(max_input),  logsum(std::log(sum)) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(input - max_input - logsum);
}

  const AccumT max_input;
  const AccumT logsum;
};

template<typename T, typename AccumT, typename OutT>
struct LogSoftMaxBackwardEpilogue {
  __device__ __forceinline__ LogSoftMaxBackwardEpilogue(AccumT sum)
    : sum(sum) {}

  __device__ __forceinline__ T operator()(OutT gradOutput, OutT output) const {
    return static_cast<T>(gradOutput - std::exp(static_cast<AccumT>(output)) * sum);
  }

  const AccumT sum;
};

template<typename T, typename AccumT, typename OutT>
struct SoftMaxForwardEpilogue {
  __device__ __forceinline__ SoftMaxForwardEpilogue(AccumT max_input, AccumT sum)
    : max_input(max_input)
    , sum(sum) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(std::exp(input - max_input) / sum);
  }

  const AccumT max_input;
  const AccumT sum;
};

template<typename T, typename AccumT, typename OutT>
struct SoftMaxBackwardEpilogue {
  __device__ __forceinline__ SoftMaxBackwardEpilogue(AccumT sum)
    : sum(sum) {}

  // XXX: gradOutput that we get here is really gradOutput * output
  // Look for cmul in SoftMax_updateGradInput
  __device__ __forceinline__ T operator()(OutT gradOutput, OutT output) const {
    return static_cast<T>(gradOutput - output * sum);
  }

  const AccumT sum;
};

template<typename T, typename AccumT, typename OutT>
 struct SoftMaxForwardWithMulEpilogue {
   __device__ __forceinline__ SoftMaxForwardWithMulEpilogue(AccumT max_input, AccumT sum)
     : max_input(max_input)
     , sum(sum) {}

   __device__ __forceinline__ OutT operator()(T input) const {
     return static_cast<OutT>(__expf(input - max_input) * sum);
   }

   const AccumT max_input;
   const AccumT sum;
 };




////////////////////////////////////////////////////////////////////////////////
// Spatial kernel (fast with large inner_size and small dim_size)
////////////////////////////////////////////////////////////////////////////////
// Let's assume that our input has been flattened to have only three dimension:
//     outer x dim x inner
// The spatial algorithm tries to parallelize along all of them.
// Within a 2d block threadIdx.y parallelizes over dim slices, and threads that
// share it will speed up reductions over dim (along axis x).
// The 2d grid is used to parallelize inner dimension over y axis and outer over x.
inline dim3 SpatialSoftMax_getGridSize(
    dim3 block, uint32_t max_active_blocks,
    uint64_t outer_size, uint64_t inner_size) {
  // First, tile as many blocks as we can over the y axis
  uint32_t inner_blocks = (inner_size + block.y - 1) / block.y;
  if (inner_blocks > max_active_blocks)
    inner_blocks = max_active_blocks;
  // Fill the x axis with as many blocks as we can fit (a little more is ok too)
  uint32_t outer_blocks = (max_active_blocks + inner_blocks - 1) / inner_blocks;
  if (outer_blocks > outer_size)
    outer_blocks = outer_size;
  return dim3(outer_blocks, inner_blocks);
}

const int max_threads = 1024;

inline dim3 SpatialSoftMax_getBlockSize(
  uint64_t dim_size, uint64_t inner_size) {
  uint32_t inner_threads = inner_size;
  inner_threads = std::min(inner_threads, static_cast<uint32_t>(max_threads));
  uint32_t dim_threads = 1;
  if (inner_threads <= 64 && dim_size >= 64) {
    while (inner_threads * dim_threads <= max_threads && dim_threads <= dim_size)
      dim_threads *= 2;
    dim_threads /= 2;
  }
  return dim3(dim_threads, inner_threads);
}


template<typename accscalar_t, typename Kernel>
void SpatialSoftMax_getLaunchSizes(
    Kernel k,
    uint64_t outer_size, uint64_t dim_size, uint64_t inner_size,
    dim3& grid, dim3& block, uint32_t& smem_size) {
  block = SpatialSoftMax_getBlockSize(dim_size, inner_size);
  uint32_t block_threads = block.x * block.y;
  smem_size = block.x == 1 ? 0 : block_threads * sizeof(accscalar_t);
  int max_active_blocks;
  AT_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                                                              k, block_threads, smem_size));
  max_active_blocks *= at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  grid = SpatialSoftMax_getGridSize(block, max_active_blocks, outer_size, inner_size);
}

inline dim3 SoftMax_getBlockSize(int ILP, uint64_t dim_size) {
  uint64_t block_size = 1;
  uint64_t max_block_size = std::min(dim_size / ILP, static_cast<uint64_t>(max_threads));

  // In the vectorized case we want to trade off allowing more of the buffers to be accessed
  // in a vectorized way against wanting a larger block size to get better utilisation.
  // In general with ILP you can have (ILP-1)/ILP of the buffer accessed vectorised, at the risk
  // of having a very small block size. We choose to keep >= 1/2 of the buffer vectorised while
  // allowing a larger block size.
  if (ILP > 1) {
    max_block_size /= 2;
  }

  while (block_size < (max_block_size)) block_size *= 2;
  // Launch at least a single warp - the kernel assumes that.
  block_size = std::max(block_size, static_cast<uint64_t>(at::cuda::warp_size()));
  return dim3(block_size);
}

inline dim3 SoftMaxForward_getBlockSize(uint64_t dim_size) {
  uint64_t block_size = 1;
  uint64_t max_block_size = std::min(dim_size, static_cast<uint64_t>(max_threads));

  // We need a block size that is a multiple of at::cuda::warp_size() in order
  // to perform block size reductions using warp shuffle instructions.
  // Since max_threads is also a multiple of at::cuda::warp_size() we do not
  // risk creating a block size larger than the limit.

  int warp_size = at::cuda::warp_size();
  if (max_block_size % warp_size == 0) {
    block_size = max_block_size;
  } else {
    block_size = (max_block_size / warp_size + 1) * warp_size;
  }

  return dim3(block_size);
}

template<typename T>
struct Add {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a + b;
  }

  __device__ __forceinline__ T combine(T a, T b) const {
    return a + b;
  }

  // Needed to allow warp level reduction as a first step in the
  // thread block reduction
  __device__ __forceinline__ T warp_shfl_down(T data, int offset) const {
    return WARP_SHFL_DOWN(data, offset);
  }
};

template<typename T>
struct Max {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a < b ? b : a;
  }

  __device__ __forceinline__ T combine(T a, T b) const {
    return a < b ? b : a;
  }

  // Needed to allow warp level reduction as a first step in the
  // thread block reduction
  __device__ __forceinline__ T warp_shfl_down(T data, int offset) const {
    return WARP_SHFL_DOWN(data, offset);
  }
};

// Note that it's not a complete block-wide reduction.
// Only threads that share threadIdx.y reduce values.
template<typename T, template<typename> class ReduceOp>
__forceinline__ __device__
T spatialBlockReduceX(T *shared, T val) {
  ReduceOp<T> r;
  shared += threadIdx.y * blockDim.x;

  __syncthreads();

  shared[threadIdx.x] = val;

  // NOTE: loop starts with __syncthreads()
  int offset = blockDim.x / 2;
  while (offset > 0) {
    __syncthreads();
    if (threadIdx.x < offset)
      shared[threadIdx.x] = r(shared[threadIdx.x], shared[threadIdx.x + offset]);
    offset /= 2;
  }

  __syncthreads();

  return shared[0];
}

template <typename scalar_t, typename accscalar_t, typename outscalar_t, typename index_t, template<typename, typename, typename> class Epilogue>
__global__ void cunn_SpatialSoftMaxForward(
    outscalar_t *output, const scalar_t *input,
    index_t outer_size, index_t dim_size, index_t inner_size)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  const index_t outer_stride = inner_size * dim_size;
  const index_t dim_stride = inner_size;

  for (index_t outer_index = blockIdx.x; outer_index < outer_size; outer_index += gridDim.x) {
    const index_t outer_offset = outer_index * outer_stride;
    for (index_t inner_index = blockIdx.y * blockDim.y + threadIdx.y; inner_index < inner_size; inner_index += blockDim.y * gridDim.y) {
      const index_t data_offset = outer_offset + inner_index;
      ////////////////////////////////////////////////////////////
      // These two blocks are really equivalent, but specializing on
      // blockDim.x == 1 makes the kernel faster when it's unused.
      // I didn't want to thread an extra template parameter, and nvcc
      // seems to be smart enough to hoist the if outside of the loops.
      ////////////////////////////////////////////////////////////

      if (blockDim.x > 1) {
        accscalar_t max_input = at::numeric_limits<accscalar_t>::lowest();
        for (index_t d = threadIdx.x; d < dim_size; d += blockDim.x) {
          const accscalar_t value = static_cast<accscalar_t>(input[data_offset + d * dim_stride]);
          max_input = Max<accscalar_t>()(max_input, value);
        }
        max_input = spatialBlockReduceX<accscalar_t, Max>(sdata,max_input);

        accscalar_t sum = 0;
        for (index_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          sum += std::exp(static_cast<accscalar_t>(input[data_offset + d * dim_stride])
                 - max_input);
        sum = spatialBlockReduceX<accscalar_t, Add>(sdata, sum);

        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_input, sum);
        for (index_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          output[data_offset + d * dim_stride] = epilogue(input[data_offset + d * dim_stride]);
      } else {
        accscalar_t max_input = at::numeric_limits<accscalar_t>::lowest();
        for (index_t d = threadIdx.x; d < dim_size; d += blockDim.x) {
          const accscalar_t value = static_cast<accscalar_t>(input[data_offset + d * dim_stride]);
          max_input = Max<accscalar_t>()(max_input, value);
        }
        accscalar_t sum = 0;
        for (index_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          sum += std::exp(static_cast<accscalar_t>(input[data_offset + d * dim_stride])
                 - max_input);
        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_input, sum);
        for (index_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          output[data_offset + d * dim_stride] = epilogue(input[data_offset + d * dim_stride]);
      }
    }
  }
}



template <typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void cunn_SpatialSoftMaxBackward(
    scalar_t *gradInput, const outscalar_t *output, const outscalar_t *gradOutput,
    uint32_t outer_size, uint32_t dim_size, uint32_t inner_size)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  const uint32_t outer_stride = inner_size * dim_size;
  const uint32_t dim_stride = inner_size;

  for (uint32_t outer_index = blockIdx.x; outer_index < outer_size; outer_index += gridDim.x) {
    const uint32_t outer_offset = outer_index * outer_stride;
    for (uint32_t inner_index = blockIdx.y * blockDim.y + threadIdx.y; inner_index < inner_size; inner_index += blockDim.y * gridDim.y) {
      const uint32_t data_offset = outer_offset + inner_index;
      // See the comment in forward kernel
      if (blockDim.x > 1) {
        accscalar_t sum = 0;
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          sum += gradOutput[data_offset + d * dim_stride];
        sum = spatialBlockReduceX<accscalar_t, Add>(sdata, sum);

        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum);
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x) {
          gradInput[data_offset + d * dim_stride] =
            epilogue(gradOutput[data_offset + d * dim_stride],
                    output[data_offset + d * dim_stride]);
        }
      } else {
        accscalar_t sum = 0;
        for (uint32_t d = 0; d < dim_size; d++)
          sum += gradOutput[data_offset + d * dim_stride];

        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum);
        for (uint32_t d = 0; d < dim_size; d++) {
          gradInput[data_offset + d * dim_stride] =
            epilogue(gradOutput[data_offset + d * dim_stride],
                    output[data_offset + d * dim_stride]);
        }
      }
    }
  }
}


////////////////////////////////////////////////////////////////////////////////
// Regular kernel (fast when dim_size is large; requires inner_size == 1)
////////////////////////////////////////////////////////////////////////////////


template <typename T, typename AccumT>
struct MaxFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT max, T v) const {
    return ::max(max, (AccumT)v);
  }
};

template<typename T, typename AccumT>
struct AddFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + v;
  }
};

template<typename T, typename AccumT>
struct SumExpFloat
{
  __device__ __forceinline__ SumExpFloat(AccumT v)
    : max_k(v) {}

  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + std::exp(v - max_k);
  }

  const AccumT max_k;
};

template<typename T, typename AccumT>
struct SumExpfFloat
{
  __device__ __forceinline__ SumExpfFloat(AccumT v)
    : max_k(v) {}

  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + __expf(v - max_k);
  }

  const AccumT max_k;
};

template <template<typename> class Reduction, typename AccumT>
__device__ __forceinline__ AccumT
blockReduce(AccumT* smem, AccumT val,
            const Reduction<AccumT>& r,
            AccumT defaultVal)
{
  // To avoid RaW races from chaining blockReduce calls together, we need a sync here
  __syncthreads();

  smem[threadIdx.x] = val;

  __syncthreads();

  AccumT warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  uint32_t mask = (((uint64_t)1) << (blockDim.x / C10_WARP_SIZE)) - 1;
  if (threadIdx.x < C10_WARP_SIZE) {
    int lane = threadIdx.x % C10_WARP_SIZE;
    if (lane < blockDim.x / C10_WARP_SIZE) {
#pragma unroll
      for (int i = 0; i < C10_WARP_SIZE; ++i) {
        warpVal = r(warpVal, smem[lane * C10_WARP_SIZE + i]);
      }
#if !defined(USE_ROCM)
      __syncwarp(mask);
#endif
      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  AccumT blockVal = defaultVal;

  if (threadIdx.x == 0) {
    for (int i = 0; i < blockDim.x / C10_WARP_SIZE; ++i) {
      blockVal = r(blockVal, smem[i]);
    }
    smem[0] = blockVal;
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

// Performs a thread block reduction with a given functor but uses
// warp shuffles as the first step in the reduction
template <template<typename> class Reduction, typename T>
__device__ __forceinline__
T blockReduceWarp(T* smem_cache, T value, const Reduction<T>& op, T defaultVal)
{
  T result = cuda_utils::BlockReduce<T, Reduction<T>>(value, op, defaultVal, smem_cache);
  if (threadIdx.x == 0) {
    smem_cache[0] = result;
  }
  __syncthreads();
  return smem_cache[0];
}


template <template<typename> class Reduction, typename T>
__device__ __forceinline__
T blockReduceWarpInverse(T* smem_cache, T value, const Reduction<T>& op, T defaultVal)
{
  T result = cuda_utils::BlockReduce<T, Reduction<T>>(value, op, defaultVal, smem_cache);
  if (threadIdx.x == 0) {
    smem_cache[0] = 1 / result;
  }
  __syncthreads();
  return smem_cache[0];
}

template <template<typename, typename> class Reduction, int ILP, typename T, typename AccumT, typename index_t=int>
__device__ __forceinline__ AccumT
ilpReduce(index_t shift,
          const T* data,
          index_t size,
          const Reduction<T, AccumT>& r,
          AccumT defaultVal)
{
  using LoadT = at::native::memory::aligned_vector<T, ILP>;
  AccumT threadVal = defaultVal;
  index_t offset = threadIdx.x;

  // shift and do 1
  if(shift > 0){
    data -= shift;
    size += shift;
    if (offset >= shift && offset < size) {
      threadVal = r(threadVal, data[offset]);
    }
    size -= blockDim.x > size ? size : blockDim.x;
    data += blockDim.x;
  }
  index_t last = size % (ILP * blockDim.x);

  T v[ILP];
  LoadT* value = reinterpret_cast<LoadT*>(&v);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *value = reinterpret_cast<const LoadT*>(data)[offset];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      threadVal = r(threadVal, v[j]);
    }
  }

  offset = size - last + threadIdx.x;
  // Epilogue
  for (; offset < size; offset += blockDim.x)
    threadVal = r(threadVal, data[offset]);

  return threadVal;
}

int32_t potential_register_count(int32_t dim_size, int32_t thread_count){
  // This method calculate the potential register count for ilpReduce method (it's just a rough number).
  int reg_cnt = (dim_size + thread_count - 1) / thread_count;
  return reg_cnt;
}

/**
 * This will apply the Epilogue with vectorized reads & writes when input & output have the same shift
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__device__ __forceinline__ void
WriteFpropResultsVectorized(
             int size,
             const int shift,
             const scalar_t *input,
             outscalar_t *output,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  using LoadT = at::native::memory::aligned_vector<scalar_t, ILP>;
  using StoreT = at::native::memory::aligned_vector<outscalar_t, ILP>;

  int offset = threadIdx.x;

  // if unaligned, do one value / thread and move on, guaranteeing aligned reads/writes later
  if (shift > 0) {
    input -= shift;
    output -= shift;
    size += shift;

    if (offset >= shift && offset < size) {
      output[offset] = epilogue(input[offset]);
    }
    size -= blockDim.x > size ? size : blockDim.x;
    input += blockDim.x;
    output += blockDim.x;
  }

  const int last = size % (ILP * blockDim.x);

  scalar_t in_v[ILP];
  LoadT* in_value = reinterpret_cast<LoadT*>(&in_v);

  outscalar_t out_v[ILP];
  const StoreT* out_value = reinterpret_cast<const StoreT*>(&out_v);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *in_value = reinterpret_cast<const LoadT*>(input)[offset];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      out_v[j] = epilogue(in_v[j]);
    }

    reinterpret_cast<StoreT*>(output)[offset] = *out_value;
  }

  offset = size - last + threadIdx.x;
  // handle the tail
  for (; offset < size; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}

template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue, typename index_t = int32_t>
__device__ __forceinline__ void
WriteBpropResultsVectorized(
             index_t size,
             const index_t shift,
             scalar_t *gradInput,
             const outscalar_t *output,
             const outscalar_t *gradOutput,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  using gradInputT = at::native::memory::aligned_vector<scalar_t, ILP>;
  using outputT = at::native::memory::aligned_vector<outscalar_t, ILP>;

  index_t offset = threadIdx.x;

  // if unaligned, do one value / thread and move on, guaranteeing aligned reads/writes later
  if (shift > 0) {
    gradInput -= shift;
    output -= shift;
    gradOutput -= shift;
    size += shift;

    if (threadIdx.x >= shift) {
      gradInput[offset] = epilogue(gradOutput[offset], output[offset]);
    }
    size -= blockDim.x > size ? size : blockDim.x;
    gradInput += blockDim.x;
    output += blockDim.x;
    gradOutput += blockDim.x;
  }

  const index_t last = size % (ILP * blockDim.x);

  scalar_t dX[ILP];
  gradInputT *dX_v = reinterpret_cast<gradInputT*>(&dX);

  outscalar_t Y[ILP];
  outputT *Y_v = reinterpret_cast<outputT*>(&Y);

  outscalar_t dY[ILP];
  outputT *dY_v = reinterpret_cast<outputT*>(&dY);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *Y_v = reinterpret_cast<const outputT*>(output)[offset];
    *dY_v = reinterpret_cast<const outputT*>(gradOutput)[offset];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      dX[j] = epilogue(dY[j], Y[j]);
    }

    reinterpret_cast<gradInputT*>(gradInput)[offset] = *dX_v;
  }

  offset = size - last + threadIdx.x;
  for (; offset < size; offset += blockDim.x) {
    gradInput[offset] = epilogue(gradOutput[offset], output[offset]);
  }
}

/**
 * This will apply the Epilogue with non-vectorized reads & writes for the general case
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__device__ __forceinline__ void
WriteFpropResults(
             int classes,
             const scalar_t *input,
             outscalar_t *output,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  for (int offset = threadIdx.x; offset < classes; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}

template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue, typename index_t>
__device__ __forceinline__ void
WriteBpropResults(
             int classes,
             scalar_t *gradInput,
             const outscalar_t *output,
             const outscalar_t *gradOutput,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {

  index_t offset = threadIdx.x;

  index_t last = classes % (ILP * blockDim.x);

  for (; offset < classes - last; offset += blockDim.x * ILP) {
    outscalar_t tmpOutput[ILP];
    outscalar_t tmpGradOutput[ILP];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmpOutput[j] = output[offset + j * blockDim.x];
      tmpGradOutput[j] = gradOutput[offset + j * blockDim.x];
    }

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      gradInput[offset + j * blockDim.x] = epilogue(tmpGradOutput[j], tmpOutput[j]);
    }
  }

  // Remainder - no ILP
  for (; offset < classes; offset += blockDim.x) {
    gradInput[offset] = epilogue(gradOutput[offset], output[offset]);
  }
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template <typename, typename, typename> class EpilogueWithMul>
__global__ void
cunn_SoftMaxForwardFast(outscalar_t *output, const scalar_t *input, int classes)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);

  // each block handles a sample in the mini-batch
  input += static_cast<int64_t>(blockIdx.x) * classes;
  output += static_cast<int64_t>(blockIdx.x) * classes;

  const int shift = ((uint64_t)input) % ALIGN_BYTES / sizeof(scalar_t);

  // find the max
  accscalar_t threadMax = ilpReduce<MaxFloat, ILP, scalar_t, accscalar_t>(
    shift, input, classes, MaxFloat<scalar_t, accscalar_t>(), -at::numeric_limits<accscalar_t>::max());
  accscalar_t max_k = blockReduceWarp<Max, accscalar_t>(sdata, threadMax,
    Max<accscalar_t>(), -at::numeric_limits<accscalar_t>::max());

  // reduce all values
  accscalar_t threadExp = ilpReduce<SumExpfFloat, ILP, scalar_t, accscalar_t>(
    shift, input, classes, SumExpfFloat<scalar_t, accscalar_t>(max_k), static_cast<accscalar_t>(0));
  accscalar_t sumAll = blockReduceWarpInverse<Add, accscalar_t>(sdata, threadExp,
    Add<accscalar_t>(), static_cast<accscalar_t>(0));

  EpilogueWithMul<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);

  for (int offset = threadIdx.x; offset < classes; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template <typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxForward(outscalar_t *output, const scalar_t *input, int classes)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);

  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += static_cast<int64_t>(blockIdx.x) * classes;
  output += static_cast<int64_t>(blockIdx.x) * classes;

  const int shift = ((uint64_t)input) % ALIGN_BYTES / sizeof(scalar_t);
  const int output_shift = ((uint64_t)output) % ALIGN_BYTES / sizeof(outscalar_t);

  // find the max
  accscalar_t threadMax = ilpReduce<MaxFloat, ILP, scalar_t, accscalar_t>(
    shift, input, classes, MaxFloat<scalar_t, accscalar_t>(), -at::numeric_limits<accscalar_t>::max());
  accscalar_t max_k = blockReduceWarp<Max, accscalar_t>(sdata, threadMax,
    Max<accscalar_t>(), -at::numeric_limits<accscalar_t>::max());

  // reduce all values
  accscalar_t threadExp = ilpReduce<SumExpFloat, ILP, scalar_t, accscalar_t>(
    shift, input, classes, SumExpFloat<scalar_t, accscalar_t>(max_k), static_cast<accscalar_t>(0));
  accscalar_t sumAll = blockReduceWarp<Add, accscalar_t>(sdata, threadExp,
    Add<accscalar_t>(), static_cast<accscalar_t>(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);

  if (shift == output_shift) {
    WriteFpropResultsVectorized<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, shift, input, output, epilogue);
  } else {
    WriteFpropResults<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, input, output, epilogue);
  }
}

template <typename scalar_t, typename accscalar_t, typename outscalar_t, template <typename, typename, typename> class Epilogue, typename index_t, int32_t reg_cnt>
__global__ void
cunn_SoftMaxForwardReg(outscalar_t *output, const scalar_t *input, index_t classes)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);

  scalar_t reg[reg_cnt];

  input += static_cast<int64_t>(blockIdx.x) * classes;
  output += static_cast<int64_t>(blockIdx.x) * classes;

  accscalar_t threadMax = -at::numeric_limits<accscalar_t>::max();
  accscalar_t threadExp = static_cast<accscalar_t>(0);

  // Load the elements from gmem into reg, and get the max for current thread.
  MaxFloat<scalar_t, accscalar_t> maxFunc;

  #pragma unroll
  for(int reg_idx = 0; reg_idx < reg_cnt; reg_idx ++){
    int offset = threadIdx.x + reg_idx * blockDim.x;
    if(offset < classes) {
      reg[reg_idx] = input[offset];
      threadMax = maxFunc(threadMax, reg[reg_idx]);
    }
  }

  // Reduce to the max for block
  accscalar_t max_k = blockReduceWarp<Max, accscalar_t>(sdata, threadMax,
    Max<accscalar_t>(), -at::numeric_limits<accscalar_t>::max());

  SumExpFloat<scalar_t, accscalar_t> sumExpFunc(max_k);
  // reduce all values
  #pragma unroll
  for(int reg_idx = 0; reg_idx < reg_cnt; reg_idx ++){
    int offset = threadIdx.x + reg_idx * blockDim.x;
    if(offset < classes) {
      threadExp = sumExpFunc(threadExp, reg[reg_idx]);
    }
  }
  accscalar_t sumAll = blockReduceWarp<Add, accscalar_t>(sdata, threadExp,
    Add<accscalar_t>(), static_cast<accscalar_t>(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);

  // Write back the value
  #pragma unroll
  for(int reg_idx = 0; reg_idx < reg_cnt; reg_idx ++){
    int offset = threadIdx.x + reg_idx * blockDim.x;
    if(offset < classes) {
      output[offset] = epilogue(reg[reg_idx]);
    }
  }
}


template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t,
  template <typename, typename, typename> class EpilogueWithMul, typename index_t = int32_t>
__global__ void
cunn_SoftMaxForwardGmem(outscalar_t *output, const scalar_t *input, index_t classes)
{
  // Each thread block processes a sample in the batch
  input += static_cast<int64_t>(blockIdx.x) * classes;
  output += static_cast<int64_t>(blockIdx.x) * classes;

  accscalar_t threadMax = -at::numeric_limits<accscalar_t>::max();
  accscalar_t threadExp = static_cast<accscalar_t>(0);

  // The first smem segment is used to cache input values and the last
  // segment is used for thread block reductions
  extern __shared__ unsigned char smem[];
  auto smem_reduction_cache = reinterpret_cast<accscalar_t*>(smem);

  using LoadT = at::native::memory::aligned_vector<scalar_t, ILP>;
  const LoadT* const input_vec_ptr = reinterpret_cast<const LoadT*>(input);

  // Do the first step in max calculation:
  MaxFloat<scalar_t, accscalar_t> maxFunc;
  for (index_t offset = threadIdx.x; offset * ILP < classes; offset += blockDim.x) {
    LoadT crnt_vec = input_vec_ptr[offset];
    #pragma unroll
    for (int i = 0; i < ILP; ++i) {
      threadMax = maxFunc(threadMax, crnt_vec.val[i]);
    }
  }

  accscalar_t max_k = blockReduceWarp<Max, accscalar_t>(smem_reduction_cache, threadMax,
    Max<accscalar_t>(), -at::numeric_limits<accscalar_t>::max());

  // Do the second step in sum exp calculation:
  SumExpfFloat<scalar_t, accscalar_t> sumExpFunc(max_k);
  for (index_t offset = threadIdx.x; offset * ILP < classes; offset += blockDim.x) {
    LoadT crnt_vec = input_vec_ptr[offset];
    #pragma unroll
    for (int i = 0; i < ILP; ++i) {
      threadExp = sumExpFunc(threadExp, crnt_vec.val[i]);
    }
  }

  accscalar_t sumAll = blockReduceWarpInverse<Add, accscalar_t>(smem_reduction_cache, threadExp,
    Add<accscalar_t>(), static_cast<accscalar_t>(0));

  EpilogueWithMul<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);

  using StoreT = at::native::memory::aligned_vector<outscalar_t, ILP>;
  StoreT* output_vec_ptr = reinterpret_cast<StoreT*>(output);
  for (index_t offset = threadIdx.x; offset * ILP < classes; offset += blockDim.x) {
    LoadT crnt_vec = input_vec_ptr[offset];
    StoreT out_vec;
    #pragma unroll
    for (int i = 0; i < ILP; ++i) {
      out_vec.val[i] = epilogue(crnt_vec.val[i]);
    }
    output_vec_ptr[offset] = out_vec;
  }
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t,
  template <typename, typename, typename> class Epilogue, typename index_t = int32_t>
__global__ void
cunn_SoftMaxForwardSmem(outscalar_t *output, const scalar_t *input, index_t classes)
{
  // Each thread block processes a sample in the batch
  input += static_cast<int64_t>(blockIdx.x) * classes;
  output += static_cast<int64_t>(blockIdx.x) * classes;

  accscalar_t threadMax = -at::numeric_limits<accscalar_t>::max();
  accscalar_t threadExp = static_cast<accscalar_t>(0);

  // The first smem segment is used to cache input values and the last
  // segment is used for thread block reductions
  extern __shared__ unsigned char smem[];
  auto smem_input_cache = reinterpret_cast<scalar_t*>(smem);
  auto smem_reduction_cache = reinterpret_cast<accscalar_t*>(smem +
    classes * sizeof(scalar_t));

  using LoadT = at::native::memory::aligned_vector<scalar_t, ILP>;
  const LoadT* const input_vec_ptr = reinterpret_cast<const LoadT*>(input);
  LoadT* const smem_input_cache_vec_ptr = reinterpret_cast<LoadT*>(smem_input_cache);

  // Download inputs to shared memory while doing the first step
  // in max calculation
  MaxFloat<scalar_t, accscalar_t> maxFunc;
  for (index_t offset = threadIdx.x; offset * ILP < classes; offset += blockDim.x) {
    LoadT crnt_vec = input_vec_ptr[offset];
    smem_input_cache_vec_ptr[offset] = crnt_vec;

    #pragma unroll
    for (int i = 0; i < ILP; ++i) {
      threadMax = maxFunc(threadMax, crnt_vec.val[i]);
    }
  }

  accscalar_t max_k = blockReduceWarp<Max, accscalar_t>(smem_reduction_cache, threadMax,
    Max<accscalar_t>(), -at::numeric_limits<accscalar_t>::max());

  // Reload input from shared memory to compute the sum. The previous
  // reduce has performed a __syncthreads() so the smem contents are populated.
  SumExpFloat<scalar_t, accscalar_t> sumExpFunc(max_k);
  for (index_t offset = threadIdx.x; offset * ILP < classes; offset += blockDim.x) {
    LoadT crnt_vec = smem_input_cache_vec_ptr[offset];

    #pragma unroll
    for (int i = 0; i < ILP; ++i) {
      threadExp = sumExpFunc(threadExp, crnt_vec.val[i]);
    }
  }

  accscalar_t sumAll = blockReduceWarp<Add, accscalar_t>(smem_reduction_cache, threadExp,
    Add<accscalar_t>(), static_cast<accscalar_t>(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);

  // Use vectorized stores to save the output
  using StoreT = at::native::memory::aligned_vector<outscalar_t, ILP>;
  StoreT* output_vec_ptr = reinterpret_cast<StoreT*>(output);
  for (index_t offset = threadIdx.x; offset * ILP < classes; offset += blockDim.x) {
    LoadT crnt_vec = smem_input_cache_vec_ptr[offset];
    StoreT out_vec;

    #pragma unroll
    for (int i = 0; i < ILP; ++i) {
      out_vec.val[i] = epilogue(crnt_vec.val[i]);
    }

    output_vec_ptr[offset] = out_vec;
  }
}

C10_DEVICE bool inline is_32bit_representable(const int64_t value) {
  return value < static_cast<int64_t>(std::numeric_limits<int32_t>::max());
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxBackward(scalar_t *gradInput, const outscalar_t *output, const outscalar_t *gradOutput, int64_t classes)
{
  using LoadT = at::native::memory::aligned_vector<scalar_t, ILP>;
  using StoreT = at::native::memory::aligned_vector<outscalar_t, ILP>;

  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  gradInput += static_cast<int64_t>(blockIdx.x) * classes;
  output += static_cast<int64_t>(blockIdx.x) * classes;
  gradOutput += static_cast<int64_t>(blockIdx.x) * classes;

  const int64_t shift = ((uint64_t)gradInput) % ALIGN_BYTES / sizeof(scalar_t);
  const int64_t output_shift = ((uint64_t)output) % ALIGN_BYTES / sizeof(outscalar_t);
  const int64_t grad_output_shift = ((uint64_t)gradOutput) % ALIGN_BYTES / sizeof(outscalar_t);

  const bool can_use_32bit_indexing = is_32bit_representable(shift) && is_32bit_representable(output_shift) && is_32bit_representable(grad_output_shift) && is_32bit_representable(classes);
  accscalar_t threadSum;
  if (can_use_32bit_indexing) {
    threadSum = ilpReduce<AddFloat, ILP, outscalar_t, accscalar_t, int32_t>(
        static_cast<int32_t>(grad_output_shift), gradOutput, classes, AddFloat<outscalar_t, accscalar_t>(), accscalar_t(0));
  } else {
    threadSum = ilpReduce<AddFloat, ILP, outscalar_t, accscalar_t, int64_t>(
        grad_output_shift, gradOutput, classes, AddFloat<outscalar_t, accscalar_t>(), accscalar_t(0));
  }
  accscalar_t sum_k = blockReduce<Add, accscalar_t>(
        sdata, threadSum, Add<accscalar_t>(), accscalar_t(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum_k);

  if (shift == output_shift && shift == grad_output_shift) {
    if (can_use_32bit_indexing) {
      WriteBpropResultsVectorized<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue, int32_t>(classes, static_cast<int32_t>(shift), gradInput, output, gradOutput, epilogue);
    } else {
      WriteBpropResultsVectorized<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue, int64_t>(classes, shift, gradInput, output, gradOutput, epilogue);
    }
  } else {
    if (can_use_32bit_indexing) {
      WriteBpropResults<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue, int32_t>(classes, gradInput, output, gradOutput, epilogue);
    } else {
      WriteBpropResults<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue, int64_t>(classes, gradInput, output, gradOutput, epilogue);
    }
  }
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxBackwardSmem(scalar_t *gradInput, const outscalar_t *output, const outscalar_t *gradOutput, int64_t classes)
{
  // The first smem segment is used to cache input values and the last
  // segment is used for thread block reductions
  extern __shared__ unsigned char smem[];
  auto smem_input_cache = reinterpret_cast<outscalar_t*>(smem);
  auto smem_reduction_cache = reinterpret_cast<accscalar_t*>(smem +
    classes * sizeof(outscalar_t));

  gradInput += static_cast<int64_t>(blockIdx.x) * classes;
  output += static_cast<int64_t>(blockIdx.x) * classes;
  gradOutput += static_cast<int64_t>(blockIdx.x) * classes;

  accscalar_t threadSum = 0;

  using LoadT = at::native::memory::aligned_vector<outscalar_t, ILP>;
  const LoadT* const gradOutput_vec_ptr = reinterpret_cast<const LoadT*>(gradOutput);
  LoadT* const smem_gradOutput_cache_vec_ptr = reinterpret_cast<LoadT*>(smem_input_cache);

  // Download inputs to shared memory while doing the first step
  // in sum calculation
  for (int32_t offset = threadIdx.x; offset * ILP < classes; offset += blockDim.x) {
    LoadT crnt_vec = gradOutput_vec_ptr[offset];
    smem_gradOutput_cache_vec_ptr[offset] = crnt_vec;

    #pragma unroll
    for (int i = 0; i < ILP; ++i) {
      threadSum = threadSum + crnt_vec.val[i];
    }
  }

  // We need a __syncthreads() here to be safe. However, blockReduceWarp's code
  // calls a __syncthreads() before reading shared memory so we are safe.

  accscalar_t sum_k = blockReduceWarp<Add, accscalar_t>(smem_reduction_cache, threadSum, Add<accscalar_t>(), accscalar_t(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum_k);

  // Use vectorized stores to save the output
  using StoreT = at::native::memory::aligned_vector<scalar_t, ILP>;
  StoreT* gradInput_vec_ptr = reinterpret_cast<StoreT*>(gradInput);
  const LoadT* const output_vec_ptr = reinterpret_cast<const LoadT*>(output);
  for (int32_t offset = threadIdx.x; offset * ILP < classes; offset += blockDim.x) {
    LoadT crnt_vec = smem_gradOutput_cache_vec_ptr[offset];
    LoadT crnt_out = output_vec_ptr[offset];
    StoreT out_vec;

    #pragma unroll
    for (int i = 0; i < ILP; ++i) {
      out_vec.val[i] = epilogue(crnt_vec.val[i], crnt_out.val[i]);
    }

    gradInput_vec_ptr[offset] = out_vec;
  }
}


 template<template<typename, typename, typename> class Epilogue,
          template<typename, typename, typename> class EpilogueWithMul, bool is_log_softmax, bool use_fast_softmax>
Tensor host_softmax(const Tensor & input_, const int64_t dim_, const bool half_to_float, const Tensor& output){
  if (half_to_float) {
    TORCH_CHECK(input_.scalar_type() == ScalarType::Half, "conversion is supported for Half type only");
  }
  auto input = input_.contiguous();
  static_assert(std::is_same_v<acc_type<at::Half, true>, float>, "accscalar_t for half should be float");
  if (input.dim() == 0) input = input.view(1);
  int64_t dim = maybe_wrap_dim(dim_, input.dim());
  TORCH_CHECK(dim >=0 && dim < input.dim(), "dim must be non-negative and less than input dimensions");
  int64_t outer_size = 1;
  int64_t dim_size = input.size(dim);

  if (input.numel() > 0) {
    int64_t inner_size = 1;
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    for (int64_t i = 0; i < dim; ++i)
      outer_size *= input.size(i);
    for (int64_t i = dim + 1; i < input.dim(); ++i)
      inner_size *= input.size(i);
    // This kernel spawns a block per each element in the batch.
    // XXX: it assumes that inner_size == 1

    if (inner_size == 1) {
      dim3 grid(outer_size);
      AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "host_softmax", [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        if (!half_to_float) {
          auto output_ptr = output.mutable_data_ptr<scalar_t>();
          auto input_ptr = input.const_data_ptr<scalar_t>();
          if (dim_size <= 2048 && dim_size*sizeof(scalar_t) <= 8192) {
            int64_t remaining = outer_size;
            int64_t chunk_size = (1L << 30L) / dim_size;
            while(remaining > 0) {
              dispatch_softmax_forward<scalar_t, scalar_t, accscalar_t, is_log_softmax, false>(
                output_ptr, input_ptr, dim_size, dim_size, std::min<int64_t>(remaining, chunk_size), nullptr/* not masked */);
              input_ptr += chunk_size * dim_size;
              output_ptr += chunk_size * dim_size;
              remaining -= chunk_size;
            }
          } else {
            constexpr int ILP = sizeof(float4) / sizeof(scalar_t);
            if constexpr (use_fast_softmax) {
              dim3 block(512);
              size_t smem_reduction_sz = block.x / at::cuda::warp_size() * sizeof(accscalar_t);
              if (dim_size % ILP == 0) {
                cunn_SoftMaxForwardGmem<ILP, scalar_t, accscalar_t, scalar_t, EpilogueWithMul>
                    <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
              } else {
                cunn_SoftMaxForwardFast<ILP, scalar_t, accscalar_t, scalar_t, EpilogueWithMul>
                    <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
              }
            } else {
              dim3 block = SoftMaxForward_getBlockSize(dim_size);
              size_t smem_reduction_sz = block.x / at::cuda::warp_size() * sizeof(accscalar_t);
              auto max_elements_per_smem = (at::cuda::getCurrentDeviceProperties()->sharedMemPerBlock -
                smem_reduction_sz) / sizeof(scalar_t);

              bool can_use_smem = static_cast<size_t>(dim_size) < max_elements_per_smem;
              can_use_smem &= !(reinterpret_cast<uintptr_t>(input_ptr) % ALIGN_BYTES);
              can_use_smem &= (!(reinterpret_cast<uintptr_t>(output_ptr) % ALIGN_BYTES));
              can_use_smem &= !(dim_size % ILP);

              int32_t potential_reg_cnt = potential_register_count(dim_size, block.x);
              if(potential_reg_cnt < 10){
                TORCH_INTERNAL_ASSERT(potential_reg_cnt > 0, "potential_reg_cnt for softmax with register should be greater than 0.");
                switch (potential_reg_cnt) {
                  // TODO(Wenqin): try to investigate why we couldn't use macro for below code,
                  // because it seems on MSVS, it seems the macro way didn't expand correct.
                  case 1:
                    cunn_SoftMaxForwardReg<scalar_t, accscalar_t, scalar_t, Epilogue, int64_t, 1>
                      <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
                    break;
                  case 2:
                    cunn_SoftMaxForwardReg<scalar_t, accscalar_t, scalar_t, Epilogue, int64_t, 2>
                      <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
                    break;
                  case 3:
                    cunn_SoftMaxForwardReg<scalar_t, accscalar_t, scalar_t, Epilogue, int64_t, 3>
                      <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
                    break;
                  case 4:
                    cunn_SoftMaxForwardReg<scalar_t, accscalar_t, scalar_t, Epilogue, int64_t, 4>
                      <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
                    break;
                  case 5:
                    cunn_SoftMaxForwardReg<scalar_t, accscalar_t, scalar_t, Epilogue, int64_t, 5>
                      <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
                    break;
                  case 6:
                    cunn_SoftMaxForwardReg<scalar_t, accscalar_t, scalar_t, Epilogue, int64_t, 6>
                      <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
                    break;
                  case 7:
                    cunn_SoftMaxForwardReg<scalar_t, accscalar_t, scalar_t, Epilogue, int64_t, 7>
                      <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
                    break;
                  case 8:
                    cunn_SoftMaxForwardReg<scalar_t, accscalar_t, scalar_t, Epilogue, int64_t, 8>
                      <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
                    break;
                  case 9:
                    cunn_SoftMaxForwardReg<scalar_t, accscalar_t, scalar_t, Epilogue, int64_t, 9>
                      <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
                    break;
                }
              } else if (can_use_smem) {
                size_t smem_sz = dim_size * sizeof(scalar_t) + smem_reduction_sz;
                cunn_SoftMaxForwardSmem<ILP, scalar_t, accscalar_t, scalar_t, Epilogue>
                  <<<grid, block, smem_sz, stream>>>(output_ptr, input_ptr, dim_size);
              } else {
                cunn_SoftMaxForward<ILP, scalar_t, accscalar_t, scalar_t, Epilogue>
                  <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
              }
            }

            C10_CUDA_KERNEL_LAUNCH_CHECK();
          }
        } else {
          auto output_ptr = output.mutable_data_ptr<accscalar_t>();
          auto input_ptr = input.const_data_ptr<scalar_t>();
          if (dim_size <= 1024 && dim_size*sizeof(scalar_t) <= 4096) {
            int64_t remaining = outer_size;
            int64_t chunk_size = (1<<30) / dim_size;
            while(remaining > 0) {
              dispatch_softmax_forward<scalar_t, accscalar_t, accscalar_t, is_log_softmax, false>(
                  output_ptr, input_ptr, dim_size, dim_size, std::min<int64_t>(remaining, chunk_size), nullptr/* not masked */);
              input_ptr += chunk_size * dim_size;
              output_ptr += chunk_size * dim_size;
              remaining -= chunk_size;
            }
          } else {
            constexpr int ILP = sizeof(float4) / sizeof(scalar_t);
            if constexpr (use_fast_softmax) {
              dim3 block(512);
              size_t smem_reduction_sz = block.x / at::cuda::warp_size() * sizeof(accscalar_t);
              if (dim_size % ILP == 0) {
                cunn_SoftMaxForwardGmem<ILP, scalar_t, accscalar_t, accscalar_t, EpilogueWithMul>
                    <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
              } else {
                cunn_SoftMaxForwardFast<ILP, scalar_t, accscalar_t, accscalar_t, EpilogueWithMul>
                    <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
              }
            } else {
              dim3 block = SoftMaxForward_getBlockSize(dim_size);
              size_t smem_reduction_sz = block.x / at::cuda::warp_size() * sizeof(accscalar_t);
              auto max_elements_per_smem = (at::cuda::getCurrentDeviceProperties()->sharedMemPerBlock -
                smem_reduction_sz) / sizeof(scalar_t);

              bool can_use_smem = static_cast<size_t>(dim_size) < max_elements_per_smem;
              can_use_smem &= !(reinterpret_cast<uintptr_t>(input_ptr) % ALIGN_BYTES);
              can_use_smem &= (!(reinterpret_cast<uintptr_t>(output_ptr) % ALIGN_BYTES));
              can_use_smem &= !(dim_size % ILP);

              if (can_use_smem) {
                size_t smem_sz = dim_size * sizeof(scalar_t) + smem_reduction_sz;
                cunn_SoftMaxForwardSmem<ILP, scalar_t, accscalar_t, accscalar_t, Epilogue>
                  <<<grid, block, smem_sz, stream>>>(output_ptr, input_ptr, dim_size);
              } else {
                cunn_SoftMaxForward<ILP, scalar_t, accscalar_t, accscalar_t, Epilogue>
                  <<<grid, block, smem_reduction_sz, stream>>>(output_ptr, input_ptr, dim_size);
              }
            }

            C10_CUDA_KERNEL_LAUNCH_CHECK();
          }
        }
      });
    // This kernel runs in a 2D grid, where each application along y dimension has a fixed
    // outer_size, and runs in parallel over inner_size. Dimension x is parallel over outer_size.
    // Reductions over dim are done in a single-threaded manner.
    } else {
      uint32_t smem_size;
      dim3 grid, block;
      AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "host_softmax", [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        AT_DISPATCH_INDEX_TYPES(
            at::native::canUse32BitIndexMath(input, INT_MAX) ? ScalarType::Int : ScalarType::Long,
        "host_softmax_launcher", [&] {
            if (!half_to_float) {
                SpatialSoftMax_getLaunchSizes<accscalar_t>(
                    &cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, scalar_t, index_t, Epilogue>,
                    outer_size, dim_size, inner_size,
                    grid, block, smem_size);
                cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, scalar_t, index_t, Epilogue>
                  <<<grid, block, smem_size, stream>>>(
                  output.mutable_data_ptr<scalar_t>(), input.const_data_ptr<scalar_t>(), outer_size, dim_size, inner_size);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
            } else {
                SpatialSoftMax_getLaunchSizes<accscalar_t>(
                    &cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, accscalar_t, index_t, Epilogue>,
                    outer_size, dim_size, inner_size,
                    grid, block, smem_size);
                cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, accscalar_t, index_t, Epilogue>
                  <<<grid, block, smem_size, stream>>>(
                  output.mutable_data_ptr<accscalar_t>(), input.const_data_ptr<scalar_t>(), outer_size, dim_size, inner_size);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
            }
         });
      });
    }
  }
  return output;
}

template<typename input_t, typename output_t, typename accscalar_t, template<typename, typename, typename> class Epilogue>
void dispatch_host_softmax_backward(int64_t dim_size, dim3 grid, Tensor &grad, Tensor &output, const Tensor &gI) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  constexpr int ILP = sizeof(float4) / sizeof(output_t);
  dim3 block = SoftMax_getBlockSize(ILP, dim_size);

  size_t smem_reduction_sz = block.x / at::cuda::warp_size() * sizeof(accscalar_t);
  auto max_elements_per_smem = (at::cuda::getCurrentDeviceProperties()->sharedMemPerBlock -
    smem_reduction_sz) / sizeof(output_t);
  bool can_use_smem = static_cast<size_t>(dim_size) < max_elements_per_smem;
  can_use_smem &= (!(reinterpret_cast<uintptr_t>(gI.const_data_ptr<input_t>()) % ALIGN_BYTES));
  can_use_smem &= (!(reinterpret_cast<uintptr_t>(output.const_data_ptr<output_t>()) % ALIGN_BYTES));
  can_use_smem &= !(reinterpret_cast<uintptr_t>(grad.const_data_ptr<output_t>()) % ALIGN_BYTES);
  can_use_smem &= !(dim_size % ILP);
  // This should not be needed on current generation GPUs because the size of shared memory is so low.
  // But we add this check to be defensive and future-proof just in case shared memory size goes up
  // to be so large as to requires 64-bits of addressing.
  can_use_smem &= (dim_size < std::numeric_limits<int32_t>::max());

  if (can_use_smem) {
    size_t smem_sz = dim_size * sizeof(output_t) + smem_reduction_sz;
    cunn_SoftMaxBackwardSmem<ILP, input_t, accscalar_t, output_t, Epilogue>
    <<<grid, block, smem_sz, stream>>>(
      gI.mutable_data_ptr<input_t>(), output.const_data_ptr<output_t>(), grad.const_data_ptr<output_t>(), dim_size);
  } else {
    cunn_SoftMaxBackward<ILP, input_t, accscalar_t, output_t, Epilogue>
    <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
        gI.mutable_data_ptr<input_t>(), output.const_data_ptr<output_t>(), grad.const_data_ptr<output_t>(), dim_size
      );
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<template<typename, typename, typename> class Epilogue, bool is_log_softmax>
void host_softmax_backward(const Tensor &grad_, const Tensor &output_, int64_t dim_, bool half_to_float, const Tensor &gI){
  int64_t dim = maybe_wrap_dim(dim_, grad_.dim());
  if (grad_.numel() == 0) {
    return;
  }
  auto grad = grad_.contiguous();
  static_assert(std::is_same_v<acc_type<at::Half, true>, float>, "accscalar_t for half should be float");
  if (grad.dim() == 0) grad = grad.view(1);
  TORCH_CHECK(dim >=0 && dim < grad.dim(), "dim must be non-negative and less than input dimensions");
  auto output = output_.contiguous();
  if (output.dim() == 0) output = output.view(1);
  int64_t outer_size = 1;
  int64_t dim_size = output.size(dim);
  int64_t inner_size = 1;
  for (int64_t i = 0; i < dim; ++i)
    outer_size *= output.size(i);
  for (int64_t i = dim + 1; i < output.dim(); ++i)
    inner_size *= output.size(i);
// See descriptions of kernels above.
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  if (inner_size == 1) {
    dim3 grid(outer_size);
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, gI.scalar_type(), "host_softmax_backward", [&] {
    using accscalar_t = acc_type<scalar_t, true>;
    if (!half_to_float) {
      if (dim_size <= 1024 && dim_size*sizeof(scalar_t) <= 4096) {
        auto gI_ptr = gI.mutable_data_ptr<scalar_t>();
        auto grad_ptr = grad.const_data_ptr<scalar_t>();
        auto output_ptr = output.const_data_ptr<scalar_t>();
        int64_t remaining = outer_size;
        int64_t chunk_size = (1<<30) / dim_size;
        while(remaining > 0) {
          dispatch_softmax_backward<scalar_t, scalar_t, accscalar_t, is_log_softmax, false /* masked_softmax */>(
            gI_ptr, grad_ptr, output_ptr, dim_size, dim_size, std::min<int64_t>(remaining, chunk_size));
          gI_ptr += chunk_size * dim_size;
          grad_ptr += chunk_size * dim_size;
          output_ptr += chunk_size * dim_size;
          remaining -= chunk_size;
        }
      } else {
        dispatch_host_softmax_backward<scalar_t, scalar_t, accscalar_t, Epilogue>(dim_size, grid, grad, output, gI);
      }
    } else {
      if (dim_size <= 1024 && dim_size*sizeof(scalar_t) <= 4096) {
        auto gI_ptr = gI.mutable_data_ptr<scalar_t>();
        auto grad_ptr = grad.const_data_ptr<accscalar_t>();
        auto output_ptr = output.const_data_ptr<accscalar_t>();
        int64_t remaining = outer_size;
        int64_t chunk_size = (1<<30) / dim_size;
        while(remaining > 0) {
          dispatch_softmax_backward<accscalar_t, scalar_t, accscalar_t, is_log_softmax, false /* masked_softmax */>(
            gI_ptr, grad_ptr, output_ptr, dim_size, dim_size, std::min<int64_t>(remaining, chunk_size));
          gI_ptr += chunk_size * dim_size;
          grad_ptr += chunk_size * dim_size;
          output_ptr += chunk_size * dim_size;
          remaining -= chunk_size;
        }
      } else {
        dispatch_host_softmax_backward<scalar_t, accscalar_t, accscalar_t, Epilogue>(dim_size, grid, grad, output, gI);
      }
    }
    });
  } else {
    uint32_t smem_size;
    dim3 grid, block;
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, gI.scalar_type(), "host_softmax_backward", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      if (!half_to_float) {
          SpatialSoftMax_getLaunchSizes<accscalar_t>(
              &cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, scalar_t, Epilogue>,
              outer_size, dim_size, inner_size,
              grid, block, smem_size);

          cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, scalar_t, Epilogue>
            <<<grid, block, smem_size, stream>>>(
              gI.mutable_data_ptr<scalar_t>(), output.const_data_ptr<scalar_t>(), grad.const_data_ptr<scalar_t>(),
              outer_size, dim_size, inner_size
          );
          C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
          SpatialSoftMax_getLaunchSizes<accscalar_t>(
              &cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, accscalar_t, Epilogue>,
              outer_size, dim_size, inner_size,
              grid, block, smem_size);

          cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, accscalar_t, Epilogue>
            <<<grid, block, smem_size, stream>>>(
              gI.mutable_data_ptr<scalar_t>(), output.const_data_ptr<accscalar_t>(), grad.const_data_ptr<accscalar_t>(),
              outer_size, dim_size, inner_size
          );
          C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
}
}

TORCH_IMPL_FUNC(log_softmax_cuda_out) (
  const Tensor &input,
  const int64_t dim,
  const bool half_to_float,
  const Tensor &output) {
  host_softmax<LogSoftMaxForwardEpilogue, LogSoftMaxForwardEpilogue, true, false>(input, dim, half_to_float, output);
}

TORCH_IMPL_FUNC(log_softmax_backward_cuda_out) (
  const Tensor& grad,
  const Tensor& output,
  int64_t dim,
  ScalarType input_dtype,
  const Tensor& grad_input) {
  bool half_to_float = grad.scalar_type() != input_dtype;
  if (half_to_float) {
    TORCH_CHECK(
        (grad.scalar_type() == ScalarType::Float &&
         input_dtype == ScalarType::Half),
        "expected input and grad types to match, or input to be at::Half and grad to be at::Float");
  }
  host_softmax_backward<LogSoftMaxBackwardEpilogue, true>(grad, output, dim, half_to_float, grad_input);
}

TORCH_IMPL_FUNC(softmax_cuda_out) (
  const Tensor &input,
  const int64_t dim,
  const bool half_to_float,
  const Tensor &output) {
#if defined(USE_ROCM)
   host_softmax<SoftMaxForwardEpilogue, SoftMaxForwardWithMulEpilogue, false, true>(input, dim, half_to_float, output);
 #else
   host_softmax<SoftMaxForwardEpilogue, SoftMaxForwardWithMulEpilogue, false, false>(input, dim, half_to_float, output);
 #endif
}

TORCH_IMPL_FUNC(softmax_backward_cuda_out)
(const Tensor& grad,
 const Tensor& output,
 int64_t dim,
 ScalarType input_dtype,
 const Tensor& grad_input) {
  bool half_to_float = grad.scalar_type() != input_dtype;
  if (half_to_float) {
    TORCH_CHECK(
        (grad.scalar_type() == ScalarType::Float &&
         input_dtype == ScalarType::Half),
        "expected input and grad types to match, or input to be at::Half and grad to be at::Float");
  }
  Tensor tmp = grad * output;
  host_softmax_backward<SoftMaxBackwardEpilogue, false>(tmp, output, dim, half_to_float, grad_input);
}

Tensor masked_softmax_cuda(const Tensor& input_, const Tensor& mask_, const std::optional<int64_t> dim_, const std::optional<int64_t> mask_type_) {
  Tensor output = at::empty_like(input_, input_.options());
  TORCH_CHECK(mask_.scalar_type() == ScalarType::Bool, "Mask should be a boolean tensor");

  TORCH_CHECK(mask_type_.has_value(), "Mask Type should be defined");
  int64_t mask_type = mask_type_.value();
  TORCH_CHECK((mask_type == 0) || (mask_type == 1) || (mask_type == 2), "Mask Type should be 0 (src_mask), 1 (src_key_padding_mask), or 2 (default_mask)");

  // If input is [B, H, T, T] and mask is [B, T]
  // we have special fast kernel
  // mask_type == 1 => mask_ is a src_key_padding_mask
  bool is_BxT_mask = (mask_type == 1) && (input_.dim() == 4 && mask_.dim() == 2 && input_.size(0) == mask_.size(0) && input_.size(2) == mask_.size(1) && input_.size(3) == mask_.size(1));

  // If input is [B, H, T, T] and mask is [T, T]
  // expand mask to [B, H, T, T] and treat it like regular mask
  // TODO We should have special fast kernel for TxT mask as well
  // mask_type == 0 => mask_ is a src_mask
  bool is_TxT_mask = (mask_type == 0) && input_.dim() == 4 && mask_.dim() == 2 && input_.size(3) == mask_.size(1) && input_.size(2) == mask_.size(0) && mask_.size(0) == mask_.size(1);
  // If mask_type == 2, then mask_.sizes() must equal input_.sizes()
  TORCH_CHECK(mask_.sizes() == input_.sizes() || is_BxT_mask || is_TxT_mask, "Mask shape should match input. mask: ", mask_.sizes(), " input: ", input_.sizes());

  auto input = input_.dim() == 0 ? input_.view(1) : input_;
  auto mask = mask_.dim() == 0 ? mask_.view(1) : mask_;
  if (is_TxT_mask) {
    mask = mask.expand(input.sizes());
  }
  int64_t dim = dim_.has_value() ? dim_.value() : input.dim() - 1;

  int softmax_elements = input.size(dim);
  // Persistent softmax is only supported when all of the conditions are held:
  //     1) softmax_elements <= 1024
  //     2) softmax_elements * input.element_size() <= 4096
  //     3) mask.is_contiguous()
  //     4) dim == input.dim() - 1
  // Otherwise, we fallback to vanilla softmax (where we do not support transformer_mask since converting the mask is expensive)
  if (softmax_elements > 1024 || softmax_elements * input.element_size() > 4096 || !mask.is_contiguous() || dim < input.dim()-1) {
    if (is_BxT_mask) {
      mask = mask.view({mask_.size(0), 1, 1, mask_.size(1)}).expand(input.sizes());
    }
    AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      input.scalar_type(),
      "masked_softmax",
      [&] {
        output = at::softmax(input.masked_fill(mask, -std::numeric_limits<scalar_t>::infinity()), dim);
      });
    return output;
  }
  int batch_count = input.numel() / softmax_elements;
  int chunk_size = input.numel() / input.size(0);
  if (is_BxT_mask) {
    // Only support when num_heads is even in transformer
    TORCH_CHECK(input.size(1) % 2 == 0, "Only support when num_heads is even in transformer");
    AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      input.scalar_type(),
      "masked_softmax",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        dispatch_softmax_forward<scalar_t, scalar_t, accscalar_t, false/* is_log_softmax */, true/* is_masked */>(
          output.mutable_data_ptr<scalar_t>(),    // dst
          input.const_data_ptr<scalar_t>(),       // src
          softmax_elements,
          softmax_elements,
          batch_count,
          mask.const_data_ptr<bool>(),
          chunk_size,
          true // is_transformer_mask
        );
      });

  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      input.scalar_type(),
      "masked_softmax",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        dispatch_softmax_forward<scalar_t, scalar_t, accscalar_t, false/* is_log_softmax */, true/* is_masked */>(
          output.mutable_data_ptr<scalar_t>(),    // dst
          input.const_data_ptr<scalar_t>(),       // src
          softmax_elements,
          softmax_elements,
          batch_count,
          mask.const_data_ptr<bool>()
        );
      });
  }
  return output;
}

Tensor masked_softmax_backward_cuda(
    const Tensor& grad_,
    const Tensor& output_,
    const Tensor& mask_,
    const std::optional<int64_t> dim_) {
  Tensor grad_input = at::empty_like(grad_, grad_.options());
  if (grad_.numel() == 0) {
    return grad_input;
  }

  auto grad = grad_.contiguous();
  auto output = output_.contiguous();
  auto mask = mask_.contiguous();
  int64_t dim = dim_.has_value() ? maybe_wrap_dim(dim_.value(), output.dim()) : output.dim() - 1;

  grad = grad.dim() == 0 ? grad.view(1) : grad;
  mask = mask.dim() == 0 ? mask.view(1) : mask;
  output = output.dim() == 0 ? output.view(1) : output;

  TORCH_CHECK(dim >=0 && dim < grad.dim(), "dim must be non-negative and less than input dimensions");
  TORCH_CHECK(grad.sizes() == mask.sizes(), "Mask shape should match grad shape");
  TORCH_CHECK(mask.scalar_type() == ScalarType::Bool, "Mask should be a boolean tensor");

  int softmax_elements = output.size(dim);
  int64_t batch_count = grad.numel() / softmax_elements;

  if (softmax_elements > 1024 || softmax_elements * grad.element_size() > 4096 || dim < grad.dim()-1) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      grad_input.scalar_type(),
      "masked_softmax_backward",
      [&] {
        grad_input = at::_softmax_backward_data(
          grad,
          output.masked_fill(mask, 0),
          dim,
          grad.scalar_type()
        );
      });
  } else {
    grad = grad * output;
    AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      grad_input.scalar_type(),
      "masked_softmax_backward",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        dispatch_softmax_backward<scalar_t, scalar_t, accscalar_t, false, true /* masked_softmax */>(
          grad_input.mutable_data_ptr<scalar_t>(),  // gI_ptr
          grad.const_data_ptr<scalar_t>(),  // grad_ptr
          output.const_data_ptr<scalar_t>(),  // output_ptr
          softmax_elements,  // softmax_elements
          softmax_elements,   // softmax_elements_stride
          batch_count,  // batch_count
          mask.const_data_ptr<bool>()  /* not masked */
        );
      });
  }
  return grad_input;
}

} // namespace at::native
