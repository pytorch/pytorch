#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if 0 && AT_USE_JITERATOR()
constexpr char tanh_name[] = "tanh_impl";
#endif

void tanh_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
    // Disabled due to accuracy issues
#if 0 && AT_USE_JITERATOR()
    static const auto tanh_string = jiterator_stringify(
        template <typename T> T tanh_impl(T a) { return std::tanh(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "tanh_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/tanh_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, tanh_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "tanh_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::tanh(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        ScalarType::Half,
        ScalarType::BFloat16,
        common_dtype,
        "tanh_cuda",
        [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::tanh(a);
          });
        });
  }
}

REGISTER_DISPATCH(tanh_stub, &tanh_kernel_cuda)

} // namespace at::native
