#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Reduce.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/SharedReduceOps.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/ReduceOps.h>

namespace at::native {

template <typename scalar_t, typename out_t=scalar_t>
void std_var_kernel_impl(TensorIterator& iter, double correction, bool take_sqrt) {
  // reducing unrolling factor to 2 for welford kernel
  // This is necessary to lower register usage that leads to register spills.
  using accscalar_t = at::acc_type<scalar_t, true>;
  using ops_t = WelfordOps<scalar_t, accscalar_t, int32_t, thrust::pair<out_t, out_t>>;
  ops_t ops(static_cast<accscalar_t>(correction), take_sqrt);
  gpu_reduce_kernel<scalar_t, out_t, 2>(iter, ops, typename ops_t::acc_t{});
}

static void std_var_kernel_cuda(TensorIterator& iter, double correction, bool take_sqrt) {
  const auto input_dtype = iter.input_dtype();
  if (input_dtype == kHalf && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    std_var_kernel_impl<at::Half, float>(iter, correction, take_sqrt);
  } else if (input_dtype == kBFloat16 && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    std_var_kernel_impl<at::BFloat16, float>(iter, correction, take_sqrt);
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
                                    iter.dtype(), "std_cuda", [&]() {
      std_var_kernel_impl<scalar_t>(iter, correction, take_sqrt);
    });
  }
}

template <typename scalar_t, typename acc_t=scalar_t, typename out_t=scalar_t>
void mean_kernel_impl(TensorIterator& iter) {
  //  returns acc_t for all non-complex dtypes and returns T for c10::complex<T>
  constexpr bool is_16_bits = sizeof(scalar_t) == 2;
  using factor_t = typename c10::scalar_value_type<acc_t>::type;
  factor_t factor = static_cast<factor_t>(iter.num_output_elements()) / iter.numel();
  if constexpr (is_16_bits) {
    gpu_reduce_kernel<scalar_t, out_t, /*vt0=*/4, /*input_vec_size=*/8>(iter, MeanOps<scalar_t, acc_t, factor_t, out_t> {factor});
  } else {
    gpu_reduce_kernel<scalar_t, out_t>(iter, MeanOps<scalar_t, acc_t, factor_t, out_t> {factor});
  }
}

static void mean_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == kHalf) {
    mean_kernel_impl<at::Half, float>(iter);
  } else if (iter.dtype(1) == kHalf && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    mean_kernel_impl<at::Half, float, float>(iter);
  } else if(iter.dtype() == kBFloat16) {
    mean_kernel_impl<at::BFloat16, float>(iter);
  } else if (iter.dtype(1) == kBFloat16 && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    mean_kernel_impl<at::BFloat16, float, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX(iter.dtype(), "mean_cuda", [&]() {
      mean_kernel_impl<scalar_t>(iter);
    });
  }
}

REGISTER_DISPATCH(std_var_stub, &std_var_kernel_cuda)
REGISTER_DISPATCH(mean_stub, &mean_kernel_cuda)

} // namespace at::native
