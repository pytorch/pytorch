#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/ForeachUtils.h>
#include <ATen/native/cuda/ForeachFunctors.cuh>
#include <ATen/native/cuda/ForeachMinMaxFunctors.cuh>
#include <functional>
#include <type_traits>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_foreach_add_native.h>
#include <ATen/ops/_foreach_clamp_max_native.h>
#include <ATen/ops/_foreach_clamp_min_native.h>
#include <ATen/ops/_foreach_copy_native.h>
#include <ATen/ops/_foreach_div_native.h>
#include <ATen/ops/_foreach_mul_native.h>
#include <ATen/ops/_foreach_pow_native.h>
#include <ATen/ops/_foreach_sub_native.h>

#include <ATen/ops/empty_like_native.h>
#endif

namespace at::native {

template <typename T, template <class> class Op>
std::vector<Tensor> foreach_tensor_list_op(
    TensorList tensors1,
    TensorList tensors2,
    const Scalar& alpha = 1) {
  std::vector<std::vector<at::Tensor>> tensor_lists;
  std::vector<at::Tensor> vec_res;
  vec_res.reserve(tensors1.size());
  for (const auto& t : tensors1) {
    vec_res.emplace_back(at::native::empty_like(t));
  }

  tensor_lists.emplace_back(tensors1.vec());
  tensor_lists.emplace_back(tensors2.vec());
  tensor_lists.emplace_back(std::move(vec_res));

  using opmath_t = at::opmath_type<T>;
  multi_tensor_apply<3>(
      tensor_lists,
      BinaryOpListAlphaFunctor<
          T,
          /* depth */ 3,
          /* r_args_depth */ 2,
          /* res_arg_index */ 2>(),
      Op<opmath_t>(),
      alpha.to<opmath_t>());

  return tensor_lists[2];
}

template <typename T, template <class> class Op>
void foreach_tensor_list_op_(
    TensorList tensors1,
    TensorList tensors2,
    const Scalar& alpha = 1) {
  std::vector<std::vector<at::Tensor>> tensor_lists;
  tensor_lists.emplace_back(tensors1.vec());
  tensor_lists.emplace_back(tensors2.vec());

  using opmath_t = at::opmath_type<T>;
  multi_tensor_apply<2>(
      tensor_lists,
      BinaryOpListAlphaFunctor<
          T,
          /* depth */ 2,
          /* r_args_depth */ 2,
          /* res_arg_index */ 0>(),
      Op<opmath_t>(),
      alpha.to<opmath_t>());
  increment_version(tensors1);
}

template <template <class> class Op>
std::vector<Tensor> all_types_complex_bool_half_bfloat16(
    TensorList tensors1,
    TensorList tensors2,
    const Scalar& alpha = 1) {
  return AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      kBool,
      kBFloat16,
      kHalf,
      tensors1[0].scalar_type(),
      "foreach_binary_op_list_cuda",
      [&]() {
        return foreach_tensor_list_op<scalar_t, Op>(tensors1, tensors2, alpha);
      });
}

template <template <class> class Op>
void all_types_complex_bool_half_bfloat16_(
    TensorList tensors1,
    TensorList tensors2,
    const Scalar& alpha = 1) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      kBool,
      kBFloat16,
      kHalf,
      tensors1[0].scalar_type(),
      "foreach_binary_op_list_cuda_",
      [&]() {
        foreach_tensor_list_op_<scalar_t, Op>(tensors1, tensors2, alpha);
      });
}

template <template <class> class Op>
std::vector<Tensor> all_types_half_bfloat16(
    TensorList tensors1,
    TensorList tensors2,
    const Scalar& alpha = 1) {
  return AT_DISPATCH_ALL_TYPES_AND2(
      kBFloat16,
      kHalf,
      tensors1[0].scalar_type(),
      "foreach_binary_op_list_cuda",
      [&]() {
        return foreach_tensor_list_op<scalar_t, Op>(tensors1, tensors2, alpha);
      });
}

template <template <class> class Op>
void all_types_complex_half_bfloat16_(
    TensorList tensors1,
    TensorList tensors2,
    const Scalar& alpha = 1) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
      kBFloat16,
      kHalf,
      tensors1[0].scalar_type(),
      "foreach_binary_op_list_cuda_",
      [&]() {
        foreach_tensor_list_op_<scalar_t, Op>(tensors1, tensors2, alpha);
      });
}

template <template <class> class Op>
void all_types_half_bfloat16_(
    TensorList tensors1,
    TensorList tensors2,
    const Scalar& alpha = 1) {
  AT_DISPATCH_ALL_TYPES_AND2(
      kBFloat16,
      kHalf,
      tensors1[0].scalar_type(),
      "foreach_binary_op_list_cuda_",
      [&]() {
        foreach_tensor_list_op_<scalar_t, Op>(tensors1, tensors2, alpha);
      });
}

template <template <class> class Op>
std::vector<Tensor> all_types_complex_half_bfloat16(
    TensorList tensors1,
    TensorList tensors2,
    const Scalar& alpha = 1) {
  return AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
      kBFloat16,
      kHalf,
      tensors1[0].scalar_type(),
      "foreach_binary_op_list_cuda",
      [&]() {
        return foreach_tensor_list_op<scalar_t, Op>(tensors1, tensors2, alpha);
      });
}

#define FOREACH_BINARY_OP_LIST(FUNCTION, NAME, OP, DIVISION_OP)     \
  void foreach_tensor_##NAME##_list_kernel_cuda_(                   \
      TensorList tensors1, TensorList tensors2) {                   \
    check_foreach_api_restrictions(tensors1, tensors2);             \
    if (!can_use_fast_route(tensors1, tensors2, DIVISION_OP)) {     \
      return at::native::foreach_tensor_##NAME##_list_kernel_slow_( \
          tensors1, tensors2);                                      \
    }                                                               \
                                                                    \
    FUNCTION##_<OP>(tensors1, tensors2);                            \
  }                                                                 \
                                                                    \
  std::vector<Tensor> foreach_tensor_##NAME##_list_kernel_cuda(     \
      TensorList tensors1, TensorList tensors2) {                   \
    check_foreach_api_restrictions(tensors1, tensors2);             \
    if (!can_use_fast_route(tensors1, tensors2, DIVISION_OP)) {     \
      return at::native::foreach_tensor_##NAME##_list_kernel_slow(  \
          tensors1, tensors2);                                      \
    }                                                               \
                                                                    \
    return FUNCTION<OP>(tensors1, tensors2);                        \
  }

#define FOREACH_BINARY_OP_LIST_ALPHA(FUNCTION, NAME, OP)               \
  void foreach_tensor_##NAME##_list_kernel_cuda_(                      \
      TensorList tensors1, TensorList tensors2, const Scalar& alpha) { \
    check_foreach_api_restrictions(tensors1, tensors2);                \
    if (!can_use_fast_route({tensors1, tensors2}, alpha)) {            \
      return at::native::foreach_tensor_##NAME##_list_kernel_slow_(    \
          tensors1, tensors2, alpha);                                  \
    }                                                                  \
                                                                       \
    FUNCTION##_<OP>(tensors1, tensors2, alpha);                        \
  }                                                                    \
                                                                       \
  std::vector<Tensor> foreach_tensor_##NAME##_list_kernel_cuda(        \
      TensorList tensors1, TensorList tensors2, const Scalar& alpha) { \
    check_foreach_api_restrictions(tensors1, tensors2);                \
    if (!can_use_fast_route({tensors1, tensors2}, alpha)) {            \
      return at::native::foreach_tensor_##NAME##_list_kernel_slow(     \
          tensors1, tensors2, alpha);                                  \
    }                                                                  \
                                                                       \
    return FUNCTION<OP>(tensors1, tensors2, alpha);                    \
  }

FOREACH_BINARY_OP_LIST_ALPHA(
    all_types_complex_bool_half_bfloat16,
    add,
    std::plus);
FOREACH_BINARY_OP_LIST_ALPHA(
    all_types_complex_bool_half_bfloat16,
    sub,
    std::minus);
FOREACH_BINARY_OP_LIST(
    all_types_complex_bool_half_bfloat16,
    mul,
    std::multiplies,
    /*division_op*/ false);
FOREACH_BINARY_OP_LIST(
    all_types_complex_bool_half_bfloat16,
    div,
    std::divides,
    /*division_op*/ true);
// NOTE(crcrpar): `all_types_half_bfloat16` does not cover bool, so temporarily
// set `division_op` to true.
FOREACH_BINARY_OP_LIST(
    all_types_half_bfloat16,
    clamp_max,
    minimum,
    /*division_op*/ true);
FOREACH_BINARY_OP_LIST(
    all_types_half_bfloat16,
    clamp_min,
    maximum,
    /*division_op*/ true);
// NOTE(crcrpar): [Why is foreach_pow's division_op=true?]
// To push integer inputs to slow path. This is because with integer type inputs
// the fast path behaves differently from the slow one. Need to investigate
// later.
FOREACH_BINARY_OP_LIST(
    all_types_complex_half_bfloat16,
    pow,
    power_functor,
    /*division_op*/ true);

template <typename dst_t, typename src_t = dst_t>
struct Copy {
  __device__ __forceinline__ dst_t operator()(const src_t& x) {
    return static_cast<dst_t>(x);
  }
};

template <typename dst_t>
struct Copy<dst_t, c10::complex<double>> {
  __device__ __forceinline__ dst_t operator()(const c10::complex<double>& x) {
    if constexpr (!(std::is_same_v<dst_t, c10::complex<double>> ||
                    std::is_same_v<dst_t, c10::complex<float>>)) {
      return static_cast<dst_t>(x.real());
    } else {
      return static_cast<dst_t>(x);
    }
  }
};

template <typename dst_t>
struct Copy<dst_t, c10::complex<float>> {
  __device__ __forceinline__ dst_t operator()(const c10::complex<float>& x) {
    if constexpr (!(std::is_same_v<dst_t, c10::complex<double>> ||
                    std::is_same_v<dst_t, c10::complex<float>>)) {
      return static_cast<dst_t>(x.real());
    } else {
      return static_cast<dst_t>(x);
    }
  }
};

#define AT_DISPATCH_SOURCE_TYPES(TYPE, NAME, ...)                                                \
  AT_DISPATCH_SWITCH(                                                                            \
      TYPE,                                                                                      \
      NAME,                                                                                      \
      AT_PRIVATE_CASE_TYPE_USING_HINT(                                                           \
          at::ScalarType::Byte,                                                                  \
          src_t,                                                                                 \
          __VA_ARGS__) AT_PRIVATE_CASE_TYPE_USING_HINT(at::ScalarType::Char, src_t, __VA_ARGS__) \
          AT_PRIVATE_CASE_TYPE_USING_HINT(                                                       \
              at::ScalarType::Long, src_t, __VA_ARGS__)                                          \
              AT_PRIVATE_CASE_TYPE_USING_HINT(                                                   \
                  at::ScalarType::Short, src_t, __VA_ARGS__)                                     \
                  AT_PRIVATE_CASE_TYPE_USING_HINT(                                               \
                      at::ScalarType::Int, src_t, __VA_ARGS__)                                   \
                      AT_PRIVATE_CASE_TYPE_USING_HINT(                                           \
                          at::ScalarType::Double, src_t, __VA_ARGS__)                            \
                          AT_PRIVATE_CASE_TYPE_USING_HINT(                                       \
                              at::ScalarType::Float, src_t, __VA_ARGS__)                         \
                              AT_PRIVATE_CASE_TYPE_USING_HINT(                                   \
                                  at::ScalarType::ComplexDouble,                                 \
                                  src_t,                                                         \
                                  __VA_ARGS__)                                                   \
                                  AT_PRIVATE_CASE_TYPE_USING_HINT(                               \
                                      at::ScalarType::ComplexFloat,                              \
                                      src_t,                                                     \
                                      __VA_ARGS__)                                               \
                                      AT_PRIVATE_CASE_TYPE_USING_HINT(                           \
                                          at::ScalarType::Half,                                  \
                                          src_t,                                                 \
                                          __VA_ARGS__)                                           \
                                          AT_PRIVATE_CASE_TYPE_USING_HINT(                       \
                                              at::ScalarType::BFloat16,                          \
                                              src_t,                                             \
                                              __VA_ARGS__)                                       \
                                              AT_PRIVATE_CASE_TYPE_USING_HINT(                   \
                                                  at::ScalarType::Bool,                          \
                                                  src_t,                                         \
                                                  __VA_ARGS__)                                   \
                                                  AT_PRIVATE_CASE_TYPE_USING_HINT(               \
                                                      at::ScalarType::                           \
                                                          Float8_e4m3fn,                         \
                                                      src_t,                                     \
                                                      __VA_ARGS__)                               \
                                                      AT_PRIVATE_CASE_TYPE_USING_HINT(           \
                                                          at::ScalarType::                       \
                                                              Float8_e4m3fnuz,                   \
                                                          src_t,                                 \
                                                          __VA_ARGS__)                           \
                                                          AT_PRIVATE_CASE_TYPE_USING_HINT(       \
                                                              at::ScalarType::                   \
                                                                  Float8_e5m2,                   \
                                                              src_t,                             \
                                                              __VA_ARGS__)                       \
                                                              AT_PRIVATE_CASE_TYPE_USING_HINT(   \
                                                                  at::ScalarType::               \
                                                                      Float8_e5m2fnuz,           \
                                                                  src_t,                         \
                                                                  __VA_ARGS__))

namespace {

template <
    typename T,
    typename src_t,
    int depth,
    int r_args_depth,
    int res_arg_index>
struct CopyFunctor {
  static_assert(depth == 2 && r_args_depth == 1 && res_arg_index == 1);
  template <typename Op>
  __device__ __forceinline__ void operator()(
      int64_t chunk_size,
      TensorListMetadata<depth>& tl,
      Op op) {
    const auto tensor_loc = tl.block_to_tensor[blockIdx.x];
    const auto chunk_idx = tl.block_to_chunk[blockIdx.x];
    auto n = tl.numel_for_tensor[tensor_loc];

    src_t* src_ptr = (src_t*)tl.addresses[0][tensor_loc];
    src_ptr += chunk_idx * chunk_size;
    T* self_ptr = (T*)tl.addresses[1][tensor_loc];
    self_ptr += chunk_idx * chunk_size;

    const bool all_aligned{is_aligned(src_ptr) && is_aligned(self_ptr)};

    n -= chunk_idx * chunk_size;
    src_t src_args[kILP];
    T r_args[kILP];

    // to make things simple, we put aligned case in a different code path
    if (n % kILP == 0 && chunk_size % kILP == 0 && all_aligned) {
      for (int64_t i_start = threadIdx.x;
           i_start * kILP < n && i_start * kILP < chunk_size;
           i_start += blockDim.x) {
        // load
        load_store(src_args, src_ptr, 0, i_start);
#pragma unroll
        for (int ii = 0; ii < kILP; ii++) {
          r_args[ii] = static_cast<T>(op(src_args[ii]));
        }
        // store
        load_store(self_ptr, r_args, i_start, 0);
      }
    } else {
      for (int64_t i_start = 0; i_start < n && i_start < chunk_size;
           i_start += blockDim.x * kILP) {
#pragma unroll
        for (int ii = 0; ii < kILP; ii++) {
          const auto i = i_start + threadIdx.x + ii * blockDim.x;
          if (i < n && i < chunk_size) {
            src_args[ii] = src_ptr[i];
          }
        }
#pragma unroll
        for (int ii = 0; ii < kILP; ii++) {
          r_args[ii] = static_cast<T>(op(src_args[ii]));
        }
        store_args(self_ptr, r_args, i_start, chunk_size, n);
      }
    }
  }
};

} // anonymous namespace

void foreach_tensor_copy_list_kernel_cuda_(
    TensorList self,
    TensorList src,
    const bool non_blocking) {
  check_foreach_api_restrictions(self, src);
  if (!(_check_tensors_share_device_and_dtype(
            {self, src}, /* skip_dtype_check */ true) &&
        std::all_of(
            src.cbegin(),
            src.cend(),
            [&](const auto& t) -> bool {
              return t.dtype() == src[0].dtype();
            }) &&
        _check_tensors_share_sizes_and_strides({self, src}))) {
    return at::native::foreach_tensor_copy_list_kernel_slow_(
        self, src, non_blocking);
  }

  std::vector<std::vector<at::Tensor>> tensor_lists{src.vec(), self.vec()};

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND7(
      ScalarType::Half,
      ScalarType::BFloat16,
      ScalarType::Bool,
      ScalarType::Float8_e4m3fn,
      ScalarType::Float8_e4m3fnuz,
      ScalarType::Float8_e5m2,
      ScalarType::Float8_e5m2fnuz,
      self[0].scalar_type(),
      "foreach_tensor_copy",
      [&]() {
        AT_DISPATCH_SOURCE_TYPES(src[0].scalar_type(), "foreach_tensor_copy", [&] {
          if constexpr (std::is_same_v<scalar_t, src_t>) {
            multi_tensor_apply<2>(
                tensor_lists,
                UnaryOpFunctor<
                    scalar_t,
                    /* depth */ 2,
                    /* r_args_depth */ 1,
                    /* res_arg_index */ 1>(),
                Copy<scalar_t, scalar_t>());
          } else {
            // Ref:
            // https://github.com/pytorch/pytorch/blob/656134c38f4737d13c3f43fc5c59470bc23c1d2f/aten/src/ATen/native/Copy.cpp#L299-L301
            if (!self[0].is_complex() && src[0].is_complex()) {
              TORCH_WARN_ONCE(
                  "Casting complex values to real discards the imaginary part");
            }
            multi_tensor_apply<2>(
                tensor_lists,
                CopyFunctor<
                    scalar_t,
                    src_t,
                    /* depth */ 2,
                    /* r_args_depth */ 1,
                    /* res_arg_index */ 1>(),
                Copy<scalar_t, src_t>());
          }
        });
      });
  increment_version(self);
}

#undef AT_DISPATCH_SOURCE_TYPES

} // namespace at::native
