#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>
#include <ATen/native/cuda/DistributionTemplates.h>
#include <ATen/native/Resize.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty_like.h>
#include <ATen/ops/leaky_relu.h>
#include <ATen/ops/rrelu_with_noise_native.h>
#endif


namespace at::native {

template <typename scalar_t, int unroll_factor, typename F>
#if __CUDA_ARCH__ >= 350 || defined USE_ROCM
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void rrelu_with_noise_cuda_kernel(
    int numel,
    PhiloxCudaState philox_args,
    scalar_t* output,
    const scalar_t* input,
    scalar_t* noise,
    double lower,
    double upper,
    const F& random_func) {
  auto seeds = at::cuda::philox::unpack(philox_args);
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds),
              idx,
              std::get<1>(seeds),
              &state);

  int grid_stride = blockDim.x * gridDim.x * unroll_factor;
  int rounded_size = ((numel - 1) / grid_stride + 1) * grid_stride;
  double range = upper - lower;

  for (int linear_index = idx; linear_index < rounded_size; linear_index += grid_stride) {
    auto rand = random_func(&state);

    // ensure that (&rand.x)[ii] is safe
    static_assert(sizeof(rand)/sizeof(rand.x) == unroll_factor, "");

    #pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li >= numel) {
        continue;
      }
      scalar_t r = static_cast<scalar_t>((&rand.x)[ii]);
      r = r * range + lower;
      if (input[li] <= 0) {
        output[li] = input[li] * r;
        noise[li] = r;
      } else {
        output[li] = input[li];
        noise[li] = static_cast<scalar_t>(1);
      }
    }
    __syncthreads();
  }
}

template <typename scalar_t>
inline void _rrelu_with_noise_cuda_train(
    Tensor& output,
    const Tensor& input_,
    Tensor& noise_,
    const Scalar& lower_,
    const Scalar& upper_,
    std::optional<Generator> generator) {
  auto input = input_.contiguous();
  auto noise = noise_.contiguous();
  Tensor tmp_output = output.contiguous();

  int64_t numel = input.numel();
  const int unroll_factor = std::is_same_v<scalar_t, double> ? 2 : 4;
  auto [counter_offset, grid, block] = calc_execution_policy(numel, unroll_factor);

  auto gen = get_generator_or_default<CUDAGeneratorImpl>(
      generator, cuda::detail::getDefaultCUDAGenerator());
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(counter_offset);
  }

  const scalar_t* input_data = input.const_data_ptr<scalar_t>();
  scalar_t* noise_data = noise.mutable_data_ptr<scalar_t>();
  scalar_t* output_data = tmp_output.mutable_data_ptr<scalar_t>();

  double lower = lower_.to<double>();
  double upper = upper_.to<double>();

  auto stream = at::cuda::getCurrentCUDAStream();

  if (std::is_same_v<scalar_t, double>) {
    rrelu_with_noise_cuda_kernel<scalar_t, 2><<<grid, block, 0, stream>>>(
        numel,
        rng_engine_inputs,
        output_data,
        input_data,
        noise_data,
        lower,
        upper,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) {
          return hiprand_uniform2_double(state);
        });
        C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    // half and float
    rrelu_with_noise_cuda_kernel<scalar_t, 4><<<grid, block, 0, stream>>>(
        numel,
        rng_engine_inputs,
        output_data,
        input_data,
        noise_data,
        lower, upper,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) {
          return hiprand_uniform4(state);
        });
        C10_CUDA_KERNEL_LAUNCH_CHECK();
  }

  if (!output.is_contiguous()) {
    output.copy_(tmp_output);
  }
}

Tensor& rrelu_with_noise_out_cuda(const Tensor& self,
    Tensor& noise,
    const Scalar& lower,
    const Scalar& upper,
    bool training,
    std::optional<Generator> generator,
    Tensor& output) {
  at::native::resize_output(output, self.sizes());

  if (self.numel() == 0) {
    return output;
  }

  TensorArg self_arg{self, "self", 1}, noise_arg{noise, "noise", 2},
      output_arg{output, "output", 3};
  checkAllSameGPU("rrelu_with_noise_out_cuda", {self_arg, noise_arg, output_arg});

  if (training) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
        self.scalar_type(), "rrelu_with_noise_out_cuda", [&] {
          _rrelu_with_noise_cuda_train<scalar_t>(
              output, self, noise, lower, upper, generator);
        });
  }
  else {
    auto lower_tensor = lower.to<double>();
    auto upper_tensor = upper.to<double>();
    Scalar negative_slope = (lower_tensor + upper_tensor) / 2;
    at::leaky_relu_out(output, self, negative_slope);
  }
  return output;
}

Tensor rrelu_with_noise_cuda(
    const Tensor& self,
    Tensor& noise,
    const Scalar& lower,
    const Scalar& upper,
    bool training,
    std::optional<Generator> generator) {
  Tensor output = at::empty_like(self, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  return at::native::rrelu_with_noise_out_cuda(self, noise, lower, upper, training, generator, output);
}

Tensor& rrelu_with_noise_cuda_(
    Tensor& self,
    Tensor& noise,
    const Scalar& lower,
    const Scalar& upper,
    bool training,
    std::optional<Generator> generator) {
  return at::native::rrelu_with_noise_out_cuda(
      self, noise, lower, upper, training, generator, self);
}

}  // namespace at::native
