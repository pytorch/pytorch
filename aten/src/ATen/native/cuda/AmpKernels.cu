#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#define _USE_MATH_DEFINES

#include <math.h>

#include <ATen/core/Tensor.h>
#include <ATen/DeviceGuard.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/ForeachFunctors.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/ForeachUtils.h>
#include <ATen/native/TensorIterator.h>


namespace {
// Thin wrapper around https://docs.nvidia.com/cuda/cuda-math-api/cuda_math_api/group__CUDA__MATH__SINGLE.html,
// to ensure the Cuda math library's isfinite is actually what gets called in
// _amp_non_finite_check_and_unscale_cuda_'s gpu_kernel lambda.
//
// isfinite_ensure_cuda_math is defined outside at::native because:
// - A bare call to "isfinite(val)" inside at::native causes nvcc to prefer the unrelated
//   Tensor at::native::isfinite(const Tensor&), resulting in an error:
//   "no suitable constructor exists to convert from "float" to "at::Tensor""
// - Unfortunately, the Cuda math library documentation doesn't say how (or if) you can provide a full namespace path
//   to ensure that its version of a particular function is invoked.  It only shows bare (not-namespaced)
//   calls to its routines inside kernel or device functions.
// - "std::isfinite(val)" in the gpu_kernel lambda causes an "unspecified launch failure" at runtime with cuda 9 on Windows.
//
// isfinite_ensure_cuda_math, declared at file scope outside the at::native region, uses isfinite as math library docs
// suggest and allows disambiguated usage in the lambda within the at::native region.
// GPU_LAMBDA is defined as __host__ __device__ (see Loops.cuh), so I need the __host__ keyword or else nvcc complains that
// "calling a __device__ function("isfinite_ensure_cuda_math") from a __host__ __device__ function("operator()") is not allowed."
static __host__ __device__ __forceinline__ int isfinite_ensure_cuda_math(float val) {
  return isfinite(val);
}
}

namespace at::native {

namespace {
// Single-tensor fallback for _amp_foreach_non_finite_check_and_unscale_cuda_.
// Handles individual tensors that are acceptable to unscale but not MTA-safe.
void _amp_non_finite_check_and_unscale_cuda_(Tensor& scaled_grad,
                                             Tensor& found_inf,
                                             const Tensor& inv_scale)
{
  // The only way we reach this function is through _amp_foreach_non_finite_check_and_unscale_cuda_, so no input checks.

  // It's not obvious gpu_kernel always guards onto its argument.  Guarding here just in case.
  const OptionalDeviceGuard device_guard(device_of(scaled_grad));

  // Acts on scaled_grad in place.
  auto iter = TensorIterator::unary_op(scaled_grad, scaled_grad);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    iter.dtype(),
    "_amp_non_finite_check_and_unscale_cuda",
    [&iter, &found_inf, &inv_scale] {
      auto* found_inf_ptr = found_inf.mutable_data_ptr<float>();
      auto* inv_scale_ptr = inv_scale.const_data_ptr<float>();

      using opmath_t = at::opmath_type<scalar_t>;

      gpu_kernel(iter,
                 [found_inf_ptr, inv_scale_ptr] GPU_LAMBDA (scalar_t val_in) -> scalar_t {
                   auto val = static_cast<opmath_t>(val_in);
                   if (!isfinite_ensure_cuda_math(val)) {
                     *found_inf_ptr = 1.f;
                   }
                   // Every thread accesses inv_scale, but it will hit in cache.
                   const auto inv_scale_val = *inv_scale_ptr;
                   return static_cast<scalar_t>(inv_scale_val == 1.f ? val : val * inv_scale_val);
                 });
    });
}
} // anonymous namespace


// Multiplies each tensor in scaled_grads by inv_scale in-place.
// If any element of any tensor in scaled_grads is inf or NaN, sets found_inf to 1.0.
// Uses multi tensor apply (MTA) to process all MTA-safe tensors.
//
// Args:
// scaled_grads:  A TensorList of scaled gradient tensors.  May contain infs or NaNs.
// found_inf:  A single-element float tensor to which 1.0 will be written if any gradient contain infs/nans.
//             Pre-zeroing found_inf, if appropriate, is the responsibility of the caller.
// inv_scale:  The inverse of the scale factor by which scaled_grads are currently multiplied.
void _amp_foreach_non_finite_check_and_unscale_cuda_(TensorList scaled_grads,
                                                     Tensor& found_inf,
                                                     const Tensor& inv_scale)
{
  if (scaled_grads.size() == 0) {
    return;
  }

  TORCH_CHECK(inv_scale.is_cuda(), "inv_scale must be a CUDA tensor.");
  TORCH_CHECK(found_inf.is_cuda(), "found_inf must be a CUDA tensor.");
  TORCH_CHECK(inv_scale.numel() == 1, "inv_scale must be a 1-element tensor.");
  TORCH_CHECK(found_inf.numel() == 1, "found_inf must be a 1-element tensor.");
  TORCH_CHECK(inv_scale.scalar_type() == at::ScalarType::Float, "inv_scale must be a float tensor.");
  TORCH_CHECK(found_inf.scalar_type() == at::ScalarType::Float, "found_inf must be a float tensor.");

  // Ensures client code (GradScaler) filtered scaled_grads by dtype.
  check_foreach_api_restrictions(scaled_grads);

  std::vector<std::vector<at::Tensor>> tensor_lists;

  // is_non_overlapping_and_dense() is not available in Python.
  // GradScaler can't filter for it. We need to filter here.
  if (can_use_fast_route(scaled_grads)) {
    // Hopefully common case.
    // can_use_fast_route is true, which confirms:
    //  - all scaled_grads are strided
    //  - all scaled_grads are non overlapping and dense
    //  - all scaled_grads are on the same device
    //  - all scaled_grads are of the same dtype
    TORCH_CHECK(scaled_grads[0].is_cuda(), "scaled_grads must be CUDA tensors.");
    // Sets up MTA launch to use scaled_grads as-is.
    tensor_lists.emplace_back(scaled_grads.vec());
  } else {
    // Hopefully uncommon case.
    // can_use_fast_route is an all-or-nothing check.  In this path it was false,
    // so any of the above confirmations could have gone wrong.
    // We filter MTA-safe tensors into an MTA-able list.
    // If a tensor is acceptable but not MTA-safe, we fall back to the TensorIterator kernel.
    // If a tensor is unacceptable, we throw an error to blame GradScaler.
    tensor_lists.resize(1);
    tensor_lists[0].reserve(scaled_grads.size());
    auto expected_device = scaled_grads[0].device();
    const auto expected_dtype = scaled_grads[0].scalar_type();
    for (const Tensor& t : scaled_grads) {
      // Ensures GradScaler filtered scaled_grads by device.
      TORCH_CHECK(t.is_cuda(), "one of scaled_grads was not a CUDA tensor.");
      TORCH_CHECK(t.device() == expected_device, "scaled_grads must be on the same device.");
      TORCH_CHECK(t.layout() == at::kStrided, "one of scaled_grads was not a strided tensor.");
      if (!t.is_non_overlapping_and_dense() || t.scalar_type() != expected_dtype) {
        // t is acceptable but not MTA-safe.  Falls back to single-tensor TensorIterator kernel.
        _amp_non_finite_check_and_unscale_cuda_(const_cast<Tensor&>(t),
                                                found_inf,
                                                inv_scale);
      } else {
        tensor_lists[0].push_back(t);
      }
    }
    if (tensor_lists[0].size() == 0) {
      return;
    }
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    tensor_lists[0][0].scalar_type(),
    "_amp_foreach_non_finite_check_and_unscale_cuda",
    [&tensor_lists, &found_inf, &inv_scale] {
      auto* found_inf_ptr = found_inf.mutable_data_ptr<float>();
      auto* inv_scale_ptr = inv_scale.const_data_ptr<float>();

      using opmath_t = at::opmath_type<scalar_t>;

      // multi_tensor_apply guards onto tensor_lists[0][0], no need to guard explicitly.
      multi_tensor_apply<1>(tensor_lists,
                            UnaryOpFunctor<scalar_t,
                                           /* depth */ 1,
                                           /* r_args_depth */ 1,
                                           /* res_arg_index */ 0>(),
                            [found_inf_ptr, inv_scale_ptr] GPU_LAMBDA (opmath_t val) -> opmath_t {
                              // There is a slight asymmetry here with the TensorIterator kernel above.
                              // MTA Functors ensure val comes in as opmath_t rather than scalar_t.
                              if (!isfinite_ensure_cuda_math(val)) {
                                *found_inf_ptr = 1.f;
                              }
                              // Every thread accesses inv_scale, but it will hit in cache.
                              const auto inv_scale_val = *inv_scale_ptr;
                              return static_cast<opmath_t>(inv_scale_val == 1.f ? val : val * inv_scale_val);
                            });
    });
}


// amp_update_scale_cuda_kernel is launched with a single thread to compute the new scale.
// The scale factor is maintained and updated on the GPU to avoid synchronization.
__global__ void amp_update_scale_cuda_kernel(float* current_scale,
                                             int* growth_tracker,
                                             const float* found_inf,
                                             double growth_factor,
                                             double backoff_factor,
                                             int growth_interval)
{
  if (*found_inf) {
    *current_scale = (*current_scale)*backoff_factor;
    *growth_tracker = 0;
  } else {
    // Entering this branch means we just carried out a successful step,
    // so growth_tracker is incremented before comparing to growth_interval.
    auto successful = (*growth_tracker) + 1;
    if (successful == growth_interval) {
      auto new_scale = static_cast<float>((*current_scale)*growth_factor);
      // Do not grow the scale past fp32 bounds to inf.
      if (isfinite_ensure_cuda_math(new_scale)) {
          *current_scale = new_scale;
      }
      *growth_tracker = 0;
    } else {
      *growth_tracker = successful;
    }
  }
}


// _amp_update_scale_cuda asynchronously updates the scale tensor in place.
//
// Args:
// current_scale:  A one-element cuda float tensor containing the scale value.
// growth_tracker:  A one-element torch.cuda.IntTensor containing the number of recent consecutive unskipped steps.
// found_inf:  A one-element cuda float tensor. If > 0, indicates that infs/nans were found by the relevant
//             prior _amp_non_finite_check_and_unscale_cuda call, and 0 if no infs/nans were found.
// growth_factor:  Multiplier if no infs/NaNs were found (typically slightly > 1).
// backoff_factor:  Multiplier if infs/NaNs were found (typically 0.5).
// growth_interval:  Number of consecutive unskipped steps that must occur for current_scale to be multiplied by
//                   growth_factor.
//
// Returns:
// current_scale
Tensor& _amp_update_scale_cuda_(Tensor& current_scale,
                                Tensor& growth_tracker,
                                const Tensor& found_inf,
                                double growth_factor,
                                double backoff_factor,
                                int64_t growth_interval)
{
  TORCH_CHECK(growth_tracker.is_cuda(), "growth_tracker must be a CUDA tensor.");
  TORCH_CHECK(current_scale.is_cuda(), "current_scale must be a CUDA tensor.");
  TORCH_CHECK(found_inf.is_cuda(), "found_inf must be a CUDA tensor.");
  TORCH_CHECK(growth_tracker.numel() == 1, "growth_tracker must be a 1-element tensor.");
  TORCH_CHECK(current_scale.numel() == 1, "current_scale must be a 1-element tensor.");
  TORCH_CHECK(found_inf.numel() == 1, "found_inf must be a 1-element tensor.");
  TORCH_CHECK(growth_tracker.scalar_type() == at::ScalarType::Int, "growth_tracker must be an int tensor.");
  TORCH_CHECK(current_scale.scalar_type() == at::ScalarType::Float, "current_scale must be a float tensor.");
  TORCH_CHECK(found_inf.scalar_type() == at::ScalarType::Float, "found_inf must be a float tensor.");

  amp_update_scale_cuda_kernel<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
    current_scale.mutable_data_ptr<float>(),
    growth_tracker.mutable_data_ptr<int>(),
    found_inf.const_data_ptr<float>(),
    growth_factor,
    backoff_factor,
    growth_interval);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return current_scale;
}

} // namespace at::native
