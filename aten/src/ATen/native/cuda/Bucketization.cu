#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/BucketizationUtils.h>
#include <ATen/native/Resize.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/bucketize_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/searchsorted_native.h>
#endif

namespace at::native {

// Implement a numpy like searchsorted and a TF like bucketize function running on cuda
// See details in ATen/native/Bucketization.cpp

namespace {

template<typename input_t>
__device__ int64_t lower_bound(const input_t *data_ss, int64_t start, int64_t end, const input_t val, const int64_t *data_sort) {
  // sorter gives relative ordering for ND tensors, so we need to save and add the non-updated start as an offset
  // i.e. the second row of a 3x3 tensors starts at element 3 but sorter's second row only contains 0, 1, or 2
  const int64_t orig_start = start;
  while (start < end) {
    const int64_t mid = start + ((end - start) >> 1);
    const input_t mid_val = data_sort ? data_ss[orig_start + data_sort[mid]] : data_ss[mid];
    if (!(mid_val >= val)) {
      start = mid + 1;
    }
    else {
      end = mid;
    }
  }
  return start;
}

template<typename input_t>
__device__ int64_t upper_bound(const input_t *data_ss, int64_t start, int64_t end, const input_t val, const int64_t *data_sort) {
  // sorter gives relative ordering for ND tensors, so we need to save and add the non-updated start as an offset
  // i.e. the second row of a 3x3 tensors starts at element 3 but sorter's second row only contains 0, 1, or 2
  const int64_t orig_start = start;
  while (start < end) {
    const int64_t mid = start + ((end - start) >> 1);
    const input_t mid_val = data_sort ? data_ss[orig_start + data_sort[mid]] : data_ss[mid];
    if (!(mid_val > val)) {
      start = mid + 1;
    }
    else {
      end = mid;
    }
  }
  return start;
}

template<typename input_t, typename output_t>
__global__ void searchsorted_cuda_kernel(
  output_t *data_out,
  const input_t *data_in,
  const input_t *data_bd,
  const int64_t *data_sort,
  int64_t idim_in,
  int64_t idim_bd,
  int64_t numel_in,
  bool right,
  bool is_1d_boundaries) {

  for (int64_t tid = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x; tid < numel_in; tid += blockDim.x * gridDim.x) {
    // If boundaries tensor is 1d, we always search the entire boundary tensor
    int64_t start_bd = is_1d_boundaries ? 0 : tid / idim_in * idim_bd;
    int64_t end_bd = start_bd + idim_bd;

    int64_t pos = !right ?
      lower_bound<input_t>(data_bd, start_bd, end_bd, data_in[tid], data_sort) - start_bd :
      upper_bound<input_t>(data_bd, start_bd, end_bd, data_in[tid], data_sort) - start_bd;

    // type conversion might happen here
    data_out[tid] = pos;
  }
}

template<typename input_t, typename output_t>
void searchsorted_cuda_contiguous(Tensor& result, const Tensor& input, const Tensor& boundaries, const bool& right, const Tensor& sorter) {
  int64_t numel_in = input.numel();
  bool is_scalar_input = input.dim() == 0 && numel_in == 1;
  // inner most dim size of input and boundaries
  int64_t idim_in = is_scalar_input ? 1 : input.sizes().back();
  int64_t idim_bd = boundaries.sizes().back();

  const input_t *data_in = input.const_data_ptr<input_t>();
  const input_t *data_bd = boundaries.const_data_ptr<input_t>();
  const int64_t *data_sort = sorter.defined() ? sorter.const_data_ptr<int64_t>() : nullptr;
  output_t *data_out = result.mutable_data_ptr<output_t>();

  int64_t maxThread = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t maxGrid = 1024;
  dim3 block = dim3(std::min(maxThread, numel_in));
  dim3 grid  = dim3(std::min(maxGrid, ceil_div<int64_t>(numel_in, block.x)));
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

  searchsorted_cuda_kernel<<<grid, block, 0, stream>>>(
    data_out, data_in, data_bd, data_sort, idim_in, idim_bd, numel_in, right, boundaries.dim() == 1);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void dispatch(
    Tensor& result,
    const Tensor& input,
    const Tensor& boundaries,
    bool out_int32,
    bool right,
    const Tensor& sorter) {
  if (!out_int32) {
    AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "searchsorted_out_cuda", [&] {
      searchsorted_cuda_contiguous<scalar_t, int64_t>(result, input, boundaries, right, sorter);
    });
  }
  else {
    AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "searchsorted_out_cuda", [&] {
      searchsorted_cuda_contiguous<scalar_t, int>(result, input, boundaries, right, sorter);
    });
  }
}

}

Tensor& searchsorted_out_cuda(
    const Tensor& sorted_sequence,
    const Tensor& self,
    bool out_int32,
    bool right,
    const std::optional<std::string_view> side_opt,
    const std::optional<Tensor>& sorter_opt,
    Tensor& result) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> sorter_maybe_owned = at::borrow_from_optional_tensor(sorter_opt);
  const Tensor& sorter = *sorter_maybe_owned;
  searchsorted_pre_check(sorted_sequence, self, result, out_int32, right, side_opt, sorter);
  resize_output(result, self.sizes());

  // we have two inputs to set right, pre_check checks that they aren't set to opposites
  bool is_right = (side_opt && *side_opt == "right") || right;
  if (self.numel() == 0) {
    return result;
  }

  // for non-contiguous result tensors, we write the output to a contiguous copy so we can later copy back, maintaining the original result tensor
  Tensor out = result;
  if (!result.is_contiguous()) {
    out = result.contiguous();
  }
  if (sorted_sequence.is_contiguous() && self.is_contiguous() && sorted_sequence.dtype() == self.dtype() && sorter.is_contiguous()) {
   dispatch(out, self, sorted_sequence, out_int32, is_right, sorter);
  }
  else {
    Tensor trimmed_input;
    Tensor trimmed_boundaries;
    Tensor trimmed_sorter;
    searchsorted_maybe_trim_input_tensors(trimmed_input, trimmed_boundaries, trimmed_sorter, self, sorted_sequence, sorter);
    const Tensor& final_input = trimmed_input.defined() ? trimmed_input : self;
    const Tensor& final_boundaries = trimmed_boundaries.defined() ? trimmed_boundaries : sorted_sequence;
    const Tensor& final_sorter = trimmed_sorter.defined() ? trimmed_sorter : sorter;
    dispatch(out, final_input, final_boundaries, out_int32, is_right, final_sorter);
  }

  // if result is non-contiguous, we wrote the answer to a copied version, so we copy back to the original result tensor
  if (!result.is_contiguous()) {
    result.copy_(out);
  }
  return result;
}

Tensor& searchsorted_out_cuda(
    const Tensor& sorted_sequence,
    const Scalar& self,
    bool out_int32,
    bool right,
    const std::optional<std::string_view> side_opt,
    const std::optional<Tensor>& sorter_opt,
    Tensor& result) {
  const Tensor& scalar_tensor = searchsorted_scalar_tensor(self, sorted_sequence.device());
  return searchsorted_out_cuda(sorted_sequence, scalar_tensor, out_int32, right, side_opt, sorter_opt, result);
}

Tensor searchsorted_cuda(
    const Tensor& sorted_sequence,
    const Tensor& self,
    bool out_int32,
    bool right,
    const std::optional<std::string_view> side_opt,
    const std::optional<Tensor>& sorter) {
  ScalarType scalar_type = out_int32 ? ScalarType::Int : ScalarType::Long;
  c10::TensorOptions options = TensorOptions().device(self.options().device()).dtype(scalar_type);
  Tensor result = at::empty({0}, options, MemoryFormat::Contiguous);
  at::native::searchsorted_out_cuda(sorted_sequence, self, out_int32, right, side_opt, sorter, result);
  return result;
}

Tensor searchsorted_cuda(
    const Tensor& sorted_sequence,
    const Scalar& self,
    bool out_int32,
    bool right,
    const std::optional<std::string_view> side_opt,
    const std::optional<Tensor>& sorter) {
  const Tensor& scalar_tensor = searchsorted_scalar_tensor(self, sorted_sequence.device());
  return searchsorted_cuda(sorted_sequence, scalar_tensor, out_int32, right, side_opt, sorter);
}

Tensor& bucketize_out_cuda(const Tensor& self, const Tensor& boundaries, bool out_int32, bool right, Tensor& result) {
  TORCH_CHECK(boundaries.dim() == 1, "boundaries tensor must be 1 dimension, but got dim(", boundaries.dim(), ")");
  at::native::searchsorted_out_cuda(boundaries, self, out_int32, right, std::nullopt, std::nullopt, result);
  return result;
}

Tensor bucketize_cuda(const Tensor& self, const Tensor& boundaries, bool out_int32, bool right) {
  ScalarType scalar_type = out_int32 ? ScalarType::Int : ScalarType::Long;
  c10::TensorOptions options = TensorOptions().device(self.options().device()).dtype(scalar_type);
  Tensor result = at::empty({0}, options, MemoryFormat::Contiguous);
  at::native::bucketize_out_cuda(self, boundaries, out_int32, right, result);
  return result;
}

Tensor bucketize_cuda(const Scalar& self, const Tensor& boundaries, bool out_int32, bool right) {
  return bucketize_cuda(searchsorted_scalar_tensor(self, boundaries.device()), boundaries, out_int32, right);
}

} // namespace at::native
