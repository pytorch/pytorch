#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/Resize.h>
#include <ATen/native/TensorShape.h>
#include <c10/cuda/CUDAGraphsC10Utils.h>
#include <c10/util/TypeCast.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_chunk_cat_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/split_with_sizes_copy_native.h>
#endif

namespace at::native {

namespace detail {

// NOTE [CUDA fast path for split_with_sizes_copy.out]
// split_with_sizes_copy.out for contiguous operands has the following
// properties:
// - Each src split consists of multiple chunks that are separated by a fixed
// stride. The number of chunks and the strides are the same across all src
// splits.
// - Each dst split is the concatenation of the chunks in its corresponding src
// splits.
// - The sizes of chunks vary across splits.
// - A (src, dst) chunk pair is not guaranteed to have the
// same alignment.
//
// The following strategies are employed to optimize for this workload:
// - The entire workload is fused into a single kernel to maximize I/O
// throughput and minimize wave quantization.
// - To account for both small and large chunk sizes, a "jagged grid" is used.
// Each chunk is processed by one or more blocks depending on its size.
// - Within each chunk, the region in which writes can be vectorized is
// identified. Within this region, writes are always vectorized and reads are
// oppurtunistically vectorized.
static constexpr int64_t BLOCK_SIZE = 128;
static constexpr int64_t BYTES_PER_THREAD = 16;
static constexpr int64_t BYTES_PER_BLOCK = BYTES_PER_THREAD * BLOCK_SIZE;

static __host__ __device__ inline int64_t div_up(int64_t a, int64_t b) {
  return (a + b - 1) / b;
}

template <typename T>
__device__ inline void stream_load128(uint4& val, const T* addr) {
  uint64_t low, high;
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  low = reinterpret_cast<const uint64_t*>(addr)[0];
  high = reinterpret_cast<const uint64_t*>(addr)[1];
#else
  asm("ld.global.nc.v2.u64 {%0, %1}, [%2];"
      : "=l"(low), "=l"(high)
      : "l"(addr));
#endif
  reinterpret_cast<uint64_t*>(&val)[0] = low;
  reinterpret_cast<uint64_t*>(&val)[1] = high;
}

template <typename T>
__device__ inline void stream_store128(T* addr, const uint4& val) {
  uint64_t low, high;
  low = reinterpret_cast<const uint64_t*>(&val)[0];
  high = reinterpret_cast<const uint64_t*>(&val)[1];
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  reinterpret_cast<uint64_t*>(addr)[0] = low;
  reinterpret_cast<uint64_t*>(addr)[1] = high;
#else
  asm("st.global.cs.v2.u64 [%0], {%1, %2};" : : "l"(addr), "l"(low), "l"(high));
#endif
}

template <typename T>
static __device__ inline bool is_aligned(const void* addr) {
  return reinterpret_cast<uintptr_t>(addr) % sizeof(T) == 0;
}

template <typename T>
static __device__ inline void load128(uint4& val, const char* addr) {
  for (size_t i = 0; i < detail::BYTES_PER_THREAD / sizeof(T); ++i) {
    reinterpret_cast<T*>(&val)[i] = reinterpret_cast<const T*>(addr)[i];
  }
}

template <>
__device__ inline void load128<uint4>(uint4& val, const char* addr) {
  stream_load128(val, addr);
}

static __device__ inline void load128(uint4& val, const char* addr) {
  if (is_aligned<uint4>(addr)) {
    load128<uint4>(val, addr);
  } else if (is_aligned<int64_t>(addr)) {
    load128<uint64_t>(val, addr);
  } else if (is_aligned<uint32_t>(addr)) {
    load128<uint32_t>(val, addr);
  } else {
    load128<uint8_t>(val, addr);
  }
}

static __device__ __inline__ void get_aligned_region(
    char* ptr,
    const int64_t chunk_size,
    const int64_t alignment,
    int64_t& align_off,
    int64_t& aligned_size) {
  const int64_t ptr_val = reinterpret_cast<uintptr_t>(ptr);
  align_off = detail::div_up(ptr_val, alignment) * alignment - ptr_val;
  aligned_size = (chunk_size - align_off) / alignment * alignment;
}

static __device__ __inline__ void copy_chunk(
    char* dst,
    const char* src,
    int64_t chunk_size,
    int64_t thread_idx,
    int64_t num_threads) {
  if (chunk_size < num_threads) {
    if (thread_idx < chunk_size) {
      dst[thread_idx] = src[thread_idx];
    }
    return;
  }

  // Identify the region in which writes are guaranteed to be 128-bit aligned
  int64_t align_off, aligned_size;
  get_aligned_region(
      dst, chunk_size, detail::BYTES_PER_THREAD, align_off, aligned_size);

  for (int64_t off = align_off + thread_idx * detail::BYTES_PER_THREAD;
       off < align_off + aligned_size;
       off += num_threads * detail::BYTES_PER_THREAD) {
    uint4 val;
    // Oppurtunistically vectorize reads
    load128(val, &src[off]);
    stream_store128(&dst[off], val);
  }

  // Handle unaligned regions
  if (thread_idx < align_off && thread_idx < chunk_size) {
    dst[thread_idx] = src[thread_idx];
  }
  if (align_off + aligned_size + thread_idx < chunk_size) {
    dst[align_off + aligned_size + thread_idx] =
        src[align_off + aligned_size + thread_idx];
  }
}

static __global__ void split_with_sizes_copy_out_contiguous_no_cast_kernel(
    char** dst_base_addrs,
    char** src_base_addrs,
    int64_t* split_chunk_sizes,
    int64_t* block_idx_to_split_idx,
    int64_t* blocks_cumsums,
    int64_t src_stride,
    int64_t num_chunks) {
  const int64_t split_idx = block_idx_to_split_idx[blockIdx.x];
  const int64_t split_blocks =
      blocks_cumsums[split_idx + 1] - blocks_cumsums[split_idx];
  const int64_t split_threads = split_blocks * blockDim.x;
  const int64_t split_thread_idx =
      (blockIdx.x - blocks_cumsums[split_idx]) * blockDim.x + threadIdx.x;
  const int64_t split_chunk_size = split_chunk_sizes[split_idx];

  char* dst_base_addr = dst_base_addrs[split_idx];
  char* src_base_addr = src_base_addrs[split_idx];

  for (int64_t i = blockIdx.y; i < num_chunks; i += gridDim.y) {
    copy_chunk(
        dst_base_addr + i * split_chunk_size,
        src_base_addr + i * src_stride,
        split_chunk_size,
        split_thread_idx,
        split_threads);
  }
}

// Calculate the base addr for each split.
static inline std::vector<int64_t> get_split_base_addrs(
    const at::Tensor& tensor,
    at::IntArrayRef split_sizes,
    int64_t dim) {
  const auto* data_ptr = static_cast<const char*>(tensor.const_data_ptr());
  const auto strides = tensor.strides();
  const auto element_sz = tensor.element_size();
  int64_t off = 0;
  std::vector<int64_t> split_base_addrs;
  split_base_addrs.reserve(split_sizes.size());
  for (const auto& split_size : split_sizes) {
    split_base_addrs.push_back(reinterpret_cast<int64_t>(data_ptr + off));
    off += split_size * strides[dim] * element_sz;
  }
  return split_base_addrs;
}

static inline std::vector<int64_t> get_dst_addrs(at::TensorList out) {
  std::vector<int64_t> addrs;
  addrs.reserve(out.size());
  for (const auto& tensor : out) {
    addrs.push_back(reinterpret_cast<int64_t>(tensor.data_ptr()));
  }
  return addrs;
}

// Calculate the chunk size for each split in bytes.
static inline std::vector<int64_t> get_split_chunk_sizes(
    const at::Tensor& tensor,
    at::IntArrayRef split_sizes,
    int64_t dim) {
  const auto stride = tensor.stride(dim);
  const auto element_sz = tensor.element_size();
  std::vector<int64_t> split_chunk_sizes;
  split_chunk_sizes.reserve(split_sizes.size());
  for (const auto& split_size : split_sizes) {
    split_chunk_sizes.push_back(split_size * stride * element_sz);
  }
  return split_chunk_sizes;
}

// Calculate the chunk stride in bytes. This is the same for all splits.
static inline int64_t get_chunk_stride(const at::Tensor& tensor, int64_t dim) {
  int64_t stride = 1;
  for (int64_t d = dim; d < tensor.dim(); ++d) {
    stride *= tensor.sizes()[d];
  }
  return stride * tensor.element_size();
}

// Calculate the number of chunks. This is the same for all splits.
static inline int64_t get_num_chunks(const at::Tensor& tensor, int64_t dim) {
  int64_t num_chunks = tensor.numel();
  for (int64_t d = dim; d < tensor.dim(); ++d) {
    num_chunks /= tensor.sizes()[d];
  }
  return num_chunks;
}

// Pack multiple std::vector<int64_t> into a single cuda tensor.
std::pair<at::Tensor, std::vector<int64_t*>> pack_vecs(
    std::vector<const std::vector<int64_t>*> vecs,
    const at::Device& device) {
  int64_t numel = 0;
  for (const auto* vec : vecs) {
    numel += vec->size();
  }

  auto packed = at::empty(
      {numel}, at::TensorOptions().dtype(at::kLong).pinned_memory(true));
  size_t offset = 0;
  for (const auto* vec : vecs) {
    memcpy(
        packed.data_ptr<int64_t>() + offset,
        vec->data(),
        sizeof(int64_t) * vec->size());
    offset += vec->size();
  }
  packed = packed.to(device, /*non_blocking=*/true);

  std::vector<int64_t*> ptrs;
  ptrs.reserve(vecs.size());
  offset = 0;
  for (const auto* vec : vecs) {
    ptrs.push_back(packed.data_ptr<int64_t>() + offset);
    offset += vec->size();
  }
  return std::make_pair(std::move(packed), std::move(ptrs));
}

static inline std::vector<int64_t> get_chunk_cat_out_sizes(
    IntArrayRef input_tensor_sizes,
    int64_t dim,
    int64_t num_chunks,
    int64_t chunk_size,
    int64_t out_element_size) {
  std::vector<int64_t> view_sizes = std::vector<int64_t>(
      input_tensor_sizes.begin(), input_tensor_sizes.begin() + dim);
  view_sizes.insert(
      view_sizes.end(), {num_chunks, chunk_size / out_element_size});
  return view_sizes;
}

// Copy `max_chunk_size` bytes from `src` to `dst` by `num_threads`, and pad
// zero when `src` size (i.e., actual_chunk_size) is less than `max_chunk_size`.
// Assume elements of src and dst have the same data type.
template <typename dst_t, typename src_t>
__device__ __inline__ void copy_chunk_with_pad(
    dst_t* dst_ptr,
    src_t* src_ptr,
    int64_t max_chunk_size,
    int64_t actual_chunk_size,
    int64_t thread_idx,
    int64_t num_threads) {
  // Supports type cast
  if (!std::is_same_v<dst_t, src_t>) {
    const int64_t max_num_elems = max_chunk_size / sizeof(dst_t);
    const int64_t actual_num_elems = actual_chunk_size / sizeof(src_t);
    int64_t elem_index = thread_idx;
    while (elem_index < actual_num_elems) {
      dst_ptr[elem_index] =
          static_cast_with_inter_type<dst_t, src_t>::apply(src_ptr[elem_index]);
      elem_index += num_threads;
    }
    while (elem_index < max_num_elems) {
      dst_ptr[elem_index] = static_cast_with_inter_type<dst_t, int>::apply(0);
      elem_index += num_threads;
    }
    return;
  }
  char* dst = reinterpret_cast<char*>(dst_ptr);
  char* src = reinterpret_cast<char*>(src_ptr);
  // Fast path when the number of threads is larger than the number of bytes to
  // be copied (i.e., max_chunk_size). In this case, each thread only copies 1
  // byte. For 0 <= thread_idx < actual_chunk_size, the thread copies data from
  // `src`. For actual_chunk_size <= thread_idx < max_chunk_size, the thread set
  // the val=0 for padding.
  if (max_chunk_size < num_threads) {
    char val = static_cast<char>(0);
    if (thread_idx < actual_chunk_size) {
      val = src[thread_idx];
    }
    if (thread_idx < max_chunk_size) {
      dst[thread_idx] = val;
    }
    return;
  }
  // Split dst array into three parts:
  // [dst, dst+align_off), [dst+align_off, dst+align_end), [dst+align_end,
  // dst+max_chunk_size) The second part is aligned with BYTES_PER_THREAD(=16
  // bytes) to enable `stream_store128`.
  int64_t align_off, aligned_size;
  get_aligned_region(
      dst, actual_chunk_size, BYTES_PER_THREAD, align_off, aligned_size);
  int64_t align_end = align_off + aligned_size;
  for (int64_t i = align_off + thread_idx * BYTES_PER_THREAD; i < align_end;
       i += num_threads * BYTES_PER_THREAD) {
    uint4 val;
    if (is_aligned<uint4>(src + i)) {
      stream_load128(val, src + i);
    } else {
      for (size_t j = 0; j < BYTES_PER_THREAD; ++j) {
        reinterpret_cast<char*>(&val)[j] = src[i + j];
      }
    }
    stream_store128(&dst[i], val);
  }
  // Copy data for the first part of dst array [dst, dst+align_off).
  // Check `thread_idx<max_chunk_sze` for the edge case that max_chunk_size <
  // align_off.
  if (thread_idx < align_off && thread_idx < max_chunk_size) {
    char val = (char)0;
    if (thread_idx < actual_chunk_size) {
      val = src[thread_idx];
    }
    dst[thread_idx] = val;
  }
  // Copy data for the third part of dst array [dst+align_end,
  // dst+max_chunk_size).
  while (align_end + thread_idx < max_chunk_size) {
    char val = (char)0;
    if (align_end + thread_idx < actual_chunk_size) {
      val = src[align_end + thread_idx];
    }
    dst[align_end + thread_idx] = val;
    align_end += num_threads;
  }
}

// NOTE [CUDA kernel for chunk_cat]
// chunk_cat_cuda adopts a "jagged grid" strategy, inspired by NOTE [CUDA fast
// path for split_with_sizes_copy.out]. In addition, chunk_cat_cuda supports
// padding via copy_chunk_with_pad when src chunk size is less than dst chunk
// size.
template <typename dst_t, typename src_t>
static __global__ void chunk_cat_cuda_kernel(
    src_t** src,
    dst_t* dst,
    int64_t* block_idx_to_tensor_idx,
    int64_t* tensor_idx_to_start_tensor_bytes,
    int64_t* start_block_idx_per_tensor_chunk,
    int64_t* actual_tensor_sizes,
    int64_t* pad_tensor_chunk_sizes,
    int64_t* num_blocks_per_tensor_chunk,
    int64_t slice_size,
    int64_t chunk_size,
    int64_t dst_to_src_ratio) {
  const int64_t slice_idx = blockIdx.z;
  const int64_t chunk_idx = blockIdx.y;
  const int64_t tensor_idx = block_idx_to_tensor_idx[blockIdx.x];
  const int64_t tile_idx =
      blockIdx.x - start_block_idx_per_tensor_chunk[tensor_idx];
  // Number of threads for the `tensor_idx`-th tensor chunk.
  const int64_t num_threads =
      num_blocks_per_tensor_chunk[tensor_idx] * BLOCK_SIZE;
  const int64_t thread_idx = tile_idx * BLOCK_SIZE + threadIdx.x;
  char* src_addr = reinterpret_cast<char**>(src)[tensor_idx] +
      slice_idx * actual_tensor_sizes[tensor_idx] +
      chunk_idx * pad_tensor_chunk_sizes[tensor_idx] / dst_to_src_ratio;
  char* dst_addr = reinterpret_cast<char*>(dst) + slice_idx * slice_size +
      chunk_idx * chunk_size + tensor_idx_to_start_tensor_bytes[tensor_idx];
  // Compute the actual number of bytes to copy from src.
  const int64_t actual_copy_size = std::min(
      pad_tensor_chunk_sizes[tensor_idx] / dst_to_src_ratio,
      std::max(
          (int64_t)0,
          actual_tensor_sizes[tensor_idx] -
              chunk_idx * pad_tensor_chunk_sizes[tensor_idx] /
                  dst_to_src_ratio));
  copy_chunk_with_pad<dst_t, src_t>(
      reinterpret_cast<dst_t*>(dst_addr),
      reinterpret_cast<src_t*>(src_addr),
      pad_tensor_chunk_sizes[tensor_idx],
      actual_copy_size,
      thread_idx,
      num_threads);
}

bool all_contiguous(TensorList tensors) {
  for (const auto& t : tensors) {
    if (!t.is_contiguous()) {
      return false;
    }
  }
  return true;
}

// Get leading dimensions before `dim`-th dimension.
static inline int64_t get_leading_dim(at::IntArrayRef sizes, int64_t dim) {
  int64_t leading_dim = 1;
  if (dim > 0) {
    leading_dim = c10::multiply_integers(sizes.slice(0, dim));
  }
  return leading_dim;
}

// Get trailing dimensions after `dim`-th dimension and padded size along
// `dim`-th dimension.
static inline std::pair<int64_t, int64_t> get_pad_size(
    at::IntArrayRef sizes,
    int64_t dim,
    int64_t num_chunks) {
  int64_t trailing_numel = 1;
  if (sizes.size() > (uint64_t)dim + 1) {
    trailing_numel =
        c10::multiply_integers(sizes.slice(dim + 1, sizes.size() - dim - 1));
  }
  int64_t pad_size_along_dim =
      detail::div_up(sizes[dim], num_chunks) * num_chunks;
  return std::make_pair(pad_size_along_dim, trailing_numel);
}

// Get the padded chunk size.
static inline int64_t get_chunk_size(
    TensorList tensors,
    int64_t dim,
    int64_t num_chunks,
    int64_t elem_size) {
  auto num_tensors = tensors.size();
  int64_t chunk_size = 0;
  for (const auto i : c10::irange(num_tensors)) {
    auto [pad_size_along_dim, trailing_numel] =
        get_pad_size(tensors[i].sizes(), dim, num_chunks);
    const int64_t pad_tensor_chunk_size =
        pad_size_along_dim * trailing_numel * elem_size / num_chunks;
    chunk_size += pad_tensor_chunk_size;
  }
  return chunk_size;
}

// Get metadata for chunk_cat.
std::tuple<
    int64_t,
    int64_t,
    int64_t,
    int64_t,
    std::vector<int64_t>,
    std::vector<int64_t>,
    std::vector<int64_t>,
    std::vector<int64_t>,
    std::vector<int64_t>,
    std::vector<int64_t>,
    std::vector<int64_t>>
get_chunk_cat_metadata(
    TensorList tensors,
    int64_t dim,
    int64_t num_chunks,
    int64_t dst_elem_size,
    int64_t src_elem_size) {
  TORCH_CHECK(
      dst_elem_size % src_elem_size == 0,
      "get_chunk_cat_metadata error: only support dst_elem_size % src_elem_size == 0");
  auto num_tensors = tensors.size();
  int64_t leading_dim = get_leading_dim(tensors[0].sizes(), dim);
  std::vector<int64_t> pad_tensor_chunk_sizes;
  std::vector<int64_t> num_blocks_per_tensor_chunk;
  std::vector<int64_t> start_block_idx_per_tensor_chunk{0};
  std::vector<int64_t> actual_tensor_sizes;
  std::vector<int64_t> tensor_idx_to_start_tensor_bytes{0};
  std::vector<int64_t> srcs;
  pad_tensor_chunk_sizes.reserve(num_tensors);
  num_blocks_per_tensor_chunk.reserve(num_tensors);
  start_block_idx_per_tensor_chunk.reserve(num_tensors + 1);
  actual_tensor_sizes.reserve(num_tensors);
  tensor_idx_to_start_tensor_bytes.reserve(num_tensors + 1);
  srcs.reserve(num_tensors);
  // block_idx_to_tensor_idx cannot be reserved since the number of blocks is
  // data dependent
  std::vector<int64_t> block_idx_to_tensor_idx;
  // Inline computing `chunk_size` to avoid redundant computation
  int64_t chunk_size = 0;
  for (const auto i : c10::irange(num_tensors)) {
    at::Tensor tensor = tensors[i];
    srcs.push_back(reinterpret_cast<int64_t>(tensor.data_ptr()));
    auto sizes = tensor.sizes();
    auto [pad_size_along_dim, trailing_numel] =
        get_pad_size(sizes, dim, num_chunks);
    const int64_t pad_tensor_chunk_size =
        pad_size_along_dim * trailing_numel * dst_elem_size / num_chunks;
    pad_tensor_chunk_sizes.push_back(pad_tensor_chunk_size);
    chunk_size += pad_tensor_chunk_size;
    // Number of blocks required to process this tensor chunk.
    const int64_t num_blocks =
        detail::div_up(pad_tensor_chunk_size, detail::BYTES_PER_BLOCK);
    num_blocks_per_tensor_chunk.push_back(num_blocks);
    start_block_idx_per_tensor_chunk.push_back(
        start_block_idx_per_tensor_chunk.back() + num_blocks);
    block_idx_to_tensor_idx.insert(
        block_idx_to_tensor_idx.end(), num_blocks, i);
    tensor_idx_to_start_tensor_bytes.push_back(
        tensor_idx_to_start_tensor_bytes.back() + pad_tensor_chunk_size);
    actual_tensor_sizes.push_back(sizes[dim] * trailing_numel * src_elem_size);
  }
  const int64_t num_blocks_per_chunk = start_block_idx_per_tensor_chunk.back();
  const int64_t slice_size = num_chunks * chunk_size;
  return std::make_tuple(
      chunk_size,
      leading_dim,
      num_blocks_per_chunk,
      slice_size,
      srcs,
      block_idx_to_tensor_idx,
      tensor_idx_to_start_tensor_bytes,
      start_block_idx_per_tensor_chunk,
      actual_tensor_sizes,
      pad_tensor_chunk_sizes,
      num_blocks_per_tensor_chunk);
}

// See [CUDA kernel for chunk_cat_cuda]
template <typename dst_t, typename src_t>
void _chunk_cat_out_cuda_contiguous(
    TensorList tensors,
    int64_t dim,
    int64_t num_chunks,
    Tensor& out,
    int64_t dst_elem_size,
    int64_t src_elem_size) {
  const auto device = tensors[0].device();
  // `get_chunk_cat_metadata` must return vectors and `pack_vecs` cannot be
  // moved into `get_chunk_cat_metadata`. Otherwise `packed` would point to
  // vectors allocated inside `get_chunk_cat_metadata` which become out of local
  // scope.
  auto
      [chunk_size,
       leading_dim,
       num_blocks_per_chunk,
       slice_size,
       srcs,
       block_idx_to_tensor_idx,
       tensor_idx_to_start_tensor_bytes,
       start_block_idx_per_tensor_chunk,
       actual_tensor_sizes,
       pad_tensor_chunk_sizes,
       num_blocks_per_tensor_chunk] =
          get_chunk_cat_metadata(
              tensors, dim, num_chunks, dst_elem_size, src_elem_size);
  auto packed = pack_vecs(
      {&srcs,
       &block_idx_to_tensor_idx,
       &tensor_idx_to_start_tensor_bytes,
       &start_block_idx_per_tensor_chunk,
       &actual_tensor_sizes,
       &pad_tensor_chunk_sizes,
       &num_blocks_per_tensor_chunk},
      device);
  std::vector<int64_t> view_sizes = get_chunk_cat_out_sizes(
      tensors[0].sizes(), dim, num_chunks, chunk_size, dst_elem_size);
  at::native::resize_output(out, view_sizes);
  dim3 blocks(num_blocks_per_chunk, num_chunks, leading_dim);
  dim3 threads(detail::BLOCK_SIZE, 1, 1);
  detail::chunk_cat_cuda_kernel<<<
      blocks,
      threads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      /*srcs=*/reinterpret_cast<src_t**>(packed.second[0]),
      reinterpret_cast<dst_t*>(out.data_ptr()),
      /*block_idx_to_tensor_idx=*/packed.second[1],
      /*tensor_idx_to_start_tensor_bytes=*/packed.second[2],
      /*start_block_idx_per_tensor_chunk=*/packed.second[3],
      /*actual_tensor_sizes=*/packed.second[4],
      /*pad_tensor_chunk_sizes=*/packed.second[5],
      /*num_blocks_per_tensor_chunk=*/packed.second[6],
      slice_size,
      chunk_size,
      dst_elem_size / src_elem_size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

} // namespace detail

// See [CUDA fast path for split_with_sizes_copy.out]
void split_with_sizes_copy_out_cuda_contiguous_no_cast(
    const at::Tensor& self,
    at::IntArrayRef split_sizes,
    int64_t dim,
    at::TensorList out) {
  const auto device = self.device();
  const auto src_base_addrs =
      detail::get_split_base_addrs(self, split_sizes, dim);
  const auto dst_base_addrs = detail::get_dst_addrs(out);
  const auto src_stride = detail::get_chunk_stride(self, dim);
  const auto split_chunk_sizes =
      detail::get_split_chunk_sizes(self, split_sizes, dim);
  const auto num_chunks = detail::get_num_chunks(self, dim);

  // Calculate the number of blocks required for the first chunk across all
  // splits, assuming each thread only processes BYTES_PER_THREAD bytes.
  int64_t num_blocks = 0;
  for (const auto& split_chunk_size : split_chunk_sizes) {
    num_blocks += detail::div_up(
        split_chunk_size, detail::BLOCK_SIZE * detail::BYTES_PER_THREAD);
  }

  // Calculate the maximum number of blocks to launch. Only consider
  // maxThreadsPerMultiProcessor as a limiting factor as the kernel uses no
  // shared memory and little registers. Over-subscribe the SMs to hide I/O
  // latency.
  const auto num_sms =
      at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  const auto max_threads_per_sm =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor;
  const int64_t max_blocks =
      num_sms * max_threads_per_sm / detail::BLOCK_SIZE * 2.0;

  // Make each thread process BYTES_PER_THREAD * iter_factor bytes to regulate
  // block size. Spread iter_factor evenly between chunks_per_block and
  // iters_per_chunk.
  int64_t iter_factor = detail::div_up(num_blocks * num_chunks, max_blocks);
  int64_t chunks_per_block = std::ceil(std::sqrt(iter_factor));
  chunks_per_block = std::min(chunks_per_block, num_chunks);
  const int64_t iters_per_chunk = detail::div_up(iter_factor, chunks_per_block);

  // Launch a logically jagged grid of shape
  // (chunk_size*, num_splits, num_chunks / chunks_per_block)
  // backed by a physical grid of shape
  // (sum(chunk_size), num_chunks / chunks_per_block).
  // A block can find its split_idx via block_idx_to_split_idx.
  std::vector<int64_t> block_idx_to_split_idx;
  std::vector<int64_t> blocks_cumsums{0};
  block_idx_to_split_idx.reserve(num_blocks);
  for (size_t split_idx = 0; split_idx < split_sizes.size(); ++split_idx) {
    const auto blocks = detail::div_up(
        split_chunk_sizes[split_idx],
        detail::BLOCK_SIZE * detail::BYTES_PER_THREAD * iters_per_chunk);
    block_idx_to_split_idx.insert(
        block_idx_to_split_idx.end(), blocks, split_idx);
    blocks_cumsums.push_back(blocks_cumsums.back() + blocks);
  }

  dim3 blocks(blocks_cumsums.back(), num_chunks / chunks_per_block, 1);
  dim3 threads(detail::BLOCK_SIZE, 1, 1);

  auto [_, ptrs] = detail::pack_vecs(
      {&dst_base_addrs,
       &src_base_addrs,
       &split_chunk_sizes,
       &block_idx_to_split_idx,
       &blocks_cumsums},
      device);

  detail::split_with_sizes_copy_out_contiguous_no_cast_kernel<<<
      blocks,
      threads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      /*dst_base_addrs=*/reinterpret_cast<char**>(ptrs[0]),
      /*src_base_addrs=*/reinterpret_cast<char**>(ptrs[1]),
      /*split_chunk_sizes=*/ptrs[2],
      /*block_idx_to_split_idx=*/ptrs[3],
      /*blocks_cumsums=*/ptrs[4],
      src_stride,
      num_chunks);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void split_with_sizes_copy_out_cuda(
    const Tensor& self,
    IntArrayRef split_sizes,
    int64_t dim,
    TensorList out) {
  const bool is_capturing = at::cuda::currentStreamCaptureStatusMayInitCtx() !=
      at::cuda::CaptureStatus::None;
  bool contiguous_no_cast = self.is_non_overlapping_and_dense();
  for (const auto& t : out) {
    contiguous_no_cast &= t.is_non_overlapping_and_dense();
    contiguous_no_cast &= (t.dtype() == self.dtype());
  }
  // TODO(yifu): make the fast path work for CUDA graph
  if (!is_capturing && contiguous_no_cast) {
    // Perform equivalent checks performed by the composite impl
    if (dim < 0) {
      dim = at::maybe_wrap_dim(dim, self.dim());
    }
    TORCH_CHECK(
        self.dim() != 0, "split expects at least a 1-dimensional tensor")

    const int64_t dim_size = self.size(dim);
    int64_t split_sizes_sum = 0;
    for (const auto i : c10::irange(split_sizes.size())) {
      TORCH_CHECK(
          split_sizes[i] >= 0,
          "split_with_sizes expects split_sizes have only non-negative ",
          "entries, but got split_sizes=",
          split_sizes[i]);
      split_sizes_sum += split_sizes[i];
    }
    TORCH_CHECK(
        split_sizes_sum == dim_size,
        "split_with_sizes expects split_sizes to sum exactly to ",
        dim_size,
        " (input tensor's size at dimension ",
        dim,
        "), ",
        "but got split_sizes=",
        split_sizes);

    TORCH_CHECK(
        out.size() == split_sizes.size(),
        "split_with_sizes_copy_out() expected an out= argument of size ",
        split_sizes.size(),
        ", got size ",
        out.size());

    auto out_shape = self.sizes().vec();
    for (const auto i : c10::irange(split_sizes.size())) {
      out_shape[dim] = split_sizes[i];
      if (resize_output_check(out[i], out_shape)) {
        out[i].resize_(out_shape);
      }
      TORCH_CHECK(
          out[i].dtype() == self.dtype(),
          "Expected out tensor to have dtype ",
          self.dtype(),
          ", but got ",
          out[i].dtype(),
          " instead");
      TORCH_CHECK(
          out[i].device() == self.device(),
          "Expected out tensor to have device ",
          self.device(),
          ", but got ",
          out[i].device(),
          " instead");
    }
    split_with_sizes_copy_out_cuda_contiguous_no_cast(
        self, split_sizes, dim, out);
  } else {
    at::native::split_with_sizes_copy_out(self, split_sizes, dim, out);
  }
}

Tensor _chunk_cat_cuda(TensorList tensors, int64_t dim, int64_t num_chunks) {
  dim = at::native::preprocess_chunk_cat_inputs(tensors, dim, num_chunks);
  if (detail::all_contiguous(tensors)) {
    // Return a tensor with the same dtype as input tensors
    int64_t elem_size = tensors[0].element_size();
    int64_t chunk_size =
        detail::get_chunk_size(tensors, dim, num_chunks, elem_size);
    int64_t leading_dim = detail::get_leading_dim(tensors[0].sizes(), dim);
    auto view_sizes = detail::get_chunk_cat_out_sizes(
        tensors[0].sizes(), dim, num_chunks, chunk_size, elem_size);
    Tensor out =
        tensors[0]
            .new_empty(chunk_size * num_chunks * leading_dim / elem_size)
            .view(view_sizes);
    // Type-agnostic copy since out and input tensors have the same type.
    detail::_chunk_cat_out_cuda_contiguous<char, char>(
        tensors, dim, num_chunks, out, elem_size, elem_size);
    return out;
  } else {
    return at::native::_chunk_cat(tensors, dim, num_chunks);
  }
}

Tensor& _chunk_cat_out_cuda(
    TensorList tensors,
    int64_t dim,
    int64_t num_chunks,
    Tensor& out) {
  dim = at::native::preprocess_chunk_cat_inputs(tensors, dim, num_chunks);
  TORCH_CHECK(
      tensors[0].device() == out.device(),
      "_chunk_cat_out_cuda: mismatch between input and out tensor devices");
  bool both_input_output_contiguous =
      detail::all_contiguous(tensors) && out.is_non_overlapping_and_dense();
  if (both_input_output_contiguous &&
      (tensors[0].dtype() == at::ScalarType::BFloat16) &&
      (out.dtype() == at::ScalarType::Float)) {
    // _chunk_cat_out_cuda_contiguous should also support other types, thanks to
    // static_cast_with_inter_type. Here, we dispatch to BFloat16 in and float32
    // out since it is the only known use case.
    detail::_chunk_cat_out_cuda_contiguous<float, BFloat16>(
        tensors,
        dim,
        num_chunks,
        out,
        out.element_size(),
        tensors[0].element_size());
  } else if (
      both_input_output_contiguous && tensors[0].dtype() == out.dtype()) {
    // Type-agnostic copy since out and input tensors have the same type.
    detail::_chunk_cat_out_cuda_contiguous<char, char>(
        tensors,
        dim,
        num_chunks,
        out,
        out.element_size(),
        tensors[0].element_size());
  } else {
    at::native::_chunk_cat_out(tensors, dim, num_chunks, out);
  }
  return out;
}

} // namespace at::native
