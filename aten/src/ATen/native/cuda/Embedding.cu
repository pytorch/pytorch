#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/ceil_div.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Exception.h>
#include <c10/macros/Macros.h>

#include <ATen/cuda/cub.cuh>

#include <ATen/native/cuda/EmbeddingBackwardKernel.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <ATen/native/cuda/block_reduce.cuh>
#include <ATen/native/cuda/thread_constants.h>

#if CUB_SUPPORTS_SCAN_BY_KEY()
#include <thrust/iterator/reverse_iterator.h>
#endif

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/arange.h>
#include <ATen/ops/embedding_dense_backward_native.h>
#include <ATen/ops/embedding_renorm_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/zeros.h>
#endif

namespace at::native {

namespace {

#if defined(USE_ROCM)
static const int BLOCKDIMY = 16;
#else
static const int BLOCKDIMY = 32;
#endif

template
  <typename scalar_t,
   typename accscalar_t,
   typename index_t>
__global__ void embedding_backward_feature_kernel
  (const index_t* indices,
   const scalar_t* __restrict__ grad,
   scalar_t* __restrict__ grad_weight,
   int n, // OK to pass as int, we don't expect 2 billion+ samples in one shot
   int64_t stride,
   int padding_idx)
{
  extern __shared__ char buf[];
  accscalar_t* smem = (accscalar_t*)buf;
  accscalar_t* my_s = smem + C10_WARP_SIZE*threadIdx.y;
  int* indices_batch = (int*)(buf + sizeof(accscalar_t)*C10_WARP_SIZE*blockDim.y);

  const int s = (int)stride; // OK to make int, we don't expect 2 billion+ embedding row size

  const int f = threadIdx.x + blockIdx.x*blockDim.x; // feature_dim

  for(int batch_start = 0; batch_start < n; batch_start += blockDim.x*blockDim.y)
  {
    // Entire block cooperates to load a batch of 1024 indices to process
    int tid = threadIdx.x + threadIdx.y*blockDim.x;
    if(batch_start + tid < n)
      indices_batch[tid] = (int)indices[batch_start + tid];

    int batch_end = batch_start + blockDim.x*blockDim.y < n ?
                    batch_start + blockDim.x*blockDim.y : n;

    // Loop over the batch of <= 1024 loaded indices in chunks of blockDim.y = 32
    for(int chunk_start = batch_start; chunk_start < batch_end; chunk_start += blockDim.y)
    {
      // This does double duty:  it makes sure indices_batch is ready, and it makes sure match-group
      // leaders are done with their accumulates before other warps start loading again.
      __syncthreads();

      int n_this_chunk = (batch_end - chunk_start) < blockDim.y ?
                         (batch_end - chunk_start) : blockDim.y;

      int src_row = chunk_start + threadIdx.y;
      int dst_row = indices_batch[src_row - batch_start]; // This warp's target row in grad_weight

      // All warps load their smem segments with incoming grad data
      if(src_row < n && f < s && dst_row != padding_idx)
        my_s[threadIdx.x] = static_cast<accscalar_t>(grad[src_row*stride + f]);

      __syncthreads();

      // To ensure determinism, we can't just have each warp add its grad data to its dst_row.
      // We need to check if any other warps pulled grad data targeting dst_row.
      // If so, we elect the first warp in each matching group as the leader.
      // Each leader warp serializes the accumulates targeting dst_row in shared memory,
      // then finishes by adding the accumulated buffer to dst_row in grad_weight.
      if(dst_row != padding_idx && src_row < n) // Per-warp exit condition, safe with ballot_sync
      {
        int match_found_this_thread = 0;
        if(threadIdx.x < n_this_chunk)
          match_found_this_thread = (dst_row == indices_batch[chunk_start - batch_start + threadIdx.x]);
#if defined(USE_ROCM)
        unsigned long long int matchmask = WARP_BALLOT(match_found_this_thread);
        int first_remaining_peer = __ffsll(matchmask) - 1;
#else
        unsigned int matchmask = WARP_BALLOT(match_found_this_thread);
        int first_remaining_peer = __ffs(matchmask) - 1;
#endif

        if(threadIdx.y == first_remaining_peer) // Nominate lowest-indexed warp as the leader
        {
          matchmask ^= (1 << first_remaining_peer);
          while(matchmask)
          {
#if defined(USE_ROCM)
            first_remaining_peer = __ffsll(matchmask) - 1;
#else
            first_remaining_peer = __ffs(matchmask) - 1;
#endif
            my_s[threadIdx.x] += smem[threadIdx.x + C10_WARP_SIZE*first_remaining_peer];
            matchmask ^= (1 << first_remaining_peer);
          }
          if(f < s)
            grad_weight[dst_row*stride + f] += static_cast<scalar_t>(my_s[threadIdx.x]);
        }
      }
    }
  }
}


template <typename scalar_t, typename index_t>
__global__ void embedding_backward_kernel(
  index_t* input, index_t* indices, scalar_t* grad_output, scalar_t* grad_weight,
  index_t* count, int64_t numel, int64_t stride, int padding_idx) {

  using accscalar_t = acc_type<scalar_t, true>;
  int idx = blockIdx.x * 4 + threadIdx.y;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceding input has the same as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values processed by each thread (grain size)
  const int SZ = 4;

  if (idx < numel
      && (idx == 0 || input[idx] != input[idx - 1])
      && input[idx] != padding_idx) {
    do {
      const int start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
      const int weight_row = ((int) input[idx]) * stride;
      const int grad_row = ((int) indices[idx]) * stride;
      const accscalar_t scale = count ? (accscalar_t)1.0 / count[idx] : 1.0;

      accscalar_t gradient[SZ];
      accscalar_t weight[SZ];

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        int feature_dim = start_feature + ii * C10_WARP_SIZE;
        if (feature_dim < stride) {
          gradient[ii] = static_cast<accscalar_t>(grad_output[grad_row + feature_dim]);
          weight[ii] = static_cast<accscalar_t>(grad_weight[weight_row + feature_dim]);
        }
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        weight[ii] += gradient[ii] * scale;
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        int feature_dim = start_feature + ii * C10_WARP_SIZE;
        if (feature_dim < stride) {
            grad_weight[weight_row + feature_dim] = static_cast<scalar_t>(weight[ii]);
        }
      }

      idx++;
    } while (idx < numel && input[idx] == input[idx - 1]);
  }
}

/* Calculate norms of the rows of weight_ptr given by idx_ptr and capture them in norms */
template <typename scalar_t, typename accscalar_t, typename index_t>
__global__ void renorm_kernel(
    scalar_t* weights, index_t* indices, accscalar_t max_norm,
    accscalar_t norm_type, int64_t dim,
    int64_t weights_stride0, int64_t weights_stride1,
    const int64_t *num_unique_indices) {
  if (blockIdx.x >= *num_unique_indices) {
    return;
  }

  // Some casting hacks since dynamic shared memory and templates don't work together:
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);

  int tid = threadIdx.x;
  int base_index = indices[blockIdx.x] * weights_stride0;

  accscalar_t v = 0;
  for (int i = tid; i < dim; i += blockDim.x) {
    auto x = static_cast<accscalar_t>(weights[base_index + i * weights_stride1]);
    if (norm_type == 1) {
      v += std::abs(x);
    } else if (norm_type == 2) {
      v += x * x;
    } else {
      v += std::pow(x, norm_type);
    }
  }

  v = cuda_utils::BlockReduceSum(v, sdata);

  if (tid == 0) {
    sdata[0] = std::pow(v, static_cast<accscalar_t>(1.0 / norm_type));
  }
  __syncthreads();

  // now we renormalize the blocks that need it
  if (sdata[0] > max_norm) {
    auto factor = static_cast<scalar_t>(max_norm / (sdata[0] + 1e-7));
    for (int i = tid; i < dim; i += blockDim.x) {
      weights[base_index + i * weights_stride1] *= factor;
    }
  }
}

} // anonymous namespace

#if !CUB_SUPPORTS_SCAN_BY_KEY()
template<typename index_t>
void embedding_dense_backward_cuda_scan(Tensor &sorted_indices, Tensor &count);
#endif

Tensor embedding_dense_backward_cuda(const Tensor & grad_, const Tensor & indices_,
                               int64_t num_weights, int64_t padding_idx,
                               bool scale_grad_by_freq) {
  auto grad_arg = TensorArg(grad_, "grad", 1);
  auto indices_arg = TensorArg(indices_, "indices", 1);
  checkScalarTypes("embedding_backward", indices_arg, {kLong, kInt});
  checkSameGPU("embedding_backward", grad_arg, indices_arg);

  auto indices = indices_.contiguous();

  auto num_indices = indices.numel();
  auto grad = grad_.contiguous().view({num_indices, grad_.size(-1)});
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (num_indices <= 3072 && !scale_grad_by_freq) {
    auto indices_contig = indices.contiguous();
    auto grad_weight = at::zeros({num_weights, grad_.size(-1)}, grad_.options());
    int64_t stride = grad_weight.stride(0);
    int warp_size = at::cuda::warp_size();
    dim3 grid(ceil_div(stride, (int64_t)warp_size));
    dim3 block(warp_size, BLOCKDIMY);

    AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      grad.scalar_type(),
       "embedding_backward",
       [&]
       {
          using accscalar_t = acc_type<scalar_t, true>;
          AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_dense_backward_cuda", [&] () {
          embedding_backward_feature_kernel<scalar_t, accscalar_t, index_t>
            <<<grid,
                block,
                sizeof(accscalar_t)*warp_size*BLOCKDIMY + sizeof(int)*warp_size*BLOCKDIMY,
                stream>>>
            (indices_contig.const_data_ptr<index_t>(),
              grad.const_data_ptr<scalar_t>(),
              grad_weight.mutable_data_ptr<scalar_t>(),
              static_cast<int>(num_indices),
              static_cast<int64_t>(stride),
              static_cast<int>(padding_idx));
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
       });
    return grad_weight;
  }

  auto sorted_indices = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto orig_indices = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor count;
  AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_dense_backward_cuda", [&] () {
    auto range = at::arange(num_indices, indices.options());
    int64_t nbits = cuda::hipcub::get_num_bits(num_weights);
    cuda::hipcub::radix_sort_pairs(
      indices.const_data_ptr<index_t>(), sorted_indices.mutable_data_ptr<index_t>(),
      range.const_data_ptr<index_t>(), orig_indices.mutable_data_ptr<index_t>(),
      num_indices, false/*, 0, nbits*/);
  });

  if (scale_grad_by_freq) {
    count = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
#if CUB_SUPPORTS_SCAN_BY_KEY()
    AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_dense_backward_cuda", [&] () {
      hipStream_t stream = at::cuda::getCurrentCUDAStream();

      // Compute an increasing sequence per unique item in sortedIndices:
      // sorted: 2 5 5 5 7 7 8 9 9
      //  count: 1 1 2 3 1 2 1 1 2
      auto sorted_data = sorted_indices.const_data_ptr<index_t>();
      auto count_data = count.mutable_data_ptr<index_t>();
      cuda::hipcub::inclusive_sum_by_key(
        sorted_data,
        ATEN_CUB_CONSTANT_ITERATOR(index_t)(1),
        count_data,
        num_indices
      );

      // Take the maximum of each count per unique key in reverse:
      // sorted: 2 5 5 5 7 7 8 9 9
      //  count: 1 3 3 3 2 2 1 2 2
      cuda::hipcub::inclusive_scan_by_key(
        thrust::make_reverse_iterator(sorted_data + num_indices),
        thrust::make_reverse_iterator(static_cast<const index_t*>(count_data) + num_indices),
        thrust::make_reverse_iterator(count_data + num_indices),
        ATEN_CUB_MAXIMUM(),
        num_indices
      );
    });
#else
    AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_dense_backward_cuda", [&] () {
      embedding_dense_backward_cuda_scan<index_t>(sorted_indices, count);
    });
#endif
  }

  return embedding_backward_cuda_kernel(grad, orig_indices,
      sorted_indices, count, num_weights, padding_idx);
}

Tensor & embedding_renorm_cuda_(Tensor & self, const Tensor & indices,
                                double max_norm, double norm_type) {
  auto self_arg = TensorArg(self, "self", 1);
  auto indices_arg = TensorArg(indices, "indices", 1);
  checkDim("embedding_renorm_", self_arg, 2);
  checkSameGPU("embedding_renorm", self_arg, indices_arg);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_renorm_cuda_", [&] () {

    auto num_indices = indices.numel();
    auto indices_contig = std::get<0>(indices.sort()).contiguous();
    auto unique_indices = at::empty(indices.numel(), indices.options());
    auto num_unique_indices = at::empty({}, indices.options().dtype(kLong));

    cuda::hipcub::unique(
      indices_contig.const_data_ptr<index_t>(),
      unique_indices.mutable_data_ptr<index_t>(),
      num_unique_indices.mutable_data_ptr<int64_t>(),
      num_indices
    );

    int warp_size = at::cuda::warp_size();
    TORCH_INTERNAL_ASSERT(num_threads() % warp_size == 0 &&
                  num_threads() <= static_cast<uint32_t>(cuda_utils::kCUDABlockReduceMaxThreads()),
                  "BlockReduceSum requires all warps be active");
    const int64_t *num_unique_indices_ptr = num_unique_indices.const_data_ptr<int64_t>();
    dim3 grid = unique_indices.numel();
    dim3 block = num_threads();
    int dim = self.stride(0);

    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "embedding_renorm_cuda_", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      renorm_kernel<<<grid, block, (block.x / warp_size) * sizeof(accscalar_t), stream>>>(
        self.mutable_data_ptr<scalar_t>(),
        unique_indices.const_data_ptr<index_t>(),
        static_cast<accscalar_t>(max_norm),
        static_cast<accscalar_t>(norm_type),
        dim, self.stride(0), self.stride(1),
        num_unique_indices_ptr);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  });
  return self;
}


}  // namespace at::native
