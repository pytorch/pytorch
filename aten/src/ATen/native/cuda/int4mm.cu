#include "hip/hip_runtime.h"
#if (defined(USE_ROCM) && ROCM_VERSION >= 50700) || ((defined(CUDA_VERSION) && CUDA_VERSION >= 12000) && (!defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 800)))
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#if !defined(USE_ROCM)
#include <mma.h>
#endif
#endif
#include <ATen/ATen.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/DeviceGuard.h>
#include <c10/cuda/CUDAGuard.h>


namespace at::native {

template <typename U, typename V>
constexpr __host__ __device__ auto divDown(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral_v<U> && std::is_integral_v<V>, "");
  return (a / b);
}

template <typename U, typename V>
constexpr __host__ __device__ auto divUp(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral_v<U> && std::is_integral_v<V>, "");
  // Overflow safe variant of (a + b - 1) / b
  const uint64_t blocks = a / b + (a % b != 0);
  return blocks;
}

template <typename U, typename V>
constexpr __host__ __device__ auto roundDown(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral_v<U> && std::is_integral_v<V>, "");
  return divDown(a, b) * b;
}

template <typename U, typename V>
constexpr __host__ __device__ auto roundUp(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral_v<U> && std::is_integral_v<V>, "");
  return divUp(a, b) * b;
}

template <typename U, typename V>
constexpr __host__ __device__ bool isEvenDivisor(U a, V b) {
  static_assert(std::is_integral_v<U> && std::is_integral_v<V>, "");
  return (a % V(b) == 0) && ((a / V(b)) >= 1);
}

template <class T>
constexpr __host__ __device__ T pow(T n, int power) {
  return (power > 0 ? n * pow(n, power - 1) : 1);
}

template <class T>
constexpr __host__ __device__ T pow2(int power) {
  return pow(2, power);
}

static_assert(pow2<int>(8) == 256, "pow2");

template <typename T>
constexpr __host__ __device__ int log2(T n, int p = 0) {
  return (n <= 1) ? p : log2(n / 2, p + 1);
}

static_assert(log2(2) == 1, "log2");
static_assert(log2(3) == 1, "log2");
static_assert(log2(4) == 2, "log2");

template <typename T>
constexpr __host__ __device__ bool isPowerOf2(T v) {
  static_assert(std::is_integral_v<T>, "");
  return (v && !(v & (v - 1)));
}

static_assert(isPowerOf2(2048), "isPowerOf2");
static_assert(!isPowerOf2(3333), "isPowerOf2");

template <typename T>
constexpr __host__ __device__ T nextHighestPowerOf2(T v) {
  static_assert(std::is_integral_v<T>, "");
  return (isPowerOf2(v) ? (T)2 * v : ((T)1 << (log2(v) + 1)));
}

static_assert(nextHighestPowerOf2(1) == 2, "nextHighestPowerOf2");
static_assert(nextHighestPowerOf2(2) == 4, "nextHighestPowerOf2");
static_assert(nextHighestPowerOf2(3) == 4, "nextHighestPowerOf2");
static_assert(nextHighestPowerOf2(4) == 8, "nextHighestPowerOf2");

static_assert(nextHighestPowerOf2(15) == 16, "nextHighestPowerOf2");
static_assert(nextHighestPowerOf2(16) == 32, "nextHighestPowerOf2");
static_assert(nextHighestPowerOf2(17) == 32, "nextHighestPowerOf2");

static_assert(
    nextHighestPowerOf2(1536000000u) == 2147483648u,
    "nextHighestPowerOf2");
static_assert(
    nextHighestPowerOf2((size_t)2147483648ULL) == (size_t)4294967296ULL,
    "nextHighestPowerOf2");

template <typename T>
constexpr __host__ __device__ T nextLowestPowerOf2(T v) {
  static_assert(std::is_integral_v<T>, "");
  return (isPowerOf2(v) ? v / (T)2 : ((T)1 << (log2(v))));
}

static_assert(nextLowestPowerOf2(1) == 0, "nextLowestPowerOf2");
static_assert(nextLowestPowerOf2(2) == 1, "nextLowestPowerOf2");
static_assert(nextLowestPowerOf2(3) == 2, "nextLowestPowerOf2");
static_assert(nextLowestPowerOf2(4) == 2, "nextLowestPowerOf2");

static_assert(nextLowestPowerOf2(15) == 8, "nextLowestPowerOf2");
static_assert(nextLowestPowerOf2(16) == 8, "nextLowestPowerOf2");
static_assert(nextLowestPowerOf2(17) == 16, "nextLowestPowerOf2");

inline __host__ __device__ bool isPointerAligned(const void* p, int align) {
  return reinterpret_cast<uintptr_t>(p) % align == 0;
}

// Returns the increment needed to aligned the pointer to the next highest
// aligned address
template <int Align>
inline __host__ __device__ uint32_t getAlignmentRoundUp(const void* p) {
  static_assert(isPowerOf2(Align), "");
  const uint32_t diff = uint32_t(uintptr_t(p) & uintptr_t(Align - 1));
  return diff == 0 ? 0 : uint32_t(Align) - diff;
}

#if defined (__gfx90a__) || defined(__gfx942__)
#define CDNA2_OR_LATER 1
#else
#define CDNA2_OR_LATER 0
#endif

#if (defined(USE_ROCM) && ROCM_VERSION >= 50700) || ((defined(CUDA_VERSION) && CUDA_VERSION >= 12000) && (!defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 800)))

#if defined(USE_ROCM)
// TODO: Support RDNA
constexpr int32_t kWarpSize = 64;

template<typename T, uint32_t Rank>
using VecT = T __attribute__((ext_vector_type(Rank)));

static bool isCDNA2orLater(int index) {
    return at::detail::getCUDAHooks().isGPUArch({"gfx90a", "gfx942"}, index);
}

#else
constexpr int32_t kWarpSize = 32;
#endif

// f16 vector types
struct __align__(2) f16x1 {
  __half vals[1];
};

struct __align__(4) f16x2 {
  __half vals[2];
};

struct __align__(8) f16x4 {
  __half vals[4];
};

struct __align__(16) f16x8 {
  __half vals[8];
};

// bf16 vector types
struct __align__(2) bf16x1 {
  __hip_bfloat16 vals[1];
};

struct __align__(4) bf16x2 {
  __hip_bfloat16 vals[2];
};

struct __align__(8) bf16x4 {
  __hip_bfloat16 vals[4];
};

struct __align__(16) bf16x8 {
  __hip_bfloat16 vals[8];
};

// bf162 vector types
struct __align__(4) bf16x2x1 {
  __hip_bfloat162 vals[1];
};

struct __align__(8) bf16x2x2 {
  __hip_bfloat162 vals[2];
};

struct __align__(16) bf16x2x4 {
  __hip_bfloat162 vals[4];
};

struct __align__(16) bf16x2x4_u32 {
#if defined(USE_ROCM)
  VecT<short, 4> val[2];
#else
  uint32_t vals[4];
#endif
};

struct __align__(8) bf16x2x2_u32 {
#if defined(USE_ROCM)
  VecT<short, 4> val;
#else
  uint32_t vals[2];
#endif
};

struct __align__(4) bf16x2x1_u32 {
  uint32_t vals[1];
};

template <typename T, int N>
struct __align__(sizeof(T) * N) VectorType {
  T vals[N];
};

// from
// https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
inline __device__ bf16x2x4 convert_i4x8_to_bf16x2x4(uint32_t source) {
  bf16x2x4 result;
  constexpr int kElements = 8;

  uint32_t* h = reinterpret_cast<uint32_t*>(&result);
  uint32_t const source_i4s = source;

  // First, we extract the i4s and construct an intermediate fp16 number.
#if !defined(USE_ROCM)
  static constexpr uint32_t immLut = (0xf0 & 0xcc) | 0xaa;
#endif
  static constexpr uint32_t MASK = 0x000f000f;
  static constexpr uint32_t I4s_TO_BF16s_MAGIC_NUM = 0x43004300;

  // We don't have enough mantissa to remove as much shift overhead as FP16, so
  // we must loop. No shift needed for first item.
  uint32_t i4s = source_i4s;

#if defined(USE_ROCM)
  asm volatile("v_and_or_b32 %0, %1, %2, %3"
               : "=v"(h[0])
               : "v"(i4s), "v"(MASK), "v"(I4s_TO_BF16s_MAGIC_NUM));
#else
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[0])
               : "r"(i4s), "n"(MASK), "n"(I4s_TO_BF16s_MAGIC_NUM), "n"(immLut));
#endif

#pragma unroll
  for (int ii = 1; ii < kElements / 2; ++ii) {
    i4s >>= 4; // or is it 8?
    // (i4s & 0x000f000f) | 0x43004300
#if defined(USE_ROCM)
    asm volatile("v_and_or_b32 %0, %1, %2, %3"
        : "=v"(h[ii])
        : "v"(i4s), "v"(MASK), "v"(I4s_TO_BF16s_MAGIC_NUM));
#else
    asm volatile(
        "lop3.b32 %0, %1, %2, %3, %4;\n"
        : "=r"(h[ii])
        : "r"(i4s), "n"(MASK), "n"(I4s_TO_BF16s_MAGIC_NUM), "n"(immLut));
#endif
  }

  // This is the BF16 {-136, -136} represented as an integer.
#if defined(USE_ROCM)
#if ROCM_VERSION >= 60200
  auto BF16_BIAS = __bfloat162bfloat162(__hip_bfloat16(__hip_bfloat16_raw{0xC308}));
  auto BF16_ONE = __bfloat162bfloat162(__hip_bfloat16(__hip_bfloat16_raw{0x3F80}));
#else
  auto BF16_BIAS = __bfloat162bfloat162(__hip_bfloat16{0xC308});
  auto BF16_ONE = __bfloat162bfloat162(__hip_bfloat16{0x3F80});
#endif
#else
  static constexpr uint32_t BF16_BIAS = 0xC308C308;
  static constexpr uint32_t BF16_ONE = 0x3F803F80;
#endif

// Finally, we construct the output numbers.
#pragma unroll
  for (int ii = 0; ii < kElements / 2; ++ii) {
    // Since this section is for Ampere+, we use bf16 fma to do the bias
    // subtraction
#if defined(USE_ROCM)
     result.vals[ii] = __hfma2(result.vals[ii], BF16_ONE, BF16_BIAS);
#else
    asm("fma.rn.bf16x2 %0, %1, %2, %3;\n"
        : "=r"(h[ii])
        : "r"(h[ii]), "r"(BF16_ONE), "r"(BF16_BIAS));
#endif
  }

  return result;
}



enum class KReductionType {
  // No k-reduction is needed between blocks as the number of k-tiles processed
  // per block are exact and we can directly write the output
  None,
};

// Loads the A matrix in 16-bit standard m x k row major layout, and writes
// the C matrix in 16-bit standard m x n row major layout:
//
// size [m][k]
template <KReductionType ReduceType>
struct ALayout_RM {
  static constexpr int32_t kMTileSize = 16;
#if defined(USE_ROCM)
  static constexpr int32_t kNTileSize = 16;
#else
  static constexpr int32_t kNTileSize = 8;
#endif
  static constexpr int32_t kKTileSize = 16;

  template <int KTilesToLoad>
  static __device__ void load(
      const void* A,
      int32_t m,
      int32_t k,
      int32_t mTiles,
      int32_t mTile,
      int32_t kTiles,
      int32_t kTileStart,
      int32_t laneId,
#if defined(USE_ROCM)
      bf16x2x2_u32 out[KTilesToLoad]
#else
      bf16x2x4_u32 out[KTilesToLoad]
#endif
  ) {
#if defined(USE_ROCM)
    const auto mLane = mTile * kMTileSize + (laneId % kMTileSize);
    const auto kLane = kTileStart * kKTileSize + (laneId / kMTileSize) * 4;
#else
    const auto mLane = mTile * kMTileSize + (laneId / 4);
    const auto kLane = kTileStart * kKTileSize + (laneId % 4) * 2;
#endif

    // access
    // [mTile * kMTileSize + (laneId / 4)]
    // [kTileStart * kKTileSize + (laneId % 4) * 2]
    auto aPtr = reinterpret_cast<const __hip_bfloat16*>(A) + mLane * k + kLane;
    bool m0InBounds = mLane < m;

#if !defined(USE_ROCM)
    auto aPtrPlus8Rows = aPtr + 8 * k;

    bool m1InBounds = (mLane + 8) < m;
#endif

#pragma unroll
    for (int i = 0; i < KTilesToLoad; ++i) {
#if defined(USE_ROCM)
      out[i].val = m0InBounds ? *((VecT<short, 4> *)(aPtr + i * kKTileSize)) : VecT<short, 4>{0, 0, 0, 0};
#else
      out[i].vals[0] = m0InBounds
          ? *reinterpret_cast<const uint32_t*>(aPtr + i * kKTileSize)
          : uint32_t(0);
      out[i].vals[1] = m1InBounds
          ? *reinterpret_cast<const uint32_t*>(aPtrPlus8Rows + i * kKTileSize)
          : uint32_t(0);

      out[i].vals[2] = m0InBounds
          ? *reinterpret_cast<const uint32_t*>(aPtr + i * kKTileSize + 8)
          : uint32_t(0);
      out[i].vals[3] = m1InBounds ? *reinterpret_cast<const uint32_t*>(
                                        aPtrPlus8Rows + i * kKTileSize + 8)
                                  : uint32_t(0);
#endif
    }
  }

  static __device__ void store(
      void* C,
      int32_t m,
      int32_t n,
      int32_t mOutTiles,
      int32_t mTile,
      int32_t nOutTiles,
      int32_t nTile,
      int32_t laneId,
      const float4& out) {
    static_assert(ReduceType == KReductionType::None, "");

    if constexpr (ReduceType == KReductionType::None) {
#if defined(USE_ROCM)
      const int outRow = mTile * kMTileSize + (laneId / kNTileSize) * 4;
      const int outCol = nTile * kNTileSize + (laneId % kNTileSize);
#else
      // sum.x / sum.y are written at
      // [laneId / 4], [(laneId % 4) * 2, (laneId % 4) * 2 + 1]
      // sum.z / sum.w are written at
      // [8 + (laneId / 4)], [(laneId % 4) * 2, (laneId % 4) * 2 + 1]
      // i.e., same columns, different row.
      const int outRow = mTile * kMTileSize + (laneId / 4);
      const int outCol = nTile * kNTileSize + (laneId % 4) * 2;
#endif

      // Pointer where sum.x / sum.y is written
      auto cPtr = reinterpret_cast<__hip_bfloat16*>(C) + outRow * n + outCol;

#if defined(USE_ROCM)
      if (outRow < m)
        cPtr[0] = __float2bfloat16(out.x);
      if ((outRow + 1) < m)
        cPtr[n] = __float2bfloat16(out.y);
      if ((outRow + 2) < m)
        cPtr[2*n] = __float2bfloat16(out.z);
      if ((outRow + 3) < m)
        cPtr[3*n] = __float2bfloat16(out.w);
#else
      auto v01 = __float22bfloat162_rn(float2{out.x, out.y});
      auto v23 = __float22bfloat162_rn(float2{out.z, out.w});

      if (outRow < m) {
        *reinterpret_cast<__hip_bfloat162*>(cPtr) = v01;
      }

      // sum.z, sum.w at +8 rows from cPtr
      if (outRow + 8 < m) {
        *reinterpret_cast<__hip_bfloat162*>(cPtr + 8 * n) = v23;
      }
#endif
    }
  }
};

template <int InnerKTiles, int QGroupSize>
struct BLayout_TC_int4 {
  static constexpr int32_t kInnerKTiles = InnerKTiles;
  static constexpr int32_t kMTileSize = 16;
#if defined(USE_ROCM)
  static constexpr int32_t kNTileSize = 16;
#else
  static constexpr int32_t kNTileSize = 8;
#endif
  static constexpr int32_t kKTileSize = 16;

  template <int KTilesToLoad>
  static __device__ void load(
      // type uint32, size [n / 8][k / (InnerKTiles * 16)][32][InnerKTiles / 2]
      // n-tiles: n / 8 for NV, n /16 for AMD
      // k / (InnerKTiles * 16): TC size per k-tile is 16 (m16n8k16 for NV, m16n16k16 for AMD)
      // value per warp lane: 32 for NV, 64 for AMD
      // (InnerKTiles / 2): B layout has 4 values per lane (16 bits) per k-tile.
      // 2 k-tiles packed is a uint32 (hence InnerKTiles == 2 is our smallest
      // value) 4 k-tiles packed is a uint32x2 (64 bits) 8 k-tiles packed is a
      // uint32x4 (128 bits)
      const void* __restrict__ B,
      // size [k / qGroupSize][n][2]
      // Contains the scale and zero point of each of the quantized int4 values
      // within B
      // v_reconstructed = (bf16(B_int4_val) * scale) - zero
      const void* __restrict__ quantizationInfo,
      int32_t n,
      int32_t k,
      int32_t nTiles,
      int32_t nTile,
      int32_t kTiles,
      int32_t kTileStart,
      int32_t laneId,
      bf16x2x4_u32 out[KTilesToLoad / InnerKTiles][InnerKTiles / 2]) {
    // offset [nTile][kTileStart / InnerKTiles][laneId][0]
    auto bPtr = reinterpret_cast<const int32_t*>(B) +
        (((nTile * (kTiles / InnerKTiles) + (kTileStart / InnerKTiles)) *
          kWarpSize) +
         laneId) *
            (InnerKTiles / 2);

    int32_t b_int4[KTilesToLoad / InnerKTiles][InnerKTiles / 2];

#pragma unroll
    for (int i = 0; i < KTilesToLoad / InnerKTiles; ++i) {
      auto bPtrCur = bPtr + i * kWarpSize * (InnerKTiles / 2);

      if constexpr (InnerKTiles == 2) {
        b_int4[i][0] = bPtrCur[0];
      }

      if constexpr (InnerKTiles == 4) {
        // asm volatile("ld.global.cs.v2.u32 {%0, %1}, [%2];\n"
        //              : "=r"(b_int4[i][0]), "=r"(b_int4[i][1])
        //              : "l"(bPtrCur));

        int2 load8 = reinterpret_cast<const int2*>(bPtrCur)[0];
        b_int4[i][0] = load8.x;
        b_int4[i][1] = load8.y;
      }

      if constexpr (InnerKTiles == 8) {
        // asm volatile("ld.global.cs.v4.u32 {%0, %1, %2, %3}, [%4];\n"
        //              : "=r"(b_int4[i][0]), "=r"(b_int4[i][1]),
        //              "=r"(b_int4[i][2]), "=r"(b_int4[i][3]) : "l"(bPtrCur));

        int4 load16 = reinterpret_cast<const int4*>(bPtrCur)[0];
        b_int4[i][0] = load16.x;
        b_int4[i][1] = load16.y;
        b_int4[i][2] = load16.z;
        b_int4[i][3] = load16.w;
      }
    }

    // Load needed info for dequantization

    static_assert(isPowerOf2(QGroupSize), "");
    static_assert(isEvenDivisor(QGroupSize, kKTileSize), "");
    // smallest quantization group size is 32 (2 k-tiles are packed in an int32)
    static_assert(QGroupSize >= kKTileSize * 2, "");
    constexpr int kKTilesPerQGroup = (QGroupSize / kKTileSize);
    // a q-group could be larger than what we are handling in a single warp
    constexpr int kNumQGroups = (KTilesToLoad / kKTilesPerQGroup) < 1
        ? 1
        : (KTilesToLoad / kKTilesPerQGroup);

    __hip_bfloat162 qScaleAndZero[kNumQGroups];
    {
#if defined(USE_ROCM)
      int32_t laneN = nTile * kNTileSize + (laneId % kNTileSize);
#else
      int32_t laneN = nTile * kNTileSize + (laneId / 4);
#endif
      int32_t groupStart = (kTileStart * kKTileSize) / QGroupSize;

      int32_t n = nTiles * kNTileSize;

      // offset [qScale_kGroup][qScale_n][0]
      auto qInfoPtr = reinterpret_cast<const __hip_bfloat16*>(quantizationInfo) +
          (groupStart * n + laneN) * 2;

#pragma unroll
      for (int i = 0; i < kNumQGroups; ++i) {
        qScaleAndZero[i] =
            *reinterpret_cast<const __hip_bfloat162*>(qInfoPtr + i * n * 2);
      }
    }

    //
    // De-quantize int4 values to bf16. Values are dequantized as truly int4
    // [-8, 7] range; dequant = (bf16(int4_value) * bf16_scale) + bf16_zero
    //
    {
      // FIXME: does this negatively affect register counts, or will nvcc
      // move this expansion (and data loads above) closer to the point of use?
      __hip_bfloat162 qScale[kNumQGroups];
      __hip_bfloat162 qZero[kNumQGroups];

#pragma unroll
      for (int i = 0; i < kNumQGroups; ++i) {
        qScale[i] = __bfloat162bfloat162(qScaleAndZero[i].x);
        qZero[i] = __bfloat162bfloat162(qScaleAndZero[i].y);
      }

#pragma unroll
      for (int i = 0; i < KTilesToLoad / InnerKTiles; ++i) {
#pragma unroll
        for (int j = 0; j < InnerKTiles / 2; ++j) {
          bf16x2x4 v = convert_i4x8_to_bf16x2x4(b_int4[i][j]);

          int curKTile = i * InnerKTiles + j * 2;
          int curQGroup = (curKTile * kKTileSize) / QGroupSize;

          // The dequantized values in `v` for a given lane have the same n
          // dimension (the B tensor core layout has all values in the same
          // thread along the same n) but different k dimension, but all are
          // guaranteed to occur within the same quantization group, so we need
          // only load a single scale + zero to cover what this lane has
#pragma unroll
          for (int k = 0; k < 4; ++k) {
            v.vals[k] = __hfma2(v.vals[k], qScale[curQGroup], qZero[curQGroup]);
          }

          // type pun, the __hip_bfloat162 value in bf16x2x4 is a struct and
          // can't be used as a 32-bit asm register argument for `mma`
          static_assert(sizeof(bf16x2x4) == sizeof(out[0][0]), "");
          std::memcpy(&out[i][j], &v, sizeof(bf16x2x4_u32));
        }
      }
    }
  }
};

template <
    typename ALayout,
    typename BLayout,
    typename CLayout,
    int Warps,
    int KTilesPerIteration>
__global__
__launch_bounds__(Warps* kWarpSize) void tinygemm_m16n8k16_chunk_kernel(
    // Data for the A matrix, loaded as per ALayout
    const void* const __restrict__ A,

    // Data for the B matrix, loaded as per BLayout
    const void* const __restrict__ B,

    // Optional quantization data for dequantizing B, loaded as per BLayout
    const void* const __restrict__ B_quantizationInfo,

    // Output data for the C matrix, stored as per CLayout
    void* __restrict__ C,

    // The size of the matrix multiplication
    int32_t m,
    int32_t n,
    int32_t k,

    // The size of the matrix multiplication, in multiples of our TC tile size
    int32_t mTiles,
    int32_t nTiles,
    int32_t kTiles) {
  constexpr int32_t kMTileSize = 16;
#if defined(USE_ROCM)
  constexpr int32_t kNTileSize = 16;
#else
  constexpr int32_t kNTileSize = 8;
#endif
  constexpr int32_t kKTileSize = 16;

#if !defined(USE_ROCM) || CDNA2_OR_LATER

  static_assert(
      ALayout::kMTileSize == kMTileSize && ALayout::kNTileSize == kNTileSize &&
          ALayout::kKTileSize == kKTileSize,
      "");

  static_assert(
      BLayout::kMTileSize == kMTileSize && BLayout::kNTileSize == kNTileSize &&
          BLayout::kKTileSize == kKTileSize,
      "");

  static_assert(
      CLayout::kMTileSize == kMTileSize && CLayout::kNTileSize == kNTileSize &&
          CLayout::kKTileSize == kKTileSize,
      "");

  constexpr int kInnerKTiles = BLayout::kInnerKTiles;

  // 2/4/8 inner k-tiles correspond to 4, 8 and 16 byte innermost loads
  static_assert(
      kInnerKTiles == 2 || kInnerKTiles == 4 || kInnerKTiles == 8, "");

  // We always process at least kInnerKTiles k-tiles back to back in a warp
  static_assert(
      KTilesPerIteration >= kInnerKTiles &&
          isEvenDivisor(KTilesPerIteration, kInnerKTiles),
      "");

  auto warpId = threadIdx.y;
  auto laneId = threadIdx.x;

  int32_t mTile = blockIdx.z;
  int32_t nTile = blockIdx.y;

#if defined(USE_ROCM)
  VecT<float, 4> c{0.0f, 0.0f, 0.0f, 0.0f};
#else
  float4 c{0.0f, 0.0f, 0.0f, 0.0f};
#endif

  // First, handle whole multiples of KTilesPerIteration
  auto kTilesLimit = roundDown(kTiles, KTilesPerIteration);

  // Each warp handles a set of KTilesPerIteration under the above limit
  for (int32_t kTileBase = (blockIdx.x * Warps + warpId) * KTilesPerIteration;
       kTileBase < kTilesLimit;
       kTileBase += Warps * KTilesPerIteration) {
    //
    // Load data from A
    //
#if defined(USE_ROCM)
    bf16x2x2_u32 a[KTilesPerIteration];
#else
    bf16x2x4_u32 a[KTilesPerIteration];
#endif
    ALayout::template load<KTilesPerIteration>(
        A, m, k, mTiles, mTile, kTiles, kTileBase, laneId, a);

    //
    // Load data from B and de-quantize as needed
    // Each k-tile is bf16x2x2
    //
    bf16x2x4_u32 b[KTilesPerIteration / kInnerKTiles][kInnerKTiles / 2];
    BLayout::template load<KTilesPerIteration>(
        B,
        B_quantizationInfo,
        n,
        k,
        nTiles,
        nTile,
        kTiles,
        kTileBase,
        laneId,
        b);

    //
    // Now, perform the matrix multiplication
    //

    // We accumulate across k-tiles here
#pragma unroll
    for (int i = 0; i < KTilesPerIteration / kInnerKTiles; ++i) {
      static_assert(isEvenDivisor(kInnerKTiles, 2) && kInnerKTiles >= 2, "");
#pragma unroll
      for (int j = 0; j < kInnerKTiles / 2; ++j) {
        // We don't simply accumulate into `c` as this creates a too-strong
        // execution dependency. Instead, we only periodically accumulate into
        // `c`
#if defined(USE_ROCM)
        VecT<float, 4> cTmp[2];
#else
        float4 cTmp[2];
#endif

#pragma unroll
        for (int k = 0; k < 2; ++k) {
#if defined(USE_ROCM)
          cTmp[k] = VecT<float, 4>{0.0f, 0.0f, 0.0f, 0.0f};
#else
          cTmp[k] = float4{0.0f, 0.0f, 0.0f, 0.0f};
#endif
        }

#pragma unroll
        for (int k = 0; k < 2; ++k) {
#if defined(USE_ROCM)
          cTmp[k] = __builtin_amdgcn_mfma_f32_16x16x16bf16_1k(
              a[i * kInnerKTiles + j * 2 + k].val,
              b[i][(j * 2 + k) / 2].val[((j * 2 + k) % 2)],
              cTmp[k], 0, 0, 0);
#else
          asm volatile(
              "mma.sync.aligned.m16n8k16.row.col.f32.bf16.bf16.f32 "
              "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};"
              : "=f"(cTmp[k].x),
                "=f"(cTmp[k].y),
                "=f"(cTmp[k].z),
                "=f"(cTmp[k].w)
              : "r"(a[i * kInnerKTiles + j * 2 + k].vals[0]),
                "r"(a[i * kInnerKTiles + j * 2 + k].vals[1]),
                "r"(a[i * kInnerKTiles + j * 2 + k].vals[2]),
                "r"(a[i * kInnerKTiles + j * 2 + k].vals[3]),
                "r"(b[i][(j * 2 + k) / 2].vals[((j * 2 + k) % 2) * 2 + 0]),
                "r"(b[i][(j * 2 + k) / 2].vals[((j * 2 + k) % 2) * 2 + 1]),
                "f"(cTmp[k].x),
                "f"(cTmp[k].y),
                "f"(cTmp[k].z),
                "f"(cTmp[k].w));
#endif
        }

#pragma unroll
        for (int k = 0; k < 2; ++k) {
#if defined(USE_ROCM)
          c[0] += cTmp[k][0];
          c[1] += cTmp[k][1];
          c[2] += cTmp[k][2];
          c[3] += cTmp[k][3];
#else
          c.x += cTmp[k].x;
          c.y += cTmp[k].y;
          c.z += cTmp[k].z;
          c.w += cTmp[k].w;
#endif
        }
      }
    }
  } // for all tiles under kTilesLimit

  // Now, there could be a remainder of 1 to KTilesPerIteration - 1 k-tiles
  // remaining. We guarantee that the number of warps is >= KTilesPerIteration /
  // kInnerKTiles, so that each warp can simply load kInnerKTiles and do its
  // thing without needing more warps
  static_assert(Warps >= KTilesPerIteration / kInnerKTiles, "");

  auto kTileBaseRemaining = kTilesLimit + warpId * kInnerKTiles;

  // If we have any remainder k-tiles, some warps will handle them, processing
  // kInnerKTiles k-tiles at a time
  if (kTileBaseRemaining < kTiles) {
#if defined(USE_ROCM)
    bf16x2x2_u32 a[kInnerKTiles];
#else
    bf16x2x4_u32 a[kInnerKTiles];
#endif
    ALayout::template load<kInnerKTiles>(
        A, m, k, mTiles, mTile, kTiles, kTileBaseRemaining, laneId, a);

    bf16x2x4_u32 b[1][kInnerKTiles / 2];
    BLayout::template load<kInnerKTiles>(
        B,
        B_quantizationInfo,
        n,
        k,
        nTiles,
        nTile,
        kTiles,
        kTileBaseRemaining,
        laneId,
        b);

#pragma unroll
    for (int j = 0; j < kInnerKTiles / 2; ++j) {
      // We don't simply accumulate into `c` as this creates a too-strong
      // execution dependency. Instead, we only periodically accumulate into
      // `c`
#if defined(USE_ROCM)
      VecT<float, 4> cTmp[2];
#else
      float4 cTmp[2];
#endif

#pragma unroll
      for (int k = 0; k < 2; ++k) {
#if defined(USE_ROCM)
        cTmp[k] = VecT<float, 4>{0.0f, 0.0f, 0.0f, 0.0f};
#else
        cTmp[k] = float4{0.0f, 0.0f, 0.0f, 0.0f};
#endif
      }

#pragma unroll
      for (int k = 0; k < 2; ++k) {
#if defined(USE_ROCM)
        cTmp[k] = __builtin_amdgcn_mfma_f32_16x16x16bf16_1k(
          a[j * 2 + k].val,
          b[0][(j * 2 + k) / 2].val[((j * 2 + k) % 2)],
          cTmp[k], 0, 0, 0);
#else
        asm volatile(
            "mma.sync.aligned.m16n8k16.row.col.f32.bf16.bf16.f32 "
            "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};"
            : "=f"(cTmp[k].x), "=f"(cTmp[k].y), "=f"(cTmp[k].z), "=f"(cTmp[k].w)
            : "r"(a[j * 2 + k].vals[0]),
              "r"(a[j * 2 + k].vals[1]),
              "r"(a[j * 2 + k].vals[2]),
              "r"(a[j * 2 + k].vals[3]),
              "r"(b[0][(j * 2 + k) / 2].vals[((j * 2 + k) % 2) * 2 + 0]),
              "r"(b[0][(j * 2 + k) / 2].vals[((j * 2 + k) % 2) * 2 + 1]),
              "f"(cTmp[k].x),
              "f"(cTmp[k].y),
              "f"(cTmp[k].z),
              "f"(cTmp[k].w));
#endif
      }

#pragma unroll
      for (int k = 0; k < 2; ++k) {
#if defined(USE_ROCM)
        c[0] += cTmp[k][0];
        c[1] += cTmp[k][1];
        c[2] += cTmp[k][2];
        c[3] += cTmp[k][3];
#else
        c.x += cTmp[k].x;
        c.y += cTmp[k].y;
        c.z += cTmp[k].z;
        c.w += cTmp[k].w;
#endif
      }
    }
  }

  //
  // Reduce independent k-tiles (same m/n) across warps
  //
  __shared__ float4 smem_sum[Warps][kWarpSize];

  // FIXME: this likely doesn't need to be a true reduction tree, can just be a
  // serial sum, maybe (unless nvcc/ptxas goes back to its old ways)
  // smem_sum[warpId][laneId] = TreeReduce4<KTilesPerIteration>::reduce(c);
#if defined(USE_ROCM)
  smem_sum[warpId][laneId].x = c[0];
  smem_sum[warpId][laneId].y = c[1];
  smem_sum[warpId][laneId].z = c[2];
  smem_sum[warpId][laneId].w = c[3];
#else
  smem_sum[warpId][laneId] = c;
#endif

  __syncthreads();

  if (warpId == 0) {
    float4 sum_f32{0.0f, 0.0f, 0.0f, 0.0f};

    // Reduce across the block in the first warp
    for (int i = 0; i < Warps; ++i) {
      float4 v = smem_sum[i][laneId];
      sum_f32.x += v.x;
      sum_f32.y += v.y;
      sum_f32.z += v.z;
      sum_f32.w += v.w;
    }

    // Write the reduced result (in the first warp) into the output
    CLayout::store(
        C,
        m,
        n,
        mTiles,
        mTile,
        // n for C output becomes k for A input, so for m16n8k16,
        // we need to halve the tiles
        nTiles / 2,
        nTile,
        laneId,
        sum_f32);
  }
#else
    printf("__builtin_amdgcn_mfma_f32_16x16x16bf16_1k is only supported on AMD gpu arch greater than or equal to CDNA2\n");
#endif
}


template <
    typename ALayout,
    typename BLayout,
    typename CLayout,
    int Warps,
    int KTilesPerWarp>
void launch_tinygemm_kernel(
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor* qScaleAndZeros, /* optional */
    at::Tensor& C_final,
    int32_t mTiles,
    int32_t nTiles,
    int32_t kTiles,
    int32_t m,
    int32_t n,
    int32_t k,
    hipStream_t stream) {
  // The chunking kernel requires that kTiles is a multiple of kInnerKTiles
  TORCH_CHECK(
      kTiles >= BLayout::kInnerKTiles &&
      isEvenDivisor(kTiles, BLayout::kInnerKTiles));

  TORCH_CHECK(
      KTilesPerWarp >= BLayout::kInnerKTiles &&
      isEvenDivisor(KTilesPerWarp, BLayout::kInnerKTiles));

  // After intra-block reduction across the k dimension, we are left with this
  // many tiles
  //  int32_t postKernelKTiles = kTiles / (Warps * KTilesPerWarp);
  int32_t postKernelKTiles = 1; // we loop

  auto grid = dim3(postKernelKTiles, nTiles, mTiles);
  auto block = dim3(kWarpSize, Warps);

  auto func =
      tinygemm_m16n8k16_chunk_kernel<ALayout, BLayout, CLayout, Warps, KTilesPerWarp>;

  func<<<grid, block, 0, stream>>>(
      A.data_ptr(),
      B.data_ptr(),
      qScaleAndZeros ? qScaleAndZeros->data_ptr() : nullptr,
      C_final.data_ptr(),
      m,
      n,
      k,
      mTiles,
      nTiles,
      kTiles);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  hipFuncAttributes funcAttr;
#if defined(USE_ROCM)
  C10_CUDA_CHECK(hipFuncGetAttributes(
      &funcAttr,
      reinterpret_cast<const void*>((void *))func
  ));
#else
  C10_CUDA_CHECK(hipFuncGetAttributes(
      &funcAttr,
      reinterpret_cast<const void*>(func
  )));
#endif
}

// FIXME: parallelize better, smem staging etc?
template <int InnerKTiles>
__global__ void matrix_to_m16n8k16_Bint4_layout(
    // size [n][k / 2]
    const at::PackedTensorAccessor32<uint8_t, 2, at::RestrictPtrTraits> in,
    // size [ceil(n / 8)][ceil(k / (InnerKTiles * 16))][32][InnerKTiles / 2]
    at::PackedTensorAccessor32<int32_t, 4, at::RestrictPtrTraits> out) {
  // int4 values are packed into int32 values, which require at least 8. Given
  // m16n8k16 B layout requires 4 scalar values/lane, the minimum number of
  // innermost k-tiles that we can use is 2.
  static_assert(InnerKTiles >= 2 && isPowerOf2(InnerKTiles), "");

#if defined(USE_ROCM)
  constexpr int32_t kNTileSize = 16;
#else
  constexpr int32_t kNTileSize = 8;
#endif
  constexpr int32_t kKTileSize = 16;

  // gridDim.x corresponds to the number of k-tiles divided by InnerKTiles
  auto kOuterTile = blockIdx.x;
  auto nTile = blockIdx.y;
  auto t = threadIdx.x;

  // Two k-tiles are packed into an int32 at a time
#pragma unroll
  for (int innerKTile = 0; innerKTile < InnerKTiles; innerKTile += 2) {
    // n dimension that this lane loads from
#if defined(USE_ROCM)
    auto n0 = nTile * kNTileSize + (t % kNTileSize);
#else
    auto n0 = nTile * kNTileSize + (t / 4);
#endif

    bool n0Valid = n0 < in.size(0);

    // Four uint8 are packed into an int32
    int32_t ks[4];

    auto kBase0 = (kOuterTile * InnerKTiles + innerKTile) * kKTileSize / 2;

#if defined(USE_ROCM)
    ks[0] = kBase0 + (t / kNTileSize) * 2;
    ks[1] = ks[0] + 1;

    auto kBase1 = kBase0 + kKTileSize / 2;
    ks[2] = kBase1 + (t / kNTileSize) * 2;
    ks[3] = ks[2] + 1;
#else
    ks[0] = kBase0 + t % 4;
    ks[1] = ks[0] + 4;

    auto kBase1 = kBase0 + kKTileSize / 2;
    ks[2] = kBase1 + t % 4;
    ks[3] = ks[2] + 4;
#endif

    auto pIn = &in[n0][0];

    uint8_t v[4];
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      v[i] = (n0Valid && ks[i] < in.size(1)) ? pIn[ks[i]] : uint8_t(0);
    }

    // To clearly explain the packed result with 8 int4 values (4 uint8)
    // into one int32, we use the follow figure:
    // [n][k]     int32: v[0] v[1] v[2] v[3] v[4] v[5] v[6] v[7]
    // [n][k / 2] uint8:    v[0]     v[1]      v[2]      v[3]
    // When using int32 weight as input, the packed result is consisted of
    // v[7] | v[5] | v[3] | v[1] | v[6] | v[4] | v[2] | v[0],
    // which epuals to
    // v[3]L | v[2]L | v[1]L | v[0]L | v[3]H | v[2]H | v[1]H | v[0]H
    // when using uint8 weight as input.
    int32_t pack = ((uint32_t)(v[3] & 0xF) << 28) |
        ((uint32_t)(v[2] & 0xF) << 24) | ((uint32_t)(v[1] & 0xF) << 20) |
        ((uint32_t)(v[0] & 0xF) << 16) | ((uint32_t)(v[3] & 0xF0) << 8) |
        ((uint32_t)(v[2] & 0xF0) << 4) | ((uint32_t)(v[1] & 0xF0)) |
        ((uint32_t)(v[0] & 0xF0) >> 4);

    // inner k-tiles pack two at a time
#if defined(USE_ROCM)
    // The output tensor shape is [ceil(n / 8)][ceil(k / (InnerKTiles * 16))][32][InnerKTiles / 2], which is specific to Nvidia
    // But AMD needs [ceil(n / 16)][ceil(k / (InnerKTiles * 16))][64][InnerKTiles / 2]
    // So construct the pointer accordingly
    auto bPtr = out.data() +
      ((nTile * out.size(1) * kWarpSize * (InnerKTiles / 2)) +
        (kOuterTile * kWarpSize * (InnerKTiles / 2)) +
          (t * (InnerKTiles / 2)) +
            (innerKTile / 2));
    *bPtr = pack;
#else
    out[nTile][kOuterTile][t][innerKTile / 2] = pack;
#endif
  }
}

#endif


at::Tensor _weight_int4pack_mm_cuda(
    const at::Tensor& A,
    const at::Tensor& B,
    int64_t qGroupSize,
    const at::Tensor& qScaleAndZeros) {
  c10::cuda::CUDAGuard g(A.device());

  TORCH_CHECK(
      A.device() == B.device() && A.device() == qScaleAndZeros.device());

#if defined(USE_ROCM)
  if (!isCDNA2orLater(A.device().index())) {
    TORCH_CHECK(false, "_weight_int4pack_mm_cuda is only supported on AMD gpu arch greater than or equal to CDNA2");
  }
#endif

  constexpr int32_t kMTileSize = 16;
#if defined(USE_ROCM)
  constexpr int32_t kNTileSize = 16;
#else
  constexpr int32_t kNTileSize = 8;
#endif
  constexpr int32_t kKTileSize = 16;

  // row major layout
  auto m = A.size(0);
  auto mTiles = divUp(m, kMTileSize);

  // To convert the nTiles from tensor storage layout to the actual matrix core layout
  constexpr int32_t kNTileSizeTensor = 8;
  auto nTileScaleFactor = (kNTileSize / kNTileSizeTensor);

  // tensor core layout
  auto nTiles = (B.size(0) / nTileScaleFactor);
  auto n = nTiles * kNTileSize;

  // row major layout
  auto k = A.size(1);
  auto kTiles = divUp(k, kKTileSize);

  // The number of inner k tiles is the innermost dimension of  times 2
  // 2 k-tiles (4 values per lane per tile, 8 values total) quantized to int4
  // packed into 1 int32 for int4 B
  auto B_innerKTiles = B.size(3) * 2;
  TORCH_CHECK(B_innerKTiles == 2 || B_innerKTiles == 4 || B_innerKTiles == 8);

  // A is standard row major
  TORCH_CHECK(A.dtype() == at::kBFloat16);
  TORCH_CHECK(A.is_contiguous());
  TORCH_CHECK(A.dim() == 2);

  // B has B_innerKTiles k-tiles in the innermost dimension
  TORCH_CHECK(B.dtype() == at::kInt);
  TORCH_CHECK(B.is_contiguous());
  TORCH_CHECK(B.dim() == 4);
  TORCH_CHECK(B.size(1) == k / (B_innerKTiles * kKTileSize));
  TORCH_CHECK(B.size(2) == 32);

  // Validate the scale and zero point tensor for dequantization
  // These are the only versions handled at the moment
  TORCH_CHECK(
      qGroupSize == 32 || qGroupSize == 64 || qGroupSize == 128 ||
      qGroupSize == 256);

  TORCH_CHECK(qScaleAndZeros.dim() == 3);
  auto numQGroups = qScaleAndZeros.size(0);
  TORCH_CHECK(
      kTiles * kKTileSize >= qGroupSize &&
      isEvenDivisor(kTiles * kKTileSize, qGroupSize));
  TORCH_CHECK(qScaleAndZeros.size(1) == n);
  TORCH_CHECK(qScaleAndZeros.size(2) == 2);

  // Output is a standard row-major matrix
  auto C_final = at::empty(
      {m, n}, at::TensorOptions().dtype(at::kBFloat16).device(A.device()));

#if (defined(USE_ROCM) && ROCM_VERSION >= 50700) || ((defined(CUDA_VERSION) && CUDA_VERSION >= 12000) && (!defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 800)))
  auto stream = at::cuda::getCurrentCUDAStream();
#define RUN_GEMM(WARPS, K_TILES_PER_WARP, Q_GROUP_SIZE, REDUCE_TYPE) \
  do {                                                               \
    using ACLayout = ALayout_RM<REDUCE_TYPE>;                        \
                                                                     \
    TORCH_CHECK(                                                     \
        K_TILES_PER_WARP >= B_innerKTiles &&                         \
        isEvenDivisor(K_TILES_PER_WARP, B_innerKTiles));             \
                                                                     \
    switch (B_innerKTiles) {                                         \
      case 2:                                                        \
        if constexpr (K_TILES_PER_WARP >= 2) {                       \
          using BLayout = BLayout_TC_int4<2, Q_GROUP_SIZE>;          \
          launch_tinygemm_kernel<                                    \
              ACLayout,                                              \
              BLayout,                                               \
              ACLayout,                                              \
              WARPS,                                                 \
              K_TILES_PER_WARP>(                                     \
              A,                                                     \
              B,                                                     \
              &qScaleAndZeros,                                       \
              C_final,                                               \
              mTiles,                                                \
              nTiles,                                                \
              kTiles,                                                \
              m,                                                     \
              n,                                                     \
              k,                                                     \
              stream);                                               \
        }                                                            \
        break;                                                       \
      case 4:                                                        \
        if constexpr (K_TILES_PER_WARP >= 4) {                       \
          using BLayout = BLayout_TC_int4<4, Q_GROUP_SIZE>;          \
          launch_tinygemm_kernel<                                    \
              ACLayout,                                              \
              BLayout,                                               \
              ACLayout,                                              \
              WARPS,                                                 \
              K_TILES_PER_WARP>(                                     \
              A,                                                     \
              B,                                                     \
              &qScaleAndZeros,                                       \
              C_final,                                               \
              mTiles,                                                \
              nTiles,                                                \
              kTiles,                                                \
              m,                                                     \
              n,                                                     \
              k,                                                     \
              stream);                                               \
        }                                                            \
        break;                                                       \
      case 8:                                                        \
        if constexpr (K_TILES_PER_WARP >= 8) {                       \
          using BLayout = BLayout_TC_int4<8, Q_GROUP_SIZE>;          \
          launch_tinygemm_kernel<                                    \
              ACLayout,                                              \
              BLayout,                                               \
              ACLayout,                                              \
              WARPS,                                                 \
              K_TILES_PER_WARP>(                                     \
              A,                                                     \
              B,                                                     \
              &qScaleAndZeros,                                       \
              C_final,                                               \
              mTiles,                                                \
              nTiles,                                                \
              kTiles,                                                \
              m,                                                     \
              n,                                                     \
              k,                                                     \
              stream);                                               \
        }                                                            \
        break;                                                       \
      default:                                                       \
        break;                                                       \
    }                                                                \
  } while (false)

#define HANDLE_Q_GROUP(WARPS, K_TILES_PER_WARP, REDUCE_TYPE) \
  do {                                                       \
    switch (qGroupSize) {                                    \
      case 32:                                               \
        RUN_GEMM(WARPS, K_TILES_PER_WARP, 32, REDUCE_TYPE);  \
        break;                                               \
      case 64:                                               \
        RUN_GEMM(WARPS, K_TILES_PER_WARP, 64, REDUCE_TYPE);  \
        break;                                               \
      case 128:                                              \
        RUN_GEMM(WARPS, K_TILES_PER_WARP, 128, REDUCE_TYPE); \
        break;                                               \
      case 256:                                              \
        RUN_GEMM(WARPS, K_TILES_PER_WARP, 256, REDUCE_TYPE); \
        break;                                               \
    }                                                        \
  } while (false)

  HANDLE_Q_GROUP(8, 8, KReductionType::None);

#undef HANDLE_Q_GROUP
#undef RUN_GEMM

  return C_final;
#endif
  TORCH_CHECK(false, "_weight_int4pack_mm_cuda is not available for build.")
  return C_final;
}

// input is [n][k / 2] (uint8 dtype)
// output is [n / 8][k / (InnerKTiles * 16)][32][innerKTiles / 2] (int32 dtype)
at::Tensor _convert_weight_to_int4pack_cuda(
    const at::Tensor& in,
    int64_t innerKTiles) {
  c10::cuda::CUDAGuard g(in.device());

  TORCH_CHECK(in.dim() == 2);
  TORCH_CHECK(in.dtype() == at::kByte);
  TORCH_CHECK(in.is_contiguous());

  // At least 2 k-tiles need to be packed back to back in the innermost
  // dimension, as the m16n8k16 tensor core tile presents 4 scalar values for
  // the B matrix, but the minimum word size for the packed format is 4 bytes
  // (int32). 4 inner K-tiles = 8 byte load, 8 inner k-tiles = 16 byte load
  // which is the maximum vectorized load/store size
  TORCH_CHECK(innerKTiles == 2 || innerKTiles == 4 || innerKTiles == 8);

#if defined(USE_ROCM)
  if (!isCDNA2orLater(in.device().index())) {
    TORCH_CHECK(false, "_convert_weight_to_int4pack_cuda is only supported on AMD gpu arch greater than or equal to CDNA2");
  }
#endif

#if defined(USE_ROCM)
  constexpr int32_t kNTileSize = 16;
#else
  constexpr int32_t kNTileSize = 8;
#endif
  constexpr int32_t kKTileSize = 16;

  // GPT-FAST assumes nTileSize of 8 for quantized weight tensor.
  // See https://github.com/pytorch-labs/gpt-fast/blob/091515ab5b06f91c0d6a3b92f9c27463f738cc9b/quantize.py#L510
  // Torch dynamo also requires the torch ops has the same output shape for each device.
  // See https://github.com/pytorch/pytorch/blob/ec284d3a74ec1863685febd53687d491fd99a161/torch/_meta_registrations.py#L3263
  constexpr int32_t kNTileSizeTensor = 8;

  auto nTiles = divUp(in.size(0), kNTileSize);
  auto nTilesTensor = divUp(in.size(0), kNTileSizeTensor);

  // k-tiles are packed back to back in the innermost dimension in order to
  // allow for 4/8/16 byte loads
  TORCH_CHECK(isEvenDivisor(in.size(1) * 2, innerKTiles * kKTileSize));
  // kSuperTiles is the number of k-tiles assuming k is innerKTiles * kKTileSize
  auto kSuperTiles = divUp(in.size(1) * 2, innerKTiles * kKTileSize);

  // each block handles `innerKTiles` k-tiles.
  // 2 k-tiles are a single int32
  //
  // We use the same shape for AMD gpus also to match the GPT-FAST spec.
  // Will index it correctly when dereferencing the quantized weight tensor pointer.
  auto out = at::empty(
      {nTilesTensor, kSuperTiles, 32, innerKTiles / 2},
      at::TensorOptions().dtype(at::kInt).device(in.device()));

#if (defined(USE_ROCM) && ROCM_VERSION >= 50700) || ((defined(CUDA_VERSION) && CUDA_VERSION >= 12000) && (!defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 800)))
  auto stream = at::cuda::getCurrentCUDAStream();
  dim3 grid(kSuperTiles, nTiles);

  if (innerKTiles == 2) {
    matrix_to_m16n8k16_Bint4_layout<2><<<grid, kWarpSize, 0, stream>>>(
        in.packed_accessor32<uint8_t, 2, at::RestrictPtrTraits>(),
        out.packed_accessor32<int32_t, 4, at::RestrictPtrTraits>());
  } else if (innerKTiles == 4) {
    matrix_to_m16n8k16_Bint4_layout<4><<<grid, kWarpSize, 0, stream>>>(
        in.packed_accessor32<uint8_t, 2, at::RestrictPtrTraits>(),
        out.packed_accessor32<int32_t, 4, at::RestrictPtrTraits>());
  } else if (innerKTiles == 8) {
    matrix_to_m16n8k16_Bint4_layout<8><<<grid, kWarpSize, 0, stream>>>(
        in.packed_accessor32<uint8_t, 2, at::RestrictPtrTraits>(),
        out.packed_accessor32<int32_t, 4, at::RestrictPtrTraits>());
  }

  return out;
#endif
  TORCH_CHECK(false, "_convert_weight_to_int4pack_cuda is not available for build.")
  return out;
}


} // namespace at::native
