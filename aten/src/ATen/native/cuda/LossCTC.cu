#include "hip/hip_runtime.h"
// Copyright (c) 2018 MathInf GmbH, Thomas Viehmann
// Licensed under the BSD-3-Clause license
// This is the GPU implementation of the Connectionist Temporal Loss.
// We mostly follow Graves.
// 1. Graves et al.: http://www.cs.toronto.edu/~graves/icml_2006.pdf
// We use the equations from above link, but note that [1] has 1-based indexing and we (of course) use 0-based.
// Graves et al. call the probabilities y, we use log_probs (also calling them inputs)
// A few optimizations (similar to those here, but also some I didn't take) are described in
// 2. Minmin Sun: http://on-demand.gputechconf.com/gtc/2016/presentation/s6383-minmin-sun-speech-recognition.pdf
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/TensorUtils.h>
#include <c10/util/Exception.h>
#include <c10/macros/Macros.h>
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorOperators.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_ctc_loss_backward_native.h>
#include <ATen/ops/_ctc_loss_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/exp.h>
#include <ATen/ops/full_like.h>
#include <ATen/ops/imag.h>
#include <ATen/ops/logsumexp.h>
#include <ATen/ops/tensor.h>
#include <ATen/ops/where.h>
#include <ATen/ops/zeros.h>
#endif

#include <type_traits>
#include <numeric>

namespace at::native {

namespace {

// this ad-hoc converts from targets (l in [1]) to augmented targets (l' in [1])
// so if l is l_0 l_1 ... l_(tl-1) then this looks up idx in
// l' = BLANK l_0 BLANK l_1 BLANK ... BLANK l_(tl-1) BLANK
// - note that no bound-checking is done
// - it is important to only call it with idx == 0 if the target length is 0
// - __restrict__ impact to be measured, see
//   https://devblogs.nvidia.com/cuda-pro-tip-optimize-pointer-aliasing/
template <typename target_t>
__device__ static inline int64_t get_target_prime(
    const target_t* __restrict__ target,
    int64_t offset,
    int64_t stride,
    int64_t idx,
    int64_t BLANK) {
  if (idx % 2 == 0) {
    return BLANK;
  } else {
    return target[offset + stride * (idx / 2)];
  }
}

// this kernel is a relatively straightforward implementation of the alpha calculation in the forward backward algorithm (section 4.1).
// A (minor) twist is that we are using log-calculations to enhance numerical stability (log_probs and log_alpha).
// In total it would be more efficient to compute the beta in the same kernel (e.g. cudnn does this). While the beta are not
// needed for the loss itself (just the grad), we can return log_alpha+log_beta (so same space as currently) and the overhead
// is small and the use-case for loss without grad is relatively limited.
// We parallelize by batch and target sequence. Empirically, it is faster to loop over the input (log probs) sequence  and do
// target in parallel, even if it means more frequent __syncthreads.
// In contrast to the cuDNN implementation, we allow large target lengths. For this we need that all previous `s` have been
// computed when we start a new block_s. This is why we have our own for loop here.
template<typename scalar_t, typename target_t>
__global__ void
#if defined (USE_ROCM)
C10_LAUNCH_BOUNDS_2((std::is_same_v<scalar_t, float> ? 1024 : 896), 1)
#endif
ctc_loss_log_alpha_gpu_kernel(scalar_t* __restrict__ log_alpha_data,
                                    const scalar_t*log_probs_data, const int64_t* __restrict__ input_lengths, int64_t max_input_length,
                                    const target_t* __restrict__ targets_data, const int64_t* __restrict__ target_lengths, int64_t max_target_length,
                                    scalar_t* __restrict__ neg_log_likelihood_data,
                                    int64_t lp_input_stride, int64_t lp_batch_stride, int64_t lp_char_stride,
                                    int64_t la_batch_stride, int64_t la_input_stride, int64_t la_target_stride,
                                    const int64_t* __restrict__ tg_batch_offsets, int64_t tg_target_stride,
                                    int64_t batch_size, int64_t BLANK) {

  constexpr scalar_t neginf = -INFINITY;

  // bookkeeping
  int64_t b = threadIdx.y + blockIdx.y * blockDim.y;
  int64_t input_length = input_lengths[b];
  int64_t target_length = target_lengths[b];
  int64_t lp_batch_offset = b*lp_batch_stride;
  int64_t la_batch_offset = b*la_batch_stride;
  int64_t tg_batch_offset = tg_batch_offsets[b];

  if (b >= batch_size)
    return;

  if (input_length == 0) {
    if (threadIdx.x == 0) {
      scalar_t log_likelihood = target_length == 0 ? 0 : neginf;
      neg_log_likelihood_data[b] = -log_likelihood;
    }
    return;
  }

  // first row (t=0), the three equations for alpha_1 above eq (6)
  for (int64_t block_s = 0; block_s < 2*max_target_length+1; block_s += blockDim.x) {
    int64_t s = threadIdx.x + block_s;
    scalar_t la;
    switch (s) {
    case 0:
      la = log_probs_data[lp_batch_offset + lp_char_stride * BLANK];
      break;
    case 1:
      la = target_length == 0 ? neginf
                              : log_probs_data
                                    [lp_batch_offset +
                                     lp_char_stride *
                                         get_target_prime(
                                             targets_data,
                                             tg_batch_offset,
                                             tg_target_stride,
                                             1,
                                             BLANK)];
      break;
    default:
      la = neginf;
    }
    if (s < 2*max_target_length+1)
      log_alpha_data[la_batch_offset + /* la_input_stride * 0 */ + la_target_stride * s] = la;
  }

  for (int64_t block_s = 0; block_s < 2*max_target_length+1; block_s += blockDim.x) {
    int64_t s = threadIdx.x + block_s;

    // These two only depend on s, so we can cache them.
    int64_t current_char;       // l_s in eq (6)
    bool have_three;            // flag which of the two cases in eq (6) we have
    if (s < 2 * target_length + 1 && target_length > 0) {
      current_char = get_target_prime(
          targets_data,
          tg_batch_offset,
          tg_target_stride,
          s,
          BLANK);
      have_three =
          ((s > 1) &&
           (get_target_prime(
                targets_data,
                tg_batch_offset,
                tg_target_stride,
                s - 2,
                BLANK) != current_char));
    } else {
      current_char = BLANK;
      have_three = false;
    }
    for (int64_t t=1; t < max_input_length; t++) {
      __syncthreads(); // on cuda 9 we might use partial synchronization of only the threads within the same batch
      if ((t < input_length) && (s < 2 * target_length + 1)) {
        // only for valid t, s. This is equation (6) and (7), la1, la2, la3 are the three summands,
        // lamax is the maximum for the logsumexp trick.
        scalar_t la1 = log_alpha_data[la_batch_offset + la_input_stride * (t-1) + la_target_stride * s];
        scalar_t lamax = la1;
        scalar_t la2, la3;
        if (s > 0) {
          la2 = log_alpha_data[la_batch_offset + la_input_stride * (t-1) + la_target_stride * (s-1)];
          if (la2 > lamax)
            lamax = la2;
        } else {
          la2 = neginf;
        }
        if (have_three) {
          la3 = log_alpha_data[la_batch_offset + la_input_stride * (t-1) + la_target_stride * (s-2)];
          if (la3 > lamax)
            lamax = la3;
        } else {
          la3 = neginf;
        }
        if (lamax == neginf) // when all are neginf. (then the whole thing is neginf, but we can pretend)
          lamax = 0;

        log_alpha_data[la_batch_offset + la_input_stride * t + la_target_stride * s] = std::log(std::exp(la1-lamax)+std::exp(la2-lamax)+std::exp(la3-lamax))+lamax
          + log_probs_data[lp_batch_offset + t * lp_input_stride + lp_char_stride * current_char];
      } else {
        // otherwise we just set to neginf
        if (s < 2*max_target_length+1)
          log_alpha_data[la_batch_offset + la_input_stride * t + la_target_stride * s] = neginf;
      }
    }
  }
  __syncthreads(); // on cuda 9 we might use partial synchronization of only the threads within the same batch

  // compute the loss (eq (8))
  if (threadIdx.x == 0) {
    scalar_t l1 = log_alpha_data[la_batch_offset + la_input_stride * (input_length-1) + la_target_stride * (target_length*2)];
    scalar_t l2 = target_length > 0
        ? log_alpha_data
              [la_batch_offset + la_input_stride * (input_length - 1) +
               la_target_stride * (target_length * 2 - 1)]
        : neginf;
    scalar_t m = ((l1 > l2) ? l1 : l2);
    m = ((m == neginf) ? 0 : m);
    scalar_t log_likelihood = std::log(std::exp(l1-m)+std::exp(l2-m))+m;
    neg_log_likelihood_data[b] = -log_likelihood;
  }
}

// The forward computation. Lot's of admin and a call to the alpha kernel.
// Note: we do not check that the labels are in the valid range. As we use
// them for indexing in the kernels, you'll see memory errors when you
// pass corrupt labels.
// We support both a 2-dimensional tensor as targets (one set of targets in each row) and
// a 1-dimensional tensor where all targets are concatenated (and we use target_lengths
// to figure out where they begin).
// We return log_alpha (currently, might change to (log_alpha+log_beta) to be passed to the
// backward. The dispatch function will only return the loss.
template<typename scalar_t, ScalarType target_scalar_type>
std::tuple<Tensor, Tensor> ctc_loss_gpu_template(const Tensor& log_probs, const Tensor& targets, IntArrayRef input_lengths, IntArrayRef target_lengths, int64_t BLANK) {
  TORCH_CHECK(log_probs.numel() > 0, "log_probs tensor must not be empty");
  // log_probs: input_len x batch_size x num_labels
  // targets [int64]: batch_size x target_length OR sum(target_lengths)
  CheckedFrom c = "ctc_loss_gpu";
  using target_t = typename std::conditional_t<target_scalar_type == kInt, int, int64_t>;
  auto log_probs_arg = TensorArg(log_probs, "log_probs", 1);
  auto targets_arg = TensorArg(targets, "targets", 2);
  checkAllSameGPU(c, {log_probs_arg, targets_arg});

  checkScalarType(c, targets_arg, target_scalar_type);
  checkDim(c, log_probs_arg, 3);
  checkDimRange(c, targets_arg, 1, 3);

  int64_t batch_size = log_probs.size(1);
  int64_t num_labels = log_probs.size(2);
  TORCH_CHECK((0 <= BLANK) && (BLANK < num_labels), "blank must be in label range");
  TORCH_CHECK(input_lengths.size() == static_cast<size_t>(batch_size), "input_lengths must be of size batch_size");
  TORCH_CHECK(target_lengths.size() == static_cast<size_t>(batch_size), "target_lengths must be of size batch_size");

  int64_t tg_target_stride;

  int64_t max_target_length = 0;
  auto tg_batch_offsets = at::empty({batch_size}, at::device(at::kCPU).dtype(at::kLong));
  auto tg_batch_offsets_data = tg_batch_offsets.mutable_data_ptr<int64_t>();
  if (targets.dim() == 1) { // concatenated targets
    int64_t pos = 0;
    for (int64_t i = 0; i < batch_size; i++) {
      TORCH_CHECK(target_lengths[i] >= 0,
                  "Expected target_lengths to have value at least ", 0, ", but got value ", target_lengths[i],
                  " (while checking arguments for ", c, ")");
      tg_batch_offsets_data[i] = pos;
      pos += target_lengths[i];
      if (max_target_length < target_lengths[i])
        max_target_length = target_lengths[i];
    }
    tg_target_stride = targets.stride(0);
    checkSize(c, targets_arg, 0, pos);
  }
  else { // batch x max_target_length
    // dim is 2
    int64_t tg_batch_stride = targets.stride(0);
    for (int64_t i = 0; i < batch_size; i++) {
      TORCH_CHECK(target_lengths[i] >= 0,
                  "Expected target_lengths to have value at least ", 0, ", but got value ", target_lengths[i],
                  " (while checking arguments for ", c, ")");
      tg_batch_offsets_data[i] = i * tg_batch_stride;
      if (max_target_length < target_lengths[i])
        max_target_length = target_lengths[i];
    }
    tg_target_stride = targets.stride(1);
    checkSize(c, targets_arg, 0, batch_size);
    TORCH_CHECK(targets.size(1) >= max_target_length,
             "Expected tensor to have size at least ", max_target_length, " at dimension 1, but got size ", targets.size(1), " for ", targets_arg,
             " (while checking arguments for ", c, ")");
  }
  int64_t max_input_length = log_probs.size(0);
  for (int64_t b = 0; b < batch_size; b++) {
    TORCH_CHECK(input_lengths[b] >= 0,
             "Expected input_lengths to have value at least ", 0, ", but got value ", input_lengths[b],
             " (while checking arguments for ", c, ")");
    TORCH_CHECK(input_lengths[b] <= max_input_length,
             "Expected input_lengths to have value at most ", max_input_length, ", but got value ", input_lengths[b],
             " (while checking arguments for ", c, ")");
  }

  auto target_lengths_t = at::tensor(target_lengths, targets.options().dtype(kLong));
  auto input_lengths_t = at::tensor(input_lengths, targets.options().dtype(kLong));
  tg_batch_offsets = tg_batch_offsets.cuda();

  Tensor log_alpha = at::empty({batch_size, log_probs.size(0), 2*max_target_length+1}, log_probs.options());
  Tensor neg_log_likelihood = at::empty({batch_size}, log_probs.options());

  // Very likely, we could be more clever here, e.g. learning (or generalizing and reusing) from SoftMax.cu...
  constexpr int max_threads = std::is_same_v<scalar_t, float> ? 1024 : 768; // we need 72 or so 32 bit registers for double
  int threads_target = max_threads;
  while (threads_target / 2 >= 2*max_target_length+1) {
    threads_target /= 2;
  }
  int threads_batch = std::min(max_threads / threads_target, (int) batch_size);
  dim3 block(threads_target, threads_batch);
  dim3 grid(1, (batch_size+threads_batch-1)/threads_batch);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  ctc_loss_log_alpha_gpu_kernel<scalar_t, target_t><<<grid, block, 0, stream>>>(
                      log_alpha.mutable_data_ptr<scalar_t>(),
                      log_probs.const_data_ptr<scalar_t>(), input_lengths_t.const_data_ptr<int64_t>(), log_probs.size(0),
                      targets.const_data_ptr<target_t>(), target_lengths_t.const_data_ptr<int64_t>(), max_target_length,
                      neg_log_likelihood.mutable_data_ptr<scalar_t>(),
                      log_probs.stride(0), log_probs.stride(1), log_probs.stride(2),
                      log_alpha.stride(0), log_alpha.stride(1), log_alpha.stride(2),
                      tg_batch_offsets.const_data_ptr<int64_t>(), tg_target_stride,
                      batch_size, BLANK);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return std::make_tuple(neg_log_likelihood, log_alpha);
}

// The second (backward) half of the forward backward algorithm, (10) and (11). This is parallel to the
// alpha kernel above. (As mentioned above, it might make sense do the calculation in the alpha kernel.)
template<typename scalar_t, typename target_t>
__global__ void
C10_LAUNCH_BOUNDS_2((std::is_same_v<scalar_t, float> ? 1024 : 896), 1)
ctc_loss_backward_log_beta_gpu_kernel(scalar_t* __restrict__ log_beta_data,
                                      const scalar_t*log_probs_data, const int64_t* __restrict__ input_lengths, int64_t max_input_length,
                                      const target_t* __restrict__ targets_data, const int64_t* __restrict__ target_lengths, int64_t max_target_length,
                                      int64_t lp_input_stride, int64_t lp_batch_stride, int64_t lp_char_stride,
                                      int64_t lb_batch_stride, int64_t lb_input_stride, int64_t lb_target_stride,
                                      const int64_t* __restrict__ tg_batch_offsets, int64_t tg_target_stride,
                                      int64_t batch_size, int64_t BLANK) {
  constexpr scalar_t neginf = -INFINITY;

  int64_t b = threadIdx.y + blockIdx.y * blockDim.y;

  if (b >= batch_size)
    return;

  int64_t input_length = input_lengths[b];
  int64_t target_length = target_lengths[b];
  int64_t lp_batch_offset = b*lp_batch_stride;
  int64_t lb_batch_offset = b*lb_batch_stride;
  int64_t tg_batch_offset = tg_batch_offsets[b];

  if (input_length == 0)
    return;

  // "first" row, the beta initialization before eq (10) (t=target_length - differes per batch)
  for (int64_t block_s = 2*max_target_length - (2*max_target_length % blockDim.x); block_s >= 0; block_s -= blockDim.x) {
    int64_t s = threadIdx.x + block_s;
    scalar_t lb;
    if (s == 2*target_length) {
      lb = log_probs_data[lp_batch_offset + (input_length-1) * lp_input_stride + lp_char_stride * BLANK];
    } else if (s == 2 * target_length - 1) { // false for target_length == 0
      int64_t current_target_prime = get_target_prime(
          targets_data,
          tg_batch_offset,
          tg_target_stride,
          s,
          BLANK);
      lb = log_probs_data[lp_batch_offset + (input_length-1) * lp_input_stride + lp_char_stride * current_target_prime];
    } else {
      lb = neginf;
    }
    if (s < 2*max_target_length+1) {
      log_beta_data[lb_batch_offset + (input_length-1) * lb_input_stride + lb_target_stride * s] = lb;
    }
  }

  // go backward in s
  for (int64_t block_s = 2*max_target_length - (2*max_target_length % blockDim.x); block_s >= 0; block_s -= blockDim.x) {
    int64_t s = threadIdx.x + block_s;
    int64_t current_target_prime;
    bool have_three;
    if (s < 2 * target_length + 1 && target_length > 0) {
      current_target_prime = get_target_prime(
          targets_data,
          tg_batch_offset,
          tg_target_stride,
          s,
          BLANK);
      have_three =
          ((s < 2 * target_length - 1) &&
           (get_target_prime(
                targets_data,
                tg_batch_offset,
                tg_target_stride,
                s + 2,
                BLANK) != current_target_prime));
    } else {
      current_target_prime = BLANK;
      have_three = false;
    }
    // now go backward in t. Note that we need to skip the last timestep that we did above.
    for (int64_t t=max_input_length-2; t>=0; t--) {
      __syncthreads(); // on cuda 9 we might use partial synchronization of only the threads within the same batch item
      if ((t < input_length - 1) && (s < 2 * target_length + 1)) {
        scalar_t lb1 = log_beta_data[lb_batch_offset + lb_input_stride * (t+1) + lb_target_stride * s];
        scalar_t lbmax = lb1;
        scalar_t lb2, lb3;

        if (s < 2*target_length) {
          lb2 = log_beta_data[lb_batch_offset + lb_input_stride * (t+1) + lb_target_stride * (s+1)];
          if (lb2 > lbmax)
            lbmax = lb2;
        } else {
          lb2 = neginf;
        }
        if (have_three) {
          lb3 = log_beta_data[lb_batch_offset + lb_input_stride * (t+1) + lb_target_stride * (s+2)];
          if (lb3 > lbmax)
            lbmax = lb3;
        } else {
          lb3 = neginf;
        }
        if (lbmax == neginf)
          lbmax = 0;

        scalar_t lb = std::log(std::exp(lb1-lbmax)+std::exp(lb2-lbmax)+std::exp(lb3-lbmax))+lbmax
          + log_probs_data[lp_batch_offset + t * lp_input_stride + lp_char_stride * current_target_prime];

        log_beta_data[lb_batch_offset + lb_input_stride * t + lb_target_stride * s] = lb;
      } else if (
          (s < 2 * max_target_length + 1) &&
          (((target_length == 0) && (s > 0)) || (s >= 2 * target_length + 1) ||
           (t >= input_length))) {
        log_beta_data
            [lb_batch_offset + lb_input_stride * t + lb_target_stride * s] =
                neginf;
      }
    }
  }
}

// This implements the subtrahend of equation (16) for all *nonblank* characters.
// It assumes you have probs in gradient_data when called
// and it modifies gradient_data to be, the gradient.
// In order to facilitate this inplace update, We don't actually do this in logspace.
// (The other variant implemented uses log_space and the differences seem to be
//  not so problematic at least with unit normal distributed test activations.)
// Internally this uses atomicAdd because different threads may write to the same
// gradient position.
// This is parallelised over b and s again.
// Note that for us, the Z of eqn (16) is actually constant for all t and it is the
// likelihood - this is why we use the negative log likelihood below.
// We also multiply by the input gradient to keep with standard autograd style.
// I took this trick from [2], for moderate alphabet sizes a log-space
// calculation (with an atomic log add) is similarly in performance, but for large
// alphabets the inplace nature is a considerable advantage.
template<typename scalar_t, typename target_t>
__global__ void
#if defined (USE_ROCM)
C10_LAUNCH_BOUNDS_2((std::is_same_v<scalar_t, float> ? 1024 : 896), 1)
#endif
ctc_loss_backward_collect_nonblank_gpu_kernel(scalar_t* __restrict__ gradient_data,
                                                     const scalar_t* __restrict__ grad_out_data, int64_t grad_out_batch_stride,
                                                     const scalar_t* __restrict__ log_alpha_data, const scalar_t* __restrict__ log_beta_data,
                                                     const scalar_t*log_probs_data, const int64_t* __restrict__ input_lengths,
                                                     const target_t* __restrict__ targets_data, const int64_t* __restrict__ target_lengths,
                                                     const scalar_t* __restrict__ neg_log_likelihood_data,
                                                     int64_t gr_input_stride, int64_t gr_batch_stride, int64_t gr_char_stride,
                                                     int64_t lp_input_stride, int64_t lp_batch_stride, int64_t lp_char_stride,
                                                     int64_t la_batch_stride, int64_t la_input_stride, int64_t la_target_stride,
                                                     int64_t lb_batch_stride, int64_t lb_input_stride, int64_t lb_target_stride,
                                                     const int64_t* __restrict__ tg_batch_offsets, int64_t tg_target_stride,
                                              int64_t batch_size, bool zero_infinity) {
  int64_t b = threadIdx.y + blockIdx.y * blockDim.y;
  int64_t s = threadIdx.x + ((int64_t) blockIdx.x) * blockDim.x; // note, this directly indexes into targets, not targets prime!

  if (b >= batch_size)
    return;

  int64_t input_length = input_lengths[b];
  int64_t target_length = target_lengths[b];
  int64_t gr_batch_offset = b*gr_batch_stride;
  int64_t lp_batch_offset = b*lp_batch_stride;
  int64_t la_batch_offset = b*la_batch_stride;
  int64_t lb_batch_offset = b*lb_batch_stride;
  int64_t tg_batch_offset = tg_batch_offsets[b];

  if (s >= target_length)
    return;

  int64_t target = targets_data[tg_batch_offset + s * tg_target_stride];
  scalar_t nll = neg_log_likelihood_data[b];
  scalar_t gr =  grad_out_data[b * grad_out_batch_stride];

  if (zero_infinity && nll == INFINITY)
    return;

  for (int64_t t = 0; t < input_length; t++) {
    scalar_t lp = log_probs_data[lp_batch_offset + t * lp_input_stride + lp_char_stride * target];
    gpuAtomicAddNoReturn(&gradient_data[gr_batch_offset + t * gr_input_stride + gr_char_stride * target],
              -std::exp(log_alpha_data[la_batch_offset + la_input_stride * t + la_target_stride * (s*2+1)]
                        + log_beta_data[lb_batch_offset + lb_input_stride * t + lb_target_stride * (s*2+1)]
                        + nll - lp) * gr);
  }
}

// This is the naive implementation of equation (16). It is parallelised in batch and input timestep.
// It appears to be faster than the above method for small batch sizes.
template<typename scalar_t, typename target_t>
__global__ void
#if defined (USE_ROCM)
C10_LAUNCH_BOUNDS_2((std::is_same_v<scalar_t, float> ? 1024 : 896), 1)
#endif
ctc_loss_backward_collect_gpu_kernel(scalar_t* __restrict__ gradient_data,
                                                     const scalar_t* __restrict__ grad_out_data, int64_t grad_out_batch_stride,
                                                     const scalar_t* __restrict__ log_alpha_data, const scalar_t* __restrict__ log_beta_data,
                                                     const scalar_t*log_probs_data, const int64_t* __restrict__ input_lengths, int64_t max_input_length,
                                                     const target_t* __restrict__ targets_data, const int64_t* __restrict__ target_lengths, int64_t max_target_length,
                                                     const scalar_t* __restrict__ neg_log_likelihood_data,
                                                     int64_t gr_input_stride, int64_t gr_batch_stride, int64_t gr_char_stride,
                                                     int64_t lp_input_stride, int64_t lp_batch_stride, int64_t lp_char_stride,
                                                     int64_t la_batch_stride, int64_t la_input_stride, int64_t la_target_stride,
                                                     int64_t lb_batch_stride, int64_t lb_input_stride, int64_t lb_target_stride,
                                                     const int64_t* __restrict__ tg_batch_offsets, int64_t tg_target_stride,
                                     int64_t batch_size, int64_t num_labels, int64_t BLANK, bool zero_infinity) {

  constexpr scalar_t neginf = -INFINITY;
  int64_t b = threadIdx.y + blockIdx.y * blockDim.y;
  int64_t t = threadIdx.x + ((int64_t) blockIdx.x) * blockDim.x;

  if ((t >= max_input_length) || (b >= batch_size))
    return;

  int64_t input_length = input_lengths[b];
  int64_t target_length = target_lengths[b];
  int64_t gr_batch_offset = b*gr_batch_stride;
  int64_t lp_batch_offset = b*lp_batch_stride;
  int64_t la_batch_offset = b*la_batch_stride;
  int64_t lb_batch_offset = b*lb_batch_stride;
  int64_t tg_batch_offset = tg_batch_offsets[b];

  // collected[b, t, target'[s]] "log+=" log_alpha[t, s]+log_beta[t, s]
  for (int s = 0; s < 2*max_target_length+1; s++) {
    if (s < 2 * target_length + 1) { // if target_length == 0, s == 0
      int64_t current_target_prime = get_target_prime(
          targets_data,
          tg_batch_offset,
          tg_target_stride,
          s,
          BLANK);
      scalar_t log_alpha_beta = (log_alpha_data[la_batch_offset + la_input_stride * t + la_target_stride * s]
                                 + log_beta_data[lb_batch_offset + lb_input_stride * t + lb_target_stride * s]);
      scalar_t& lcab = gradient_data[gr_batch_offset + t * gr_input_stride + gr_char_stride * current_target_prime];
      if (lcab == neginf) {
        lcab = log_alpha_beta;
      } else {
        scalar_t max = ((lcab > log_alpha_beta) ? lcab : log_alpha_beta);
        lcab = std::log(std::exp(lcab-max)+std::exp(log_alpha_beta-max))+max;
      }
    }
  }

  scalar_t nll = neg_log_likelihood_data[b];
  scalar_t gr =  grad_out_data[b * grad_out_batch_stride];

  for (int64_t c = 0; c < num_labels; c++) {
    scalar_t& res = gradient_data[gr_batch_offset + t * gr_input_stride + gr_char_stride * c];
    if (t < input_length && (! zero_infinity || nll != INFINITY)) {
      scalar_t lp = log_probs_data[lp_batch_offset + t * lp_input_stride + lp_char_stride * c];
      res = (std::exp(lp)-std::exp(res + nll - lp)) * gr;
    }
    else {
      res = 0.;
    }
  }
}

// This is to zero gradients which corresponding to the out-of-sequence position
// Those gradients should not be used in any model update since the input
// elements are padded
template<typename scalar_t>
__global__ void
#if defined (USE_ROCM)
C10_LAUNCH_BOUNDS_2((std::is_same_v<scalar_t, float> ? 1024 : 896), 1)
#endif
ctc_loss_zero_padded_gradients(
    scalar_t* __restrict__ gradient_data,   /* (T, B, D) layout */
    const int64_t* __restrict__ input_lengths, /* (B, ) layout */
    int64_t gr_timestep_stride,
    int64_t gr_batch_stride,
    int64_t gr_label_stride,
    int64_t max_input_length, /* T */
    int64_t batch_size, /* B */
    int64_t num_labels  /* D */ ) {
      int64_t b = threadIdx.y + blockIdx.y * blockDim.y;
      int64_t t = threadIdx.x + ((int64_t) blockIdx.x) * blockDim.x;

      if (b >= batch_size || t >= max_input_length) {
        return;
      }

      scalar_t input_length = input_lengths[b];
      if (t >= input_length) {
        for (int l = 0; l < num_labels; l++)
          gradient_data[
            t * gr_timestep_stride + b * gr_batch_stride + l * gr_label_stride]
          = 0.0f;
      }
  }


// The backward. It essentially computes eq 16 by using the above kernels.
// We don't do a lot of checking as we envision this to be called only when backpropagating through a (well-checked) forward.
template<typename scalar_t, ScalarType target_scalar_type>
Tensor ctc_loss_backward_gpu_template(const Tensor& grad_out, const Tensor& log_probs, const Tensor& targets, IntArrayRef input_lengths, IntArrayRef target_lengths,
                                      const Tensor& neg_log_likelihood, const Tensor& log_alpha, int64_t BLANK, bool zero_infinity) {
  constexpr scalar_t neginf = -INFINITY;
  using target_t = typename std::conditional_t<target_scalar_type == kInt, int, int64_t>;
  int64_t batch_size = log_probs.size(1);
  int64_t num_labels = log_probs.size(2);
  int64_t tg_target_stride;

  int64_t max_target_length;
  auto tg_batch_offsets = at::empty({batch_size}, TensorOptions(at::CPU(kLong)));
  auto tg_batch_offsets_data = tg_batch_offsets.mutable_data_ptr<int64_t>();
  if (targets.dim() == 1) { // concatenated targets
    int64_t pos = 0;
    max_target_length = 0;
    for (int64_t i = 0; i < batch_size; i++) {
      tg_batch_offsets_data[i] = pos;
      pos += target_lengths[i];
      if (max_target_length < target_lengths[i])
        max_target_length = target_lengths[i];
    }
    tg_target_stride = targets.stride(0);
  }
  else { // batch x max_target_length
    // dim is 2
    int64_t tg_batch_stride = targets.stride(0);
    for (int64_t i = 0; i < batch_size; i++) {
      tg_batch_offsets_data[i] = i * tg_batch_stride;
    }
    tg_target_stride = targets.stride(1);
    max_target_length = log_alpha.size(2)/2; // targets.size(1) might be larger
  }
  auto target_lengths_t = at::tensor(target_lengths, targets.options().dtype(kLong));
  auto input_lengths_t = at::tensor(input_lengths, targets.options().dtype(kLong));
  tg_batch_offsets = tg_batch_offsets.cuda();

  Tensor log_beta = at::empty_like(log_alpha, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  log_beta.fill_(neginf);

  Tensor grad = at::full_like(log_probs, neginf, LEGACY_CONTIGUOUS_MEMORY_FORMAT); // initialization for log(sum (alpha beta))

  // As above, there may be better configurations to use.
  constexpr int max_threads = std::is_same_v<scalar_t, float> ? 1024 : 896; // we need 72 or so 32 bit registers for double
  int threads_target = max_threads;
  while (threads_target / 2 >= 2*max_target_length+1) {
    threads_target /= 2;
  }
  int threads_batch = std::min(max_threads / threads_target, (int) batch_size);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  {
    dim3 block(threads_target, threads_batch);
    dim3 grid(1, (batch_size+threads_batch-1)/threads_batch);
    ctc_loss_backward_log_beta_gpu_kernel<scalar_t, target_t><<<grid, block, 0, stream>>>
      (log_beta.mutable_data_ptr<scalar_t>(),
       log_probs.const_data_ptr<scalar_t>(), input_lengths_t.const_data_ptr<int64_t>(), log_probs.size(0),
       targets.const_data_ptr<target_t>(), target_lengths_t.const_data_ptr<int64_t>(), max_target_length,
       log_probs.stride(0), log_probs.stride(1), log_probs.stride(2),
       log_beta.stride(0), log_beta.stride(1), log_beta.stride(2),
       tg_batch_offsets.const_data_ptr<int64_t>(), tg_target_stride,
       batch_size, BLANK);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }

  // Very crude heuristic for what is a small problem., based on linearly regressing problem dimensions on
  // the (capped) difference of timings.
  // Note that for OK problems target length <= input length, so we
  // only consider input length.
  bool is_large = (2*log_probs.size(0)+(24*batch_size)/10+(2*num_labels)/10) > 450;
  if (is_large) { // large alphabet, large batch
    // this computes the probs, minuend in (16)
    at::exp_out(grad, log_probs);
    // now we compute the subtrahend for the blanks. It is a straightforward reduction because we know that
    // blanks are in every other position.
    // maybe we should kernelize this, too.
    auto grad_blank = grad.narrow(2, BLANK, 1);
    grad_blank -= (at::logsumexp(log_alpha.as_strided({batch_size, log_alpha.size(1), max_target_length+1},
                                                      {log_alpha.stride(0), log_alpha.stride(1), log_alpha.stride(2)*2})
                                 + log_beta.as_strided({batch_size, log_beta.size(1), max_target_length+1},
                                                       {log_beta.stride(0), log_beta.stride(1), log_beta.stride(2)*2}),
                                 2, true)
                   .permute({1, 0, 2})
                   .add_(neg_log_likelihood.view({1, batch_size, 1}))
                   .sub_(log_probs.narrow(2, BLANK, 1))
                   .exp_()
                   );
    // scale by output gradient (blanks and first summand of non-blanks)
    grad *= grad_out.view({1, batch_size, 1});
    if (zero_infinity) {
      grad = at::where(neg_log_likelihood.view({1, batch_size, 1}) == Scalar(INFINITY), at::zeros({}, grad.options()), grad);
    }

    // For the non-blank characters, we use a kernel to compute the subtrahend.
    // Again we might configure block and grid in a better way.
    int threads_target = max_threads;
    while (threads_target / 2 >= max_target_length && threads_target > 1) {
      threads_target /= 2;
    }
    int threads_batch = std::min(max_threads / threads_target, (int) batch_size);
    dim3 block(threads_target, threads_batch);
    dim3 grid(
        std::max<int>(
            (max_target_length + threads_target - 1) / threads_target, 1),
        (batch_size + threads_batch - 1) / threads_batch,
        1);
    ctc_loss_backward_collect_nonblank_gpu_kernel<scalar_t, target_t><<<grid, block, 0, stream>>>
      (grad.mutable_data_ptr<scalar_t>(),
       grad_out.const_data_ptr<scalar_t>(), grad_out.stride(0),
       log_alpha.const_data_ptr<scalar_t>(), log_beta.const_data_ptr<scalar_t>(),
       log_probs.const_data_ptr<scalar_t>(), input_lengths_t.const_data_ptr<int64_t>(),
       targets.const_data_ptr<target_t>(), target_lengths_t.const_data_ptr<int64_t>(),
       neg_log_likelihood.const_data_ptr<scalar_t>(),
       grad.stride(0), grad.stride(1), grad.stride(2),
       log_probs.stride(0), log_probs.stride(1), log_probs.stride(2),
       log_alpha.stride(0), log_alpha.stride(1), log_alpha.stride(2),
       log_beta.stride(0), log_beta.stride(1), log_beta.stride(2),
       tg_batch_offsets.const_data_ptr<int64_t>(), tg_target_stride,
       batch_size, zero_infinity);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else { // small problem, use naive algorithm
    // Still no block/grid configuration guru...
    int threads_input = max_threads;
    while (threads_input / 2 >= log_probs.size(0) && threads_input > 1) {
      threads_input /= 2;
    }
    threads_batch = std::min(max_threads / threads_input, (int) batch_size);
    dim3 block(threads_input, threads_batch);
    dim3 grid((log_probs.size(0) + threads_input-1)/threads_input, (batch_size+threads_batch-1)/threads_batch);
    ctc_loss_backward_collect_gpu_kernel<scalar_t, target_t><<<grid, block, 0, stream>>>
      (grad.mutable_data_ptr<scalar_t>(),
       grad_out.const_data_ptr<scalar_t>(), grad_out.stride(0),
       log_alpha.const_data_ptr<scalar_t>(), log_beta.const_data_ptr<scalar_t>(),
       log_probs.const_data_ptr<scalar_t>(), input_lengths_t.const_data_ptr<int64_t>(), log_probs.size(0),
       targets.const_data_ptr<target_t>(), target_lengths_t.const_data_ptr<int64_t>(), max_target_length,
       neg_log_likelihood.const_data_ptr<scalar_t>(),
       grad.stride(0), grad.stride(1), grad.stride(2),
       log_probs.stride(0), log_probs.stride(1), log_probs.stride(2),
       log_alpha.stride(0), log_alpha.stride(1), log_alpha.stride(2),
       log_beta.stride(0), log_beta.stride(1), log_beta.stride(2),
       tg_batch_offsets.const_data_ptr<int64_t>(), tg_target_stride,
       batch_size, num_labels, BLANK, zero_infinity);
    C10_CUDA_KERNEL_LAUNCH_CHECK(); // catch launch errors
  }

  // zero those invalid graident elements due to padding
  {
    int threads_input = max_threads;
    while (threads_input / 2 >= log_probs.size(0)) {
      threads_input /= 2;
    }
    threads_batch = std::min(max_threads / threads_input, (int) batch_size);
    dim3 block(threads_input, threads_batch);
    dim3 grid(
      (log_probs.size(0) + threads_input-1)/threads_input,
      (batch_size+threads_batch-1)/threads_batch);
    ctc_loss_zero_padded_gradients<scalar_t><<<grid, block, 0, stream>>>(
      grad.mutable_data_ptr<scalar_t>(),
      input_lengths_t.const_data_ptr<int64_t>(),
      grad.stride(0),
      grad.stride(1),
      grad.stride(2),
      grad.size(0),
      grad.size(1),
      grad.size(2)
    );
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }

  return grad;
}

} // namespace

std::tuple<Tensor, Tensor> ctc_loss_gpu(const Tensor& log_probs, const Tensor& targets, IntArrayRef input_lengths, IntArrayRef target_lengths, int64_t BLANK, bool zero_infinity) {
  (void)zero_infinity; // only used for backward
  return AT_DISPATCH_FLOATING_TYPES(log_probs.scalar_type(), "ctc_loss_cuda", [&] {
      if (targets.scalar_type() == kLong) {
        return ctc_loss_gpu_template<scalar_t, kLong>(log_probs, targets, input_lengths, target_lengths, BLANK);
      } else {
        return ctc_loss_gpu_template<scalar_t, kInt>(log_probs, targets, input_lengths, target_lengths, BLANK);
      }
    });
}

Tensor ctc_loss_backward_gpu(const Tensor& grad, const Tensor& log_probs, const Tensor& targets, IntArrayRef input_lengths, IntArrayRef target_lengths,
                             const Tensor& neg_log_likelihood, const Tensor& log_alpha, int64_t BLANK, bool zero_infinity) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("ctc_loss_backward_gpu");
  return AT_DISPATCH_FLOATING_TYPES(log_probs.scalar_type(), "ctc_loss_backward_cuda", [&] {
      if (targets.scalar_type() == kLong) {
        return ctc_loss_backward_gpu_template<scalar_t, kLong>(grad, log_probs, targets, input_lengths, target_lengths, neg_log_likelihood, log_alpha, BLANK, zero_infinity);
      } else {
        return ctc_loss_backward_gpu_template<scalar_t, kInt>(grad, log_probs, targets, input_lengths, target_lengths, neg_log_likelihood, log_alpha, BLANK, zero_infinity);
      }
    });
}

} // at::native
