#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/DeviceUtils.cuh>
#include <ATen/native/EmbeddingBag.h>
#include <ATen/TensorUtils.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/arange.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/zeros.h>
#include <ATen/ops/_embedding_bag_native.h>
#include <ATen/ops/_embedding_bag_forward_only_native.h>
#include <ATen/ops/_embedding_bag_dense_backward_native.h>
#include <ATen/ops/_embedding_bag_per_sample_weights_backward_native.h>
#endif

#include <ATen/cuda/cub.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <ATen/native/cuda/EmbeddingBackwardKernel.cuh>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#include <c10/macros/Macros.h>

#if CUB_SUPPORTS_SCAN_BY_KEY()
#include <thrust/iterator/reverse_iterator.h>
#endif

namespace at::native {

#if !CUB_SUPPORTS_SCAN_BY_KEY()
template<typename index_t>
void embedding_dense_backward_cuda_scan(Tensor &sorted_indices, Tensor &count);
#endif

namespace {

std::pair<Tensor, Tensor> promoteIndicesAndOffsets(
    const Tensor& indices,
    const Tensor& offsets) {
  const auto commonType =
      promoteTypes(offsets.scalar_type(), indices.scalar_type());
  return {
      indices.scalar_type() == commonType ? indices
                                          : indices.toType(commonType),
      offsets.scalar_type() == commonType ? offsets
                                          : offsets.toType(commonType)};
}

// This kernel assumes that all input tensors except `weight` and
// per_sample_weights are contiguous.
template <typename scalar_t, typename index_t>
__global__ void EmbeddingBag_updateOutputKernel_max(
    const index_t *input, const index_t *offsets, const scalar_t *weight, scalar_t *output,
    index_t *offset2bag, int64_t numIndices, int64_t numBags,
    int64_t featureSize, int64_t weight_stride0, int64_t weight_stride1,
    index_t *bag_size, index_t *max_indices,
    index_t padding_idx, int64_t numRows) {

  // the strategy here is that each bag x feature is handled by a single thread

  int64_t chunksPerBag = ceil_div(featureSize, (int64_t)blockDim.x);
  int64_t numChunks = numBags * chunksPerBag;
  int64_t chunkOffset = blockIdx.x * blockDim.y + threadIdx.y;
  int64_t chunkStride = gridDim.x * blockDim.y;

  for (int64_t chunk = chunkOffset; chunk < numChunks; chunk += chunkStride) {
    int64_t featureDim = (chunk % chunksPerBag) * blockDim.x + threadIdx.x;
    if (featureDim < featureSize) {
      int64_t bag = chunk / chunksPerBag;
      const scalar_t *weightFeat = weight + featureDim * weight_stride1;
      int64_t begin = bag == 0 ? 0 : offsets[bag]; // forces first offset to be 0 instead of asserting on it
      int64_t end = (bag < numBags - 1) ? (offsets[bag + 1]) : numIndices;
      CUDA_KERNEL_ASSERT(end >= begin);
      scalar_t weightFeatMax = 0;
      int64_t bag_size_ = 0;
      int64_t maxWord = -1;
      for (int64_t emb = begin; emb < end; emb++) {
        bool pad = (input[emb] == padding_idx);
        CUDA_KERNEL_ASSERT(input[emb] < numRows);
        const int64_t weightRow = input[emb] * weight_stride0;
        scalar_t weightValue = weightFeat[weightRow];
        if (bag_size_ == 0 || weightValue > weightFeatMax) {
          weightFeatMax = pad ? weightFeatMax : weightValue;
          maxWord = pad ? maxWord : input[emb];
        }
        bag_size_ += pad ? 0 : 1;

        if (featureDim == 0) {
          offset2bag[emb] = bag;
        }
      }
      bag_size[bag] = bag_size_;
      max_indices[bag * featureSize + featureDim] = maxWord;
      output[bag * featureSize + featureDim] = weightFeatMax;
    }
  }
}

// This kernel assumes that all input tensors except `weight` and
// per_sample_weights are contiguous.
template <typename scalar_t, typename index_t>
__global__ void EmbeddingBag_updateOutputKernel_sum_mean(
    const index_t *input, const index_t *offsets, const scalar_t *weight, scalar_t *output,
    index_t *offset2bag, int64_t numIndices, int64_t numBags,
    int64_t featureSize, int64_t weight_stride0, int64_t weight_stride1,
    int mode, index_t *bag_size,
    const scalar_t* per_sample_weights, int64_t per_sample_weights_stride,
    index_t padding_idx, int64_t numRows) {

  // the strategy here is that each bag x feature is handled by a single thread

  using accscalar_t = acc_type<scalar_t, true>;
  int64_t chunksPerBag = ceil_div(featureSize, (int64_t)blockDim.x);
  int64_t numChunks = numBags * chunksPerBag;
  int64_t chunkOffset = blockIdx.x * blockDim.y + threadIdx.y;
  int64_t chunkStride = gridDim.x * blockDim.y;

  for (int64_t chunk = chunkOffset; chunk < numChunks; chunk += chunkStride) {
    int64_t featureDim = (chunk % chunksPerBag) * blockDim.x + threadIdx.x;
    if (featureDim < featureSize) {
      int64_t bag = chunk / chunksPerBag;
      const scalar_t *weightFeat = weight + featureDim * weight_stride1;
      int64_t begin = bag == 0 ? 0 : offsets[bag]; // forces first offset to be 0 instead of asserting on it
      int64_t end = (bag < numBags - 1) ? (offsets[bag + 1]) : numIndices;
      CUDA_KERNEL_ASSERT(end >= begin);
      accscalar_t weightFeatSum = 0;
      int64_t bag_size_ = 0;
      for (int64_t emb = begin; emb < end; emb++) {
        index_t input_idx = input[emb];
        bool pad = (input_idx == padding_idx);
        CUDA_KERNEL_ASSERT(0 <= input_idx && input_idx < numRows);
        const int64_t weightRow = input_idx * weight_stride0;
        scalar_t weightValue = weightFeat[weightRow];
        weightValue = pad ? static_cast<scalar_t>(0) : weightValue;
        if (per_sample_weights) {
          accscalar_t scaleWeightBy = static_cast<accscalar_t>(
              per_sample_weights[emb * per_sample_weights_stride]);
          weightFeatSum += scaleWeightBy * static_cast<accscalar_t>(weightValue);
        } else {
          weightFeatSum += static_cast<accscalar_t>(weightValue);
        }
        bag_size_ += pad ? 0 : 1;

        if (featureDim == 0) {
          offset2bag[emb] = bag;
        }
      }
      if (mode == static_cast<int64_t>(EmbeddingBagMode::MEAN)) {
        if (bag_size_ != 0) {
          weightFeatSum = weightFeatSum / static_cast<accscalar_t>(bag_size_);
        }
      }
      bag_size[bag] = bag_size_;
      output[bag * featureSize + featureDim] = static_cast<scalar_t>(weightFeatSum);
    }
  }
}

Tensor embedding_bag_backward_cuda_sum_avg(
                                   const Tensor &grad,
                                   const Tensor &indices_,
                                   const Tensor &offset2bag,
                                   const Tensor &bag_size,
                                   int64_t num_weights,
                                   bool scale_grad_by_freq, int64_t mode,
                                   const Tensor& per_sample_weights,
                                   int64_t padding_idx) {
  auto indices = indices_.contiguous();

  ptrdiff_t num_indices = indices.numel();

  if (num_indices == 0) {
    // all empty bags
    return at::zeros({num_weights, grad.size(1)}, grad.options());
  }

  auto sorted_indices = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto orig_indices = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor count;

  AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_bag_backward_cuda_sum_avg", [&] () {
    auto range = at::arange(num_indices, indices.options());
    // int64_t nbits = cuda::hipcub::get_num_bits(num_weights);
    cuda::hipcub::radix_sort_pairs(
      indices.const_data_ptr<index_t>(), sorted_indices.mutable_data_ptr<index_t>(),
      range.const_data_ptr<index_t>(), orig_indices.mutable_data_ptr<index_t>(),
      num_indices, false/*, 0, nbits*/);
  });

  if (scale_grad_by_freq) {
    count = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
#if CUB_SUPPORTS_SCAN_BY_KEY()
    AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_bag_backward_cuda_sum_avg", [&] () {
      hipStream_t stream = at::cuda::getCurrentCUDAStream();

      // Compute an increasing sequence per unique item in sortedIndices:
      // sorted: 2 5 5 5 7 7 8 9 9
      //  count: 1 1 2 3 1 2 1 1 2
      auto sorted_data = sorted_indices.const_data_ptr<index_t>();
      auto count_data = count.mutable_data_ptr<index_t>();
      cuda::hipcub::inclusive_sum_by_key(
        sorted_data,
        ATEN_CUB_CONSTANT_ITERATOR(index_t)(1),
        count_data,
        num_indices
      );

      // Take the maximum of each count per unique key in reverse:
      // sorted: 2 5 5 5 7 7 8 9 9
      //  count: 1 3 3 3 2 2 1 2 2
      cuda::hipcub::inclusive_scan_by_key(
        thrust::make_reverse_iterator(sorted_data + num_indices),
        thrust::make_reverse_iterator(count_data + num_indices),
        thrust::make_reverse_iterator(count_data + num_indices),
        ATEN_CUB_MAXIMUM(),
        num_indices
      );
    });
#else
    AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_bag_backward_cuda_sum_avg", [&] () {
      embedding_dense_backward_cuda_scan<index_t>(sorted_indices, count);
    });
#endif
  }
  return embedding_backward_cuda_kernel(grad, orig_indices, sorted_indices,
      count, num_weights, padding_idx, mode == EmbeddingBagMode::MEAN, offset2bag,
      bag_size, per_sample_weights);
}

template <typename scalar_t, typename index_t>
__global__ void EmbeddingBag_accGradParametersKernel_max(
    const index_t *max_indices, const scalar_t *gradOutput,
    scalar_t *gradWeight, int64_t stride, int64_t numBags,
    index_t padding_idx, const index_t numel) {

  using accscalar_t = acc_type<scalar_t, true>;

  int64_t chunksPerBag = ceil_div(stride, (int64_t)blockDim.x);
  int64_t numChunks = numBags * chunksPerBag;
  int64_t chunkOffset = blockIdx.x * blockDim.y + threadIdx.y;
  int64_t chunkStride = gridDim.x * blockDim.y;

  for (int64_t chunk = chunkOffset; chunk < numChunks; chunk += chunkStride) {
    int64_t featureDim = (chunk % chunksPerBag) * blockDim.x + threadIdx.x;
    if (featureDim < stride) {
      int64_t bag = chunk / chunksPerBag;

      index_t word_idx = max_indices[bag * stride + featureDim];
      if (word_idx >= 0 && word_idx != padding_idx) {
        // If bag is empty, we have max_indices[idx] set to -1 in forward.
        fastAtomicAdd(
            gradWeight, static_cast<index_t>(word_idx * stride + featureDim),
            numel, gradOutput[bag * stride + featureDim], true);
      }
    }
  }
}

Tensor embedding_bag_backward_cuda_max(const Tensor &grad,
                                   const Tensor &max_indices,
                                   int64_t num_weights,
                                   int64_t padding_idx) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("embedding_bag_backward_cuda_max");

  auto grad_weight = at::zeros({num_weights, grad.size(1)}, grad.options());

  int64_t stride = grad_weight.stride(0);

  int64_t numBags = grad.size(0);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

#if defined(USE_ROCM)
  dim3 block = dim3(64, 4);
#else
  dim3 block = dim3(32, 8);
#endif
  int grid = 1024;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad.scalar_type(), "embedding_bag_backward_cuda_max", [&] {
        AT_DISPATCH_INDEX_TYPES(max_indices.scalar_type(), "embedding_bag_backward_cuda_max", [&] () {
          EmbeddingBag_accGradParametersKernel_max<
              scalar_t, index_t><<<grid, block, 0, stream>>>(
              max_indices.const_data_ptr<index_t>(), grad.const_data_ptr<scalar_t>(),
              grad_weight.mutable_data_ptr<scalar_t>(), stride, numBags,
              padding_idx, grad_weight.numel());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
  });

  return grad_weight;
}
}

// Assumes all input tensors are contiguous.
// See NOTE [ embedding_bag Native Functions ] in native_functions.yaml for details
std::tuple<Tensor, Tensor, Tensor, Tensor>
_embedding_bag_forward_only_cuda(const Tensor &weight, const Tensor &indices,
                   const Tensor &offsets, const bool scale_grad_by_freq,
                   const int64_t mode, bool sparse, const std::optional<Tensor>& per_sample_weights_opt,
                   bool include_last_offset, int64_t padding_idx) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> per_sample_weights_maybe_owned = at::borrow_from_optional_tensor(per_sample_weights_opt);
  const Tensor& per_sample_weights = *per_sample_weights_maybe_owned;

  return _embedding_bag_cuda(
      weight,
      indices,
      offsets,
      scale_grad_by_freq,
      mode,
      sparse,
      per_sample_weights,
      include_last_offset,
      padding_idx);
}

// Assumes all input tensors are contiguous.
// See NOTE [ embedding_bag Native Functions ] in native_functions.yaml for details
std::tuple<Tensor, Tensor, Tensor, Tensor>
_embedding_bag_cuda(const Tensor &weight, const Tensor &indices_,
                   const Tensor &offsets_, const bool scale_grad_by_freq,
                   const int64_t mode, bool sparse, const std::optional<Tensor>& per_sample_weights_opt,
                   bool include_last_offset, int64_t padding_idx) {
  TORCH_CHECK(indices_.dim() == 1 || indices_.dim() == 2,
      "input has to be a 1D or 2D Tensor, but got Tensor of dimension ",
      indices_.dim());
  if (indices_.dim() == 1) {
    TORCH_CHECK(offsets_.dim() == 1,
        "offsets has to be a 1D Tensor, but got Tensor of dimension ",
        offsets_.dim());
  }
  TORCH_CHECK(weight.dim() == 2,
      "weight has to be a 2D Tensor, but got Tensor of dimension ",
      weight.dim());
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> per_sample_weights_maybe_owned = at::borrow_from_optional_tensor(per_sample_weights_opt);
  const Tensor& per_sample_weights = *per_sample_weights_maybe_owned;

  Tensor indices, offsets;
  std::tie(indices, offsets) = promoteIndicesAndOffsets(indices_, offsets_);
  auto indices_arg = TensorArg(indices, "indices", 1);
  checkScalarTypes("embedding_bag_cuda", indices_arg, {kLong, kInt});
  auto offsets_arg = TensorArg(offsets, "offsets", 1);
  checkScalarTypes("embedding_bag_cuda", offsets_arg, {kLong, kInt});
  checkSameType("embedding_bag_cuda", indices_arg, offsets_arg);
  auto weight_arg = TensorArg(weight, "weight", 1);
  checkSameGPU("embedding_bag_cuda", weight_arg, indices_arg);
  checkSameGPU("embedding_bag_cuda", weight_arg, offsets_arg);

  int64_t numIndices = indices.size(0);
  int64_t numBags = offsets.size(0);
  if (include_last_offset) {
    // Check https://github.com/pytorch/pytorch/issues/29019
    // We plan to add one more element in offsets, which is equal to the size of
    // indices. Currently for cuda devices, we still use the legacy
    // implementation even this flag is enabled.
    TORCH_CHECK(
        numBags >= 1, "include_last_offset: numBags should be at least 1");
    numBags -= 1;
  }
  int64_t featureSize = weight.size(1);

  auto bag_size = at::empty(offsets.sizes(), indices.options());
  auto offset2bag =
      at::empty({indices.size(0)}, indices.options()); // offset2bag = [0 0 0 0 0]

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto output = at::empty({numBags, featureSize}, weight.options());

  Tensor max_indices;

  if (mode == EmbeddingBagMode::MAX) {
    max_indices = at::empty({numBags, featureSize}, indices.options());
  } else {
    // No need to allocate if we aren't doing a backwards pass
    max_indices = at::empty({0}, indices.options());
  }

#if defined(USE_ROCM)
  dim3 block = dim3(64, 4);
#else
  dim3 block = dim3(32, 8);
#endif
  int grid = 1024;
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, weight.scalar_type(), "embedding_bag_cuda", [&] {
    AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_bag_cuda", [&] () {
      if (mode == EmbeddingBagMode::MAX) {
        EmbeddingBag_updateOutputKernel_max<scalar_t, index_t><<<grid, block, 0, stream>>>(
            indices.const_data_ptr<index_t>(), offsets.const_data_ptr<index_t>(),
            weight.const_data_ptr<scalar_t>(), output.mutable_data_ptr<scalar_t>(),
            offset2bag.mutable_data_ptr<index_t>(), numIndices, numBags, featureSize,
            weight.stride(0), weight.stride(1), bag_size.mutable_data_ptr<index_t>(),
            max_indices.mutable_data_ptr<index_t>(),
            padding_idx, weight.size(0));
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        EmbeddingBag_updateOutputKernel_sum_mean<scalar_t, index_t><<<grid, block, 0, stream>>>(
            indices.const_data_ptr<index_t>(), offsets.const_data_ptr<index_t>(),
            weight.const_data_ptr<scalar_t>(), output.mutable_data_ptr<scalar_t>(),
            offset2bag.mutable_data_ptr<index_t>(), numIndices, numBags, featureSize,
            weight.stride(0), weight.stride(1), mode, bag_size.mutable_data_ptr<index_t>(),
            per_sample_weights.defined() ? per_sample_weights.const_data_ptr<scalar_t>() : NULL,
            per_sample_weights.defined() ? per_sample_weights.stride(0) : 0,
            padding_idx, weight.size(0));
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  });

  return std::tuple<Tensor, Tensor, Tensor, Tensor>(output, offset2bag, bag_size, max_indices);
}

Tensor _embedding_bag_dense_backward_cuda(const Tensor &grad_, const Tensor &indices,
                                   const Tensor &offset2bag,
                                   const Tensor &bag_size_,
                                   const Tensor &max_indices,
                                   int64_t num_weights,
                                   bool scale_grad_by_freq, int64_t mode, const std::optional<Tensor>& per_sample_weights_opt,
                                   int64_t padding_idx) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> per_sample_weights_maybe_owned = at::borrow_from_optional_tensor(per_sample_weights_opt);
  const Tensor& per_sample_weights = *per_sample_weights_maybe_owned;

  // indices, offsets and offset2bag are assumed having correct dtypes and
  // contiguous here due to the checks in _embedding_bag_backward in
  // EmbeddingBag.cpp.
  // Also see NOTE [ embedding_bag Native Functions ] in native_functions.yaml
  // for more details.

  Tensor grad = grad_.contiguous();
  auto indices_arg = TensorArg(indices, "indices", 1);
  auto grad_arg = TensorArg(grad, "grad", 1);
  checkSameGPU("embedding_bag_cuda", grad_arg, indices_arg);


  switch (static_cast<EmbeddingBagMode>(mode)) {
    case EmbeddingBagMode::SUM:
    case EmbeddingBagMode::MEAN:
      if (mode == EmbeddingBagMode::MEAN)
        AT_ASSERT(!per_sample_weights.defined());
      return embedding_bag_backward_cuda_sum_avg(grad, indices, offset2bag,
              bag_size_, num_weights, scale_grad_by_freq, mode,
              per_sample_weights, padding_idx);

    case EmbeddingBagMode::MAX:
      AT_ASSERT(!per_sample_weights.defined());
      return embedding_bag_backward_cuda_max(grad, max_indices, num_weights,
              padding_idx);

    default:
      TORCH_CHECK(false,
          "Unknown mode for embedding_bag_backward_cuda ", mode);
  }
}

template <typename scalar_t, typename index_t>
__global__ static void _embedding_bag_per_sample_weights_backward_kernel(
    const scalar_t* grad, int64_t grad_stride0, int64_t grad_stride1,
    const scalar_t* weight, int64_t weight_stride0, int64_t weight_stride1,
    const index_t* indices,  // contiguous
    const index_t* offset2bag,  // contiguous
    int64_t num_samples,
    int64_t embedding_features,
    scalar_t* output,
    index_t padding_idx) {
  using accscalar_t = acc_type<scalar_t, true>;
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int warp = idx / C10_WARP_SIZE;
  const int thread_in_warp = idx % C10_WARP_SIZE;
  const int num_warps = blockDim.x * gridDim.x / C10_WARP_SIZE;

  // Each warp is responsible for the accumulation of one sample.
  // This involves doing one dot product between grad[bag_idx] and weight[embedding_idx].
  for (int sample_idx = warp; sample_idx < num_samples; sample_idx += num_warps) {
    accscalar_t result = 0.;
    const int bag_idx = (int)offset2bag[sample_idx];
    const int embedding_idx = (int)indices[sample_idx];
    if (embedding_idx != padding_idx) {
      for (int feature_idx = thread_in_warp; feature_idx < embedding_features;
          feature_idx += C10_WARP_SIZE) {
        result +=
            grad[grad_stride0 * bag_idx + grad_stride1 * feature_idx] *
            weight[weight_stride0 * embedding_idx + weight_stride1 * feature_idx];
      }
    }
    result = cuda_utils::WarpReduceSum<accscalar_t>(result);
    if (thread_in_warp == 0) {
      output[sample_idx] = result;
    }
  }
}

Tensor _embedding_bag_per_sample_weights_backward_cuda(
    const Tensor& grad,
    const Tensor& weight,  // NB: embedding table, not per_sample_weights
    const Tensor& indices_,
    const Tensor& offsets_,
    const Tensor& offset2bag,
    int64_t mode,
    int64_t padding_idx) {
  TORCH_CHECK(
      mode == EmbeddingBagMode::SUM,
      "embedding_bag_backward: per_sample_weights only supported for mode='sum'");

  AT_ASSERT(grad.dim() == 2);
  auto embedding_features = grad.size(1);

  Tensor indices, offsets;
  std::tie(indices, offsets) = promoteIndicesAndOffsets(indices_, offsets_);
  AT_ASSERT(indices.dim() == 1);
  auto num_samples = indices.size(0);

  AT_ASSERT(weight.dim() == 2);
  AT_ASSERT(weight.size(1) == embedding_features);

  const int threads_per_block = 512;
  const int warps_per_block = threads_per_block / at::cuda::warp_size();

  dim3 block(threads_per_block);
  dim3 grid((num_samples + warps_per_block - 1) / warps_per_block);

  auto output = at::empty({num_samples}, grad.options());

  // Early return when there is no samples in the batch. This saves unnecessary kernel
  // launch, but also prevents hipGetLastError() to complain about invalid launch args
  if (num_samples == 0) {
    return output;
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad.scalar_type(), "_embedding_bag_per_sample_weights_backward_cuda", [&]() {
      AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "_embedding_bag_per_sample_weights_backward_cuda", [&]() {
        _embedding_bag_per_sample_weights_backward_kernel<scalar_t, index_t>
          <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            grad.const_data_ptr<scalar_t>(), grad.stride(0), grad.stride(1),
            weight.const_data_ptr<scalar_t>(), weight.stride(0), weight.stride(1),
            indices.const_data_ptr<index_t>(),
            offset2bag.const_data_ptr<index_t>(),
            num_samples,
            embedding_features,
            output.mutable_data_ptr<scalar_t>(),
            padding_idx);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
    }
  );
  return output;
}

} // namespace at::native
