#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Exception.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/adaptive_avg_pool3d_backward_native.h>
#include <ATen/ops/adaptive_avg_pool3d_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#endif

#include <ATen/native/AdaptivePooling.h>

#include <algorithm>
#include <cfloat>
#include <cmath>


namespace at::native {

namespace {

__device__ inline int64_t start_index(int64_t a, int64_t b, int64_t c) {
  return (a / b) * c + ((a % b) * c) / b;
}

__device__ inline int64_t end_index(int64_t a, int64_t b, int64_t c) {
  return 1 + ((a + 1) * c - 1) / b;
}

// 5d tensor B x D x T x H x W
// All kernels view batch dim B and dim D as collapsed.

/*
 * Description:
 *    this function adaptively average pools an input 5D tensor along dimensions
 * 2, 3, and 4 5D input, 5D output
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 */
template <typename scalar_t, typename accscalar_t>
__global__ void adaptiveaveragepool(
    const scalar_t *input, scalar_t *output,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW,
    int64_t sizeD, int64_t istrideB, int64_t istrideD,
    int64_t istrideT, int64_t istrideH, int64_t istrideW,
    int64_t offsetZ) {
  // iterates on output pixels
  int ot, oh, ow;

  // compute offsets based on thread/block ID
  int ostartH = blockIdx.y * blockDim.y + threadIdx.y;
  int oendH = osizeH;
  int ostepH = gridDim.y * blockDim.y;
  int ostartW = threadIdx.x;
  int oendW = osizeW;
  int ostepW = blockDim.x;

  // select output plane
  int64_t o_plane = blockIdx.x + offsetZ;
  ot = o_plane % osizeT; // output frame/time
  int d = o_plane / osizeT; // flattened (batch, channel) index

  // Decompose d into batch and channel indices
  int batch_idx = d / sizeD;
  int channel_idx = d % sizeD;

  // input frame/time range is fixed.
  int istartT = start_index(ot, osizeT, isizeT);
  int iendT = end_index(ot, osizeT, isizeT);
  int kT = iendT - istartT;

  // output offset by slice/feature and frame/time
  scalar_t *output_dt = output + o_plane*osizeH*osizeW;

  // For all output pixels...
  for (oh = ostartH; oh < oendH; oh += ostepH) {
    int istartH = start_index(oh, osizeH, isizeH);
    int iendH = end_index(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for (ow = ostartW; ow < oendW; ow += ostepW) {
      int istartW = start_index(ow, osizeW, isizeW);
      int iendW = end_index(ow, osizeW, isizeW);
      int kW = iendW - istartW;

      scalar_t *ptr_output = output_dt + oh*osizeW + ow;
      accscalar_t sum = static_cast<accscalar_t>(0);

      int it, ih, iw;
      for (it = 0; it < kT; ++it) {
        for (ih = 0; ih < kH; ++ih) {
          for (iw = 0; iw < kW; ++iw) {
            int64_t input_offset = batch_idx * istrideB + channel_idx * istrideD +
                                   (istartT + it) * istrideT +
                                   (istartH + ih) * istrideH + (istartW + iw) * istrideW;
            scalar_t val = input[input_offset];
            sum += static_cast<accscalar_t>(val);
          }
        }
      }
      // Update output
      const accscalar_t divide_factor = static_cast<accscalar_t>(kT * kH * kW);
      *ptr_output = static_cast<scalar_t>(sum / divide_factor);
    }
  }
}

template <typename scalar_t, typename accscalar_t>
void adaptiveaveragepool_loop(
    const scalar_t *input_data, scalar_t *output_data,
    int64_t totalZ,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW,
    int64_t sizeD, int64_t istrideB, int64_t istrideD, int64_t istrideT, int64_t istrideH, int64_t istrideW) {
  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = std::max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);
    adaptiveaveragepool<scalar_t, accscalar_t>
      <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        input_data, output_data,
        isizeT, isizeH, isizeW,
        osizeT, osizeH, osizeW,
        sizeD, istrideB, istrideD,
        istrideT, istrideH, istrideW,
        offsetZ);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    totalZ -= 65535;
    offsetZ += 65535;
  }
}

/*
 * Description:
 *    This function computes the gradInput from gradOutput.
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 */
template <typename scalar_t, typename accscalar_t>
__global__ void adaptiveaveragegradinput(
    scalar_t *gradInput, const scalar_t *gradOutput,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW,
    int64_t offsetZ)
{
  // iterators on input pixels
  int it, ih, iw;

  // compute offsets based on thread/block ID
  int istartH = blockIdx.y * blockDim.y + threadIdx.y;
  int iendH = isizeH;
  int istepH = gridDim.y * blockDim.y;
  int istartW = threadIdx.x;
  int iendW = isizeW;
  int istepW = blockDim.x;

  // select input plane
  int64_t i_plane = blockIdx.x + offsetZ;
  it = i_plane % isizeT; // output frame/time
  int d = i_plane / isizeT; // slice/feature

  // output frame/time range is fixed.
  int ostartT = start_index(it, isizeT, osizeT);
  int oendT = end_index(it, isizeT, osizeT);

  // gradInput offset by slice/feature and frame/time.
  scalar_t *gradInput_dt = gradInput + i_plane*isizeH*isizeW;
  // gradOutput offset by slice/feature and earliest relevant frame/time
  const scalar_t *gradOutput_dt = gradOutput + (d*osizeT + ostartT)*osizeH*osizeW;

  // For all input pixels...
  for (ih = istartH; ih < iendH; ih += istepH) {
    int ostartH = start_index(ih, isizeH, osizeH);
    int oendH = end_index(ih, isizeH, osizeH);

    for (iw = istartW; iw < iendW; iw += istepW) {
      int ostartW = start_index(iw, isizeW, osizeW);
      int oendW = end_index(iw, isizeW, osizeW);

      // Compute the gradients from corresponding output pixels
      scalar_t *ptr_gradInput = gradInput_dt + ih*isizeW + iw;
      const scalar_t *ptr_gradOutput = gradOutput_dt;

      // for all relevant output pixels
      int ot, oh, ow;
      for (ot = ostartT; ot < oendT; ++ot) {
        int kT = end_index(ot, osizeT, isizeT) - start_index(ot, osizeT, isizeT);
        for (oh = ostartH; oh < oendH; ++oh) {
          int kH = end_index(oh, osizeH, isizeH) - start_index(oh, osizeH, isizeH);
          for (ow = ostartW; ow < oendW; ++ow) {
            int kW = end_index(ow, osizeW, isizeW) - start_index(ow, osizeW, isizeW);
            const accscalar_t divide_factor = kW * kH * kT;
            accscalar_t grad_delta = static_cast<accscalar_t>(ptr_gradOutput[oh*osizeW + ow] / divide_factor);
            *ptr_gradInput += static_cast<scalar_t>(grad_delta);
          }
        }
        ptr_gradOutput += osizeH*osizeW; // next output frame
      }
    }
  }
}

template <typename scalar_t, typename accscalar_t>
void adaptiveaveragegradinput_loop(
    scalar_t *gradInput_data, const scalar_t *gradOutput_data,
    int64_t totalZ,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW) {
  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = std::max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);
    adaptiveaveragegradinput<scalar_t, accscalar_t>
      <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        gradInput_data, gradOutput_data,
        isizeT, isizeH, isizeW,
        osizeT, osizeH, osizeW,
        offsetZ);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    totalZ -= 65535;
    offsetZ += 65535;
  }
}

/*
 * Description:
 *    This function computes the gradInput from gradOutput.
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 *
 *    (uses atomic add)
 *
 */
template <typename scalar_t>
__global__ void atomicadaptiveaveragegradinput(
    scalar_t *gradInput, const scalar_t *gradOutput,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW,
    int64_t offsetZ)
{
  // iterators on output pixels
  int ot, oh, ow;

  // compute offsets based on thread/block ID
  int ostartH = blockIdx.y * blockDim.y + threadIdx.y;
  int oendH = osizeH;
  int ostepH = gridDim.y * blockDim.y;
  int ostartW = threadIdx.x;
  int oendW = osizeW;
  int ostepW = blockDim.x;

  // select output plane
  int64_t o_plane = blockIdx.x + offsetZ;
  ot = o_plane % osizeT; // output frame/time
  int d = o_plane / osizeT; // output slice/feature

  // input frame/time range is fixed.
  int istartT = start_index(ot, osizeT, isizeT);
  int iendT = end_index(ot, osizeT, isizeT);
  int kT = iendT - istartT;

  // gradInput offset by slice/feature and earliest relevant frame/time
  scalar_t *gradInput_nt = gradInput + (d*isizeT + istartT)*isizeH*isizeW;
  // gradOutput offset by slice/feature and frame/time
  const scalar_t *gradOutput_nt = gradOutput + o_plane*osizeH*osizeW;

  // For all output pixels...
  for (oh = ostartH; oh < oendH; oh += ostepH) {
    int istartH = start_index(oh, osizeH, isizeH);
    int iendH = end_index(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for (ow = ostartW; ow < oendW; ow += ostepW) {
      int istartW = start_index(ow, osizeW, isizeW);
      int iendW = end_index(ow, osizeW, isizeW);
      int kW = iendW - istartW;

      // Compute the gradients from corresponding input pixels
      scalar_t *ptr_gradInput = gradInput_nt + istartH*isizeW + istartW;
      const scalar_t *ptr_gradOutput = gradOutput_nt + oh*osizeW + ow;
      scalar_t grad_delta = *ptr_gradOutput / kT / kH / kW;

      int it, ih, iw;
      for (it = 0; it < kT; ++it) {
        for (ih = 0; ih < kH; ++ih) {
          for (iw = 0; iw < kW; ++iw) {
            gpuAtomicAddNoReturn(&(ptr_gradInput[ih*isizeW + iw]), grad_delta);
          }
        }
        ptr_gradInput += isizeH*isizeW; // next input frame
      }
    }
  }
}

template <typename scalar_t>
void atomicadaptiveaveragegradinput_loop(
    scalar_t* gradInput_data, const scalar_t* gradOutput_data,
    int64_t totalZ,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW) {
  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  int blocksH = std::max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);
    atomicadaptiveaveragegradinput<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        gradInput_data, gradOutput_data,
        isizeT, isizeH, isizeW,
        osizeT, osizeH, osizeW,
        offsetZ);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    totalZ -= 65535;
    offsetZ += 65535;
  }
}

// 5D tensor B x D x T x H x w

void adaptive_avg_pool3d_out_cuda_template(
    Tensor& output,
    const Tensor& input_,
    IntArrayRef& output_size) {
  TensorArg output_arg{output, "output", 1};
  TensorArg input_arg{input_, "input_", 2};

  checkAllSameGPU("adaptive_avg_pool3d_cuda", {output_arg, input_arg});

  for (int64_t i = 1; i < input_.ndimension(); i++) {
    TORCH_CHECK(
        input_.size(i) > 0,
        "adaptive_avg_pool3d_cuda(): Expected input to have non-zero size for non-batch dimensions, "
        "but input has sizes ", input_.sizes(),
        " with dimension ", i, " being empty");
  }

  TORCH_CHECK(
      (input_.ndimension() == 4 || input_.ndimension() == 5),
      "adaptive_avg_pool3d_cuda(): Expected 4D or 5D tensor, but got ", input_.sizes());

  // the jit sometimes passes output_size.size() == 1
  TORCH_CHECK(
      output_size.size() == 1 || output_size.size() == 3,
      "adaptive_avg_pool3d: internal error: output_size.size() must be 1 or 3");

  int64_t osizeT = output_size[0];
  int64_t osizeH = output_size[1];
  int64_t osizeW = output_size[2];

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t istrideB, istrideD, istrideT, istrideH, istrideW;
  int64_t totalZ;

  const Tensor& input = input_.ndimension() == 4 ? input_ : input_.contiguous();

  if (input.ndimension() == 4) {
    sizeD = input.size(0);
    isizeT = input.size(1);
    isizeH = input.size(2);
    isizeW = input.size(3);

    istrideB = 0;
    istrideD = input.stride(0);
    istrideT = input.stride(1);
    istrideH = input.stride(2);
    istrideW = input.stride(3);

    output.resize_({sizeD, osizeT, osizeH, osizeW});

    totalZ = sizeD * osizeT;
  } else {
    int64_t sizeB = input.size(0);
    sizeD = input.size(1);
    isizeT = input.size(2);
    isizeH = input.size(3);
    isizeW = input.size(4);

    istrideB = input.stride(0);
    istrideD = input.stride(1);
    istrideT = input.stride(2);
    istrideH = input.stride(3);
    istrideW = input.stride(4);

    output.resize_({sizeB, sizeD, osizeT, osizeH, osizeW});

    totalZ = sizeB * sizeD * osizeT;
  }

  if (output.numel() == 0) {
    return;
  }

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(), "adaptive_avg_pool3d_cuda", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;
        const scalar_t* input_data = input.const_data_ptr<scalar_t>();
        scalar_t* output_data = output.mutable_data_ptr<scalar_t>();

        adaptiveaveragepool_loop<scalar_t, accscalar_t>(
            input_data, output_data,
            totalZ,
            isizeT, isizeH, isizeW,
            osizeT, osizeH, osizeW,
            sizeD, istrideB, istrideD, istrideT, istrideH, istrideW);
      });
}

void adaptive_avg_pool3d_backward_out_cuda_template(
    Tensor& gradInput,
    const Tensor& gradOutput_,
    const Tensor& input) {
  TensorArg grad_input_arg{gradInput, "gradInput", 1};
  TensorArg grad_output_arg{gradOutput_, "gradOutput_", 2};
  TensorArg input_arg{input, "input", 3};

  adaptive_pool_empty_output_check(gradOutput_, "adaptive_avg_pool3d_backward");
  TORCH_CHECK(input.dim() == gradOutput_.dim(),
    __func__, ": Expected dimensions ", input.dim(), " for `gradOutput_` but got dimensions ", gradOutput_.dim());

  checkAllSameGPU(
      "adaptive_avg_pool3d_out_cuda",
      {grad_input_arg, grad_output_arg, input_arg});

  const Tensor gradOutput = gradOutput_.contiguous();

  gradInput.resize_as_(input);
  if (gradInput.numel() == 0) {
    return;
  }

  gradInput.zero_();

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t osizeT, osizeH, osizeW;
  int64_t totalZ;

  if (input.ndimension() == 4) {
    sizeD = input.size(0);
    isizeT = input.size(1);
    isizeH = input.size(2);
    isizeW = input.size(3);

    osizeT = gradOutput.size(1);
    osizeH = gradOutput.size(2);
    osizeW = gradOutput.size(3);
  } else {
    sizeD = input.size(1);
    isizeT = input.size(2);
    isizeH = input.size(3);
    isizeW = input.size(4);

    osizeT = gradOutput.size(2);
    osizeH = gradOutput.size(3);
    osizeW = gradOutput.size(4);
  }

  bool atomic = (isizeW%osizeW != 0) || (isizeH%osizeH != 0) || (isizeT%osizeT != 0);

  if (input.ndimension() == 4) {
    totalZ = atomic ? sizeD * osizeT : sizeD * isizeT;
  } else {
    int sizeB = input.size(0);
    totalZ = atomic ? sizeB * sizeD * osizeT : sizeB * sizeD * isizeT;
  }

  if (atomic) {
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
        input.scalar_type(), "adaptive_avg_pool3d_backward_cuda", [&] {
          scalar_t* gradInput_data = gradInput.mutable_data_ptr<scalar_t>();
          const scalar_t* gradOutput_data = gradOutput.const_data_ptr<scalar_t>();

          atomicadaptiveaveragegradinput_loop(
              gradInput_data, gradOutput_data,
              totalZ,
              isizeT, isizeH, isizeW,
              osizeT, osizeH, osizeW);
        });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
        input.scalar_type(), "adaptive_avg_pool3d_backward_cuda", [&] {
          using accscalar_t = at::acc_type<scalar_t, true>;

          scalar_t* gradInput_data = gradInput.mutable_data_ptr<scalar_t>();
          const scalar_t* gradOutput_data = gradOutput.const_data_ptr<scalar_t>();

          adaptiveaveragegradinput_loop<scalar_t, accscalar_t>(
              gradInput_data, gradOutput_data,
              totalZ,
              isizeT, isizeH, isizeW,
              osizeT, osizeH, osizeW);
        });
  }
}

} // namespace

Tensor& adaptive_avg_pool3d_out_cuda(const Tensor& input,
    IntArrayRef output_size,
    Tensor& output) {
  adaptive_avg_pool3d_out_cuda_template(output, input, output_size);
  return output;
}

Tensor adaptive_avg_pool3d_cuda(
    const Tensor& input,
    IntArrayRef output_size) {
  auto output = at::empty({0}, input.options());
  adaptive_avg_pool3d_out_cuda_template(output, input, output_size);
  return output;
}

Tensor& adaptive_avg_pool3d_backward_out_cuda(const Tensor& gradOutput_,
    const Tensor& input,
    Tensor& gradInput) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("adaptive_avg_pool3d_backward_out_cuda");
  adaptive_avg_pool3d_backward_out_cuda_template(gradInput, gradOutput_, input);
  return gradInput;
}

Tensor adaptive_avg_pool3d_backward_cuda(
    const Tensor& gradOutput_,
    const Tensor& input) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("adaptive_avg_pool3d_backward_cuda");
  auto gradInput = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  adaptive_avg_pool3d_backward_out_cuda_template(gradInput, gradOutput_, input);
  return gradInput;
}

} // namespace at::native
