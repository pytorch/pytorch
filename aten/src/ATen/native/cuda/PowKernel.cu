#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Pow.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/Pow.h>
#include <c10/core/Scalar.h>

namespace at::native {

// Forward declare some unary kernels
void rsqrt_kernel_cuda(TensorIteratorBase& iter);
void sqrt_kernel_cuda(TensorIteratorBase& iter);
void reciprocal_kernel_cuda(TensorIteratorBase& iter);

namespace {

void pow_tensor_scalar_kernel(TensorIteratorBase& iter, const Scalar& exp_scalar);

template <typename scalar_t>
void pow_scalar_tensor_impl(TensorIteratorBase& iter, scalar_t base) {
  gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t exp) -> scalar_t {
    return pow_(base, exp);
  });
}

template <typename value_t>
void pow_scalar_tensor_impl(TensorIteratorBase& iter, c10::complex<value_t> base) {
  // For complex, thrust::pow uses the identity
  // pow(a, b) = exp(log(a) * b)
  const auto fct = std::log(base);
  gpu_kernel(iter, [=]GPU_LAMBDA(c10::complex<value_t> exp) -> c10::complex<value_t> {
    return std::exp(fct * exp);
  });
}

/* complex<Half> support impl */
constexpr char pow_scalar_base_name[] = "pow_scalar_base_kernel";
template <>
void pow_scalar_tensor_impl(TensorIteratorBase& iter, c10::complex<at::Half> base) {
  using scalar_t = c10::complex<at::Half>;
  using opmath_t = at::opmath_type<scalar_t>;
  // For complex, thrust::pow uses the identity
  // pow(a, b) = exp(log(a) * b)
  const auto fct = std::log(opmath_t{base});
#if AT_USE_JITERATOR()
  static const auto pow_kernel_string =
      jiterator_stringify(template <typename T> T pow_scalar_base_kernel(T exp, T fct) {
        return std::exp(fct * exp);
      });
  jitted_gpu_kernel<pow_scalar_base_name, scalar_t, scalar_t, 1>(
      iter,
      pow_kernel_string,
      /*scalar_pos=*/at::cuda::jit::BinaryFuncVariant::NoScalar,
      /*scalar_val=*/0,
      /*extra_args=*/std::make_tuple(fct));
#else
  gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t exp) -> scalar_t {
    return std::exp(fct * opmath_t{exp});
  });
#endif
}

namespace {

#if AT_USE_JITERATOR()
/* complex<Half> support impl */
constexpr char pow_name[] = "pow_kernel";
static const auto pow_kernel_string =
    jiterator_stringify(template <typename T> T pow_kernel(T base, T exp) {
      return std::pow(base, exp);
    });
#endif

/* complex<Half> support impl */
void pow_chalf_tensor_scalar_impl(TensorIteratorBase& iter, const Scalar& exp_scalar) {
  using scalar_t = c10::complex<at::Half>;
  using opmath_t = at::opmath_type<scalar_t>;
  auto exp = exp_scalar.to<opmath_t>();
#if AT_USE_JITERATOR()
  jitted_gpu_kernel<pow_name, scalar_t, scalar_t, 1>(
      iter,
      pow_kernel_string,
      /*scalar_pos=*/at::cuda::jit::BinaryFuncVariant::NoScalar,
      /*scalar_val=*/0,
      /*extra_args=*/std::make_tuple(exp));
#else
  gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t base) -> scalar_t {
    return std::pow(opmath_t{base}, exp);
  });
#endif
}

}  // anonymous namespace

void pow_tensor_tensor_kernel(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (common_dtype == kComplexHalf) {
    using scalar_t = c10::complex<at::Half>;
    if (iter.is_cpu_scalar(1)) {
      const auto base = iter.scalar_value<scalar_t>(1);
      iter.remove_operand(1);
      pow_scalar_tensor_impl(iter, base);
    } else if (iter.is_cpu_scalar(2)) {
      const auto exp = iter.scalar_value<scalar_t>(2);
      iter.remove_operand(2);
      pow_chalf_tensor_scalar_impl(iter, exp);
    } else {
      using opmath_t = at::opmath_type<scalar_t>;
      TORCH_INTERNAL_ASSERT(!iter.is_cpu_scalar(1) && !iter.is_cpu_scalar(2));
#if AT_USE_JITERATOR()
      jitted_gpu_kernel<pow_name, scalar_t, scalar_t, 2>(
          iter, pow_kernel_string);
#else
      gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return pow_(opmath_t{base}, opmath_t{exp});
          });
#endif
    }
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
        kHalf, kBFloat16, iter.common_dtype(), "pow_cuda", [&] {
      if (iter.is_cpu_scalar(1)) {
        const auto base = iter.scalar_value<scalar_t>(1);
        iter.remove_operand(1);
        pow_scalar_tensor_impl(iter, base);
      } else if (iter.is_cpu_scalar(2)) {
        const auto exp = iter.scalar_value<scalar_t>(2);
        iter.remove_operand(2);
        pow_tensor_scalar_kernel(iter, exp);
      } else {
        gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
          return pow_(base, exp);
        });
      }
    });
  }
}


template<typename Base_type, typename Exp_type>
void pow_tensor_scalar_kernel_impl(TensorIteratorBase& iter,
                                                 Exp_type exp) {
  const auto d_exp = static_cast<double>(exp);
  // .5 (sqrt), -.5 (rsqrt) and -1 (reciprocal) specializations are handled
  // in pow_tensor_scalar_kernel
  if (d_exp == 2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base;
    });
  } else if (d_exp == 3) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base * base;
    });
  } else if (d_exp == -2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return 1.0 / (base * base);
    });
  } else {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return pow_(base, exp);
    });
  }
}

void pow_tensor_scalar_kernel(TensorIteratorBase& iter, const Scalar& exp_scalar) {
  // Dispatch to fast specialization for sqrt, rsqrt and reciprocal
  if (!exp_scalar.isComplex()) {
    if (exp_scalar.equal(.5)) {
      return sqrt_kernel_cuda(iter);
    } else if (exp_scalar.equal(-0.5)) {
      return rsqrt_kernel_cuda(iter);
    } else if (exp_scalar.equal(-1.0)) {
      return reciprocal_kernel_cuda(iter);
    }
  }
  if (isComplexType(iter.common_dtype()) || exp_scalar.isComplex()) {
    if (iter.common_dtype() == kComplexHalf) {
      using scalar_t = c10::complex<at::Half>;
      pow_chalf_tensor_scalar_impl(iter, exp_scalar);
      return;
    }
    AT_DISPATCH_COMPLEX_TYPES(iter.common_dtype(), "pow_cuda", [&]() {
      if (exp_scalar.equal(2.0)) {
        gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base) -> scalar_t {
          return base * base;
        });
        return;
      }
      const auto exp = exp_scalar.to<scalar_t>();
      gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base) -> scalar_t {
        return pow_(base, exp);
      });
    });
  } else if (isFloatingType(iter.common_dtype()) || exp_scalar.isIntegral(false)) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  } else {
    TORCH_INTERNAL_ASSERT(false, "invalid combination of type in Pow function, common dtype:", iter.common_dtype(),
                                 "exp is integral?", exp_scalar.isIntegral(false));
  }
}

} // anonymous namespace

REGISTER_DISPATCH(pow_tensor_tensor_stub, &pow_tensor_tensor_kernel)
REGISTER_DISPATCH(pow_tensor_scalar_stub, &pow_tensor_scalar_kernel)

} // namespace at::native
