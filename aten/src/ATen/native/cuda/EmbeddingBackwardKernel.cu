#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/native/cuda/EmbeddingBackwardKernel.cuh>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/cub.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/native/cuda/SortingCommon.cuh>

#include <c10/macros/Macros.h>

#if CUB_SUPPORTS_UNIQUE_BY_KEY()
#include <thrust/iterator/counting_iterator.h>
#endif

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/zeros.h>
#endif

namespace at::native {

namespace {

/* This code computes the sum of the weights in two-steps:
  1) Each GPU warp sums `NROWS_PER_THREAD` number of row given by `indeces`
  2) Each partial-sum from 1) are summed and scatter into `grad_weight`

  Notice, `NROWS_PER_THREAD` impacts the Achieved Occupancy of the
  kernel execution. If it is high, the size of the thread blocks will be
  too small to achieve good occupancy. Similarly, a very low value will
  make the size of the thread blocks in the final sum in step 2) too small.
*/
constexpr int NROWS_PER_THREAD = 10;

// Fast ceil division (no overflow checking)
__host__ __device__ __forceinline__
int64_t ceil_div(int64_t x, int64_t y) {
  return (x + y - 1) / y;
}

template <typename index_t>
__global__
void krn_partials_per_segment(index_t *ret, const index_t *segment_offsets,
                              const int64_t *num_of_segments_ptr, int64_t numel) {
  int64_t num_of_segments = *num_of_segments_ptr;
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id < num_of_segments) {
    const int64_t idx_start = segment_offsets[id];
    const int64_t idx_end = (id == num_of_segments-1)?numel:segment_offsets[id+1];
    const int64_t size = idx_end - idx_start;
    ret[id] = ceil_div(size, NROWS_PER_THREAD);
  }
}

template <typename index_t>
__global__
void krn_partial_segment_offset(
        index_t *ret,
        const index_t *partials_per_segment,
        const index_t *partials_per_segment_offset,
        const index_t *segment_offsets,
        const int64_t *num_of_segments_ptr) {
  int64_t num_of_segments = *num_of_segments_ptr;
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id < num_of_segments) {
    index_t idx = partials_per_segment_offset[id];
    const index_t num_partials = partials_per_segment[id];
    const index_t segment_offset = segment_offsets[id];
    for (int64_t i=0; i<num_partials; ++i) {
      ret[idx++] = segment_offset + i * NROWS_PER_THREAD;
    }
  }
}


template <typename scalar_t, typename index_t>
__global__ void compute_grad_weight_bags(
    const index_t *indices, const scalar_t *gradOutput,
    const index_t *offset2bag, const index_t *count, ptrdiff_t numel,
    int64_t stride, int mode_mean, const index_t *bag_size,
    const scalar_t* per_sample_weights, int64_t per_sample_weights_stride,
    const index_t* segment_offsets, const int64_t *num_of_segments_ptr,
    acc_type<scalar_t, true> *grad_weight_per_segment,
    const int64_t stride_warped) {

  int64_t num_of_segments = *num_of_segments_ptr;
  const int64_t gid = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int64_t id = gid / stride_warped;
  const int64_t startFeature = gid % stride_warped;
  if (startFeature >= stride) {
    return;
  }
  if (id >= num_of_segments) {
    return;
  }
  const int idx_begin = segment_offsets[id];
  const int idx_end = (id == num_of_segments-1)?numel:segment_offsets[id+1];

  acc_type<scalar_t, true> weight = 0;
  for (int idx=idx_begin; idx < idx_end; ++idx) {
    const int origRow = indices[idx];
    const int seq_number = offset2bag[origRow];
    const int gradOutputRow = seq_number * stride;

    acc_type<scalar_t, true> scale = count ? 1.0 / count[idx] : 1.0;
    if (per_sample_weights) {
      scale *= per_sample_weights[origRow * per_sample_weights_stride];
    }

    acc_type<scalar_t, true> gradient = gradOutput[gradOutputRow + startFeature];
    if (mode_mean) {
      gradient /= bag_size[seq_number];
    }
    weight += gradient * scale;
  }
  grad_weight_per_segment[id * stride + startFeature] = weight;
}

template <typename scalar_t, typename index_t>
__global__ void compute_grad_weight(
    const index_t *indices,
    const scalar_t *gradOutput,
    const index_t *count,
    ptrdiff_t numel,
    int64_t stride,
    const index_t* segment_offsets,
    const int64_t *num_of_segments_ptr,
    acc_type<scalar_t, true> *grad_weight_per_segment,
    const int64_t stride_warped) {

  int64_t num_of_segments = *num_of_segments_ptr;
  using accscalar_t = acc_type<scalar_t, true>;
  const int64_t gid = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int64_t id = gid / stride_warped;
  const int64_t startFeature = gid % stride_warped;
  if (startFeature >= stride) {
    return;
  }
  if (id >= num_of_segments) {
    return;
  }
  const int idx_begin = segment_offsets[id];
  const int idx_end = (id == num_of_segments-1)?numel:segment_offsets[id+1];

  accscalar_t weight = 0;
  for (int idx=idx_begin; idx < idx_end; ++idx) {
    const index_t target_row = indices[idx];
    const accscalar_t scale = count ? (accscalar_t)1.0 / count[idx] : 1.0;
    weight += gradOutput[target_row * stride + startFeature] * scale;
  }
  grad_weight_per_segment[id * stride + startFeature] = weight;
}

// This kernel assumes that all input tensors are contiguous.
template <typename scalar_t, typename index_t>
__global__ void sum_and_scatter(
    const index_t *input, scalar_t *gradWeight, int64_t stride,
    const index_t* segment_offsets, const int64_t *num_of_segments_ptr,
    const acc_type<scalar_t, true> *grad_weight_per_segment,
    const index_t *segment_sizes_offsets, const int64_t *num_of_partial_segments_ptr,
    const int64_t padding_idx,
    const int64_t stride_warped) {

  int64_t num_of_segments = *num_of_segments_ptr;
  int64_t num_of_partial_segments = *num_of_partial_segments_ptr;
  const int64_t gid = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int64_t id = gid / stride_warped;
  const int64_t startFeature = gid % stride_warped;
  if (startFeature >= stride) {
    return;
  }
  if (id >= num_of_segments) {
    return;
  }

  const int idx_begin = segment_sizes_offsets[id];
  const int idx_end = (id == num_of_segments-1)?num_of_partial_segments:segment_sizes_offsets[id+1];
  acc_type<scalar_t, true> weight = 0;
  for (int idx=idx_begin; idx < idx_end; ++idx) {
    weight += grad_weight_per_segment[idx*stride + startFeature];
  }
  int64_t target_row = input[segment_offsets[id]];
  if (target_row != padding_idx) {
    gradWeight[target_row * stride + startFeature] = weight;
  }
}

template<typename index_t>
__global__ void compute_num_of_partial_segments(const index_t *partials_per_segment, const index_t *partials_per_segment_offset, const int64_t *num_of_segments_ptr, int64_t *output) {
  int64_t num_of_segments = *num_of_segments_ptr;
  *output = partials_per_segment[num_of_segments-1] +
            partials_per_segment_offset[num_of_segments-1];
}

#if !CUB_SUPPORTS_UNIQUE_BY_KEY()
__global__ void write_num_of_segments_for_legacy_thrust_path(int64_t *num_of_segments_ptr, int64_t num_of_segments) {
  *num_of_segments_ptr = num_of_segments;
}
#endif

} // anon namespace

#if !CUB_SUPPORTS_UNIQUE_BY_KEY()
template<typename index_t>
int64_t embedding_backward_cuda_kernel_unique_by_key(const Tensor &sorted_indices, Tensor &segment_offsets);
#endif

Tensor embedding_backward_cuda_kernel(
        const Tensor &grad,
        const Tensor &orig_indices,
        const Tensor &sorted_indices,
        const Tensor &count,
        int64_t num_weights,
        int padding_idx,
        bool mode_mean,
        const Tensor &offset2bag,
        const Tensor &bag_size,
        const Tensor &per_sample_weights) {

  auto stream = at::cuda::getCurrentCUDAStream();
  const ptrdiff_t numel = sorted_indices.numel();

  auto grad_weight = at::zeros({num_weights, grad.size(-1)}, grad.options());
  const int64_t stride = grad_weight.stride(0);

  // Compute the number of segments and their start position so that we do not have to
  // spawn a warp per index. In this context, a segment is a number of rows that should
  // be summarized.
  // Unit: index in `sorted_indices` and `orig_indices`
  auto segment_offsets = at::empty({numel}, orig_indices.options());
  auto num_of_segments_tensor = at::empty({}, grad.options().dtype(kLong));
  int64_t *num_of_segments_ptr = num_of_segments_tensor.mutable_data_ptr<int64_t>();
#if !CUB_SUPPORTS_UNIQUE_BY_KEY()
  AT_DISPATCH_INDEX_TYPES(orig_indices.scalar_type(), "embedding_backward_cuda_kernel", [&] () {
    int64_t num_of_segments = embedding_backward_cuda_kernel_unique_by_key<index_t>(sorted_indices, segment_offsets);
    write_num_of_segments_for_legacy_thrust_path<<<1, 1, 0, c10::cuda::getCurrentCUDAStream()>>>(num_of_segments_ptr, num_of_segments);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
#else
  AT_DISPATCH_INDEX_TYPES(orig_indices.scalar_type(), "embedding_backward_cuda_kernel", [&] () {
    cuda::hipcub::unique_by_key(
      sorted_indices.const_data_ptr<index_t>(), thrust::make_counting_iterator(0),
      segment_offsets.mutable_data_ptr<index_t>(),
      num_of_segments_ptr, sorted_indices.numel());
  });
#endif

  int64_t max_segments = std::min<int64_t>(numel, num_weights);

  AT_DISPATCH_INDEX_TYPES(orig_indices.scalar_type(), "embedding_backward_cuda_kernel", [&] () {
    // We split the segments up into sizes of `NROWS_PER_THREAD`
    // Compute the number partial-segments per segment (some partial-segments
    // may not be the full `NROWS_PER_THREAD` number of rows)
    auto partials_per_segment = at::empty({max_segments}, orig_indices.options());
    {
      krn_partials_per_segment<<<ceil_div(max_segments, 32), 32, 0, stream>>> (
              partials_per_segment.mutable_data_ptr<index_t>(),
              segment_offsets.const_data_ptr<index_t>(),
              num_of_segments_ptr,
              numel);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }

    // In order to compute `partial_segment_offset`, which is the start index
    // of each partial-segment in `sorted_indices`, we need to compute the
    // start position of each _segment_ in `partial_segment_offset`.
    // Unit: index in `partial_segment_offset`
    auto partials_per_segment_offset = at::empty({max_segments}, orig_indices.options());
    cuda::hipcub::exclusive_sum(
        partials_per_segment.const_data_ptr<index_t>(),
        partials_per_segment_offset.mutable_data_ptr<index_t>(),
        max_segments);

    // The total number of partial-segments is the sum of `partials_per_segment_offset`
    auto num_of_partial_segments_tensor = at::empty({}, grad.options().dtype(kLong));
    int64_t *num_of_partial_segments_ptr = num_of_partial_segments_tensor.mutable_data_ptr<int64_t>();
    compute_num_of_partial_segments<index_t><<<1, 1, 0, c10::cuda::getCurrentCUDAStream()>>>(
      partials_per_segment.const_data_ptr<index_t>(),
      partials_per_segment_offset.const_data_ptr<index_t>(),
      num_of_segments_ptr, num_of_partial_segments_ptr);
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    auto max_partial_segment = numel / NROWS_PER_THREAD + max_segments;

    // Now we can compute the start position of each partial-segment
    // Unit: index in `sorted_indices` and `orig_indices`
    auto partial_segment_offset = at::empty({max_partial_segment}, orig_indices.options());
    {
      krn_partial_segment_offset<<<ceil_div(max_segments, 32), 32, 0, stream>>> (
              partial_segment_offset.mutable_data_ptr<index_t>(),
              partials_per_segment.const_data_ptr<index_t>(),
              partials_per_segment_offset.const_data_ptr<index_t>(),
              segment_offsets.const_data_ptr<index_t>(),
              num_of_segments_ptr);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }

    const int warp_size = at::cuda::warp_size();
    const int stride_warped = ceil_div(stride, warp_size)*warp_size;
    const int block = std::min(stride_warped, MAX_BLOCK_SIZE);
    const int grid = ceil_div(max_partial_segment*stride_warped, block);

    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
      grad.scalar_type(), "embedding_bag_backward_cuda_compute_grad_weight", [&] {
        // For numerical stability, the dtype of `grad_weight_per_segment`
        // should match `acc_type`
        using partial_weight_t = acc_type<scalar_t, true>;
        TensorOptions op;
        if(grad.dtype() == at::kHalf || grad.dtype() == at::kBFloat16) {
            op = grad.options().dtype(at::kFloat);
        } else {
            op = grad.options();
        }
        auto grad_weight_per_segment = at::empty({max_partial_segment, stride}, op);
        // Compute the sum of each partial-segment and handle bags
        if (offset2bag.defined()) {
              compute_grad_weight_bags<scalar_t><<<grid, block, 0, stream>>>(
                orig_indices.const_data_ptr<index_t>(),
                grad.const_data_ptr<scalar_t>(),
                offset2bag.const_data_ptr<index_t>(),
                count.defined() ? count.const_data_ptr<index_t>() : nullptr, numel, stride,
                mode_mean, bag_size.const_data_ptr<index_t>(),
                per_sample_weights.defined() ? per_sample_weights.const_data_ptr<scalar_t>() : NULL,
                per_sample_weights.defined() ? per_sample_weights.stride(0) : 0,
                partial_segment_offset.const_data_ptr<index_t>(),
                num_of_partial_segments_ptr, grad_weight_per_segment.mutable_data_ptr<partial_weight_t>(),
                stride_warped);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
        } else {
              compute_grad_weight<scalar_t><<<grid, block, 0, stream>>>(
                orig_indices.const_data_ptr<index_t>(),
                grad.const_data_ptr<scalar_t>(),
                count.defined() ? count.const_data_ptr<index_t>() : nullptr,
                numel, stride,
                partial_segment_offset.const_data_ptr<index_t>(),
                num_of_partial_segments_ptr,
                grad_weight_per_segment.mutable_data_ptr<partial_weight_t>(),
                stride_warped);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
        }

        // Finally, we sum all the partial-sums and scatter them
        // into `grad_weight`.
        const int grid2 = ceil_div(max_segments*stride_warped, block);
            sum_and_scatter<scalar_t><<<grid2, block, 0, stream>>>(
              sorted_indices.const_data_ptr<index_t>(),
              grad_weight.mutable_data_ptr<scalar_t>(),
              stride,
              segment_offsets.const_data_ptr<index_t>(),
              num_of_segments_ptr, grad_weight_per_segment.const_data_ptr<partial_weight_t>(),
              partials_per_segment_offset.const_data_ptr<index_t>(),
              num_of_partial_segments_ptr,
              padding_idx,
              stride_warped);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  });
  return grad_weight;
}

} // namespace at::native
