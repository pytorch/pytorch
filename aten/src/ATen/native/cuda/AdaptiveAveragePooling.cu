#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/TensorUtils.h>
#include <ATen/OpMathType.h>
#include <ATen/Utils.h>
#include <c10/util/Exception.h>
#include <ATen/native/cuda/LaunchUtils.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_adaptive_avg_pool2d_backward_native.h>
#include <ATen/ops/_adaptive_avg_pool2d_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/zeros_like.h>
#endif

#include <ATen/native/AdaptivePooling.h>

#include <algorithm>
#include <cfloat>
#include <cmath>

#define START_IND(a,b,c) ((int64_t)((a / b) * c + ((a % b) * c) / b))
#define END_IND(a,b,c) (1 + ((int64_t)(a + 1) * c - 1) / b)

#define START_IND_INT(a,b,c) ((a * c) / b)
#define END_IND_INT(a,b,c) (((a + 1) * c + b - 1) / b)
// #define START_IND(a,b,c) a * c / b
// #define END_IND(a,b,c)  (a + 1) * c / b + ((a + 1) * c % b > 0)?1:0

#define CUDA_MAX_THREADS 1024 // this is safe, in reality 256 is our limit
#define BLOCK_STRIDE 2 // increasing block_stride to lower # of blocks launched

namespace at::native {

namespace {

  // 4d tensor B x D x H x W
  // All kernels view batch dim B and feature dim D as collapsed.

  /*
   * Description:
   *    this function adaptively average pools an input 4D tensor along dimensions 2 and 3
   *    4D input, 4D output
   */
   template <typename scalar_t>
  __global__ void adaptive_average_pool(const scalar_t *input, scalar_t *output,
                          int isizeH, int isizeW,
                          int osizeH, int osizeW,
                          int64_t istrideD, int64_t istrideH, int64_t istrideW)
  {
    using opmath_t = at::opmath_type<scalar_t>;
    // iterators on output pixels
    int oh, ow;

    // select input/output plane based on thread/block ID
    int o_plane = blockIdx.x;
    int i_plane = o_plane;

    output = output + o_plane*osizeH*osizeW;
    input = input + i_plane*istrideD;

    int ostartH = blockDim.y*blockIdx.y + threadIdx.y;
    int oendH = osizeH;
    const int ostepH = blockDim.y*gridDim.y;

    int ostartW = threadIdx.x;
    int oendW = osizeW;
    const int ostepW = blockDim.x;

    // For all output pixels...
    for(oh = ostartH; oh < oendH; oh += ostepH) {

      int istartH = START_IND(oh, osizeH, isizeH);
      int iendH   = END_IND(oh, osizeH, isizeH);
      int kH = iendH - istartH;

      for(ow = ostartW; ow < oendW; ow += ostepW) {

        int istartW = START_IND(ow, osizeW, isizeW);
        int iendW   = END_IND(ow, osizeW, isizeW);
        int kW = iendW - istartW;

        // Compute the average pooling over corresponding input pixels
        const scalar_t *ptr_input = input + istartH*istrideH + istartW*istrideW;
        scalar_t *ptr_output = output + oh*osizeW + ow;
        opmath_t sum = static_cast<opmath_t>(0);
        int ih, iw;
        for(ih = 0; ih < kH; ++ih) {
          for(iw = 0; iw < kW; ++iw) {
            scalar_t val = ptr_input[iw*istrideW];
            sum += val;
          }
          ptr_input += istrideH; // next input line
        }
        // Update output
        *ptr_output = sum / kH / kW;
      }
    }
  }

  /*
   * Description:
   *    this function computes the gradInput from gradOutput
   */
   template <typename T>
  __global__ void adaptive_average_gradinput(
    T *gradInput, const T *gradOutput,
    int isizeH, int isizeW, int osizeH, int osizeW
  )
  {
    // iterators on input pixels
    int ih, iw;

    // select input/output plane based on thread/block ID
    int i_plane = blockIdx.x;
    int o_plane = i_plane;

    gradOutput = gradOutput + o_plane*osizeH*osizeW;
    gradInput = gradInput + i_plane*isizeH*isizeW;

    int istartH = blockDim.y*blockIdx.y + threadIdx.y;
    int iendH = isizeH;
    int istepH = blockDim.y*gridDim.y;

    int istartW = threadIdx.x;
    int iendW = isizeW;
    int istepW = blockDim.x;

    // compute gradInput
    for(ih = istartH; ih < iendH; ih += istepH) {

      int ostartH = START_IND(ih, isizeH, osizeH);
      int oendH   = END_IND(ih, isizeH, osizeH);

      for(iw = istartW; iw < iendW; iw += istepW) {

        int ostartW = START_IND(iw, isizeW, osizeW);
        int oendW   = END_IND(iw, isizeW, osizeW);

        // Compute the gradients over corresponding output pixels
        T *ptr_gradInput = gradInput + ih*isizeW + iw;

        int oh, ow;
        for(oh = ostartH; oh < oendH; ++oh) {
          int kH = START_IND(oh, osizeH, isizeH) - END_IND(oh, osizeH, isizeH);
          for(ow = ostartW; ow < oendW; ++ow) {
            int kW = START_IND(ow, osizeW, isizeW) - END_IND(ow, osizeW, isizeW);
            T grad_delta = gradOutput[ow + oh*osizeW] / kH / kW;
            *ptr_gradInput += grad_delta;
          }
        }
      }
    }
  }

  /*
   * Description:
   *    this function computes the gradInput from gradOutput
   *    (uses atomic add)
   */
   template <typename T>
  __global__ void atomic_adaptive_average_gradinput(
    T *gradInput, const T *gradOutput,
    int isizeH, int isizeW, int osizeH, int osizeW
  )
  {
    // iterators on output indices
    int oh, ow;

    // select input/output plane based on thread/block ID
    int o_plane = blockIdx.x;
    int i_plane = o_plane;

    gradOutput = gradOutput + o_plane*osizeW*osizeH;
    gradInput = gradInput + i_plane*isizeW*isizeH;

    int ostartH = blockDim.y*blockIdx.y + threadIdx.y;
    int oendH = osizeH;
    int ostepH = blockDim.y*gridDim.y;

    int ostartW = threadIdx.x;
    int oendW = osizeW;
    int ostepW = blockDim.x;

    // For all output pixels...
    for(oh = ostartH; oh < oendH; oh += ostepH) {

      int istartH = START_IND(oh, osizeH, isizeH);
      int iendH   = END_IND(oh, osizeH, isizeH);
      int kH = iendH - istartH;

      for(ow = ostartW; ow < oendW; ow += ostepW) {

        int istartW = START_IND(ow, osizeW, isizeW);
        int iendW   = END_IND(ow, osizeW, isizeW);
        int kW = iendW - istartW;

        // Compute the gradients for over corresponding input pixels
        T *ptr_gradInput = gradInput + istartH*isizeW + istartW;
        const T *ptr_gradOutput = gradOutput + oh*osizeW + ow;
        T grad_delta = *ptr_gradOutput / kW / kH;

        int ih, iw;
        for(ih = 0; ih < kH; ++ih) {
          for(iw = 0; iw < kW; ++iw) {
            // atomic add since different threads could update same variable
            gpuAtomicAddNoReturn(&(ptr_gradInput[iw]), grad_delta);
          }
          ptr_gradInput += isizeW; // next input line
        }
      }
    }
  }

  /*
   * Description:
   *    this function adaptively average pools an input 4D tensor along dimensions 2 and 3
   *    NHWC layout for both input and output tensor
   *    4D input, 4D output
   */
   template <typename index_t, typename scalar_t>
  C10_LAUNCH_BOUNDS_1(CUDA_MAX_THREADS)
  __global__ void adaptive_average_pool_nhwc(const scalar_t* __restrict__ input, scalar_t* __restrict__ output,
                          int sizeB, int sizeC,
                          int isizeH, int isizeW,
                          int osizeH, int osizeW,
                          int kernel_stride_C, int kernel_size_C,
                          index_t istrideB, index_t istrideC,
                          index_t istrideH, index_t istrideW)
  {
    using opmath_t = at::opmath_type<scalar_t>;
    extern __shared__ int smem[];
    opmath_t *out_cached = reinterpret_cast<opmath_t*>(smem);

    // flattening cta for pre-computation & smem initialization;
    int thread_id = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);
    int block_size = blockDim.x * blockDim.y * blockDim.z;

    // use shared memory to store temporary output value. This is simply to
    // reduce register usage.
    for (index_t i = thread_id; i < kernel_size_C*blockDim.x*blockDim.y*blockDim.z; i+= block_size) {
      out_cached[i] = opmath_t(0.0);
    }

    __syncthreads();

    // each CTA handles a portion of a single slice on batch dimension;
    int batch_id = blockIdx.x % sizeB;
    int channel_id = blockIdx.x / sizeB;
    int channel_offset = threadIdx.x + channel_id * blockDim.x;

    // each CTA handles a single slice on batch dimension;
    // We use gridDim.x to handle striding on C as well.
    output = output + batch_id * osizeH * osizeW * sizeC;
    input = input + batch_id * istrideB;

    // split out_cached and exclusively it assigned to each thread;
    out_cached = &out_cached[(threadIdx.z * blockDim.y + threadIdx.y) * kernel_size_C * blockDim.x];

    // iterate on output H & W.
    // Each CTA handles a consecutive H & W section (TILE); Do NOT stride CTA on
    // tile so there's a better chance to hit L1 cache.
    index_t oH = (osizeH + gridDim.z-1) / gridDim.z;
    index_t oW = (osizeW + gridDim.y-1) / gridDim.y;
    index_t ostartH = threadIdx.z + blockIdx.z*oH;
    index_t oendH = ::min(ostartH+oH, osizeH);
    index_t ostartW = threadIdx.y + blockIdx.y*oW;
    index_t oendW = ::min(ostartW+oW, osizeW);

    // Stride for threads, each warp can reuse L1 as they go. So theoretically
    // better chance to survive cache eviction.
    for (int oh = ostartH; oh < oendH; oh+=blockDim.z) {
      int istartH = START_IND_INT(oh, osizeH, isizeH);
      int iendH = END_IND_INT(oh, osizeH, isizeH);
      for (int ow = ostartW; ow < oendW; ow+=blockDim.y) {
        int istartW = START_IND_INT(ow, osizeW, isizeW);
        int iendW = END_IND_INT(ow, osizeW, isizeW);
        scalar_t factor = scalar_t(1.0) / ((iendH-istartH) * (iendW-istartW));

        // loop on input: hierarchy h->w->c, use shared memory here hopefully
        // would not stall global memory read;
        for (index_t ih = istartH; ih < iendH; ih++) {
          for (index_t iw = istartW; iw < iendW; iw++) {
            int cached_index = threadIdx.x;
            const scalar_t *ptr_input = input + ih*istrideH + iw*istrideW;
            for (index_t c = channel_offset;
                 c < sizeC;
                 c += blockDim.x*kernel_stride_C) {
              out_cached[cached_index] += ptr_input[c*istrideC];
              cached_index += blockDim.x;
            }
          }
        }
        scalar_t *ptr_output = output + (oh * osizeW + ow) * sizeC;

        int cached_index = threadIdx.x;
        // write accumulated output to global memory;
        for (index_t c = channel_offset;
             c < sizeC;
             c += blockDim.x*kernel_stride_C) {
          // This causes numerical issueptr when unit test with NCHW kernel;
          // switch to could verify the correctness;
          // output[c] = out_cached[c] / (iendH-istartH) / (iendW-istartW);
          ptr_output[c] = out_cached[cached_index] * factor;
          out_cached[cached_index] = opmath_t(0.0);
          cached_index += blockDim.x;
        }
        // no need to __syncthreads() since out_cached is not shared.
      }
    }
  }

  /*
   * Description:
   *    this function computes the gradInput from gradOutput
   *    NHWC layout for both input and output tensor
   *    4D input, 4D output
   */
   template <typename index_t, typename scalar_t>
  C10_LAUNCH_BOUNDS_1(CUDA_MAX_THREADS)
  __global__ void adaptive_average_gradinput_nhwc(scalar_t* __restrict__ gradInput, const scalar_t* __restrict__ gradOutput,
                          int sizeB, int sizeC,
                          int isizeH, int isizeW,
                          int osizeH, int osizeW,
                          int kernel_stride_C, int kernel_size_C,
                          index_t ostrideB, index_t ostrideC,
                          index_t ostrideH, index_t ostrideW)
  {
    extern __shared__ int smem[];
    index_t *ostartW_cached = smem;
    index_t *oendW_cached = &ostartW_cached[isizeW];

    // be careful with alignment, in case scalar_t is fp16, we want to assign
    // int pointers first.
    scalar_t *r_kW_cached = reinterpret_cast<scalar_t*>(&oendW_cached[isizeW]);
    scalar_t *r_kH_cached = &r_kW_cached[osizeW];
    scalar_t *out_cached = &r_kH_cached[osizeH];

    // flattening cta for pre-computation & smem initialization;
    int thread_id = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);
    int block_size = blockDim.x * blockDim.y * blockDim.z;

    // Precompute output start/end index per input index on width dimension;
    // Not doing this for height dimension, as that's our out-most loop.
    for (index_t i = thread_id; i < isizeW; i+= block_size) {
      ostartW_cached[i] = START_IND_INT(i, isizeW, osizeW);
      oendW_cached[i] = END_IND_INT(i, isizeW, osizeW);
    }

    // Precompute pooling height/weight factor for each output element;
    // This is used to weight output gradient when accumulate them on input
    // gradient.
    // Technically we don't have to compute it for the whole `osizeH`, since
    // each cta only covers a consecutive portion of the entire output. But it's
    // not going to save us from code divergence, and shared memory save is not
    // an issue neither, so just leave it as is for now.
    for (index_t i = thread_id; i < osizeH; i+= block_size) {
      r_kH_cached[i] = scalar_t(1.0) / (END_IND_INT(i, osizeH, isizeH) - START_IND_INT(i, osizeH, isizeH));
    }
    for (index_t i = thread_id; i < osizeW; i+= block_size) {
      r_kW_cached[i] = scalar_t(1.0) / (END_IND_INT(i, osizeW, isizeW) - START_IND_INT(i, osizeW, isizeW));
    }

    // each CTA handles a portion of a single slice on batch dimension;
    int batch_id = blockIdx.x % sizeB;
    int channel_id = blockIdx.x / sizeB;
    int channel_offset = threadIdx.x + channel_id * blockDim.x;

    // use shared memory to store temporary output value. This is simply to
    // reduce register usage.
    for (index_t i = thread_id; i < kernel_size_C*blockDim.x*blockDim.y*blockDim.z; i+= block_size) {
      out_cached[i] = scalar_t(0.0);
    }

    __syncthreads();

    // each CTA handles a portion of a single slice on batch dimension;
    // We use gridDim.x to handle striding on C as well.
    gradInput = gradInput + batch_id * isizeH * isizeW * sizeC;
    gradOutput = gradOutput + batch_id * ostrideB;

    // split out_cached and exclusively it assigned to each thread;
    out_cached = &out_cached[(threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x * kernel_size_C];

    // iterate on input H & W.
    // Each CTA handles a consecutive H & W section (TILE); Do NOT stride CTA on
    // tile so there's a better chance to hit L1 cache.
    index_t iH = (isizeH + gridDim.z-1) / gridDim.z;
    index_t iW = (isizeW + gridDim.y-1) / gridDim.y;
    index_t istartH = threadIdx.z + blockIdx.z*iH;
    index_t iendH = ::min(istartH+iH, isizeH);
    index_t istartW = threadIdx.y + blockIdx.y*iW;
    index_t iendW = ::min(istartW+iW, isizeW);

    // Stride for threads, each warp can reuse L1 as they go. So theoretically
    // better chance to survive cache eviction.
    for (index_t ih = istartH; ih < iendH; ih+=blockDim.z) {
      index_t ostartH = START_IND_INT(ih, isizeH, osizeH);
      index_t oendH = END_IND_INT(ih, isizeH, osizeH);
      for (index_t iw = istartW; iw < iendW; iw+=blockDim.y) {
        // loop on output: hierarchy h->w->c, so we could reuse weight factor f
        // because it remains the same for given oh & ow
        for(index_t oh = ostartH; oh < oendH; ++oh) {
          for(index_t ow = ostartW_cached[iw]; ow < oendW_cached[iw]; ++ow) {
            scalar_t f = r_kW_cached[ow] * r_kH_cached[oh];
            const scalar_t* ptr_gradOutput = gradOutput + oh*ostrideH + ow*ostrideW;
            int cached_index = threadIdx.x;
            for (index_t c = channel_offset;
                 c < sizeC;
                 c += blockDim.x*kernel_stride_C) {
              out_cached[cached_index] += ptr_gradOutput[c*ostrideC] * f;
              cached_index += blockDim.x;
            }
          }
        }
        scalar_t *ptr_gradInput = gradInput + (ih * isizeW + iw) * sizeC;
        int cached_index = threadIdx.x;
        // write accumulated gradIput to global memory;
        for (index_t c = channel_offset;
             c < sizeC;
             c += blockDim.x*kernel_stride_C) {
          ptr_gradInput[c] = out_cached[cached_index];
          out_cached[cached_index] = scalar_t(0.0);
          cached_index += blockDim.x;
        }
        // no need to __syncthreads() since out_cached is not shared.
      }
    }
  }

  // 4d tensor B x D x H x W

  void adaptive_avg_pool2d_out_cuda_template(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size)
  {
    TensorArg input_arg{ input, "input", 1 },
              output_arg{ output, "output", 2 };
    checkAllSameGPU(__func__, {input_arg, output_arg});

    TORCH_CHECK(output_size.size() == 2, "adaptive_avg_pool2d: output_size must be 2");
    int64_t ndim = input.dim();
    TORCH_CHECK((ndim == 3 || ndim == 4),
      "adaptive_avg_pool2d(): Expected 3D or 4D tensor, but got ", input.sizes());
    for (const auto i : {-2, -1}) {
      TORCH_CHECK(input.size(i) > 0,
        "adaptive_avg_pool2d(): Expected input to have non-zero size for non-batch dimensions, "
        "but input has sizes ", input.sizes(), " with dimension ", i + ndim, " being "
        "empty");
    }

    Tensor input_ = input;
    switch (input.suggest_memory_format()) {
      case at::MemoryFormat::ChannelsLast: {
        // special case for tensor memory format in channels_last
        TORCH_CHECK(input.ndimension() == 4,
                    "adaptive_avg_pool2d(): Expected 4D tensor, but got ",
                    input.sizes());

        int sizeB = input_.size(0);
        int sizeC = input_.size(1);
        int isizeH = input_.size(2);
        int isizeW = input_.size(3);

        int64_t istrideB = input_.stride(0);
        int64_t istrideC = input_.stride(1);
        int64_t istrideH = input_.stride(2);
        int64_t istrideW = input_.stride(3);

        int osizeH = output_size[0];
        int osizeW = output_size[1];
        // preserve channels_last stride on output tensor;
        if (!output.is_contiguous(at::MemoryFormat::ChannelsLast)) {
          // TODO: modify this after resize_ added `memory_format` tag
          output.resize_({sizeB, sizeC, osizeH, osizeW}).as_strided_({sizeB, sizeC, osizeH, osizeW}, {sizeC*osizeH*osizeW, 1, osizeW*sizeC, sizeC});
        }

        if (output.numel() == 0) {
          return;
        }

        const int max_threads = std::min<int>(
            at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, CUDA_MAX_THREADS);
        int* maxThreadsDim = at::cuda::getCurrentDeviceProperties()->maxThreadsDim;
        int* maxGridSize = at::cuda::getCurrentDeviceProperties()->maxGridSize;
        size_t sharedMemPerBlock = at::cuda::getCurrentDeviceProperties()->sharedMemPerBlock;

        // Launch kernel on output tensor elements. Logic behind launch config:
        // output tensor size NCHW, strides NHWC;
        // Launch on:
        // N -> grid.x
        // H -> grid.z * block.z
        // W -> grid.y * block.y
        // C -> block.x
        // encourage larger block_y & block_z for better cache hit while maintain
        // reasonable block_x for coalesced memory access;
        int block_x = std::min<int>(
            maxThreadsDim[0], std::min<int>(lastPow2(sizeC), at::cuda::warp_size()));
        int block_y = std::min<int>(
            maxThreadsDim[1], std::min<int>(lastPow2(osizeW), max_threads / block_x));
        int block_z = std::min<int>(
            maxThreadsDim[2], std::min<int>(lastPow2(osizeH), max_threads / block_x / block_y));
        block_x = std::min<int>(
            maxThreadsDim[0], std::min<int>(lastPow2(sizeC), max_threads / block_y / block_z));
        const dim3 block(block_x, block_y, block_z);
        int kernel_stride_C = ceil_div(sizeC, block_x * 4);
        int kernel_size_C = ceil_div(sizeC, block_x * kernel_stride_C);

        // Do NOT clip grid_x, striding on Batch dimension is not in the kernel,
        // although it could be easily implemented given current kernel.
        int grid_x = sizeB*kernel_stride_C;
        // it's OK to clip grid_y & grid_z, as we block the two dimensions in the kernel;
        int grid_y = std::min<int>(
            maxGridSize[1], ceil_div(osizeW, block_y*BLOCK_STRIDE));
        int grid_z = std::min<int>(
            maxGridSize[2], ceil_div(osizeH, block_z*BLOCK_STRIDE));
        const dim3 grid(grid_x, grid_y, grid_z);


        // we are dealing with packed tensor here. max index is the same as numel.
        // TODO: to really support input tensor large enough to go beyond int32,
        // we will need to restrict out shared memory usage and adjust the launch
        // config;
        AT_ASSERT(input_.numel() < std::numeric_limits<int32_t>::max());
        AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
            input_.scalar_type(), "adaptive_avg_pool2d_nhwc_cuda", [&] {
              using opmath_t = at::opmath_type<scalar_t>;
              size_t shmem_size = (kernel_size_C * block_x * block_y * block_z) * sizeof(opmath_t);
              AT_ASSERT(shmem_size <= sharedMemPerBlock);
              adaptive_average_pool_nhwc<int32_t><<<grid, block, shmem_size, at::cuda::getCurrentCUDAStream()>>> (
                input_.const_data_ptr<scalar_t>(),
                output.mutable_data_ptr<scalar_t>(),
                sizeB, sizeC, isizeH, isizeW, osizeH, osizeW,
                kernel_stride_C, kernel_size_C,
                istrideB, istrideC, istrideH, istrideW);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }
          );
        break;
      }
      case at::MemoryFormat::Contiguous: {
        int64_t grid_x = input.size(-3);
        if (input.ndimension() == 4) {
           input_ = input.contiguous();
           grid_x *= input_.size(-4);
        }
        int64_t sizeD  = input_.size(-3);
        int64_t isizeH = input_.size(-2);
        int64_t isizeW = input_.size(-1);

        int64_t istrideD = input_.stride(-3);
        int64_t istrideH = input_.stride(-2);
        int64_t istrideW = input_.stride(-1);

        int64_t osizeH = output_size[0];
        int64_t osizeW = output_size[1];
        if (input.ndimension() == 4) {
           output.resize_({input_.size(-4), sizeD, osizeH, osizeW});
        } else {
           output.resize_({sizeD, osizeH, osizeW});
        }
        if (output.numel() == 0) {
          return;
        }

        AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
            input_.scalar_type(), "adaptive_avg_pool2d_cuda", [&] {
              const scalar_t *input_data = input_.const_data_ptr<scalar_t>();
              scalar_t *output_data = output.mutable_data_ptr<scalar_t>();

              // cuda blocks & threads:
              int blocksH = std::max<int64_t>((int)(16L / sizeD), 1);
              dim3 blocks(grid_x, blocksH);
              dim3 threads(32, 8);

              // run averagepool kernel
              adaptive_average_pool <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>> (
                input_data, output_data,
                isizeH, isizeW, osizeH, osizeW,
                istrideD, istrideH, istrideW);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }
          );
        break;
      }
      default:
        TORCH_CHECK(
          false,
          "Unsupported memory format. Supports only ChannelsLast, Contiguous");
    }
  }

  void adaptive_avg_pool2d_backward_out_cuda_template(
    Tensor& gradInput,
    const Tensor& gradOutput_,
    const Tensor& input)
  {
    TensorArg grad_input_arg{ gradInput, "gradInput", 1 },
              grad_output_arg{ gradOutput_, "gradOutput_", 2 },
              input_arg{ input, "input", 3 };

    adaptive_pool_empty_output_check(gradOutput_, "adaptive_avg_pool2d_backward");
    TORCH_CHECK(input.dim() == gradOutput_.dim(),
      __func__, ": Expected dimensions ", input.dim(), " for `gradOutput_` but got dimensions ", gradOutput_.dim());

    checkAllSameGPU(__func__, {grad_input_arg, grad_output_arg, input_arg});

    switch (input.suggest_memory_format()) {
      case at::MemoryFormat::ChannelsLast: {
        // special case for tensor memory format in channels_last
        TORCH_CHECK(input.ndimension() == 4,
                    "adaptive_avg_pool2d_backward_cuda(): Expected 4D tensor, but got ", input.ndimension());

        int sizeB = input.size(0);
        int sizeC = input.size(1);
        int isizeH = input.size(2);
        int isizeW = input.size(3);

        Tensor gradOutput = gradOutput_;

        int64_t ostrideB = gradOutput.stride(0);
        int64_t ostrideC = gradOutput.stride(1);
        int64_t ostrideH = gradOutput.stride(2);
        int64_t ostrideW = gradOutput.stride(3);

        int osizeH = gradOutput.size(-2);
        int osizeW = gradOutput.size(-1);

        // preserve channels_last stride on input tensor;
        if (!gradInput.is_contiguous(at::MemoryFormat::ChannelsLast)) {
          gradInput.as_strided_(
              {sizeB, sizeC, isizeH, isizeW},
              {sizeC*isizeH*isizeW, 1, isizeW*sizeC, sizeC});
        }

        int max_threads = std::min<int>(
            at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, CUDA_MAX_THREADS);
        int* maxThreadsDim = at::cuda::getCurrentDeviceProperties()->maxThreadsDim;
        int* maxGridSize = at::cuda::getCurrentDeviceProperties()->maxGridSize;
        size_t sharedMemPerBlock = at::cuda::getCurrentDeviceProperties()->sharedMemPerBlock;

        // Launch kernel on input tensor elements. Logic behind launch config:
        // input tensor size NCHW, strides NHWC;
        // Launch on:
        // N(C) -> grid.x (striding on C to reduce sh_mem usage)
        // H    -> grid.z * block.z
        // W    -> grid.y * block.y
        // C    -> block.x
        // encourage larger block_y & block_z for better cache hit while maintain
        // reasonable block_x for coalesced memory access;
        bool done = false;
        do {
          int block_x = std::max<int>(std::min<int>(
              maxThreadsDim[0], std::min<int>(lastPow2(sizeC), at::cuda::warp_size())), 1);
          int block_y = std::max<int>(std::min<int>(
              maxThreadsDim[1], std::min<int>(lastPow2(isizeW), max_threads / block_x)), 1);
          int block_z = std::max<int>(std::min<int>(
              maxThreadsDim[2], std::min<int>(lastPow2(isizeH), max_threads / block_x / block_y)), 1);
          block_x = std::max<int>(std::min<int>(
              maxThreadsDim[0], std::min<int>(lastPow2(sizeC), max_threads / block_y / block_z)), 1);
          const dim3 block(block_x, block_y, block_z);
          int kernel_stride_C = ceil_div(sizeC, block_x * 4);
          int kernel_size_C = ceil_div(sizeC, block_x * kernel_stride_C);

          // Do NOT clip grid_x, striding on Batch dimension is not in the kernel,
          // although it could be easily implemented given current kernel.
          int grid_x = sizeB*kernel_stride_C;
          // it's OK to clip grid_y & grid_z, as we block the two dimensions in the kernel;
          int grid_y = std::min<int>(
              maxGridSize[1], ceil_div(isizeW, block_y*BLOCK_STRIDE));
          int grid_z = std::min<int>(
              maxGridSize[2], ceil_div(isizeH, block_z*BLOCK_STRIDE));
          const dim3 grid(grid_x, grid_y, grid_z);

          // we are dealing with packed tensor here. max index is the same as numel.
          // TODO: to really support input tensor large enough to go beyond int32,
          // we will need to restrict out shared memory usage and adjust the launch
          // config;
          AT_ASSERT(input.numel() < std::numeric_limits<int32_t>::max());
          AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
              input.scalar_type(), "adaptive_avg_pool2d_backward_nhwc_cuda", [&] {
                size_t shmem_size = (kernel_size_C * block_x * block_y * block_z + osizeH + osizeW) * sizeof(scalar_t) + 2 * isizeW * sizeof(int32_t);
                if (shmem_size <= sharedMemPerBlock) {
                  adaptive_average_gradinput_nhwc<int32_t><<<grid, block, shmem_size, at::cuda::getCurrentCUDAStream()>>> (
                    gradInput.mutable_data_ptr<scalar_t>(),
                    gradOutput.const_data_ptr<scalar_t>(),
                    sizeB, sizeC, isizeH, isizeW, osizeH, osizeW,
                    kernel_stride_C, kernel_size_C,
                    ostrideB, ostrideC, ostrideH, ostrideW);
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                  done = true;
                } else {
                  TORCH_WARN_ONCE("Requested shmem_size exceeds sharedMemPerBlock limit! Reducing max_threads...");
                  max_threads /= 2;
                }
              }
            );
        } while (!done && max_threads);
        if (!done) {
          TORCH_INTERNAL_ASSERT(false, "Couldn't reduce launch bounds to accommodate sharedMemPerBlock limit");
        }
        break;
      }
      case at::MemoryFormat::Contiguous: {
        bool atomic = true; // suboptimal, but without atomic it doesn't pass the tests

        Tensor gradOutput = gradOutput_.contiguous();

        int64_t sizeD  = input.size(-3);
        int64_t isizeH = input.size(-2);
        int64_t isizeW = input.size(-1);

        int64_t osizeH = gradOutput.size(-2);
        int64_t osizeW = gradOutput.size(-1);

        int64_t grid_x = sizeD;
        if (input.ndimension() == 4) grid_x *= input.size(-4);

          //bool atomic = (isizeW%osizeW != 0) || (isizeH%osizeH != 0);
        AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
            input.scalar_type(), "adaptive_avg_pool2d_backward_cuda", [&] {
              const scalar_t *gradOutput_data = gradOutput.const_data_ptr<scalar_t>();
              scalar_t *gradInput_data = gradInput.mutable_data_ptr<scalar_t>();

              // cuda blocks & threads:
              int blocksH = std::max((int)(16L / sizeD), 1);
              dim3 blocks(grid_x, blocksH);
              dim3 threads(32, 8);

              if(atomic)
              {
                // run updateGradInput kernel, accumulate gradients atomically
                atomic_adaptive_average_gradinput <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>> (
                  gradInput_data, gradOutput_data,
                  isizeH, isizeW, osizeH, osizeW);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
              else
              {
                // run updateGradInput kernel
                adaptive_average_gradinput <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>> (
                  gradInput_data, gradOutput_data,
                  isizeH, isizeW, osizeH, osizeW);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
            }
          );
        break;
      }
      default:
        TORCH_CHECK(
          false,
          "Unsupported memory format. Supports only ChannelsLast, Contiguous");

    }
  }

} // namespace

  Tensor& adaptive_avg_pool2d_out_cuda(
    const Tensor& input,
    IntArrayRef output_size,
    Tensor& output)
  {
    adaptive_avg_pool2d_out_cuda_template(
      output, input, output_size);
    return output;
  }

  Tensor adaptive_avg_pool2d_cuda(
    at::Tensor const& input,
    IntArrayRef output_size)
  {
    auto output = at::empty({0}, input.options());
    adaptive_avg_pool2d_out_cuda_template(
      output, input, output_size);
    return output;
  }

  Tensor& adaptive_avg_pool2d_backward_out_cuda(
    Tensor& gradInput,
    const Tensor& gradOutput,
    const Tensor& input)
  {
    // See Note [Writing Nondeterministic Operations]
    // Nondeterministic because of atomicAdd usage
    globalContext().alertNotDeterministic("adaptive_avg_pool2d_backward_out_cuda");
    gradInput.resize_as_(input);
    if (gradInput.numel() != 0) {
      adaptive_avg_pool2d_backward_out_cuda_template(
        gradInput, gradOutput, input);
    }
    return gradInput;
  }

  Tensor adaptive_avg_pool2d_backward_cuda(
    const Tensor& gradOutput,
    const Tensor& input)
  {
    // See Note [Writing Nondeterministic Operations]
    // Nondeterministic because of atomicAdd usage
    globalContext().alertNotDeterministic("adaptive_avg_pool2d_backward_cuda");
    auto gradInput = at::zeros_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
    if (gradInput.numel() != 0) {
      adaptive_avg_pool2d_backward_out_cuda_template(
        gradInput, gradOutput, input);
    }
    return gradInput;
  }

} // namespace at::native

#undef BLOCK_STRIDE
#undef CUDA_MAX_THREADS
#undef START_IND
#undef END_IND
