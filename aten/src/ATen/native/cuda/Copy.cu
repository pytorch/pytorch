#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/Dispatch_v2.h>
#include <ATen/cuda/CachingHostAllocator.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAEvent.h>
#include <ATen/cuda/PeerToPeerAccess.h>
#include <ATen/native/Copy.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#else
#include <ATen/ops/empty_like.h>
#endif

#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/cuda/CUDAStream.h>

// TODO(NS): Investigate why FP8 conversion intrinsics end up being slower
#ifdef AT_USE_NV_CVT_INTRINSICS
#include <hip/hip_fp8.h>
#endif

namespace at::native {

void neg_kernel_cuda(TensorIteratorBase &iter);
void conj_kernel_cuda(TensorIteratorBase &iter);

void float16_copy_kernel_cuda(TensorIteratorBase &iter) {
    gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
        return static_cast<at::Half>(value);
    });
}

void bfloat16_copy_kernel_cuda(TensorIteratorBase &iter) {
    gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
        return static_cast<at::BFloat16>(value);
    });
}

void float8_copy_kernel_cuda(TensorIteratorBase &iter) {
  ScalarType dtype = iter.dtype(0);
  ScalarType other_dtype = iter.dtype(1);
  if (dtype == kFloat8_e4m3fn) {
    switch (other_dtype) {
      case kFloat:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
             return Float8_e4m3fn(value);
         });
         break;
      case kHalf:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(Half value) {
             return Float8_e4m3fn(value);
         });
         break;
      case kBFloat16:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(BFloat16 value) {
             return Float8_e4m3fn(value);
         });
         break;
      default:
        gpu_kernel(iter, [] GPU_LAMBDA(Float8_e4m3fn x) { return x; });
        break;
    }
  } else if (dtype == kFloat8_e5m2) {
    switch (other_dtype) {
      case kFloat:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
#ifdef AT_USE_NV_CVT_INTRINSICS
             const auto x =  __hip_cvt_float_to_fp8(value, __HIP_NOSAT, __HIP_E5M2_FNUZ);
             return Float8_e5m2(x, Float8_e5m2::from_bits());
#else
             return Float8_e5m2(value);
#endif
         });
         break;
      case kHalf:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(Half value) {
#ifdef AT_USE_NV_CVT_INTRINSICS
             const auto x =  __hip_cvt_halfraw_to_fp8(static_cast<__half>(value), __HIP_NOSAT, __HIP_E5M2_FNUZ);
             return Float8_e5m2(x, Float8_e5m2::from_bits());
#else
             return Float8_e5m2(value);
#endif
         });
         break;
      case kBFloat16:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(BFloat16 value) {
#ifdef AT_USE_NV_CVT_INTRINSICS
             const auto x =  __hip_cvt_bfloat16raw_to_fp8(static_cast<__hip_bfloat16>(value), __HIP_NOSAT, __HIP_E5M2_FNUZ);
             return Float8_e5m2(x, Float8_e5m2::from_bits());
#else
             return Float8_e5m2(value);
#endif
         });
         break;
      default:
         gpu_kernel(iter, [] GPU_LAMBDA(Float8_e5m2 x) { return x; });
         break;
    }
  } else if (dtype == kFloat8_e4m3fnuz) {
    switch (other_dtype) {
      case kFloat:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
             return Float8_e4m3fnuz(value);
         });
         break;
      case kHalf:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(Half value) {
             return Float8_e4m3fnuz(value);
         });
         break;
      case kBFloat16:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(BFloat16 value) {
             return Float8_e4m3fnuz(value);
         });
         break;
      default:
        gpu_kernel(iter, [] GPU_LAMBDA(Float8_e4m3fnuz x) { return x; });
        break;
    }
  } else if (dtype == kFloat8_e5m2fnuz) {
    switch (other_dtype) {
      case kFloat:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
             return Float8_e5m2fnuz(value);
         });
         break;
      case kHalf:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(Half value) {
             return Float8_e5m2fnuz(value);
         });
         break;
      case kBFloat16:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(BFloat16 value) {
             return Float8_e5m2fnuz(value);
         });
         break;
      default:
         gpu_kernel(iter, [] GPU_LAMBDA(Float8_e5m2fnuz x) { return x; });
         break;
    }
  } else if (dtype == kFloat8_e8m0fnu) {
    // TODO(#146647): clean this up, too much copy-pasta
    switch (other_dtype) {
      case kFloat:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
             return Float8_e8m0fnu(value);
         });
         break;
      case kHalf:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(Half value) {
             return Float8_e8m0fnu(value);
         });
         break;
      case kBFloat16:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(BFloat16 value) {
             return Float8_e8m0fnu(value);
         });
         break;
      default:
         gpu_kernel(iter, [] GPU_LAMBDA(Float8_e8m0fnu x) { return x; });
         break;
    }
  } else {
    TORCH_CHECK(false, "This supposed ot be called only for Float8 types");
  }
}

// TODO: We probably can use the opaque type trick to avoid creating duplicate
// kernels for equivalent bit lengths
void direct_copy_kernel_cuda(TensorIteratorBase &iter) {
  ScalarType dtype = iter.dtype(0);
  if (isQIntType(dtype)) {
    AT_DISPATCH_QINT_TYPES(dtype, "copy_", [&] {
      gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
    });
  } else if (isFloat8Type(dtype)) {
     float8_copy_kernel_cuda(iter);
  } else if (iter.dtype(1) == kFloat && (dtype == kBFloat16 || dtype == kHalf)) {
     if (dtype == kBFloat16) {
       bfloat16_copy_kernel_cuda(iter);
     } else {
       float16_copy_kernel_cuda(iter);
     }
  } else if (isBitsType(dtype)) {
    TORCH_CHECK(dtype == iter.dtype(1), "copy_() does not support casting "
      "bits types to different bits types. Source dtype is ", iter.dtype(1), "target dtype is ", dtype);
    AT_DISPATCH_BIT_TYPES(dtype, "copy_", [&] {
      gpu_kernel_nocast(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
    });
  } else {
    AT_DISPATCH_V2(
        dtype, "copy_", AT_WRAP([&] {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
    }), AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX), kHalf, kBool, kBFloat16, kComplexHalf, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
  }
}

void neg_conj_kernel_cuda(TensorIteratorBase &iter) {
  AT_DISPATCH_COMPLEX_TYPES(iter.common_dtype(), "neg_conj_cuda", [&] {
    gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return -std::conj(x); });
  });
}

using namespace at::cuda;

// device-to-device copy, does type conversion
void copy_device_to_device(TensorIterator& iter,
                           bool non_blocking,
                           bool p2p_enabled) {
  int64_t numel = iter.numel();

  // We can memcpy the memory if both tensors have the same type AND both
  // tensors are contiguous after dimension coalescing and reordering.
  bool same_type = iter.dtype(0) == iter.dtype(1);
  bool same_conj = iter.tensor(0).is_conj() == iter.tensor(1).is_conj();
  bool same_neg = iter.tensor(0).is_neg() == iter.tensor(1).is_neg();
  bool memcpy_eligible = same_type && same_conj && same_neg && iter.is_contiguous();

  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  CUDAGuard device_guard(src_device);

  // We always perform the copy on the source device, using the current stream
  // on the source device, and we fully synchronize on both src and dst's
  // current streams for completion of the copy. We have to explicitly do this
  // for non-contig copies. This mimics the behavior of cross-device
  // hipMemcpyAsync on the default stream.
  CUDAStream copy_stream = getCurrentCUDAStream(src_device.index());
  if (src_device != dst_device) {
    // This is a cross-device copy on the src current stream and dst current
    // stream. We perform a two-way barrier between both devices' streams
    // before the copy. This ensures that any write-after-write and
    // write-after-read dependencies on the destination side are handled, so
    // that no one is operating on the dst memory when we perform the copy.
    // src waits on dst barrier (src already waits on src)
    CUDAEvent dst_ready;
    device_guard.set_device(dst_device);
    dst_ready.record(getCurrentCUDAStream(dst_device.index()));

    device_guard.set_device(src_device);
    dst_ready.block(copy_stream);
  }

  if (memcpy_eligible) {
    void *dst = iter.data_ptr(0);
    void *src = iter.data_ptr(1);
    size_t size = numel * iter.element_size(0);
    if (src != dst || src_device != dst_device) {
      // Due to bizarre cuda driver intricacies, copies of
      // cudaMallocAsynced memory between devices that aren't
      // peer-to-peer-capable need "hipMemcpyPeerAsync".
      // So we let the allocator implement the correct call
      // (either hipMemcpyAsync or hipMemcpyPeerAsync)
      AT_CUDA_CHECK(CUDACachingAllocator::memcpyAsync(
        dst, dst_device.index(),
        src, src_device.index(),
        size, copy_stream, p2p_enabled));
    }
  } else {
    if (same_neg) {
      if (!same_conj) {
        conj_kernel_cuda(iter);
      } else {
        direct_copy_kernel_cuda(iter);
      }
    } else {
      if (!same_conj) {
        neg_conj_kernel_cuda(iter);
      } else {
        neg_kernel_cuda(iter);
      }
    }
  }

  if (src_device != dst_device) {
    // dst waits on src barrier (dst already waits on dst). We cannot
    // operate on dst's copy until the copy is complete.

    // Still on src_device, record stream event
    CUDAEvent src_ready;
    src_ready.record(copy_stream);

    device_guard.set_device(dst_device);
    src_ready.block(getCurrentCUDAStream(dst_device.index()));
  }

  AT_CUDA_CHECK(hipGetLastError());
}

static bool copy_requires_temporaries(TensorIterator& iter, bool p2p_enabled) {
  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  if (dst_device == src_device) {
    // We never require temporaries for copies on the same GPU.
    TORCH_INTERNAL_ASSERT(dst_device.is_cuda() && src_device.is_cuda());
    return false;
  }

  bool same_dtype = iter.dtype(0) == iter.dtype(1);
  if (same_dtype && iter.is_contiguous()) {
    // Contiguous same-dtype copies can always use hipMemcpyAsync
    return false;
  } else if (dst_device.is_cuda() && src_device.is_cuda()) {
    // Copies between GPUs can use the copy kernel if P2P is supported
    return !p2p_enabled;
  } else {
    // The remaining cases require temporaries. For example, this includes
    // non-contiguous copies between CPU and GPU.
    return true;
  }
}

static bool maybe_enable_p2p_access(Device dst_device, Device src_device) {
  if (dst_device.is_cpu() || src_device.is_cpu()) {
    return false;
  }
  return at::cuda::get_p2p_access(src_device.index(), dst_device.index());
}

static void copy_kernel_cuda(TensorIterator& iter, bool non_blocking) {
  TORCH_CHECK(iter.ntensors() == 2);

  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  // Enable p2p access between devices. (No-op if it involves the CPU)
  bool p2p_enabled = maybe_enable_p2p_access(dst_device, src_device);

  if (copy_requires_temporaries(iter, p2p_enabled)) {
    // NB: this involves recursive calls to copy. Be careful that those copies
    // don't require temporaries or you will cause an infinite recursion!
    auto& dst = iter.tensor(0);
    Tensor dst_contig;
    Tensor src_contig;

    // If non_blocking is true - type conversions are performed on the GPU
    // For blocking transfers conversions are performed on CPU to avoid allocating
    // extra GPU memory
    // for GPU-GPU transfers conversions are performed on the source device
    auto conversion_device = non_blocking ? kCUDA : kCPU;
    if (iter.device_type(1) == conversion_device) {
      dst_contig = dst.is_contiguous() ? dst : at::empty_like(dst, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      src_contig = iter.tensor(1).to(iter.dtype(0)).expand_as(dst).contiguous();
    } else {
      bool same_type = iter.dtype(0) == iter.dtype(1);
      dst_contig = (dst.is_contiguous() && same_type) ? dst : at::empty_like(dst, iter.dtype(1), LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      src_contig = iter.tensor(1).expand_as(dst).contiguous();
    }

    // propagate the correct conjugate bit
    dst_contig._set_conj(dst.is_conj());
    src_contig._set_conj(iter.tensor(1).is_conj());

    dst_contig._set_neg(dst.is_neg());
    src_contig._set_neg(iter.tensor(1).is_neg());

    // perform a same-dtype copy on contiguous tensors
    TORCH_INTERNAL_ASSERT(dst_contig.sizes().equals(src_contig.sizes()));
    TORCH_INTERNAL_ASSERT(dst_contig.scalar_type() == src_contig.scalar_type());
    dst_contig.copy_(src_contig, non_blocking);

    // if necessary, copy back into dst
    if (!dst_contig.is_same(dst)) {
      TORCH_INTERNAL_ASSERT(dst_contig.device() == dst.device());
      dst.copy_(dst_contig, non_blocking);
    }
    return;
  }

  // Copy on GPU (or between GPUs)
  if (dst_device.is_cuda() && src_device.is_cuda()) {
    copy_device_to_device(iter, non_blocking, p2p_enabled);
    return;
  }

  // Copy between CPU and GPU
  cuda::OptionalCUDAGuard device_guard;
  hipMemcpyKind kind;
  if (dst_device.is_cuda() && src_device.is_cpu()) {
    device_guard.set_device(dst_device);
    kind = hipMemcpyHostToDevice;
  } else if (dst_device.is_cpu() && src_device.is_cuda()) {
    device_guard.set_device(src_device);
    kind = hipMemcpyDeviceToHost;
  } else {
    TORCH_INTERNAL_ASSERT(false, "unsupported devices in GPU copy_()");
  }

  void* dst = iter.data_ptr(0);
  void* src = iter.data_ptr(1);
  int64_t nbytes = iter.numel() * iter.element_size(0);
  CUDAStream stream = getCurrentCUDAStream();

  if (non_blocking) {
    AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));
    // we use both the storage context and the tensor data pointer as the key
    // for the caching host allocator. This allows us to better attribute the
    // events to the original tensor allocation correctly. The cases we seek to
    // handle are:

    // 1: a user can pass a pinned memory tensor with an alternative
    // context, for example if allocating memory directly from the pinned memory
    // allocator and constructing a tensor with torch::from_blob.

    // 2: a user can pass a tensor with a different base pointer to the original
    // allocation (via slicing).
    const auto& dst_tensor = iter.tensor(0);
    const auto& src_tensor = iter.tensor(1);
    const auto& host_tensor = (dst_device == kCPU ? dst_tensor : src_tensor);
    auto* ptr = (dst_device == kCPU ? dst : src);
    auto* ctx = host_tensor.storage().data_ptr().get_context();
    // TODO: warn on the return value.
    at::getHostAllocator(at::kCUDA)->record_event(ptr, ctx, stream.unwrap());
  } else {
    at::cuda::memcpy_and_sync(dst, src, nbytes, kind, stream);
  }

  if (iter.tensor(0).is_conj() != iter.tensor(1).is_conj()) {
     iter.tensor(0).conj_physical_();
  }
  if (iter.tensor(0).is_neg() != iter.tensor(1).is_neg()) {
     iter.tensor(0).neg_();
  }
}

REGISTER_DISPATCH(copy_stub, &copy_kernel_cuda)

} // namespace at::native
