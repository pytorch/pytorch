#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/native/SegmentReduce.h>
#include <hip/hip_runtime.h>

#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>
#include <ATen/NumericUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/cub.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/zeros.h>
#include <ATen/ops/cat.h>
#include <ATen/ops/cumsum.h>
#endif

// SegmentReduce compilation with CUDA-12.9 causes  NVCC crash on Windows
// See https://github.com/pytorch/pytorch/issues/156181
#if !defined(_WIN32) || CUDART_VERSION < 12090

namespace at::native {

namespace {
struct CustomMax {
  template <typename OutputT>
  __host__ __device__ __forceinline__ OutputT
  operator()(const OutputT& a, const OutputT& b) const {
    if (at::_isnan(a)) {
      return a;
    } else if (at::_isnan(b)) {
      return b;
    }
    return std::max<OutputT>(a, b);
  }
};

struct CustomSum {
  template <typename OutputT>
  __host__ __device__ __forceinline__ OutputT
  operator()(const OutputT& a, const OutputT& b) const {
    return a + b;
  }
};

struct CustomProd {
  template <typename OutputT>
  __host__ __device__ __forceinline__ OutputT
  operator()(const OutputT& a, const OutputT& b) const {
    return a * b;
  }
};

struct CustomMin {
  template <typename OutputT>
  __host__ __device__ __forceinline__ OutputT
  operator()(const OutputT& a, const OutputT& b) const {
    if (at::_isnan(a)) {
      return a;
    } else if (at::_isnan(b)) {
      return b;
    }
    return std::min<OutputT>(a, b);
  }
};

template <typename scalar_t, typename index_t>
__global__ static void post_sum_div_kernel(
    scalar_t* output_data,
    const index_t* lengths_data,
    const int64_t segment_count,
    bool is_initial_set,
    scalar_t initial) {
  CUDA_KERNEL_LOOP(index, segment_count) {
    CUDA_KERNEL_ASSERT(lengths_data[index] >= 0);
    if (lengths_data[index] == 0) {
      if (is_initial_set) {
        output_data[index] = initial;
      } else {
        output_data[index] = NAN;
      }
    } else if (!at::_isnan(output_data[index])) {
      output_data[index] = output_data[index] / lengths_data[index];
    }
  }
}

template <typename scalar_t, typename index_t>
__global__ void segment_reduce_forward_kernel(
    ReductionType reduction,
    scalar_t* output_data,
    const scalar_t* values_data,
    const index_t* lengths_data,
    const index_t* lengths_cumsum_data,
    const int64_t segment_count,
    const int64_t lengths_stride_axis,
    bool is_initial_set,
    scalar_t initial_value,
    const int64_t outer_offset,
    const int64_t inner_offset,
    const int64_t data_stride_axis,
    const int64_t data_size_axis,
    const int64_t output_stride_axis,
    const int64_t output_size_axis,
    const int64_t lengths_cumsum_stride_axis) {
  int64_t idx = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  if (idx >= (outer_offset * segment_count * inner_offset)) {
    return;
  }
  int64_t row_id = idx / inner_offset;
  int64_t lane_id = idx % inner_offset;   // lane_id is the inner_idx
  int64_t outer_idx = row_id / segment_count;
  int64_t dim_idx = row_id % segment_count;

  int64_t offset_idx = outer_idx * lengths_cumsum_stride_axis * (segment_count + 1) + dim_idx;
  index_t offset_start = lengths_cumsum_data[offset_idx];
  index_t offset_end = lengths_cumsum_data[offset_idx + 1];

  // ===== step2: apply reduction
  for (index_t j = offset_start; j < offset_end; ++j) {
    int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                         + j * data_stride_axis + lane_id;
    const auto data = values_data[data_index];
    // TODO: There is no need to branch with every element
    if (reduction == ReductionType::MAX) {
      initial_value =
          at::_isnan(data) ? data : std::max<scalar_t>(initial_value, data);
    } else if (
        reduction == ReductionType::MEAN ||
        reduction == ReductionType::SUM) {
      initial_value = initial_value + data;
    } else if (reduction == ReductionType::MIN) {
      initial_value =
          at::_isnan(data) ? data : std::min<scalar_t>(initial_value, data);
    } else if (
      reduction == ReductionType::PROD) {
      initial_value = initial_value * data;
    }
  }

  // ===== step3: finalize reduction
  int64_t lengths_idx = outer_idx * lengths_stride_axis * segment_count + dim_idx;
  CUDA_KERNEL_ASSERT(lengths_data[lengths_idx] >= 0);
  if (lengths_data[lengths_idx] == 0 && !is_initial_set &&
      reduction == ReductionType::MEAN) {
    initial_value = static_cast<scalar_t>(NAN);
  } else if (
      reduction == ReductionType::MEAN && lengths_data[lengths_idx] > 0 &&
      !at::_isnan(initial_value)) {
    initial_value = initial_value / lengths_data[lengths_idx];
  }
  int64_t output_index = outer_idx * output_stride_axis * output_size_axis
                         + dim_idx * output_stride_axis + lane_id;
  output_data[output_index] = initial_value;
}


template <typename scalar_t, typename index_t>
__global__ void segment_reduce_backward_kernel(
    ReductionType reduction,
    scalar_t* grad_input_data,
    const scalar_t* grad_data,
    const scalar_t* output_data,
    const scalar_t* values_data,
    const index_t* lengths_data,
    const index_t* lengths_cumsum_data,
    const int64_t segment_count,
    const int64_t lengths_stride_axis,
    scalar_t initial_prod_value,
    const int64_t outer_offset,
    const int64_t inner_offset,
    const int64_t data_stride_axis,
    const int64_t data_size_axis,
    const int64_t output_stride_axis,
    const int64_t output_size_axis,
    const int64_t lengths_cumsum_stride_axis) {
  int64_t idx = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  if (idx >= (outer_offset * segment_count * inner_offset)) {
    return;
  }
  int64_t row_id = idx / inner_offset;
  int64_t lane_id = idx % inner_offset;  // lane_id is the inner_idx
  int64_t outer_idx = row_id / segment_count;
  int64_t dim_idx = row_id % segment_count;

  int64_t lengths_idx = outer_idx * lengths_stride_axis * segment_count + dim_idx;
  auto segment_length = lengths_data[lengths_idx];
  if (segment_length == 0) {
    return;
  }

  int64_t offset_idx = outer_idx * lengths_cumsum_stride_axis * (segment_count + 1) + dim_idx;
  index_t offset_start = lengths_cumsum_data[offset_idx];
  index_t offset_end = lengths_cumsum_data[offset_idx + 1];

  int64_t output_index = outer_idx * output_stride_axis * output_size_axis
                         + dim_idx * output_stride_axis + lane_id;

  if (reduction == ReductionType::MAX ||
      reduction == ReductionType::MIN) {
    int64_t counter = 0;
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      if (at::_isnan(values_data[data_index]) ||
          values_data[data_index] == output_data[output_index]) {
        grad_input_data[data_index] = grad_data[output_index];
        counter++;
      }
    }
    // Average gradient based on number of maximum elements in the
    // segment
    if (counter < 2) {
      return;
    }
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      if (grad_input_data[data_index] > 0) {
        grad_input_data[data_index] =
            grad_input_data[data_index] / counter;
      }
    }
  } else if (reduction == ReductionType::MEAN) {
    auto grad_val = grad_data[output_index] / segment_length;
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      grad_input_data[data_index] = grad_val;
    }
  } else if (reduction == ReductionType::SUM) {
    const auto& grad_val = grad_data[output_index];
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      grad_input_data[data_index] = grad_val;
    }
  } else if (reduction == ReductionType::PROD) {
    const auto& grad_val = grad_data[output_index] * output_data[output_index];
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      if (at::_isnan(values_data[data_index]) ||
          values_data[data_index] == 0) {
        // explicitly compute exclusive prod
        scalar_t exclusive_prod = initial_prod_value;
        int64_t prod_idx;
        for (int64_t k = offset_start; k < offset_end; ++k) {
          if (k != j) {
            prod_idx = outer_idx * data_stride_axis * data_size_axis
                       + k * data_stride_axis + lane_id;
            exclusive_prod *= values_data[prod_idx];
          }
        }
        grad_input_data[data_index] = grad_data[output_index] * exclusive_prod;
      } else {
        grad_input_data[data_index] = grad_val / values_data[data_index];
      }
    }
  }
}
} // namespace

Tensor _segment_reduce_lengths_offsets_backward_cuda_kernel(
    const Tensor& grad_contig,
    const Tensor& output_contig,
    const Tensor& data_contig,
    ReductionType reduction,
    const Tensor& lengths_or_offsets_contig,
    int64_t axis,
    const std::optional<Scalar>& initial,
    bool is_offsets_like) {
  axis = lengths_or_offsets_contig.dim() - 1;
  int64_t segment_count = is_offsets_like ?
                          lengths_or_offsets_contig.size(axis) - 1 :
                          lengths_or_offsets_contig.size(axis);
  int64_t lengths_stride_axis = lengths_or_offsets_contig.stride(axis);
  auto grad_input = at::zeros({data_contig.sizes()}, grad_contig.options());

  auto offsets = lengths_or_offsets_contig;
  auto lengths = lengths_or_offsets_contig;
  if (is_offsets_like) {
    lengths = lengths.diff();
  } else {
    auto zeros_shape = offsets.sizes().vec();
    zeros_shape[axis] = 1;
    offsets = at::cat({at::zeros(zeros_shape, offsets.options()), offsets}, axis);
    offsets.cumsum_(axis);
  }

  // outer_offset is the size of the outer dimensions of output (before axis)
  // inner_offset is the size of the inner dimensions of output (after axis)
  int64_t outer_offset = 1, inner_offset = 1;
  for (int64_t d = 0; d < axis; d++) {
    outer_offset *= output_contig.size(d);
  }
  for (int64_t d = axis + 1; d < output_contig.dim(); d++) {
    inner_offset *= output_contig.size(d);
  }

  constexpr int threads_per_block = 256;
  int64_t num_blocks = (outer_offset * inner_offset * segment_count + threads_per_block - 1) / threads_per_block;

  num_blocks = std::max(num_blocks, (int64_t)1);

  auto data_stride_axis = data_contig.stride(axis);
  auto data_size_axis = data_contig.size(axis);
  auto output_stride_axis = output_contig.stride(axis);
  auto output_size_axis = output_contig.size(axis);
  auto offsets_stride_axis = offsets.stride(axis);

  AT_DISPATCH_INDEX_TYPES(
      lengths_or_offsets_contig.scalar_type(), "_segment_reduce_cuda_lengths_offsets_backward_kernel1", ([&] {
        const auto* lengths_data = lengths.const_data_ptr<index_t>();
        auto* offsets_data = offsets.const_data_ptr<index_t>();

        // TODO: Switch to TensorIterator for better maintainablility and
        // readability
        AT_DISPATCH_FLOATING_TYPES_AND2(
            kBFloat16,
            kHalf,
            data_contig.scalar_type(),
            "_segment_reduce_cpu",
            ([&]() {
              auto* output_data = output_contig.const_data_ptr<scalar_t>();
              auto* grad_data = grad_contig.const_data_ptr<scalar_t>();
              auto* grad_input_data = grad_input.mutable_data_ptr<scalar_t>();
              const auto* values_data = data_contig.const_data_ptr<scalar_t>();

              scalar_t initial_prod_value;
              if (initial.has_value()) {
                initial_prod_value = initial.value().to<scalar_t>();
              } else {
                initial_prod_value = 1;
              }

              segment_reduce_backward_kernel<scalar_t>
                  <<<num_blocks,
                     threads_per_block,
                     0,
                     at::cuda::getCurrentCUDAStream()>>>(
                      reduction,
                      grad_input_data,
                      grad_data,
                      output_data,
                      values_data,
                      lengths_data,
                      offsets_data,
                      segment_count,
                      lengths_stride_axis,
                      initial_prod_value,
                      outer_offset,
                      inner_offset,
                      data_stride_axis,
                      data_size_axis,
                      output_stride_axis,
                      output_size_axis,
                      offsets_stride_axis
                    );
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }));
      }));
  return grad_input;
}

Tensor _segment_reduce_lengths_backward_cuda_kernel(
  const Tensor& grad_contig,
  const Tensor& output_contig,
  const Tensor& data_contig,
  ReductionType reduction,
  const Tensor& lengths_contig,
  int64_t axis,
  const std::optional<Scalar>& initial) {
  return _segment_reduce_lengths_offsets_backward_cuda_kernel(
    grad_contig, output_contig, data_contig, reduction, lengths_contig, axis, initial, /*is_offsets_like=*/false);
}

Tensor _segment_reduce_offsets_backward_cuda_kernel(
  const Tensor& grad_contig,
  const Tensor& output_contig,
  const Tensor& data_contig,
  ReductionType reduction,
  const Tensor& offsets_contig,
  int64_t axis,
  const std::optional<Scalar>& initial) {
  return _segment_reduce_lengths_offsets_backward_cuda_kernel(
    grad_contig, output_contig, data_contig, reduction, offsets_contig, axis, initial, /*is_offsets_like=*/true);
}

Tensor _segment_reduce_lengths_offsets_cuda_kernel(
  ReductionType reduction,
  const Tensor& data,
  const Tensor& lengths_or_offsets,
  int64_t axis,
  const std::optional<Scalar>& initial,
  bool is_offsets_like) {
  // data and lengths_or_offsets should be contiguous from the call to .contiguous in segment_reduce_kernel
  TORCH_CHECK(data.is_contiguous());
  TORCH_CHECK(lengths_or_offsets.is_contiguous());
  axis = lengths_or_offsets.dim() - 1;
  int64_t segment_count = is_offsets_like ? lengths_or_offsets.size(axis) - 1 : lengths_or_offsets.size(axis);
  int64_t lengths_stride_axis = lengths_or_offsets.stride(axis);
  auto output_shape = data.sizes().vec();
  output_shape[axis] = segment_count;
  auto output = at::empty(output_shape, data.options());


  auto offsets = lengths_or_offsets;
  auto lengths = lengths_or_offsets;
  if (is_offsets_like) {
    lengths = lengths.diff();
  } else {
    auto zeros_shape = offsets.sizes().vec();
    zeros_shape[axis] = 1;
    offsets = at::cat({at::zeros(zeros_shape, offsets.options()), offsets}, axis);
    offsets.cumsum_(axis);
  }

  // outer_offset is the size of the outer dimensions of output (before axis)
  // inner_offset is the size of the inner dimensions of output (after axis)
  int64_t outer_offset = 1, inner_offset = 1;
  for (int64_t d = 0; d < axis; d++) {
    outer_offset *= output.size(d);
  }
  for (int64_t d = axis + 1; d < output.dim(); d++) {
    inner_offset *= output.size(d);
  }

  constexpr int threads_per_block = 256;
  // segment_count * stride_count is just output.numel() ?
  int64_t num_blocks = (output.numel() + threads_per_block - 1) / threads_per_block;

  num_blocks = std::max(num_blocks, (int64_t)1);

  auto data_stride_axis = data.stride(axis);
  auto data_size_axis = data.size(axis);
  auto output_stride_axis = output.stride(axis);
  auto output_size_axis = output.size(axis);
  auto offsets_stride_axis = offsets.stride(axis);

  AT_DISPATCH_INDEX_TYPES(
      lengths_or_offsets.scalar_type(), "_segment_reduce_cuda_kernel1", ([&] {
        auto* offsets_data_ptr = offsets.const_data_ptr<index_t>();
        auto* lengths_data_ptr = lengths.const_data_ptr<index_t>();
        AT_DISPATCH_FLOATING_TYPES_AND2(
            at::ScalarType::Half,
            at::ScalarType::BFloat16,
            data.scalar_type(),
            "segment_reduce_cuda",
            [&]() {
              auto* data_data_ptr = data.const_data_ptr<scalar_t>();
              auto* output_data_ptr = output.mutable_data_ptr<scalar_t>();

              // initialize starting value
              scalar_t initial_value = 0;
              if (initial.has_value()) {
                initial_value = initial.value().to<scalar_t>();
              } else if (reduction == ReductionType::MAX) {
                initial_value = -std::numeric_limits<scalar_t>::infinity();
              } else if (
                  reduction == ReductionType::MEAN ||
                  reduction == ReductionType::SUM) {
                initial_value = 0;
              } else if (reduction == ReductionType::MIN) {
                initial_value = std::numeric_limits<scalar_t>::infinity();
              } else if (reduction == ReductionType::PROD) {
                initial_value = 1;
              }

              if (output_shape.size() > 1) {
                segment_reduce_forward_kernel<scalar_t>
                    <<<num_blocks,
                       threads_per_block,
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        reduction,
                        output_data_ptr,
                        data_data_ptr,
                        lengths_data_ptr,
                        offsets_data_ptr,
                        segment_count,
                        lengths_stride_axis,
                        initial.has_value(),
                        initial_value,
                        outer_offset,
                        inner_offset,
                        data_stride_axis,
                        data_size_axis,
                        output_stride_axis,
                        output_size_axis,
                        offsets_stride_axis
                      );
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              } else {
                if (reduction == ReductionType::MAX) {
                  CustomMax max_op{};
                  CUB_WRAPPER(
                      hipcub::DeviceSegmentedReduce::Reduce,
                      data_data_ptr,
                      output_data_ptr,
                      segment_count,
                      offsets_data_ptr,
                      offsets_data_ptr + 1,
                      max_op,
                      initial_value,
                      at::cuda::getCurrentCUDAStream());
                } else if (reduction == ReductionType::MEAN) {
                  CustomSum sum_op{};
                  CUB_WRAPPER(
                      hipcub::DeviceSegmentedReduce::Reduce,
                      data_data_ptr,
                      output_data_ptr,
                      segment_count,
                      offsets_data_ptr,
                      offsets_data_ptr + 1,
                      sum_op,
                      initial_value,
                      at::cuda::getCurrentCUDAStream());

                  post_sum_div_kernel<scalar_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          output_data_ptr,
                          lengths_data_ptr,
                          segment_count,
                          initial.has_value(),
                          initial_value);
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                } else if (reduction == ReductionType::MIN) {
                  CustomMin min_op{};
                  CUB_WRAPPER(
                      hipcub::DeviceSegmentedReduce::Reduce,
                      data_data_ptr,
                      output_data_ptr,
                      segment_count,
                      offsets_data_ptr,
                      offsets_data_ptr + 1,
                      min_op,
                      initial_value,
                      at::cuda::getCurrentCUDAStream());
                } else if (reduction == ReductionType::SUM) {
                  CustomSum sum_op{};
                  CUB_WRAPPER(
                      hipcub::DeviceSegmentedReduce::Reduce,
                      data_data_ptr,
                      output_data_ptr,
                      segment_count,
                      offsets_data_ptr,
                      offsets_data_ptr + 1,
                      sum_op,
                      initial_value,
                      at::cuda::getCurrentCUDAStream());
                } else if (reduction == ReductionType::PROD) {
                  CustomProd prod_op{};
                  CUB_WRAPPER(
                      hipcub::DeviceSegmentedReduce::Reduce,
                      data_data_ptr,
                      output_data_ptr,
                      segment_count,
                      offsets_data_ptr,
                      offsets_data_ptr + 1,
                      prod_op,
                      initial_value,
                      at::cuda::getCurrentCUDAStream());
                }
              }
            });
      }));

  return output;
}

Tensor _segment_reduce_lengths_cuda_kernel(
  ReductionType reduction,
  const Tensor& data,
  const Tensor& lengths,
  int64_t axis,
  const std::optional<Scalar>& initial) {
  return _segment_reduce_lengths_offsets_cuda_kernel(
    reduction, data, lengths, axis, initial, /*is_offsets_like=*/false);
}

Tensor _segment_reduce_offsets_cuda_kernel(
  ReductionType reduction,
  const Tensor& data,
  const Tensor& offsets,
  int64_t axis,
  const std::optional<Scalar>& initial) {
  return _segment_reduce_lengths_offsets_cuda_kernel(
    reduction, data, offsets, axis, initial, /*is_offsets_like=*/true);
}

REGISTER_DISPATCH(_segment_reduce_lengths_stub, &_segment_reduce_lengths_cuda_kernel)
REGISTER_DISPATCH(_segment_reduce_offsets_stub, &_segment_reduce_offsets_cuda_kernel)
REGISTER_DISPATCH(
    _segment_reduce_lengths_backward_stub,
    &_segment_reduce_lengths_backward_cuda_kernel);
REGISTER_DISPATCH(
  _segment_reduce_offsets_backward_stub,
  &_segment_reduce_offsets_backward_cuda_kernel);

} // namespace at::native

#endif
