#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/native/group_norm.h>

#include <type_traits>

#include <thrust/tuple.h>

#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/SharedReduceOps.h>
#include <ATen/native/TensorIterator.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#else
#include <ATen/ops/empty.h>
#endif

namespace at::native {

namespace {

constexpr int kCUDANumThreads = 256;
constexpr int kReduceTileSize = 32;

template <typename T>
__global__ void RowwiseMomentsCUDAKernel(
    int64_t N,
    T eps,
    const T* X,
    T* mean,
    T* rstd) {
  using T_ACC = acc_type<T, true>;
  using WelfordType = WelfordData<T_ACC, int64_t>;
  using WelfordOp =
      WelfordOps<T_ACC, T_ACC, int64_t, thrust::pair<T_ACC, T_ACC>>;

  const int64_t i = blockIdx.x;
  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    val = welford_op.reduce(val, static_cast<T_ACC>(X[index]), index);
  }
  if (blockDim.x <= C10_WARP_SIZE) {
    val = cuda_utils::WarpReduce(val, welford_op);
  } else {
    // There will be a warning if we declare a __shared__ WelfordType array.
    // https://github.com/pytorch/pytorch/pull/13967
    __shared__ typename std::aligned_storage<
        sizeof(WelfordType),
        alignof(WelfordType)>::type val_shared[C10_WARP_SIZE];
    WelfordType* val_shared_ptr = reinterpret_cast<WelfordType*>(val_shared);
    val = cuda_utils::BlockReduce(
        val,
        welford_op,
        /*identity_element=*/WelfordType(0, 0, 0, 0),
        val_shared_ptr);
  }
  if (threadIdx.x == 0) {
    T_ACC m1;
    T_ACC m2;
    thrust::tie(m2, m1) = welford_op.project(val);
    mean[i] = m1;
    rstd[i] = c10::cuda::compat::rsqrt(m2 + static_cast<T_ACC>(eps));
  }
}

template <typename T>
__global__ void ComputeFusedParamsCUDAKernel(
    int64_t N,
    int64_t C,
    int64_t group,
    const T* mean,
    const T* rstd,
    const T* gamma,
    const T* beta,
    acc_type<T, true>* a,
    acc_type<T, true>* b) {
  using T_ACC = acc_type<T, true>;
  const int64_t index = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  if (index < N * C) {
    const int64_t ng = index / (C / group);
    const int64_t c = index % C;
    const T_ACC scale = (gamma == nullptr)
        ? static_cast<T_ACC>(rstd[ng])
        : static_cast<T_ACC>(rstd[ng]) * static_cast<T_ACC>(gamma[c]);
    a[index] = scale;
    b[index] = -scale * static_cast<T_ACC>(mean[ng]) +
        ((beta == nullptr) ? 0 : static_cast<T_ACC>(beta[c]));
  }
}

template <typename T>
__global__ void Compute1dBackwardFusedParamsCUDAKernel(
    int64_t C,
    int64_t group,
    const T* dY,
    const T* X,
    const T* mean,
    const T* rstd,
    const T* gamma,
    acc_type<T, true>* c2,
    acc_type<T, true>* c3) {
  using T_ACC = acc_type<T, true>;
  const int64_t G = group;
  const int64_t D = C / G;
  const int64_t n = blockIdx.x;
  const int64_t g = blockIdx.y;
  const int64_t ng = n * G + g;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t i = threadIdx.x; i < D; i += blockDim.x) {
    const int64_t index = ng * D + i;
    const int64_t c = g * D + i;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[c]);
    sum1 += dY[index] * X[index] * gamma_v;
    sum2 += dY[index] * gamma_v;
  }
  if (blockDim.x <= C10_WARP_SIZE) {
    sum1 = cuda_utils::WarpReduceSum<T_ACC>(sum1);
    sum2 = cuda_utils::WarpReduceSum<T_ACC>(sum2);
  } else {
    __shared__ T_ACC ds_shared[C10_WARP_SIZE];
    __shared__ T_ACC db_shared[C10_WARP_SIZE];
    sum1 = cuda_utils::BlockReduceSum<T_ACC>(sum1, ds_shared);
    sum2 = cuda_utils::BlockReduceSum<T_ACC>(sum2, db_shared);
  }
  if (threadIdx.x == 0) {
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>(D);
    const T_ACC x = (sum2 * static_cast<T_ACC>(mean[ng]) - sum1) *
        static_cast<T_ACC>(rstd[ng]) * static_cast<T_ACC>(rstd[ng]) *
        static_cast<T_ACC>(rstd[ng]) * s;
    c2[ng] = x;
    c3[ng] = -x * static_cast<T_ACC>(mean[ng]) -
        sum2 * static_cast<T_ACC>(rstd[ng]) * s;
  }
}

template <typename T>
__global__ void GammaBeta1dBackwardCUDAKernel1(
    int64_t N,
    int64_t C,
    int64_t group,
    const T* dY,
    const T* X,
    const T* mean,
    const T* rstd,
    T* dgamma,
    T* dbeta) {
  using T_ACC = acc_type<T, true>;
  const int64_t c = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  if (c < C) {
    const int64_t G = group;
    const int64_t D = C / G;
    T_ACC sum1 = 0;
    T_ACC sum2 = 0;
    for (int64_t n = 0; n < N; ++n) {
      const int64_t nc = n * C + c;
      const int64_t ng = n * G + c / D;
      const T_ACC dy_acc = static_cast<T_ACC>(dY[nc]);
      const T_ACC x_acc = static_cast<T_ACC>(X[nc]);
      sum1 += (dgamma == nullptr)
          ? T_ACC(0)
          : ((dy_acc * x_acc - dy_acc * static_cast<T_ACC>(mean[ng])) *
             static_cast<T_ACC>(rstd[ng]));
      sum2 += (dbeta == nullptr) ? T_ACC(0) : dy_acc;
    }
    if (dgamma != nullptr) {
      dgamma[c] = sum1;
    }
    if (dbeta != nullptr) {
      dbeta[c] = sum2;
    }
  }
}

template <typename T>
__global__ void GammaBeta1dBackwardCUDAKernel2(
    int64_t N,
    int64_t C,
    int64_t group,
    const T* dY,
    const T* X,
    const T* mean,
    const T* rstd,
    T* dgamma,
    T* dbeta) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC g_shared[kReduceTileSize][kReduceTileSize + 1];
  __shared__ T_ACC b_shared[kReduceTileSize][kReduceTileSize + 1];
  const int64_t c = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  T_ACC dg_sum1 = 0;
  T_ACC dg_sum2 = 0;
  T_ACC db_sum1 = 0;
  T_ACC db_sum2 = 0;
  if (c < C) {
    const int64_t G = group;
    const int64_t D = C / G;
    // Accumulate each 32 cols into a 32 * 32 tile.
    // Since the blockDim is (32, 16), accumulate twice for 1st and 2nd 16 rows
    // of a 32 contiguous elements.
    for (int64_t n = threadIdx.y; n < N; n += blockDim.y * 2) {
      const int64_t n1 = n;
      const int64_t n2 = n + blockDim.y;
      const int64_t nc1 = n1 * C + c;
      const int64_t nc2 = n2 * C + c;
      const int64_t ng1 = n1 * G + c / D;
      const int64_t ng2 = n2 * G + c / D;
      const T_ACC dy1_acc = static_cast<T_ACC>(dY[nc1]);
      const T_ACC x1_acc = static_cast<T_ACC>(X[nc1]);
      dg_sum1 += dgamma == nullptr
          ? T_ACC(0)
          : ((dy1_acc * x1_acc - dy1_acc * static_cast<T_ACC>(mean[ng1])) *
             static_cast<T_ACC>(rstd[ng1]));
      db_sum1 += dbeta == nullptr ? T_ACC(0) : dy1_acc;
      if (n2 < N) {
        const T_ACC dy2_acc = static_cast<T_ACC>(dY[nc2]);
        const T_ACC x2_acc = static_cast<T_ACC>(X[nc2]);
        dg_sum2 += dgamma == nullptr
            ? T_ACC(0)
            : ((dy2_acc * x2_acc - dy2_acc * static_cast<T_ACC>(mean[ng2])) *
               static_cast<T_ACC>(rstd[ng2]));
        db_sum2 += dbeta == nullptr ? T_ACC(0) : dy2_acc;
      }
    }
  }

  // Write accumulated tile to shared memory.
  g_shared[threadIdx.y][threadIdx.x] = dg_sum1;
  g_shared[threadIdx.y + blockDim.y][threadIdx.x] = dg_sum2;
  b_shared[threadIdx.y][threadIdx.x] = db_sum1;
  b_shared[threadIdx.y + blockDim.y][threadIdx.x] = db_sum2;
  __syncthreads();

  // Do warp reduce for the 1st 16 cols in the tile.
  T_ACC sum1 = g_shared[threadIdx.x][threadIdx.y];
  T_ACC sum2 = b_shared[threadIdx.x][threadIdx.y];
  sum1 = cuda_utils::WarpReduceSum<T_ACC>(sum1);
  sum2 = cuda_utils::WarpReduceSum<T_ACC>(sum2);
  if (threadIdx.x == 0) {
    const int64_t c = blockIdx.x * blockDim.x + threadIdx.y;
    if (c < C) {
      if (dgamma != nullptr) {
        dgamma[c] = sum1;
      }
      if (dbeta != nullptr) {
        dbeta[c] = sum2;
      }
    }
  }

  // Do warp reduce for the 2nd 16 cols in the tile.
  sum1 = g_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum2 = b_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum1 = cuda_utils::WarpReduceSum<T_ACC>(sum1);
  sum2 = cuda_utils::WarpReduceSum<T_ACC>(sum2);
  if (threadIdx.x == 0) {
    const int64_t c = blockIdx.x * blockDim.x + threadIdx.y + blockDim.y;
    if (c < C) {
      if (dgamma != nullptr) {
        dgamma[c] = sum1;
      }
      if (dbeta != nullptr) {
        dbeta[c] = sum2;
      }
    }
  }
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    int64_t HxW,
    const T* dY,
    const T* X,
    acc_type<T, true>* ds,
    acc_type<T, true>* db) {
  using T_ACC = acc_type<T, true>;
  const int64_t nc = blockIdx.x;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t hw = threadIdx.x; hw < HxW; hw += blockDim.x) {
    const int64_t index = nc * HxW + hw;
    sum1 += static_cast<T_ACC>(dY[index]) * static_cast<T_ACC>(X[index]);
    sum2 += static_cast<T_ACC>(dY[index]);
  }
  if (blockDim.x <= C10_WARP_SIZE) {
    sum1 = cuda_utils::WarpReduceSum<T_ACC>(sum1);
    sum2 = cuda_utils::WarpReduceSum<T_ACC>(sum2);
  } else {
    __shared__ T_ACC ds_shared[C10_WARP_SIZE];
    __shared__ T_ACC db_shared[C10_WARP_SIZE];
    sum1 = cuda_utils::BlockReduceSum<T_ACC>(sum1, ds_shared);
    sum2 = cuda_utils::BlockReduceSum<T_ACC>(sum2, db_shared);
  }
  if (threadIdx.x == 0) {
    ds[nc] = sum1;
    db[nc] = sum2;
  }
}

template <typename T>
__global__ void ComputeBackwardFusedParamsCUDAKernel(
    int64_t C,
    int64_t HxW,
    int64_t group,
    const T* mean,
    const T* rstd,
    const T* gamma,
    const acc_type<T, true>* ds,
    const acc_type<T, true>* db,
    acc_type<T, true>* c2,
    acc_type<T, true>* c3) {
  using T_ACC = acc_type<T, true>;
  const int64_t G = group;
  const int64_t D = C / G;
  const int64_t n = blockIdx.x;
  const int64_t g = blockIdx.y;
  const int64_t ng = n * G + g;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t i = threadIdx.x; i < D; i += blockDim.x) {
    const int64_t index = ng * D + i;
    const int64_t c = g * D + i;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[c]);
    sum1 += ds[index] * gamma_v;
    sum2 += db[index] * gamma_v;
  }
  if (blockDim.x <= C10_WARP_SIZE) {
    sum1 = cuda_utils::WarpReduceSum<T_ACC>(sum1);
    sum2 = cuda_utils::WarpReduceSum<T_ACC>(sum2);
  } else {
    __shared__ T_ACC ds_shared[C10_WARP_SIZE];
    __shared__ T_ACC db_shared[C10_WARP_SIZE];
    sum1 = cuda_utils::BlockReduceSum<T_ACC>(sum1, ds_shared);
    sum2 = cuda_utils::BlockReduceSum<T_ACC>(sum2, db_shared);
  }
  if (threadIdx.x == 0) {
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>(D * HxW);
    const T_ACC x = (sum2 * static_cast<T_ACC>(mean[ng]) - sum1) *
        static_cast<T_ACC>(rstd[ng]) * static_cast<T_ACC>(rstd[ng]) *
        static_cast<T_ACC>(rstd[ng]) * s;
    c2[ng] = x;
    c3[ng] = -x * static_cast<T_ACC>(mean[ng]) -
        sum2 * static_cast<T_ACC>(rstd[ng]) * s;
  }
}

template <typename T>
__global__ void GammaBetaBackwardCUDAKernel1(
    int64_t N,
    int64_t C,
    int64_t group,
    const T* mean,
    const T* rstd,
    const acc_type<T, true>* ds,
    const acc_type<T, true>* db,
    T* dgamma,
    T* dbeta) {
  using T_ACC = acc_type<T, true>;
  const int64_t c = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  if (c < C) {
    const int64_t G = group;
    const int64_t D = C / G;
    T_ACC sum1 = 0;
    T_ACC sum2 = 0;
    for (int64_t n = 0; n < N; ++n) {
      const int64_t nc = n * C + c;
      const int64_t ng = n * G + c / D;
      sum1 += (dgamma == nullptr)
          ? T_ACC(0)
          : ((ds[nc] - db[nc] * static_cast<T_ACC>(mean[ng])) *
             static_cast<T_ACC>(rstd[ng]));
      sum2 += (dbeta == nullptr) ? T_ACC(0) : db[nc];
    }
    if (dgamma != nullptr) {
      dgamma[c] = sum1;
    }
    if (dbeta != nullptr) {
      dbeta[c] = sum2;
    }
  }
}

template <typename T>
__global__ void GammaBetaBackwardCUDAKernel2(
    int64_t N,
    int64_t C,
    int64_t group,
    const T* mean,
    const T* rstd,
    const acc_type<T, true>* ds,
    const acc_type<T, true>* db,
    T* dgamma,
    T* dbeta) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC g_shared[kReduceTileSize][kReduceTileSize + 1];
  __shared__ T_ACC b_shared[kReduceTileSize][kReduceTileSize + 1];
  const int64_t c = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  T_ACC dg_sum1 = 0;
  T_ACC dg_sum2 = 0;
  T_ACC db_sum1 = 0;
  T_ACC db_sum2 = 0;
  if (c < C) {
    const int64_t G = group;
    const int64_t D = C / G;
    // Accumulate each 32 cols into a 32 * 32 tile.
    // Since the blockDim is (32, 16), accumulate twice for 1st and 2nd 16 rows
    // of a 32 contiguous elements.
    for (int64_t n = threadIdx.y; n < N; n += blockDim.y * 2) {
      const int64_t n1 = n;
      const int64_t n2 = n + blockDim.y;
      const int64_t nc1 = n1 * C + c;
      const int64_t nc2 = n2 * C + c;
      const int64_t ng1 = n1 * G + c / D;
      const int64_t ng2 = n2 * G + c / D;
      dg_sum1 += dgamma == nullptr
          ? T_ACC(0)
          : ((ds[nc1] - db[nc1] * static_cast<T_ACC>(mean[ng1])) *
             static_cast<T_ACC>(rstd[ng1]));
      db_sum1 += dbeta == nullptr ? T_ACC(0) : db[nc1];
      if (n2 < N) {
        dg_sum2 += dgamma == nullptr
            ? T_ACC(0)
            : ((ds[nc2] - db[nc2] * static_cast<T_ACC>(mean[ng2])) *
               static_cast<T_ACC>(rstd[ng2]));
        db_sum2 += dbeta == nullptr ? T_ACC(0) : db[nc2];
      }
    }
  }

  // Write accumulated tile to shared memory.
  g_shared[threadIdx.y][threadIdx.x] = dg_sum1;
  g_shared[threadIdx.y + blockDim.y][threadIdx.x] = dg_sum2;
  b_shared[threadIdx.y][threadIdx.x] = db_sum1;
  b_shared[threadIdx.y + blockDim.y][threadIdx.x] = db_sum2;
  __syncthreads();

  // Do warp reduce for the 1st 16 cols in the tile.
  T_ACC sum1 = g_shared[threadIdx.x][threadIdx.y];
  T_ACC sum2 = b_shared[threadIdx.x][threadIdx.y];
  sum1 = cuda_utils::WarpReduceSum<T_ACC>(sum1);
  sum2 = cuda_utils::WarpReduceSum<T_ACC>(sum2);
  if (threadIdx.x == 0) {
    const int64_t c = blockIdx.x * blockDim.x + threadIdx.y;
    if (c < C) {
      if (dgamma != nullptr) {
        dgamma[c] = sum1;
      }
      if (dbeta != nullptr) {
        dbeta[c] = sum2;
      }
    }
  }

  // Do warp reduce for the 2st 16 cols in the tile.
  sum1 = g_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum2 = b_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum1 = cuda_utils::WarpReduceSum<T_ACC>(sum1);
  sum2 = cuda_utils::WarpReduceSum<T_ACC>(sum2);
  if (threadIdx.x == 0) {
    const int64_t c = blockIdx.x * blockDim.x + threadIdx.y + blockDim.y;
    if (c < C) {
      if (dgamma != nullptr) {
        dgamma[c] = sum1;
      }
      if (dbeta != nullptr) {
        dbeta[c] = sum2;
      }
    }
  }
}

template <typename T>
void GroupNorm1dForward(
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t N,
    int64_t C,
    int64_t group,
    Tensor& Y) {
  using T_ACC = acc_type<T, true>;
  const int64_t G = group;
  const int64_t D = C / G;
  if (gamma.defined() && beta.defined()) {
    auto iter = TensorIteratorConfig()
                    .resize_outputs(false)
                    .add_owned_output(Y.view({N, G, D}))
                    .add_owned_const_input(X.view({N, G, D}))
                    .add_owned_input(mean.view({N, G, 1}))
                    .add_owned_input(rstd.view({N, G, 1}))
                    .add_owned_const_input(gamma.view({1, G, D}))
                    .add_owned_const_input(beta.view({1, G, D}))
                    .build();
    gpu_kernel(iter, [] GPU_LAMBDA(T x, T mean, T rstd, T gamma, T beta) -> T {
      return (static_cast<T_ACC>(x) - static_cast<T_ACC>(mean)) *
          static_cast<T_ACC>(rstd) * static_cast<T_ACC>(gamma) +
          static_cast<T_ACC>(beta);
    });
  } else if (gamma.defined()) {
    auto iter = TensorIteratorConfig()
                    .resize_outputs(false)
                    .add_owned_output(Y.view({N, G, D}))
                    .add_owned_const_input(X.view({N, G, D}))
                    .add_owned_input(mean.view({N, G, 1}))
                    .add_owned_input(rstd.view({N, G, 1}))
                    .add_owned_const_input(gamma.view({1, G, D}))
                    .build();
    gpu_kernel(iter, [] GPU_LAMBDA(T x, T mean, T rstd, T gamma) -> T {
      return (static_cast<T_ACC>(x) - static_cast<T_ACC>(mean)) *
          static_cast<T_ACC>(rstd) * static_cast<T_ACC>(gamma);
    });
  } else if (beta.defined()) {
    auto iter = TensorIteratorConfig()
                    .resize_outputs(false)
                    .add_owned_output(Y.view({N, G, D}))
                    .add_owned_const_input(X.view({N, G, D}))
                    .add_owned_input(mean.view({N, G, 1}))
                    .add_owned_input(rstd.view({N, G, 1}))
                    .add_owned_const_input(beta.view({1, G, D}))
                    .build();
    gpu_kernel(iter, [] GPU_LAMBDA(T x, T mean, T rstd, T beta) -> T {
      return (static_cast<T_ACC>(x) - static_cast<T_ACC>(mean)) *
          static_cast<T_ACC>(rstd) +
          static_cast<T_ACC>(beta);
    });
  } else {
    auto iter = TensorIteratorConfig()
                    .resize_outputs(false)
                    .add_owned_output(Y.view({N * G, D}))
                    .add_owned_const_input(X.view({N * G, D}))
                    .add_owned_input(mean.view({N * G, 1}))
                    .add_owned_input(rstd.view({N * G, 1}))
                    .build();
    gpu_kernel(iter, [] GPU_LAMBDA(T x, T mean, T rstd) -> T {
      return (static_cast<T_ACC>(x) - static_cast<T_ACC>(mean)) *
          static_cast<T_ACC>(rstd);
    });
  }
  AT_CUDA_CHECK(hipGetLastError());
}

template <typename T>
void GroupNormKernelImplInternal(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t N,
    int64_t C,
    int64_t HxW,
    int64_t group,
    T eps,
    Tensor& Y,
    Tensor& mean,
    Tensor& rstd) {
  using T_ACC = acc_type<T, true>;
  TORCH_CHECK(X.numel() == N * C * HxW);
  TORCH_CHECK(!gamma.defined() || gamma.numel() == C);
  TORCH_CHECK(!beta.defined() || beta.numel() == C);
  if (N == 0) {
    return;
  }
  const int64_t G = group;
  const int64_t D = C / G;
  const T* X_data = X.const_data_ptr<T>();
  T* mean_data = mean.mutable_data_ptr<T>();
  T* rstd_data = rstd.mutable_data_ptr<T>();

  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  const int64_t num_threads = D * HxW < cuda_utils::kCUDABlockReduceNumThreads
      ? at::cuda::warp_size()
      : cuda_utils::kCUDABlockReduceNumThreads;
  RowwiseMomentsCUDAKernel<T><<<N * G, num_threads, 0, hip_stream>>>(
      D * HxW, eps, X_data, mean_data, rstd_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  if (HxW == 1) {
    GroupNorm1dForward<T>(X, mean, rstd, gamma, beta, N, C, G, Y);
  } else if (!gamma.defined() && !beta.defined()) {
    auto iter = TensorIteratorConfig()
                    .resize_outputs(false)
                    .add_owned_output(Y.view({N * G, D * HxW}))
                    .add_owned_const_input(X.view({N * G, D * HxW}))
                    .add_owned_input(mean.view({N * G, 1}))
                    .add_owned_input(rstd.view({N * G, 1}))
                    .build();
    gpu_kernel(iter, [] GPU_LAMBDA(T x, T mean, T rstd) -> T {
      return (static_cast<T_ACC>(x) - static_cast<T_ACC>(mean)) *
          static_cast<T_ACC>(rstd);
    });
  } else {
    const auto kAccType =
        (X.scalar_type() == kHalf || X.scalar_type() == kBFloat16)
        ? kFloat
        : X.scalar_type();
    Tensor a = at::empty({N, C}, X.options().dtype(kAccType));
    Tensor b = at::empty({N, C}, X.options().dtype(kAccType));
    const T* gamma_data = gamma.defined() ? gamma.const_data_ptr<T>() : nullptr;
    const T* beta_data = beta.defined() ? beta.const_data_ptr<T>() : nullptr;
    T_ACC* a_data = a.mutable_data_ptr<T_ACC>();
    T_ACC* b_data = b.mutable_data_ptr<T_ACC>();

    // TODO: Since there is some issues in gpu_kernel_multiple_outputs, we are
    // using manual kernel here. Make it using gpu_kernel_multiple_outputs once
    // the issue fixed.
    const int64_t B = (N * C + kCUDANumThreads - 1) / kCUDANumThreads;
    ComputeFusedParamsCUDAKernel<T><<<B, kCUDANumThreads, 0, hip_stream>>>(
        N, C, G, mean_data, rstd_data, gamma_data, beta_data, a_data, b_data);
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    auto iter = TensorIteratorConfig()
                    .check_all_same_dtype(std::is_same_v<T, T_ACC>)
                    .resize_outputs(false)
                    .add_owned_output(Y.view({N * C, HxW}))
                    .add_owned_const_input(X.view({N * C, HxW}))
                    .add_owned_input(a.view({N * C, 1}))
                    .add_owned_input(b.view({N * C, 1}))
                    .build();
    gpu_kernel(iter, [] GPU_LAMBDA(T x, T_ACC a, T_ACC b) -> T {
      return a * static_cast<T_ACC>(x) + b;
    });
  }
  AT_CUDA_CHECK(hipGetLastError());
}

void GroupNormKernelImpl(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t N,
    int64_t C,
    int64_t HxW,
    int64_t group,
    double eps,
    Tensor& Y,
    Tensor& mean,
    Tensor& rstd) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      X.scalar_type(),
      "GroupNormKernelImpl",
      [&]() {
        GroupNormKernelImplInternal<scalar_t>(
            X,
            gamma,
            beta,
            N,
            C,
            HxW,
            group,
            static_cast<scalar_t>(eps),
            Y,
            mean,
            rstd);
      });
}

template <typename T>
void GroupNorm1dBackward(
    const Tensor dY,
    const Tensor X,
    const Tensor mean,
    const Tensor rstd,
    const Tensor gamma,
    int64_t N,
    int64_t C,
    int64_t group,
    Tensor& dX,
    Tensor& dgamma,
    Tensor& dbeta) {
  using T_ACC = acc_type<T, true>;
  const int64_t G = group;
  const int64_t D = C / G;
  const T* dY_data = dY.const_data_ptr<T>();
  const T* X_data = X.const_data_ptr<T>();
  const T* mean_data = mean.const_data_ptr<T>();
  const T* rstd_data = rstd.const_data_ptr<T>();

  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  if (dX.defined()) {
    const T* gamma_data = gamma.defined() ? gamma.const_data_ptr<T>() : nullptr;
    const auto kAccType =
        (X.scalar_type() == kHalf || X.scalar_type() == kBFloat16)
        ? kFloat
        : X.scalar_type();
    Tensor c2 = at::empty({N, G}, X.options().dtype(kAccType));
    Tensor c3 = at::empty({N, G}, X.options().dtype(kAccType));
    T_ACC* c2_data = c2.mutable_data_ptr<T_ACC>();
    T_ACC* c3_data = c3.mutable_data_ptr<T_ACC>();
    const int64_t num_threads = (C / G) < cuda_utils::kCUDABlockReduceNumThreads
        ? at::cuda::warp_size()
        : cuda_utils::kCUDABlockReduceNumThreads;
    Compute1dBackwardFusedParamsCUDAKernel<T>
        <<<dim3(N, G), num_threads, 0, hip_stream>>>(
            C,
            G,
            dY_data,
            X_data,
            mean_data,
            rstd_data,
            gamma_data,
            c2_data,
            c3_data);
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    if (gamma.defined()) {
      auto iter = TensorIteratorConfig()
                      .check_all_same_dtype(std::is_same_v<T, T_ACC>)
                      .resize_outputs(false)
                      .add_owned_output(dX.view({N, G, D}))
                      .add_owned_const_input(dY.view({N, G, D}))
                      .add_owned_const_input(X.view({N, G, D}))
                      .add_owned_const_input(rstd.view({N, G, 1}))
                      .add_owned_const_input(gamma.view({1, G, D}))
                      .add_owned_const_input(c2.view({N, G, 1}))
                      .add_owned_const_input(c3.view({N, G, 1}))
                      .build();
      gpu_kernel(
          iter,
          [] GPU_LAMBDA(T dy, T x, T rstd, T gamma, T_ACC c2, T_ACC c3) -> T {
            const T_ACC c1 =
                static_cast<T_ACC>(rstd) * static_cast<T_ACC>(gamma);
            return c1 * static_cast<T_ACC>(dy) + c2 * static_cast<T_ACC>(x) +
                c3;
          });
    } else {
      auto iter = TensorIteratorConfig()
                      .check_all_same_dtype(std::is_same_v<T, T_ACC>)
                      .resize_outputs(false)
                      .add_owned_output(dX.view({N * G, D}))
                      .add_owned_const_input(dY.view({N * G, D}))
                      .add_owned_const_input(X.view({N * G, D}))
                      .add_owned_const_input(rstd.view({N * G, 1}))
                      .add_owned_const_input(c2.view({N * G, 1}))
                      .add_owned_const_input(c3.view({N * G, 1}))
                      .build();
      gpu_kernel(
          iter, [] GPU_LAMBDA(T dy, T x, T rstd, T_ACC c2, T_ACC c3) -> T {
            const T_ACC c1 = static_cast<T_ACC>(rstd);
            return c1 * static_cast<T_ACC>(dy) + c2 * static_cast<T_ACC>(x) +
                c3;
          });
    }
  }
  if (dgamma.defined() || dbeta.defined()) {
    T* dgamma_data = dgamma.defined() ? dgamma.mutable_data_ptr<T>() : nullptr;
    T* dbeta_data = dbeta.defined() ? dbeta.mutable_data_ptr<T>() : nullptr;
    if (N <= 128) {
      const int64_t B = (C + kCUDANumThreads - 1) / kCUDANumThreads;
      GammaBeta1dBackwardCUDAKernel1<T><<<B, kCUDANumThreads, 0, hip_stream>>>(
          N,
          C,
          G,
          dY_data,
          X_data,
          mean_data,
          rstd_data,
          dgamma_data,
          dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      const int64_t B = (C + kReduceTileSize - 1) / kReduceTileSize;
      // The algorithm for colwise reduction here is to accumulate each 32 cols
      // to a 32 * 32 tile and write the tile to shared memory. Then do warp
      // reduce for each col in the tile. So here the blockDim must be (32, 16).
      constexpr int kThreadX = kReduceTileSize;
      constexpr int kThreadY = kReduceTileSize / 2;
      GammaBeta1dBackwardCUDAKernel2<T>
          <<<B, dim3(kThreadX, kThreadY), 0, hip_stream>>>(
              N,
              C,
              G,
              dY_data,
              X_data,
              mean_data,
              rstd_data,
              dgamma_data,
              dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
}

template <typename T>
void GroupNormBackwardKernelImplInternal(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t N,
    int64_t C,
    int64_t HxW,
    int64_t group,
    Tensor& dX,
    Tensor& dgamma,
    Tensor& dbeta) {
  using T_ACC = acc_type<T, true>;
  const int64_t G = group;
  const int64_t D = C / G;
  TORCH_CHECK(dY.numel() == N * C * HxW);
  TORCH_CHECK(X.numel() == N * C * HxW);
  TORCH_CHECK(mean.numel() == N * G);
  TORCH_CHECK(rstd.numel() == N * G);
  TORCH_CHECK(!gamma.defined() || gamma.numel() == C);
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();

  if (N == 0) {
    if (dgamma.defined()) {
      dgamma.fill_(T(0));
    }
    if (dbeta.defined()) {
      dbeta.fill_(T(0));
    }
    return;
  }

  const T* dY_data = dY.const_data_ptr<T>();
  const T* X_data = X.const_data_ptr<T>();
  const T* mean_data = mean.const_data_ptr<T>();
  const T* rstd_data = rstd.const_data_ptr<T>();
  const T* gamma_data = gamma.defined() ? gamma.const_data_ptr<T>() : nullptr;
  const auto kAccType =
      (X.scalar_type() == kHalf || X.scalar_type() == kBFloat16)
      ? kFloat
      : X.scalar_type();
  Tensor ds = at::empty({N, C}, X.options().dtype(kAccType));
  Tensor db = at::empty({N, C}, X.options().dtype(kAccType));
  T_ACC* ds_data = ds.mutable_data_ptr<T_ACC>();
  T_ACC* db_data = db.mutable_data_ptr<T_ACC>();

  if (HxW == 1) {
    GroupNorm1dBackward<T>(
        dY, X, mean, rstd, gamma, N, C, G, dX, dgamma, dbeta);
    return;
  }

  int warp_size = at::cuda::warp_size();
  int64_t num_threads = HxW < cuda_utils::kCUDABlockReduceNumThreads
      ? warp_size
      : cuda_utils::kCUDABlockReduceNumThreads;
  ComputeInternalGradientsCUDAKernel<T><<<N * C, num_threads, 0, hip_stream>>>(
      HxW, dY_data, X_data, ds_data, db_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  if (dX.defined()) {
    Tensor c1 = at::empty({0}, X.options().dtype(kAccType));
    Tensor c2 = at::empty({N, G}, X.options().dtype(kAccType));
    Tensor c3 = at::empty({N, G}, X.options().dtype(kAccType));
    T_ACC* c2_data = c2.mutable_data_ptr<T_ACC>();
    T_ACC* c3_data = c3.mutable_data_ptr<T_ACC>();

    if (gamma.defined()) {
      auto iter = TensorIteratorConfig()
                      .check_all_same_dtype(std::is_same_v<T, T_ACC>)
                      .add_output(c1)
                      .add_owned_const_input(rstd.view({N, G, 1}))
                      .add_owned_const_input(gamma.view({1, G, D}))
                      .build();
      gpu_kernel(iter, [] GPU_LAMBDA(T rstd, T gamma) -> T_ACC {
        return static_cast<T_ACC>(rstd) * static_cast<T_ACC>(gamma);
      });
    }

    num_threads = (C / G) < cuda_utils::kCUDABlockReduceNumThreads
        ? warp_size
        : cuda_utils::kCUDABlockReduceNumThreads;
    ComputeBackwardFusedParamsCUDAKernel<T>
        <<<dim3(N, G), num_threads, 0, hip_stream>>>(
            C,
            HxW,
            G,
            mean_data,
            rstd_data,
            gamma_data,
            ds_data,
            db_data,
            c2_data,
            c3_data);
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    if (gamma.defined()) {
      auto iter = TensorIteratorConfig()
                      .check_all_same_dtype(std::is_same_v<T, T_ACC>)
                      .resize_outputs(false)
                      .add_owned_output(dX.view({N * G, D, HxW}))
                      .add_owned_const_input(dY.view({N * G, D, HxW}))
                      .add_owned_const_input(X.view({N * G, D, HxW}))
                      .add_owned_const_input(c1.view({N * G, D, 1}))
                      .add_owned_const_input(c2.view({N * G, 1, 1}))
                      .add_owned_const_input(c3.view({N * G, 1, 1}))
                      .build();
      gpu_kernel(
          iter, [] GPU_LAMBDA(T dy, T x, T_ACC c1, T_ACC c2, T_ACC c3) -> T {
            return c1 * static_cast<T_ACC>(dy) + c2 * static_cast<T_ACC>(x) +
                c3;
          });
    } else {
      auto iter = TensorIteratorConfig()
                      .check_all_same_dtype(std::is_same_v<T, T_ACC>)
                      .resize_outputs(false)
                      .add_owned_output(dX.view({N * G, D * HxW}))
                      .add_owned_const_input(dY.view({N * G, D * HxW}))
                      .add_owned_const_input(X.view({N * G, D * HxW}))
                      .add_owned_const_input(rstd.view({N * G, 1}))
                      .add_owned_const_input(c2.view({N * G, 1}))
                      .add_owned_const_input(c3.view({N * G, 1}))
                      .build();
      gpu_kernel(
          iter, [] GPU_LAMBDA(T dy, T x, T_ACC c1, T_ACC c2, T_ACC c3) -> T {
            return c1 * static_cast<T_ACC>(dy) + c2 * static_cast<T_ACC>(x) +
                c3;
          });
    }
  }
  if (dgamma.defined() || dbeta.defined()) {
    T* dgamma_data = dgamma.defined() ? dgamma.mutable_data_ptr<T>() : nullptr;
    T* dbeta_data = dbeta.defined() ? dbeta.mutable_data_ptr<T>() : nullptr;
    if (N <= 128) {
      // For small batch size, do colwise reduce directly.
      const int64_t B = (C + kCUDANumThreads - 1) / kCUDANumThreads;
      GammaBetaBackwardCUDAKernel1<T><<<B, kCUDANumThreads, 0, hip_stream>>>(
          N,
          C,
          G,
          mean_data,
          rstd_data,
          ds_data,
          db_data,
          dgamma_data,
          dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      const int64_t B = (C + kReduceTileSize - 1) / kReduceTileSize;
      // The algorithm for colwise reduction here is to accumulate each 32 cols
      // to a 32 * 32 tile and write the tile to shared memory. Then do warp
      // reduce for each col in the tile. So here the blockDim must be (32, 16).
      constexpr int kThreadX = kReduceTileSize;
      constexpr int kThreadY = kReduceTileSize / 2;
      GammaBetaBackwardCUDAKernel2<T>
          <<<B, dim3(kThreadX, kThreadY), 0, hip_stream>>>(
              N,
              C,
              G,
              mean_data,
              rstd_data,
              ds_data,
              db_data,
              dgamma_data,
              dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
}

void GroupNormBackwardKernelImpl(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t N,
    int64_t C,
    int64_t HxW,
    int64_t group,
    Tensor& dX,
    Tensor& dgamma,
    Tensor& dbeta) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      X.scalar_type(),
      "GroupNormBackwardKernelImpl",
      [&]() {
        GroupNormBackwardKernelImplInternal<scalar_t>(
            dY, X, mean, rstd, gamma, N, C, HxW, group, dX, dgamma, dbeta);
      });
}

} // namespace

REGISTER_DISPATCH(GroupNormKernel, &GroupNormKernelImpl)
REGISTER_DISPATCH(GroupNormBackwardKernel, &GroupNormBackwardKernelImpl)

} // namespace at::native
