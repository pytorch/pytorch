#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/EmptyTensor.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/native/cuda/Resize.h>
#include <ATen/native/TensorFactories.h>
#include <c10/util/accumulate.h>
#include <c10/util/Exception.h>
#include <ATen/native/cuda/Loops.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_efficientzerotensor_native.h>
#include <ATen/ops/empty_native.h>
#include <ATen/ops/empty_strided_native.h>
#include <ATen/ops/eye_native.h>
#include <ATen/ops/tril_indices_native.h>
#include <ATen/ops/tril_native.h>
#include <ATen/ops/triu_indices_native.h>
#include <ATen/ops/triu_native.h>
#endif

#include <algorithm>
#include <cmath>
#include <cstddef>

namespace at::native {

Tensor& eye_out_cuda(int64_t n, Tensor& result) {
  // the default value of `m` equals to `n`
  return at::native::eye_out_cuda(n, n, result);
}

Tensor& eye_out_cuda(int64_t n, int64_t m, Tensor& result) {
  TORCH_CHECK(n >= 0, "n must be greater or equal to 0, got ", n);
  TORCH_CHECK(m >= 0, "m must be greater or equal to 0, got ", m);

  result.resize_({n, m});
  result.zero_();

  int64_t sz = std::min<int64_t>(n, m);
  int64_t stride = result.stride(0) + result.stride(1);

  Tensor diag = result.as_strided({sz}, {stride});
  diag.fill_(1);
  return result;
}

Tensor empty_cuda(IntArrayRef size, std::optional<ScalarType> dtype_opt, std::optional<Layout> layout_opt, std::optional<Device> device_opt, std::optional<bool> pin_memory_opt, std::optional<c10::MemoryFormat> memory_format_opt) {
  Tensor result = at::detail::empty_cuda(size, dtype_opt, layout_opt, device_opt, pin_memory_opt, memory_format_opt);
  // See Note [Enabling Deterministic Operations]
  if (C10_UNLIKELY(at::globalContext().deterministicAlgorithms() && at::globalContext().deterministicFillUninitializedMemory())) {
    fill_empty_deterministic_(result);
  }
  return result;
}

Tensor _efficientzerotensor_cuda(IntArrayRef size,
    std::optional<ScalarType> dtype,
    std::optional<Layout> layout,
    std::optional<Device> device,
    std::optional<bool> pin_memory) {
    auto device_ = device_or_default(device);
    if (!device_.has_index()) {
      device_.set_index(at::cuda::current_device());
    }
    auto allocator = at::native::ZeroTensorAllocator(device_);
    auto dtype_ = dtype_or_default(dtype);
    auto zero_ks = at::DispatchKeySet(c10::DispatchKey::CUDA) | at::DispatchKeySet(c10::DispatchKey::ZeroTensor);
    auto out = at::detail::empty_generic(size, &allocator, zero_ks, dtype_, std::nullopt);
    return out;
}


Tensor empty_strided_cuda(IntArrayRef size, IntArrayRef stride, std::optional<ScalarType> dtype_opt, std::optional<Layout> layout_opt, std::optional<Device> device_opt, std::optional<bool> pin_memory_opt) {
  Tensor result = at::detail::empty_strided_cuda(size, stride, dtype_opt, layout_opt, device_opt, pin_memory_opt);
  // See Note [Enabling Deterministic Operations]
  if (C10_UNLIKELY(at::globalContext().deterministicAlgorithms() && at::globalContext().deterministicFillUninitializedMemory())) {
    fill_empty_deterministic_(result);
  }
  return result;
}

// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ triangle ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

namespace {
// To find the max integer that does not exceed the root of an int64_t variable,
// we could use a loop to test one bit at a time, which takes up to 31
// iterations. This would give the accurate result, but is relatively slow and
// is an overkill for most cases where double's precision suffice.
//
// If we directly use sqrt to calculate the root, the conversion from int64_t
// to double would lose 11 bits precision.
//
// The following solution uses sqrt directly for most cases, and would only
// special handle it if there is indeed precision loss.
__device__
inline int64_t resolve_root_int(
    int64_t b, int64_t cX4, int64_t x, int32_t sign) {
  int64_t bXb_cX4 = b*b - cX4;
  // potential precision loss could occur here when casting int64_t (63 bits
  // precision) to double (52 bits precision)
  double sr = ::sqrt((double)bXb_cX4);
  int64_t res = ::__double2ll_rd((-b + sign * sr)/2);

  // have to cast double to int64_t, otherwise it would only compare up to the
  // precision of a double variable, ignoring the precision loss
  if (bXb_cX4 != (int64_t) (sr * sr)) {
    // handle precision loss by using binary search
    int64_t llsr = ::__double2ll_rd(sr);
    // Use the following math to reduce search space.
    // Suppose z is the accurate result of sqrt(bXb_cX4) without precision loss
    // let d = abs(bXb_cX4 - llsr * llsr), then we have:
    // z = sqrt(bXb_cX4) <= sqrt(llsr * llsr + d) <= llsr + sqrt(d)
    // z = sqrt(bXb_cX4) >= sqrt(llsr * llsr - d) >= llsr - sqrt(d)
    // Hence, it is sufficient to search range [llsr - sqrt(d), llsr + sqrt(d)).
    // And the true value of row would also be with in range,
    //            [res - sqrt(d), res + sqrt(d) + 1)
    // as the denominator would only reduce the precision penalty.
    int64_t diff =
      ::__double2ll_ru(::sqrt(::fabs((double)(bXb_cX4 - llsr * llsr))));
    // l never exceeds (could equal to) the target row index
    auto l = res > diff ? res - diff : 0;
    // r is always larger than the target row index
    auto r = res + diff + 1;

    // binary search for the correct answer
    x <<= 1; // the loop always compares with 2x, so do it once here
    while (l + 1 < r) {
      auto m = (l + r) >> 1;
      // for tril:
      //    b = 2f - 1, sign = 1, hence (2f + m - 1) * m / 2
      // for triu:
      //    b = -2f - 1, sign = -1, hence (2f - m + 1) * m / 2
      if (sign * (b + m) * m > x) {
        r = m;
      } else {
        l = m;
      }
    }
    res = l;
  }

  return res;
}

// f: the number of elements in the first row of the trapezoid.
// x: the index of the target coordinates ordered by row and then column.
//
// View the tril as a top trapezoid stacked on a bottom rectangle. Assume x
// corresponds to the coordinate (row, col) in the trapezoid, where the row and
// the col both start from 0, then we have:
//
//                   (f + f + row - 1) * row / 2 <= x                       [1]
//                 (f + f + row) * (row + 1) / 2  > x                       [2]
//
// Therefore, row is the maximum integer satisfying the following inequality:
//
//                       (row + 2f - 1)row <= 2x
//                  row^2 + (2f-1)row - 2x <= 0.                            [3]
//
// Based on inequality [3], we have the following coefficients for formula of
// root:
//                               a = 1
//                               b = 2f - 1
//                               c = -2x
// There are two roots, and we should use the largest integer that does not
// exceed the root on the right. Intuitively, it is because:
//  i)  the valid solution range of row is between two roots, as it is <= 0;
//  ii) as we count in more rows, the total # of elements should always
//      increase, hence so does the left-hand side row^2 + (2f-1)row - 2x.
//      Therefore, the valid range of row lies in between the nadir point and
//      the larger root on the right.
// Full proof can be derived from inequality [2]. So, we calculate the result
// coordinate as:
//
//                   row = floor((-b + sqrt(b^2 - 4c)) / 2)
//                   col = x - (f + f + row - 1) * row / 2
__device__
inline void get_coordinate_in_tril_trapezoid(
    int64_t f, int64_t x, int64_t & row, int64_t & col) {
  f <<= 1; // all statements use 2f, so only calculate it once here.
  auto b = f - 1;
  auto cX4 = - (x << 3); // 4 * c = 4 * (-2x) = -8x;
  row = resolve_root_int(b, cX4, x, 1);
  col = x - ((f + row - 1) * row >> 1);
}

// f: the number of elements in the first row of the bottom trapezoid.
// x: the index of the target coordinates ordered by row and then column.
//
// View the triu as a top rectangle stacked on a bottom trapezoid, where the
// trapezoid is upside down. Assume x corresponds to the coordinate (row, col)
// in the bottom trapezoid, where the row and the col start from 0, then we
// have:
//
//                   (f + f - row + 1) * row / 2 <= x                       [1]
//                 (f + f - row) * (row + 1) / 2  > x                       [2]
//
// Therefore, row is the maximum integer satisfying the following inequality:
//
//                       (-row + 2f + 1)row <= 2x
//                   row^2 - (2f+1)row + 2x >= 0.                           [3]
//
// Based on inequality [3], we have the following coefficients for formula of
// root:
//                               a = 1
//                               b = -1 - 2f
//                               c = 2x
// There are two roots, and we should use the largest integer that does not
// exceed the root on the left. Intuitively, it is because:
//  i)  the valid solution range of row is outside of the two roots, as it is <
//      > 0;
//  ii) as we count in more rows, the total # of elements should always
//      increase, hence so does the left-hand side row^2 - (2f+1)row + 2x.
//      Therefore, the valid range of row lies to the left of the smaller root
//      on the left.
// Full proof can be derived from inequality [2]. So, we calculate the result
// coordinate as:
//
//                   row = floor((-b - sqrt(b^2 - 4c)) / 2)
//                   col = x - (f + f - row + 1) * row / 2
__device__
inline void get_coordinate_in_triu_trapezoid(
    int64_t f, int64_t x, int64_t & row, int64_t & col) {
  f <<= 1; // all statements use 2f, so only calculate it once here.
  auto b = -1 - f;
  auto cX4 = x << 3; // 4 * c = 4 * (2x) = 8x;
  row = resolve_root_int(b, cX4, x, -1);
  col = x - ((f - row + 1) * row >> 1) + row;
}

} // namespace

template <typename scalar_t>
__global__
#if defined(USE_ROCM)
C10_LAUNCH_BOUNDS_1(512)
#endif
void tril_indices_kernel(scalar_t * tensor,
                         int64_t row_offset,
                         int64_t m_first_row,
                         int64_t col,
                         int64_t trapezoid_size,
                         int64_t tril_size) {
  int64_t linear_index = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;

  if (linear_index < tril_size) {
    int64_t r, c;
    if (linear_index < trapezoid_size) {
      // the coordinate is within the top trapezoid
      get_coordinate_in_tril_trapezoid(m_first_row, linear_index, r, c);
    } else {
      // the coordinate falls in the bottom rectangle
      auto surplus = linear_index - trapezoid_size;
      // add the height of trapezoid: m_last_row (col) - m_first_row + 1
      r = surplus / col + col - m_first_row + 1;
      c = surplus % col;
    }
    r += row_offset;

    tensor[linear_index] = r;
    tensor[linear_index + tril_size] = c;
  }
}

// Some Large test cases for the fallback binary search path is disabled by
// default to speed up CI tests and to avoid OOM error. When modifying the
// implementation, please enable them in test/test_cuda.py and make sure they
// pass on your local server.
Tensor tril_indices_cuda(
    int64_t row, int64_t col, int64_t offset, std::optional<ScalarType> dtype_opt,
    std::optional<Layout> layout_opt, std::optional<Device> device_opt, std::optional<bool> pin_memory_opt) {
  check_args(row, col, layout_opt);

  auto tril_size = get_tril_size(row, col, offset);
  auto tensor = empty_cuda({2, tril_size}, dtype_opt, layout_opt, device_opt, pin_memory_opt);

  if (tril_size > 0) {
    auto m_first_row = offset > 0 ?
      std::min<int64_t>(col, 1 + offset) : // upper bounded by col
      row + offset > 0; // either 0 or 1
    auto trapezoid_row_offset = std::max<int64_t>(0, -offset);
    auto rectangle_row_offset = trapezoid_row_offset + col - m_first_row + 1;
    int64_t rectangle_size = 0;
    if (rectangle_row_offset < row) {
      rectangle_size = (row - rectangle_row_offset) * col;
    }

    dim3 dim_block = cuda::getApplyBlock();
    dim3 dim_grid;
    // using tril_size instead of tensor.numel(), as each thread takes care of
    // two elements in the tensor.
    TORCH_CHECK(
      cuda::getApplyGrid(tril_size, dim_grid, tensor.get_device()),
      "unable to get dim grid");

    AT_DISPATCH_INDEX_TYPES(tensor.scalar_type(), "tril_indices_cuda", [&] {
      tril_indices_kernel<<<
          dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
        tensor.mutable_data_ptr<index_t>(),
        trapezoid_row_offset,
        m_first_row,
        col,
        tril_size - rectangle_size,
        tril_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  }

  return tensor;
}

template <typename scalar_t>
__global__
void triu_indices_kernel(scalar_t * tensor,
                         int64_t col_offset,
                         int64_t m_first_row,
                         int64_t col,
                         int64_t rectangle_size,
                         int64_t triu_size) {
  int64_t linear_index = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;

  if (linear_index < triu_size) {
    int64_t r, c;
    if (linear_index < rectangle_size) {
      // the coordinate is within the top rectangle
      r = linear_index / col;
      c = linear_index % col;
    } else {
      // the coordinate falls in the bottom trapezoid
      get_coordinate_in_triu_trapezoid(
        m_first_row, linear_index - rectangle_size, r, c);
      r += rectangle_size / col;
    }

    c += col_offset;
    tensor[linear_index] = r;
    tensor[linear_index + triu_size] = c;
  }
}

// Some Large test cases for the fallback binary search path is disabled by
// default to speed up CI tests and to avoid OOM error. When modifying the
// implementation, please enable them in test/test_cuda.py and make sure they
// pass on your local server.
Tensor triu_indices_cuda(
    int64_t row, int64_t col, int64_t offset, std::optional<ScalarType> dtype_opt,
    std::optional<Layout> layout_opt, std::optional<Device> device_opt, std::optional<bool> pin_memory_opt) {
  check_args(row, col, layout_opt);

  auto triu_size = row * col - get_tril_size(row, col, offset - 1);
  auto tensor = empty_cuda({2, triu_size}, dtype_opt, layout_opt, device_opt, pin_memory_opt);

  if (triu_size > 0) {
    // # of triu elements in the first row
    auto m_first_row = offset > 0 ?
      std::max<int64_t>(col - offset, 0) : // upper bounded by col
      col;

    // size of the top rectangle
    int64_t rectangle_size = 0;
    if (offset < 0) {
      rectangle_size = std::min<int64_t>(row, -offset) * col;
    }

    dim3 dim_block = cuda::getApplyBlock();
    dim3 dim_grid;

    // using triu_size instead of tensor.numel(), as each thread takes care of
    // two elements in the tensor.
    TORCH_CHECK(
      cuda::getApplyGrid(triu_size, dim_grid, tensor.get_device()),
      "unable to get dim grid");

    AT_DISPATCH_INDEX_TYPES(tensor.scalar_type(), "triu_indices_cuda", [&] {
      triu_indices_kernel<<<
          dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
        tensor.mutable_data_ptr<index_t>(),
        std::max<int64_t>(0, offset),
        m_first_row,
        col,
        rectangle_size,
        triu_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  }

  return tensor;
}

} // namespace at::native
