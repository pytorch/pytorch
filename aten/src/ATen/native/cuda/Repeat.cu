#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/Repeat.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/repeat_interleave_native.h>
#endif

template <typename index_t>
__global__ static void compute_cuda_kernel(
    const index_t* repeat_ptr,
    const int64_t* cumsum_ptr,
    index_t* result_ptr,
    int64_t size,
    int64_t result_size) {
  if (C10_UNLIKELY((result_size != cumsum_ptr[size - 1]))) {
    printf("%s:%d:%s: block: [%d,%d,%d], thread: [%d,%d,%d] "
      "Invalid input! In `repeat_interleave`, the `output_size` argument (%ld) must be the same as the sum of the elements in the `repeats` tensor (%ld).\n",
      __FILE__, __LINE__, __func__,blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, result_size, cumsum_ptr[size - 1 ]);
    CUDA_KERNEL_ASSERT(result_size == cumsum_ptr[size - 1])
  }

  int64_t idx = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  int64_t stride = (blockDim.x * gridDim.x) / C10_WARP_SIZE;
  int warp_id = idx / C10_WARP_SIZE;
  int tid_in_warp = idx % C10_WARP_SIZE;
  for (int64_t i = warp_id; i < size; i += stride) {
    int64_t end = cumsum_ptr[i];
    index_t repeat = repeat_ptr[i];
    CUDA_KERNEL_ASSERT(repeat >= 0);
    int64_t start = end - repeat;
    for (int64_t j = start + tid_in_warp; j < end; j += C10_WARP_SIZE) {
      result_ptr[j] = i;
    }
  }
}

template <typename index_t>
static void compute_cuda(
    const index_t* repeat_ptr,
    const int64_t* cumsum_ptr,
    index_t* result_ptr,
    int64_t size,
    int64_t result_size) {
  int64_t block = 512;
  int64_t warps_per_block = block / at::cuda::warp_size();
  int64_t grid =
      std::min<int64_t>((size + warps_per_block - 1) / warps_per_block, 2048L);

  compute_cuda_kernel<<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
      repeat_ptr, cumsum_ptr, result_ptr, size, result_size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

namespace at::native {

Tensor repeat_interleave_cuda(
    const Tensor& repeat,
    std::optional<int64_t> output_size) {
  Tensor output;
  AT_DISPATCH_INDEX_TYPES(
      repeat.scalar_type(), "repeat_interleave_cuda", [&]() {
        output = repeat_interleave_common<index_t, compute_cuda<index_t>>(
            repeat, output_size);
      });
  return output;
}

} // namespace at::native
