#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/native/cuda/UpSample.cuh>

#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/upsample_nearest3d.h>
#include <ATen/ops/upsample_nearest3d_native.h>
#include <ATen/ops/upsample_nearest3d_backward.h>
#include <ATen/ops/upsample_nearest3d_backward_native.h>
#include <ATen/ops/_upsample_nearest_exact3d.h>
#include <ATen/ops/_upsample_nearest_exact3d_native.h>
#include <ATen/ops/_upsample_nearest_exact3d_backward.h>
#include <ATen/ops/_upsample_nearest_exact3d_backward_native.h>
#endif

namespace at::native {
namespace {

#define MAX_THREADS 512

// Define a typedef to dispatch to nearest_neighbor_compute_source_index or
// nearest_neighbor_exact_compute_source_index
typedef int (*nn_compute_source_index_fn_t)(const float, int, int);

// Define a typedef to dispatch to nearest_neighbor_bw_compute_source_index or
// nearest_neighbor_exact_bw_compute_source_index
typedef int (*nn_bw_compute_source_index_fn_t)(const float, int, int);

// see NOTE [ Nearest neighbor upsampling kernel implementation ]
template <typename scalar_t, nn_compute_source_index_fn_t nn_compute_source_index_fn>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest3d_out_frame(
    const scalar_t* input,
    size_t dim_b,
    size_t dim_c,
    size_t src_dim_d,
    size_t src_dim_h,
    size_t src_dim_w,
    size_t dst_dim_d,
    size_t dst_dim_h,
    size_t dst_dim_w,
    scalar_t* output,
    float depth_scale,
    float height_scale,
    float width_scale) {

  int64_t dst_idx = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (dst_idx >= dim_c * dst_dim_d * dst_dim_h * dst_dim_w)
    return;

  int64_t dst_c_stride = dst_dim_d * dst_dim_h * dst_dim_w;
  int64_t src_c_stride = src_dim_d * src_dim_h * src_dim_w;

  int c = (dst_idx / (dst_c_stride)) % dim_c;

  int dst_z = (dst_idx / dst_dim_h / dst_dim_w) % dst_dim_d;
  int src_z = nn_compute_source_index_fn(depth_scale, dst_z, src_dim_d);
  int dst_y = (dst_idx / dst_dim_w) % dst_dim_h;
  int src_y = nn_compute_source_index_fn(height_scale, dst_y, src_dim_h);

  int dst_x = dst_idx % dst_dim_w;
  int src_x = nn_compute_source_index_fn(width_scale, dst_x, src_dim_w);

  int64_t src_idx = c * src_c_stride + src_z * src_dim_h * src_dim_w +
      src_y * src_dim_w + src_x;
  for (int b = 0; b < dim_b; b++) {
    output[dst_idx] = input[src_idx];
    src_idx += dim_c * src_c_stride;
    dst_idx += dim_c * dst_c_stride;
  }
}

// see NOTE [ Nearest neighbor upsampling kernel implementation ]
// Backward operation
template <typename scalar_t, typename accscalar_t, nn_bw_compute_source_index_fn_t nn_bw_compute_source_index_fn>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest3d_backward_out_frame(
    const scalar_t* grad_o,
    size_t dim_b,
    size_t dim_c,
    size_t src_dim_d,
    size_t src_dim_h,
    size_t src_dim_w,
    size_t dst_dim_d,
    size_t dst_dim_h,
    size_t dst_dim_w,
    scalar_t* grad_i,
    float depth_scale,
    float height_scale,
    float width_scale) {

  int64_t dst_idx = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  if (dst_idx >= dim_c * dst_dim_d * dst_dim_h * dst_dim_w)
    return;

  int64_t dst_c_stride = dst_dim_d * dst_dim_h * dst_dim_w;
  int64_t src_c_stride = src_dim_d * src_dim_h * src_dim_w;

  int c = (dst_idx / (dst_c_stride)) % dim_c;

  int dst_z = (dst_idx / dst_dim_h / dst_dim_w) % dst_dim_d;
  // note that we do not want to clamp src_z to src_dim_z, since we might
  // intentionally want to skip in case of scale_factor < 1.0
  int src_z = nn_bw_compute_source_index_fn(depth_scale, dst_z, src_dim_d);
  int src_z_up = nn_bw_compute_source_index_fn(depth_scale, dst_z+1, src_dim_d);

  int dst_y = (dst_idx / dst_dim_w) % dst_dim_h;
  // note that we do not want to clamp src_y to src_dim_y, since we might
  // intentionally want to skip in case of scale_factor < 1.0
  int src_y = nn_bw_compute_source_index_fn(height_scale, dst_y, src_dim_h);
  int src_y_up = nn_bw_compute_source_index_fn(height_scale, dst_y+1, src_dim_h);

  int dst_x = dst_idx % dst_dim_w;
  // note that we do not want to clamp src_x to src_dim_w, since we might
  // intentionally want to skip in case of scale_factor < 1.0
  int src_x = nn_bw_compute_source_index_fn(width_scale, dst_x, src_dim_w);
  int src_x_up = nn_bw_compute_source_index_fn(width_scale, dst_x+1, src_dim_w);

  for (int b = 0; b < dim_b; b++) {
    accscalar_t grad = 0;
    for (int z = src_z; z < src_z_up; z++) {
      for (int y = src_y; y < src_y_up; y++) {
        for (int x = src_x; x < src_x_up; x++) {
          int64_t src_idx = b * dim_c * src_c_stride + c * src_c_stride +
              z * src_dim_h * src_dim_w + y * src_dim_w + x;
          grad += grad_o[src_idx];
        }
      }
    }
    grad_i[dst_idx] = grad;
    dst_idx += dim_c * dst_c_stride;
  }
}

template<nn_compute_source_index_fn_t nn_compute_source_index_fn>
static void upsample_nearest3d_out_cuda_template(
    const Tensor& output,
    const Tensor& input_,
    IntArrayRef output_size,
    std::optional<double> scales_d,
    std::optional<double> scales_h,
    std::optional<double> scales_w) {
  TensorArg input_arg{input_, "input_", 1}, output_arg{output, "output", 2};
  checkAllSameGPU(__func__, {input_arg, output_arg});

  // TODO: remove this when the cuda kernel is updated to support the channels_last memory format.
  // This is a temporary hack to prevent a silence correctness issue when calling this kernel
  // with tensors in channels_last format.
  auto output_c = output.is_contiguous() ? output : at::empty(output.sizes(), output.options());

  int output_depth = output_size[0];
  int output_height = output_size[1];
  int output_width = output_size[2];

  int nbatch = input_.size(0);
  int channels = input_.size(1);
  int input_depth = input_.size(2);
  int input_height = input_.size(3);
  int input_width = input_.size(4);

  Tensor input = input_.contiguous();

  if (input.numel() == 0) {
    return;
  }

  // upsample_nearest3d meta call makes sure `nbatch != 0`
  unsigned int n = output.numel() / nbatch;
  dim3 bdim{std::min<unsigned int>(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS)};
  dim3 gdim{ceil_div(n, bdim.x)};
  // safe check for int64 indexing; implicitly restrict launch config for kernel
  TORCH_CHECK(output.numel() <= std::numeric_limits<int64_t>::max(),
        "upsample_nearest3d only supports output tensors with less than INT64_MAX elements, but got ", output.sizes());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Byte,input.scalar_type(), "upsample_nearest3d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.const_data_ptr<scalar_t>();
        auto odata = output_c.mutable_data_ptr<scalar_t>();

        const float depth_scale = compute_scales_value<float>(scales_d, input_depth, output_depth);
        const float height_scale = compute_scales_value<float>(scales_h, input_height, output_height);
        const float width_scale = compute_scales_value<float>(scales_w, input_width, output_width);

        upsample_nearest3d_out_frame<scalar_t, nn_compute_source_index_fn>
          <<<gdim, bdim, 0, stream>>>(
            idata,
            nbatch,
            channels,
            input_depth,
            input_height,
            input_width,
            output_depth,
            output_height,
            output_width,
            odata,
            depth_scale,
            height_scale,
            width_scale);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });

  if (!output.is_contiguous()) {
      output.copy_(output_c);
  }
}

template<nn_bw_compute_source_index_fn_t nn_bw_compute_source_index_fn>
static void upsample_nearest3d_backward_out_cuda_template(
    const Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    std::optional<double> scales_d,
    std::optional<double> scales_h,
    std::optional<double> scales_w) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(
      __func__,
      {grad_output_arg, grad_input_arg});

  int output_depth = output_size[0];
  int output_height = output_size[1];
  int output_width = output_size[2];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_depth = input_size[2];
  int input_height = input_size[3];
  int input_width = input_size[4];

  Tensor grad_output = grad_output_.contiguous();

  if (grad_input.numel() == 0) {
    return;
  }

  // upsample_nearest3d meta call makes sure `nbatch != 0`
  unsigned int n = grad_input.numel() / nbatch;
  dim3 bdim{std::min<unsigned int>(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS)};
  dim3 gdim{ceil_div(n, bdim.x)};
  // safe check for int64 indexing; implicitly restrict launch config for kernel
  TORCH_CHECK(grad_input.numel() <= std::numeric_limits<int64_t>::max(),
    "upsample_nearest3d_backward only supports input tensors with less than INT64_MAX elements, but got ", grad_input.sizes());
  TORCH_CHECK(grad_output.numel() <= std::numeric_limits<int64_t>::max(),
    "upsample_nearest3d_backward only supports output tensors with less than INT64_MAX elements, but got ", grad_output.sizes());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Byte, grad_output.scalar_type(), "upsample_nearest3d_backward_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.mutable_data_ptr<scalar_t>();
        auto odata = grad_output.const_data_ptr<scalar_t>();

        float depth_scale = compute_scales_value_backwards<float>(scales_d, output_depth, input_depth);
        float height_scale = compute_scales_value_backwards<float>(scales_h, output_height, input_height);
        float width_scale = compute_scales_value_backwards<float>(scales_w, output_width, input_width);

        upsample_nearest3d_backward_out_frame<scalar_t, accscalar_t, nn_bw_compute_source_index_fn>
            <<<gdim, bdim, 0, stream>>>(
                odata,
                nbatch,
                channels,
                output_depth,
                output_height,
                output_width,
                input_depth,
                input_height,
                input_width,
                idata,
                depth_scale,
                height_scale,
                width_scale);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace

TORCH_IMPL_FUNC(upsample_nearest3d_out_cuda) (
    const Tensor& input,
    IntArrayRef output_size,
    std::optional<double> scales_d,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& output) {
  upsample_nearest3d_out_cuda_template<nearest_neighbor_compute_source_index>(
      output, input, output_size, scales_d, scales_h, scales_w);
}

TORCH_IMPL_FUNC(_upsample_nearest_exact3d_out_cuda) (
    const Tensor& input,
    IntArrayRef output_size,
    std::optional<double> scales_d,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& output) {
  upsample_nearest3d_out_cuda_template<nearest_neighbor_exact_compute_source_index>(output, input, output_size, scales_d, scales_h, scales_w);
}

TORCH_IMPL_FUNC(upsample_nearest3d_backward_out_cuda) (
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    std::optional<double> scales_d,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& grad_input) {
  upsample_nearest3d_backward_out_cuda_template<nearest_neighbor_bw_compute_source_index>(
      grad_input, grad_output, output_size, input_size, scales_d, scales_h, scales_w);
}

TORCH_IMPL_FUNC(_upsample_nearest_exact3d_backward_out_cuda) (
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    std::optional<double> scales_d,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& grad_input) {
  upsample_nearest3d_backward_out_cuda_template<nearest_neighbor_exact_bw_compute_source_index>(
      grad_input, grad_output, output_size, input_size, scales_d, scales_h, scales_w);
}

using at::native::upsample::compute_output_size;
using at::native::upsample_cuda::get_scale_value;

} // namespace at::native
