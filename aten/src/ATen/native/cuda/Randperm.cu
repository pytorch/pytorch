#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/TensorFactories.h>
#include <ATen/cuda/cub.h>
#include <ATen/native/cuda/Randperm.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/arange.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/randperm_native.h>
#endif

#include <limits>

namespace at::native {

// [Algorithm of randperm]
//
// randperm is implemented by sorting an arange tensor of size n with randomly
// generated keys. When random keys are different from each other, all different
// permutations have the same probability.
//
// However, there is a pitfall here:
// For better performance, these N random keys are generated independently,
// and there is no effort to make sure they are different at the time of generation.
// When two keys are identical, stable sorting algorithms will not permute these two keys.
// As a result, (0, 1) will appear more often than (1, 0).
//
// To overcome this pitfall we first carefully choose the number of bits in these keys,
// so that the probability of having duplicate keys is under a threshold. Let q be the
// threshold probability for having non-duplicate keys, then it can be proved that[1]
// the number of bits required is: ceil(log2(n - (6 n^2 + 1) / (12 log(q))))
//
// Then after sort, we launch a separate kernel that additionally shuffles any islands
// of values whose keys matched. The algorithm of this kernel is as follows:
// Each thread reads its key and the keys of its neighbors to tell if it's part of an island.
// For each island, the first thread in the island sees a key match at index i+1 but not index i-1.
// This thread considers itself the "island leader". The island leader then reads more indices to
// the right to figure out how big the island is. Most likely, the island will be very small,
// just a few values. The island leader then rolls that many RNG, uses them to additionally
// shuffle values within the island using serial Fisher-Yates, and writes them out.
//
// Reference
// [1] https://osf.io/af2hy/

// The kernels are templated on an opaque, self-aligned type of the correct
// size to avoid redundant kernels for different types of the same size.
namespace {
template <int N> struct alignas(N) OpaqueType { char data[N]; };
}

Tensor& randperm_out_cuda(int64_t n, std::optional<Generator> generator, Tensor& result) {
  TORCH_CHECK(n >= 0, "n must be non-negative, got", n);

  check_supported_max_int_with_precision(n, result);

  result.resize_({n});

  auto range = at::arange(n, result.options());

  // shuffled_data points to the underlying data of the output tensor if the tensor is contiguous; otherwise it
  // points to a new tensor.
  Tensor shuffled;
  void *shuffled_data;
  if (result.is_contiguous()) {
    shuffled_data = result.data_ptr();
  } else {
    shuffled = at::empty(n, result.options());
    shuffled_data = shuffled.data_ptr();
  }

  auto opt = TensorOptions().device(result.device());

  // See note [Algorithm of randperm]
  const double log_threshold_12 = std::log(0.9) * 12;
  double nd = static_cast<double>(n);

  int bits = std::min(64,
    static_cast<int>(std::ceil(std::log2(nd - (6 * nd * nd + 1) / log_threshold_12))));

  if (n == 0) {
    return result;
  } else if (bits <= 32) {
    // For asserting device type match of the generator and result,
    // we deligate that to the 'random_' function below.

    auto keys = at::empty(result.sizes(), opt.dtype(kInt)).random_(
      std::numeric_limits<int>::min(), std::numeric_limits<int>::max(), generator);
    auto keys_tmp = at::empty_like(keys);
    auto keys_out = keys_tmp.mutable_data_ptr<int>();
    AT_DISPATCH_ALL_TYPES_AND(kHalf, result.scalar_type(), "randperm_out_cuda", [&] {
      using dtype = OpaqueType<sizeof(scalar_t)>;
      auto shuffled_data_ = reinterpret_cast<dtype*>(shuffled_data);
      auto* range_data = reinterpret_cast<const dtype*>(range.const_data_ptr());
      at::cuda::hipcub::radix_sort_pairs<int, dtype>(
        keys.const_data_ptr<int>(), keys_out,
        range_data, shuffled_data_,
        n, false, 0, bits);

      randperm_handle_duplicate_keys(keys_out, shuffled_data_, bits, n, generator);
    });
  } else {
    auto keys = at::empty(result.sizes(), opt.dtype(kLong)).random_(
      std::numeric_limits<int64_t>::min(), std::numeric_limits<int64_t>::max(), generator);
    auto keys_tmp = at::empty_like(keys);
    auto keys_out = keys_tmp.mutable_data_ptr<int64_t>();
    AT_DISPATCH_ALL_TYPES_AND(kHalf, result.scalar_type(), "randperm_out_cuda", [&] {
      using dtype = OpaqueType<sizeof(scalar_t)>;
      auto shuffled_data_ = reinterpret_cast<dtype*>(shuffled_data);
      auto* range_data = reinterpret_cast<const dtype*>(range.data_ptr());
      at::cuda::hipcub::radix_sort_pairs<int64_t, dtype>(
        keys.const_data_ptr<int64_t>(), keys_out,
        range_data, shuffled_data_,
        n, false, 0, bits);

      randperm_handle_duplicate_keys(keys_out, shuffled_data_, bits, n, generator);
    });
  }

  if (!result.is_contiguous()) {
    result.copy_(shuffled);
  }

  return result;
}

} // namespace at::native
