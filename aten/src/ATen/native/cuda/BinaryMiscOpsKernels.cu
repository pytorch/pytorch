#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/NumericUtils.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at::native {

void smooth_l1_kernel_cuda(TensorIteratorBase& iter, double beta) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "smooth_l1_cuda", [&iter, beta]() {
    scalar_t beta_val(beta);
    gpu_kernel(iter, [beta_val] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      auto z = ::abs(a - b);
      return z < beta_val ? scalar_t(0.5) * z * z / beta_val : z - scalar_t(0.5) * beta_val;
    });
  });
}

void huber_kernel_cuda(TensorIterator& iter, double delta) {
  AT_DISPATCH_FLOATING_TYPES_AND2(kBFloat16, kHalf, iter.dtype(), "huber_cuda", [&iter, delta] {
    scalar_t delta_val(delta);
    gpu_kernel(iter, [delta_val] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      auto z = ::abs(a - b);
      return z < delta_val ? scalar_t(0.5) * z * z : delta_val * (z - scalar_t(0.5) * delta_val);
    });
  });
}

void mse_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "mse_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto diff = a - b;
      return diff * diff;
    });
  });
}

void xlogy_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "xlogy_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t x, scalar_t y) -> scalar_t {
      if (at::_isnan(y)){
        return NAN;
      }
      if (x == 0){
        return 0;
      }
      return x * std::log(y);
    });
  });
}

void xlog1py_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "xlog1py_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t x, scalar_t y) -> scalar_t {
      if (at::_isnan(y)){
        return NAN;
      }
      if (x == 0){
        return 0;
      }
      return x * std::log1p(y);
    });
  });
}

REGISTER_DISPATCH(smooth_l1_stub, &smooth_l1_kernel_cuda)
REGISTER_DISPATCH(huber_stub, &huber_kernel_cuda)
REGISTER_DISPATCH(mse_stub, &mse_kernel_cuda)
REGISTER_DISPATCH(xlogy_stub, &xlogy_kernel_cuda)
REGISTER_DISPATCH(xlog1py_stub, &xlog1py_kernel_cuda)

// DO NOT ADD ANY NEW KERNELS HERE
// CUDA compilation times grow quickly.  It's perfectly acceptable to have a file per kernel.

} // namespace at::native
