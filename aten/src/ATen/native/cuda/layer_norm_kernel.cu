#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/native/layer_norm.h>

#include <type_traits>

#include <thrust/tuple.h>

#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/block_reduce.cuh>
#include <ATen/native/cuda/thread_constants.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like_native.h>
#include <ATen/ops/native_layer_norm_native.h>
#include <ATen/ops/native_layer_norm_backward_native.h>
#include <ATen/ops/zeros_like_native.h>
#endif

#include <c10/cuda/CUDAMathCompat.h>
#include <c10/util/env.h>


namespace at::native {

namespace {

constexpr int kCUDANumThreads = 256;
constexpr unsigned int kWarpSize = C10_WARP_SIZE;
constexpr int vec_size = 4; //we could make it dependent on dtype, but that would lead to different results between float and low-p types

// aligned vector generates vectorized load/store on CUDA (copy-pasted from MemoryAccess.cuh)
template<typename scalar_t, int vec_size>
struct alignas(sizeof(scalar_t) * vec_size) aligned_vector {
  scalar_t val[vec_size];
};

// Checks alignment of buffers for using vectorized loads / stores
template<typename T>
bool can_vectorize(const T * ptr, int alignment) {
  uint64_t addr = reinterpret_cast<uint64_t>(ptr);
  return addr % alignment == 0;
};


template <typename T, typename T_ACC>
__global__ void RowwiseMomentsCUDAKernel(
    int64_t N,
    T_ACC eps,
    const T* X,
    T_ACC* mean,
    T_ACC* rstd) {
  using WelfordType = WelfordData<T_ACC, int64_t>;
  using WelfordOp =
      WelfordOps<T_ACC, T_ACC, int64_t, thrust::pair<T_ACC, T_ACC>>;

  __shared__
      typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::
          type val_shared[C10_WARP_SIZE];
  WelfordType* val_shared_ptr = reinterpret_cast<WelfordType*>(val_shared);

  const int64_t i = blockIdx.x;
  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);

  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    val = welford_op.reduce(val, static_cast<T_ACC>(X[index]), index);
  }
  val = cuda_utils::BlockReduce(
      val,
      welford_op,
      /*identity_element=*/WelfordType(0, 0, 0, 0),
      val_shared_ptr);

  if (threadIdx.x == 0) {
    T_ACC m1;
    T_ACC m2;
    thrust::tie(m2, m1) = welford_op.project(val);
    mean[i] = m1;
    rstd[i] = c10::cuda::compat::rsqrt(m2 + eps);
  }
}

template <typename T, typename T_ACC>
__global__ void LayerNormForwardCUDAKernel(
    int64_t N,
    const T* X,
    const T_ACC* mean,
    const T_ACC* rstd,
    const T* gamma,
    const T* beta,
    T* Y) {
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    const T_ACC beta_v =
        beta == nullptr ? T_ACC(0) : static_cast<T_ACC>(beta[j]);
    Y[index] = (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
            static_cast<T_ACC>(rstd[i]) * gamma_v +
        beta_v;
  }
}

struct WelfordDataLN{
  float mean;
  float sigma2;
  float count;
  C10_HOST_DEVICE WelfordDataLN(): mean(0.f), sigma2(0.f), count(0.f){}
  C10_HOST_DEVICE WelfordDataLN(float mean, float sigma2, float count): mean(mean), sigma2(sigma2), count(count) {}
};

template<typename U> __device__
WelfordDataLN cuWelfordOnlineSum(
  const U val,
  const WelfordDataLN& curr_sum)
{
  U delta = val - curr_sum.mean;
  U new_count = curr_sum.count + 1.f;
  U new_mean = curr_sum.mean + delta * (1.f/new_count); //proper division is slow, this is less accurate but noticeably faster
  return {new_mean, curr_sum.sigma2 + delta * (val - new_mean), new_count};
}

__device__
WelfordDataLN cuWelfordCombine(
  const WelfordDataLN dataB,
  const WelfordDataLN dataA
) {
  using U = decltype(dataB.count);
  U delta = dataB.mean - dataA.mean;
  U count = dataA.count + dataB.count;
  U mean, sigma2;
  if (count > decltype(dataB.count){0}) {
    auto coef = 1.f/count; //NB we don't use --use_fast_math, but this is emulation, 1./count goes to intrinsic, `* coef` is multiplication, instead of slow fp division
    auto nA = dataA.count * coef;
    auto nB = dataB.count * coef;
    mean = nA*dataA.mean + nB*dataB.mean;
    sigma2 = dataA.sigma2 + dataB.sigma2 + delta * delta * dataA.count * nB;
  } else {
    mean = U(0);
    sigma2 = U(0);
  }
  return {mean, sigma2, count};
}

template<typename T>
__device__ WelfordDataLN compute_stats(
  const T*  __restrict__ X,
  const int N,
  float * buf
  ) {
    //X points to the row to read
    using vec_t = aligned_vector<T, vec_size>;
    using acc_t = acc_type<T, true>;
    const vec_t * X_vec = reinterpret_cast<const vec_t*>(X);
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const int n_vec_to_read = N/vec_size;
    WelfordDataLN wd(0.f, 0.f, 0.f);
    //no tail, we check that N is multiple of vec_size
    for (int i = thrx; i < n_vec_to_read; i += numx) {
      vec_t data = X_vec[i];
      #pragma unroll
      for (int ii=0; ii < vec_size; ii++){
        wd = cuWelfordOnlineSum(static_cast<acc_t>(data.val[ii]), wd);
      }
    }
    // intra-warp reduction
    for (int offset = (C10_WARP_SIZE >> 1); offset > 0; offset >>= 1) {
        WelfordDataLN wdB{WARP_SHFL_DOWN(wd.mean, offset),
        WARP_SHFL_DOWN(wd.sigma2, offset), WARP_SHFL_DOWN(wd.count, offset)};
        wd = cuWelfordCombine(wd, wdB);
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      float * meansigmabuf = buf;
      float * countbuf = buf + blockDim.y;
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_y = threadIdx.y - offset;
          meansigmabuf[2*wrt_y] = wd.mean;
          meansigmabuf[2*wrt_y+1] = wd.sigma2;
          countbuf[wrt_y] = wd.count;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          WelfordDataLN wdB{meansigmabuf[2*threadIdx.y],
                          meansigmabuf[2*threadIdx.y+1],
                          countbuf[threadIdx.y]};
          wd = cuWelfordCombine(wd, wdB);
        }
        __syncthreads();
      }
      if (threadIdx.x == 0 && threadIdx.y ==0) {
        meansigmabuf[0] = wd.mean;
        meansigmabuf[1] = wd.sigma2/float(N);
      }
      __syncthreads();
      return WelfordDataLN{meansigmabuf[0], meansigmabuf[1],0.f};

    } else {
      return WelfordDataLN{WARP_SHFL(wd.mean,0), WARP_SHFL(wd.sigma2,0)/float(N), 0.f};
    }
}


template <typename T, typename T_ACC,
typename std::enable_if_t<!std::is_same_v<T, double>, int> = 0>
__device__ __inline__ void vectorized_layer_norm_kernel_impl(
  const int N,
  T_ACC eps,
  const  T* __restrict__ X,
  const  T* gamma,
  const  T* beta,
  T_ACC* mean,
  T_ACC* rstd,
  T* Y){
    extern __shared__ float s_data[]; //if we made smem WelfordDataLN type, there would be bank conflicts,
    //as one thread would have to write 3 consecutive floats
    auto i1 = blockIdx.x;
    const T * block_row = X + i1 * N;
    WelfordDataLN wd = compute_stats(block_row, N, s_data);

    using vec_t = aligned_vector<T, vec_size>;
    const vec_t * X_vec = reinterpret_cast<const vec_t*>(block_row);
    const vec_t * gamma_vec = (gamma != nullptr) ? reinterpret_cast<const vec_t*>(gamma) : nullptr;
    const vec_t * beta_vec = (beta != nullptr) ? reinterpret_cast<const vec_t*>(beta) : nullptr;
    vec_t * Y_vec = reinterpret_cast<vec_t*>(Y + i1 * N);

    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const int n_vec_to_read = N/vec_size;

    T_ACC rstd_val = c10::cuda::compat::rsqrt(wd.sigma2 + eps);

    // No tail, N is guaranteed to be multiple of vec size
    for (int i = thrx; i < n_vec_to_read; i += numx) {
      vec_t data = X_vec[i];
      vec_t out;

      // Computation is performed in T_ACC, X is cast to T_ACC and result is implicitly cast to T
      if (gamma_vec != nullptr && beta_vec != nullptr) {
        #pragma unroll
        for (int ii=0; ii < vec_size; ii++){
          out.val[ii] = static_cast<T_ACC>(gamma_vec[i].val[ii]) * (rstd_val * (static_cast<T_ACC>(data.val[ii]) - wd.mean))
            + static_cast<T_ACC>(beta_vec[i].val[ii]);
        }
      } else if (gamma_vec != nullptr) {
        #pragma unroll
        for (int ii=0; ii < vec_size; ii++){
          out.val[ii] = static_cast<T_ACC>(gamma_vec[i].val[ii]) * (rstd_val * (static_cast<T_ACC>(data.val[ii]) - wd.mean));
        }
      } else if (beta_vec != nullptr) {
        #pragma unroll
        for (int ii=0; ii < vec_size; ii++){
          out.val[ii] = (rstd_val * (static_cast<T_ACC>(data.val[ii]) - wd.mean)) + static_cast<T_ACC>(beta_vec[i].val[ii]);
        }
      } else {
        #pragma unroll
        for (int ii=0; ii < vec_size; ii++){
          out.val[ii] = rstd_val * (static_cast<T_ACC>(data.val[ii]) - wd.mean);
        }
      }
      Y_vec[i] = out;
    }
    if (thrx == 0) {
      mean[i1] = wd.mean;
      rstd[i1] = rstd_val;
    }
}

template <typename T, typename T_ACC,
typename std::enable_if_t<std::is_same_v<T, double>, int> = 0>
__device__ __inline__ void vectorized_layer_norm_kernel_impl(
  const int /*N*/,
  T_ACC /*eps*/,
  const  T* __restrict__ /*X*/,
  const  T* /*gamma*/,
  const  T* /*beta*/,
  T_ACC* /*mean*/,
  T_ACC* /*rstd*/,
  T* /*Y*/){
    CUDA_KERNEL_ASSERT(false && "doesn't work with double");
  }

//to avoid windows SFINAE errors
template <typename T, typename T_ACC>
__global__ void vectorized_layer_norm_kernel(
  const int N,
  T_ACC eps,
  const  T* __restrict__ X,
  const  T* gamma,
  const  T* beta,
  T_ACC* mean,
  T_ACC* rstd,
  T* Y){
    vectorized_layer_norm_kernel_impl(N, eps, X, gamma, beta, mean, rstd, Y);
  }


template<typename T, typename T_ACC>
__device__ __inline__ void compute_gI(
  const T* __restrict__ dY,
  const T* __restrict__ X,
  const T_ACC* __restrict__ mean,
  const T_ACC* __restrict__ rstd,
  const T* __restrict__ gamma,
  T* dX,
  const int N,
  T_ACC * buf){
    const auto i1 = blockIdx.x;
    const T_ACC mean_val = mean[i1];
    const T_ACC rstd_val = rstd[i1];
    T_ACC stats_x1{0}, stats_x2{0};
    constexpr int unroll = 4;
    auto l = unroll * threadIdx.x;
    const T * X_i = X + i1 * N;
    const T * dY_i = dY + i1 * N;
    T * dX_i = dX + i1 * N;
    //vectorized reads don't improve perf, so use regular unrolling

    for (; l+unroll - 1 < N; l += blockDim.x * unroll){
      #pragma unroll
      for (int k=0; k< unroll; k++){
          const auto gamma_val = (gamma != nullptr) ? static_cast<T_ACC>(gamma[l+k]) : T_ACC(1);
          const auto c_h = static_cast<T_ACC>(X_i[l+k]);
          const auto c_loss = static_cast<T_ACC>(dY_i[l+k]);
          stats_x1 += c_loss * gamma_val;
          stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
      }
    }
    for (;  l < N; l ++) {
          const auto gamma_val = (gamma != nullptr) ? static_cast<T_ACC>(gamma[l]) : T_ACC(1);
          const auto c_h = static_cast<T_ACC>(X_i[l]);
          const auto c_loss = static_cast<T_ACC>(dY_i[l]);
          stats_x1 += c_loss * gamma_val;
          stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
    }

    stats_x1 = cuda_utils::BlockReduceSum(stats_x1, buf);
    stats_x2 = cuda_utils::BlockReduceSum(stats_x2, buf);
    if (threadIdx.x == 0) {
      buf[0] = stats_x1;
      buf[1] = stats_x2;
    }
    __syncthreads();
    stats_x1 = buf[0];
    stats_x2 = buf[1];
    T_ACC fH = N;
    T_ACC term1 = (T_ACC(1) / fH) * rstd_val;

    for (int l = threadIdx.x; l < N; l += blockDim.x){
        const auto x = X_i[l];
        const auto dy = dY_i[l];
        const auto gamma_val = (gamma != nullptr) ? static_cast<T_ACC>(gamma[l]) : T_ACC(1);

        T_ACC f_grad_input = fH * gamma_val * dy;
        f_grad_input -= (x - mean_val) * rstd_val * stats_x2;
        f_grad_input -= stats_x1;
        f_grad_input *= term1;
        dX_i[l] = f_grad_input;
    }
  }


template<typename T, typename T_ACC>
__global__ void layer_norm_grad_input_kernel(
  const T* __restrict__ dY,
  const T* __restrict__ X,
  const T_ACC* __restrict__ mean,
  const T_ACC* __restrict__ rstd,
  const T* __restrict__ gamma,
  T*  dX,
  const int N){
    alignas(sizeof(double)) extern __shared__ char s_data1[];
    T_ACC * buf = reinterpret_cast<T_ACC*>(&s_data1);

    compute_gI(dY, X, mean, rstd, gamma, dX, N, buf);
  }


// This implementation gets called when input buffers (dY, X, gamma and dX) are aligned
// to vec_size * sizeof(T). Compared to the unvectorized implementation, it is about 10%
// faster measured at PT operator level, with cases seeing a 2X speedup (where N >> M).
// There are no noticeable regressions on the rest of the sizes.

template<typename T, typename T_ACC>
__global__ void layer_norm_grad_input_kernel_vectorized(
  const T* __restrict__ dY,
  const T* __restrict__ X,
  const T_ACC* __restrict__ mean,
  const T_ACC* __restrict__ rstd,
  const T* __restrict__ gamma,
  T* dX,
  const int N) {
  alignas(sizeof(double)) extern __shared__ char shared_data[];
  T_ACC* reduce_buf = reinterpret_cast<T_ACC*>(&shared_data);

  const auto bIdx = blockIdx.x;
  const T_ACC mean_val = mean[bIdx];
  const T_ACC rstd_val = rstd[bIdx];
  const T* X_i = X + bIdx * N;
  const T* dY_i = dY + bIdx * N;
  T* dX_i = dX + bIdx * N;

  using vec_t = aligned_vector<T, vec_size>;
  const vec_t* const X_i_vec_ptr = reinterpret_cast<const vec_t*>(X_i);
  const vec_t* const dY_i_vec_ptr = reinterpret_cast<const vec_t*>(dY_i);
  const vec_t* const gamma_vec_ptr = (gamma != nullptr) ? reinterpret_cast<const vec_t*>(gamma) : nullptr;
  vec_t* const dX_i_vec = reinterpret_cast<vec_t*>(dX_i);

  vec_t X_i_vec_reg, dY_i_vec_reg, gamma_vec_reg, dX_i_vec_reg;
  for (int k = 0; k < vec_size; ++k) {
    gamma_vec_reg.val[k] = T(1);
  }

  T_ACC stats_x1{0}, stats_x2{0};
  unsigned int l = threadIdx.x * vec_size;
  for (; l + vec_size - 1 < N; l += blockDim.x * vec_size) {
    unsigned int vec_idx = l / vec_size;
    if (gamma != nullptr) {
      gamma_vec_reg = gamma_vec_ptr[vec_idx];
    }

    X_i_vec_reg = X_i_vec_ptr[vec_idx];
    dY_i_vec_reg = dY_i_vec_ptr[vec_idx];

    for (int k = 0; k < vec_size; ++k) {
      const auto gamma_val = static_cast<T_ACC>(gamma_vec_reg.val[k]);
      const auto c_h = static_cast<T_ACC>(X_i_vec_reg.val[k]);
      const auto c_loss = static_cast<T_ACC>(dY_i_vec_reg.val[k]);
      stats_x1 += c_loss * gamma_val;
      stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
    }
  }

  // Tail Loop
  for (; l < N; l++) {
    const auto gamma_val = (gamma != nullptr) ? static_cast<T_ACC>(gamma[l]) : T_ACC(1);
    const auto c_h = static_cast<T_ACC>(X_i[l]);
    const auto c_loss = static_cast<T_ACC>(dY_i[l]);
    stats_x1 += c_loss * gamma_val;
    stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
  }

  // Reduction in Shared Memory
  stats_x1 = cuda_utils::BlockReduceSum(stats_x1, reduce_buf);
  stats_x2 = cuda_utils::BlockReduceSum(stats_x2, reduce_buf);
  if (threadIdx.x == 0) {
    reduce_buf[0] = stats_x1;
    reduce_buf[1] = stats_x2;
  }
  __syncthreads();
  stats_x1 = reduce_buf[0];
  stats_x2 = reduce_buf[1];

  T_ACC fH = N;
  T_ACC term1 = (T_ACC(1) / fH) * rstd_val;

  l = threadIdx.x * vec_size;
  for (; l + vec_size - 1 < N; l += blockDim.x * vec_size) {
    unsigned int vec_idx = l / vec_size;
    if (gamma != nullptr) {
      gamma_vec_reg = gamma_vec_ptr[vec_idx];
    }

    X_i_vec_reg = X_i_vec_ptr[vec_idx];
    dY_i_vec_reg = dY_i_vec_ptr[vec_idx];

    for (int k = 0; k < vec_size; ++k) {
      const auto gamma_val = static_cast<T_ACC>(gamma_vec_reg.val[k]);
      const auto x = static_cast<T_ACC>(X_i_vec_reg.val[k]);
      const auto dy = static_cast<T_ACC>(dY_i_vec_reg.val[k]);

      T_ACC f_grad_input = fH * gamma_val * dy;
      f_grad_input -= (x - mean_val) * rstd_val * stats_x2;
      f_grad_input -= stats_x1;
      f_grad_input *= term1;
      dX_i_vec_reg.val[k] = f_grad_input;
    }

    dX_i_vec[vec_idx] = dX_i_vec_reg;
  }

  // Tail Loop
  for (; l < N; l += blockDim.x) {
    const auto x = X_i[l];
    const auto dy = dY_i[l];
    const auto gamma_val = (gamma != nullptr) ? static_cast<T_ACC>(gamma[l]) : T_ACC(1);

    T_ACC f_grad_input = fH * gamma_val * dy;
    f_grad_input -= (x - mean_val) * rstd_val * stats_x2;
    f_grad_input -= stats_x1;
    f_grad_input *= term1;
    dX_i[l] = f_grad_input;
  }
}


template <typename T, typename T_ACC>
__global__ void GammaBetaBackwardSimpleCUDAKernel(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T_ACC* mean,
    const T_ACC* rstd,
    T* dg,
    T* db) {
  const int64_t j = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  if (j < N) {
    T_ACC sum1 = 0;
    T_ACC sum2 = 0;
    for (int64_t i = 0; i < M; ++i) {
      const int64_t index = i * N + j;
      sum1 += dg == nullptr ? T_ACC(0)
                            : static_cast<T_ACC>(dY[index]) *
              (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
              static_cast<T_ACC>(rstd[i]);
      sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index]);
    }
    if (dg != nullptr) {
      dg[j] = sum1;
    }
    if (db != nullptr) {
      db[j] = sum2;
    }
  }
}

template <typename T, typename T_ACC,
unsigned int block_dim_x,
unsigned int block_dim_y,
unsigned int rows_per_block_y,
bool check_x,
bool check_y>
__device__
__forceinline__
void
blockReduceGammaBetaBackwardsHelper(
    int64_t M_start,
    int64_t M,
    int64_t N,
    const T* __restrict__ dY,
    const T* __restrict__ X,
    const T_ACC* __restrict__ mean,
    const T_ACC* __restrict__ rstd,
    T* __restrict__ dg,
    T* __restrict__ db,
    T_ACC &dg_sum,
    T_ACC &db_sum
) {
  constexpr int rows_per_thread_y = rows_per_block_y / block_dim_y;
  int64_t thread_x = blockIdx.x * block_dim_x + threadIdx.x;

    int lane_id = (threadIdx.y * blockDim.x + threadIdx.x) & (kWarpSize - 1);
    int64_t mean_index = M_start + threadIdx.y * rows_per_thread_y;
    T_ACC warp_mean = 0, warp_rstd = 0;
    if (lane_id < rows_per_thread_y && mean_index + lane_id < M) {
      warp_mean = mean[mean_index + lane_id];
      warp_rstd = rstd[mean_index + lane_id];
    }
    // We do a WARP_SYNC() here because we use WARP_SHFL below to access
    // warp_mean and warp_rstd.
    WARP_SYNC();

    T_ACC dY_regs[rows_per_thread_y] = {0};
    T_ACC X_regs[rows_per_thread_y] = {0};
    #pragma unroll
    for (int i = 0; i < rows_per_thread_y; ++i) {
      int64_t current_y = M_start + threadIdx.y * rows_per_thread_y + i;
      bool active = true;
      if (check_x && thread_x >= N) {
        active = false;
      }
      if (check_y && current_y >= M) {
        active = false;
      }
      if (active) {
        dY_regs[i] = dY[current_y * N + thread_x];
        X_regs[i] = X[current_y * N + thread_x];
      }
    }

    #pragma unroll
    for (int i = 0; i < rows_per_thread_y; ++i) {
      T_ACC mean_reg = WARP_SHFL(warp_mean, i, kWarpSize);
      T_ACC rstd_reg = WARP_SHFL(warp_rstd, i, kWarpSize);
      dg_sum += dY_regs[i] * (X_regs[i] - mean_reg) * rstd_reg;
      db_sum += dY_regs[i];
    }
}

template <typename T, typename T_ACC,
unsigned int block_dim_x,
unsigned int block_dim_y,
unsigned int rows_per_block_y,
bool check_x,
bool check_y>
__device__
__forceinline__
void
blockReduceGammaBetaBackwardsWithChecks(
    int64_t M,
    int64_t N,
    const T* __restrict__ dY,
    const T* __restrict__ X,
    const T_ACC* __restrict__ mean,
    const T_ACC* __restrict__ rstd,
    T* __restrict__ dg,
    T* __restrict__ db,
    T_ACC &dg_sum,
    T_ACC &db_sum
) {
  for (int64_t M_start = blockIdx.y * rows_per_block_y;
        M_start < M;
        M_start += rows_per_block_y * gridDim.y) {
    int64_t M_end = M_start + rows_per_block_y - 1;
    if (!check_y || M_end < M) {
      blockReduceGammaBetaBackwardsHelper<T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, check_x, false>
      (M_start, M, N, dY, X, mean, rstd, dg, db, dg_sum, db_sum);
    } else {
      blockReduceGammaBetaBackwardsHelper<T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, check_x, true>
      (M_start, M, N, dY, X, mean, rstd, dg, db, dg_sum, db_sum);
    }
  }
}

// block_dim_x is the number of threads in the x dimension per block.
// block_dim_y is the number of threads in the y dimension per block.
// rows_per_block_y is the size of the tile (number of data elements)
// in the y dimension per block.
// partial_reduction indicates whether we need to reduce across threads
// or not. If set to true, we will not reduce across threads. This can
// be faster in the M >> N case but requires another kernel to do a full
// final reduction.
// aligned_grid means the data size is a multiple of tile size. In that
// case we don't need to check for boundary conditions which can provide
// a further speedup by not needing instructions to check for edge cases
// and not needing predicate registers.
template <typename T, typename T_ACC,
unsigned int block_dim_x, unsigned int block_dim_y,
unsigned int rows_per_block_y,
bool partial_reduction,
bool aligned_grid
>
__global__
void
__launch_bounds__(block_dim_x * block_dim_y)
 GammaBetaBackwardCUDAKernelTemplate(
    int64_t M,
    int64_t N,
    const T* __restrict__ dY,
    const T* __restrict__ X,
    const T_ACC* __restrict__ mean,
    const T_ACC* __restrict__ rstd,
    T* __restrict__ dg,
    T* __restrict__ db) {
  // This assert is a compile-time check only.
  constexpr int rows_per_thread_y = rows_per_block_y / block_dim_y;
  static_assert(rows_per_thread_y <= kWarpSize);

  T_ACC dg_sum = 0;
  T_ACC db_sum = 0;

  if (aligned_grid) {
    // When N and M align perfectly with block_dim_x and block_dim_y, we
    // can skip boundary condition checks that waste instruction issue slots.
    blockReduceGammaBetaBackwardsWithChecks
          <T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, false, false>
          (M, N, dY, X, mean, rstd, dg, db, dg_sum, db_sum);
  } else {
    // In the general case we need to check boundary conditions in the M
    // dimension. However, we can still avoid boundary checks in the N dimension
    // for the inner blocks. So try to avoid those checks when possible.
    if (blockIdx.x * block_dim_x + block_dim_x - 1 < N) {
      blockReduceGammaBetaBackwardsWithChecks
          <T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, false, true>
          (M, N, dY, X, mean, rstd, dg, db, dg_sum, db_sum);
    } else {
      blockReduceGammaBetaBackwardsWithChecks
          <T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, true, true>
          (M, N, dY, X, mean, rstd, dg, db, dg_sum, db_sum);
    }
  }

  int64_t thread_x = ((int64_t)blockIdx.x) * block_dim_x + threadIdx.x;

  // When partial_reduction is requested, we don't reduce within a block.
  // We also don't reduce if we are only a single block in the y dimension.
  if (partial_reduction || (blockDim.y == 1 && gridDim.y == 1)) {
    if (aligned_grid || thread_x < N) {
      int64_t thread_y = ((int64_t)blockIdx.y) * blockDim.y + threadIdx.y;
      if (dg) {
        dg[thread_y * N + thread_x] = dg_sum;
      }
      if (db) {
        db[thread_y * N + thread_x] = db_sum;
      }
    }
  } else {
    // The caller requested a full reduction so we must reduce across
    // warps using shared memory and warp shuffles.
    static_assert(rows_per_thread_y <= C10_WARP_SIZE);
    alignas(sizeof(double)) extern __shared__ char s_data1[];
    T_ACC* s_data_typed = reinterpret_cast<T_ACC*>(&s_data1);
    T_ACC* s_dg;
    T_ACC* s_db;
    int padded_bx = (block_dim_x + 1);
    // Transpose dg and db.
    s_dg = s_data_typed;
    s_db = s_data_typed + (padded_bx * block_dim_y);
    s_dg[threadIdx.y * padded_bx + threadIdx.x] = dg_sum;
    s_db[threadIdx.y * padded_bx + threadIdx.x] = db_sum;
    __syncthreads();

    // Load transposed so that a warp holds an entire column
    // Because block_dim_x != block_dim_y in the general case, we need
    // some code to handle the general case.
    static_assert(block_dim_x * block_dim_y % C10_WARP_SIZE == 0);
    constexpr int warps_available_to_reduce = block_dim_x * block_dim_y / C10_WARP_SIZE;
    int thread_id = threadIdx.y * block_dim_x + threadIdx.x;
    int warp_id = thread_id / C10_WARP_SIZE;
    int lane_id = thread_id & (C10_WARP_SIZE - 1);
    #pragma unroll
    for (int i = warp_id; i < block_dim_x; i += warps_available_to_reduce) {
      T_ACC reg_db, reg_dg;
      if (lane_id < block_dim_y) {
        reg_dg = s_dg[lane_id * padded_bx + i];
        reg_db = s_db[lane_id * padded_bx + i];
      }
      #pragma unroll
      for (unsigned delta = block_dim_y >> 1; delta >= 1; delta >>= 1) {
        reg_dg += WARP_SHFL_XOR(reg_dg, delta, kWarpSize);
        reg_db += WARP_SHFL_XOR(reg_db, delta, kWarpSize);
      }
      // Reduce is done. Now write it out to global memory.
      int64_t out_index = ((int64_t)blockIdx.x) * block_dim_x + i;
      if (threadIdx.x == 0 && (aligned_grid || out_index < N)) {
        if (dg) {
          dg[out_index] = reg_dg;
        }
        if (db) {
          db[out_index] = reg_db;
        }
      }
    }
  }
}

template<typename T, typename T_ACC,
int block_dim_x, int block_dim_y,
int rows_per_block_y,
bool partial_reduction>
void LaunchAndCheckGammaBetaBackwardKernel(
  bool aligned_grid,
  dim3 blocks,
  dim3 threads,
  size_t shmem_sz,
  hipStream_t hip_stream,
  const T* dY_data,
  const T* X_data,
  const T_ACC* mean_data,
  const T_ACC* rstd_data,
  int64_t M,
  int64_t N,
  T* dgamma_data,
  T* dbeta_data) {
if (aligned_grid) {
    GammaBetaBackwardCUDAKernelTemplate<T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, partial_reduction, true>
        <<<blocks, threads, shmem_sz, hip_stream>>>(
            M,
            N,
            dY_data,
            X_data,
            mean_data,
            rstd_data,
            dgamma_data,
            dbeta_data);
  } else {
    GammaBetaBackwardCUDAKernelTemplate<T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, partial_reduction, false>
        <<<blocks, threads, shmem_sz, hip_stream>>>(
            M,
            N,
            dY_data,
            X_data,
            mean_data,
            rstd_data,
            dgamma_data,
            dbeta_data);
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename T, typename T_ACC,
int block_dim_x, int block_dim_y,
int rows_per_block_y>
void ConfigureAndLaunchGammaBetaBackwardKernel(
    const T* dY_data,
    const T* X_data,
    const T_ACC* mean_data,
    const T_ACC* rstd_data,
    int64_t M,
    int64_t N,
    Tensor* dgamma,
    Tensor* dbeta,
    hipStream_t hip_stream) {
  T* dgamma_data =
    dgamma->defined() ? dgamma->template data_ptr<T>() : nullptr;
  T* dbeta_data = dbeta->defined() ? dbeta->template data_ptr<T>() : nullptr;
  bool aligned_grid = (M % rows_per_block_y == 0) && (N % block_dim_x == 0);
  dim3 threads{block_dim_x, block_dim_y};
  dim3 blocks;
  blocks.x = (N + block_dim_x - 1) / block_dim_x;
  blocks.y = 1;
  size_t shmem_sz = (block_dim_x + 1) * block_dim_y * sizeof(T_ACC) * 2;
  if (blocks.y == 1 && threads.y == 1) {
    // Optimization: since there is just one thread doing all the summation, we don't need a reduction
    // across threads. So we set partial_reduction to true.
    LaunchAndCheckGammaBetaBackwardKernel<T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, true>(
      aligned_grid, blocks, threads, shmem_sz, hip_stream, dY_data, X_data, mean_data, rstd_data, M, N, dgamma_data, dbeta_data);
  } else {
    LaunchAndCheckGammaBetaBackwardKernel<T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, false>(
      aligned_grid, blocks, threads, shmem_sz, hip_stream, dY_data, X_data, mean_data, rstd_data, M, N, dgamma_data, dbeta_data);
  }

}

template<typename T, typename T_ACC>
void LaunchGammaBetaBackwardCUDAKernel(
    const T* dY_data,
    const T* X_data,
    const T_ACC* mean_data,
    const T_ACC* rstd_data,
    int64_t M,
    int64_t N,
    Tensor* dgamma,
    Tensor* dbeta,
    hipStream_t hip_stream) {
  constexpr int block_dim_x = 32;
  const int sm_count = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  if (M > 64 * 1024 && N / block_dim_x < sm_count / 2) {
    // We have a situation where M >> N and N is small.
    // In this case we can speed up the computation by parallelizing in the M dimension.
    // We launch multiple blocks in the y-dimension, and compute partial sums for the
    // gradient in the first pass. Then we do a .sum(0) to do a final reduction.
    // Although we launch 2 kernels, we can get up to a 10x speedup for large M.
    constexpr int block_dim_y = 1;
    constexpr int rows_per_block_y = 32;
    bool aligned_grid = (M % rows_per_block_y == 0) && (N % block_dim_x == 0);
    dim3 threads{block_dim_x, block_dim_y};
    dim3 blocks;
    blocks.x = (N + block_dim_x - 1) / block_dim_x;
    // int rows_per_block = my_gamma_beta_unroll_factor *
    blocks.y = (M + rows_per_block_y - 1) / rows_per_block_y;
    constexpr int max_grid_size = 64 * 1024 / 2;
    blocks.y = std::min<unsigned int>(max_grid_size / blocks.x, blocks.y);
    Tensor dgamma_blocks;
    Tensor dbeta_blocks;
    T * dgamma_blocks_ptr = nullptr;
    T * dbeta_blocks_ptr = nullptr;
    if (dgamma->defined()) {
      auto options = dgamma->options();
      dgamma_blocks = at::empty({blocks.y * threads.y, dgamma->size(-1)}, options);
      dgamma_blocks_ptr = dgamma_blocks.data_ptr<T>();
    }
    if (dbeta->defined()) {
      auto options = dbeta->options();
      dbeta_blocks = at::empty({blocks.y * threads.y, dgamma->size(-1)}, options);
      dbeta_blocks_ptr = dbeta_blocks.data_ptr<T>();
    }
    LaunchAndCheckGammaBetaBackwardKernel<T, T_ACC, block_dim_x, block_dim_y, rows_per_block_y, true>(
      aligned_grid, blocks, threads, 0, hip_stream, dY_data, X_data, mean_data, rstd_data, M, N, dgamma_blocks_ptr, dbeta_blocks_ptr);

    if (dgamma_blocks.defined()) {
      *dgamma = dgamma_blocks.sum(0);
    }
    if (dbeta_blocks.defined()) {
      *dbeta = dbeta_blocks.sum(0);
    }
  } else {
    // We are in the normal case where M is not that large.
    // We can change the tile shape (which is the last template parameter) in accordance with M.
    // For small M it is faster to have a smaller tile, otherwise we could have idle threads.
    // For larger M we use a bigger tile size.
    if (M < 64) {
      ConfigureAndLaunchGammaBetaBackwardKernel<T, T_ACC, block_dim_x, 1, 8>(dY_data, X_data, mean_data, rstd_data, M, N, dgamma, dbeta, hip_stream);
    } else if (M < 128) {
      ConfigureAndLaunchGammaBetaBackwardKernel<T, T_ACC, block_dim_x, 8, 64>(dY_data, X_data, mean_data, rstd_data, M, N, dgamma, dbeta, hip_stream);
    } else if (M < 256) {
      ConfigureAndLaunchGammaBetaBackwardKernel<T, T_ACC, block_dim_x, 16, 128>(dY_data, X_data, mean_data, rstd_data, M, N, dgamma, dbeta, hip_stream);
    } else {
      ConfigureAndLaunchGammaBetaBackwardKernel<T, T_ACC, block_dim_x, 32, 256>(dY_data, X_data, mean_data, rstd_data, M, N, dgamma, dbeta, hip_stream);
    }
  }
}

template <typename T, typename T_ACC>
void launch_vectorized_layer_norm_kernel(
  int N,
  int64_t M,
  T_ACC eps,
  const T* X_data,
  const T* gamma_data,
  const T* beta_data,
  T* Y_data,
  T_ACC* mean_data,
  T_ACC* rstd_data
) {
    //constexpr int alignment = 16; //currently unused to make sure float and half results are bw accurate
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    const int warp_size = at::cuda::warp_size();
    const dim3 threads(warp_size, num_threads() / warp_size, 1);
    dim3 blocks(M);

#ifdef USE_ROCM
    uint64_t workgroupSize = static_cast<uint64_t>(blocks.x) * static_cast<uint64_t>(threads.x);
    // this caused invalid configuration problem
    if (workgroupSize > std::numeric_limits<uint32_t>::max()) {
      // Fix invalid configuration https://github.com/pytorch/pytorch/issues/136291
      blocks.x = std::numeric_limits<uint32_t>::max() / threads.x;
    }
#endif

    TORCH_INTERNAL_ASSERT_DEBUG_ONLY(threads.y % 2 == 0 || threads.y == 1);
    int nshared = threads.y > 1 ? threads.y * 3/2 *sizeof(T_ACC) : 0;
    vectorized_layer_norm_kernel<<<blocks, threads, nshared, stream>>>(N, eps, X_data,
    gamma_data, beta_data, mean_data, rstd_data, Y_data);
    C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef USE_ROCM
    // the blocks.x contains the max grid x dimention without invalid configuration error
    // Fix invalid configuration https://github.com/pytorch/pytorch/issues/136291
    // Ensure all elements are processed. Prepare for next round
    int64_t remaining = M - blocks.x;
    const T* X_data2 = X_data;
    T_ACC* mean_data2 = mean_data;
    T_ACC* rstd_data2 = rstd_data;
    T* Y_data2 = Y_data;

    while (remaining > 0) {
      X_data2 += N * blocks.x;
      mean_data2 += blocks.x;
      rstd_data2 += blocks.x;
      Y_data2 += N * blocks.x;

      blocks.x = (remaining > blocks.x) ? blocks.x : remaining;

      vectorized_layer_norm_kernel<<<blocks, threads, nshared, stream>>>(N, eps, X_data2,
        gamma_data, beta_data, mean_data2, rstd_data2, Y_data2);
      C10_CUDA_KERNEL_LAUNCH_CHECK();

      remaining -= blocks.x;
    }
#endif

}

template <typename T, typename T_ACC>
void LayerNormKernelImplInternal(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t M,
    int64_t N,
    T_ACC eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  // assumes input, gamma and beta are of proper shape, this was checked in _check_layer_norm_inputs
  // assumes all tensors are contiguous
  TORCH_CHECK(M <= at::cuda::getCurrentDeviceProperties()->maxGridSize[0], "M should be less than maximum CUDA grid size, \
  file a support request to support bigger batches");
  const T* X_data = X.const_data_ptr<T>();
  const T* gamma_data = gamma.defined() ? gamma.const_data_ptr<T>() : nullptr;
  const T* beta_data = beta.defined() ? beta.const_data_ptr<T>() : nullptr;
  T* Y_data = Y->data_ptr<T>();
  T_ACC* mean_data = mean->data_ptr<T_ACC>();
  T_ACC* rstd_data = rstd->data_ptr<T_ACC>();

  // check if can take fast path - all tensors are properly aligned, N is less than 2^24 (to use float count),
  // N is multiple of vec_size (so that all rows are aligned if tensor is aligned)
  constexpr int num_vec_elems = vec_size;
  constexpr int alignment = num_vec_elems * sizeof(T);
  bool can_vec_X = can_vectorize(X_data, alignment);
  bool can_vec_Y = can_vectorize(Y_data, alignment);
  bool can_vec_gamma = gamma.defined() ? can_vectorize(gamma_data, alignment) : true;
  bool can_vec_beta = beta.defined() ? can_vectorize(beta_data, alignment) : true;

  if ((std::is_same_v<T, float> || std::is_same_v<T, at::Half> || std::is_same_v<T, at::BFloat16>) &&
  N <= static_cast<int64_t>(1ULL << std::numeric_limits<float>::digits) && N % num_vec_elems == 0 &&
  can_vec_X && can_vec_Y && can_vec_gamma && can_vec_beta) {
    launch_vectorized_layer_norm_kernel(static_cast<int>(N), M, eps, X_data, gamma_data, beta_data, Y_data, mean_data, rstd_data);
  } else {
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  RowwiseMomentsCUDAKernel<T, T_ACC>
      <<<M, cuda_utils::kCUDABlockReduceNumThreads, 0, hip_stream>>>(
          N, eps, X_data, mean_data, rstd_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  LayerNormForwardCUDAKernel<T, T_ACC><<<M, kCUDANumThreads, 0, hip_stream>>>(
      N, X_data, mean_data, rstd_data, gamma_data, beta_data, Y_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

void LayerNormKernelImpl(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t M,
    int64_t N,
    double eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      X.scalar_type(),
      "LayerNormKernelImpl",
      [&]() {
        using acc_t = acc_type<scalar_t, true>;
        LayerNormKernelImplInternal<scalar_t, acc_t>(
            X, gamma, beta, M, N, static_cast<acc_t>(eps), Y, mean, rstd);
      });
}

template<typename T, typename T_ACC> __device__
void cuLoadWriteStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    T_ACC* warp_buf1,
    T_ACC* warp_buf2,
    const T* input,
    const T* dout,
    const int i1_end,
    const int64_t N,
    const T_ACC* __restrict__ mean,
    const T_ACC* __restrict__ rstd)
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    T_ACC curr_mean = mean[i1];
    T_ACC curr_rstd = rstd[i1];
    for (int k = 0;  k < blockDim.y;  ++k) {
      int i2 = i2_off + k;
      int load_idx = i1*N+i2;
      int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<N) {
        T curr_input = static_cast<T>(input[load_idx]);
        T curr_dout = static_cast<T>(dout[load_idx]);
        warp_buf1[write_idx] = curr_dout;
        warp_buf2[write_idx] = curr_dout * (curr_input - curr_mean) * curr_rstd;
      } else {
        warp_buf1[write_idx] = T(0);
        warp_buf2[write_idx] = T(0);
      }
    }
  } else {
    for (int k = 0;  k < blockDim.y;  ++k) {
      int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      warp_buf1[write_idx] = T(0);
      warp_buf2[write_idx] = T(0);
    }
  }
}

template<typename T, typename T_ACC> __device__
void cuLoadAddStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    T_ACC* warp_buf1,
    T_ACC* warp_buf2,
    const T* input,
    const T* dout,
    const int i1_end,
    const int64_t N,
    const T_ACC* __restrict__ mean,
    const T_ACC* __restrict__ rstd)
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    T_ACC curr_mean = mean[i1];
    T_ACC curr_rstd = rstd[i1];
    for (int k = 0;  k < blockDim.y;  ++k) {
      int i2 = i2_off + k;
      int load_idx = i1*N+i2;
      int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<N) {
        T_ACC curr_input = static_cast<T_ACC>(input[load_idx]);
        T_ACC curr_dout = static_cast<T_ACC>(dout[load_idx]);
        warp_buf1[write_idx] += curr_dout;
        warp_buf2[write_idx] += curr_dout * (curr_input - curr_mean) * curr_rstd;
      }
    }
  }
}

template<typename T, typename T_ACC> __global__
void cuComputePartGradGammaBeta(
    const T* __restrict__ dout,
    const T* __restrict__ input,
    const int64_t M,
    const int64_t N,
    const T_ACC* __restrict__ mean,
    const T_ACC* __restrict__ rstd,
    T_ACC* part_grad_gamma,
    T_ACC* part_grad_beta)
{
    const int numsegs_M = (M+blockDim.y*blockDim.y-1) / (blockDim.y*blockDim.y);
    const int segs_per_block = (numsegs_M + gridDim.y - 1) / gridDim.y;
    const int i1_beg = blockIdx.y * segs_per_block * blockDim.y*blockDim.y;
    const int i1_beg_plus_one = (blockIdx.y+1) * segs_per_block * blockDim.y*blockDim.y;
    const int i1_end = i1_beg_plus_one < M ? i1_beg_plus_one : M;
    const int row_stride = blockDim.x+1;
    const int thr_load_col_off = (threadIdx.x*blockDim.y)&(blockDim.x-1);
    const int thr_load_row_off = (threadIdx.x*blockDim.y)/blockDim.x + threadIdx.y*blockDim.y;
    const int i2_off = blockIdx.x * blockDim.x + thr_load_col_off;
    alignas(sizeof(double)) extern __shared__ char shared[];
    T_ACC * buf = reinterpret_cast<T_ACC*>(&shared); // buf has at least blockDim.x * blockDim.y * blockDim.y + (blockDim.y - 1)*(blockDim.x/blockDim.y) elements
    T_ACC* warp_buf1 = (T_ACC*)buf;
    T_ACC* warp_buf2 = warp_buf1 + blockDim.y * blockDim.y * row_stride;
    // compute partial sums from strided inputs
    // do this to increase number of loads in flight
    cuLoadWriteStridedInputs(i1_beg,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf1,warp_buf2,input,dout,i1_end,N,mean,rstd);
    for (int i1_block = i1_beg+blockDim.y*blockDim.y;  i1_block < i1_end;  i1_block+=blockDim.y*blockDim.y) {
      cuLoadAddStridedInputs(i1_block,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf1,warp_buf2,input,dout,i1_end,N,mean,rstd);
    }
    __syncthreads();
    // inter-warp reductions
    // sum within each warp
    T_ACC acc1 = T_ACC(0);
    T_ACC acc2 = T_ACC(0);
    for (int k = 0;  k < blockDim.y;  ++k) {
      int row1 = threadIdx.y + k*blockDim.y;
      int idx1 = row1*row_stride + threadIdx.x;
      acc1 += warp_buf1[idx1];
      acc2 += warp_buf2[idx1];
    }
    warp_buf1[threadIdx.y*row_stride+threadIdx.x] = acc1;
    warp_buf2[threadIdx.y*row_stride+threadIdx.x] = acc2;
    __syncthreads();
    // sum all warps
    for (int offset = blockDim.y/2;  offset > 1;  offset /= 2) {
      if (threadIdx.y < offset) {
        int row1 = threadIdx.y;
        int row2 = threadIdx.y + offset;
        int idx1 = row1*row_stride + threadIdx.x;
        int idx2 = row2*row_stride + threadIdx.x;
        warp_buf1[idx1] += warp_buf1[idx2];
        warp_buf2[idx1] += warp_buf2[idx2];
      }
      __syncthreads();
    }
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.y == 0 && i2 < N) {
      int row1 = threadIdx.y;
      int row2 = threadIdx.y + 1;
      int idx1 = row1*row_stride + threadIdx.x;
      int idx2 = row2*row_stride + threadIdx.x;
      part_grad_beta[blockIdx.y*N+i2] = warp_buf1[idx1] + warp_buf1[idx2];
      part_grad_gamma[blockIdx.y*N+i2] = warp_buf2[idx1] + warp_buf2[idx2];
    }
}

template<typename T, typename T_ACC> __global__
void cuComputeGradGammaBeta(
    const T_ACC* part_grad_gamma,
    const T_ACC* part_grad_beta,
    const int part_size,
    const int64_t M,
    const int64_t N,
    T* grad_gamma,
    T* grad_beta)
{
    // sum partial gradients for gamma and beta
    alignas(sizeof(double)) extern __shared__ char shared[];
    T_ACC * buf = reinterpret_cast<T_ACC*>(&shared);
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;

    // each warp does sequential reductions until reduced part_size is num_warps
    int num_warp_reductions = part_size / blockDim.y;
    T_ACC sum_gamma = T_ACC(0);
    T_ACC sum_beta = T_ACC(0);
    const T_ACC* part_grad_gamma_ptr = part_grad_gamma + threadIdx.y * num_warp_reductions * N + i2;
    const T_ACC* part_grad_beta_ptr = part_grad_beta + threadIdx.y * num_warp_reductions * N + i2;

    if (i2 < N) {
        for (int warp_offset = 0;  warp_offset < num_warp_reductions;  ++warp_offset) {
          sum_gamma += part_grad_gamma_ptr[warp_offset*N];
          sum_beta += part_grad_beta_ptr[warp_offset*N];
        }
    }

    // inter-warp reductions
    const int nbsize3 = blockDim.x * blockDim.y / 2;
    for (int offset = blockDim.y/2;  offset >= 1;  offset /= 2) {
      // top half write to shared memory
      if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
        const int write_idx = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
        buf[write_idx] = sum_gamma;
        buf[write_idx+nbsize3] = sum_beta;
      }
      __syncthreads();
      // bottom half sums
      if (threadIdx.y < offset) {
        const int read_idx = threadIdx.y * blockDim.x + threadIdx.x;
        sum_gamma += buf[read_idx];
        sum_beta += buf[read_idx+nbsize3];
      }
      __syncthreads();
    }

    // write out fully summed gradients
    if (threadIdx.y == 0 && i2 < N) {
      if (grad_gamma) {
          grad_gamma[i2] = sum_gamma;
      }
      if (grad_beta) {
          grad_beta[i2] = sum_beta;
      }
    }
}

template<typename T, typename T_ACC> __global__
void cuComputeGradInput(
    const T* __restrict__ dout,
    const T* __restrict__ input,
    const int64_t M,
    const int64_t N,
    const T_ACC* __restrict__ mean,
    const T_ACC* __restrict__ rstd,
    const T* gamma,
    T* grad_input)
{
  for (int i1=blockIdx.y; i1 < M; i1 += gridDim.y) {
    T_ACC sum_loss1 = T_ACC(0);
    T_ACC sum_loss2 = T_ACC(0);
    T_ACC c_mean = mean[i1];
    const T_ACC c_rstd = rstd[i1];
    const T* k_input = input + i1*N;
    const T* k_dout = dout + i1*N;
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL) {
      // Optimization for ROCm MI100
      for( int l = 0; l < N ; l += numx) {
        int idx = l + thrx;
        const T_ACC gamma_idx = static_cast<T_ACC>((idx<N) ? gamma[idx] : T(0));
        const T_ACC c_h = static_cast<T_ACC>((idx<N) ? k_input[idx] : T(0));
        const T_ACC c_loss = static_cast<T_ACC>((idx<N) ? k_dout[idx] : T(0));
        sum_loss1 += c_loss * gamma_idx;
        sum_loss2 += c_loss * gamma_idx * (c_h - c_mean) * c_rstd;
      }
    } else {
      for( int l = 0; l < N ; l += numx) {
        int idx = l + thrx;
        const T_ACC c_h = static_cast<T_ACC>((idx<N) ? k_input[idx] : T(0));
        const T_ACC c_loss = static_cast<T_ACC>((idx<N) ? k_dout[idx] : T(0));
        sum_loss1 += c_loss;
        sum_loss2 += c_loss * (c_h - c_mean) * c_rstd;
      }
    }
    // intra-warp reductions
    for (int mask = blockDim.x/2;  mask > 0;  mask /= 2) {
      sum_loss1 += WARP_SHFL_XOR(sum_loss1, mask);
      sum_loss2 += WARP_SHFL_XOR(sum_loss2, mask);
    }
    // inter-warp reductions
    if (blockDim.y > 1) {
      alignas(sizeof(double)) extern __shared__ char shared[];
      T_ACC * buf = reinterpret_cast<T_ACC*>(&shared);
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_i = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          buf[2*wrt_i] = sum_loss1;
          buf[2*wrt_i+1] = sum_loss2;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.y < offset) {
          const int read_i = threadIdx.y * blockDim.x + threadIdx.x;
          sum_loss1 += buf[2*read_i];
          sum_loss2 += buf[2*read_i+1];
        }
        __syncthreads();
      }
      if (threadIdx.y == 0) {
        buf[2*threadIdx.x] = sum_loss1;
        buf[2*threadIdx.x+1] = sum_loss2;
      }
      __syncthreads();
      if (threadIdx.y !=0) {
        sum_loss1 = buf[2*threadIdx.x];
        sum_loss2 = buf[2*threadIdx.x+1];
      }
    }
    // all threads now have the two sums over l
    T_ACC fH = (T_ACC)N;
    T_ACC term1 = (T_ACC(1) / fH) * c_rstd;
    T* k_grad_input = grad_input + i1*N;
    if (gamma != NULL) {
      for (int l = thrx;  l < N;  l+=numx) {
        const T_ACC c_h = static_cast<T_ACC>(k_input[l]);
        const T_ACC c_loss = static_cast<T_ACC>(k_dout[l]);
        T_ACC f_grad_input = fH * c_loss * gamma[l];
        f_grad_input -= sum_loss1;
        f_grad_input -= (c_h - c_mean) * c_rstd * sum_loss2;
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    } else {
      for (int l = thrx;  l < N;  l+=numx) {
        const T_ACC c_h = static_cast<T_ACC>(k_input[l]);
        const T_ACC c_loss = static_cast<T_ACC>(k_dout[l]);
        T_ACC f_grad_input = fH * c_loss;
        f_grad_input -= sum_loss1;
        f_grad_input -= (c_h - c_mean) * c_rstd * sum_loss2;
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    }
    // prevent race where buf is written again before reads are done
    __syncthreads();
  }
}

template <typename T>
void LayerNormBackwardKernelImplInternal(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t M,
    int64_t N,
    Tensor* dX,
    Tensor* dgamma,
    Tensor* dbeta) {
  using T_ACC = acc_type<T, true>;
  TORCH_CHECK(dY.numel() == M * N);
  TORCH_CHECK(mean.numel() == M);
  TORCH_CHECK(rstd.numel() == M);
  TORCH_CHECK(M <= at::cuda::getCurrentDeviceProperties()->maxGridSize[0], "M should be less than maximum CUDA grid size, \
  file a support request to support bigger batches");
  TORCH_CHECK(N <= std::numeric_limits<int>::max(), "Normalized shape should have less than INT_MAX elements, \
  file a support request to support bigger normalized shapes");
  const T* dY_data = dY.template const_data_ptr<T>();
  const T* X_data = X.template const_data_ptr<T>();
  const T_ACC* mean_data = mean.template const_data_ptr<T_ACC>();
  const T_ACC* rstd_data = rstd.template const_data_ptr<T_ACC>();
  const T* gamma_data =
      gamma.defined() ? gamma.template const_data_ptr<T>() : nullptr;
  T* dX_data = dX->defined() ? dX->template data_ptr<T>() : nullptr;
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  const int warp_size = at::cuda::warp_size();
  if (dX_data != nullptr) {
#ifdef USE_ROCM
    if (M >= 32768) {
      const uint64_t maxGridY = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
      const dim3 blocks1(1, std::min((uint64_t)M, maxGridY), 1);
      dim3 threads1(warp_size, 4, 1);
      threads1.y = 2; // Optimization for ROCm
      int nshared =
              threads1.y > 1 ?
              threads1.y*threads1.x*sizeof(T_ACC) :
              0;
      cuComputeGradInput<<<blocks1, threads1, nshared, hip_stream>>>(
              dY_data,
              X_data,
              M, N,
              mean_data,
              rstd_data,
              gamma_data,
              dX_data);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      const dim3 blocks(M);
      int nshared = (num_threads()/warp_size) * sizeof(T_ACC);
      layer_norm_grad_input_kernel<<<blocks, num_threads(), nshared, hip_stream>>>(dY_data,
      X_data, mean_data, rstd_data, gamma_data, dX_data, N);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
#else
    const dim3 blocks(M);
    int nshared = (num_threads() / warp_size) * sizeof(T_ACC);

    bool bVectorSizeMultiple = (N % vec_size == 0);
    bool bTargetDataTypes = (std::is_same_v<T, float> || std::is_same_v<T, at::Half> ||
      std::is_same_v<T, at::BFloat16>);
    const unsigned int alignment = sizeof(T) * vec_size;
    bool bAlignedBuffers = can_vectorize(dY_data, alignment) && can_vectorize(X_data, alignment) &&
      can_vectorize(gamma_data, alignment) && can_vectorize(dX_data, alignment);

    if (bAlignedBuffers && bTargetDataTypes && bVectorSizeMultiple) {
      layer_norm_grad_input_kernel_vectorized<<<blocks, num_threads(), nshared, hip_stream>>>(dY_data,
          X_data, mean_data, rstd_data, gamma_data, dX_data, N);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      layer_norm_grad_input_kernel<<<blocks, num_threads(), nshared, hip_stream>>>(dY_data,
          X_data, mean_data, rstd_data, gamma_data, dX_data, N);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
#endif
  }

  if (dgamma->defined() || dbeta->defined()) {
    T* dgamma_data =
        dgamma->defined() ? dgamma->template data_ptr<T>() : nullptr;
    T* dbeta_data = dbeta->defined() ? dbeta->template data_ptr<T>() : nullptr;

#if defined(USE_ROCM)
    if (M < 128) {
      // For small batch size, do colwise reduce directly.
      const int64_t B = (N + kCUDANumThreads - 1) / kCUDANumThreads;
      GammaBetaBackwardSimpleCUDAKernel<T, T_ACC>
          <<<B, kCUDANumThreads, 0, hip_stream>>>(
              M,
              N,
              dY_data,
              X_data,
              mean_data,
              rstd_data,
              dgamma_data,
              dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      // For small batch size, do colwise reduce directly.
      const int part_size = warp_size;
      const dim3 threads2(warp_size, 4, 1);
      const dim3 blocks2((N + threads2.x - 1) / threads2.x, part_size, 1);
      const int nshared2_a = 2 * sizeof(T_ACC) * threads2.y * threads2.y * (threads2.x + 1);
      const int nshared2_b = threads2.x * threads2.y * sizeof(T_ACC);
      const int nshared2 = nshared2_a > nshared2_b ? nshared2_a : nshared2_b;

      const auto part_grad_dtype = at::toAccumulateType(X.scalar_type(), true);
      Tensor part_grad_gamma = at::empty({part_size,N}, gamma.options().dtype(part_grad_dtype));
      Tensor part_grad_beta = at::native::empty_like(part_grad_gamma);

      cuComputePartGradGammaBeta<<<blocks2, threads2, nshared2, hip_stream>>>(
                      dY_data,
                      X_data,
                      M,N,
                      mean_data,
                      rstd_data,
                      part_grad_gamma.template data_ptr<T_ACC>(),
                      part_grad_beta.template data_ptr<T_ACC>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();

      const dim3 threads3(warp_size, 8, 1); // Optimization for ROCm
      const dim3 blocks3((N + threads3.x - 1) / threads3.x, 1, 1);
      const int nshared3 = threads3.x * threads3.y * sizeof(T_ACC);

      cuComputeGradGammaBeta<<<blocks3, threads3, nshared3, hip_stream>>>(
                      part_grad_gamma.template data_ptr<T_ACC>(),
                      part_grad_beta.template data_ptr<T_ACC>(),
                      part_size,
                      M,N,
                      dgamma_data,
                      dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
#else
    LaunchGammaBetaBackwardCUDAKernel(
      dY_data, X_data, mean_data, rstd_data, M, N, dgamma, dbeta, hip_stream);
#endif
  }
}

void LayerNormBackwardKernelImpl(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t M,
    int64_t N,
    Tensor* dX,
    Tensor* dgamma,
    Tensor* dbeta) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      X.scalar_type(),
      "LayerNormBackwardKernelImpl",
      [&]() {
        LayerNormBackwardKernelImplInternal<scalar_t>(
            dY.contiguous(), X, mean, rstd, gamma, M, N, dX, dgamma, dbeta);
      });
}

} // namespace

std::tuple<Tensor, Tensor, Tensor> layer_norm_cuda(
    const Tensor& input,
    IntArrayRef normalized_shape,
    const std::optional<Tensor>& weight_opt /* optional */,
    const std::optional<Tensor>& bias_opt /* optional */,
    double eps) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> weight_maybe_owned =
      at::borrow_from_optional_tensor(weight_opt);
  const Tensor& weight = *weight_maybe_owned;
  c10::MaybeOwned<Tensor> bias_maybe_owned =
      at::borrow_from_optional_tensor(bias_opt);
  const Tensor& bias = *bias_maybe_owned;

  auto M_N = _check_layer_norm_inputs(input, normalized_shape, weight, bias);
  auto M = M_N.first;
  auto N = M_N.second;
  auto X = input.expect_contiguous();
  auto gamma = weight.expect_contiguous();
  auto beta = bias.expect_contiguous();

  Tensor Y = at::native::empty_like(
      *X,
      std::nullopt /* dtype */,
      std::nullopt /* layout */,
      std::nullopt /* device */,
      std::nullopt /* pin_memory */,
      LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto acc_type = at::toAccumulateType(input.scalar_type(), /*is_cuda=*/true);
  Tensor mean = at::empty({M}, X->options().dtype(acc_type));
  Tensor rstd = at::empty({M}, X->options().dtype(acc_type));
  // Calling the kernel for M==0 gives a CUDA error
  // See: https://github.com/pytorch/pytorch/pull/28614
  if (M > 0) {
    LayerNormKernelImpl(*X, *gamma, *beta, M, N, eps, &Y, &mean, &rstd);
  }
  const auto input_shape = input.sizes();
  const size_t axis = input.dim() - normalized_shape.size();

  std::vector<int64_t> stat_shape;
  for (const auto idx: c10::irange(axis)) {
    stat_shape.push_back(input_shape[idx]);
  }
  for ([[maybe_unused]] const auto idx : c10::irange(axis, input.dim())) {
    stat_shape.push_back(1);
  }

  mean = mean.view(stat_shape);
  rstd = rstd.view(stat_shape);

  return std::make_tuple(std::move(Y), std::move(mean), std::move(rstd));
}

std::tuple<Tensor, Tensor, Tensor> layer_norm_backward_cuda(
    const Tensor& dY,
    const Tensor& input,
    IntArrayRef normalized_shape,
    const Tensor& mean,
    const Tensor& rstd,
    const std::optional<Tensor>& weight_opt /* optional */,
    const std::optional<Tensor>& bias_opt /* optional */,
    std::array<bool, 3> grad_input_mask) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> weight_maybe_owned =
      at::borrow_from_optional_tensor(weight_opt);
  const Tensor& weight = *weight_maybe_owned;
  c10::MaybeOwned<Tensor> bias_maybe_owned =
      at::borrow_from_optional_tensor(bias_opt);
  const Tensor& bias = *bias_maybe_owned;

  auto M_N = _check_layer_norm_inputs(input, normalized_shape, weight, bias);
  auto M = M_N.first;
  auto N = M_N.second;
  auto X = input.expect_contiguous();
  auto gamma = weight.expect_contiguous();
  auto beta = bias.expect_contiguous();

  Tensor dX;
  Tensor dgamma;
  Tensor dbeta;
  if (grad_input_mask[0]) {
    dX = at::native::empty_like(
        *X,
        std::nullopt /* dtype */,
        std::nullopt /* layout */,
        std::nullopt /* device */,
        std::nullopt /* pin_memory */,
        LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  }
  if (grad_input_mask[1]) {
    dgamma = M > 0 ? at::native::empty_like(
                         *gamma,
                         std::nullopt /* dtype */,
                         std::nullopt /* layout */,
                         std::nullopt /* device */,
                         std::nullopt /* pin_memory */,
                         LEGACY_CONTIGUOUS_MEMORY_FORMAT)
                   : at::native::zeros_like(
                         *gamma,
                         std::nullopt /* dtype */,
                         std::nullopt /* layout */,
                         std::nullopt /* device */,
                         std::nullopt /* pin_memory */,
                         LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  }
  if (grad_input_mask[2]) {
    dbeta = M > 0 ? at::native::empty_like(
                        *beta,
                        std::nullopt /* dtype */,
                        std::nullopt /* layout */,
                        std::nullopt /* device */,
                        std::nullopt /* pin_memory */,
                        LEGACY_CONTIGUOUS_MEMORY_FORMAT)
                  : at::native::zeros_like(
                        *beta,
                        std::nullopt /* dtype */,
                        std::nullopt /* layout */,
                        std::nullopt /* device */,
                        std::nullopt /* pin_memory */,
                        LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  }
  if (M > 0 && N > 0) {
    LayerNormBackwardKernelImpl(
        dY, *X, mean, rstd, *gamma, M, N, &dX, &dgamma, &dbeta);
  }
  return std::make_tuple(std::move(dX), std::move(dgamma), std::move(dbeta));
}

REGISTER_DISPATCH(LayerNormKernel, &LayerNormKernelImpl)
REGISTER_DISPATCH(LayerNormBackwardKernel, &LayerNormBackwardKernelImpl)

} // namespace at::native
