#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <c10/macros/Macros.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/_thnn_fused_lstm_cell_native.h>
#include <ATen/ops/_thnn_fused_lstm_cell_backward_impl_native.h>
#include <ATen/ops/_thnn_fused_gru_cell_native.h>
#include <ATen/ops/_thnn_fused_gru_cell_backward_native.h>
#endif

namespace at::native {

namespace {

using at::cuda::detail::TensorInfo;
using at::cuda::detail::getTensorInfo;
using at::cuda::detail::IndexToOffset;
using at::cuda::detail::canUse32BitIndexMath;

// Factor will be 3 for GRU and 4 for LSTM
void checkSizes(CheckedFrom c,
                const TensorArg& input_gates, const TensorArg& hidden_gates,
                const TensorArg& input_bias, const TensorArg& hidden_bias,
                int64_t factor, const TensorArg& prev_hidden) {
  checkDim(c, input_gates, 2);
  checkSameSize(c, input_gates, hidden_gates);
  int64_t gates_size = input_gates->size(1);

  if (input_bias->defined()) {
    checkDim(c, input_bias, 1);
    checkNumel(c, input_bias, gates_size);
    checkSameSize(c, input_bias, hidden_bias);
  }

  checkDim(c, prev_hidden, 2);
  checkNumel(c, prev_hidden, input_gates->size(0) * gates_size / factor);

  checkAllSameGPU(c, {input_gates, hidden_gates, input_bias, hidden_bias, prev_hidden});
}

bool allContiguous(at::TensorList tensors) {
  return std::all_of(tensors.begin(), tensors.end(),
                     [](const at::Tensor& t) { return !t.defined() || t.is_contiguous(); });
}

void getLaunchConfig(dim3* block, dim3* grid, int64_t numel) {
  c10::DeviceIndex curDevice = -1;
  AT_CUDA_CHECK(c10::cuda::GetDevice(&curDevice));
  *block = cuda::getApplyBlock();
  TORCH_INTERNAL_ASSERT(cuda::getApplyGrid(numel, *grid, curDevice),
                        "Could not get grid size for pointwise apply.");
}

template<typename T, typename T2>
TensorInfo<T, T2> tryGetTensorInfo(const at::Tensor& t) {
  return t.defined() ? getTensorInfo<T, T2>(t) : TensorInfo<T, T2>{};
}

void collapseDims() {};
template<typename T, typename T2, typename... Args>
void collapseDims(TensorInfo<T, T2>& info, Args&... infos) {
  info.collapseDims();
  collapseDims(infos...);
}

#define DEVICE_LINEAR_GET(D_TENSOR, INDEX)                              \
  D_TENSOR.data[IndexToOffset<scalar_t, index_type, indexing_kind>::get(INDEX, D_TENSOR)]

// Biases are always 1D
#define DEVICE_BIAS_GET(D_TENSOR, INDEX)                              \
  D_TENSOR.data[IndexToOffset<scalar_t, index_type, 1>::get(INDEX, D_TENSOR)]

#define H2F(input) static_cast<accscalar_t>(input)
#define F2H(input) static_cast<scalar_t>(input)

template<typename T>
__device__ __forceinline__
T sigmoid(T in)  {
  T one = static_cast<T>(1.0);
  return one / (one + ::exp(-in));
}

namespace kernel {

template <typename scalar_t, typename accscalar_t, typename index_type, int indexing_kind>
#if __CUDA_ARCH__ >= 350 || defined(USE_ROCM)
C10_LAUNCH_BOUNDS_2(512, 4)
#endif
__global__ void lstm_cell_forward(
            TensorInfo<scalar_t, index_type> input,
            TensorInfo<scalar_t, index_type> hidden,
            TensorInfo<scalar_t, index_type> bias1,
            TensorInfo<scalar_t, index_type> bias2,
            TensorInfo<scalar_t, index_type> _cx,
            TensorInfo<scalar_t, index_type> _hy,
            TensorInfo<scalar_t, index_type> _cy,
            TensorInfo<scalar_t, index_type> workspace,
            index_type hsz,
            index_type totalElements) {
    bool has_bias = bias1.data != nullptr;
    for (index_type linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalElements;
       linearIndex += gridDim.x * blockDim.x) {
      index_type offset = (linearIndex/hsz)*4*hsz+linearIndex%hsz;

      scalar_t iig = DEVICE_LINEAR_GET(input, offset+0*hsz);
      scalar_t ifg = DEVICE_LINEAR_GET(input, offset+1*hsz);
      scalar_t icg = DEVICE_LINEAR_GET(input, offset+2*hsz);
      scalar_t iog = DEVICE_LINEAR_GET(input, offset+3*hsz);

      scalar_t hig = DEVICE_LINEAR_GET(hidden, offset+0*hsz);
      scalar_t hfg = DEVICE_LINEAR_GET(hidden, offset+1*hsz);
      scalar_t hcg = DEVICE_LINEAR_GET(hidden,  offset+2*hsz);
      scalar_t hog = DEVICE_LINEAR_GET(hidden,  offset+3*hsz);

      scalar_t* wig = &DEVICE_LINEAR_GET(workspace, offset+0*hsz);
      scalar_t* wfg = &DEVICE_LINEAR_GET(workspace, offset+1*hsz);
      scalar_t* wcg = &DEVICE_LINEAR_GET(workspace, offset+2*hsz);
      scalar_t* wog = &DEVICE_LINEAR_GET(workspace, offset+3*hsz);

      scalar_t cx = DEVICE_LINEAR_GET(_cx, linearIndex);

      scalar_t* hy = &DEVICE_LINEAR_GET(_hy, linearIndex);
      scalar_t* cy = &DEVICE_LINEAR_GET(_cy, linearIndex);

      scalar_t b1i, b1f, b1c, b1o;
      scalar_t b2i, b2f, b2c, b2o;

      if (has_bias) {
        b1i = DEVICE_BIAS_GET(bias1, linearIndex % hsz + 0 * hsz);
        b1f = DEVICE_BIAS_GET(bias1, linearIndex % hsz + 1 * hsz);
        b1c = DEVICE_BIAS_GET(bias1, linearIndex % hsz + 2 * hsz);
        b1o = DEVICE_BIAS_GET(bias1, linearIndex % hsz + 3 * hsz);

        b2i = DEVICE_BIAS_GET(bias2, linearIndex % hsz + 0 * hsz);
        b2f = DEVICE_BIAS_GET(bias2, linearIndex % hsz + 1 * hsz);
        b2c = DEVICE_BIAS_GET(bias2, linearIndex % hsz + 2 * hsz);
        b2o = DEVICE_BIAS_GET(bias2, linearIndex % hsz + 3 * hsz);
      } else {
#ifndef THC_REAL_IS_HALF
        b1i = 0.0; b1f = 0.0; b1c = 0.0; b1o = 0.0;
        b2i = 0.0; b2f = 0.0; b2c = 0.0; b2o = 0.0;
#else
        b1i = F2H(0.0); b1f = F2H(0.0); b1c = F2H(0.0); b1o = F2H(0.0);
        b2i = F2H(0.0); b2f = F2H(0.0); b2c = F2H(0.0); b2o = F2H(0.0);
#endif
      }

      accscalar_t ig, fg, cg, og;
      accscalar_t f_hy, f_cy;

      ig = sigmoid(H2F(iig) + H2F(hig) + H2F(b1i) + H2F(b2i));
      fg = sigmoid(H2F(ifg) + H2F(hfg) + H2F(b1f) + H2F(b2f));
      cg = ::tanh(H2F(icg) + H2F(hcg) + H2F(b1c) + H2F(b2c));
      og = sigmoid(H2F(iog) + H2F(hog) + H2F(b1o) + H2F(b2o));

      f_cy = (fg * H2F(cx)) + (ig * cg);
      f_hy = og * ::tanh(f_cy);

      *hy = F2H(f_hy);
      *cy = F2H(f_cy);

      //SAVE FOR BACKWARDS
      //Also need cy and cx but can be saved easily in python
      *wig = F2H(ig);
      *wfg = F2H(fg);
      *wcg = F2H(cg);
      *wog = F2H(og);
    }
}

template <typename scalar_t, typename accscalar_t, typename index_type, int indexing_kind>
#if __CUDA_ARCH__ >= 350 || defined(USE_ROCM)
C10_LAUNCH_BOUNDS_2(512, 4)
#endif
__global__ void lstm_cell_backward(
              TensorInfo<scalar_t, index_type> storage,
              TensorInfo<scalar_t, index_type> gradInGates,
              TensorInfo<scalar_t, index_type> _cx,
              TensorInfo<scalar_t, index_type> _cy,
              TensorInfo<scalar_t, index_type> gradoutput,
              TensorInfo<scalar_t, index_type> gradoutputcell,
              TensorInfo<scalar_t, index_type> gradInputCx,
              index_type hsz,
              index_type totalElements) {
  bool has_gradoutput = gradoutput.data != nullptr;
  bool has_gradoutputcell = gradoutputcell.data != nullptr;
  for (index_type linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalElements;
       linearIndex += gridDim.x * blockDim.x) {
    index_type offset = (linearIndex/hsz)*4*hsz+linearIndex%hsz;

    scalar_t ig = DEVICE_LINEAR_GET(storage, offset+0*hsz);
    scalar_t fg = DEVICE_LINEAR_GET(storage, offset+1*hsz);
    scalar_t cg = DEVICE_LINEAR_GET(storage, offset+2*hsz);
    scalar_t og = DEVICE_LINEAR_GET(storage, offset+3*hsz);

    scalar_t* ih = &DEVICE_LINEAR_GET(gradInGates, offset+0*hsz);
    scalar_t* fh = &DEVICE_LINEAR_GET(gradInGates, offset+1*hsz);
    scalar_t* ch = &DEVICE_LINEAR_GET(gradInGates, offset+2*hsz);
    scalar_t* oh = &DEVICE_LINEAR_GET(gradInGates, offset+3*hsz);

    //will return hidden grads here
    scalar_t cx = DEVICE_LINEAR_GET(_cx, linearIndex);
    scalar_t cy = DEVICE_LINEAR_GET(_cy, linearIndex);

    scalar_t* gi = &DEVICE_LINEAR_GET(gradInputCx, linearIndex);

    accscalar_t go  = has_gradoutput ? H2F(DEVICE_LINEAR_GET(gradoutput, linearIndex)) : 0.f;
    accscalar_t goc = has_gradoutputcell ? H2F(DEVICE_LINEAR_GET(gradoutputcell, linearIndex)) : 0.f;

    accscalar_t gcx = ::tanh(H2F(cy));

    accscalar_t gog = go * gcx;
    gcx = go * H2F(og) * (1 - gcx*gcx) + goc;

    accscalar_t gig = gcx * H2F(cg);
    accscalar_t gfg = gcx * H2F(cx);
    accscalar_t gcg = gcx * H2F(ig);

    gcx = gcx * H2F(fg);

    gig = gig * (1-H2F(ig)) * H2F(ig);
    gfg = gfg * (1-H2F(fg)) * H2F(fg);
    gcg = gcg * (1-H2F(cg)*H2F(cg));
    gog = gog * (1-H2F(og)) * H2F(og);

    *ih = F2H(gig);
    *fh = F2H(gfg);
    *ch = F2H(gcg);
    *oh = F2H(gog);

    *gi = F2H(gcx);
  }
}

template <typename scalar_t, typename accscalar_t, typename index_type, int indexing_kind>
#if __CUDA_ARCH__ >= 350 || defined(USE_ROCM)
C10_LAUNCH_BOUNDS_2(512, 4)
#endif
__global__ void gru_cell_forward(
            TensorInfo<scalar_t, index_type> Input,
            TensorInfo<scalar_t, index_type> Hidden,
            TensorInfo<scalar_t, index_type> Bias1,
            TensorInfo<scalar_t, index_type> Bias2,
            TensorInfo<scalar_t, index_type> _hx,
            TensorInfo<scalar_t, index_type> _hy,
            TensorInfo<scalar_t, index_type> storage,
            index_type hsz,
            index_type totalElements) {
  bool has_bias = Bias1.data != nullptr;
  for (index_type linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalElements;
       linearIndex += gridDim.x * blockDim.x) {
      index_type offset = (linearIndex/hsz)*3*hsz+linearIndex%hsz;

      scalar_t ir = DEVICE_LINEAR_GET(Input, offset+0*hsz);
      scalar_t ii = DEVICE_LINEAR_GET(Input, offset+1*hsz);
      scalar_t in = DEVICE_LINEAR_GET(Input, offset+2*hsz);
      scalar_t hr = DEVICE_LINEAR_GET(Hidden,offset+0*hsz);
      scalar_t hi = DEVICE_LINEAR_GET(Hidden,offset+1*hsz);
      scalar_t hn = DEVICE_LINEAR_GET(Hidden,  offset+2*hsz);

      scalar_t hx = DEVICE_LINEAR_GET(_hx, linearIndex);
      scalar_t* hy = &DEVICE_LINEAR_GET(_hy, linearIndex);

      scalar_t b1r, b1i, b1n, b2r, b2i, b2n;

      if (has_bias) {
        b1r = DEVICE_BIAS_GET(Bias1, linearIndex%hsz+0*hsz);
        b1i = DEVICE_BIAS_GET(Bias1, linearIndex%hsz+1*hsz);
        b1n = DEVICE_BIAS_GET(Bias1, linearIndex%hsz+2*hsz);

        b2r = DEVICE_BIAS_GET(Bias2, linearIndex%hsz+0*hsz);
        b2i = DEVICE_BIAS_GET(Bias2, linearIndex%hsz+1*hsz);
        b2n = DEVICE_BIAS_GET(Bias2, linearIndex%hsz+2*hsz);
      } else {
#ifndef THC_REAL_IS_HALF
        b1r = 0.0; b1i = 0.0; b1n = 0.0;
        b2r = 0.0; b2i = 0.0; b2n = 0.0;
#else
        b1r = F2H(0.0); b1i = F2H(0.0); b1n = F2H(0.0);
        b2r = F2H(0.0); b2i = F2H(0.0); b2n = F2H(0.0);
#endif
      }

      offset = (linearIndex/hsz)*5*hsz+linearIndex%hsz;

      accscalar_t rg, ig, ng;

      rg = sigmoid(H2F(ir) + H2F(hr) + H2F(b1r) + H2F(b2r));
      ig = sigmoid(H2F(ii) + H2F(hi) + H2F(b1i) + H2F(b2i));

      ng = H2F(in) + H2F(b1n) + rg*( H2F(hn)+H2F(b2n) );
      ng = ::tanh(ng);
      *hy = F2H( ng + ig * ( H2F(hx)-ng ) );

      //SAVE FOR BACKWARDS
      DEVICE_LINEAR_GET(storage, offset+0*hsz) = F2H(rg);
      DEVICE_LINEAR_GET(storage, offset+1*hsz) = F2H(ig);
      DEVICE_LINEAR_GET(storage, offset+2*hsz) = F2H(ng);
      DEVICE_LINEAR_GET(storage, offset+3*hsz) = hx;
      DEVICE_LINEAR_GET(storage, offset+4*hsz) = F2H(H2F(hn) + H2F(b2n));
    }
}

template <typename scalar_t, typename accscalar_t, typename index_type, int indexing_kind>
#if __CUDA_ARCH__ >= 350 || defined(USE_ROCM)
C10_LAUNCH_BOUNDS_2(512, 4)
#endif
__global__ void gru_cell_backward(
             TensorInfo<scalar_t, index_type> gradInInput,
             TensorInfo<scalar_t, index_type> gradInHidden,
             TensorInfo<scalar_t, index_type> gradOutput,
             TensorInfo<scalar_t, index_type> gradInputHx,
             TensorInfo<scalar_t, index_type> storage,
             index_type hsz,
             index_type totalElements) {
  for (index_type linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalElements;
       linearIndex += gridDim.x * blockDim.x) {
    index_type offset = (linearIndex/hsz)*5*hsz+linearIndex%hsz;

    scalar_t rg = DEVICE_LINEAR_GET(storage, offset+0*hsz);
    scalar_t ig = DEVICE_LINEAR_GET(storage, offset+1*hsz);
    scalar_t ng = DEVICE_LINEAR_GET(storage, offset+2*hsz);
    scalar_t hx = DEVICE_LINEAR_GET(storage, offset+3*hsz);
    scalar_t hn = DEVICE_LINEAR_GET(storage, offset+4*hsz);

    scalar_t go = DEVICE_LINEAR_GET(gradOutput, linearIndex);

    offset = (linearIndex/hsz)*3*hsz+linearIndex%hsz;

    accscalar_t gig = H2F(go)*( H2F(hx)-H2F(ng) )*( 1-H2F(ig) )*H2F(ig);
    accscalar_t ghx = H2F(go)*H2F(ig);
    accscalar_t gin = H2F(go)*( 1-H2F(ig) )*( 1-H2F(ng)*H2F(ng) );
    accscalar_t ghn = gin * H2F(rg);
    accscalar_t grg = gin *H2F(hn)*( 1-H2F(rg) )*H2F(rg);

    DEVICE_LINEAR_GET(gradInInput, offset+0*hsz) = F2H(grg);
    DEVICE_LINEAR_GET(gradInInput, offset+1*hsz) = F2H(gig);
    DEVICE_LINEAR_GET(gradInInput, offset+2*hsz) = F2H(gin);

    DEVICE_LINEAR_GET(gradInHidden, offset+0*hsz) = F2H(grg);
    DEVICE_LINEAR_GET(gradInHidden, offset+1*hsz) = F2H(gig);
    DEVICE_LINEAR_GET(gradInHidden, offset+2*hsz) = F2H(ghn);
    DEVICE_LINEAR_GET(gradInputHx, linearIndex) = F2H(ghx);
  }
}

#undef DEVICE_LINEAR_GET
#undef DEVICE_BIAS_GET
#undef H2F
#undef F2H

} // namespace kernel

template<typename scalar_t, typename index_type>
void lstm_forward_impl(const Tensor& input_gates, const Tensor& hidden_gates,
                       const Tensor& input_bias, const Tensor& hidden_bias,
                       const Tensor& cx,
                       const Tensor& hy, const Tensor& cy, const Tensor& workspace) {
  using accscalar_t = acc_type<scalar_t, /*is_cuda=*/true>;

  dim3 block, grid;
  int64_t numel = cx.numel();
  if (numel == 0) return;
  getLaunchConfig(&block, &grid, numel);

  auto input_gatesI = getTensorInfo<scalar_t, index_type>(input_gates);
  auto hidden_gatesI = getTensorInfo<scalar_t, index_type>(hidden_gates);
  auto input_biasI = tryGetTensorInfo<scalar_t, index_type>(input_bias);
  auto hidden_biasI = tryGetTensorInfo<scalar_t, index_type>(hidden_bias);
  auto cxI = getTensorInfo<scalar_t, index_type>(cx);
  auto hyI = getTensorInfo<scalar_t, index_type>(hy);
  auto cyI = getTensorInfo<scalar_t, index_type>(cy);
  auto workspaceI = getTensorInfo<scalar_t, index_type>(workspace);
  index_type hidden_size = cxI.sizes[cxI.dims-1];

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  if (allContiguous({input_gates, hidden_gates, input_bias, hidden_bias, cx, hy, cy, workspace})) {
    collapseDims(input_gatesI, hidden_gatesI, input_biasI, hidden_biasI, cxI, hyI, cyI, workspaceI);
    kernel::lstm_cell_forward<scalar_t, accscalar_t, index_type, 1>
      <<<grid, block, 0, stream>>>
        (input_gatesI, hidden_gatesI, input_biasI, hidden_biasI, cxI, hyI, cyI, workspaceI, hidden_size, numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    kernel::lstm_cell_forward<scalar_t, accscalar_t, index_type, 2>
      <<<grid, block, 0, stream>>>
        (input_gatesI, hidden_gatesI, input_biasI, hidden_biasI, cxI, hyI, cyI, workspaceI, hidden_size, numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

template<typename scalar_t, typename index_type>
void lstm_backward_impl(const Tensor& grad_hy, const Tensor& grad_cy,
                        const Tensor& cx, const Tensor& cy,
                        const Tensor& workspace,
                        const Tensor& grad_gates, const Tensor& grad_cx) {
  using accscalar_t = acc_type<scalar_t, /*is_cuda=*/true>;

  dim3 block, grid;
  int64_t numel = cx.numel();
  getLaunchConfig(&block, &grid, numel);
  if (numel == 0) return;

  auto grad_hyI = tryGetTensorInfo<scalar_t, index_type>(grad_hy);
  auto grad_cyI = tryGetTensorInfo<scalar_t, index_type>(grad_cy);
  auto cxI = getTensorInfo<scalar_t, index_type>(cx);
  auto cyI = getTensorInfo<scalar_t, index_type>(cy);
  auto workspaceI = getTensorInfo<scalar_t, index_type>(workspace);
  auto grad_gatesI = getTensorInfo<scalar_t, index_type>(grad_gates);
  auto grad_cxI = getTensorInfo<scalar_t, index_type>(grad_cx);
  index_type hidden_size = cxI.sizes[cxI.dims-1];

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  if (allContiguous({grad_hy, grad_cy, cx, cy, workspace, grad_gates, grad_cx})) {
    collapseDims(grad_hyI, grad_cyI, cxI, cyI, workspaceI, grad_gatesI, grad_cxI);
    kernel::lstm_cell_backward<scalar_t, accscalar_t, index_type, 1>
      <<<grid, block, 0, stream>>>
        (workspaceI, grad_gatesI, cxI, cyI, grad_hyI, grad_cyI, grad_cxI, hidden_size, numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    kernel::lstm_cell_backward<scalar_t, accscalar_t, index_type, 2>
      <<<grid, block, 0, stream>>>
        (workspaceI, grad_gatesI, cxI, cyI, grad_hyI, grad_cyI, grad_cxI, hidden_size, numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

template<typename scalar_t, typename index_type>
void gru_forward_impl(const Tensor& input_gates, const Tensor& hidden_gates,
                      const Tensor& input_bias, const Tensor& hidden_bias,
                      const Tensor& hx,
                      const Tensor& hy, const Tensor& workspace) {
  using accscalar_t = acc_type<scalar_t, /*is_cuda=*/true>;

  dim3 block, grid;
  int64_t numel = hx.numel();
  if (numel == 0) return;
  getLaunchConfig(&block, &grid, numel);

  auto input_gatesI = getTensorInfo<scalar_t, index_type>(input_gates);
  auto hidden_gatesI = getTensorInfo<scalar_t, index_type>(hidden_gates);
  auto input_biasI = tryGetTensorInfo<scalar_t, index_type>(input_bias);
  auto hidden_biasI = tryGetTensorInfo<scalar_t, index_type>(hidden_bias);
  auto hxI = getTensorInfo<scalar_t, index_type>(hx);
  auto hyI = getTensorInfo<scalar_t, index_type>(hy);
  auto workspaceI = getTensorInfo<scalar_t, index_type>(workspace);
  index_type hidden_size = hxI.sizes[hxI.dims-1];

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  if (allContiguous({input_gates, hidden_gates, input_bias, hidden_bias, hx, hy, workspace})) {
    collapseDims(input_gatesI, hidden_gatesI, input_biasI, hidden_biasI, hxI, hyI, workspaceI);
    kernel::gru_cell_forward<scalar_t, accscalar_t, index_type, 1>
      <<<grid, block, 0, stream>>>
        (input_gatesI, hidden_gatesI, input_biasI, hidden_biasI, hxI, hyI, workspaceI, hidden_size, numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    kernel::gru_cell_forward<scalar_t, accscalar_t, index_type, 2>
      <<<grid, block, 0, stream>>>
        (input_gatesI, hidden_gatesI, input_biasI, hidden_biasI, hxI, hyI, workspaceI, hidden_size, numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

template<typename scalar_t, typename index_type>
void gru_backward_impl(const Tensor& grad_hy, const Tensor& workspace,
                       const Tensor& grad_input_gates, const Tensor& grad_hidden_gates, const Tensor& grad_hx) {
  using accscalar_t = acc_type<scalar_t, /*is_cuda=*/true>;

  dim3 block, grid;
  int64_t numel = grad_hy.numel();
  if (numel == 0) return;
  getLaunchConfig(&block, &grid, numel);

  auto grad_hyI = getTensorInfo<scalar_t, index_type>(grad_hy);
  auto workspaceI = getTensorInfo<scalar_t, index_type>(workspace);
  auto grad_input_gatesI = getTensorInfo<scalar_t, index_type>(grad_input_gates);
  auto grad_hidden_gatesI = getTensorInfo<scalar_t, index_type>(grad_hidden_gates);
  auto grad_hxI = getTensorInfo<scalar_t, index_type>(grad_hx);
  index_type hidden_size = grad_hyI.sizes[grad_hyI.dims-1];

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  if (allContiguous({grad_hy, workspace, grad_input_gates, grad_hidden_gates, grad_hx})) {
    collapseDims(grad_hyI, workspaceI, grad_input_gatesI, grad_hidden_gatesI, grad_hxI);
    kernel::gru_cell_backward<scalar_t, accscalar_t, index_type, 1>
      <<<grid, block, 0, stream>>>
        (grad_input_gatesI, grad_hidden_gatesI, grad_hyI, grad_hxI, workspaceI, hidden_size, numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    kernel::gru_cell_backward<scalar_t, accscalar_t, index_type, 2>
      <<<grid, block, 0, stream>>>
        (grad_input_gatesI, grad_hidden_gatesI, grad_hyI, grad_hxI, workspaceI, hidden_size, numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

} // anonymous namespace

// Note [64-bit index math check elision]
// It's enough to perform the check for 64-bit math on the largest tensor only.
// If 32-bit is enough for it, it will suffice for all other tensors too, and we
// can save some work using this trick.

std::tuple<Tensor, Tensor, Tensor> _thnn_fused_lstm_cell_cuda(
      const Tensor& input_gates, const Tensor& hidden_gates,
      const Tensor& cx, const std::optional<Tensor>& input_bias_opt, const std::optional<Tensor>& hidden_bias_opt) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> input_bias_maybe_owned = at::borrow_from_optional_tensor(input_bias_opt);
  const Tensor& input_bias = *input_bias_maybe_owned;
  const Tensor& hidden_bias = hidden_bias_opt.value_or(Tensor());

  checkSizes("_thnn_fused_lstm_cell_cuda",
             {input_gates, "input_gates", 1}, {hidden_gates, "hidden_gates", 2},
             {input_bias, "input_bias", 3}, {hidden_bias, "hidden_bias", 4},
             /*factor=*/4, {cx, "prev_hidden", 5});

  auto workspace = at::empty_like(input_gates, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto hy = at::empty_like(cx, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto cy = at::empty_like(cx, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    input_gates.scalar_type(),
    "_thnn_fused_lstm_cell_cuda",
    [&] {
      if (canUse32BitIndexMath(workspace)) { // See Note [64-bit index math check elision]
        lstm_forward_impl<scalar_t, int32_t>(input_gates, hidden_gates, input_bias, hidden_bias, cx, hy, cy, workspace);
      } else {
        lstm_forward_impl<scalar_t, int64_t>(input_gates, hidden_gates, input_bias, hidden_bias, cx, hy, cy, workspace);
      }
  });
  return std::make_tuple(std::move(hy), std::move(cy), std::move(workspace));
}

void checkLSTMBackwardSizes(const TensorArg& grad_hy, const TensorArg& grad_cy,
                            const TensorArg& cx, const TensorArg& cy,
                            const TensorArg& workspace) {
  CheckedFrom c = "fused_lstm_cell_backward";
  const TensorArg& defined_grad = grad_hy->defined() ? grad_hy : grad_cy;
  checkDim(c, defined_grad, 2);
  auto exp_size = defined_grad->sizes();
  if (grad_hy->defined()) {
    checkSize(c, grad_hy, exp_size);
  }
  if (grad_cy->defined()) {
    checkSize(c, grad_cy, exp_size);
  }
  checkSize(c, cx, exp_size);
  checkSize(c, cy, exp_size);
  checkDim(c, workspace, 2);
  checkNumel(c, workspace, exp_size[0] * exp_size[1] * 4);
}

std::tuple<Tensor, Tensor, Tensor> _thnn_fused_lstm_cell_backward_impl_cuda( const std::optional<Tensor>& grad_hy_opt, const std::optional<Tensor>& grad_cy_opt,
      const Tensor& cx, const Tensor& cy,
      const Tensor& workspace, bool has_bias) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> grad_hy_maybe_owned = at::borrow_from_optional_tensor(grad_hy_opt);
  const Tensor& grad_hy = *grad_hy_maybe_owned;
  const Tensor& grad_cy = grad_cy_opt.value_or(Tensor());

  if (!grad_hy.defined() && !grad_cy.defined()) {
    return std::tuple<Tensor, Tensor, Tensor>();
  }
  checkLSTMBackwardSizes({grad_hy, "grad_hy", 1}, {grad_cy, "grad_cy", 2},
                         {cx, "cx", 3}, {cy, "cy", 4},
                         {workspace, "workspace", 5});

  auto grad_gates = at::empty_like(workspace, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto grad_cx = at::empty_like(cx, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    workspace.scalar_type(),
    "_thnn_fused_lstm_cell_cuda_backward",
    [&] {
      if (canUse32BitIndexMath(workspace)) { // See Note [64-bit index math check elision]
        lstm_backward_impl<scalar_t, int32_t>(grad_hy, grad_cy, cx, cy, workspace, grad_gates, grad_cx);
      } else {
        lstm_backward_impl<scalar_t, int64_t>(grad_hy, grad_cy, cx, cy, workspace, grad_gates, grad_cx);
      }
  });

  auto grad_bias = has_bias ? grad_gates.sum(0, /*keepdim=*/false) : at::Tensor{};
  return std::make_tuple(std::move(grad_gates), std::move(grad_cx), std::move(grad_bias));
}

static constexpr int64_t GRU_WORKSPACE_MULTIPLIER = 5;

std::tuple<Tensor, Tensor> _thnn_fused_gru_cell_cuda(
      const Tensor& input_gates, const Tensor& hidden_gates,
      const Tensor& hx, const std::optional<Tensor>& input_bias_opt, const std::optional<Tensor>& hidden_bias_opt) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> input_bias_maybe_owned = at::borrow_from_optional_tensor(input_bias_opt);
  const Tensor& input_bias = *input_bias_maybe_owned;
  const Tensor& hidden_bias = hidden_bias_opt.value_or(Tensor());

  checkSizes("_thnn_fused_gru_cell_cuda",
             {input_gates, "input_gates", 1}, {hidden_gates, "hidden_gates", 2},
             {input_bias, "input_bias", 3}, {hidden_bias, "hidden_bias", 4},
             /*factor=*/3, {hx, "prev_hidden", 5});

  auto workspace = at::empty({hx.size(0), hx.size(1) * GRU_WORKSPACE_MULTIPLIER}, hx.options());
  auto hy = at::empty_like(hx, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    input_gates.scalar_type(),
    "_thnn_fused_gru_cell_cuda",
    [&] {
      if (canUse32BitIndexMath(workspace)) { // See Note [64-bit index math check elision]
        gru_forward_impl<scalar_t, int32_t>(input_gates, hidden_gates, input_bias, hidden_bias, hx, hy, workspace);
      } else {
        gru_forward_impl<scalar_t, int64_t>(input_gates, hidden_gates, input_bias, hidden_bias, hx, hy, workspace);
      }
  });
  return std::make_tuple(std::move(hy), std::move(workspace));
}

void checkGRUBackwardSizes(const TensorArg& grad_hy, const TensorArg& workspace) {
  CheckedFrom c = "fused_gru_cell_backward";
  checkDim(c, grad_hy, 2);
  checkSize(c, workspace, {grad_hy->size(0), grad_hy->size(1) * GRU_WORKSPACE_MULTIPLIER});
}

std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor> _thnn_fused_gru_cell_backward_cuda(
      const Tensor& grad_hy, const Tensor& workspace, bool has_bias) {
  checkGRUBackwardSizes({grad_hy, "grad_hy", 1}, {workspace, "workspace", 2});

  int64_t hidden_size = workspace.size(1) / GRU_WORKSPACE_MULTIPLIER;
  auto grad_input_gates = at::empty({workspace.size(0), hidden_size * 3}, workspace.options());
  auto grad_hidden_gates = at::empty({workspace.size(0), hidden_size * 3}, workspace.options());
  auto grad_hx = at::empty_like(grad_hy, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    grad_hy.scalar_type(),
    "_thnn_fused_gru_cell_cuda_backward",
    [&] {
      if (canUse32BitIndexMath(workspace)) { // See Note [64-bit index math check elision]
        gru_backward_impl<scalar_t, int32_t>(grad_hy, workspace, grad_input_gates, grad_hidden_gates, grad_hx);
      } else {
        gru_backward_impl<scalar_t, int64_t>(grad_hy, workspace, grad_input_gates, grad_hidden_gates, grad_hx);
      }
  });

  at::Tensor grad_input_bias, grad_hidden_bias;
  if (has_bias) {
    grad_input_bias = grad_input_gates.sum(0, /*keepdim=*/false);
    grad_hidden_bias = grad_hidden_gates.sum(0, /*keepdim=*/false);
  }

  return std::make_tuple(
    std::move(grad_input_gates),
    std::move(grad_hidden_gates),
    std::move(grad_hx),
    std::move(grad_input_bias),
    std::move(grad_hidden_bias)
  );
}

} // namespace at::native
