#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <c10/cuda/CUDAException.h>
#include <c10/macros/Macros.h>
#include <ATen/native/IndexingUtils.h>
#include <ATen/native/Resize.h>
#include <ATen/native/cuda/block_reduce.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/nll_loss2d_forward_native.h>
#include <ATen/ops/nll_loss2d_backward_native.h>
#endif

namespace at::native {

namespace {

// Returns a contiguous tensor if the source tensor
// is defined. Otherwise returns the undefined
// source tensor unmodified.
inline Tensor optional_contiguous(const Tensor& source) {
  return source.defined() ? source.contiguous() : source;
}

// Returns the address of the first element of a tensor
// or nullptr if the tensor is undefined.
template <typename scalar_t>
inline const scalar_t* optional_data(const Tensor& source) {
  return source.defined() ? source.const_data_ptr<scalar_t>() : nullptr;
}

using at::cuda::detail::CUDA_NUM_THREADS;
using at::cuda::detail::GET_BLOCKS;

// TODO(crcrpar): Think about introducing `canUse32BitIndexMath` and choose int or int64_t for `target`.
template <typename scalar_t>
C10_LAUNCH_BOUNDS_1(CUDA_NUM_THREADS)
__global__ void nll_loss2d_forward_no_reduce_kernel(
  int64_t n_threads,
  PackedTensorAccessor64<scalar_t, 4> input,
  PackedTensorAccessor64<int64_t, 3> target,
  PackedTensorAccessor64<scalar_t, 3> output,
  const scalar_t* weight,
  int64_t ignore_index
) {
  int64_t batch_size = input.size(0);
  int64_t n_classes = input.size(1);
  int64_t H = input.size(2);
  int64_t W = input.size(3);

  CUDA_KERNEL_LOOP(index, n_threads) {
    const int64_t b = index % batch_size;
    const int64_t h = (index / batch_size) % H;
    const int64_t w = (index / (batch_size * H)) % W;

    int64_t cur_target = target[b][h][w];
    if (cur_target == ignore_index) {
      output[b][h][w] = static_cast<scalar_t>(0);
      continue;
    }
    CUDA_KERNEL_ASSERT(cur_target >= 0 && cur_target < n_classes);
    scalar_t value = input[b][cur_target][h][w];
    scalar_t cur_weight = weight != nullptr ? weight[cur_target] : static_cast<scalar_t>(1);
    output[b][h][w] = -value * cur_weight;
  }
}

template <typename scalar_t, typename accscalar_t, typename index_t>
C10_LAUNCH_BOUNDS_1(CUDA_NUM_THREADS)
__global__ void nll_loss2d_forward_kernel(
  scalar_t* output,
  scalar_t* total_weight,
  const scalar_t* input,
  const int64_t* target,
  const scalar_t* weight,
  int n_classes,
  int map_nelem,
  int blocks_per_sample,
  int64_t ignore_index) {

  scalar_t cur_weight;
  accscalar_t input_sum = 0;
  accscalar_t acc_weight = 0;

  index_t sample = blockIdx.x / blocks_per_sample;
  index_t toffset = sample * map_nelem;
  index_t ioffset = sample * map_nelem * n_classes;
  int step = blockDim.x * blocks_per_sample;
  for (int i = (blockIdx.x % blocks_per_sample) * blockDim.x + threadIdx.x;
       i < map_nelem;
       i += step) {
    index_t t = target[toffset + i];
    if (t != ignore_index) {
      CUDA_KERNEL_ASSERT(t >= 0 && t < n_classes);
      cur_weight = weight != nullptr ? weight[t] : static_cast<scalar_t>(1);
      const auto input_index = ioffset + i + map_nelem * t;
      CUDA_KERNEL_ASSERT(input_index >= 0);
      input_sum -= input[input_index] * cur_weight;
      acc_weight += cur_weight;
    }
  }

  __shared__ accscalar_t acc_weight_smem[CUDA_NUM_THREADS];
  __shared__ accscalar_t input_sum_smem[CUDA_NUM_THREADS];

  auto acc_weight_ = cuda_utils::BlockReduceSum(acc_weight, acc_weight_smem);
  auto input_sum_ = cuda_utils::BlockReduceSum(input_sum, input_sum_smem);

  if (threadIdx.x == 0) {
    gpuAtomicAdd(total_weight, static_cast<scalar_t>(acc_weight_));
    gpuAtomicAdd(output, static_cast<scalar_t>(input_sum_));
  }
}

template <typename scalar_t>
C10_LAUNCH_BOUNDS_1(CUDA_NUM_THREADS)
__global__ void nll_loss2d_forward_size_average_kernel(
  scalar_t* output,
  const scalar_t* total_weight
) {
  *output /= *total_weight;
}

template <typename scalar_t>
C10_LAUNCH_BOUNDS_1(CUDA_NUM_THREADS)
__global__ void nll_loss2d_backward_no_reduce_kernel(
  int64_t n_threads,
  PackedTensorAccessor64<int64_t, 3> target,
  PackedTensorAccessor64<scalar_t, 3> grad_output,
  PackedTensorAccessor64<scalar_t, 4> grad_input,
  const scalar_t* weight,
  int64_t ignore_index
) {
  int64_t batch_size = target.size(0);
  int64_t H = target.size(1);
  int64_t W = target.size(2);
  int64_t n_classes = grad_input.size(1);

  CUDA_KERNEL_LOOP(index, n_threads) {
    const int64_t b = index % batch_size;
    const int64_t h = (index / batch_size) % H;
    const int64_t w = (index / (batch_size * H)) % W;

    int64_t cur_target = target[b][h][w];
    if (cur_target == ignore_index) {
      continue;
    }
    CUDA_KERNEL_ASSERT(cur_target >= 0 && cur_target < n_classes);
    scalar_t value = -(weight != nullptr ? weight[cur_target] : static_cast<scalar_t>(1));
    grad_input[b][cur_target][h][w] = value * grad_output[b][h][w];
  }
}

template <typename scalar_t>
C10_LAUNCH_BOUNDS_1(CUDA_NUM_THREADS)
__global__ void nll_loss2d_backward_kernel(
  scalar_t* grad_input,
  const scalar_t* grad_output,
  const int64_t* target,
  const scalar_t* weights,
  const scalar_t* total_weight,
  bool size_average,
  int n_classes,
  int map_nelem,
  int blocks_per_sample,
  int64_t ignore_index
) {
  const auto grad = -(size_average ? *grad_output / *total_weight
                                   : *grad_output);

  const int sample = blockIdx.x / blocks_per_sample;
  const int step = blockDim.x * blocks_per_sample;

  const int toffset = sample * map_nelem;
  const auto* const target_thread = target + toffset;

  const int ioffset = sample * map_nelem * n_classes;
  auto* const grad_input_thread = grad_input + ioffset;

  for (int i = (blockIdx.x % blocks_per_sample) * blockDim.x + threadIdx.x;
       i < map_nelem;
       i += step) {
    const int64_t t = target_thread[i];
    if (t != ignore_index) {
      CUDA_KERNEL_ASSERT(t >= 0 && t < n_classes);
      const auto grad_input_index = i + map_nelem * t;
      CUDA_KERNEL_ASSERT(grad_input_index >= 0);
      grad_input_thread[i + map_nelem * t] = weights != nullptr ? weights[t] * grad
                                                                : grad;
    }
  }
}

void check_inputs_nll_loss2d(
    const Tensor& input,
    const Tensor& target,
    const Tensor& weight) {
  TORCH_CHECK(
      target.dim() == 3,
      "only batches of spatial targets supported (3D tensors)"
      " but got targets of size: : ",
      target.sizes());
  TORCH_CHECK(
      input.dim() == 4,
      "only batches of spatial inputs supported (4D tensors), "
      "but got input of size: ",
      input.sizes());
  TORCH_CHECK(
      !weight.defined() || weight.numel() == input.size(1),
      "weight tensor should be defined either for all or no classes");

  TORCH_CHECK(
      input.size(0) == target.size(0) && input.size(2) == target.size(1) &&
          input.size(3) == target.size(2),
      "input and target batch or spatial sizes don't match: target ",
      target.sizes(),
      ", input ",
      input.sizes());
}

void nll_loss2d_forward_out_cuda_template(
    Tensor& output,
    Tensor& total_weight,
    const Tensor& input,
    const Tensor& target,
    const std::optional<Tensor>& weight_opt,
    int64_t reduction,
    int64_t ignore_index) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage in 'sum' or 'mean' reductions.
  if (reduction != at::Reduction::None) {
    at::globalContext().alertNotDeterministic("nll_loss2d_forward_out_cuda_template");
  }

  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> weight_maybe_owned =
      at::borrow_from_optional_tensor(weight_opt);
  const Tensor& weight = *weight_maybe_owned;

  check_inputs_nll_loss2d(input, target, weight);
  total_weight.resize_({});

  if (reduction == at::Reduction::None) {
    int64_t batch_size = input.size(0);
    int64_t H = input.size(2);
    int64_t W = input.size(3);
    int64_t count = batch_size * H * W;

    at::native::resize_output(output, {batch_size, H, W});
    if (count == 0) {
      // This guards from unnecessary operations and launching CUDA kernel with
      // 0 blocks.
      return;
    }
    auto weight_ = optional_contiguous(weight);
    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half,
        at::ScalarType::BFloat16,
        input.scalar_type(),
        "nll_loss2d_forward_no_reduce_kernel",
        [&] {
          nll_loss2d_forward_no_reduce_kernel<scalar_t>
              <<<GET_BLOCKS(count),
                 CUDA_NUM_THREADS,
                 0,
                 at::cuda::getCurrentCUDAStream()>>>(
                  count,
                  input.packed_accessor64<scalar_t, 4>(),
                  target.packed_accessor64<int64_t, 3>(),
                  output.packed_accessor64<scalar_t, 3>(),
                  optional_data<scalar_t>(weight_),
                  ignore_index);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
    return;
  }

  // produce scalar outputs for the reduction case
  at::native::resize_output(output, {});

  if (target.numel() == 0) {
    // Here target (and input) have zero elements
    // Mean reduction on empty tensors produces NaN. See the discussion in
    // https://github.com/pytorch/pytorch/pull/64572#issuecomment-926504162
    if (reduction == Reduction::Mean) {
      output.fill_(std::numeric_limits<double>::quiet_NaN());
    } else {
      output.zero_();
    }
    total_weight.zero_();
    return;
  }

  auto input_ = input.contiguous();
  auto weight_ = optional_contiguous(weight);
  auto target_ = target.contiguous();

  output.zero_();
  total_weight.zero_();

  auto batch_size = target.size(0);
  int64_t map_nelem = target.numel() / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      input.scalar_type(),
      "nll_loss2d_forward_kernel",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
    AT_DISPATCH_INDEX_TYPES(
        at::native::canUse32BitIndexMath(input_, INT_MAX) ? ScalarType::Int : ScalarType::Long,
        "nll_loss2d_forward_launcher", [&] {
            nll_loss2d_forward_kernel<scalar_t, accscalar_t, index_t>
                <<<total_blocks,
                  CUDA_NUM_THREADS,
                  0,
                  at::cuda::getCurrentCUDAStream()>>>(
                    output.mutable_data_ptr<scalar_t>(),
                    total_weight.mutable_data_ptr<scalar_t>(),
                    input_.const_data_ptr<scalar_t>(),
                    target_.const_data_ptr<int64_t>(),
                    optional_data<scalar_t>(weight_),
                    input_.size(1),
                    input_.size(2) * input_.size(3),
                    blocks_per_sample,
                    ignore_index);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            // Divide by total_weight
            if (reduction == at::Reduction::Mean) {
              nll_loss2d_forward_size_average_kernel<scalar_t>
                  <<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                      output.mutable_data_ptr<scalar_t>(),
                      total_weight.const_data_ptr<scalar_t>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }
    });
      });
}

void nll_loss2d_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& target,
    const std::optional<Tensor>& weight_opt,
    int64_t reduction,
    int64_t ignore_index,
    const Tensor& total_weight) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> weight_maybe_owned =
      at::borrow_from_optional_tensor(weight_opt);
  const Tensor& weight = *weight_maybe_owned;

  check_inputs_nll_loss2d(input, target, weight);
  grad_input.resize_as_(input);
  grad_input.zero_();
  TORCH_CHECK(grad_input.is_contiguous(), "grad_input must be contiguous");
  TORCH_CHECK(
      total_weight.numel() == 1,
      "expected total_weight to be a single element tensor, got: ",
      total_weight.sizes(),
      " (",
      total_weight.numel(),
      " elements)");


  if (reduction == at::Reduction::None) {
    TORCH_CHECK(
        grad_output.dim() == 3,
        "grad_output must have same dimension as target (3) but got dimension: ",
        grad_output.sizes());
    TORCH_CHECK(
        grad_output.size(0) == target.size(0) &&
            grad_output.size(1) == target.size(1) &&
            grad_output.size(2) == target.size(2),
        "grad_output sizes don't match target sizes: target ",
        target.sizes(),
        ", grad_output ",
        grad_output.sizes())
    int64_t batch_size = input.size(0);
    int64_t H = input.size(2);
    int64_t W = input.size(3);
    int64_t count = batch_size * H * W;

    if (count == 0) {
      // This guards from unnecessary operations and launching CUDA kernel with
      // 0 blocks.
      return;
    }
    auto weight_ = optional_contiguous(weight);
    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half,
        at::ScalarType::BFloat16,
        input.scalar_type(),
        "nll_loss2d_backward_no_reduce_kernel",
        [&] {
          nll_loss2d_backward_no_reduce_kernel<scalar_t>
              <<<GET_BLOCKS(count),
                 CUDA_NUM_THREADS,
                 0,
                 at::cuda::getCurrentCUDAStream()>>>(
                  count,
                  target.packed_accessor64<int64_t, 3>(),
                  grad_output.packed_accessor64<scalar_t, 3>(),
                  grad_input.packed_accessor64<scalar_t, 4>(),
                  optional_data<scalar_t>(weight_),
                  ignore_index);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
    return;
  }

  int64_t batch_size = target.size(0);
  auto target_numel = target.numel();
  if (batch_size != 0 && target_numel != 0) {
    // This guards from unnecessary operations and launching CUDA kernel with 1
    // blocks.
    auto target_ = target.contiguous();
    auto weight_ = optional_contiguous(weight);

    int64_t map_nelem = target_numel / batch_size;
    int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
    blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
    int total_blocks = blocks_per_sample * batch_size;

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half,
        at::ScalarType::BFloat16,
        input.scalar_type(),
        "nll_loss2d_backward_kernel",
        [&] {
          nll_loss2d_backward_kernel<scalar_t>
              <<<total_blocks,
                CUDA_NUM_THREADS,
                0,
                at::cuda::getCurrentCUDAStream()>>>(
                  grad_input.mutable_data_ptr<scalar_t>(),
                  grad_output.const_data_ptr<scalar_t>(),
                  target_.const_data_ptr<int64_t>(),
                  optional_data<scalar_t>(weight_),
                  total_weight.const_data_ptr<scalar_t>(),
                  reduction == at::Reduction::Mean,
                  input.size(1),
                  map_nelem,
                  blocks_per_sample,
                  ignore_index);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
  }
}
} // namespace

std::tuple<Tensor&, Tensor&> nll_loss2d_forward_out_cuda(
    const Tensor& self,
    const Tensor& target,
    const std::optional<Tensor>& weight_opt,
    int64_t reduction,
    int64_t ignore_index,
    Tensor& output,
    Tensor& total_weight) {
  nll_loss2d_forward_out_cuda_template(
      output, total_weight, self, target, weight_opt, reduction, ignore_index);
  return std::tuple<Tensor&, Tensor&>(output, total_weight);
}

std::tuple<Tensor, Tensor> nll_loss2d_forward_cuda(
    const Tensor& self,
    const Tensor& target,
    const std::optional<Tensor>& weight_opt,
    int64_t reduction,
    int64_t ignore_index) {
  auto output = at::empty({0}, self.options());
  auto total_weight = at::empty({0}, self.options());
  nll_loss2d_forward_out_cuda_template(
      output, total_weight, self, target, weight_opt, reduction, ignore_index);
  return std::make_tuple(output, total_weight);
}

Tensor& nll_loss2d_backward_out_cuda(
    const Tensor& grad_output,
    const Tensor& self,
    const Tensor& target,
    const std::optional<Tensor>& weight_opt,
    int64_t reduction,
    int64_t ignore_index,
    const Tensor& total_weight,
    Tensor& grad_input) {
  nll_loss2d_backward_out_cuda_template(
      grad_input,
      grad_output,
      self,
      target,
      weight_opt,
      reduction,
      ignore_index,
      total_weight);
  return grad_input;
}

Tensor nll_loss2d_backward_cuda(
    const Tensor& grad_output,
    const Tensor& self,
    const Tensor& target,
    const std::optional<Tensor>& weight_opt,
    int64_t reduction,
    int64_t ignore_index,
    const Tensor& total_weight) {
  auto grad_input = at::empty_like(self);
  nll_loss2d_backward_out_cuda_template(
      grad_input,
      grad_output,
      self,
      target,
      weight_opt,
      reduction,
      ignore_index,
      total_weight);
  return grad_input;
}

} // namespace at::native
