#include "hip/hip_runtime.h"
#include <c10/core/ScalarType.h>
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/NumericUtils.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/Resize.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/bincount_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/histc_native.h>
#include <ATen/ops/zeros.h>
#endif

namespace at {
namespace cuda {
#define RATIO_OF_GMEM_ATOMIC_ADD_TO_SMEM_ATOMIC_ADD 8
#define FOR_KERNEL_LOOP(i, lim)                                      \
  for (IndexType i = blockIdx.x * blockDim.x + threadIdx.x; i < lim; \
       i += gridDim.x * blockDim.x)

/*
  Memory types used for the 3 histogram implementations.
  See `CUDA_tensor_histogram` below.
 */
enum class CUDAHistogramMemoryType { SHARED, GLOBAL };
namespace {
template <typename input_t, typename IndexType>
__device__ static IndexType getBin(
    input_t bVal,
    at::acc_type<input_t, /*is_cuda=*/true> minvalue,
    at::acc_type<input_t, /*is_cuda=*/true> maxvalue,
    int64_t nbins) {
  IndexType bin = (int)(((bVal - minvalue)) * nbins / (maxvalue - minvalue));
  // (only applicable for histc)
  // while each bin is inclusive at the lower end and exclusive at the higher,
  // i.e. [start, end) the last bin is inclusive at both, i.e. [start, end], in
  // order to include maxvalue if exists therefore when bin == nbins, adjust bin
  // to the last bin
  if (bin == nbins)
    bin -= 1;
  return bin;
}
}

/*
  Kernel for computing the histogram of the input.
 */
template <
    typename output_t,
    typename input_t,
    typename IndexType,
    int ADims,
    int PDims,
    int BDims,
    CUDAHistogramMemoryType MemoryType,
    typename Op>
C10_LAUNCH_BOUNDS_1(cuda::getApplyBlockSize())
__global__ void kernelHistogram1D(
    detail::TensorInfo<output_t, IndexType> a, /* output */
    detail::TensorInfo<output_t, IndexType> p, /* partial output */
    detail::TensorInfo<const input_t, IndexType> b, /* input */
    int64_t nbins,
    at::acc_type<input_t, /*is_cuda=*/true> minvalue,
    at::acc_type<input_t, /*is_cuda=*/true> maxvalue,
    IndexType totalElements,
    Op getOp) {
  extern __shared__ unsigned char my_smem[];
  output_t* smem = nullptr;

  if (MemoryType == CUDAHistogramMemoryType::SHARED) {
    ////////////////////////// Shared memory //////////////////////////
    // atomically add to block specific shared memory
    // then atomically add to the global output tensor
    smem = reinterpret_cast<output_t*>(my_smem);
    for (IndexType i = threadIdx.x; i < a.sizes[0]; i += blockDim.x) {
      smem[i] = 0;
    }
    __syncthreads();
    FOR_KERNEL_LOOP(linearIndex, totalElements) {
      // Convert `linearIndex` into an offset of `b`
      const IndexType bOffset =
          detail::IndexToOffset<const input_t, IndexType, BDims>::get(linearIndex, b);
      const auto bVal = b.data[bOffset];
      if (bVal >= minvalue && bVal <= maxvalue) {
        // Use value at `b` as an offset of `smem`
        const IndexType bin =
            getBin<input_t, IndexType>(bVal, minvalue, maxvalue, nbins);
        gpuAtomicAddNoReturn(&smem[bin], getOp(linearIndex));
      }
    }
    __syncthreads();
    // NOTE: atomically update output bin count.
    //   Atomic update is imp since __syncthread() will only synchronize threads
    //   in a given block, not across blocks.
    for (IndexType i = threadIdx.x; i < a.sizes[0]; i += blockDim.x) {
      const IndexType aOffset =
          detail::IndexToOffset<output_t, IndexType, ADims>::get(i, a);
      gpuAtomicAddNoReturn(&a.data[aOffset], smem[i]);
    }

  } else {
    ////////////////////////// Global memory //////////////////////////
    // atomically add to the output tensor
    // compute histogram for the block
    FOR_KERNEL_LOOP(linearIndex, totalElements) {
      // Convert `linearIndex` into an offset of `b`
      const IndexType bOffset =
          detail::IndexToOffset<const input_t, IndexType, BDims>::get(linearIndex, b);
      const auto bVal = b.data[bOffset];
      if (bVal >= minvalue && bVal <= maxvalue) {
        // Use value at `b` as an offset of `a`
        const IndexType bin =
            getBin<input_t, IndexType>(bVal, minvalue, maxvalue, nbins);
        const IndexType aOffset =
            detail::IndexToOffset<output_t, IndexType, ADims>::get(bin, a);
        gpuAtomicAddNoReturn(&a.data[aOffset], getOp(linearIndex));
      }
    }
  }
}

#define HANDLE_CASE(MEMORY_TYPE, WEIGHTS_OP, SHARED_MEM)                 \
  kernelHistogram1D<                                                     \
      output_t,                                                          \
      input_t,                                                           \
      IndexType,                                                         \
      1,                                                                 \
      2,                                                                 \
      -1,                                                                \
      MEMORY_TYPE><<<grid, block, SHARED_MEM, getCurrentCUDAStream()>>>( \
      aInfo,                                                             \
      pInfo,                                                             \
      bInfo,                                                             \
      nbins,                                                             \
      minvalue,                                                          \
      maxvalue,                                                          \
      totalElements,                                                     \
      WEIGHTS_OP);                                                       \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define HANDLE_SWITCH_CASE(mType, getOp)                                   \
  switch (mType) {                                                         \
    case CUDAHistogramMemoryType::SHARED:                                  \
      HANDLE_CASE(CUDAHistogramMemoryType::SHARED, getOp, sharedMem);      \
      break;                                                               \
    default:                                                               \
      HANDLE_CASE(CUDAHistogramMemoryType::GLOBAL, getOp, 0);              \
  }

/*
  Calculate the frequency of the input values.

  `a` contains the final output or the histogram.
  Input `b` is assumed to be 1-D non-negative int array.
  `c` optionally contains the weight vector.
  See `help torch.bincount` for details on the math.

  3 implementations based of input size and memory usage:
    case: enough shared mem
        SHARED: Each block atomically adds to it's own **shared** hist copy,
        then atomically updates the global tensor.
    case: no enough shared mem
        GLOBAL: all threads atomically update to a single **global** hist copy.
 */
template <typename output_t, typename input_t, bool HasWeights>
bool CUDA_tensor_histogram(
    at::Tensor a, /* output */
    at::Tensor b, /* input */
    at::Tensor c, /* weights(optional) */
    int64_t nbins,
    at::acc_type<input_t, /*is_cuda=*/true> minvalue,
    at::acc_type<input_t, /*is_cuda=*/true> maxvalue,
    TensorArgType aType = TensorArgType::ReadWrite,
    TensorArgType bType = TensorArgType::ReadOnly,
    TensorArgType cType = TensorArgType::ReadOnly) {
  checkBackend("CUDA_tensor_histogram", {a, b}, Backend::CUDA);
  if (HasWeights) {
    checkBackend("CUDA_tensor_histogram", {c}, Backend::CUDA);
  }
  auto totalElements = b.numel();

  if (totalElements == 0) {
    return false;
  }

  const dim3 block = getApplyBlock();
  dim3 grid;
  auto curDevice = current_device();
  if (curDevice == -1 || !getApplyGrid(totalElements, grid, curDevice)) {
    return false;
  }

  CUDAHistogramMemoryType memType = CUDAHistogramMemoryType::GLOBAL;
  auto maxSharedMem = getCurrentDeviceProperties()->sharedMemPerBlock;
  auto sharedMem = nbins * sizeof(output_t) + 8; // 8 guard bytes
  // determine memory type to use in the kernel
  if (sharedMem < maxSharedMem) {
    // Solve equations:
    // (1) #(smem atomicAdd per SM) = totalElements / min(grid.x, #SM)
    // (2) #(gmem atomicAdd) = grid.x * nbins
    // (3) RATIO_OF_GMEM_ATOMIC_ADD_TO_SMEM_ATOMIC_ADD = #(gmem atomicAdd) / #(smem atomicAdd per SM)
    unsigned optimalGrid = ceil_div<size_t>(RATIO_OF_GMEM_ATOMIC_ADD_TO_SMEM_ATOMIC_ADD * totalElements,
                                            nbins * getCurrentDeviceProperties()->multiProcessorCount);
    if (optimalGrid < (unsigned)getCurrentDeviceProperties()->multiProcessorCount) {
      optimalGrid = 1 + (unsigned)std::sqrt(RATIO_OF_GMEM_ATOMIC_ADD_TO_SMEM_ATOMIC_ADD * totalElements / nbins);
    }
    auto optimalSteps = ceil_div<size_t>(totalElements, optimalGrid * block.x);
    optimalGrid = ceil_div<size_t>(totalElements, optimalSteps * block.x);
    grid.x = std::min(grid.x, optimalGrid);
    memType = CUDAHistogramMemoryType::SHARED;
  }

  using IndexType = int64_t;
  auto aInfo = detail::getTensorInfo<output_t, IndexType>(a);
  auto bInfo = detail::getTensorInfo<const input_t, IndexType>(b);
  detail::TensorInfo<output_t, IndexType> pInfo(nullptr, 0, {}, {});

  if (HasWeights) {
    auto cInfo = detail::getTensorInfo<output_t, IndexType>(c);
    const auto getWeightsOp = [cInfo] __device__(IndexType cIndex) {
      const IndexType cOffset =
          detail::IndexToOffset<output_t, IndexType, 1>::get(cIndex, cInfo);
      return cInfo.data[cOffset];
    };
    HANDLE_SWITCH_CASE(memType, getWeightsOp)
  } else {
    static const auto getDummyOp = [] __device__(IndexType) { return 1L; };
    HANDLE_SWITCH_CASE(memType, getDummyOp)
  }
  return true;
}

#undef HANDLE_CASE
#undef HANDLE_SWITCH_CASE
#undef FOR_KERNEL_LOOP
#undef RATIO_OF_GMEM_ATOMIC_ADD_TO_SMEM_ATOMIC_ADD
} // namespace cuda

namespace {
///////////////// bincount /////////////////
template <typename input_t, typename weights_t>
Tensor _bincount_cuda_template(
    const Tensor& self,
    const Tensor& weights,
    int64_t minlength) {
  if (minlength < 0) {
    TORCH_CHECK(false, "minlength should be >= 0");
  }
  if (self.dim() == 1 && self.numel() == 0) {
    return at::zeros(
        {minlength},
        kLong,
        std::nullopt /* layout */,
        kCUDA,
        std::nullopt /* pin_memory */);
  }
  if (self.dim() != 1 ||
      (!std::is_same_v<input_t, uint8_t> &&
       *self.min().cpu().const_data_ptr<input_t>() < 0)) {
    TORCH_CHECK(false, "bincount only supports 1-d non-negative integral inputs.");
  }

  bool has_weights = weights.defined();
  if (has_weights && (weights.dim() != 1 || weights.size(0) != self.size(0))) {
    TORCH_CHECK(false, "weights should be 1-d and have the same length as input");
  }

  const int64_t nbins =
      std::max(self.max().item<input_t>() + (int64_t)1, minlength);

  // we are using acc_type for the bounds, in particular int64_t for integers
  // in order to avoid overflows (e.g. using 256 bins for dtype uint8)
  using bounds_t = at::acc_type<input_t, /*is_cuda=*/true>;
  const bounds_t minvalue = 0;
  const bounds_t maxvalue = nbins;
  // alloc output counter on GPU
  Tensor output;
  if (has_weights) {
    output = at::zeros(
        {nbins},
        optTypeMetaToScalarType(weights.options().dtype_opt()),
        weights.options().layout_opt(),
        weights.options().device_opt(),
        weights.options().pinned_memory_opt());
    cuda::CUDA_tensor_histogram<weights_t, input_t, true>(
        output, self, weights, nbins, minvalue, maxvalue);
  } else {
    output = at::zeros(
        {nbins},
        kLong,
        std::nullopt /* layout */,
        DeviceType::CUDA,
        std::nullopt /* pin_memory */);
    cuda::CUDA_tensor_histogram<int64_t, input_t, false>(
        output, self, weights, nbins, minvalue, maxvalue);
  }
  return output;
}

///////////////// histc /////////////////
template <typename input_t>
Tensor _histc_cuda_template(
    const Tensor& self,
    int64_t nbins,
    at::acc_type<input_t, /*is_cuda=*/true> min,
    at::acc_type<input_t, /*is_cuda=*/true> max) {
  if (nbins <= 0) {
    TORCH_CHECK(false, "bins must be > 0");
  }
  Tensor output = at::zeros(
      {nbins},
      self.scalar_type(),
      std::nullopt /* layout */,
      DeviceType::CUDA,
      std::nullopt /* pin_memory */);
  using bounds_t = at::acc_type<input_t, /*is_cuda=*/true>;
  bounds_t minvalue = min;
  bounds_t maxvalue = max;

  if (min == max && self.numel() > 0) {
    minvalue = *self.min().cpu().const_data_ptr<input_t>();
    maxvalue = *self.max().cpu().const_data_ptr<input_t>();
  }
  if (minvalue == maxvalue) {
    minvalue = minvalue - 1;
    maxvalue = maxvalue + 1;
  }

// Microsoft's STL has a problem with integer overloads of std::fpclassify used
// by std::isnan and std::isinf, as described here:
// https://stackoverflow.com/questions/61646166/how-to-resolve-fpclassify-ambiguous-call-to-overloaded-function
// This macro provides a workaround for this problem.
#if defined(USE_ROCM) && defined(_MSC_VER)
#define STL_CAST_BUG(value) static_cast<double>(value)
#else
#define STL_CAST_BUG(value) value
#endif

#if !defined(USE_ROCM)
  TORCH_CHECK(
      !(at::_isinf(minvalue) || at::_isinf(maxvalue) ||
        at::_isnan(minvalue) || at::_isnan(maxvalue)),
      "range of [",
      minvalue,
      ", ",
      maxvalue,
      "] is not finite");
#else
  TORCH_CHECK(
      !(std::isinf(STL_CAST_BUG(minvalue)) ||
        std::isinf(STL_CAST_BUG(maxvalue)) ||
        std::isnan(STL_CAST_BUG(minvalue)) ||
        std::isnan(STL_CAST_BUG(maxvalue))),
      "range of [",
      minvalue,
      ", ",
      maxvalue,
      "] is not finite");
#endif
  TORCH_CHECK(minvalue < maxvalue, "max must be larger than min");

  cuda::CUDA_tensor_histogram<input_t, input_t, false>(
      output, self, Tensor(), nbins, minvalue, maxvalue);
  return output;
}
} // namespace

namespace native {
Tensor _bincount_cuda(
    const Tensor& self, const std::optional<Tensor>& weights_opt,
    int64_t minlength) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> weights_maybe_owned = at::borrow_from_optional_tensor(weights_opt);
  const Tensor& weights = *weights_maybe_owned;

  if (weights_opt.has_value()) {
    // See Note [Writing Nondeterministic Operations]
    // Nondeterministic if weights are given, because of floating point
    // atomicAdd usage
    globalContext().alertNotDeterministic("_bincount_cuda");
  }
  return AT_DISPATCH_INTEGRAL_TYPES(self.scalar_type(), "bincount_cuda", [&] {
    const auto scalar = weights.scalar_type();
    if (scalar == ScalarType::Undefined || scalar == ScalarType::Float)
      return _bincount_cuda_template<scalar_t, float>(self, weights, minlength);
    return _bincount_cuda_template<scalar_t, double>(
        self, weights.to(kDouble), minlength);
  });
}

Tensor _histc_cuda(
    const Tensor& self,
    int64_t nbins,
    const Scalar& min,
    const Scalar& max) {
  if (self.scalar_type() == ScalarType::Half) {
    TORCH_CHECK(false, "HalfTensor is not supported");
  }
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic for floating types because of atomicAdd usage
  if (at::isFloatingType(self.scalar_type())){
    globalContext().alertNotDeterministic("_histc_cuda with floating point input");
  }
  return AT_DISPATCH_ALL_TYPES(self.scalar_type(), "histc", [&] {
    using bounds_t = at::acc_type<scalar_t, /*is_cuda=*/true>;
    return _histc_cuda_template<scalar_t>(
        self, nbins, min.to<bounds_t>(), max.to<bounds_t>());
  });
}

Tensor& _histc_out_cuda(const Tensor& self, int64_t bins, const Scalar& min, const Scalar& max, Tensor& result) {
  auto ret = _histc_cuda(self, bins, min, max);
  resize_output(result, ret.sizes());
  result.copy_(ret);
  return result;
}
} // namespace native
} // namespace at
