#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/OpMathType.h>
#include <ATen/native/cuda/GridSampler.h>
#include <ATen/native/GridSamplerUtils.h>
#include <ATen/native/cuda/GridSampler.cuh>
#include <ATen/native/cuda/UpSample.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/core/TensorBase.h>
#include <ATen/Dispatch.h>
#include <c10/macros/Macros.h>
#include <cmath>

namespace at::native {

using namespace at::cuda::detail;

using at::native::detail::GridSamplerInterpolation;
using at::native::detail::GridSamplerPadding;

namespace {
  template <typename scalar_t, typename index_t>
  C10_LAUNCH_BOUNDS_1(256)
  __global__ void grid_sampler_2d_kernel(
      const index_t nthreads,
      TensorInfo<const scalar_t, index_t> input,
      TensorInfo<const scalar_t, index_t> grid,
      TensorInfo<scalar_t, index_t> output,
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {

    using opmath_t = at::opmath_type<scalar_t>;
    index_t C = input.sizes[1];
    index_t inp_H = input.sizes[2];
    index_t inp_W = input.sizes[3];
    index_t out_H = grid.sizes[1];
    index_t out_W = grid.sizes[2];
    index_t inp_sN = input.strides[0];
    index_t inp_sC = input.strides[1];
    index_t inp_sH = input.strides[2];
    index_t inp_sW = input.strides[3];
    index_t grid_sN = grid.strides[0];
    index_t grid_sH = grid.strides[1];
    index_t grid_sW = grid.strides[2];
    index_t grid_sCoor = grid.strides[3];
    index_t out_sN = output.strides[0];
    index_t out_sC = output.strides[1];
    index_t out_sH = output.strides[2];
    index_t out_sW = output.strides[3];

    CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
      const index_t w = index % out_W;
      const index_t h = (index / out_W) % out_H;
      const index_t n = index / (out_H * out_W);
      const index_t grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y coordinates from grid
      opmath_t x = grid.data[grid_offset];
      opmath_t y = grid.data[grid_offset + grid_sCoor];

      opmath_t ix = grid_sampler_compute_source_index(x, inp_W, padding_mode, align_corners);
      opmath_t iy = grid_sampler_compute_source_index(y, inp_H, padding_mode, align_corners);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get NE, NW, SE, SW pixel values from (x, y)
        index_t ix_nw = static_cast<index_t>(::floor(ix));
        index_t iy_nw = static_cast<index_t>(::floor(iy));
        index_t ix_ne = ix_nw + 1;
        index_t iy_ne = iy_nw;
        index_t ix_sw = ix_nw;
        index_t iy_sw = iy_nw + 1;
        index_t ix_se = ix_nw + 1;
        index_t iy_se = iy_nw + 1;

        // get surfaces to each neighbor:
        opmath_t nw = (ix_se - ix)    * (iy_se - iy);
        opmath_t ne = (ix    - ix_sw) * (iy_sw - iy);
        opmath_t sw = (ix_ne - ix)    * (iy    - iy_ne);
        opmath_t se = (ix    - ix_nw) * (iy    - iy_nw);

        // calculate bilinear weighted pixel value and set output pixel
        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCHW = output.data + n * out_sN + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
          opmath_t out_acc = 0;
          if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW] * nw;
          }
          if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW] * ne;
          }
          if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW] * sw;
          }
          if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW] * se;
          }
          *out_ptr_NCHW = out_acc;
        }
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        index_t ix_nearest = static_cast<index_t>(std::nearbyint(ix));
        index_t iy_nearest = static_cast<index_t>(std::nearbyint(iy));

        // assign nearest neighbour pixel value to output pixel
        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCHW = output.data + n * out_sN + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
          if (within_bounds_2d(iy_nearest, ix_nearest, inp_H, inp_W)) {
            *out_ptr_NCHW = inp_ptr_NC[iy_nearest * inp_sH + ix_nearest * inp_sW];
          } else {
            *out_ptr_NCHW = static_cast<scalar_t>(0);
          }
        }
      } else if (interpolation_mode == GridSamplerInterpolation::Bicubic) {

        ix = grid_sampler_unnormalize(x, inp_W, align_corners);
        iy = grid_sampler_unnormalize(y, inp_H, align_corners);

        opmath_t ix_nw = std::floor(ix);
        opmath_t iy_nw = std::floor(iy);

        const opmath_t tx = ix - ix_nw;
        const opmath_t ty = iy - iy_nw;

        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCHW = output.data + n * out_sN + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
          opmath_t coefficients[4];

          #pragma unroll 4
          for (index_t i = 0; i < 4; ++i) {
            coefficients[i] = cubic_interp1d(
              get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw - 1, iy_nw - 1 + i, inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners),
              get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw + 0, iy_nw - 1 + i, inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners),
              get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw + 1, iy_nw - 1 + i, inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners),
              get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw + 2, iy_nw - 1 + i, inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners),
              tx);
          }

          *out_ptr_NCHW = cubic_interp1d(
            coefficients[0],
            coefficients[1],
            coefficients[2],
            coefficients[3],
            ty);
        }
      }
    }
  }

  template <typename scalar_t, typename index_t>
  C10_LAUNCH_BOUNDS_1(512)
  __global__ void grid_sampler_3d_kernel(
      const index_t nthreads,
      TensorInfo<const scalar_t, index_t> input,
      TensorInfo<const scalar_t, index_t> grid,
      TensorInfo<scalar_t, index_t> output,
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {

    using opmath_t = at::opmath_type<scalar_t>;
    index_t C = input.sizes[1];
    index_t inp_D = input.sizes[2];
    index_t inp_H = input.sizes[3];
    index_t inp_W = input.sizes[4];
    index_t out_D = grid.sizes[1];
    index_t out_H = grid.sizes[2];
    index_t out_W = grid.sizes[3];
    index_t inp_sN = input.strides[0];
    index_t inp_sC = input.strides[1];
    index_t inp_sD = input.strides[2];
    index_t inp_sH = input.strides[3];
    index_t inp_sW = input.strides[4];
    index_t grid_sN = grid.strides[0];
    index_t grid_sD = grid.strides[1];
    index_t grid_sH = grid.strides[2];
    index_t grid_sW = grid.strides[3];
    index_t grid_sCoor = grid.strides[4];
    index_t out_sN = output.strides[0];
    index_t out_sC = output.strides[1];
    index_t out_sD = output.strides[2];
    index_t out_sH = output.strides[3];
    index_t out_sW = output.strides[4];

    CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
      const index_t w = index % out_W;
      const index_t h = (index / out_W) % out_H;
      const index_t d = (index / (out_H * out_W)) % out_D;
      const index_t n = index / (out_D * out_H * out_W);
      const index_t grid_offset = n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y, z coordinates from grid
      opmath_t x = grid.data[grid_offset];
      opmath_t y = grid.data[grid_offset + grid_sCoor];
      opmath_t z = grid.data[grid_offset + 2 * grid_sCoor];

      opmath_t ix = grid_sampler_compute_source_index(x, inp_W, padding_mode, align_corners);
      opmath_t iy = grid_sampler_compute_source_index(y, inp_H, padding_mode, align_corners);
      opmath_t iz = grid_sampler_compute_source_index(z, inp_D, padding_mode, align_corners);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get corner pixel values from (x, y, z)
        // for 4d, we used north-east-south-west
        // for 5d, we add top-bottom
        index_t ix_tnw = static_cast<index_t>(::floor(ix));
        index_t iy_tnw = static_cast<index_t>(::floor(iy));
        index_t iz_tnw = static_cast<index_t>(::floor(iz));

        index_t ix_tne = ix_tnw + 1;
        index_t iy_tne = iy_tnw;
        index_t iz_tne = iz_tnw;

        index_t ix_tsw = ix_tnw;
        index_t iy_tsw = iy_tnw + 1;
        index_t iz_tsw = iz_tnw;

        index_t ix_tse = ix_tnw + 1;
        index_t iy_tse = iy_tnw + 1;
        index_t iz_tse = iz_tnw;

        index_t ix_bnw = ix_tnw;
        index_t iy_bnw = iy_tnw;
        index_t iz_bnw = iz_tnw + 1;

        index_t ix_bne = ix_tnw + 1;
        index_t iy_bne = iy_tnw;
        index_t iz_bne = iz_tnw + 1;

        index_t ix_bsw = ix_tnw;
        index_t iy_bsw = iy_tnw + 1;
        index_t iz_bsw = iz_tnw + 1;

        index_t ix_bse = ix_tnw + 1;
        index_t iy_bse = iy_tnw + 1;
        index_t iz_bse = iz_tnw + 1;

        // get surfaces to each neighbor:
        opmath_t tnw = (ix_bse - ix)    * (iy_bse - iy)    * (iz_bse - iz);
        opmath_t tne = (ix    - ix_bsw) * (iy_bsw - iy)    * (iz_bsw - iz);
        opmath_t tsw = (ix_bne - ix)    * (iy    - iy_bne) * (iz_bne - iz);
        opmath_t tse = (ix    - ix_bnw) * (iy    - iy_bnw) * (iz_bnw - iz);
        opmath_t bnw = (ix_tse - ix)    * (iy_tse - iy)    * (iz - iz_tse);
        opmath_t bne = (ix    - ix_tsw) * (iy_tsw - iy)    * (iz - iz_tsw);
        opmath_t bsw = (ix_tne - ix)    * (iy    - iy_tne) * (iz - iz_tne);
        opmath_t bse = (ix    - ix_tnw) * (iy    - iy_tnw) * (iz - iz_tnw);

        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCDHW = output.data + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
          //   (c, iz_tnw, iy_tnw, ix_tnw) * tnw + (c, iz_tne, iy_tne, ix_tne) * tne
          // + (c, iz_tsw, iy_tsw, ix_tsw) * tsw + (c, iz_tse, iy_tse, ix_tse) * tse
          // + (c, iz_bnw, iy_bnw, ix_bnw) * bnw + (c, iz_bne, iy_bne, ix_bne) * bne
          // + (c, iz_bsw, iy_bsw, ix_bsw) * bsw + (c, iz_bse, iy_bse, ix_bse) * bse
          opmath_t out_acc = 0;
          if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, inp_D, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW] * tnw;
          }
          if (within_bounds_3d(iz_tne, iy_tne, ix_tne, inp_D, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW] * tne;
          }
          if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, inp_D, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW] * tsw;
          }
          if (within_bounds_3d(iz_tse, iy_tse, ix_tse, inp_D, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW] * tse;
          }
          if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, inp_D, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW] * bnw;
          }
          if (within_bounds_3d(iz_bne, iy_bne, ix_bne, inp_D, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW] * bne;
          }
          if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, inp_D, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW] * bsw;
          }
          if (within_bounds_3d(iz_bse, iy_bse, ix_bse, inp_D, inp_H, inp_W)) {
            out_acc += inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW] * bse;
          }
          *out_ptr_NCDHW = out_acc;
        }
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        index_t ix_nearest = static_cast<index_t>(std::nearbyint(ix));
        index_t iy_nearest = static_cast<index_t>(std::nearbyint(iy));
        index_t iz_nearest = static_cast<index_t>(std::nearbyint(iz));

        // assign nearest neighbour pixel value to output pixel
        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCDHW = output.data + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
          if (within_bounds_3d(iz_nearest, iy_nearest, ix_nearest, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW = inp_ptr_NC[iz_nearest * inp_sD + iy_nearest * inp_sH + ix_nearest * inp_sW];
          } else {
            *out_ptr_NCDHW = static_cast<scalar_t>(0);
          }
        }
      }
    }
  }

// Note [Passing pointer and offset to fastAtomicAdd]
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// For its internal bounds checking, fastAtomicAdd needs to know where the destination address
// lies relative to the entire tensor, so we pass the base grad_input.data and full offset information,
// including batch * channel offset (NC_offset).

  template <typename scalar_t, typename index_t>
  C10_LAUNCH_BOUNDS_1(256)
  __global__ void grid_sampler_2d_backward_kernel(
      const index_t nthreads,
      TensorInfo<const scalar_t, index_t> grad_output,
      TensorInfo<const scalar_t, index_t> input,
      TensorInfo<const scalar_t, index_t> grid,
      TensorInfo<scalar_t, index_t> grad_input,  // initialized to zeros (or unused if input_requires_grad is false)
      TensorInfo<scalar_t, index_t> grad_grid,   // initialized to empty
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners,
      const index_t grad_input_memory_span,
      const bool input_requires_grad) {

    index_t C = input.sizes[1];
    index_t inp_H = input.sizes[2];
    index_t inp_W = input.sizes[3];
    index_t out_H = grid.sizes[1];
    index_t out_W = grid.sizes[2];
    index_t inp_sN = input.strides[0];
    index_t inp_sC = input.strides[1];
    index_t inp_sH = input.strides[2];
    index_t inp_sW = input.strides[3];
    index_t grid_sN = grid.strides[0];
    index_t grid_sH = grid.strides[1];
    index_t grid_sW = grid.strides[2];
    index_t grid_sCoor = grid.strides[3];
    index_t gOut_sN = grad_output.strides[0];
    index_t gOut_sC = grad_output.strides[1];
    index_t gOut_sH = grad_output.strides[2];
    index_t gOut_sW = grad_output.strides[3];
    // gInp_* (and NC_offset below) are not really needed if input_requires_grad is false.
    index_t gInp_sN;
    index_t gInp_sC;
    index_t gInp_sH;
    index_t gInp_sW;
    if (input_requires_grad) {
      gInp_sN = grad_input.strides[0];
      gInp_sC = grad_input.strides[1];
      gInp_sH = grad_input.strides[2];
      gInp_sW = grad_input.strides[3];
    }
    index_t gGrid_sW = grad_grid.strides[2];

    CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
      const index_t w = index % out_W;
      const index_t h = (index / out_W) % out_H;
      const index_t n = index / (out_H * out_W);
      const auto grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y coordinates from grid
      scalar_t x = grid.data[grid_offset];
      scalar_t y = grid.data[grid_offset + grid_sCoor];

      // multipliers for gradients on ix and iy
      scalar_t gix_mult, giy_mult;
      scalar_t ix = grid_sampler_compute_source_index_set_grad(x, inp_W, padding_mode, align_corners, &gix_mult);
      scalar_t iy = grid_sampler_compute_source_index_set_grad(y, inp_H, padding_mode, align_corners, &giy_mult);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get NE, NW, SE, SW pixel values from (x, y)
        index_t ix_nw = static_cast<index_t>(std::floor(ix));
        index_t iy_nw = static_cast<index_t>(std::floor(iy));
        index_t ix_ne = ix_nw + 1;
        index_t iy_ne = iy_nw;
        index_t ix_sw = ix_nw;
        index_t iy_sw = iy_nw + 1;
        index_t ix_se = ix_nw + 1;
        index_t iy_se = iy_nw + 1;

        // get surfaces to each neighbor:
        scalar_t nw = (ix_se - ix)    * (iy_se - iy);
        scalar_t ne = (ix    - ix_sw) * (iy_sw - iy);
        scalar_t sw = (ix_ne - ix)    * (iy    - iy_ne);
        scalar_t se = (ix    - ix_nw) * (iy    - iy_nw);

        scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0);
        const scalar_t *gOut_ptr_NCHW = grad_output.data + n * gOut_sN + h * gOut_sH + w * gOut_sW;
        index_t NC_offset = n * gInp_sN;
        const scalar_t *inp_ptr_NC = input.data + n * inp_sN;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, NC_offset += gInp_sC, gOut_ptr_NCHW += gOut_sC) {
          const scalar_t gOut = *gOut_ptr_NCHW;

          if (input_requires_grad) {
            // calculate and set grad_input. See Note [Passing pointer and offset to fastAtomicAdd].
            safe_add_2d(grad_input.data, iy_nw, ix_nw, gInp_sH, gInp_sW, inp_H, inp_W, nw * gOut, NC_offset, grad_input_memory_span);
            safe_add_2d(grad_input.data, iy_ne, ix_ne, gInp_sH, gInp_sW, inp_H, inp_W, ne * gOut, NC_offset, grad_input_memory_span);
            safe_add_2d(grad_input.data, iy_sw, ix_sw, gInp_sH, gInp_sW, inp_H, inp_W, sw * gOut, NC_offset, grad_input_memory_span);
            safe_add_2d(grad_input.data, iy_se, ix_se, gInp_sH, gInp_sW, inp_H, inp_W, se * gOut, NC_offset, grad_input_memory_span);
          }

          // calculate grad_grid
          if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
            scalar_t nw_val = inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW];
            gix -= nw_val * (iy_se - iy) * gOut;
            giy -= nw_val * (ix_se - ix) * gOut;
          }
          if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
            scalar_t ne_val = inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW];
            gix += ne_val * (iy_sw - iy) * gOut;
            giy -= ne_val * (ix - ix_sw) * gOut;
          }
          if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
            scalar_t sw_val = inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW];
            gix -= sw_val * (iy - iy_ne) * gOut;
            giy += sw_val * (ix_ne - ix) * gOut;
          }
          if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
            scalar_t se_val = inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW];
            gix += se_val * (iy - iy_nw) * gOut;
            giy += se_val * (ix - ix_nw) * gOut;
          }
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NHW
        //   2. directly assign to gGrid_ptr_NHW[0], gGrid_ptr_NHW[1]
        scalar_t *gGrid_ptr_NHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NHW[0] = gix_mult * gix;
        gGrid_ptr_NHW[1] = giy_mult * giy;
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        if (input_requires_grad) {
          index_t ix_nearest = static_cast<index_t>(std::nearbyint(ix));
          index_t iy_nearest = static_cast<index_t>(std::nearbyint(iy));

          // assign nearest neighbour pixel value to output pixel
          const scalar_t *gOut_ptr_NCHW = grad_output.data + n * gOut_sN + h * gOut_sH + w * gOut_sW;
          index_t NC_offset = n * gInp_sN;
          for (index_t c = 0; c < C; ++c, NC_offset += gInp_sC, gOut_ptr_NCHW += gOut_sC) {
            // calculate and set grad_input. See Note [Passing pointer and offset to fastAtomicAdd].
            safe_add_2d(grad_input.data, iy_nearest, ix_nearest, gInp_sH, gInp_sW, inp_H, inp_W, *gOut_ptr_NCHW, NC_offset, grad_input_memory_span);
          }
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NHW
        //   2. directly assign to gGrid_ptr_NHW[0], gGrid_ptr_NHW[1]
        scalar_t *gGrid_ptr_NHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NHW[0] = static_cast<scalar_t>(0);
        gGrid_ptr_NHW[1] = static_cast<scalar_t>(0);
      } else if (interpolation_mode == GridSamplerInterpolation::Bicubic) {

        ix = grid_sampler_unnormalize_set_grad(x, inp_W, align_corners, &gix_mult);
        iy = grid_sampler_unnormalize_set_grad(y, inp_H, align_corners, &giy_mult);

        scalar_t ix_nw = std::floor(ix);
        scalar_t iy_nw = std::floor(iy);

        const scalar_t tx = ix - ix_nw;
        const scalar_t ty = iy - iy_nw;

        scalar_t x_coeffs[4];
        scalar_t y_coeffs[4];
        scalar_t x_coeffs_grad[4];
        scalar_t y_coeffs_grad[4];

        get_cubic_upsampling_coefficients<scalar_t>(x_coeffs, tx);
        get_cubic_upsampling_coefficients<scalar_t>(y_coeffs, ty);
        get_cubic_coefficients_grad<scalar_t>(x_coeffs_grad, tx);
        get_cubic_coefficients_grad<scalar_t>(y_coeffs_grad, ty);

        scalar_t gix = static_cast<scalar_t>(0);
        scalar_t giy = static_cast<scalar_t>(0);

        const scalar_t *gOut_ptr_NCHW = grad_output.data + n * gOut_sN + h * gOut_sH + w * gOut_sW;
        index_t NC_offset = n * gInp_sN;
        const scalar_t *inp_ptr_NC = input.data + n * inp_sN;

        for (index_t c = 0; c < C; ++c, gOut_ptr_NCHW += gOut_sC, NC_offset += gInp_sC, inp_ptr_NC+= inp_sC) {
          const scalar_t gOut = *gOut_ptr_NCHW;

          #pragma unroll 4
          for (index_t i = 0; i < 4; ++i) {
            #pragma unroll 4
            for (index_t j = 0; j < 4; ++j) {

              if (input_requires_grad) {
                // set input gradient. See Note [Passing pointer and offset to fastAtomicAdd].
                add_value_bounded<scalar_t>(grad_input.data, ix_nw - 1 + i, iy_nw - 1 + j, inp_W, inp_H, gInp_sW, gInp_sH,
                  gOut * x_coeffs[i] * y_coeffs[j],
                  padding_mode,
                  align_corners,
                  NC_offset,
                  grad_input_memory_span);
              }

              // set grid gradient
              scalar_t val = get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw - 1 + i, iy_nw - 1 + j,
                inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners);

              gix -= val * x_coeffs_grad[i] * y_coeffs[j] * gOut;
              giy -= val * y_coeffs_grad[j] * x_coeffs[i] * gOut;
            }
          }
        }

        scalar_t *gGrid_ptr_NHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NHW[0] = gix_mult * gix;
        gGrid_ptr_NHW[1] = giy_mult * giy;
      }
    }
  }

  template <typename scalar_t, typename index_t>
  C10_LAUNCH_BOUNDS_1(256)
  __global__ void grid_sampler_3d_backward_kernel(
      const index_t nthreads,
      TensorInfo<const scalar_t, index_t> grad_output,
      TensorInfo<const scalar_t, index_t> input,
      TensorInfo<const scalar_t, index_t> grid,
      TensorInfo<scalar_t, index_t> grad_input,  // initialized to zeros (or unused if input_requires_grad is false)
      TensorInfo<scalar_t, index_t> grad_grid,   // initialized to empty
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners,
      const index_t grad_input_memory_span,
      const bool input_requires_grad) {

    index_t C = input.sizes[1];
    index_t inp_D = input.sizes[2];
    index_t inp_H = input.sizes[3];
    index_t inp_W = input.sizes[4];
    index_t out_D = grid.sizes[1];
    index_t out_H = grid.sizes[2];
    index_t out_W = grid.sizes[3];
    index_t inp_sN = input.strides[0];
    index_t inp_sC = input.strides[1];
    index_t inp_sD = input.strides[2];
    index_t inp_sH = input.strides[3];
    index_t inp_sW = input.strides[4];
    index_t grid_sN = grid.strides[0];
    index_t grid_sD = grid.strides[1];
    index_t grid_sH = grid.strides[2];
    index_t grid_sW = grid.strides[3];
    index_t grid_sCoor = grid.strides[4];
    index_t gOut_sN = grad_output.strides[0];
    index_t gOut_sC = grad_output.strides[1];
    index_t gOut_sD = grad_output.strides[2];
    index_t gOut_sH = grad_output.strides[3];
    index_t gOut_sW = grad_output.strides[4];
    // gInp_* (and NC_offset below) are not really needed if input_requires_grad is false.
    int64_t gInp_sN = 0;
    int64_t gInp_sC = 0;
    int64_t gInp_sD = 0;
    int64_t gInp_sH = 0;
    int64_t gInp_sW = 0;
    if (input_requires_grad) {
      gInp_sN = grad_input.strides[0];
      gInp_sC = grad_input.strides[1];
      gInp_sD = grad_input.strides[2];
      gInp_sH = grad_input.strides[3];
      gInp_sW = grad_input.strides[4];
    }
    index_t gGrid_sW = grad_grid.strides[3];

    CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
      const index_t w = index % out_W;
      const index_t h = (index / out_W) % out_H;
      const index_t d = (index / (out_H * out_W)) % out_D;
      const index_t n = index / (out_D * out_H * out_W);
      const auto grid_offset = n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y, z coordinates from grid
      scalar_t ix = grid.data[grid_offset];
      scalar_t iy = grid.data[grid_offset + grid_sCoor];
      scalar_t iz = grid.data[grid_offset + 2 * grid_sCoor];

      // multipliers for gradients on ix, iy, and iz
      scalar_t gix_mult, giy_mult, giz_mult;
      ix = grid_sampler_compute_source_index_set_grad(ix, inp_W, padding_mode, align_corners, &gix_mult);
      iy = grid_sampler_compute_source_index_set_grad(iy, inp_H, padding_mode, align_corners, &giy_mult);
      iz = grid_sampler_compute_source_index_set_grad(iz, inp_D, padding_mode, align_corners, &giz_mult);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get corner pixel values from (x, y, z)
        // for 4d, we used north-east-south-west
        // for 5d, we add top-bottom
        index_t ix_tnw = static_cast<index_t>(std::floor(ix));
        index_t iy_tnw = static_cast<index_t>(std::floor(iy));
        index_t iz_tnw = static_cast<index_t>(std::floor(iz));

        index_t ix_tne = ix_tnw + 1;
        index_t iy_tne = iy_tnw;
        index_t iz_tne = iz_tnw;

        index_t ix_tsw = ix_tnw;
        index_t iy_tsw = iy_tnw + 1;
        index_t iz_tsw = iz_tnw;

        index_t ix_tse = ix_tnw + 1;
        index_t iy_tse = iy_tnw + 1;
        index_t iz_tse = iz_tnw;

        index_t ix_bnw = ix_tnw;
        index_t iy_bnw = iy_tnw;
        index_t iz_bnw = iz_tnw + 1;

        index_t ix_bne = ix_tnw + 1;
        index_t iy_bne = iy_tnw;
        index_t iz_bne = iz_tnw + 1;

        index_t ix_bsw = ix_tnw;
        index_t iy_bsw = iy_tnw + 1;
        index_t iz_bsw = iz_tnw + 1;

        index_t ix_bse = ix_tnw + 1;
        index_t iy_bse = iy_tnw + 1;
        index_t iz_bse = iz_tnw + 1;

        // get surfaces to each neighbor:
        scalar_t tnw = (ix_bse - ix)    * (iy_bse - iy)    * (iz_bse - iz);
        scalar_t tne = (ix    - ix_bsw) * (iy_bsw - iy)    * (iz_bsw - iz);
        scalar_t tsw = (ix_bne - ix)    * (iy    - iy_bne) * (iz_bne - iz);
        scalar_t tse = (ix    - ix_bnw) * (iy    - iy_bnw) * (iz_bnw - iz);
        scalar_t bnw = (ix_tse - ix)    * (iy_tse - iy)    * (iz - iz_tse);
        scalar_t bne = (ix    - ix_tsw) * (iy_tsw - iy)    * (iz - iz_tsw);
        scalar_t bsw = (ix_tne - ix)    * (iy    - iy_tne) * (iz - iz_tne);
        scalar_t bse = (ix    - ix_tnw) * (iy    - iy_tnw) * (iz - iz_tnw);

        scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0), giz = static_cast<scalar_t>(0);
        const scalar_t *gOut_ptr_NCDHW = grad_output.data + n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
        index_t NC_offset;
        if (input_requires_grad) {
          NC_offset = n * gInp_sN;
        }
        const scalar_t *inp_ptr_NC = input.data + n * inp_sN;
        // calculate bilinear weighted pixel value and set output pixel
        for (index_t c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, NC_offset += gInp_sC, inp_ptr_NC += inp_sC) {
          scalar_t gOut = *gOut_ptr_NCDHW;

          // calculate and set grad_input. See Note [Passing pointer and offset to fastAtomicAdd].
          if (input_requires_grad) {
            safe_add_3d(grad_input.data, iz_tnw, iy_tnw, ix_tnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tnw * gOut,
                        NC_offset, grad_input_memory_span);
            safe_add_3d(grad_input.data, iz_tne, iy_tne, ix_tne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tne * gOut,
                        NC_offset, grad_input_memory_span);
            safe_add_3d(grad_input.data, iz_tsw, iy_tsw, ix_tsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tsw * gOut,
                        NC_offset, grad_input_memory_span);
            safe_add_3d(grad_input.data, iz_tse, iy_tse, ix_tse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tse * gOut,
                        NC_offset, grad_input_memory_span);
            safe_add_3d(grad_input.data, iz_bnw, iy_bnw, ix_bnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bnw * gOut,
                        NC_offset, grad_input_memory_span);
            safe_add_3d(grad_input.data, iz_bne, iy_bne, ix_bne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bne * gOut,
                        NC_offset, grad_input_memory_span);
            safe_add_3d(grad_input.data, iz_bsw, iy_bsw, ix_bsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bsw * gOut,
                        NC_offset, grad_input_memory_span);
            safe_add_3d(grad_input.data, iz_bse, iy_bse, ix_bse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bse * gOut,
                        NC_offset, grad_input_memory_span);
          }
          // calculate grad_grid
          if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, inp_D, inp_H, inp_W)) {
            scalar_t tnw_val = inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW];
            gix -= tnw_val * (iy_bse - iy)    * (iz_bse - iz)    * gOut;
            giy -= tnw_val * (ix_bse - ix)    * (iz_bse - iz)    * gOut;
            giz -= tnw_val * (ix_bse - ix)    * (iy_bse - iy)    * gOut;
          }
          if (within_bounds_3d(iz_tne, iy_tne, ix_tne, inp_D, inp_H, inp_W)) {
            scalar_t tne_val = inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW];
            gix += tne_val * (iy_bsw - iy)    * (iz_bsw - iz)    * gOut;
            giy -= tne_val * (ix    - ix_bsw) * (iz_bsw - iz)    * gOut;
            giz -= tne_val * (ix    - ix_bsw) * (iy_bsw - iy)    * gOut;
          }
          if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, inp_D, inp_H, inp_W)) {
            scalar_t tsw_val = inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW];
            gix -= tsw_val * (iy - iy_bne)    * (iz_bne - iz)    * gOut;
            giy += tsw_val * (ix_bne - ix)    * (iz_bne - iz)    * gOut;
            giz -= tsw_val * (ix_bne - ix)    * (iy    - iy_bne) * gOut;
          }
          if (within_bounds_3d(iz_tse, iy_tse, ix_tse, inp_D, inp_H, inp_W)) {
            scalar_t tse_val = inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW];
            gix += tse_val * (iy - iy_bnw)    * (iz_bnw - iz)    * gOut;
            giy += tse_val * (ix    - ix_bnw) * (iz_bnw - iz)    * gOut;
            giz -= tse_val * (ix    - ix_bnw) * (iy    - iy_bnw) * gOut;
          }
          if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, inp_D, inp_H, inp_W)) {
            scalar_t bnw_val = inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW];
            gix -= bnw_val * (iy_tse - iy)    * (iz - iz_tse)    * gOut;
            giy -= bnw_val * (ix_tse - ix)    * (iz - iz_tse)    * gOut;
            giz += bnw_val * (ix_tse - ix)    * (iy_tse - iy)    * gOut;
          }
          if (within_bounds_3d(iz_bne, iy_bne, ix_bne, inp_D, inp_H, inp_W)) {
            scalar_t bne_val = inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW];
            gix += bne_val * (iy_tsw - iy)    * (iz - iz_tsw)    * gOut;
            giy -= bne_val * (ix    - ix_tsw) * (iz - iz_tsw)    * gOut;
            giz += bne_val * (ix    - ix_tsw) * (iy_tsw - iy)    * gOut;
          }
          if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, inp_D, inp_H, inp_W)) {
            scalar_t bsw_val = inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW];
            gix -= bsw_val * (iy - iy_tne)    * (iz - iz_tne)    * gOut;
            giy += bsw_val * (ix_tne - ix)    * (iz - iz_tne)    * gOut;
            giz += bsw_val * (ix_tne - ix)    * (iy    - iy_tne) * gOut;
          }
          if (within_bounds_3d(iz_bse, iy_bse, ix_bse, inp_D, inp_H, inp_W)) {
            scalar_t bse_val = inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW];
            gix += bse_val * (iy - iy_tnw)    * (iz - iz_tnw)    * gOut;
            giy += bse_val * (ix    - ix_tnw) * (iz - iz_tnw)    * gOut;
            giz += bse_val * (ix    - ix_tnw) * (iy    - iy_tnw) * gOut;
          }
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NDHW
        //   2. directly assign to gGrid_ptr_NDHW[0], gGrid_ptr_NDHW[1], gGrid_ptr_NDHW[2]
        scalar_t *gGrid_ptr_NDHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NDHW[0] = gix_mult * gix;
        gGrid_ptr_NDHW[1] = giy_mult * giy;
        gGrid_ptr_NDHW[2] = giz_mult * giz;
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        if (input_requires_grad) {
          auto ix_nearest = static_cast<index_t>(std::nearbyint(ix));
          auto iy_nearest = static_cast<index_t>(std::nearbyint(iy));
          auto iz_nearest = static_cast<index_t>(std::nearbyint(iz));

          // assign nearest neighbour pixel value to output pixel
          const scalar_t *gOut_ptr_NCDHW = grad_output.data + n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
          index_t NC_offset = n * gInp_sN;
          for (index_t c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, NC_offset += gInp_sC) {
            // calculate and set grad_input. See Note [Passing pointer and offset to fastAtomicAdd].
            safe_add_3d(grad_input.data, iz_nearest, iy_nearest, ix_nearest,
                        gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, *gOut_ptr_NCDHW,
                        NC_offset, grad_input_memory_span);
          }
        }
        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NDHW
        //   2. directly assign to gGrid_ptr_NDHW[0], gGrid_ptr_NDHW[1], gGrid_ptr_NDHW[2]
        scalar_t *gGrid_ptr_NDHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NDHW[0] = static_cast<scalar_t>(0);
        gGrid_ptr_NDHW[1] = static_cast<scalar_t>(0);
        gGrid_ptr_NDHW[2] = static_cast<scalar_t>(0);
      }
    }
  }
}  // namespace

void launch_grid_sampler_2d_forward_kernel(
    const TensorBase &output, const TensorBase &input, const TensorBase &grid,
    int64_t interpolation_mode, int64_t padding_mode, bool align_corners) {
  // See NOTE [ grid_sampler Native Functions ].
  // Add checks here in case this is called instead of grid_sampler.
  check_grid_sampler_common(input, grid);
  check_grid_sampler_2d(input, grid);

  auto N = input.size(0);
  auto H = grid.size(1);
  auto W = grid.size(2);
  int64_t count = N * H * W;
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      input.scalar_type(), "grid_sampler_2d_cuda", [&] {
      if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
          canUse32BitIndexMath(output)) {
        grid_sampler_2d_kernel<scalar_t>
          <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<int>(count),
            getTensorInfo<const scalar_t, int>(input),
            getTensorInfo<const scalar_t, int>(grid),
            getTensorInfo<scalar_t, int>(output),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        grid_sampler_2d_kernel<scalar_t>
          <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
            getTensorInfo<const scalar_t, int64_t>(input),
            getTensorInfo<const scalar_t, int64_t>(grid),
            getTensorInfo<scalar_t, int64_t>(output),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
}

void launch_grid_sampler_3d_forward_kernel(
    const TensorBase &output, const TensorBase &input, const TensorBase &grid,
    int64_t interpolation_mode, int64_t padding_mode, bool align_corners) {
  // See NOTE [ grid_sampler Native Functions ].
  // Add checks here in case this is called instead of grid_sampler.
  check_grid_sampler_common(input, grid);
  check_grid_sampler_3d(input, grid, interpolation_mode);

  auto N = input.size(0);
  auto D = grid.size(1);
  auto H = grid.size(2);
  auto W = grid.size(3);
  int64_t count = N * D * H * W;
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      input.scalar_type(), "grid_sampler_3d_cuda", [&] {
      if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
          canUse32BitIndexMath(output)) {
        grid_sampler_3d_kernel<scalar_t>
          <<<GET_BLOCKS(count, 512), 512, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<int>(count),
            getTensorInfo<const scalar_t, int>(input),
            getTensorInfo<const scalar_t, int>(grid),
            getTensorInfo<scalar_t, int>(output),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        grid_sampler_3d_kernel<scalar_t>
          <<<GET_BLOCKS(count, 512), 512, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
            getTensorInfo<const scalar_t, int64_t>(input),
            getTensorInfo<const scalar_t, int64_t>(grid),
            getTensorInfo<scalar_t, int64_t>(output),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
}

void launch_grid_sampler_2d_backward_kernel(
    const TensorBase &grad_input, const TensorBase &grad_grid,
    const TensorBase &grad_output, const TensorBase &input,
    const TensorBase &grid, int64_t interpolation_mode, int64_t padding_mode,
    bool align_corners, std::array<bool,2> output_mask) {
  // See NOTE [ grid_sampler Native Functions ].
  // Add checks here in case this is called instead of grid_sampler.
  check_grid_sampler_common(input, grid);
  check_grid_sampler_2d(input, grid);

  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("grid_sampler_2d_backward_cuda");
  auto N = input.size(0);
  auto H = grid.size(1);
  auto W = grid.size(2);

  // If `input` gradient is not required, we skip computing it -- not needing to create
  // the tensor to hold the gradient can markedly increase performance. (`grid` gradient
  // is always computed.)
  auto input_requires_grad = output_mask[0];

  int64_t count = N * H * W;
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      input.scalar_type(), "grid_sampler_2d_backward_cuda", [&] {
      if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
          canUse32BitIndexMath(grad_output)) {
        grid_sampler_2d_backward_kernel<scalar_t>
          <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<int>(count),
            getTensorInfo<const scalar_t, int>(grad_output),
            getTensorInfo<const scalar_t, int>(input),
            getTensorInfo<const scalar_t, int>(grid),
            input_requires_grad ? getTensorInfo<scalar_t, int>(grad_input) : TensorInfo<scalar_t, int>(),
            getTensorInfo<scalar_t, int>(grad_grid),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners,
            /*grad_input_memory_span =*/input_requires_grad ? static_cast<int>(grad_input.numel()) : 0,
            input_requires_grad);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        grid_sampler_2d_backward_kernel<scalar_t>
          <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
            getTensorInfo<const scalar_t, int64_t>(grad_output),
            getTensorInfo<const scalar_t, int64_t>(input),
            getTensorInfo<const scalar_t, int64_t>(grid),
            input_requires_grad ? getTensorInfo<scalar_t, int64_t>(grad_input) : TensorInfo<scalar_t, int64_t>(),
            getTensorInfo<scalar_t, int64_t>(grad_grid),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners,
            /*grad_input_memory_span =*/input_requires_grad ? grad_input.numel() : 0,
            input_requires_grad);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
}

void launch_grid_sampler_3d_backward_kernel(
    const TensorBase &grad_input, const TensorBase &grad_grid,
    const TensorBase& grad_output, const TensorBase& input,
    const TensorBase& grid, int64_t interpolation_mode, int64_t padding_mode,
    bool align_corners, std::array<bool,2> output_mask) {
  // See NOTE [ grid_sampler Native Functions ].
  // Add checks here in case this is called instead of grid_sampler.
  check_grid_sampler_common(input, grid);
  check_grid_sampler_3d(input, grid, interpolation_mode);

  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("grid_sampler_3d_backward_cuda");
  auto N = input.size(0);
  auto D = grid.size(1);
  auto H = grid.size(2);
  auto W = grid.size(3);
  int64_t count = N * D * H * W;
  auto input_requires_grad = output_mask[0];
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      input.scalar_type(), "grid_sampler_3d_backward_cuda", [&] {
      if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
          canUse32BitIndexMath(grad_output)) {
        grid_sampler_3d_backward_kernel<scalar_t>
          <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<int>(count),
            getTensorInfo<const scalar_t, int>(grad_output),
            getTensorInfo<const scalar_t, int>(input),
            getTensorInfo<const scalar_t, int>(grid),
            input_requires_grad ? getTensorInfo<scalar_t, int>(grad_input) : TensorInfo<scalar_t, int>(),
            getTensorInfo<scalar_t, int>(grad_grid),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners,
            /*grad_input_memory_span =*/input_requires_grad ? static_cast<int>(grad_input.numel()) : 0,
            input_requires_grad);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        grid_sampler_3d_backward_kernel<scalar_t>
          <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
            getTensorInfo<const scalar_t, int64_t>(grad_output),
            getTensorInfo<const scalar_t, int64_t>(input),
            getTensorInfo<const scalar_t, int64_t>(grid),
            input_requires_grad ? getTensorInfo<scalar_t, int64_t>(grad_input) : TensorInfo<scalar_t, int64_t>(),
            getTensorInfo<scalar_t, int64_t>(grad_grid),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners,
            /*grad_input_memory_span =*/input_requires_grad ? grad_input.numel() : 0,
            input_requires_grad);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
}

}  // namespace at::native
