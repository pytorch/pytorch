#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if 0 && AT_USE_JITERATOR()
constexpr char tan_name[] = "tan_impl";
#endif

void tan_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
    // Disabled due to accuracy issues
#if 0 && AT_USE_JITERATOR()
    static const auto tan_string = jiterator_stringify(
        template <typename T> T tan_impl(T a) { return std::tan(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "tan_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/tan_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, tan_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "tan_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::tan(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        ScalarType::Half,
        ScalarType::BFloat16,
        common_dtype,
        "tan_cuda",
        [&]() {
          gpu_kernel(
              iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t { return ::tan(a); });
        });
  }
}

REGISTER_DISPATCH(tan_stub, &tan_kernel_cuda)

} // namespace at::native
