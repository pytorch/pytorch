#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/cuda/HIPContext.h>
#include <ATen/Config.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/detail/CUDAHooksInterface.h>
#include <ATen/native/SpectralOpsUtils.h>

#include <array>
#include <cmath>


namespace at::native {

// Offset calculator for indexing in Hermitian mirrored order.
// In mirrored dims, maps linear index i to (n - i) % n
template <typename index_t>
struct HermitianSymmetryOffsetCalculator {
  using offset_type = std::array<index_t, 1>;
  using dim_type = std::remove_cv_t<decltype(MAX_DIMS)>;
  dim_type dims;
  at::cuda::detail::IntDivider<index_t> sizes_[MAX_DIMS];
  index_t strides_[MAX_DIMS];
  uint32_t mirror_dim_;  // bit mask
  static_assert(MAX_DIMS < 32, "Need a bigger mask type");

  HermitianSymmetryOffsetCalculator(
      IntArrayRef sizes, IntArrayRef strides, IntArrayRef dim,
      const int64_t element_size){
    TORCH_INTERNAL_ASSERT(sizes.size() == strides.size());
    TORCH_INTERNAL_ASSERT(sizes.size() <= MAX_DIMS);
    dims = sizes.size();

    using at::cuda::detail::IntDivider;
    for (dim_type i = 0; i < MAX_DIMS; ++i) {
      if (i < dims) {
        sizes_[i] = IntDivider<index_t>(sizes[i]);
        strides_[i] = strides[i] / element_size;
      } else {
        sizes_[i] = IntDivider<index_t>(1);
        strides_[i] = 0;
      }
    }

    mirror_dim_ = 0;
    for (const auto i: c10::irange(dim.size())) {
      mirror_dim_ |= (uint32_t{1} << dim[i]);
    }
  }

  C10_HOST_DEVICE offset_type get(index_t linear_idx) const {
    index_t offset = 0;

    for (dim_type dim = 0; dim < dims; ++dim) {
      auto divmod = sizes_[dim].divmod(linear_idx);
      linear_idx = divmod.div;

      if ((mirror_dim_ & (uint32_t{1} << dim)) == 0) {
        offset += divmod.mod * strides_[dim];
      } else if (divmod.mod != 0) {
        offset += (sizes_[dim].divisor - divmod.mod) * strides_[dim];
      }
    }
    offset_type offsets;
    offsets[0] = offset;
    return offsets;
  }
};


// out[:] = conj(in[:]) where in and out ordering is generalized by offset calculators
template <typename scalar_t, typename inp_calc_t, typename out_calc_t>
C10_LAUNCH_BOUNDS_1(cuda::detail::CUDA_NUM_THREADS)
__global__ void _fft_conjugate_copy_kernel(
    int64_t numel, scalar_t * out_data, const scalar_t * in_data,
    inp_calc_t ic, out_calc_t oc) {
  CUDA_KERNEL_LOOP_TYPE(index, numel, int64_t) {
    auto in_offset = ic.get(index)[0];
    auto out_offset = oc.get(index)[0];
    out_data[out_offset] = std::conj(in_data[in_offset]);
  }
}

// In real-to-complex transform, cuFFT only fills half of the values due to
// conjugate symmetry. See native/SpectralUtils.h for more details.
// The following function fills in the other half with symmetry in
// case of real-to-complex transform with onesided=False flag.
// See NOTE [ Fourier Transform Conjugate Symmetry ] in native/SpectralOpsUtils.h.

// input should be a tensor of same size as full (twosided)
// signals, but only contains half (onesided) of the values.
// This function modifies inplace.
void _fft_fill_with_conjugate_symmetry_cuda_(
    ScalarType dtype, IntArrayRef mirror_dims, IntArrayRef signal_half_sizes,
    IntArrayRef in_strides, const void * in_data,
    IntArrayRef out_strides, void * out_data) {
  // Do the actual conjugate mirroring.
  // TODO: consider adding a 32bit indexed kernel for improved performance
  auto* in_strides_ptr = in_strides.data();
  const int ndim = in_strides.size();
  const int64_t element_size = scalarTypeToTypeMeta(dtype).itemsize();
  OffsetCalculator<1, int64_t> input_offset_calculator(
      ndim, signal_half_sizes.data(), &in_strides_ptr, &element_size);
  HermitianSymmetryOffsetCalculator<int64_t> output_offset_calculator(
      signal_half_sizes, out_strides, mirror_dims, element_size);

  const auto numel = c10::multiply_integers(signal_half_sizes);
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "_fft_fill_with_conjugate_symmetry", [&] {
      using namespace cuda::detail;
      _fft_conjugate_copy_kernel<<<
        GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            numel,
            static_cast<scalar_t*>(out_data),
            static_cast<const scalar_t*>(in_data),
            input_offset_calculator,
            output_offset_calculator);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
}

REGISTER_DISPATCH(fft_fill_with_conjugate_symmetry_stub, &_fft_fill_with_conjugate_symmetry_cuda_)

} // at::native
