#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/cuda/Sort.h>
#include <ATen/core/TensorBase.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/cub.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/NumericLimits.cuh>
#include <ATen/native/cuda/SortUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>

#include <limits>

namespace at::native {

template <typename T>
static int minimum_grid_for_occupancy(T kernel, int max_block_size) {
  int minGridSize = 0;
  int blockSize = 0;
  C10_CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(
      &minGridSize,
      &blockSize,
      kernel,
      /*dynamicSMemSize=*/0,
      max_block_size));
  return minGridSize;
}

template <typename T>
constexpr bool has_nan() {
  if constexpr (std::numeric_limits<T>::is_specialized) {
    return std::numeric_limits<T>::has_quiet_NaN;
  } else if constexpr (
      c10::is_complex<T>::value ||
      std::is_same_v<T, c10::BFloat16> ||
      std::is_same_v<T, c10::Half>) {
    return true;
  }
}

// For very small unstable sorts (n <= 32), use bitonicSortKVInPlace
// which can sort multiple arrays within the same block of threads,
// improving occupancy.
struct SmallBitonicSort {
  template <int A, typename K, typename V, typename IndexType>
  void sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {
    constexpr int sort_size = 32;
    constexpr int max_block_y = 16;
    constexpr int items_per_thread = 2;
    static_assert(sort_size % items_per_thread == 0, "");
    constexpr int block_x = sort_size / items_per_thread;

    TORCH_INTERNAL_ASSERT(keySliceSize <= sort_size);

    // Scale batch size down if the grid would be too small
    const auto min_grid = minimum_grid_for_occupancy(
        bitonicSortKVInPlace<
            A, -1, block_x, max_block_y,
            K, V, LTOp<K, true>, IndexType>,
        block_x * max_block_y);
    const auto max_batch = std::max(IndexType{1}, keySlices / min_grid);
    const int block_y = std::min(IndexType(max_block_y), max_batch);
    dim3 block(block_x, block_y);

    dim3 grid;
    const int grid_count = (keySlices + block_y - 1) / block_y;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(grid_count, grid),
                          "Too many slices to sort");
    const auto stream = at::cuda::getCurrentCUDAStream();

    if (descending) {
      bitonicSortKVInPlace<A, -1, block_x, max_block_y>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          GTOp<K, true>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      bitonicSortKVInPlace<A, -1, block_x, max_block_y>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          LTOp<K, true>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
};

#if HAS_WARP_MERGE_SORT()

// For small sorts (n <= 128) we use warpMergeSortKVInPlace which
// sorts one slice per warp and potentially multiple slices in the
// same block for improved occupancy with large batch sizes.
template <int sort_size>
struct WarpMergeSort {

  template <int A, typename K, typename V, typename IndexType>
  void sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {
    constexpr int max_block_y = 16;
    const int block_x = at::cuda::warp_size();

    TORCH_INTERNAL_ASSERT(keySliceSize <= sort_size);

    // Scale batch size down if the grid would be too small
    const auto min_grid = minimum_grid_for_occupancy(
        warpMergeSortKVInPlace<
            A, -1, sort_size, max_block_y,
            K, V, LTOp<K, true>, IndexType>,
        block_x * max_block_y);
    const auto max_batch = std::max(IndexType{1}, keySlices / min_grid);
    const int block_y = std::min(IndexType(max_block_y), max_batch);
    dim3 block(block_x, block_y);

    dim3 grid;
    const int grid_count = (keySlices + block_y - 1) / block_y;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(grid_count, grid),
                          "Too many slices to sort");
    const auto stream = at::cuda::getCurrentCUDAStream();

    if (descending) {
      const K invalid_key = at::numeric_limits<K>::lower_bound();
      warpMergeSortKVInPlace<A, -1, sort_size, max_block_y>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          GTOp<K, true>(),
          invalid_key);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      const K invalid_key = []{
        // NAN is sorted after inf
        if constexpr(has_nan<K>()) {
          return K(NAN);
        }
        return at::numeric_limits<K>::upper_bound();
      }();
      warpMergeSortKVInPlace<A, -1, sort_size, max_block_y>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          LTOp<K, true>(),
          invalid_key);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
};

#endif // !HAS_WARP_MERGE_SORT()

// For medium sizes (128 < n <= 4096) use radixSortKVInplace.
struct MediumRadixSort {

  template <int A, typename K, typename V, typename IndexType>
  void sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {

#define HANDLE_CASE(SIZE, ITEMS_PER_THREAD)         \
    fixed_size_sort<A, SIZE, ITEMS_PER_THREAD>(     \
        keyInfo,                                    \
        keySlices,                                  \
        keySliceSize,                               \
        keySliceStride,                             \
        valueInfo,                                  \
        valueSliceStride,                           \
        descending)

    int64_t ceilPowerOf2 = nextHighestPowerOf2(keySliceSize);
    TORCH_INTERNAL_ASSERT(ceilPowerOf2 <= 4096);
#ifdef USE_ROCM
    constexpr int default_ipt = 8;
#else
    constexpr int default_ipt = 32;
#endif
    switch (ceilPowerOf2) {
      case 4096:
        HANDLE_CASE(4096, default_ipt);
        break;
      case 2048:
        HANDLE_CASE(2048, default_ipt);
        break;
      case 1024:
      case 512:
      case 256:
        HANDLE_CASE(1024, default_ipt);
        break;
      case 128:
      case 64:
#if !HAS_WARP_MERGE_SORT()
        HANDLE_CASE(128, 4);
        break;
#endif
      case 32:
      case 16:
      case 8:
      case 4:
      case 2:
#if HAS_WARP_MERGE_SORT()
        TORCH_INTERNAL_ASSERT(
            false, "Expected size <= 128 to be handled by a different algorithm");
#else
        HANDLE_CASE(32, 2);
#endif
        break;
      case 1:
        /* Nothing to do, data already sorted */
        break;
      default:
        TORCH_INTERNAL_ASSERT(false);
    }
#undef HANDLE_CASE

  }

  template <int A, int sort_size, int items_per_thread,
            typename K, typename V, typename IndexType>
  void fixed_size_sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {
    static_assert(sort_size % items_per_thread == 0, "");
    constexpr int block = sort_size / items_per_thread;
    dim3 grid;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(keySlices, grid),
                          "Too many slices to sort");

    const auto stream = at::cuda::getCurrentCUDAStream();
    radixSortKVInPlace<A, -1, block, items_per_thread>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          descending);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
};

template <typename Sorter>
void sortCommon(Sorter sorter, const TensorBase &key, const TensorBase &value,
                int dim, bool descending) {
  TORCH_CHECK(key.sizes() == value.sizes(),
              "Key tensor must have same size as value tensor");
  int dims = value.dim();
  TORCH_CHECK(dims <= MAX_DIMS, "value tensor has too many dimensions");
  // if key and value tensors have the same size, we do not need to check both

  ptrdiff_t inElements = key.numel();

  if (inElements == 0) {
    return;
  }

  int64_t keySliceSize = key.size(dim);
  ptrdiff_t keySlices = inElements / keySliceSize;

#define HANDLE_SORT_CASE(TYPE, A)                   \
  sorter.template sort<A>(                          \
      keyInfo,                                      \
      (TYPE) keySlices,                             \
      (TYPE) keySliceSize,                          \
      (TYPE) keyInfo.strides[collapseKeyDim],       \
      valueInfo,                                    \
      (TYPE) valueInfo.strides[collapseValueDim],   \
      descending)

  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Half, at::ScalarType::BFloat16, at::ScalarType::Bool, key.scalar_type(), "sortKeyValueInplace", [&]  {
    if (at::cuda::detail::canUse32BitIndexMath(key)) {
      at::cuda::detail::TensorInfo<scalar_t, unsigned int> keyInfo =
        at::cuda::detail::getTensorInfo<scalar_t, unsigned int>(key);
      at::cuda::detail::TensorInfo<int64_t, unsigned int> valueInfo =
        at::cuda::detail::getTensorInfo<int64_t, unsigned int>(value);

      auto strideKey = keyInfo.strides[dim];
      keyInfo.sizes[dim] = 1;
      int collapseKeyDim = keyInfo.collapseDims(dim);
      keyInfo.strides[collapseKeyDim] = strideKey;
      auto strideValue = valueInfo.strides[dim];
      valueInfo.sizes[dim]=1;
      int collapseValueDim = valueInfo.collapseDims(dim);
      valueInfo.strides[collapseValueDim] = strideValue;

      if (keyInfo.isContiguous()) {
        HANDLE_SORT_CASE(unsigned int, -2);
      } else {
        switch (keyInfo.dims) {
          case 2:
            HANDLE_SORT_CASE(unsigned int, 2);
            break;
          default:
            HANDLE_SORT_CASE(unsigned int, -1);
            break;
        }
      }

    } else {
      at::cuda::detail::TensorInfo<scalar_t, uint64_t> keyInfo =
        at::cuda::detail::getTensorInfo<scalar_t, uint64_t>(key);
      at::cuda::detail::TensorInfo<int64_t, uint64_t> valueInfo =
        at::cuda::detail::getTensorInfo<int64_t, uint64_t>(value);

      auto strideKey = keyInfo.strides[dim];
      keyInfo.sizes[dim] = 1;
      int collapseKeyDim = keyInfo.collapseDims(dim);
      keyInfo.strides[collapseKeyDim] = strideKey;
      auto strideValue = valueInfo.strides[dim];
      valueInfo.sizes[dim]=1;
      int collapseValueDim = valueInfo.collapseDims(dim);
      valueInfo.strides[collapseValueDim] = strideValue;

      // int64_t case is rare, just instantiate the generic version
      HANDLE_SORT_CASE(uint64_t, -1);
    }
  });
#undef HANDLE_SORT_CASE
}

void sortKeyValueInplace(
    const TensorBase& key,
    const TensorBase& value,
    int64_t dim,
    bool descending,
    bool stable) {
  const auto sort_size = key.size(dim);
  if (sort_size <= 1) {
    return; // Already sorted
  } else if (!stable && sort_size <= 32) {
    // NOTE: Bitonic sort is unstable
    sortCommon(SmallBitonicSort{}, key, value, dim, descending);
#if HAS_WARP_MERGE_SORT()
  } else if (sort_size <= 128) {
    sortCommon(WarpMergeSort<128>{}, key, value, dim, descending);
#endif
  } else {
    sortCommon(MediumRadixSort{}, key, value, dim, descending);
  }
}

}  // namespace at::native
