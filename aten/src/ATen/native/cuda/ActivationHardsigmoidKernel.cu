#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/core/TensorBase.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <ATen/cuda/ApplyGridUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/Loops.cuh>

namespace at::native {
namespace {

void hardsigmoid_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "hardsigmoid_cuda",
      [&]() {
        using opmath_t = at::opmath_type<scalar_t>;
        const opmath_t zero(0.0f);
        const opmath_t one_sixth(1.0f / 6.0f);
        const opmath_t three(3.0f);
        const opmath_t six(6.0f);
        gpu_kernel(
            iter,
            [zero, one_sixth, three, six] GPU_LAMBDA(
                scalar_t self_val) -> scalar_t {
              opmath_t x = static_cast<opmath_t>(self_val);
              return std::min<opmath_t>(std::max<opmath_t>(x + three, zero), six) * one_sixth;
            });
      });
}

void hardsigmoid_backward_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "hardsigmoid_backward_cuda",
      [&]() {
        using opmath_t = at::opmath_type<scalar_t>;
        const opmath_t zero(0.0f);
        const opmath_t three(3.0f);
        const opmath_t neg_three(-3.0f);
        const opmath_t one_sixth(1.0f / 6.0f);
        gpu_kernel(
            iter,
            [zero, three, neg_three, one_sixth] GPU_LAMBDA(
                scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
              opmath_t grad_val = static_cast<opmath_t>(grad_val_);
              opmath_t self_val = static_cast<opmath_t>(self_val_);
              return (self_val > neg_three && self_val < three)
                  ? grad_val * one_sixth
                  : zero;
            });
      });
}

} // namespace

REGISTER_DISPATCH(hardsigmoid_stub, &hardsigmoid_kernel)
REGISTER_DISPATCH(hardsigmoid_backward_stub, &hardsigmoid_backward_kernel)

} // namespace at::native
