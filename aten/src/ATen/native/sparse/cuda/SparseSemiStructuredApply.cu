#include "hip/hip_runtime.h"
#include <ATen/ScalarOps.h>
#include <ATen/Tensor.h>
#include <ATen/Functions.h>
#include <ATen/Utils.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/accumulate.h>

#if defined(USE_ROCM) || defined(_MSC_VER)
#else
#include <ATen/native/sparse/cuda/SparseSemiStructuredPack.h>
#endif

namespace at::native {

#if defined(USE_ROCM) || defined(_MSC_VER)
#else
template <typename KT>
__global__ void __launch_bounds__(32 /* num_threads */)
  sparse_semi_structured_apply_kernel(typename KT::Params p)
{
  KT::sparse_semi_structured_apply_kernel(p);
}

// Apply a 2:4 sparsify pattern computed with
// `_sparse_semi_structured_tile` to another Tensor
template <bool kIsMeta, typename Element>
std::tuple<Tensor, Tensor> _sparse_semi_structured_apply_typed(Tensor input, Tensor threads_masks)
{
  using KT = KernelTypes<Element>;
  // TODO: Technically we should be able to deal with that
  // by running on the transpose of `input` and swapping
  // `packed` & `packed_t`.
  // This would require to adapt the `threads_masks` a bit tho.
  if (input.stride(1) != 1) {
    input = input.contiguous();
  }
  std::optional<at::cuda::CUDAGuard> device_guard;
  if (!kIsMeta) {
    device_guard.emplace(input.device());
  }

  TORCH_CHECK(input.dim() == 2);
  TORCH_CHECK(input.stride(1) == 1);
  TORCH_CHECK(input.stride(0) % 8 == 0);
  TORCH_CHECK(input.size(1) % 32 == 0, "Wrong alignment shape[1]");

  auto roundedx = cutlass::round_up(input.size(0), kWarpX);
  auto roundedy = cutlass::round_up(input.size(1), kWarpY);
  at::Tensor packed =
      at::empty({roundedx, cutlass::ceil_div(roundedy, 2)}, input.options());
  at::Tensor packed_trans =
      at::empty({roundedy, cutlass::ceil_div(roundedx, 2)}, input.options());

  typename KT::Params p;
  p.input = (Element const*)input.data_ptr();
  p.input_s0 = input.stride(0);
  p.input_dim0 = input.size(0);
  p.input_dim1 = input.size(1);

  p.packed = (Element*)packed.data_ptr();
  p.packed_stride = packed.stride(0);
  p.packed_trans = (Element*)packed_trans.data_ptr();
  p.packed_trans_stride = packed_trans.stride(0);

  p.threads_masks = (uint64_t*)threads_masks.data_ptr();

  TORCH_CHECK(threads_masks.dim() == 3);
  TORCH_CHECK(
      threads_masks.size(0) == p.getBlocksGrid().x * p.getThreadsGrid().x);
  TORCH_CHECK(
      threads_masks.size(1) == p.getBlocksGrid().y * p.getThreadsGrid().y);
  TORCH_CHECK(threads_masks.stride(1) == sizeof(p.threads_masks[0]));
  TORCH_CHECK(threads_masks.size(2) == sizeof(p.threads_masks[0]));
  TORCH_CHECK(threads_masks.stride(2) == 1);
  TORCH_CHECK(threads_masks.scalar_type() == at::ScalarType::Byte);

  if (!kIsMeta) {
    size_t smem_bytes = 0;
    sparse_semi_structured_apply_kernel<KT>
        <<<p.getBlocksGrid(),
           p.getThreadsGrid(),
           smem_bytes,
           at::cuda::getCurrentCUDAStream()>>>(p);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
  return std::make_tuple(packed, packed_trans);
}
#endif

std::tuple<Tensor, Tensor> _sparse_semi_structured_apply(const Tensor& input, const Tensor& threads_masks) // Returned by `_sparse_semi_structured_tile`
{
#if defined(USE_ROCM) || defined(_MSC_VER)
  TORCH_CHECK(false, "_sparse_semi_structured_apply: not supported");
  return std::make_tuple(Tensor{}, Tensor{});
#else
  TORCH_CHECK(
    input.scalar_type() == at::ScalarType::Half || input.scalar_type() == at::ScalarType::BFloat16,
    "Unsupported dtype - only `float16` and `bfloat16` are supported currently"
  );
  auto result = (input.scalar_type() == at::ScalarType::Half)
            ? _sparse_semi_structured_apply_typed<false, cutlass::half_t>(input, threads_masks)
            : _sparse_semi_structured_apply_typed<false, cutlass::bfloat16_t>(input, threads_masks);
  return result;
#endif
}

} // namespace
