#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/SparseCsrTensorImpl.h>
#include <ATen/SparseCsrTensorUtils.h>
#include <ATen/WrapDimUtilsMulti.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/Resize.h>
#include <ATen/native/SparseTensorUtils.h>
#include <algorithm>
#include <ATen/AccumulateType.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_convert_indices_from_coo_to_csr_native.h>
#include <ATen/ops/_convert_indices_from_csr_to_coo_native.h>
#include <ATen/ops/_sparse_csr_tensor_unsafe_native.h>
#include <ATen/ops/_unique.h>
#include <ATen/ops/add_native.h>
#include <ATen/ops/resize_as_sparse_native.h>
#include <ATen/ops/tensor.h>
#include <ATen/ops/zeros.h>
#endif

#include <hip/hip_runtime.h>
#include <type_traits>


#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAUtils.h>
#include <ATen/cuda/ThrustAllocator.h>
#include <c10/cuda/CUDACachingAllocator.h>

#include <ATen/native/cuda/Reduce.cuh>
#include <ATen/native/sparse/cuda/SparseBlasImpl.h>
#include <ATen/native/sparse/cuda/SparseCUDABlas.h>
#include <ATen/native/sparse/cuda/SparseCUDATensorMath.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>

namespace at::native {

namespace {

template <typename input_t, typename output_t>
__global__ void convert_indices_from_coo_to_csr_cuda_kernel(output_t* data_out, const input_t* data_in, const int64_t size, const int64_t numel) {
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid == 0) {
    for (int64_t i = 0; i <= data_in[0]; i++)
      data_out[i] = static_cast<output_t>(0);
  } else if (tid < numel) {
    for (int64_t i = data_in[tid - 1]; i < data_in[tid]; i++)
      data_out[i + 1] = static_cast<output_t>(tid);
  } else if (tid == numel) {
    for (int64_t i = data_in[numel - 1] + 1; i < size + 1; i++)
      data_out[i] = static_cast<output_t>(numel);
  }
}

template <typename input_t, typename output_t>
void convert_indices_from_coo_to_csr_cuda(const Tensor& result, const Tensor& input, const int64_t size) {
  int64_t numel = input.numel();
  const input_t* data_in = input.const_data_ptr<input_t>();
  output_t* data_out = result.mutable_data_ptr<output_t>();

  if (numel == 0) {
    result.zero_();
    return;
  }

  // Run (numel + 1) threads...
  int64_t THREADS = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t BLOCKS = (numel + THREADS) / THREADS;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  convert_indices_from_coo_to_csr_cuda_kernel<<<BLOCKS, THREADS, 0, stream>>>(data_out, data_in, size, numel);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename input_t, typename output_t>
__global__ void convert_indices_from_csr_to_coo_cuda_kernel(output_t* data_out, const input_t* data_in, const int64_t nrows, const int64_t nnz, const int64_t nbatches) {
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < nrows * nbatches) {
    int64_t b = tid / nrows;
    int64_t i_ = b * (nrows + 1) + tid % nrows;
    for (int64_t i = data_in[i_]; i < data_in[i_ + 1]; i++) {
      data_out[b * nnz + i] = static_cast<output_t>(tid % nrows);
    }
  }
}

template <typename input_t, typename output_t>
void convert_indices_from_csr_to_coo_cuda(const Tensor& indices, const Tensor& crow_indices, const Tensor& col_indices, const bool transpose=false) {
  int64_t nrows = crow_indices.size(-1) - 1;
  int64_t nnz = col_indices.size(-1);
  if (nrows == 0 || nnz == 0) {
    indices.zero_();
    return;
  }
  int64_t total_nnz = col_indices.numel();
  int64_t batch_ndim = crow_indices.dim() - 1;
  if (batch_ndim > 0) {
    auto batch_indices = indices.narrow(0, 0, batch_ndim);
    batch_indices.copy_(at::sparse::full_coo_indices(crow_indices.sizes().slice(0, batch_ndim), indices.options())
                        .repeat_interleave(nnz, 1));
  }

  auto crow_indices_ = crow_indices.expect_contiguous();
  const input_t* crow_indices_data_in = crow_indices_->const_data_ptr<input_t>();
  TORCH_INTERNAL_ASSERT(indices.is_contiguous());
  auto row0 = indices.select(0, transpose?batch_ndim + 1:batch_ndim + 0);
  auto row1 = indices.select(0, transpose?batch_ndim + 0:batch_ndim + 1);
  auto col_indices_ = col_indices.expect_contiguous();
  row1.copy_(col_indices_->view({-1}));
  output_t* data_out = row0.mutable_data_ptr<output_t>();

  // Run nrows * nbatches threads...
  int64_t nbatches = total_nnz / nnz;
  int64_t THREADS = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t BLOCKS = (nrows * nbatches + THREADS) / THREADS;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  convert_indices_from_csr_to_coo_cuda_kernel<<<BLOCKS, THREADS, 0, stream>>>(data_out, crow_indices_data_in, nrows, nnz, nbatches);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

} // namespace

using namespace at::sparse_csr;
// certain utility functions are usable from sparse COO.
using namespace at::sparse;

Tensor& add_out_dense_sparse_compressed_cuda(
    Tensor& output,
    const Tensor& dense,
    const SparseCsrTensor& src,
    const Scalar& alpha) {
  TORCH_INTERNAL_ASSERT(dense.layout() == kStrided);
  TORCH_INTERNAL_ASSERT(
      src.layout() == kSparseCsr || src.layout() == kSparseCsc);
  TORCH_INTERNAL_ASSERT(dense.is_cuda());

  TORCH_CHECK(
      output.is_contiguous(),
      "out argument must be contiguous, but got: ",
      output.suggest_memory_format());
  TORCH_CHECK(
      output.is_cuda(),
      "add: expected 'out' to be CUDA tensor, but got tensor on device: ",
      output.device());

  TORCH_CHECK(
      src.is_cuda(),
      "add: expected 'other' to be a CUDA tensor, but got tensor on device: ",
      src.device());

  TORCH_CHECK(
      dense.sizes().equals(src.sizes()),
      "add: expected 'self' and 'other' to have same size, but self has size ",
      dense.sizes(),
      " while other has size ",
      src.sizes(),
      " (FYI: dense-sparse addition does not currently support broadcasting)");

  auto commonDtype = promoteTypes(dense.scalar_type(), src.scalar_type());
  TORCH_CHECK(
      canCast(commonDtype, output.scalar_type()),
      "Can't convert result type ",
      commonDtype,
      " to output ",
      output.scalar_type(),
      " in add operation");

  Tensor src_values = src.values();

  resize_output(output, dense.sizes());

  Tensor resultBuffer = output;

  if (output.scalar_type() != commonDtype) {
    resultBuffer = dense.to(commonDtype);
  } else if (!is_same_tensor(output, dense)) {
    resultBuffer.copy_(dense);
  }

  if (src._nnz() == 0) {
    return output;
  }

  auto valuesBuffer = src_values.to(commonDtype).reshape({-1, src_values.size(-1)}).contiguous();
  resultBuffer = resultBuffer.view({-1, output.size(-2), output.size(-1)});
  Tensor src_compressed_indices;
  Tensor src_plain_indices;
  std::tie(src_compressed_indices, src_plain_indices) =
      at::sparse_csr::getCompressedPlainIndices(src);
  src_compressed_indices =
      src_compressed_indices.reshape({-1, src_compressed_indices.size(-1)});
  src_plain_indices =
      src_plain_indices.reshape({-1, src_plain_indices.size(-1)});
  auto src_layout = src.layout();

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND4(
      kComplexHalf,
      kHalf,
      kBool,
      kBFloat16,
      commonDtype,
      "add_out_op2_sparse_csr",
      [&valuesBuffer,
       &resultBuffer,
       &alpha,
       &src_compressed_indices,
       &src_plain_indices,
       &src_layout]() {
        AT_DISPATCH_INDEX_TYPES(
            src_compressed_indices.scalar_type(),
            "csr_add_out_crow_indices",
            [&valuesBuffer,
             &resultBuffer,
             &alpha,
             &src_compressed_indices,
             &src_plain_indices,
             &src_layout]() {
              auto batch_count =
                  resultBuffer.dim() > 2 ? resultBuffer.size(-3) : 1;
              scalar_t* values_accessor = valuesBuffer.mutable_data_ptr<scalar_t>();
              scalar_t* out_ptr = resultBuffer.mutable_data_ptr<scalar_t>();
              scalar_t cast_value = alpha.to<scalar_t>();

              index_t* compressed_indices_accessor =
                  src_compressed_indices.mutable_data_ptr<index_t>();
              index_t* plain_indices_accessor =
                  src_plain_indices.mutable_data_ptr<index_t>();
              int64_t out_storage_offset = resultBuffer.storage_offset();

              auto out_strides = resultBuffer.strides();
              auto const out_stride_batch = out_strides[0];
              auto const out_stride_compressed =
                  AT_DISPATCH_ROW_SPARSE_COMPRESSED_LAYOUTS(
                      src_layout,
                      "add_out_dense_sparse_compressed_cpu",
                      [&out_strides] { return out_strides[1]; },
                      [&out_strides] { return out_strides[2]; });
              auto const out_stride_plain =
                  AT_DISPATCH_ROW_SPARSE_COMPRESSED_LAYOUTS(
                      src_layout,
                      "add_out_dense_sparse_compressed_cpu",
                      [&out_strides] { return out_strides[2]; },
                      [&out_strides] { return out_strides[1]; });
              auto compressed_stride0 = src_compressed_indices.stride(0);
              auto plain_stride0 = src_plain_indices.stride(0);
              auto val_stride0 = valuesBuffer.stride(0);

              hipStream_t stream = at::cuda::getCurrentCUDAStream();
              at::cuda::ThrustAllocator allocator;
              auto policy = thrust::cuda::par(allocator).on(stream);

              // Note that this could be wildly imbalanced if the sparsity
              // pattern varies a lot between slices along the compressed
              // dimension.
              thrust::for_each(
                  policy,
                  thrust::make_counting_iterator(int64_t(0)),
                  thrust::make_counting_iterator(
                      int64_t(src_compressed_indices.size(-1) - 1)),
                  [values_accessor,
                   compressed_indices_accessor,
                   plain_indices_accessor,
                   out_ptr,
                   cast_value,
                   out_stride_batch,
                   out_stride_compressed,
                   out_stride_plain,
                   compressed_stride0,
                   plain_stride0,
                   val_stride0,
                   batch_count] __device__(int64_t i_compressed) {
                    for (index_t batch_idx = 0; batch_idx < batch_count;
                         batch_idx++) {
                      index_t start_index = compressed_indices_accessor
                          [batch_idx * compressed_stride0 + i_compressed];
                      index_t end_index = compressed_indices_accessor
                          [batch_idx * compressed_stride0 + i_compressed + 1];

                      for (index_t i = start_index; i < end_index; ++i) {
                        auto i_plain = plain_indices_accessor
                            [batch_idx * plain_stride0 + i];
                        auto index = batch_idx * out_stride_batch +
                            i_compressed * out_stride_compressed +
                            i_plain * out_stride_plain;
                        out_ptr[index] += cast_value *
                            values_accessor[batch_idx * val_stride0 + i];
                      }
                    }
                  });
            });
      });
  if (output.scalar_type() != commonDtype) {
    output.copy_(resultBuffer);
  }
  return output;
}

Tensor& add_out_sparse_compressed_cuda(
    const Tensor& self,
    const SparseCsrTensor& other,
    const Scalar& alpha,
    SparseCsrTensor& out) {
  if (self.layout() == kStrided) {
    add_out_dense_sparse_compressed_cuda(out, self, other, alpha);
  } else if (other.layout() == kStrided) {
    add_out_dense_sparse_compressed_cuda(out, other, self, alpha);
  } else {
    TORCH_CHECK(
        self.sizes().equals(other.sizes()),
        "torch.add: Expected input tensors to have the same shape, but got tensor `self` with shape ",
        self.sizes(),
        " and tensor `other` with shape ",
        other.sizes());
    TORCH_CHECK(
      self.is_cuda(),
      "add: expected 'self' to be CUDA tensor, but got tensor on device: ",
      self.device());
    TORCH_CHECK(
      other.is_cuda(),
      "add: expected 'other' to be CUDA tensor, but got tensor on device: ",
      other.device());
    TORCH_CHECK(
      out.is_cuda(),
      "add: expected 'out' to be CUDA tensor, but got tensor on device: ",
      out.device());

    if (only_sparse_compressed_add_trivial_cases(self, other, alpha, out)) {
      return out;
    }

    at::native::resize_as_sparse_compressed_(out, self);
    sparse::impl::cuda::add_out_sparse_csr(self, other, Scalar(1), alpha, out);
  }
  return out;
}

TORCH_IMPL_FUNC(_convert_indices_from_coo_to_csr_structured_cuda) (
  const Tensor& input, const int64_t size, const bool out_int32, const Tensor& result
) {
  if (out_int32) {
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "convert_indices_from_coo_to_csr_cuda", [&] {
      convert_indices_from_coo_to_csr_cuda<scalar_t, int>(result, input, size);
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "convert_indices_from_coo_to_csr_cuda", [&] {
      convert_indices_from_coo_to_csr_cuda<scalar_t, int64_t>(result, input, size);
    });
  }
}

TORCH_IMPL_FUNC(_convert_indices_from_csr_to_coo_structured_cuda) (
  const Tensor& crow_indices, const Tensor& col_indices, const bool out_int32, const bool transpose, const Tensor& result
) {
  if (out_int32) {
    AT_DISPATCH_INTEGRAL_TYPES(crow_indices.scalar_type(), "convert_indices_from_csr_to_coo_cuda", [&] {
      convert_indices_from_csr_to_coo_cuda<scalar_t, int32_t>(result, crow_indices, col_indices, transpose);
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(crow_indices.scalar_type(), "convert_indices_from_csr_to_coo_cuda", [&] {
      convert_indices_from_csr_to_coo_cuda<scalar_t, int64_t>(result, crow_indices, col_indices, transpose);
    });
  }
}

  /*
    Reductions on sparse CSR tensors using masked semantics.

    - To support a reduction operator on a CSR tensor with CUDA storage, define

template <typename scalar_t>
struct Reduction...Op {
  __device__ __forceinline__ scalar_t operator()(const scalar_t a, const scalar_t b) const {
    return a ... b;
  }
  __device__ __forceinline__ scalar_t identity() const { return ...; }
  __forceinline__ scalar_t identity_cpu() const { return ...; }
};


Tensor _sparse_csr_..._cuda(const Tensor& input, IntArrayRef dims_to_sum, bool keepdim, std::optional<ScalarType> dtype) {
  ...
      result = reduce_sparse_csr_cuda_template<scalar_t>(input_, dims_to_sum, keepdim, Reduction...Op<scalar_t>());
  ...
  return result;
}

      and add the following

        - func: _sparse_csr_op.dim_dtype(Tensor self, int[1] dim, bool keepdim=False, *, ScalarType? dtype=None) -> Tensor
          dispatch:
            SparseCsrCUDA: _sparse_csr_..._cuda

      to native_functions.yaml
  */

namespace {

template <typename scalar_t, typename index_t, typename ReductionOp, typename acc_t>
__global__ void reduce_sparse_csr_dim0_cuda_kernel(acc_t* new_values,
                                                   const index_t* new_col_indices,
                                                   const int64_t new_nnz,
                                                   const scalar_t* values,
                                                   const index_t* col_indices,
                                                   const int64_t nnz,
                                                   ReductionOp rop
                                                   ) {
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < new_nnz) {
    index_t col = new_col_indices[tid];
    acc_t v = rop.identity();
    for (int64_t j=0; j < nnz; j++) {
      if (col == col_indices[j]) {
        v = rop(v, acc_t(values[j]));
      }
    }
    new_values[tid] = v;
  }
}

template <typename scalar_t, typename ReductionOp>
Tensor reduce_sparse_csr_dim0_cuda_template(const Tensor& sparse, ReductionOp rop) {
  /*
    Consider the following sparse tensor:

      1 * * * *
      * * * 2 *
      * * 3 * *
      * * * * *
      4 * 5 * *

    that has CSR representation

      crow_indices = [0, 1, 2, 3, 3, 5]
      col_indices = [0, 3, 2, 0, 2]
      values = [1, 2, 3, 4, 5]

    Reduction with dim=0 results:

      rop(1,4) * rop(3,5) 2 *

    that has CSR representation

      new_crow_indices = [0, 3]
      new_col_indices = [0, 2, 3]
      new_values = [rop(1, 4], rop(3, 5), 2]

    In general, the CSR representation data can be computed as follows:

      nnz = col_indices.numel()
      new_col_indices = col_indices.unique(sorted=True, return_inverse=False)
      new_nnz = new_col_indices.numel()
      new_crow_indices = [0, new_nnz]
      new_values.resize(new_nnz)

      for i in range(new_nnz):
          v = identity
          col = new_col_indices[i]
          for j in range(nnz):
              if col == col_indices[j]:
                  v = rop(v, values[j])
          new_values[i] = v

    Notice this algorithm is different from the one used on CPU data.
  */

  Tensor col_indices = sparse.col_indices();
  Tensor values = sparse.values();
  auto ncols = sparse.size(1);
  auto nnz = col_indices.numel();

  auto new_col_indices = std::get<0>(at::_unique(col_indices, true, false));
  auto new_nnz = new_col_indices.numel();
  Tensor new_crow_indices = at::tensor(ArrayRef<int64_t>{0, new_nnz}, col_indices.options());

  // Set `is_cuda` = `true` in acc_type in CPU backend. Because the accumulate type
  // of float should be float in current scenario. In CUDA, float is the accumulate type
  // of float, while in CPU, double is the accumulate type of float.
  using acc_t = at::acc_type<scalar_t, true>;
  auto acc_buffer = at::sparse_csr::create_acc_buffer<acc_t, scalar_t>(
      values.options(), values.scalar_type(), new_nnz);
  Tensor new_values = std::get<0>(acc_buffer);
  Tensor new_values_acc = std::get<1>(acc_buffer);
  scalar_t* values_ptr = values.mutable_data_ptr<scalar_t>();
  acc_t* new_values_acc_ptr = new_values_acc.mutable_data_ptr<acc_t>();
  int64_t THREADS = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t BLOCKS = (new_nnz + THREADS) / THREADS;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_INDEX_TYPES(col_indices.scalar_type(), "reduce_sparse_csr_dim0_cuda_indices",
                          [&]() {
                            index_t* col_indices_ptr = col_indices.mutable_data_ptr<index_t>();
                            index_t* new_col_indices_ptr = new_col_indices.mutable_data_ptr<index_t>();
                            reduce_sparse_csr_dim0_cuda_kernel<<<BLOCKS, THREADS, 0, stream>>>(new_values_acc_ptr,
                                                                                               new_col_indices_ptr,
                                                                                               new_nnz,
                                                                                               values_ptr,
                                                                                               col_indices_ptr,
                                                                                               nnz,
                                                                                               rop
                                                                                               );
                          });
  copy_from_acc_buffer(new_values, new_values_acc);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return at::native::_sparse_csr_tensor_unsafe(new_crow_indices, new_col_indices, new_values,
                                               {1, ncols},
                                               new_values.scalar_type(),
                                               sparse.layout(),
                                               new_values.device());
}

template <typename index_t>
__global__ void reduce_crow_indices_dim1_cuda_kernel(index_t* new_crow_indices,
                                                     index_t* row_map,
                                                     const index_t* crow_indices,
                                                     const int64_t nrows
                                                     ) {
  int64_t nnz = 0;
  new_crow_indices[0] = 0;
  for(int64_t i=0; i<nrows; i++) {
    if (crow_indices[i] != crow_indices[i + 1]) {
      row_map[i] = nnz;
      nnz++;
    }
    new_crow_indices[i + 1] = nnz;
  }
}

template <typename scalar_t, typename index_t, typename ReductionOp, typename acc_t>
__global__ void reduce_sparse_csr_dim1_cuda_kernel(acc_t* new_values,
                                                   const scalar_t* values,
                                                   const index_t* crow_indices,
                                                   const index_t* row_map,
                                                   const int64_t nrows,
                                                   ReductionOp rop
                                                   ) {
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < nrows) {
    index_t i_start = crow_indices[tid];
    index_t i_end = crow_indices[tid+1];
    if (i_start != i_end) {
      acc_t acc = rop.identity();
      for (index_t i = i_start; i < i_end; i++) {
        acc = rop(acc, acc_t(values[i]));
      }
      new_values[row_map[tid]] = acc;
    }
  }
}

template <typename scalar_t, typename ReductionOp>
Tensor reduce_sparse_csr_dim1_cuda_template(const Tensor& sparse, ReductionOp rop) {
  /*
    The algorithm of computing reduce of a CSR tensor along the last
    dimension is explained in the comment of the
    reduce_sparse_csr_dim1_cpu_template function.
  */
  Tensor crow_indices = sparse.crow_indices();
  auto ioptions = crow_indices.options();
  Tensor values = sparse.values();
  auto nrows = sparse.size(0);
  auto numel = values.numel();

  Tensor new_crow_indices = at::empty({crow_indices.numel()}, ioptions);
  Tensor new_col_indices = at::empty({}, ioptions);
  Tensor row_map = at::empty({nrows}, ioptions);

  // Set `is_cuda` = `true` in acc_type in CPU backend. Because the accumulate type
  // of float should be float in current scenario. In CUDA, float is the accumulate type
  // of float, while in CPU, double is the accumulate type of float.
  using acc_t = at::acc_type<scalar_t, true>;
  auto acc_buffer = at::sparse_csr::create_acc_buffer<acc_t, scalar_t>(
      values.options(), values.scalar_type());
  Tensor new_values = std::get<0>(acc_buffer);
  Tensor new_values_acc = std::get<1>(acc_buffer);

  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  int64_t THREADS = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t BLOCKS = (nrows + THREADS) / THREADS;

  AT_DISPATCH_INDEX_TYPES(crow_indices.scalar_type(), "reduce_sparse_csr_dim1_cuda_indices",
                          [&]() {
                            index_t* crow_indices_ptr = crow_indices.mutable_data_ptr<index_t>();
                            index_t* new_crow_indices_ptr = new_crow_indices.mutable_data_ptr<index_t>();
                            index_t* row_map_ptr = row_map.mutable_data_ptr<index_t>();
                            reduce_crow_indices_dim1_cuda_kernel<<<1, 1, 0, stream>>>(new_crow_indices_ptr,
                                                                                      row_map_ptr,
                                                                                      crow_indices_ptr,
                                                                                      nrows);
                            C10_CUDA_KERNEL_LAUNCH_CHECK();
                            index_t new_nnz = new_crow_indices[-1].item<index_t>();
                            new_col_indices.resize_(new_nnz);
                            new_col_indices.fill_(index_t(0));
                            new_values.resize_(new_nnz);
                            new_values_acc.resize_(new_nnz);

                            scalar_t* values_ptr = values.mutable_data_ptr<scalar_t>();
                            acc_t* new_values_acc_ptr = new_values_acc.mutable_data_ptr<acc_t>();
                            reduce_sparse_csr_dim1_cuda_kernel<<<BLOCKS, THREADS, 0, stream>>>(new_values_acc_ptr,
                                                                                               values_ptr,
                                                                                               crow_indices_ptr,
                                                                                               row_map_ptr,
                                                                                               nrows,
                                                                                               rop);
                            C10_CUDA_KERNEL_LAUNCH_CHECK();
                          });

  copy_from_acc_buffer(new_values, new_values_acc);
  return at::native::_sparse_csr_tensor_unsafe(new_crow_indices, new_col_indices, new_values,
                                               {sparse.size(0), 1},
                                               new_values.scalar_type(),
                                               sparse.layout(),
                                               new_values.device());
}

template <typename scalar_t, typename ReductionOp>
Tensor reduce_sparse_csr_dim01_cuda_template(const Tensor& sparse, ReductionOp rop) {

  auto ioptions = sparse.col_indices().options();
  Tensor values = sparse.values();
  auto numel = values.numel();
  auto nnz = std::min<int64_t>(1, numel);

  auto result_dtype = at::isIntegralType(values.scalar_type(), /*includeBool=*/true) ? ScalarType::Long : values.scalar_type();
  Tensor new_values, new_values_acc;
  if (numel > 0) {
    new_values = at::empty({1}, values.options().dtype(result_dtype));
    new_values_acc = at::empty({1}, values.options());
    auto iter = TensorIterator::reduce_op(new_values_acc, values);
    gpu_reduce_kernel<scalar_t, scalar_t>(iter, func_wrapper<scalar_t>(rop), rop.identity_cpu());
    new_values.copy_(new_values_acc);
  } else {
    new_values = at::empty({}, values.options().dtype(result_dtype));
  }
  Tensor new_col_indices = at::zeros({nnz}, ioptions);
  Tensor new_crow_indices = at::tensor(ArrayRef<int64_t>{0, nnz}, ioptions);
  return at::native::_sparse_csr_tensor_unsafe(new_crow_indices, new_col_indices, new_values,
                                               {1, std::min<int64_t>(1, sparse.size(1))},
                                               new_values.scalar_type(),
                                               sparse.layout(),
                                               new_values.device());
}

template <typename scalar_t, typename ReductionOp>
Tensor reduce_sparse_csr_cuda_template(const Tensor& sparse, std::vector<int64_t> dims, ReductionOp rop) {
  if (dims.size() == 1) {
    if (dims[0] == 0) {
      return reduce_sparse_csr_dim0_cuda_template<scalar_t>(sparse, rop);
    } else {
      TORCH_INTERNAL_ASSERT(dims[0] == 1);
      return reduce_sparse_csr_dim1_cuda_template<scalar_t>(sparse, rop);
    }
  } else if (dims.size() == 2) {
    TORCH_INTERNAL_ASSERT(((dims[0] == 0 && dims[1] == 1) || (dims[0] == 1 && dims[1] == 0)));
    return reduce_sparse_csr_dim01_cuda_template<scalar_t>(sparse, rop);
  }
  TORCH_INTERNAL_ASSERT(dims.size() == 0);
  // effective after gh-29137 has been resolved
  return sparse.clone();
}

template <typename scalar_t, typename ReductionOp>
Tensor reduce_sparse_csr_cuda_template(const Tensor& sparse, IntArrayRef dims_to_sum, bool keepdim, ReductionOp rop) {
  TORCH_INTERNAL_ASSERT(sparse.is_sparse_csr());
  TORCH_CHECK(keepdim, "reduction operations on CSR tensors with keepdim=False is unsupported");
  TORCH_INTERNAL_ASSERT(sparse.is_cuda());

  const int64_t input_dim = sparse.dim();
  TORCH_INTERNAL_ASSERT(input_dim == 2);
  auto dims = dims_to_sum.vec();
  maybe_wrap_dims(dims, input_dim);
  if (dims.size() == 0) {
    // after gh-29137 is resolved, delete this if-block
    dims.emplace_back(0);
    dims.emplace_back(1);
  }
  return reduce_sparse_csr_cuda_template<scalar_t>(sparse, dims, rop);
}

template <typename scalar_t>
struct ReductionAddOp {
  __device__ __forceinline__ scalar_t operator()(const scalar_t a, const scalar_t b) const {
    return a + b;
  }
  __device__ __forceinline__ scalar_t identity() const { return 0; }
  __forceinline__ scalar_t identity_cpu() const { return 0; }
};

template <typename scalar_t>
struct ReductionMulOp {
  __device__ __forceinline__ scalar_t operator()(const scalar_t a, const scalar_t b) const {
    return a * b;
  }
  __device__ __forceinline__ scalar_t identity() const { return 1; }
  __forceinline__ scalar_t identity_cpu() const { return 1; }
};

void _apply_sparse_csr_linear_solve(
  const Tensor& A,
  const Tensor& b,
  const bool left,
  const Tensor& x) {
#if defined(USE_ROCM) || !defined(USE_CUDSS)
  TORCH_CHECK(
      false,
      "Calling linear solver with sparse tensors requires compiling ",
      "PyTorch with CUDA cuDSS and is not supported in ROCm build.");
#else
  // layout check
  TORCH_CHECK(A.is_sparse_csr(), "A must be a CSR matrix");
  TORCH_CHECK(b.layout() == kStrided, "b must be a strided tensor");
  TORCH_CHECK(x.layout() == kStrided, "x must be a strided tensor");
  // dim check
  TORCH_CHECK(b.dim() == 1, "b must be a 1D tensor");
  TORCH_CHECK(b.stride(0) == 1, "b must be a column major tensor");
  TORCH_CHECK(b.size(0) == A.size(0), "linear system size mismatch.");
  TORCH_CHECK(x.dim() == 1, "x must be a 1D tensor");
  TORCH_CHECK(x.stride(0) == 1, "x must be a column major tensor");
  TORCH_CHECK(x.size(0) == A.size(1), "linear system size mismatch.");
  TORCH_CHECK(A.dtype() == b.dtype() && A.dtype() == x.dtype(), "A, x, and b must have the same dtype");
  TORCH_CHECK(left == true, "only left == true is supported by the Sparse CSR backend")

  Tensor crow = A.crow_indices();
  Tensor col = A.col_indices();
  if (crow.scalar_type() != ScalarType::Int) {
    crow = crow.to(crow.options().dtype(ScalarType::Int));
    col = col.to(col.options().dtype(ScalarType::Int));
  }
  int* rowOffsets = crow.mutable_data_ptr<int>();
  int* colIndices = col.mutable_data_ptr<int>();
  Tensor values = A.values();
  // cuDSS data structures and handle initialization
  cudssConfig_t config;
  cudssMatrix_t b_mt;
  cudssMatrix_t A_mt;
  cudssMatrix_t x_mt;
  cudssData_t cudss_data;
  cudssHandle_t handle = at::cuda::getCurrentCudssHandle();

  TORCH_CUDSS_CHECK(cudssConfigCreate(&config));
  TORCH_CUDSS_CHECK(cudssDataCreate(handle, &cudss_data));

  AT_DISPATCH_FLOATING_TYPES(values.scalar_type(), "create_matrix", ([&] {
    scalar_t* values_ptr = values.mutable_data_ptr<scalar_t>();
    scalar_t* b_ptr = b.mutable_data_ptr<scalar_t>();
    scalar_t* x_ptr = x.mutable_data_ptr<scalar_t>();
    auto CUDA_R_TYP = std::is_same_v<scalar_t, double> ? HIP_R_64F : HIP_R_32F;
    TORCH_CUDSS_CHECK(cudssMatrixCreateDn(&b_mt, b.size(0), 1, b.size(0), b_ptr, CUDA_R_TYP, CUDSS_LAYOUT_COL_MAJOR));
    TORCH_CUDSS_CHECK(cudssMatrixCreateDn(&x_mt, x.size(0), 1, x.size(0), x_ptr, CUDA_R_TYP, CUDSS_LAYOUT_COL_MAJOR));
    TORCH_CUDSS_CHECK(cudssMatrixCreateCsr(&A_mt, A.size(0), A.size(1),  A._nnz(), rowOffsets, rowOffsets + crow.size(0), colIndices, values_ptr, HIP_R_32I, CUDA_R_TYP, CUDSS_MTYPE_GENERAL, CUDSS_MVIEW_FULL, CUDSS_BASE_ZERO));
  }));
  TORCH_CUDSS_CHECK(cudssExecute(handle, CUDSS_PHASE_ANALYSIS, config, cudss_data, A_mt, x_mt, b_mt));
  TORCH_CUDSS_CHECK(cudssExecute(handle, CUDSS_PHASE_FACTORIZATION, config, cudss_data, A_mt, x_mt, b_mt));
  TORCH_CUDSS_CHECK(cudssExecute(handle, CUDSS_PHASE_SOLVE, config, cudss_data, A_mt, x_mt, b_mt));
  // Destroy the opaque objects
  TORCH_CUDSS_CHECK(cudssConfigDestroy(config));
  TORCH_CUDSS_CHECK(cudssDataDestroy(handle, cudss_data));
  TORCH_CUDSS_CHECK(cudssMatrixDestroy(A_mt));
  TORCH_CUDSS_CHECK(cudssMatrixDestroy(x_mt));
  TORCH_CUDSS_CHECK(cudssMatrixDestroy(b_mt));
#endif
}
} // namespace

Tensor _sparse_csr_sum_cuda(const Tensor& input, IntArrayRef dims_to_sum, bool keepdim, std::optional<ScalarType> dtype) {
  ScalarType dtype_ = dtype.value_or(input.scalar_type());
  Tensor input_ = at::sparse_csr::to_type(input, dtype_);
  Tensor result;
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
      kHalf, kBFloat16, input_.scalar_type(), "_sparse_csr_sum_cuda", [&] {
      // Set `is_cuda` = `true` in acc_type in CPU backend. Because the accumulate type
      // of float should be float in current scenario. In CUDA, float is the accumulate type
      // of float, while in CPU, double is the accumulate type of float.
      using acc_t = at::acc_type<scalar_t, true>;
        result = reduce_sparse_csr_cuda_template<scalar_t>(
            input_, dims_to_sum, keepdim, ReductionAddOp<acc_t>());
      });
  return result;
}

Tensor _sparse_csr_prod_cuda(const Tensor& input, IntArrayRef dims_to_reduce, bool keepdim, std::optional<ScalarType> dtype) {
  ScalarType dtype_ = dtype.value_or(input.scalar_type());
  Tensor input_ = input.to(dtype_);
  Tensor result;
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
    kHalf, kBFloat16, input_.scalar_type(), "_sparse_csr_prod_cuda",
    [&] {
      result = reduce_sparse_csr_cuda_template<scalar_t>(input_, dims_to_reduce, keepdim, ReductionMulOp<scalar_t>());
    });
  return result;
}

Tensor _sparse_csr_linear_solve(const Tensor& A, const Tensor& b, const bool left) {
  Tensor b_copy = b.contiguous();
  Tensor out = b_copy.new_empty(b_copy.sizes());
  _apply_sparse_csr_linear_solve(A, b_copy, left, out);
  return out;
}


} // namespace at::native
