#include <ATen/ATen.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/CUDAUtils.h>
#include <ATen/Dispatch.h>

#if defined(USE_ROCM) || defined(_MSC_VER)
#else
#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/layout/layout.h>
#include <cutlass/tensor_ref.h>
#include <cutlass/gemm/device/gemm_sparse_with_visitor.h>
#include <cutlass/epilogue/threadblock/fusion/visitors.hpp>
#endif

#include <type_traits>
#include <tuple>

#if defined(USE_ROCM) || defined(_MSC_VER)
#else
#define CUTLASS_STATUS_CHECK(status)                                    \
  {                                                                     \
    TORCH_CHECK(status == cutlass::Status::kSuccess,                    \
                __func__, " : CUTLASS error: ",                         \
                cutlassGetStatusString(status));                        \
  }
#endif

namespace at::native {

#if defined(USE_ROCM) || defined(_MSC_VER)
#else
// Wrapper function for CUTLASS sparse GEMM implementation, used
// solely to simplify dispatching from
// sparse_semi_structured_mad_op() function below.
template <
    typename ElementInputA,
    typename ElementInputB,
    typename ElementOutput,
    typename ElementAccumulator,
    typename ThreadblockShape,
    typename WarpShape,
    typename InstructionShape,
    typename Operator,
    typename LayoutInputA,
    typename LayoutInputB,
    bool use_tensor_c>
void spgemm_cutlass(
    const Tensor& tensor_a, const at::IntArrayRef::value_type& tensor_a_stride,
    const Tensor& tensor_b, const at::IntArrayRef::value_type& tensor_b_stride,
    const Tensor& tensor_c, const Tensor& tensor_e, const Scalar& alpha,
    const Scalar& beta, Tensor& tensor_d) {
    // Fix CUTLASS sparse GEMM template arguments that are not
    // provided as template argument of this function, and create an
    // alias for particular instantiation of this template.
    using LayoutOutput = cutlass::layout::RowMajor; // Result of the operation will be provided in row-major format.
    using MMAOp = cutlass::arch::OpClassTensorOp; // Tensor cores are to be used for maximum performance.
    using SmArch = cutlass::arch::Sm80; // Only CC 8.x devices are supported at the moment.
    using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>; // This choice provides good performance across wide range of operand sizes.
    constexpr int NumStages = 3; // This choice provides good performance across wide range of operand sizes.
    constexpr int NumEVTEpilogueStages = 1;

    constexpr int AlignmentInputA = 128 / cutlass::sizeof_bits<ElementInputA>::value;
    constexpr int AlignmentInputB = 128 / cutlass::sizeof_bits<ElementInputB>::value;
    constexpr int AlignmentOutput = 128 / cutlass::sizeof_bits<ElementOutput>::value;

    using ElementComputeEpilogue = ElementAccumulator; // Typically slightly slower, but more precise than if ElementOutput used.
    constexpr int AlignmentComputeEpilogue = 128 / cutlass::sizeof_bits<ElementComputeEpilogue>::value;
    using ElementC = ElementOutput;
    using LayoutC = LayoutOutput;
    constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;

    using OutputTileThreadMap = cutlass::epilogue::threadblock::OutputTileThreadLayout<
        ThreadblockShape,
        WarpShape,
        ElementOutput,
        AlignmentOutput,
        NumEVTEpilogueStages>;

    using Accum = cutlass::epilogue::threadblock::VisitorAccFetch;

    using Alpha =
        cutlass::epilogue::threadblock::VisitorScalarBroadcast<ElementComputeEpilogue>;
    using AlphaArguments = typename Alpha::Arguments;

    using ApplyAlpha = cutlass::epilogue::threadblock::VisitorCompute<
        cutlass::multiplies, ElementComputeEpilogue, ElementComputeEpilogue,
        cutlass::FloatRoundStyle::round_to_nearest>;
    using EVTApplyAlpha = cutlass::epilogue::threadblock::Sm80EVT<
        ApplyAlpha,
        Alpha,
        Accum>;

    using Beta =
        cutlass::epilogue::threadblock::VisitorScalarBroadcast<ElementComputeEpilogue>;
    using BetaArguments = typename Beta::Arguments;

    using TensorCScalar =
        cutlass::epilogue::threadblock::VisitorScalarBroadcast<ElementC>;
    using TensorCTensor =
        cutlass::epilogue::threadblock::VisitorColBroadcast<
            OutputTileThreadMap,
            ElementC,
            cute::Stride<cute::_1, cute::_0, int64_t>>;
    using TensorC = std::conditional_t<use_tensor_c, TensorCTensor, TensorCScalar>;
    using TensorCArguments = typename TensorC::Arguments;

    using ApplyBeta = cutlass::epilogue::threadblock::VisitorCompute<
        cutlass::multiplies, ElementComputeEpilogue, ElementComputeEpilogue,
        cutlass::FloatRoundStyle::round_to_nearest>;
    using EVTApplyBeta = cutlass::epilogue::threadblock::Sm80EVT<
        ApplyBeta,
        Beta,
        TensorC>;

    using ApplySum = cutlass::epilogue::threadblock::VisitorCompute<
        cutlass::plus, ElementComputeEpilogue, ElementComputeEpilogue,
        cutlass::FloatRoundStyle::round_to_nearest>;
    using EVTApplySum = cutlass::epilogue::threadblock::Sm80EVT<
        ApplySum,
        EVTApplyAlpha,
        EVTApplyBeta>;

    using Output = cutlass::epilogue::threadblock::VisitorAuxStore<
        OutputTileThreadMap, ElementOutput, cutlass::FloatRoundStyle::round_to_nearest,
        cute::Stride<int64_t, cute::_1, int64_t>>;

    using EVTOutput = cutlass::epilogue::threadblock::Sm80EVT<
        Output,
        EVTApplySum>;

    using Gemm = cutlass::gemm::device::SparseGemmWithVisitor<
        ElementInputA,
        LayoutInputA,
        ElementInputB,
        LayoutInputB,
        ElementC,
        LayoutC,
        ElementAccumulator,
        MMAOp,
        SmArch,
        ThreadblockShape,
        WarpShape,
        InstructionShape,
        EVTOutput,
        SwizzleThreadBlock,
        NumStages,
        AlignmentInputA,
        AlignmentInputB,
        Operator,
        NumEVTEpilogueStages>;

    // Datatype and layout of metadata matrix are inferred from sparse
    // GEMM template.
    using ElementInputE = typename Gemm::ElementE;
    using LayoutInputE = cutlass::layout::RowMajor;
    using ReorderedLayoutInputE = typename Gemm::LayoutE;
    static_assert(
        std::is_same<ReorderedLayoutInputE,
                     cutlass::layout::ColumnMajorInterleaved<2>>::value,
        "Matrix layout used by CUTLASS for reordered metadata for sparse GEMM "
        "change, thus code doing conversions from/to dense matrix has to be "
        "updated.");

    constexpr auto kSparse = Gemm::kSparse;
    constexpr int kElementsPerElementE = Gemm::kElementsPerElementE;

    // Operand sizes.
    const int length_m = tensor_a.size(0);
    const int length_k = tensor_b.size(0);
    const int length_n = tensor_b.size(1);
    const auto tensor_e_ncols = length_k / kSparse / kElementsPerElementE;

    // Determine PyTorch datatype for the metadata matrix.
    auto tensor_e_dtype = at::kChar;
    switch (sizeof(ElementInputE)) {
    case 2:
        tensor_e_dtype = at::kShort;
        break;
    case 4:
        tensor_e_dtype = at::kInt;
        break;
    default:
        TORCH_CHECK(false, __func__, ": invalid size of meta tensor datatype "
                 "encountered");
    }
    TORCH_CHECK(tensor_e.dtype() == tensor_e_dtype,
                __func__, " : Expected meta datatype ", tensor_e_dtype,
                ", but got ", tensor_e.dtype());

    // Prepare arguments for CUTLASS sparse GEMM kernel.
    cutlass::gemm::GemmCoord problem_size(length_m, length_n, length_k);
    LayoutInputA layout_a(tensor_a_stride);
    LayoutInputB layout_b(tensor_b_stride);
    auto tensor_a_device_ref =
        cutlass::TensorRef<ElementInputA, LayoutInputA>(
            (ElementInputA*)tensor_a.data_ptr(), layout_a);
    auto tensor_b_device_ref =
        cutlass::TensorRef<ElementInputB, LayoutInputB>(
            (ElementInputB*)tensor_b.data_ptr(), layout_b);
    auto tensor_e_reordered_device_ref =
        cutlass::TensorRef<ElementInputE, ReorderedLayoutInputE>(
            (ElementInputE*)tensor_e.data_ptr(),
            ReorderedLayoutInputE::packed({length_m, tensor_e_ncols}));

    AlphaArguments alpha_arguments{
        [&]() -> AlphaArguments {
            if constexpr (std::is_same_v<ElementComputeEpilogue, cutlass::half_t> ||
                          std::is_same_v<ElementComputeEpilogue, cutlass::bfloat16_t>) {
                return {ElementComputeEpilogue{alpha.to<float>()}};
            } else {
                return {{alpha.to<ElementComputeEpilogue>()}};
            }
        }()
    };
    BetaArguments beta_arguments{
        [&]() -> BetaArguments {
            if constexpr (std::is_same_v<ElementComputeEpilogue, cutlass::half_t> ||
                          std::is_same_v<ElementComputeEpilogue, cutlass::bfloat16_t>) {
                return {ElementComputeEpilogue{beta.to<float>()}};
            } else {
                return {{beta.to<ElementComputeEpilogue>()}};
            }
        }()
    };
    TensorCArguments tensor_c_arguments{
        [&]() -> TensorCArguments {
            if constexpr (use_tensor_c) {
                return {(ElementC*)tensor_c.data_ptr(),
                        ElementC(0),
                        {cute::_1{}, cute::_0{}, problem_size.m()}};
            } else {
                return {{ElementC(0)}};
            }
        }()
    };
    typename Output::Arguments output_arguments{
        (ElementOutput*)tensor_d.data_ptr(),
        {problem_size.n(), cute::_1{}, problem_size.mn().product()}
    };
    typename EVTOutput::Arguments callback_arguments{
        {
            {
                alpha_arguments,     // Alpha
                {},                  // Accum
                {}                   // ApplyAlpha
            },                       // EVTApplyAlpha
            {
                beta_arguments,      // Beta
                tensor_c_arguments,  // TensorC
                {}                   // ApplyBeta
            },                       // EVTApplyBeta
            {}                       // ApplySum
        },                           // EVTApplySum
        output_arguments             // Output
    };                               // EVTOutput

    // Create a tuple of CUTLASS sparse GEMM kernel arguments.
    typename Gemm::Arguments arguments{
        problem_size,
        tensor_a_device_ref,
        tensor_b_device_ref,
        tensor_e_reordered_device_ref,
        callback_arguments};

    cutlass::Status status;

    // Create CUTLASS sparse GEMM kernel object.
    Gemm gemm_op;

    // Verify that sparse GEMM operation with given arguments can be
    // performed by CUTLASS.
    status = gemm_op.can_implement(arguments);
    CUTLASS_STATUS_CHECK(status);

    // Allocate workspace for CUTLASS sparse GEMM kernel.
    const auto workspace_size = Gemm::get_workspace_size(arguments);
    auto workspace = tensor_a.new_empty({(int64_t)workspace_size},
                                        at::TensorOptions().dtype(at::kByte));

    // Initialize CUTLASS sparse GEMM object.
    status = gemm_op.initialize(arguments, workspace.data_ptr(),
                                at::cuda::getCurrentCUDAStream());
    CUTLASS_STATUS_CHECK(status);

    // Perform sparse GEMM operation.
    status = gemm_op.run(at::cuda::getCurrentCUDAStream());
    CUTLASS_STATUS_CHECK(status);

    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

// Dispatch according to the input tensors layouts combination.
template <
    typename ElementInputA,
    typename ElementInputB,
    typename ElementOutput,
    typename ElementAccumulator,
    typename ThreadblockShape,
    typename WarpShape,
    typename InstructionShape,
    typename Operator,
    bool EnableRowMajorRowMajorLayouts,
    bool EnableRowMajorColumnMajorLayouts,
    bool EnableColumnMajorRowMajorLayouts,
    bool EnableColumnMajorColumnMajorLayouts,
    bool use_tensor_c>
void spgemm_cutlass_dispatch_layouts(
    const Tensor& tensor_a, const Tensor& tensor_b, const Tensor& tensor_c,
    const Tensor& tensor_e, const Scalar& alpha, const Scalar& beta,
    Tensor& tensor_d) {
    // Determine layouts (row-major or column-major) of input tensors.
    const auto strides_a = tensor_a.strides();
    auto tensor_a_row_major = strides_a[1] == 1;
    auto tensor_a_stride = tensor_a_row_major ? strides_a[0] : strides_a[1];
    const auto strides_b = tensor_b.strides();
    auto tensor_b_row_major = strides_b[1] == 1;
    auto tensor_b_stride = tensor_b_row_major ? strides_b[0] : strides_b[1];

    // Perform dispatching.
    if constexpr (EnableRowMajorRowMajorLayouts) {
        if (tensor_a_row_major && tensor_b_row_major) {
            spgemm_cutlass<
                ElementInputA,
                ElementInputB,
                ElementOutput,
                ElementAccumulator,
                ThreadblockShape,
                WarpShape,
                InstructionShape,
                Operator,
                cutlass::layout::RowMajor,
                cutlass::layout::RowMajor,
                use_tensor_c>(
                tensor_a,
                tensor_a_stride,
                tensor_b,
                tensor_b_stride,
                tensor_c,
                tensor_e,
                alpha,
                beta,
                tensor_d);
            return;
        }
    }
    if constexpr (EnableRowMajorColumnMajorLayouts) {
        if (tensor_a_row_major && !tensor_b_row_major) {
            spgemm_cutlass<
                ElementInputA,
                ElementInputB,
                ElementOutput,
                ElementAccumulator,
                ThreadblockShape,
                WarpShape,
                InstructionShape,
                Operator,
                cutlass::layout::RowMajor,
                cutlass::layout::ColumnMajor,
                use_tensor_c>(
                tensor_a,
                tensor_a_stride,
                tensor_b,
                tensor_b_stride,
                tensor_c,
                tensor_e,
                alpha,
                beta,
                tensor_d);
            return;
        }
    }
    if constexpr (EnableColumnMajorRowMajorLayouts) {
        if (!tensor_a_row_major && tensor_b_row_major) {
            spgemm_cutlass<
                ElementInputA,
                ElementInputB,
                ElementOutput,
                ElementAccumulator,
                ThreadblockShape,
                WarpShape,
                InstructionShape,
                Operator,
                cutlass::layout::ColumnMajor,
                cutlass::layout::RowMajor,
                use_tensor_c>(
                tensor_a,
                tensor_a_stride,
                tensor_b,
                tensor_b_stride,
                tensor_c,
                tensor_e,
                alpha,
                beta,
                tensor_d);
            return;
        }
    }
    if constexpr (EnableColumnMajorColumnMajorLayouts) {
        if (!tensor_a_row_major && !tensor_b_row_major) {
            spgemm_cutlass<
                ElementInputA,
                ElementInputB,
                ElementOutput,
                ElementAccumulator,
                ThreadblockShape,
                WarpShape,
                InstructionShape,
                Operator,
                cutlass::layout::ColumnMajor,
                cutlass::layout::ColumnMajor,
                use_tensor_c>(
                tensor_a,
                tensor_a_stride,
                tensor_b,
                tensor_b_stride,
                tensor_c,
                tensor_e,
                alpha,
                beta,
                tensor_d);
            return;
        }
    }

    TORCH_CHECK(false, __func__, "_dispatch_layouts: Combination of ",
             tensor_a_row_major ? "row-major" : "column_major", " and ",
             tensor_b_row_major ? "row-major" : "column_major",
             " layouts for input tensors is not supported");
}

// Dispatch according to the tensor_c tensor being provided or not.
template <
    typename ElementInputA,
    typename ElementInputB,
    typename ElementOutput,
    typename ElementAccumulator,
    typename ThreadblockShape,
    typename WarpShape,
    typename InstructionShape,
    typename Operator,
    bool EnableRowMajorRowMajorLayouts,
    bool EnableRowMajorColumnMajorLayouts,
    bool EnableColumnMajorRowMajorLayouts,
    bool EnableColumnMajorColumnMajorLayouts>
void spgemm_cutlass_dispatch_layouts_tensor_c(
    const Tensor& tensor_a, const Tensor& tensor_b, const Tensor& tensor_c,
    const Tensor& tensor_e, const Scalar& alpha, const Scalar& beta,
    Tensor& tensor_d) {
    if (tensor_c.numel() > 0) {
        spgemm_cutlass_dispatch_layouts<
            ElementInputA,
            ElementInputB,
            ElementOutput,
            ElementAccumulator,
            ThreadblockShape,
            WarpShape,
            InstructionShape,
            Operator,
            EnableRowMajorRowMajorLayouts,
            EnableRowMajorColumnMajorLayouts,
            EnableColumnMajorRowMajorLayouts,
            EnableColumnMajorColumnMajorLayouts,
            true>(
            tensor_a,
            tensor_b,
            tensor_c,
            tensor_e,
            alpha,
            beta,
            tensor_d);
    } else {
        spgemm_cutlass_dispatch_layouts<
            ElementInputA,
            ElementInputB,
            ElementOutput,
            ElementAccumulator,
            ThreadblockShape,
            WarpShape,
            InstructionShape,
            Operator,
            EnableRowMajorRowMajorLayouts,
            EnableRowMajorColumnMajorLayouts,
            EnableColumnMajorRowMajorLayouts,
            EnableColumnMajorColumnMajorLayouts,
            false>(
            tensor_a,
            tensor_b,
            tensor_c,
            tensor_e,
            alpha,
            beta,
            tensor_d);
    }
}
#endif

// Perform multiply-add operation, using corresponding CUTLASS
// sparse GEMM kernel, to given arguments:
//     result = alpha * mat1 @ mat2 + beta * input
// The "mat2" tensor is a dense tensor, while the "mat1" tensor is a
// sparse semi-structured matrix.  The "input" tensor is optional; if
// provided, it should be a vector, with the number of elements equal
// to the number of rows of "mat1" matrix.  It is assumed that "mat1"
// and "mat2" are 2D tensors, supplied either in row-major or
// column-major layouts (different layouts between these two tensors
// are OK, but not all combinations of formats are supported for some
// datatypes of these matrices).  The "mat1_meta" argument contains
// sparse semi-strucutred metadata.
//
// There exists numerous limitations of CUTLASS sparse GEMM kernel,
// with regards to sizes and alignments of input tensors, their
// layouts and datatypes, and so on; this is the reason for large
// number of checks throughout the code.
//
// TODO: The "input" tensor has to be a vector, such that it could be
// broadcasted to columns of mat1 * mat2.  The case of broadcasting to
// rows of mat1 * mat2 could be also supported, if "input" tensor is a
// vector of corresponding length; and same for the case when "input"
// tensor is a matrix of same size as mat1 * mat2 product.  If these
// updates made here, then remember to update corresponding bits in
// the Inductor code that are handling meta registrations and
// lowerings of aten._sparse_semi_structured_mm and
// aten._sparse_semi_structured_addmm operators.
Tensor sparse_semi_structured_mad_op(
      const Tensor& mat1, const Tensor& mat1_meta, const Tensor& mat2,
      const std::optional<Tensor>& input_opt, const Scalar& alpha,
      const Scalar& beta, const std::optional<c10::ScalarType> out_dtype_opt) {
#if defined(USE_ROCM) || defined(_MSC_VER)
    TORCH_CHECK(false, __func__, " : CUTLASS not supported");
    return Tensor{};
#else
    // No need to check that all tensors are on CUDA device, as this
    // is provided by dispatch.

    const auto& input = input_opt.value_or(Tensor{});
    const auto out_dtype = out_dtype_opt.value_or(mat2.scalar_type());

    // For now, only CC 8.x devices are supported.
    const auto dprops = at::cuda::getCurrentDeviceProperties();
    const auto is_sm8x = dprops->major == 8;
    TORCH_CHECK(is_sm8x,
                __func__, " : Supported only on GPUs with compute capability "
                "8.x");

    // Validate datatypes of input tensors.
    TORCH_CHECK(mat2.dtype() == at::kChar ||
                mat2.dtype() == at::kHalf ||
                mat2.dtype() == at::kBFloat16 ||
                mat2.dtype() == at::kFloat,
                __func__, " : The mat2 datatype ", mat2.dtype(),
                " is not supported");
    TORCH_CHECK(mat1.dtype() == mat2.dtype(),
                __func__, " : Expected mat1 datatype ", mat2.dtype(),
                ", but got ", mat1.dtype());
    if (input.numel() != 0) {
        TORCH_CHECK(input.dtype() == out_dtype,
                    __func__, " : Expected input datatype ", out_dtype,
                    ", but got ", input.dtype());
    }

    // Validate layouts of input tensors.
    TORCH_CHECK(mat1.layout() == Layout::Strided,
                __func__, " : Expected mat1 argument to be strided, but got "
                "layout ", mat1.layout());
    TORCH_CHECK(mat1.dim() == 2,
                __func__, " : Expected mat1 argument to be 2D tensor, got ",
                mat1.dim(), " dims");
    const auto strides_a = mat1.strides();
    TORCH_CHECK(strides_a[0] == 1 || strides_a[1] == 1,
                __func__, " : Invalid strides for mat1 argument: row stride = ",
                strides_a[0], ", column stride = ", strides_a[1]);
    TORCH_CHECK(mat2.layout() == Layout::Strided,
                __func__, " : Expected mat2 argument to be "
                "strided, but got layout ", mat2.layout());
    TORCH_CHECK(mat2.dim() == 2,
                __func__, " : Expected mat2 argument to be 2D tensor, got ",
                mat2.dim(), " dims");
    const auto strides_b = mat2.strides();
    TORCH_CHECK(strides_b[0] == 1 || strides_b[1] == 1,
                __func__, " : Invalid strides for mat2 argument: row stride = ",
                strides_b[0], ", column stride = ", strides_b[1]);
    if (input.numel() != 0) {
        TORCH_CHECK(input.layout() == Layout::Strided,
                    __func__, " : Expected input argument to be strided, but "
                    "got layout ", input.layout());
        TORCH_CHECK(input.dim() == 1,
                    __func__, " : Expected input argument to be 1D tensor, "
                    "got ", input.dim(), " dims");
    }

    // Validate sizes of input tensors.
    TORCH_CHECK(mat1.size(1) == mat2.size(0) / 2,
                __func__, " : Expected mat1 argument to have ",
                mat2.size(0) / 2, " columns, but got ", mat1.size(1));
    if (input.numel() != 0) {
        TORCH_CHECK(input.size(0) == mat1.size(0),
                    __func__, " : Expected input argument to have ",
                    mat1.size(0), " elements, but got ", input.size(0));
    }

    // Introduce alias names for arguments, according to the CUTLASS
    // naming conventions.
    const auto& tensor_a = mat1;
    const auto& tensor_b = mat2;
    const auto& tensor_c = input;
    const auto& tensor_e = mat1_meta;

    // Create output tensor.
    Tensor tensor_d =
        tensor_b.new_empty({tensor_a.size(0), tensor_b.size(1)},
                           at::TensorOptions().dtype(out_dtype));

    // Call wrapper function for CUTLASS sparse GEMM, dispatching on
    // the input datatype, and then on input tensors layouts.
    // According to the input tensors datatypes and layouts,
    // corresponding template arguments are supplied for instantiating
    // the wrapper function.  The tile sizes template arguments are
    // selected according to the CUTLASS profiler results, for number
    // of runs.
    AT_DISPATCH_SWITCH(
        tensor_a.scalar_type(),
        "sparse_semi_structured_mad_op",
        AT_DISPATCH_CASE(
            at::ScalarType::Char,
            [&]() {
                using ElementInputA = int8_t;
                using ElementInputB = int8_t;
                using ElementAccumulator = int32_t;
                using ThreadblockShape =
                    cutlass::gemm::GemmShape<128, 128, 128>;
                using WarpShape = cutlass::gemm::GemmShape<64, 64, 128>;
                using InstructionShape = cutlass::gemm::GemmShape<16, 8, 64>;
                using Operator = cutlass::arch::OpMultiplyAddSaturate;
                const auto EnableRowMajorRowMajorLayouts = false;
                const auto EnableRowMajorColumnMajorLayouts = true;
                const auto EnableColumnMajorRowMajorLayouts = false;
                const auto EnableColumnMajorColumnMajorLayouts = false;
                if (out_dtype == at::kInt) {
                  using ElementOutput = int32_t;
                  spgemm_cutlass_dispatch_layouts_tensor_c<
                      ElementInputA,
                      ElementInputB,
                      ElementOutput,
                      ElementAccumulator,
                      ThreadblockShape,
                      WarpShape,
                      InstructionShape,
                      Operator,
                      EnableRowMajorRowMajorLayouts,
                      EnableRowMajorColumnMajorLayouts,
                      EnableColumnMajorRowMajorLayouts,
                      EnableColumnMajorColumnMajorLayouts>(
                      tensor_a,
                      tensor_b,
                      tensor_c,
                      tensor_e,
                      alpha,
                      beta,
                      tensor_d);
                } else if (out_dtype == at::kChar) {
                  using ElementOutput = int8_t;
                  spgemm_cutlass_dispatch_layouts_tensor_c<
                      ElementInputA,
                      ElementInputB,
                      ElementOutput,
                      ElementAccumulator,
                      ThreadblockShape,
                      WarpShape,
                      InstructionShape,
                      Operator,
                      EnableRowMajorRowMajorLayouts,
                      EnableRowMajorColumnMajorLayouts,
                      EnableColumnMajorRowMajorLayouts,
                      EnableColumnMajorColumnMajorLayouts>(
                      tensor_a,
                      tensor_b,
                      tensor_c,
                      tensor_e,
                      alpha,
                      beta,
                      tensor_d);
                }
            })
        AT_DISPATCH_CASE(
            at::ScalarType::Half,
            [&]() {
                using ElementInputA = cutlass::half_t;
                using ElementInputB = cutlass::half_t;
                using ElementOutput = cutlass::half_t;
                using ElementAccumulator = float;
                using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 64>;
                using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;
                using InstructionShape = cutlass::gemm::GemmShape<16, 8, 32>;
                using Operator = cutlass::arch::OpMultiplyAdd;
                const auto EnableRowMajorRowMajorLayouts = true;
                const auto EnableRowMajorColumnMajorLayouts = true;
                const auto EnableColumnMajorRowMajorLayouts = true;
                const auto EnableColumnMajorColumnMajorLayouts = true;
                spgemm_cutlass_dispatch_layouts_tensor_c<
                    ElementInputA,
                    ElementInputB,
                    ElementOutput,
                    ElementAccumulator,
                    ThreadblockShape,
                    WarpShape,
                    InstructionShape,
                    Operator,
                    EnableRowMajorRowMajorLayouts,
                    EnableRowMajorColumnMajorLayouts,
                    EnableColumnMajorRowMajorLayouts,
                    EnableColumnMajorColumnMajorLayouts>(
                    tensor_a,
                    tensor_b,
                    tensor_c,
                    tensor_e,
                    alpha,
                    beta,
                    tensor_d);
            })
            AT_DISPATCH_CASE(
            at::ScalarType::BFloat16,
            [&]() {
                using ElementInputA = cutlass::bfloat16_t;
                using ElementInputB = cutlass::bfloat16_t;
                using ElementOutput = cutlass::bfloat16_t;
                using ElementAccumulator = float;
                using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 64>;
                using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;
                using InstructionShape = cutlass::gemm::GemmShape<16, 8, 32>;
                using Operator = cutlass::arch::OpMultiplyAdd;
                const auto EnableRowMajorRowMajorLayouts = true;
                const auto EnableRowMajorColumnMajorLayouts = true;
                const auto EnableColumnMajorRowMajorLayouts = true;
                const auto EnableColumnMajorColumnMajorLayouts = true;
                spgemm_cutlass_dispatch_layouts_tensor_c<
                    ElementInputA,
                    ElementInputB,
                    ElementOutput,
                    ElementAccumulator,
                    ThreadblockShape,
                    WarpShape,
                    InstructionShape,
                    Operator,
                    EnableRowMajorRowMajorLayouts,
                    EnableRowMajorColumnMajorLayouts,
                    EnableColumnMajorRowMajorLayouts,
                    EnableColumnMajorColumnMajorLayouts>(
                    tensor_a,
                    tensor_b,
                    tensor_c,
                    tensor_e,
                    alpha,
                    beta,
                    tensor_d);
            })
            AT_DISPATCH_CASE(
            at::ScalarType::Float,
            [&]() {
                using ElementInputA = float;
                using ElementInputB = float;
                using ElementOutput = float;
                using ElementAccumulator = float;
                using ThreadblockShape = cutlass::gemm::GemmShape<128, 64, 32>;
                using WarpShape = cutlass::gemm::GemmShape<64, 32, 32>;
                using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
                using Operator = cutlass::arch::OpMultiplyAdd;
                const auto EnableRowMajorRowMajorLayouts = true;
                const auto EnableRowMajorColumnMajorLayouts = true;
                const auto EnableColumnMajorRowMajorLayouts = true;
                const auto EnableColumnMajorColumnMajorLayouts = true;
                spgemm_cutlass_dispatch_layouts_tensor_c<
                    ElementInputA,
                    ElementInputB,
                    ElementOutput,
                    ElementAccumulator,
                    ThreadblockShape,
                    WarpShape,
                    InstructionShape,
                    Operator,
                    EnableRowMajorRowMajorLayouts,
                    EnableRowMajorColumnMajorLayouts,
                    EnableColumnMajorRowMajorLayouts,
                    EnableColumnMajorColumnMajorLayouts>(
                    tensor_a,
                    tensor_b,
                    tensor_c,
                    tensor_e,
                    alpha,
                    beta,
                    tensor_d);
            }));

    return tensor_d;
#endif
}

// Implementation of aten._sparse_semi_structured_mm operator.
Tensor _sparse_semi_structured_mm(
      const Tensor& mat1, const Tensor& mat1_meta, const Tensor& mat2,
      const std::optional<c10::ScalarType> out_dtype_opt) {
    return sparse_semi_structured_mad_op(mat1, mat1_meta, mat2,
                                         std::optional<Tensor>(), 1, 0,
                                         out_dtype_opt);
}

// Implementation of aten._sparse_semi_structured_addmm operator.
Tensor _sparse_semi_structured_addmm(
      const Tensor& input, const Tensor& mat1, const Tensor& mat1_meta,
      const Tensor& mat2, const Scalar& alpha, const Scalar& beta,
      const std::optional<c10::ScalarType> out_dtype_opt) {
    return sparse_semi_structured_mad_op(mat1, mat1_meta, mat2, input, alpha,
                                         beta, out_dtype_opt);
}

} // namespace at::native

// Following is just for testing purposes.
namespace at::native {

#if defined(USE_ROCM) || defined(_MSC_VER)
#else
// Copied from tools/util/include/host_reorder.h, from CUTLASS source
// tree.  This is for simplicity - namely, this file is not under
// include/cutlass in this tree, as other CUTLASS include files
// needed, so it would require changing PyTorch CMake configuration;
// furthermore, including this file produces build errors in PyTorch
// at the moment.
template <typename Element, typename LayoutDest, typename LayoutSrc>
static void reorder_meta(cutlass::TensorRef<Element, LayoutDest> dest,
                         cutlass::TensorRef<Element, LayoutSrc> src,
                         const int problem_size_m, const int problem_size_k) {
  for (int m = 0; m < problem_size_m; m++) {
    for (int k = 0; k < problem_size_k; k++) {
      // First reorder the rows.
      int group = (sizeof(Element) == 2) ? 32 : 16;
      int interweave = (sizeof(Element) == 2) ? 4 : 2;

      int dest_row = m / group * group + (m % 8) * interweave + (m % group) / 8;
      int dest_col = k;

      // Next swizzle the 2x2 blocks from Z to N.
      if (((dest_row % 2) == 0) && ((dest_col % 2) == 1)) {
        ++dest_row;
        --dest_col;
      } else if (((dest_row % 2) == 1) && ((dest_col % 2) == 0)) {
        --dest_row;
        ++dest_col;
      }

      dest.at({dest_row, dest_col}) = src.at({m, k});
    }
  }
}
#endif

std::tuple<Tensor, Tensor>
_to_sparse_semi_structured(const Tensor& dense) {
#if defined(USE_ROCM) || defined(_MSC_VER)
  TORCH_CHECK(false, __func__, " : CUTLASS not supported");
  return std::make_tuple(Tensor{}, Tensor{});
#else
  // Check dimensions of the dense matrix.
  TORCH_CHECK(dense.dim() == 2,
              __func__, " : Expected dense argument to be 2D tensor, got ",
              dense.dim(), " dims");

  // Determine PyTorch datatype for the metadata matrix.
  auto meta_dtype = at::kChar;
  auto ksparse = 0;
  auto dense_elems_per_meta_elem = 0;
  if (dense.dtype() == at::kChar) {
    meta_dtype = at::kInt;
    ksparse = 4;
    dense_elems_per_meta_elem = 32;
  } else if (dense.dtype() == at::kHalf || dense.dtype() == at::kBFloat16) {
    meta_dtype = at::kShort;
    ksparse = 4;
    dense_elems_per_meta_elem = 16;
  } else if (dense.dtype() == at::kFloat) {
    meta_dtype = at::kShort;
    ksparse = 2;
    dense_elems_per_meta_elem = 8;
  } else {
    TORCH_CHECK(false, "_to_sparse_semi_structured: Invalid dense argument datatype ",
             dense.dtype(), " encountered");
  }

  const auto dense_nrows = dense.size(0);
  const auto dense_ncols = dense.size(1);

  if (dense_nrows % (meta_dtype == at::kShort ? 32 : 16) != 0) {
    TORCH_CHECK(false, "_to_sparse_semi_structured: Number of rows of dense matrix must "
             "be divisible by ", (meta_dtype == at::kShort ? 32 : 16),
             ", but it is ", dense_nrows);
  }
  if (dense_ncols % dense_elems_per_meta_elem != 0) {
    TORCH_CHECK(false, "_to_sparse_semi_structured: Number of columns of dense matrix "
             "must be divisible by ", dense_elems_per_meta_elem, ", but it is ",
             dense_ncols);
  }

  const auto dense_cpu = dense.to("cpu");

  const auto mask_cpu = dense_cpu != at::zeros({1}, dense_cpu.options());

  const auto sparse_cpu =
    dense_cpu.masked_select(mask_cpu).view({dense_nrows, dense_ncols / 2});

  const auto meta_nrows = dense_nrows;
  const auto meta_ncols = dense_ncols / dense_elems_per_meta_elem;
  auto meta_cpu = dense_cpu.new_empty({meta_nrows, meta_ncols},
                                      at::TensorOptions().dtype(meta_dtype));

  auto* mask_cpu_ptr = mask_cpu.data_ptr<bool>();
  for (auto i = 0; i < meta_nrows; ++i) {
    for (auto j = 0; j < meta_ncols; ++j) {
      uint64_t meta_val = 0;
      for (auto k = 0; k < dense_elems_per_meta_elem / ksparse; ++k, mask_cpu_ptr += ksparse) {
        const auto mask_elems =
          (ksparse == 4) ? std::make_tuple(mask_cpu_ptr[0], mask_cpu_ptr[1],
                                           mask_cpu_ptr[2], mask_cpu_ptr[3])
                         : std::make_tuple(mask_cpu_ptr[0], mask_cpu_ptr[0],
                                           mask_cpu_ptr[1], mask_cpu_ptr[1]);
        auto meta_quadruple = 0;
        if (mask_elems == std::make_tuple(1, 1, 0, 0)) {
          meta_quadruple = 4; // 0100
        } else if (mask_elems == std::make_tuple(1, 0, 1, 0)) {
          meta_quadruple = 8; // 1000
        } else if (mask_elems == std::make_tuple(0, 1, 1, 0)) {
          meta_quadruple = 9; // 1001
        } else if (mask_elems == std::make_tuple(1, 0, 0, 1)) {
          meta_quadruple = 12; // 1100
        } else if (mask_elems == std::make_tuple(0, 1, 0, 1)) {
          meta_quadruple = 13; // 1101
        } else if (mask_elems == std::make_tuple(0, 0, 1, 1)) {
          meta_quadruple = 14; // 1110
        } else {
          TORCH_CHECK(false, "_to_sparse_semi_structured: dense argument does not match ",
                   (dense.dtype() != at::kFloat) ? "2:4" : "1:2",
                   "sparsity pattern");
        }
        meta_val = meta_val | (meta_quadruple << (4 * k));
      }
      const auto idx = i * meta_ncols + j;
      if (meta_dtype == at::kShort) {
        using MetaElement = int16_t;
        const auto meta_cpu_ptr = meta_cpu.data_ptr<MetaElement>();
        meta_cpu_ptr[idx] = (MetaElement)meta_val;
      } else if (meta_dtype == at::kInt) {
        using MetaElement = int32_t;
        const auto meta_cpu_ptr = meta_cpu.data_ptr<MetaElement>();
        meta_cpu_ptr[idx] = (MetaElement)meta_val;
      }
    }
  }

  auto meta_reordered_cpu = meta_cpu.new_empty({meta_nrows, meta_ncols});
  using MetaLayout = cutlass::layout::RowMajor;
  using MetaReorderedLayout = cutlass::layout::ColumnMajorInterleaved<2>;
  if (meta_dtype == at::kShort) {
    using MetaElement = int16_t;
    auto meta_cpu_ref =
      cutlass::TensorRef<MetaElement, MetaLayout>(
          meta_cpu.data_ptr<MetaElement>(),
          MetaLayout::packed({meta_nrows, meta_ncols}));
    auto meta_reordered_cpu_ref =
      cutlass::TensorRef<MetaElement, MetaReorderedLayout>(
          meta_reordered_cpu.data_ptr<MetaElement>(),
          MetaReorderedLayout::packed({meta_nrows, meta_ncols}));
    reorder_meta(meta_reordered_cpu_ref, meta_cpu_ref, meta_nrows, meta_ncols);
  } else if (meta_dtype == at::kInt) {
    using MetaElement = int32_t;
    auto meta_cpu_ref =
      cutlass::TensorRef<MetaElement, MetaLayout>(
          meta_cpu.data_ptr<MetaElement>(),
          MetaLayout::packed({meta_nrows, meta_ncols}));
    auto meta_reordered_cpu_ref =
      cutlass::TensorRef<MetaElement, MetaReorderedLayout>(
          meta_reordered_cpu.data_ptr<MetaElement>(),
          MetaReorderedLayout::packed({meta_nrows, meta_ncols}));
    reorder_meta(meta_reordered_cpu_ref, meta_cpu_ref, meta_nrows, meta_ncols);
  }

  return std::make_tuple(sparse_cpu.to(dense.device()),
                         meta_reordered_cpu.to(dense.device()));
#endif
}

}  // namespace at::native
