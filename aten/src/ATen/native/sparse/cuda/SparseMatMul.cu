#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Config.h>
#include <ATen/Dispatch.h>
#include <ATen/NamedTensorUtils.h>
#include <ATen/Parallel.h>
#include <ATen/SparseTensorImpl.h>
#include <ATen/native/Resize.h>
#include <ATen/native/SparseTensorUtils.h>
#include <hip/hip_runtime.h>
#include <type_traits>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_sparse_sparse_matmul_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like_native.h>
#endif

#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>
#include <ATen/cuda/CUDAUtils.h>
#include <ATen/cuda/ThrustAllocator.h>
#include <hipsparse.h>
#include <ATen/native/sparse/cuda/SparseCUDABlas.h>
#include <c10/cuda/CUDACachingAllocator.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/discard_iterator.h>


#if defined(__HIPCC__) && ((CUSPARSE_VERSION >= 11000) || (defined(USE_ROCM) && ROCM_VERSION >= 60300))
#define IS_CUSPARSE11_AVAILABLE() 1
#else
#define IS_CUSPARSE11_AVAILABLE() 0
#endif

#if IS_CUSPARSE11_AVAILABLE()
#include <hip/library_types.h>
#endif

namespace at::native {

namespace {

using namespace at::sparse;

Tensor _to_csr_int(const Tensor& rowIndices, int64_t dim, int64_t nnz) {
  Tensor csr = at::empty({dim + 1}, CUDA(kInt));
  Tensor rowIndicesInt = at::empty({rowIndices.size(0)}, CUDA(kInt));
  rowIndicesInt.copy_(rowIndices);
  sparse::cuda::Xcoo2csr(
      rowIndicesInt.data_ptr<int32_t>(), nnz, dim, csr.data_ptr<int32_t>());
  return csr;
}


#pragma push
// NVCC complains that confirm_mult_size is not used,
// but it is used in specializations of CusparseMatrixMultiplyOp below
#pragma nv_diag_suppress 177   // Function was declared but never referenced
int confirm_mult_size(const std::vector<int>& mat1_size, const std::vector<int>& mat2_size) {
  TORCH_CHECK(
      mat1_size[1] == mat2_size[0],
      "mat1 and mat2 shapes cannot be multiplied (",
      mat1_size[0],
      "x",
      mat1_size[1],
      " and ",
      mat2_size[0],
      "x",
      mat2_size[1],
      ")");
  return mat1_size[1];
}
#pragma pop

void create_general_description_(hipsparseMatDescr_t& description_) {
  TORCH_CUDASPARSE_CHECK(hipsparseCreateMatDescr(&description_));
  TORCH_CUDASPARSE_CHECK(hipsparseSetMatType(description_, HIPSPARSE_MATRIX_TYPE_GENERAL));
  TORCH_CUDASPARSE_CHECK(hipsparseSetMatIndexBase(description_, HIPSPARSE_INDEX_BASE_ZERO));
}

// csrMatrixRef is used to have a representation of a raw CSR matrix representation
// coming from `sparse_sparse_matmul_cuda_kernel` function.
// Moreover this implements a RAII guard for a cusparse descriptor
template<class scalar_t>
struct csrMatrixRef {
  int* csr_indices_{nullptr};
  int* csr_pointers_{nullptr};
  scalar_t* csr_values_{nullptr};
  int nnz_{0};
  std::vector<int> size_{};

  #if IS_CUSPARSE11_AVAILABLE()
    hipsparseSpMatDescr_t description_{0};
  #else
    hipsparseMatDescr_t description_{0};
  #endif

  csrMatrixRef() {
    #if !IS_CUSPARSE11_AVAILABLE()
      create_general_description_(description_);
    #endif
  }

  csrMatrixRef(
      int* csr_indices,
      int* csr_pointers,
      scalar_t* csr_values,
      int nnz,
      const std::vector<int>& size)
      : csr_indices_{csr_indices},
        csr_pointers_{csr_pointers},
        csr_values_{csr_values},
        nnz_{nnz},
        size_{size} {
    #if IS_CUSPARSE11_AVAILABLE()
      hipDataType cuda_data_type = at::cuda::getCudaDataType<scalar_t>();
      TORCH_CUDASPARSE_CHECK(hipsparseCreateCsr(
        &description_,
        this->size(0),
        this->size(1),
        this->nnz_,
        this->csr_pointers_,
        this->csr_indices_,
        this->csr_values_,
        HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO,
        cuda_data_type));
    #else
      create_general_description_(description_);
    #endif
  }

  ~csrMatrixRef() {
    #if IS_CUSPARSE11_AVAILABLE()
      hipsparseDestroySpMat(description_);
    #else
      hipsparseDestroyMatDescr(description_);
    #endif
  }

  int size(int index) const {
    return size_.at(index);
  }
};

// csrOutput is used to represent the output for `CusparseMatrixMultiplyOp`
// Note that `csrOutput` is different from `csrMatrixRef` and the purpose
// of this was to have a materialized  version of a CSR matrix.
// Moreover this implements a RAII guard for a cusparse descriptor
struct csrOutput {
  Tensor csr_indices_{};
  Tensor csr_pointers_{};
  at::Tensor csr_values_{};
  int nnz_{0};
  std::vector<int> size_;

  hipsparseMatDescr_t description_{0};

  csrOutput(const std::vector<int> &size) : size_{size} {
    create_general_description_(description_);
  }

  ~csrOutput() {
    hipsparseDestroyMatDescr(description_);
  }

  csrOutput(const csrOutput&) = delete;
  csrOutput& operator=(const csrOutput&) = delete;
  csrOutput(csrOutput&& rhs) {
    csr_indices_ = std::move(rhs.csr_indices_);
    csr_pointers_ = std::move(rhs.csr_pointers_);
    csr_values_ = std::move(rhs.csr_values_);
    nnz_ = rhs.nnz_;
    size_ = std::move(rhs.size_);
    description_ = rhs.description_;
    rhs.description_ = 0;
  }
  csrOutput& operator=(csrOutput&&) = delete;
  int size(int index) const {
    return size_.at(index);
  }
};

#if IS_CUSPARSE11_AVAILABLE()

// RAII guard helps to support cuSparse 11 API for `A @ B` operation
// This generic template exists because with cuSparse the `scalar_t` type could be a double or float
template <class scalar_t>
struct CusparseMatrixMultiplyOp {

  hipsparseSpGEMMDescr_t spgemmDesc;

  CusparseMatrixMultiplyOp() {
    static_assert(
      #if !defined(USE_ROCM)
          std::is_same_v<c10::Half, scalar_t> ||
          std::is_same_v<c10::BFloat16, scalar_t> ||
      #endif
          std::is_same_v<float, scalar_t> ||
          std::is_same_v<double, scalar_t> ||
          std::is_same_v<c10::complex<float>, scalar_t> ||
          std::is_same_v<c10::complex<double>, scalar_t>,
      #if !defined(USE_ROCM)
          "cusparseSpGEMM only supports data type of half, bfloat16, float, double and complex float, double."
      #else
          "cusparseSpGEMM only supports data type of float, double and complex float, double."
      #endif
      );
    // SpGEMM Computation
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_createDescr(&spgemmDesc));
  }

  ~CusparseMatrixMultiplyOp() {
    // destroy matrix/vector descriptors
    hipsparseSpGEMM_destroyDescr(spgemmDesc);
  }

  csrOutput operator ()(
      const csrMatrixRef<scalar_t>& A,
      const csrMatrixRef<scalar_t>& B,
      Tensor& output_values,
      Tensor& output_indices) {
    const int A_num_rows = A.size(0);

    const int B_num_cols = B.size(1);

    csrOutput out({A.size(0), B.size(1)});

    out.csr_pointers_ = at::empty({out.size(0) + 1}, output_indices.options().dtype(kInt));

    int* dC_csrOffsets = out.csr_pointers_.data_ptr<int>();
    int* dC_columns = nullptr;
    scalar_t* dC_values = nullptr;

    scalar_t alpha = 1.0f;
    scalar_t beta = 0.0f;
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    csrMatrixRef<scalar_t> C(
      dC_columns,
      dC_csrOffsets,
      dC_values,
      /*nnz*/0,
      {A_num_rows, B_num_cols}
    );

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t handle = at::cuda::getCurrentCUDASparseHandle();
    void *dBuffer1 = NULL, *dBuffer2 = NULL;
    size_t bufferSize1 = 0, bufferSize2 = 0;

    hipsparseSpMatDescr_t matA = A.description_;
    hipsparseSpMatDescr_t matB = B.description_;
    hipsparseSpMatDescr_t matC = C.description_;
    //--------------------------------------------------------------------------

    hipDataType computeType = at::cuda::getCudaDataType<scalar_t>();

    // If a specific GPU model does not provide native support for a given data type,
    // the routine returns HIPSPARSE_STATUS_ARCH_MISMATCH error
    #if defined(USE_ROCM)
    TORCH_CHECK(!(computeType == HIP_R_16F || computeType == HIP_R_16BF),
        "sparse_mm: Float16 and BFloat16 are not supported on ROCm");
    #else // defined(USE_ROCM)
    hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
    TORCH_CHECK(prop->major >= 5 && !((10*prop->major + prop->minor) < 53 && computeType == HIP_R_16F),
        "sparse_mm: CUDA Float16 requires compute capability >= 53 (current: ", prop->major, prop->minor, ")");
    TORCH_CHECK(!(prop->major < 8 && computeType == HIP_R_16BF),
        "sparse_mm: CUDA BFloat16 requires compute capability >= 80 (current: ", prop->major, prop->minor, ")");
    #endif // defined(USE_ROCM)

    // ask bufferSize1 bytes for external memory
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_workEstimation(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc,
        &bufferSize1,
        NULL));

    auto& allocator = *::c10::cuda::CUDACachingAllocator::get();

    at::DataPtr dataPtr1 = allocator.allocate(bufferSize1);
    dBuffer1 = dataPtr1.get();
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_workEstimation(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc,
        &bufferSize1,
        dBuffer1));

    // ask bufferSize2 bytes for external memory
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_compute(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc,
        &bufferSize2,
        NULL));

    at::DataPtr dataPtr2 = allocator.allocate(bufferSize2);
    dBuffer2 = dataPtr2.get();

    // compute the intermediate product of A * B
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_compute(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc,
        &bufferSize2,
        dBuffer2));
    // get matrix C non-zero entries C_num_nnz1
    int64_t C_num_rows1, C_num_cols1, C_num_nnz1;
    TORCH_CUDASPARSE_CHECK(
        hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1, &C_num_nnz1));
    // allocate matrix C
    // allocate C offsets
    out.nnz_ = C_num_nnz1;

    out.csr_indices_ = at::empty({out.nnz_}, output_indices.options().dtype(kInt));
    out.csr_values_ = at::empty({out.nnz_}, output_values.options());
    dC_columns = out.csr_indices_.data_ptr<int>();
    dC_values = out.csr_values_.data_ptr<scalar_t>();

    // update matC with the new pointers
    TORCH_CUDASPARSE_CHECK(
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values));

    // copy the final products to the matrix C
    TORCH_CUDASPARSE_CHECK(hipsparseSpGEMM_copy(
        handle,
        opA,
        opB,
        &alpha,
        matA,
        matB,
        &beta,
        matC,
        computeType,
        HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc));
    return out;
  }
};


template struct CusparseMatrixMultiplyOp<float>;

template struct CusparseMatrixMultiplyOp<double>;

#else // if not IS_CUSPARSE11_AVAILABLE()

using DcsrMatrixRef = csrMatrixRef<double>;
using ScsrMatrixRef = csrMatrixRef<float>;

// RAII guard helps to support cuSparse 10 API for `A @ B` operation
// This generic template exists because with cuSparse the `scalar_t` type could be a double or float
template <class scalar_t>
struct CusparseMatrixMultiplyOp {
  csrOutput operator()(
      const csrMatrixRef<scalar_t>& lhs,
      const csrMatrixRef<scalar_t>& rhs,
      Tensor &output_values,
      Tensor &output_indices)
  {
    static_assert(false&&sizeof(scalar_t), "cusparse csr sparse-sparse MM only supports data type of float and double.");
  }
};

// Specializacion for `A @ B` operation for double values with cuSparse
template<> struct CusparseMatrixMultiplyOp<double> {
  csrgemm2Info_t gemm2Info_;

  CusparseMatrixMultiplyOp() {
    TORCH_CUDASPARSE_CHECK(hipsparseCreateCsrgemm2Info(&gemm2Info_));
  }
  ~CusparseMatrixMultiplyOp() {
    hipsparseDestroyCsrgemm2Info(gemm2Info_);
  }

  csrOutput operator ()(
      const DcsrMatrixRef& lhs,
      const DcsrMatrixRef& rhs,
      Tensor &output_values,
      Tensor &output_indices) {
    double alpha = 1.0;
    DcsrMatrixRef empty;
    return Dgemm2(lhs, rhs, empty, &alpha, nullptr, output_values, output_indices);
  }

  csrOutput Dgemm2(
      const DcsrMatrixRef& A,
      const DcsrMatrixRef& B,
      const DcsrMatrixRef& C,
      const double* alpha,
      const double* beta,
      Tensor &output_values,
      Tensor &output_indices) {
    void* buffer_{nullptr};
    hipsparseHandle_t cusparseHandle_ = at::cuda::getCurrentCUDASparseHandle();
    TORCH_CUDASPARSE_CHECK(hipsparseSetPointerMode(cusparseHandle_, HIPSPARSE_POINTER_MODE_HOST));

    csrOutput out({A.size(0), B.size(1)});
    int innerSize = confirm_mult_size(A.size_, B.size_);
    out.csr_pointers_ = at::empty({out.size(0) + 1}, output_indices.options().dtype(kInt));

    // Compute needed buffer size
    size_t new_bubber_sz;
    TORCH_CUDASPARSE_CHECK(hipsparseDcsrgemm2_bufferSizeExt(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        alpha,
        A.description_,
        A.nnz_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_pointers_,
        B.csr_indices_,
        beta,
        C.description_,
        C.nnz_,
        C.csr_pointers_,
        C.csr_indices_,
        gemm2Info_,
        &new_bubber_sz));

    // (Re)allocate buffer if needed
    auto& allocator = *::c10::cuda::CUDACachingAllocator::get();
    at::DataPtr data_ptr = allocator.allocate(new_bubber_sz);
    buffer_ = data_ptr.get();

    // Find the resulting non-zero pattern.
    TORCH_CUDASPARSE_CHECK(hipsparseXcsrgemm2Nnz(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        A.description_,
        A.nnz_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_pointers_,
        B.csr_indices_,
        C.description_,
        C.nnz_,
        C.csr_pointers_,
        C.csr_indices_,
        out.description_,
        out.csr_pointers_.data_ptr<int>(),
        &out.nnz_,
        gemm2Info_,
        buffer_));

    out.csr_indices_ = at::empty({out.nnz_}, output_indices.options().dtype(kInt));
    out.csr_values_ = at::empty({out.nnz_}, output_values.options());

    // Perform the gemm2 operation for doubles
    // out = alpha ∗ A ∗ B + beta ∗ C
    TORCH_CUDASPARSE_CHECK(hipsparseDcsrgemm2(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        alpha,
        A.description_,
        A.nnz_,
        A.csr_values_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_values_,
        B.csr_pointers_,
        B.csr_indices_,
        beta,
        C.description_,
        C.nnz_,
        C.csr_values_,
        C.csr_pointers_,
        C.csr_indices_,
        out.description_,
        out.csr_values_.data_ptr<double>(),
        out.csr_pointers_.data_ptr<int>(),
        out.csr_indices_.data_ptr<int>(),
        gemm2Info_,
        buffer_));
    return out;
  }
};

// Specializacion for `A @ B` operation for float values with cuSparse
template<> struct CusparseMatrixMultiplyOp<float> {
  csrgemm2Info_t gemm2Info_;

  CusparseMatrixMultiplyOp() {
    TORCH_CUDASPARSE_CHECK(hipsparseCreateCsrgemm2Info(&gemm2Info_));

  }
  ~CusparseMatrixMultiplyOp() {
    hipsparseDestroyCsrgemm2Info(gemm2Info_);
  }
  csrOutput operator()(
      const ScsrMatrixRef& lhs,
      const ScsrMatrixRef& rhs,
      Tensor &output_values,
      Tensor &output_indices) {
    float alpha = 1.0;
    ScsrMatrixRef empty;
    return Sgemm2(lhs, rhs, empty, &alpha, nullptr, output_values, output_indices);
  }

  csrOutput Sgemm2(
      const ScsrMatrixRef& A,
      const ScsrMatrixRef& B,
      const ScsrMatrixRef& C,
      const float* alpha,
      const float* beta,
      Tensor &output_values,
      Tensor &output_indices) {
    void* buffer_{nullptr};
    hipsparseHandle_t cusparseHandle_ = at::cuda::getCurrentCUDASparseHandle();
    TORCH_CUDASPARSE_CHECK(hipsparseSetPointerMode(cusparseHandle_, HIPSPARSE_POINTER_MODE_HOST));

    csrOutput out({A.size(0), B.size(1)});

    int innerSize = confirm_mult_size(A.size_, B.size_);

    out.csr_pointers_ = at::empty({out.size(0) + 1}, output_indices.options().dtype(kInt));

    // Compute needed buffer size
    size_t new_bubber_sz;
    TORCH_CUDASPARSE_CHECK(hipsparseScsrgemm2_bufferSizeExt(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        alpha,
        A.description_,
        A.nnz_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_pointers_,
        B.csr_indices_,
        beta,
        C.description_,
        C.nnz_,
        C.csr_pointers_,
        C.csr_indices_,
        gemm2Info_,
        &new_bubber_sz));

    auto& allocator = *::c10::cuda::CUDACachingAllocator::get();
    at::DataPtr data_ptr = allocator.allocate(new_bubber_sz);
    buffer_ = data_ptr.get();

    // Find the resulting non-zero pattern.
    TORCH_CUDASPARSE_CHECK(hipsparseXcsrgemm2Nnz(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        A.description_,
        A.nnz_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_pointers_,
        B.csr_indices_,
        C.description_,
        C.nnz_,
        C.csr_pointers_,
        C.csr_indices_,
        out.description_,
        out.csr_pointers_.data_ptr<int>(),
        &out.nnz_,
        gemm2Info_,
        buffer_));

    out.csr_indices_ = at::empty({out.nnz_}, output_indices.options().dtype(kInt));
    out.csr_values_ = at::empty({out.nnz_}, output_values.options());

    // Perform the gemm2 operation for doubles
    // out = alpha ∗ A ∗ B + beta ∗ C
    TORCH_CUDASPARSE_CHECK(hipsparseScsrgemm2(
        cusparseHandle_,
        out.size(0),
        out.size(1),
        innerSize,
        alpha,
        A.description_,
        A.nnz_,
        A.csr_values_,
        A.csr_pointers_,
        A.csr_indices_,
        B.description_,
        B.nnz_,
        B.csr_values_,
        B.csr_pointers_,
        B.csr_indices_,
        beta,
        C.description_,
        C.nnz_,
        C.csr_values_,
        C.csr_pointers_,
        C.csr_indices_,
        out.description_,
        out.csr_values_.data_ptr<float>(),
        out.csr_pointers_.data_ptr<int>(),
        out.csr_indices_.data_ptr<int>(),
        gemm2Info_,
        buffer_));
    return out;
  }
};



#endif // IS_CUSPARSE11_AVAILABLE()

template <typename scalar_t>
void sparse_sparse_matmul_cuda_kernel(
    Tensor& result,
    const Tensor& mat1,
    const Tensor& mat2) {

  static_assert(
    std::is_same_v<c10::Half, scalar_t> ||
        std::is_same_v<c10::BFloat16, scalar_t> ||
        std::is_same_v<float, scalar_t> ||
        std::is_same_v<double, scalar_t> ||
        std::is_same_v<c10::complex<float>, scalar_t> ||
        std::is_same_v<c10::complex<double>, scalar_t>,
    "sparse_sparse_matmul_cuda_kernel only supports data type of half, bfloat16, float, double and complex float, double.");

  // older versions of cusparse on Windows segfault for complex128 dtype
#if defined(_WIN32) && defined(CUSPARSE_VERSION) && CUSPARSE_VERSION < 11400
  TORCH_CHECK(
      !(mat1.scalar_type() == ScalarType::ComplexDouble),
      "Sparse multiplication with complex128 dtype inputs is not supported with current CUDA version. Please upgrade to CUDA Toolkit 11.2.1+");
#endif

  Tensor mat1_indices_ = mat1._indices().contiguous();
  Tensor mat1_values = mat1._values().contiguous();

  Tensor mat1_row_indices = mat1_indices_.select(0, 0);
  Tensor mat1_col_indices = mat1_indices_.select(0, 1);

  Tensor mat1_indptr = _to_csr_int(mat1_row_indices, mat1.size(0), mat1._nnz());

  Tensor mat1_indices = at::empty(
      {mat1_col_indices.size(0)}, mat1_col_indices.options().dtype(kInt));

  mat1_indices.copy_(mat1_col_indices);

  Tensor mat2_indices_ = mat2._indices().contiguous();
  Tensor mat2_values = mat2._values().contiguous();
  Tensor mat2_row_indices = mat2_indices_.select(0, 0);
  Tensor mat2_col_indices = mat2_indices_.select(0, 1);

  Tensor mat2_indptr = _to_csr_int(mat2_row_indices, mat2.size(0), mat2._nnz());
  Tensor mat2_indices = at::empty({mat2_col_indices.size(0)}, mat2_col_indices.options().dtype(kInt));
  mat2_indices.copy_(mat2_col_indices);

  auto m = mat1.size(0);
  auto k1 = mat1.size(1);

  auto k2 = mat2.size(0);
  auto n = mat2.size(1);
  TORCH_CHECK((m <= INT_MAX) && (n <= INT_MAX) && (k1 <= INT_MAX),
    "At the moment, hipsparseDcsrgemm2 only supports m, n, k, nnz with the bound [val] <= ", INT_MAX, ".",
    "If you need this, please file an issue on GitHub."
  );
  auto output_indices = result._indices();
  auto output_values = result._values();

  if ((k1 == 0 && k2 == 0) || (n == 0 && m == 0)) {
    output_indices.zero_();
    output_values.zero_();
    return;
  }

  csrMatrixRef<scalar_t> csr_mat1(
      mat1_indices.data_ptr<int>(),
      mat1_indptr.data_ptr<int>(),
      mat1_values.data_ptr<scalar_t>(),
      (int)mat1._nnz(),
      {(int)mat1.size(0), (int)mat1.size(1)});

  csrMatrixRef<scalar_t> csr_mat2(
      mat2_indices.data_ptr<int>(),
      mat2_indptr.data_ptr<int>(),
      mat2_values.data_ptr<scalar_t>(),
      (int)mat2._nnz(),
      {(int)mat2.size(0), (int)mat2.size(1)});

  // Sparse matrix multiplication
  CusparseMatrixMultiplyOp<scalar_t> op;
  csrOutput csr_output = op(csr_mat1, csr_mat2, output_values, output_indices);
  auto nnz = csr_output.nnz_;

  output_values.set_(csr_output.csr_values_);
  output_indices.resize_({2, nnz});
  auto output_indices_accessor = output_indices.packed_accessor64<int64_t, 2>();

  auto csr_output_pointers_accessor =
      csr_output.csr_pointers_.packed_accessor64<int, 1>();

  auto csr_output_ind_accessor =
      csr_output.csr_indices_.packed_accessor64<int, 1>();

  auto major_dim = result.size(0);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::cuda::ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);

  // Filling the COO row indices
  thrust::for_each(
      policy,
      thrust::make_counting_iterator(int64_t(0)),
      thrust::make_counting_iterator(int64_t(major_dim)),
      [output_indices_accessor,
       csr_output_pointers_accessor,
       major_dim,
       nnz] __device__(int64_t i) {
        auto Ap = csr_output_pointers_accessor.data();
        int64_t* indices_row = output_indices_accessor[0].data();

        for (int jj = Ap[i];  jj < Ap[i + 1]; jj++) {
          indices_row[jj] = i;
        }
      });

  // Filling the COO column indices
  thrust::for_each(
    policy,
    thrust::make_counting_iterator(int64_t(0)),
    thrust::make_counting_iterator(int64_t(csr_output.nnz_)),
    [output_indices_accessor,
      csr_output_pointers_accessor,
      csr_output_ind_accessor,
      major_dim,
      nnz] __device__(int64_t i) {
      int64_t* indices_col = output_indices_accessor[1].data();
      indices_col[i] = csr_output_ind_accessor[i];
    });
}

} // end anonymous namespace

Tensor sparse_sparse_matmul_cuda(const Tensor& mat1_, const Tensor& mat2_) {
  TORCH_INTERNAL_ASSERT(mat1_.is_sparse());
  TORCH_INTERNAL_ASSERT(mat2_.is_sparse());
  TORCH_CHECK(mat1_.dim() == 2);
  TORCH_CHECK(mat2_.dim() == 2);
  TORCH_CHECK(mat1_.dense_dim() == 0, "sparse_mm: scalar values expected, mat1 got ", mat1_.dense_dim(), "D values");
  TORCH_CHECK(mat2_.dense_dim() == 0, "sparse_mm: scalar values expected, mat2 got ", mat2_.dense_dim(), "D values");

  TORCH_CHECK(
      mat1_.size(1) == mat2_.size(0), "mat1 and mat2 shapes cannot be multiplied (",
      mat1_.size(0), "x", mat1_.size(1), " and ", mat2_.size(0), "x", mat2_.size(1), ")");

  TORCH_CHECK(mat1_.scalar_type() == mat2_.scalar_type(),
           "mat1 dtype ", mat1_.scalar_type(), " does not match mat2 dtype ", mat2_.scalar_type());

  auto output = at::native::empty_like(mat1_);
  output.sparse_resize_and_clear_({mat1_.size(0), mat2_.size(1)}, mat1_.sparse_dim(), 0);

#if IS_CUSPARSE11_AVAILABLE() && !defined(USE_ROCM)
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(kHalf, kBFloat16, mat1_.scalar_type(), "sparse_matmul", [&] {
      sparse_sparse_matmul_cuda_kernel<scalar_t>(output, mat1_.coalesce(), mat2_.coalesce());
  });
#elif IS_CUSPARSE11_AVAILABLE() && defined(USE_ROCM)
  // ROCm does not support half and bfloat16 types for sparse_matmul
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(mat1_.scalar_type(), "sparse_matmul", [&] {
      sparse_sparse_matmul_cuda_kernel<scalar_t>(output, mat1_.coalesce(), mat2_.coalesce());
  });
#else
  AT_DISPATCH_FLOATING_TYPES(mat1_.scalar_type(), "sparse_matmul", [&] {
    sparse_sparse_matmul_cuda_kernel<scalar_t>(output, mat1_.coalesce(), mat2_.coalesce());
  });
#endif
  return output;
}

} // namespace at::native
