#include "hip/hip_runtime.h"
#include <ATen/ScalarOps.h>
#include <ATen/Tensor.h>
#include <ATen/Functions.h>
#include <ATen/autocast_mode.h>
#include <c10/cuda/CUDAGuard.h>

#if defined(USE_ROCM) || defined(_MSC_VER)
#else
#include <ATen/native/sparse/cuda/ComputeSparseTile.h>
#include <ATen/native/sparse/cuda/SparseSemiStructuredPack.h>
#endif

namespace at::native {

#if defined(USE_ROCM) || defined(_MSC_VER)
#else
struct Params {
  uint64_t const* threads_masks;

  uint16_t const* input;
  int64_t input_stride;
  int64_t input_dim0;
  int64_t input_dim1;

  uint16_t* output;
  int64_t output_stride;

  __host__ dim3 getBlocksGrid() const {
    return dim3(
        cutlass::ceil_div(input_dim0, kWarpX),
        cutlass::ceil_div(input_dim1, kWarpY),
        1);
  }

  static CUTLASS_HOST_DEVICE dim3 getThreadsGrid() {
    return dim3(kWarpX / kThreadX, kWarpY / kThreadY, 1);
  }

  CUTLASS_DEVICE Tile8x8Masks* getCurrentThreadIndices() const {
    Tile8x8Masks* gmem_threads_masks = (Tile8x8Masks*)threads_masks;
    gmem_threads_masks += blockIdx.y * getThreadsGrid().y + threadIdx.y;
    int64_t strideX = gridDim.y * getThreadsGrid().y;
    gmem_threads_masks +=
        (blockIdx.x * getThreadsGrid().x + threadIdx.x) * strideX;
    return gmem_threads_masks;
  }
};

template <bool kInputRowMajor = true, bool kOutputRowMajor = true>
__global__ void __launch_bounds__(32 /* num_threads */, 32) sparse_semi_structured_apply_dense_k(Params p) {
  using Fragment = cutlass::Array<uint16_t, 8>;

  // Top-left of the 8x8 tile we own
  int warp_x = blockIdx.x * kWarpX;
  int warp_y = blockIdx.y * kWarpY;
  int x = warp_x + threadIdx.x * kThreadX;
  int y = warp_y + threadIdx.y * kThreadY;

  uint16_t* output = p.output + x * p.output_stride + y;
  Tile8x8Masks indices = *p.getCurrentThreadIndices();

  // Load dense
  Fragment lines[8];
  if (kInputRowMajor) {
    uint16_t const* input = p.input + x * p.input_stride + y;
    CUTLASS_PRAGMA_UNROLL
    for (int i = 0; i < 8; ++i) {
      cutlass::arch::global_load<Fragment, sizeof(Fragment)>(
          lines[i], input + i * p.input_stride, true);
    }
  } else {
    uint16_t const* input = p.input + x + y * p.input_stride;
    Fragment columns[8];
    CUTLASS_PRAGMA_UNROLL
    for (int i = 0; i < 8; ++i) {
      cutlass::arch::global_load<Fragment, sizeof(Fragment)>(
          columns[i], input + i * p.input_stride, true);
    }
    CUTLASS_PRAGMA_UNROLL
    for (int i = 0; i < 8; ++i) {
      CUTLASS_PRAGMA_UNROLL
      for (int j = 0; j < 8; ++j) {
        lines[i][j] = columns[j][i].get();
      }
    }
  }

  CUTLASS_PRAGMA_UNROLL
  for (int row = 0; row < 2; ++row) {
    Indices4x4 masks[2];
    if (row == 0) {
      masks[0] = indices.a;
      masks[1] = indices.b;
    } else {
      masks[0] = indices.c;
      masks[1] = indices.d;
    }

    // Apply mask
    CUTLASS_PRAGMA_UNROLL
    for (int m = 0; m < 2; ++m) {
      CUTLASS_PRAGMA_UNROLL
      for (int r = 0; r < 4; ++r) {
        CUTLASS_PRAGMA_UNROLL
        for (int c = 0; c < 4; ++c) {
          lines[4 * row + r][4 * m + c] = lines[4 * row + r][4 * m + c] *
              int((masks[m] >> (4 * r + c)) & 1);
        }
      }
    }
  }
  static_assert(kOutputRowMajor, "Transpose here for ColMajor output");
  // Save dense with zeros
  CUTLASS_PRAGMA_UNROLL
  for (int i = 0; i < 8; ++i) {
    cutlass::arch::global_store<Fragment, sizeof(Fragment)>(
        lines[i], output + i * p.output_stride, true);
  }
}
#endif

Tensor _sparse_semi_structured_apply_dense(
    const Tensor& input,
    const Tensor& threads_masks) {

#if defined(USE_ROCM) || defined(_MSC_VER)
  TORCH_CHECK(false, "_sparse_semi_structured_apply_dense: not supported");
  return Tensor{};
#else
  TORCH_CHECK(
      input.scalar_type() == at::ScalarType::Half ||
          input.scalar_type() == at::ScalarType::BFloat16,
      "Unsupported `input` dtype");
  TORCH_CHECK(
      input.stride(0) == 1 || input.stride(1) == 1,
      "`input` should be either RowMajor or ColMajor. Invalid memory layout - try .contiguous()?");

  auto roundedx = cutlass::round_up(input.size(0), kWarpX);
  auto roundedy = cutlass::round_up(input.size(1), kWarpY);

  Params p;
  p.input = (uint16_t const*)input.data_ptr();
  p.input_dim0 = input.size(0);
  p.input_dim1 = input.size(1);
  p.threads_masks = (uint64_t const*)threads_masks.data_ptr();

  TORCH_CHECK(threads_masks.dim() == 3);
  TORCH_CHECK(threads_masks.size(0) == p.getBlocksGrid().x * p.getThreadsGrid().x);
  TORCH_CHECK(threads_masks.size(1) == p.getBlocksGrid().y * p.getThreadsGrid().y);
  TORCH_CHECK(threads_masks.stride(1) == sizeof(p.threads_masks[0]));
  TORCH_CHECK(threads_masks.size(2) == sizeof(p.threads_masks[0]));
  TORCH_CHECK(threads_masks.stride(2) == 1);
  TORCH_CHECK(threads_masks.scalar_type() == at::ScalarType::Byte);

  at::Tensor output = at::empty({p.input_dim0, p.input_dim1}, input.options());
  TORCH_INTERNAL_ASSERT(output.stride(-1) == 1, "expected RowMajor?");
  p.output = (uint16_t*)output.data_ptr();

  bool inputRowMajor = input.stride(-1) == 1;
  bool outputRowMajor = output.stride(-1) == 1;
  p.input_stride = input.stride(inputRowMajor ? 0 : 1);
  p.output_stride = output.stride(outputRowMajor ? 0 : 1);
  at::cuda::CUDAGuard device_guard(input.device());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  size_t smem_bytes = 0;
  if (inputRowMajor && outputRowMajor) {
    sparse_semi_structured_apply_dense_k<true, true>
        <<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes, stream>>>(p);
  } else if (!inputRowMajor && outputRowMajor) {
    sparse_semi_structured_apply_dense_k<false, true>
        <<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes, stream>>>(p);
  } else {
    TORCH_CHECK(
        false,
        "Unsupported configuration: `input` is ",
        inputRowMajor ? "RowMajor" : "ColMajor",
        ", and `output` is ",
        outputRowMajor ? "RowMajor" : "ColMajor");
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return output;
#endif
}

} // namespace
