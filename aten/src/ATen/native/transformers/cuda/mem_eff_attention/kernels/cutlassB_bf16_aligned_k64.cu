#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// This file is auto-generated. See "generate_kernels.py"
#include <ATen/native/transformers/cuda/mem_eff_attention/kernel_backward.h>
using namespace PyTorchMemEffAttention;
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, false, true, 64, 64, 64, true>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, false, true, 64, 64, 64, true>::kMinBlocksPerSm)
fmha_cutlassB_bf16_aligned_64x64_k64_seqaligned_sm80(typename AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, false, true, 64, 64, 64, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, false, true, 64, 64, 64, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_bf16_aligned_64x64_k64_seqaligned_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, false, true, 64, 64, 64>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, false, true, 64, 64, 64>::kMinBlocksPerSm)
fmha_cutlassB_bf16_aligned_64x64_k64_sm80(typename AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, false, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, false, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_bf16_aligned_64x64_k64_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
