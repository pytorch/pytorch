#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// This file is auto-generated. See "generate_kernels.py"
#include <ATen/native/transformers/cuda/mem_eff_attention/kernel_forward.h>
using namespace PyTorchMemEffAttention;
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_64x64_rf_sm50(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_64x64_rf_sm50` is for sm50-sm70, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 64, 64, 64, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 64, 64, 64, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_64x64_rf_sm70(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 64, 64, 64, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 700
#if __CUDA_ARCH__ < 750
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 64, 64, 64, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_64x64_rf_sm70` is for sm70-sm75, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_64x64_rf_sm75(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_64x64_rf_sm75` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_64x64_rf_sm80(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_64x64_rf_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_32x128_rf_sm50(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_32x128_rf_sm50` is for sm50-sm70, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 32, 128, 128, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 32, 128, 128, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_32x128_rf_sm70(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 32, 128, 128, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 700
#if __CUDA_ARCH__ < 750
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 32, 128, 128, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_32x128_rf_sm70` is for sm70-sm75, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, 128, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, 128, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_32x128_rf_sm75(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, 128, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, 128, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_32x128_rf_sm75` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_64x128_rf_sm80(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_64x128_rf_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_32x128_gmem_sm50(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_32x128_gmem_sm50` is for sm50-sm70, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 32, 128, 65536, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 32, 128, 65536, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_32x128_gmem_sm70(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 32, 128, 65536, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 700
#if __CUDA_ARCH__ < 750
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm70, true, 32, 128, 65536, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_32x128_gmem_sm70` is for sm70-sm75, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, 65536, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, 65536, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_32x128_gmem_sm75(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, 65536, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, 65536, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_32x128_gmem_sm75` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536, true, true>::kMinBlocksPerSm)
fmha_cutlassF_f16_aligned_32x128_gmem_sm80(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_f16_aligned_32x128_gmem_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
