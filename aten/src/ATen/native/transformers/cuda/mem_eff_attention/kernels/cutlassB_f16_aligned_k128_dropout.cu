#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// This file is auto-generated. See "generate_kernels.py"
#include <ATen/native/transformers/cuda/mem_eff_attention/kernel_backward.h>
using namespace PyTorchMemEffAttention;
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, true, true, false, 128, 64, 128>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, true, true, false, 128, 64, 128>::kMinBlocksPerSm)
fmha_cutlassB_f16_aligned_128x64_k128_dropout_sm70(typename AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, true, true, false, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 700
#if __CUDA_ARCH__ < 750
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, true, true, false, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_aligned_128x64_k128_dropout_sm70` is for sm70-sm75, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, true, true, false, 128, 64, 128>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, true, true, false, 128, 64, 128>::kMinBlocksPerSm)
fmha_cutlassB_f16_aligned_128x64_k128_dropout_sm75(typename AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, true, true, false, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, true, true, false, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_aligned_128x64_k128_dropout_sm75` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::half_t, true, true, true, 128, 128, 128>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::half_t, true, true, true, 128, 128, 128>::kMinBlocksPerSm)
fmha_cutlassB_f16_aligned_128x128_k128_dropout_sm80(typename AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::half_t, true, true, true, 128, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::half_t, true, true, true, 128, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_aligned_128x128_k128_dropout_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm50, cutlass::half_t, true, true, false, 64, 64, 128>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm50, cutlass::half_t, true, true, false, 64, 64, 128>::kMinBlocksPerSm)
fmha_cutlassB_f16_aligned_64x64_k128_dropout_sm50(typename AttentionBackwardKernel<cutlass::arch::Sm50, cutlass::half_t, true, true, false, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm50, cutlass::half_t, true, true, false, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_aligned_64x64_k128_dropout_sm50` is for sm50-sm70, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, true, true, false, 64, 64, 128>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, true, true, false, 64, 64, 128>::kMinBlocksPerSm)
fmha_cutlassB_f16_aligned_64x64_k128_dropout_sm70(typename AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, true, true, false, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 700
#if __CUDA_ARCH__ < 750
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm70, cutlass::half_t, true, true, false, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_aligned_64x64_k128_dropout_sm70` is for sm70-sm75, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, true, true, false, 64, 64, 128>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, true, true, false, 64, 64, 128>::kMinBlocksPerSm)
fmha_cutlassB_f16_aligned_64x64_k128_dropout_sm75(typename AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, true, true, false, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm75, cutlass::half_t, true, true, false, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_aligned_64x64_k128_dropout_sm75` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::half_t, true, true, false, 64, 64, 128>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::half_t, true, true, false, 64, 64, 128>::kMinBlocksPerSm)
fmha_cutlassB_f16_aligned_64x64_k128_dropout_sm80(typename AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::half_t, true, true, false, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::half_t, true, true, false, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_f16_aligned_64x64_k128_dropout_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
