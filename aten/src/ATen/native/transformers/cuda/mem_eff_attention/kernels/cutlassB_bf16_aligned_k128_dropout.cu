#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// This file is auto-generated. See "generate_kernels.py"
#include <ATen/native/transformers/cuda/mem_eff_attention/kernel_backward.h>
using namespace PyTorchMemEffAttention;
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, true, 128, 128, 128>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, true, 128, 128, 128>::kMinBlocksPerSm)
fmha_cutlassB_bf16_aligned_128x128_k128_dropout_sm80(typename AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, true, 128, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, true, 128, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_bf16_aligned_128x128_k128_dropout_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 64, 64, 128>::kNumThreads,
    AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 64, 64, 128>::kMinBlocksPerSm)
fmha_cutlassB_bf16_aligned_64x64_k128_dropout_sm80(typename AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionBackwardKernel<cutlass::arch::Sm80, cutlass::bfloat16_t, true, true, false, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassB_bf16_aligned_64x64_k128_dropout_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
