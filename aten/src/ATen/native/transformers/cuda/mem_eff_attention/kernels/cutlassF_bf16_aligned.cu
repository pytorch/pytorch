#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// This file is auto-generated. See "generate_kernels.py"
#include <ATen/native/transformers/cuda/mem_eff_attention/kernel_forward.h>
using namespace PyTorchMemEffAttention;
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, true, true>::kNumThreads,
    AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, true, true>::kMinBlocksPerSm)
fmha_cutlassF_bf16_aligned_64x64_rf_sm80(typename AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_bf16_aligned_64x64_rf_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, true, true>::kNumThreads,
    AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, true, true>::kMinBlocksPerSm)
fmha_cutlassF_bf16_aligned_64x128_rf_sm80(typename AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_bf16_aligned_64x128_rf_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, true, true>::kNumThreads,
    AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, true, true>::kMinBlocksPerSm)
fmha_cutlassF_bf16_aligned_32x128_gmem_sm80(typename AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, true, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ <= 1200
  if (!p.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, true, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: kernel `fmha_cutlassF_bf16_aligned_32x128_gmem_sm80` is for sm80-sm100, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
