#include "hip/hip_runtime.h"
#include <string_view>
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <cstdint>
#include <type_traits>

#include <ATen/core/Tensor.h>
#include <ATen/TensorOperators.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/util/Exception.h>
#include <c10/util/bit_cast.h>

#include <c10/core/TensorImpl.h>
#include <ATen/native/nested/NestedTensorTransformerFunctions.h>
#include <ATen/native/nested/NestedTensorUtils.h>
#include <ATen/native/transformers/attention.h>
#include <ATen/native/transformers/cuda/sdp_utils.h>
#include <ATen/native/transformers/sdp_utils_cpp.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_flash_attention_backward.h>
#include <ATen/ops/_flash_attention_backward_native.h>
#include <ATen/ops/_efficient_attention_backward.h>
#include <ATen/ops/_efficient_attention_backward_native.h>
#include <ATen/ops/_scaled_dot_product_flash_attention_backward_native.h>
#endif

#ifdef USE_FLASH_ATTENTION
// FlashAttention Specific Imports
#include <ATen/native/transformers/cuda/flash_attn/flash_api.h>
#endif
#ifdef USE_MEM_EFF_ATTENTION
#ifndef USE_ROCM
// MemoryEfficient Attention Specific Imports for CUDA
#include <ATen/native/transformers/cuda/mem_eff_attention/kernel_backward.h>
#include <ATen/native/transformers/cuda/mem_eff_attention/kernels/cutlassB.h>
#include <ATen/native/transformers/cuda/mem_eff_attention/gemm_kernel_utils.h>
#include <ATen/native/transformers/cuda/mem_eff_attention/pytorch_utils.h>
#else
// MemoryEfficient Attention Specific Imports for ROCM
#ifndef DISABLE_AOTRITON
#include <ATen/native/transformers/hip/aotriton_adapter.h>
#include <aotriton/flash.h>
#include <aotriton/runtime.h>
#endif
#include <ATen/native/transformers/hip/flash_attn/ck/me_ck_api.h>
#endif
#endif

#ifdef __HIP_PLATFORM_AMD__
#include <ATen/native/cudnn/hip/MHA.h>
#else
#include <ATen/native/cudnn/MHA.h>
#endif

namespace at::native {

std::tuple<Tensor, Tensor, Tensor> _flash_attention_backward(
    const Tensor& grad_out,
    const Tensor& query,
    const Tensor& key,
    const Tensor& value,
    const Tensor& out,
    const Tensor& logsumexp,
    const Tensor& cumulative_sequence_length_q,
    const Tensor& cumulative_sequence_length_k,
    int64_t max_seqlen_batch_q,
    int64_t max_seqlen_batch_k,
    double dropout_p,
    bool is_causal,
    const Tensor& philox_seed,
    const Tensor& philox_offset,
    std::optional<double> scale,
    std::optional<int64_t> window_size_left,
    std::optional<int64_t> window_size_right) {
#if defined(USE_FLASH_ATTENTION)
  const auto softmax_scale = sdp::calculate_scale(query, scale).expect_float();
  //  CUDA code assumes that dout is contiguous
  auto contiguous_grad_out = grad_out.contiguous();
  auto contiguous_out = out.contiguous();

  const int non_null_window_left = window_size_left.has_value() ? window_size_left.value() : -1;
  const int non_null_window_right = window_size_right.has_value() ? window_size_right.value() : -1;

  std::optional<at::Tensor> dq{std::nullopt};
  std::optional<at::Tensor> dk{std::nullopt};
  std::optional<at::Tensor> dv{std::nullopt};

  //  The kernel computes irregardless we will drop for this functions return
  Tensor grad_softmax;

  // Currently unused args:
  std::optional<at::Tensor> alibi_slopes{std::nullopt};
  const float softcap = 0.0;

  bool determinisitic{false};
  auto& ctx = at::globalContext();
  if (ctx.deterministicAlgorithms()) {
    if (ctx.deterministicAlgorithmsWarnOnly()) {
      TORCH_WARN_ONCE(
          "Flash Attention defaults to a non-deterministic algorithm. ",
          "To explicitly enable determinism call torch.use_deterministic_algorithms(True, warn_only=False).");
    } else {
      determinisitic = true;
    }
  }

  // We check the whether the cumulative_sequence_length_q is defined
  // in order to determine whether we are using varlen or dense forward
  if (cumulative_sequence_length_q.defined()) {
    // Varlen forward
    auto [dQuery, dKey, dValue, dSoftmax] = FLASH_NAMESPACE::mha_varlen_bwd(
        contiguous_grad_out,
        query,
        key,
        value,
        contiguous_out,
        logsumexp,
        dq,
        dk,
        dv,
        cumulative_sequence_length_q,
        cumulative_sequence_length_k,
        alibi_slopes,
        max_seqlen_batch_q,
        max_seqlen_batch_k,
        dropout_p,
        softmax_scale,
        false /*zero_tensors*/,
        is_causal,
        non_null_window_left,
        non_null_window_right,
        softcap,
        determinisitic,
        philox_seed,
        philox_offset);
    return std::make_tuple(std::move(dQuery), std::move(dKey), std::move(dValue));
  } else {
    // Dense forward
    auto [dQuery, dKey, dValue, dSoftmax] = FLASH_NAMESPACE::mha_bwd(
        contiguous_grad_out,
        query,
        key,
        value,
        contiguous_out,
        logsumexp,
        dq,
        dk,
        dv,
        alibi_slopes,
        dropout_p,
        softmax_scale,
        is_causal,
        non_null_window_left,
        non_null_window_right,
        softcap,
        determinisitic,
        philox_seed,
        philox_offset);
    return std::make_tuple(std::move(dQuery), std::move(dKey), std::move(dValue));
  }
#endif
  TORCH_CHECK(false, "USE_FLASH_ATTENTION was not enabled for build.");
  return std::make_tuple(Tensor(), Tensor(), Tensor());
}

std::tuple<Tensor, Tensor, Tensor> _scaled_dot_product_cudnn_attention_backward_cuda(
    const Tensor& grad_out,
    const Tensor& query,
    const Tensor& key,
    const Tensor& value,
    const Tensor& out,
    const Tensor& logsumexp,
    const Tensor& philox_seed,
    const Tensor& philox_offset,
    const Tensor& attn_bias,
    const Tensor& cum_seq_q,
    const Tensor& cum_seq_k,
    const int64_t max_q,
    const int64_t max_k,
    double dropout_p,
    bool is_causal,
    std::optional<double> scale) {

    auto& ctx = at::globalContext();
    if (ctx.deterministicAlgorithms()) {
      if (ctx.deterministicAlgorithmsWarnOnly()) {
        TORCH_WARN_ONCE(
            "cuDNN Attention defaults to a non-deterministic algorithm. ",
            "To explicitly enable determinism call torch.use_deterministic_algorithms(True, warn_only=False).");
      }
    }

    const int64_t batch_size = query.size(0);
    const int64_t num_heads = query.size(1);
    const int64_t head_dim_qk = query.size(3);
    const int64_t head_dim_v = value.size(3);
    const int64_t max_seqlen_batch_q = query.size(2);
    const int64_t max_seqlen_batch_k = key.size(2);

    // This is needed because SaveVariable automatically converts
    // std::optional to undefined tensor
    std::optional<Tensor> attn_bias_;
    if (attn_bias.defined()) {
      attn_bias_ = attn_bias;
    }
    if (attn_bias_.has_value()) {
      const auto bias_dim = attn_bias_.value().dim();
      if (bias_dim == 2) {
        attn_bias_ = attn_bias_.value().expand({batch_size, 1, max_seqlen_batch_q, max_seqlen_batch_k});
      } else if (bias_dim == 3) {
        attn_bias_ = attn_bias_.value().expand({batch_size, 1, max_seqlen_batch_q, max_seqlen_batch_k});
      } else {
        TORCH_CHECK(bias_dim == 4, "cuDNN SDPA expects either a 2D, 3D, or 4D attn_bias but got ", attn_bias_.value().dim(), "D");
        attn_bias_ = attn_bias_.value().expand({batch_size, attn_bias_.value().size(1), max_seqlen_batch_q, max_seqlen_batch_k});
      }
    }

    const auto softmax_scale = sdp::calculate_scale(query, scale).expect_float();
    auto dq = at::empty_like(query);
    auto dk = at::empty_like(key);
    auto dv = at::empty_like(value);
    run_cudnn_SDP_bprop(batch_size /*int64_t b*/,
                        num_heads /*int64_t h*/,
                        max_q/*int64_t s_q*/,
                        max_k/*int64_t s_kv*/,
                        head_dim_qk /*int64_t d_qk*/,
                        head_dim_v /*int64_t d_v*/,
                        softmax_scale /*float scaling_factor*/,
                        is_causal /*bool is_causal*/,
                        dropout_p /*float dropout_probability*/,
                        query /*const Tensor& q*/,
                        key /*const Tensor& k*/,
                        value /*const Tensor& v*/,
                        attn_bias_ /*const std::optional<Tensor>& attn_bias*/,
                        out /*const Tensor& o*/,
                        grad_out/*const Tensor& dO*/,
                        logsumexp.unsqueeze(-1)/*const Tensor& softmaxstats*/,
                        dq/*Tensor& dQ*/,
                        dk/*Tensor& dK*/,
                        dv/*Tensor& dV*/,
                        philox_seed/*Tensor& dropoutseed*/,
                        philox_offset/*Tensor& dropoutoffset*/);
    return std::make_tuple(std::move(dq), std::move(dk), std::move(dv));
}

std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor>
_efficient_attention_backward(
    const at::Tensor& grad_out_,
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    const std::optional<at::Tensor>& kernel_bias, // additive attention bias
    const at::Tensor& out,
    // (Mode 1MHK only) [b+1]: cu_seqlens_q[b] contains the
    // position of the first query token for batch $b
    const std::optional<at::Tensor>& cu_seqlens_q_dummy,
    // (Mode 1MHK only) [b+1]: cu_seqlens_k[b] contains the
    // position of the first key token for batch $b
    const std::optional<at::Tensor>& cu_seqlens_k_dummy,
    // (Mode 1MHK only) Maximum sequence length across batches
    int64_t max_seqlen_q,
    // (Mode 1MHK only) Maximum sequence length across batches
    int64_t max_seqlen_k,
    const at::Tensor& logsumexp,
    double dropout_p, // dropout probability
    const at::Tensor& philox_seed, // seed using for generating random numbers for dropout
    const at::Tensor& philox_offset, // offset into random number sequence
    int64_t custom_mask_type,
    const bool bias_requires_grad,
    const std::optional<double> scale,
    std::optional <int64_t> num_splits_key,
    const std::optional<int64_t> window_size,
    const bool shared_storage_dqdkdv) {
  #if defined(USE_MEM_EFF_ATTENTION)
  if (!grad_out_.defined()) {
    return std::make_tuple(Tensor{}, Tensor{}, Tensor{}, Tensor{});
  }
  // This path is used when we directly call _efficient_attention_forward
  // from python.
  // This is needed because SaveVariable automatically converts
  // std::optional to undefined tensor
  std::optional<Tensor> bias, cu_seqlens_q, cu_seqlens_k;
  bias = kernel_bias.has_value() && !kernel_bias->defined() ? std::nullopt : kernel_bias;
  cu_seqlens_q = cu_seqlens_q_dummy.has_value() && !cu_seqlens_q_dummy->defined() ? std::nullopt : cu_seqlens_q_dummy;
  cu_seqlens_k = cu_seqlens_k_dummy.has_value() && !cu_seqlens_k_dummy->defined() ? std::nullopt : cu_seqlens_k_dummy;

    // ndim
  TORCH_CHECK(query.dim() == grad_out_.dim());
  TORCH_CHECK(query.dim() == key.dim());
  TORCH_CHECK(query.dim() == value.dim());
  TORCH_CHECK(query.dim() == 4);

  // batch size
  TORCH_CHECK(query.size(0) == grad_out_.size(0));
  TORCH_CHECK(query.size(0) == key.size(0));
  TORCH_CHECK(query.size(0) == value.size(0));

  // seqlen
  TORCH_CHECK(key.size(1) == value.size(1));
  TORCH_CHECK(query.size(1) == grad_out_.size(1));

  // Num heads
  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(2) == value.size(2));
  TORCH_CHECK(query.size(2) == grad_out_.size(2));

  // Embedding per head
  TORCH_CHECK(query.size(3) == key.size(3));
  TORCH_CHECK(value.size(3) == grad_out_.size(3));

  // handle potentially non-contiguous grad_out through a copy
  auto grad_out = grad_out_.contiguous();
  CHECK_NOSPARSE_CONTIGUOUS_CUDA(grad_out);

  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(query);
  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(key);
  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(value);

  TORCH_CHECK(cu_seqlens_q.has_value() == cu_seqlens_k.has_value());
  TORCH_CHECK(
      !(cu_seqlens_q.has_value() && bias.has_value()),
      "cu seqlen + bias not supported");
  if (cu_seqlens_q.has_value()) {
    TORCH_CHECK(cu_seqlens_q->scalar_type() == at::ScalarType::Int);
    TORCH_CHECK(cu_seqlens_k->scalar_type() == at::ScalarType::Int);
    TORCH_CHECK(cu_seqlens_q->dim() == 1 && cu_seqlens_k->dim() == 1);
    CHECK_NOSPARSE_CONTIGUOUS_CUDA((*cu_seqlens_q));
    CHECK_NOSPARSE_CONTIGUOUS_CUDA((*cu_seqlens_k));
    TORCH_CHECK(cu_seqlens_q->size(0) == cu_seqlens_k->size(0));
    TORCH_CHECK(query.size(0) == 1, "cu_seqlen only supports batch_size=1");
    TORCH_CHECK(max_seqlen_q > 0, "max_seqlen_q required with `cu_seqlens_q`");
    TORCH_CHECK(max_seqlen_k > 0, "max_seqlen_k required with `cu_seqlens_k`");
    TORCH_CHECK(
        max_seqlen_k <= key.size(1), "Invalid max_seqlen_k:", max_seqlen_k);
    TORCH_CHECK(
        max_seqlen_q <= query.size(1), "Invalid max_seqlen_q:", max_seqlen_q);
  } else {
    max_seqlen_q = query.size(1);
    max_seqlen_k = key.size(1);
  }

  at::cuda::CUDAGuard device_guard(query.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t nH = query.size(2);
  int64_t K = query.size(3);
  int64_t Kv = value.size(3);

  at::Tensor grad_q, grad_k, grad_v, grad_bias;
  if (shared_storage_dqdkdv) {
    // Create one big contiguous chunk
    // This is because q, k and v usually come from a single
    // output of a linear layer that is chunked.
    // Creating the gradients with the right layout saves us
    // a `torch.cat` call in the backward pass
    TORCH_CHECK(
      query.size(1) == key.size(1),
      "`shared_storage_dqdkdv` is only supported when Q/K/V "
      "have the same sequence length: got ", query.size(1),
      " query tokens and ", key.size(1), " key/value tokens"
    );
    TORCH_CHECK(
      query.size(3) == key.size(3),
      "`shared_storage_dqdkdv` is only supported when Q/K/V "
      "have the same embed dim: got ", query.size(3),
      " for Q, and ", key.size(3), " for K"
    );
    at::Tensor chunk = at::empty({B, M, 3, nH, K}, query.options());
    grad_q = chunk.select(2, 0);
    grad_k = chunk.select(2, 1);
    grad_v = chunk.select(2, 2);
  } else {
    grad_q = at::empty(query.sizes(), query.options());
    grad_k = at::empty(key.sizes(), key.options());
    grad_v = at::empty(value.sizes(), value.options());
  }

  if (bias_requires_grad) {
    // force alignment for the last dim
    std::vector<int64_t> sz = bias->sizes().vec();
    int64_t lastDim = sz[sz.size() - 1];
    int64_t alignTo = 16;
    sz[sz.size() - 1] = alignTo * ((lastDim + alignTo - 1) / alignTo);
    grad_bias = at::empty(sz, bias->options())
                    .slice(/*dim=*/-1, /*start=*/0, /*end=*/lastDim);
  }

  const bool use_dropout = std::fpclassify(dropout_p) != FP_ZERO;

  // See Note [Seed and Offset Device]
  at::PhiloxCudaState rng_engine_inputs;
  if (use_dropout) {
    if (at::cuda::currentStreamCaptureStatus() ==
        at::cuda::CaptureStatus::None) {
      rng_engine_inputs = at::PhiloxCudaState(
          *philox_seed.data_ptr<int64_t>(),
          *philox_offset.data_ptr<int64_t>());
    } else { // dropout + capture
      rng_engine_inputs = at::PhiloxCudaState(
          philox_seed.data_ptr<int64_t>(),
          philox_offset.data_ptr<int64_t>(),
          0);
    }
  }

#ifdef USE_ROCM
  // ROCM Implementation
  if(at::globalContext().getROCmFAPreferredBackend() == at::ROCmFABackend::Ck)
  {
#if defined(USE_CK_FLASH_ATTENTION)
    const auto my_softmax_scale = sdp::calculate_scale(query, scale).expect_float();
    // Store grad_bias in optional
    std::optional<at::Tensor> opt_grad_bias = grad_bias;
    auto
        [dQ,
         dK,
         dV,
         dBias] =
             pytorch_flash::mem_eff_backward_ck(
                     grad_out,
                     query,
                     key,
                     value,
                     out,
                     logsumexp,
                     grad_q,
                     grad_k,
                     grad_v,
                     bias,
                     bias_requires_grad,
                     opt_grad_bias,
                     cu_seqlens_q,
                     cu_seqlens_k,
                     max_seqlen_q,
                     max_seqlen_k,
                     float(dropout_p),
                     my_softmax_scale,
                     custom_mask_type == 0 ? false : true, // is_causal
                     false, // deterministic
                     false, // zero_tensors
                     philox_seed,
                     philox_offset);
    grad_bias = dBias;
#else
    TORCH_CHECK(false, "Attempting to use CK mem_eff_backward backend in a build that has not built CK");
#endif
  } else {
#ifndef DISABLE_AOTRITON
    TORCH_CHECK(!num_splits_key.has_value(),
              "ROCM does not support num_split_keys in _efficient_attention_forward");
    TORCH_CHECK(!window_size.has_value(),
              "ROCM does not support window_size in _efficient_attention_forward");
    auto ret = aotriton::v2::flash::check_gpu(stream);
    if (hipSuccess != ret) {
      TORCH_CHECK(false,
                "[AOTriton] Accelerated SDPA only supports MI200/MI300X/7900XTX/9070XT GPUs"
                " (gfx90a/gfx942/gfx1100/gfx1201)")
    }
    const auto softmax_scale = sdp::calculate_scale(query, scale).expect_float();
    bool is_causal;
    if (static_cast<int64_t>(sdp::CustomMaskType::CausalFromTopLeft) == custom_mask_type) {
      is_causal = true;
    } else if (static_cast<int64_t>(sdp::CustomMaskType::NoCustomMask) == custom_mask_type) {
      is_causal = false;
    } else {
      TORCH_CHECK(false, "[_efficient_attention_backward] Unsupported mask type in AOTriton, for now");
    }
    at::Tensor q_t = query.permute({0,2,1,3});
    at::Tensor k_t = key.permute({0,2,1,3});
    at::Tensor v_t = value.permute({0,2,1,3});
    at::Tensor out_t = out.permute({0,2,1,3});
    at::Tensor dq_t = grad_q.permute({0,2,1,3});
    at::Tensor dk_t = grad_k.permute({0,2,1,3});
    at::Tensor dv_t = grad_v.permute({0,2,1,3});
    at::Tensor dout_t = grad_out.permute({0,2,1,3});
    at::Tensor softmax_lse = logsumexp.view({B * nH, max_seqlen_q});
    hipError_t err;
    using aotriton::v2::flash::attn_bwd;
    using aotriton::v2::flash::attn_bwd_fused;
    using aotriton::v2::flash::attn_bwd_compact_varlen;
    using sdp::aotriton_adapter::mk_aotensor;
    using sdp::aotriton_adapter::mk_aoscalartensor;
    using sdp::aotriton_adapter::cast_dtype;
    aotriton::TensorView<4> empty_t4(0, {0, 0, 0, 0}, {0, 0, 0, 0}, cast_dtype(query.dtype()));
    if (cu_seqlens_q.has_value()) {
      at::Tensor delta = at::empty_like(softmax_lse).contiguous();
      // varlen aka Nested tensor
      err = attn_bwd_compact_varlen(mk_aotensor(q_t, "q"),
                                    mk_aotensor(k_t, "k"),
                                    mk_aotensor(v_t, "v"),
                                    mk_aotensor<1>(cu_seqlens_q.value(), "cu_seqlens_q"),
                                    mk_aotensor<1>(cu_seqlens_k.value(), "cu_seqlens_k"),
                                    max_seqlen_q,
                                    max_seqlen_k,
                                    bias.has_value() ? mk_aotensor(bias.value(), "bias") : empty_t4,
                                    softmax_scale,
                                    mk_aotensor(out_t, "out"),
                                    mk_aotensor(dout_t, "dout"),
                                    mk_aotensor(dq_t, "dq"),
                                    mk_aotensor(dk_t, "dk"),
                                    mk_aotensor(dv_t, "dv"),
                                    bias_requires_grad ? mk_aotensor(grad_bias, "db") : empty_t4,
                                    mk_aotensor<2>(softmax_lse, "L"),
                                    mk_aotensor<2>(delta, "delta"),
                                    float(dropout_p),
                                    mk_aoscalartensor(philox_seed),
                                    mk_aoscalartensor(philox_offset),
                                    0,
                                    is_causal,
                                    stream);
    } else { // cu_seqlens.has_value
      auto d_head = Kv;
      bool use_fused_bwd = d_head <= 192 && d_head * max_seqlen_q < 64 * 512;
      if (use_fused_bwd) {
        err = attn_bwd_fused(mk_aotensor(q_t, "q"),
                             mk_aotensor(k_t, "k"),
                             mk_aotensor(v_t, "v"),
                             bias.has_value() ? mk_aotensor(bias.value(), "bias") : empty_t4,
                             softmax_scale,
                             mk_aotensor(out_t, "out"),
                             mk_aotensor(dout_t, "dout"),
                             mk_aotensor(dq_t, "dq"),
                             mk_aotensor(dk_t, "dk"),
                             mk_aotensor(dv_t, "dv"),
                             bias_requires_grad ? mk_aotensor(grad_bias, "db") : empty_t4,
                             mk_aotensor<2>(softmax_lse, "L"),
                             float(dropout_p),
                             mk_aoscalartensor(philox_seed),
                             mk_aoscalartensor(philox_offset),
                             0,
                             is_causal,
                             stream);
      } else {
        at::Tensor delta = at::empty_like(softmax_lse).contiguous();
        err = attn_bwd(mk_aotensor(q_t, "q"),
                     mk_aotensor(k_t, "k"),
                     mk_aotensor(v_t, "v"),
                     bias.has_value() ? mk_aotensor(bias.value(), "bias") : empty_t4,
                     softmax_scale,
                     mk_aotensor(out_t, "out"),
                     mk_aotensor(dout_t, "dout"),
                     mk_aotensor(dq_t, "dq"),
                     mk_aotensor(dk_t, "dk"),
                     mk_aotensor(dv_t, "dv"),
                     bias_requires_grad ? mk_aotensor(grad_bias, "db") : empty_t4,
                     mk_aotensor<2>(softmax_lse, "L"),
                     mk_aotensor<2>(delta, "delta"),
                     float(dropout_p),
                     mk_aoscalartensor(philox_seed),
                     mk_aoscalartensor(philox_offset),
                     0,
                     is_causal,
                     stream);
      } //used_fused_bwd
    } // cuseqlen.has_value
#else  // DISABLE_AOTRITON
    TORCH_CHECK(false, "Attempting to use aotriton mem_eff_backward backend in a build that has not built AOTriton");
#endif
  } // Use CK
#else // USE_CUDA
  at::Tensor workspace;
  hipDeviceProp_t* p = at::cuda::getDeviceProperties(query.device().index());
  int computeCapability = p->major * 10 + p->minor;
  if (computeCapability == 121) {
    computeCapability = 120;
  }

  bool kernel_launched = false;
  const auto maxK = std::max(query.size(3), value.size(3));
  const auto maxShmem = p->sharedMemPerBlockOptin;

  auto launchKernel = [&](auto _k, auto kernel_fn) {
    using Kernel = decltype(_k);
    using scalar_t = typename Kernel::scalar_t;
    (void)_k;

    if (kernel_launched) {
      return;
    }
    // Check if this kernel is compatible
    if (Kernel::kMaxK < maxK) {
      return;
    }
    // Dropout must be supported if we need it
    if (use_dropout && !Kernel::kApplyDropout) {
      return;
    }
    if (Kernel::kKeysQueriesAlignedToBlockSize &&
        (cu_seqlens_q.has_value() || M % Kernel::kBlockSizeI ||
         N % Kernel::kBlockSizeJ)) {
      return;
    }
    // Alignment
    if ((query.stride(2) % Kernel::kMinimumAlignment) ||
        (key.stride(2) % Kernel::kMinimumAlignment) ||
        (value.stride(2) % Kernel::kMinimumAlignment)) {
      return;
    }
    // Uses too much shmem
    size_t smem_bytes = sizeof(typename Kernel::SharedStorage);
    if (smem_bytes > maxShmem) {
      return;
    }

    kernel_launched = true;

    // TODO: Fuse this into a kernel?
    // This is a bottleneck for smaller sequences (M <= 128)
    auto delta = Kernel::kKernelComputesDelta
        ? at::empty({B, nH, M}, query.options().dtype(at::ScalarType::Float))
        : (grad_out.to(at::kFloat) * out.to(at::kFloat))
              .sum(-1)
              .transpose(-2, -1)
              .contiguous();
    TORCH_INTERNAL_ASSERT(delta.size(0) == B);
    TORCH_INTERNAL_ASSERT(delta.size(1) == nH);
    TORCH_INTERNAL_ASSERT(delta.size(2) == M);

    typename Kernel::Params p;
    p.query_ptr = (const scalar_t*)query.const_data_ptr();
    p.key_ptr = (const scalar_t*)key.const_data_ptr();
    p.value_ptr = (const scalar_t*)value.const_data_ptr();
    p.logsumexp_ptr = (typename Kernel::lse_scalar_t const *)logsumexp.const_data_ptr();
    p.output_ptr = (const scalar_t*)out.const_data_ptr();
    p.grad_output_ptr = (const scalar_t*)grad_out.const_data_ptr();
    p.grad_query_ptr = (scalar_t*)grad_q.data_ptr();
    p.grad_key_ptr = (scalar_t*)grad_k.data_ptr();
    p.grad_value_ptr = (scalar_t*)grad_v.data_ptr();
    p.delta_ptr = (float*)delta.data_ptr();
    p.head_dim = query.size(3);
    p.head_dim_value = value.size(3);
    p.num_queries = max_seqlen_q;
    p.num_keys = max_seqlen_k;
    p.num_batches = cu_seqlens_q.has_value() ? cu_seqlens_q->size(0) - 1 : B;
    p.num_heads = nH;
    p.custom_mask_type = custom_mask_type;
    p.scale = sdp::calculate_scale(query, scale).expect_float();
    if (cu_seqlens_q.has_value()) {
      p.cu_seqlens_q_ptr = (const int32_t*)cu_seqlens_q->const_data_ptr();
      p.cu_seqlens_k_ptr = (const int32_t*)cu_seqlens_k->const_data_ptr();
    }
    if (window_size.has_value()) {
      p.window_size = *window_size;
    }

    ASSIGN_CHECK_OVERFLOW(p.lse_strideB, logsumexp.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.lse_strideH, logsumexp.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.gO_strideB, grad_out.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.gO_strideM, grad_out.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.gO_strideH, grad_out.stride(2));

    ASSIGN_CHECK_OVERFLOW(p.o_strideB, out.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.o_strideH, out.stride(2));

    ASSIGN_CHECK_OVERFLOW(p.gQ_strideB, grad_q.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.gK_strideB, grad_k.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.gV_strideB, grad_v.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.gQ_strideH, grad_q.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.gK_strideH, grad_k.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.gV_strideH, grad_v.stride(2));
    p.gQKV_strideM_multiplier = shared_storage_dqdkdv ? 3 : 1;
    TORCH_INTERNAL_ASSERT(p.gQ_strideM() == grad_q.stride(1));
    TORCH_INTERNAL_ASSERT(p.gK_strideM() == grad_k.stride(1));
    TORCH_INTERNAL_ASSERT(p.gV_strideM() == grad_v.stride(1));

    ASSIGN_CHECK_OVERFLOW(p.q_strideB, query.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.k_strideB, key.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.v_strideB, value.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.q_strideM, query.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.k_strideM, key.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.v_strideM, value.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.q_strideH, query.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.k_strideH, key.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.v_strideH, value.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.delta_strideB, delta.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.delta_strideH, delta.stride(1));

    if (bias.has_value()) {
      CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA((*bias));
      TORCH_CHECK(
          bias->scalar_type() == CutlassToAtenDtype<scalar_t>::atScalarType(),
          "invalid dtype for bias - should match query's dtype");

      p.bias_ptr = (scalar_t*)bias->data_ptr();

      TORCH_CHECK(bias->dim() == 4, "Bias expected in BMHK format");
      TORCH_CHECK(
          bias->size(0) == query.size(0),
          "attn_bias: wrong shape (batch dimension)");
      TORCH_CHECK(
          bias->size(1) == query.size(2),
          "attn_bias: wrong shape (head dimension)");
      TORCH_CHECK(
          bias->size(2) == query.size(1),
          "attn_bias: wrong shape (seqlenQ dimension)");
      TORCH_CHECK(
          bias->size(3) == key.size(1),
          "attn_bias: wrong shape (seqlenKV dimension)");
      TORCH_CHECK(
          bias->stride(3) == 1,
          "attn_bias: wrong alignment (last dimension must be contiguous)");
      ASSIGN_CHECK_OVERFLOW(p.bias_strideB, bias->stride(0));
      ASSIGN_CHECK_OVERFLOW(p.bias_strideH, bias->stride(1));
      ASSIGN_CHECK_OVERFLOW(p.bias_strideM, bias->stride(2));

      if (bias_requires_grad) {
        p.grad_bias_ptr = (scalar_t*)grad_bias.data_ptr();

        ASSIGN_CHECK_OVERFLOW(p.gB_strideB, grad_bias.stride(0));
        ASSIGN_CHECK_OVERFLOW(p.gB_strideH, grad_bias.stride(1));
        ASSIGN_CHECK_OVERFLOW(p.gB_strideM, grad_bias.stride(2));
      }
    }

    if (use_dropout) {
      p.rng_engine_inputs = rng_engine_inputs;
      p.dropout_prob = dropout_p;
    }

    // Heuristic for finding optimal number of splits
    auto parallelism_without_split_key =
        p.getBlocksGrid().x * p.getBlocksGrid().y * p.getBlocksGrid().z;
    p.num_splits_key = cutlass::ceil_div(p.num_keys, Kernel::kBlockSizeJ);
    if (num_splits_key.has_value()) {
      p.num_splits_key =
          std::min<int64_t>(p.num_splits_key, num_splits_key.value());
    } else {
      // Keys splitting heuristic

      // If we already have enough parallelism, split-keys can help
      // better use L2 cache.
      // This is negligible when the seqlen is too small tho
      if (parallelism_without_split_key >= 256 &&
          p.num_keys <= 2 * Kernel::kBlockSizeJ) {
        p.num_splits_key = 1;
      }
      // Increasing `split_keys` leads to using more gmem for temporary storage
      // when we need a staging area for gK/gV. let's avoid that
      if (Kernel::kNeedsAccumGradK || Kernel::kNeedsAccumGradV) {
        p.num_splits_key = std::min(
            int(p.num_splits_key), 200 / (p.num_batches * p.num_heads));
      }
    }
    if (!Kernel::kEnableSplitKeys || p.num_splits_key < 1) {
      p.num_splits_key = 1;
    }

    auto& ctx = at::globalContext();
    if (ctx.deterministicAlgorithms()) {
      if (ctx.deterministicAlgorithmsWarnOnly()) {
        TORCH_WARN_ONCE(
            "Memory Efficient attention defaults to a non-deterministic algorithm. ",
            "To explicitly enable determinism call torch.use_deterministic_algorithms(True, warn_only=False).");
      } else {
        TORCH_CHECK(
            num_splits_key.value_or(1) <= 1,
            "Using `num_splits_key > 1` makes the algorithm non-deterministic, and pytorch's deterministic mode is enabled");
        p.num_splits_key = 1;
      }
    }
    int64_t size_bytes = p.workspace_size();
    if (size_bytes) {
      workspace =
          at::empty({size_bytes}, query.options().dtype(at::ScalarType::Byte));
      p.workspace = (float*)workspace.data_ptr();
      if (p.should_zero_workspace()) {
        workspace.zero_();
      }
    }

    // Handle the edge-cases where some tensors are empty
    if (p.num_queries == 0 || p.num_keys == 0 || p.num_batches == 0 ||
        p.num_heads == 0) {
      grad_k.zero_();
      grad_v.zero_();
      grad_q.zero_();
      return;
    }
    Kernel::check_supported(p);

    if (smem_bytes > 0xc000) {
      // https://docs.nvidia.com/cuda/cuda-c-programming-guide/#features-and-technical-specifications-technical-specifications-per-compute-capability
      auto err = hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
      TORCH_CHECK(
          err != hipErrorInvalidValue,
          "This GPU does not have enough shared-memory (kernel requires ",
          smem_bytes / 1024,
          " kb)");
      AT_CUDA_CHECK(err);
    }

    // second syntax resulted in the error below on windows
    // error C3495: 'kernel_fn': a simple capture must be a variable
    // with automatic storage duration declared
    // in the reaching scope of the lambda
#ifdef _WIN32
    hipFuncAttributes attr;
    AT_CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(kernel_fn)));
    TORCH_INTERNAL_ASSERT(
        attr.binaryVersion >= Kernel::ArchTag::kMinComputeCapability,
        "Something went wrong in the build process");
#else
    auto checkBinaryArchMatches = [&]() {
      hipFuncAttributes attr;
      AT_CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(kernel_fn)));
      return attr.binaryVersion >= Kernel::ArchTag::kMinComputeCapability;
    };
    TORCH_INTERNAL_ASSERT(
        checkBinaryArchMatches(), "Something went wrong in the build process");
#endif

    kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes, stream>>>(p);
  };

  DISPATCH_TYPES(query, ([&]() {
                   dispatch_cutlassB<scalar_t>(launchKernel, computeCapability);
                 }));
  TORCH_CHECK(kernel_launched, "cutlassB: no kernel found to launch!");
  AT_CUDA_CHECK(hipGetLastError());
#endif // USE_ROCM
  return std::make_tuple(std::move(grad_q), std::move(grad_k), std::move(grad_v), std::move(grad_bias));
  #endif // defined(USE_MEM_EFF_ATTENTION)
  TORCH_CHECK(false, "USE_MEM_EFF_ATTENTION was not enabled for build.")
  return std::make_tuple(Tensor{}, Tensor{}, Tensor{}, Tensor{});
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> _scaled_dot_product_flash_attention_backward_cuda(
    const at::Tensor& grad_out_,
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    const at::Tensor& out,
    const at::Tensor& logsumexp,
    const Tensor& cumulative_sequence_length_q,
    const Tensor& cumulative_sequence_length_k,
    const int64_t max_seqlen_batch_q,
    const int64_t max_seqlen_batch_k,
    double dropout_p,
    bool is_causal,
    const at::Tensor& philox_seed,
    const at::Tensor& philox_offset,
    std::optional<double> scale){
  if (!grad_out_.defined()) {
    return std::make_tuple(Tensor{}, Tensor{}, Tensor{});
  }

  Tensor q_t = query.transpose(1, 2);
  Tensor k_t = key.transpose(1, 2);
  Tensor v_t = value.transpose(1, 2);

  Tensor grad_out_t = grad_out_.transpose(1,2);
  Tensor out_t = out.transpose(1,2);

  auto [grad_q, grad_k, grad_v] = at::_flash_attention_backward(
    grad_out_t,
    q_t,
    k_t,
    v_t,
    out_t,
    logsumexp,
    cumulative_sequence_length_q,
    cumulative_sequence_length_k,
    max_seqlen_batch_q,
    max_seqlen_batch_k,
    dropout_p,
    is_causal,
    philox_seed,
    philox_offset,
    scale);

  grad_q = grad_q.transpose(1,2);
  grad_k = grad_k.transpose(1,2);
  grad_v = grad_v.transpose(1,2);

  return std::make_tuple(std::move(grad_q), std::move(grad_k), std::move(grad_v));
}


std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor> _scaled_dot_product_efficient_attention_backward_cuda(
    const at::Tensor& grad_out_,
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    const at::Tensor& attn_bias,
    const at::Tensor& out,
    const at::Tensor& logsumexp,
    const at::Tensor& philox_seed,
    const at::Tensor& philox_offset,
    double dropout_p,
    std::array<bool, 4> grad_input_mask,
    bool causal,
    std::optional<double> scale) {

  if (!grad_out_.defined()) {
    return std::make_tuple(Tensor{}, Tensor{}, Tensor{}, Tensor{});
  }
  auto grad_out = grad_out_.transpose(1, 2);
  auto out_t = out.transpose(1, 2);
  auto q_t = query.transpose(1, 2);
  auto k_t = key.transpose(1, 2);
  auto v_t = value.transpose(1, 2);

  // This is needed because SaveVariable automatically converts
  // std::optional to undefined tensor
  std::optional<Tensor> kernel_bias;
  if (attn_bias.defined()) {
    kernel_bias = attn_bias;
  }
  // Will add with signauter changes for dropout and bias
  // We are only handling Dense inputs, but this should be passed
  // from forward to backward
  int64_t max_seqlen_q = q_t.size(1);
  int64_t max_seqlen_k = k_t.size(1);

  sdp::CustomMaskType custom_mask_type = causal
    ? sdp::CustomMaskType::CausalFromTopLeft
    : sdp::CustomMaskType::NoCustomMask;
  auto [grad_q, grad_k, grad_v, grad_bias] =
      at::_efficient_attention_backward(
          grad_out,
          q_t,
          k_t,
          v_t,
          kernel_bias,
          out_t,
          std::nullopt,
          std::nullopt,
          max_seqlen_q,
          max_seqlen_k,
          logsumexp,
          dropout_p,
          philox_seed,
          philox_offset,
          static_cast<int64_t>(custom_mask_type),
          grad_input_mask[3],
          scale,
          std::nullopt);  // num_split_keys
  return std::make_tuple(
      grad_q.transpose(1, 2), grad_k.transpose(1, 2), grad_v.transpose(1, 2), grad_bias);
}

} // namespace at::native
