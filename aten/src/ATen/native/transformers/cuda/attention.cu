#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <type_traits>

#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/NestedTensorImpl.h>
#include <ATen/TensorAccessor.h>
#include <ATen/TensorOperators.h>
#include <c10/util/Logging.h>
#include <c10/util/bit_cast.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/NonSymbolicBC.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/PersistentSoftmax.cuh>
#include <ATen/native/cuda/block_reduce.cuh>
#include <optional>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_cudnn_attention_forward.h>
#include <ATen/ops/_cudnn_attention_forward_native.h>
#include <ATen/ops/_efficient_attention_forward.h>
#include <ATen/ops/_efficient_attention_forward_native.h>
#include <ATen/ops/_fill_mem_eff_dropout_mask_native.h>
#include <ATen/ops/_flash_attention_forward.h>
#include <ATen/ops/_flash_attention_forward_native.h>
#include <ATen/ops/_fused_sdp_choice_native.h>
#include <ATen/ops/_masked_softmax.h>
#include <ATen/ops/_native_multi_head_attention_native.h>
#include <ATen/ops/scaled_dot_product_attention_native.h>
#include <ATen/ops/_scaled_dot_product_efficient_attention.h>
#include <ATen/ops/_scaled_dot_product_efficient_attention_native.h>
#include <ATen/ops/_scaled_dot_product_flash_attention.h>
#include <ATen/ops/_scaled_dot_product_flash_attention_native.h>
#include <ATen/ops/_softmax.h>
#include <ATen/ops/_transform_bias_rescale_qkv.h>
#include <ATen/ops/_triton_multi_head_attention_native.h>
#include <ATen/ops/_triton_scaled_dot_attention.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/linear.h>
#include <ATen/ops/narrow_native.h>
#include <ATen/ops/scalar_tensor.h>
#include <ATen/ops/scaled_dot_product_attention.h>
#include <ATen/ops/split_native.h>
#include <ATen/ops/zeros.h>
#endif

#ifdef __HIP_PLATFORM_AMD__
#include <ATen/native/cudnn/hip/MHA.h>
#else
#include <ATen/native/cudnn/MHA.h>
#endif

#include <c10/cuda/CUDAMathCompat.h>

#include <ATen/native/transformers/attention.h>
#include <ATen/native/nested/NestedTensorUtils.h>
#include <ATen/native/nested/NestedTensorTransformerUtils.h>
#include <ATen/native/nested/NestedTensorTransformerFunctions.h>
#include <ATen/native/transformers/cuda/sdp_utils.h>
#include <ATen/native/transformers/sdp_utils_cpp.h>

#ifdef USE_FLASH_ATTENTION
// FlashAttention Specific Imports
#include <ATen/native/transformers/cuda/flash_attn/flash_api.h>
#if !defined(__HIP_PLATFORM_AMD__)
#include <namespace_config.h>
#endif
#endif
#ifdef USE_MEM_EFF_ATTENTION
#ifndef USE_ROCM
// MemoryEfficient Attention Specific Imports for CUDA
#include <ATen/native/transformers/cuda/mem_eff_attention/kernel_forward.h>
#include <ATen/native/transformers/cuda/mem_eff_attention/kernels/cutlassF.h>
#include <ATen/native/transformers/cuda/mem_eff_attention/pytorch_utils.h>
#else
// MemoryEfficient Attention Specific Imports for ROCM
#ifndef DISABLE_AOTRITON
#include <ATen/native/transformers/hip/aotriton_adapter.h>
#include <aotriton/flash.h>
#include <aotriton/runtime.h>
#endif
#include <ATen/native/transformers/hip/flash_attn/ck/me_ck_api.h>
#endif
#endif

namespace at {

namespace cuda::philox {

__global__ void unpack_cudnn(at::PhiloxCudaState arg, int64_t* seed_ptr, int64_t* offset_ptr) {
  if (arg.captured_) {
    *seed_ptr = static_cast<int64_t>(*arg.seed_.ptr);
    *offset_ptr = static_cast<int64_t>(
                    *(arg.offset_.ptr) + static_cast<int64_t>(arg.offset_intragraph_));
  } else {
    *seed_ptr = static_cast<int64_t>(arg.seed_.val);
    *offset_ptr = static_cast<int64_t>(arg.offset_.val);
  }
}

void unpack_cudnn_wrapper(at::PhiloxCudaState arg, int64_t* seed_ptr, int64_t* offset_ptr, hipStream_t stream) {
at::cuda::philox::unpack_cudnn<<<1, 1, 0, stream>>>(arg, seed_ptr, offset_ptr);
}

} // namespace cuda::philox

namespace native {

namespace {


static constexpr int TRANSFORM_BIAS_RESCALE_VEC = 4;

template <typename scalar_t, typename accscalar_t, bool assume_aligned>
__global__ void transform_bias_rescale_qkv_kernel(
    // [B, T, 3 * D]
    const PackedTensorAccessor64<scalar_t, 3, RestrictPtrTraits> qkv,
    // [3 * D]
    const PackedTensorAccessor64<scalar_t, 1, RestrictPtrTraits> qkv_bias,
    // [3, B, NH, T, DH]
    PackedTensorAccessor64<scalar_t, 5, RestrictPtrTraits> q_k_v,
    const scalar_t inv_sqrt_dim_per_head) {
  // warp per DH.
  // so launch B * NH * T warps.
  auto NH = q_k_v.size(2);
  auto T = q_k_v.size(3);
  auto DH = q_k_v.size(4);

  auto t = blockIdx.x % T;
  auto b = blockIdx.x / T;

  auto D = NH * DH;

  if (assume_aligned) {
    constexpr int VEC = TRANSFORM_BIAS_RESCALE_VEC;
    using LoadT = memory::aligned_vector<scalar_t, VEC>;
    for (int32_t d_v = threadIdx.x; d_v < D / VEC; d_v += blockDim.x) {
      auto d = d_v * VEC;
      auto nh = d / DH;
      auto dh = d % DH;
      scalar_t qkv_bias_q[VEC];
      scalar_t qkv_bias_k[VEC];
      scalar_t qkv_bias_v[VEC];
      scalar_t qkv_q[VEC];
      scalar_t qkv_k[VEC];
      scalar_t qkv_v[VEC];

      // Here we require D % VEC == 0 for these vectorized loads.
      *reinterpret_cast<LoadT*>(&qkv_bias_q) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 0 * D]);
      *reinterpret_cast<LoadT*>(&qkv_bias_k) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 1 * D]);
      *reinterpret_cast<LoadT*>(&qkv_bias_v) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 2 * D]);

      *reinterpret_cast<LoadT*>(&qkv_q) =
          *reinterpret_cast<const LoadT*>(&qkv[b][t][d + 0 * D]);
      *reinterpret_cast<LoadT*>(&qkv_k) =
          *reinterpret_cast<const LoadT*>(&qkv[b][t][d + 1 * D]);
      *reinterpret_cast<LoadT*>(&qkv_v) =
          *reinterpret_cast<const LoadT*>(&qkv[b][t][d + 2 * D]);

#pragma unroll
      // TODO: specialize for float2half2/half2float2?
      for (auto ii = 0; ii < VEC; ++ii) {
        qkv_q[ii] = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_q[ii]) +
             static_cast<accscalar_t>(qkv_bias_q[ii])) *
            static_cast<accscalar_t>(inv_sqrt_dim_per_head));
        qkv_k[ii] = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_k[ii]) +
             static_cast<accscalar_t>(qkv_bias_k[ii])));
        qkv_v[ii] = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_v[ii]) +
             static_cast<accscalar_t>(qkv_bias_v[ii])));
      }

      // Here we require DH % VEC == 0 for these vectorized stores.
      *reinterpret_cast<LoadT*>(&q_k_v[0][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_q);
      *reinterpret_cast<LoadT*>(&q_k_v[1][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_k);
      *reinterpret_cast<LoadT*>(&q_k_v[2][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_v);
    }
  } else {
    // Same as above, but we can't vectorize memory access.
    for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
      auto nh = d / DH;
      auto dh = d % DH;
      scalar_t qkv_bias_q = qkv_bias[d + 0 * D];
      scalar_t qkv_bias_k = qkv_bias[d + 1 * D];
      scalar_t qkv_bias_v = qkv_bias[d + 2 * D];
      scalar_t qkv_q = qkv[b][t][d + 0 * D];
      scalar_t qkv_k = qkv[b][t][d + 1 * D];
      scalar_t qkv_v = qkv[b][t][d + 2 * D];
      qkv_q = static_cast<scalar_t>(
          (static_cast<accscalar_t>(qkv_q) +
           static_cast<accscalar_t>(qkv_bias_q)) *
          static_cast<accscalar_t>(inv_sqrt_dim_per_head));
      qkv_k = static_cast<scalar_t>(
          (static_cast<accscalar_t>(qkv_k) +
           static_cast<accscalar_t>(qkv_bias_k)));
      qkv_v = static_cast<scalar_t>(
          (static_cast<accscalar_t>(qkv_v) +
           static_cast<accscalar_t>(qkv_bias_v)));

      q_k_v[0][b][nh][t][dh] = qkv_q;
      q_k_v[1][b][nh][t][dh] = qkv_k;
      q_k_v[2][b][nh][t][dh] = qkv_v;
    }
  }
}

template <typename scalar_t, typename accscalar_t, bool assume_aligned = false>
__global__ void transform_bias_rescale_qkv_add_padding_kernel(
    // [B, T, 3 * D], but it's a NestedTensor buffer
    const PackedTensorAccessor64<scalar_t, 1, RestrictPtrTraits> qkv,
    // [3 * D]
    const PackedTensorAccessor64<scalar_t, 1, RestrictPtrTraits> qkv_bias,
    const int* offsets,
    const int* input_sizes,
    // [3, B, NH, T, DH]
    PackedTensorAccessor64<scalar_t, 5, RestrictPtrTraits> q_k_v,
    const scalar_t inv_sqrt_dim_per_head) {
  // warp per DH.
  // so launch B * NH * T warps.
  const auto NH = q_k_v.size(2);
  const auto T = q_k_v.size(3);
  const auto DH = q_k_v.size(4);

  const auto t = blockIdx.x % T;
  const auto b = blockIdx.x / T;

  const auto D = NH * DH;
  const auto _3D = 3 * D;

  const auto offset_for_batch = offsets[b];
  const auto input_dim = 1;
  const auto* sizes_i = input_sizes + b * input_dim;
  if (assume_aligned) {
    constexpr int VEC = TRANSFORM_BIAS_RESCALE_VEC;
    using LoadT = memory::aligned_vector<scalar_t, VEC>;
    for (int32_t d_v = threadIdx.x; d_v < D / VEC; d_v += blockDim.x) {
      auto d = d_v * VEC;
      auto nh = d / DH;
      auto dh = d % DH;
      scalar_t qkv_bias_q[VEC];
      scalar_t qkv_bias_k[VEC];
      scalar_t qkv_bias_v[VEC];
      scalar_t qkv_q[VEC];
      scalar_t qkv_k[VEC];
      scalar_t qkv_v[VEC];

      const auto first_item_offset = t * _3D + d;
      const auto last_item_offset = first_item_offset + VEC - 1;
      const bool first_item_in_bounds = first_item_offset < sizes_i[0];
      const bool entire_vec_in_bounds = last_item_offset < sizes_i[0];

      // Here we require D % VEC == 0 for these vectorized loads.
      *reinterpret_cast<LoadT*>(&qkv_bias_q) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 0 * D]);
      *reinterpret_cast<LoadT*>(&qkv_bias_k) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 1 * D]);
      *reinterpret_cast<LoadT*>(&qkv_bias_v) =
          *reinterpret_cast<const LoadT*>(&qkv_bias[d + 2 * D]);

      if (entire_vec_in_bounds) {
        const auto offset = offset_for_batch + first_item_offset;
        *reinterpret_cast<LoadT*>(&qkv_q) =
            *reinterpret_cast<const LoadT*>(&qkv[offset + 0 * D]);
        *reinterpret_cast<LoadT*>(&qkv_k) =
            *reinterpret_cast<const LoadT*>(&qkv[offset + 1 * D]);
        *reinterpret_cast<LoadT*>(&qkv_v) =
            *reinterpret_cast<const LoadT*>(&qkv[offset + 2 * D]);
#pragma unroll
        // TODO: specialize for float2half2/half2float2?
        for (auto ii = 0; ii < VEC; ++ii) {
          qkv_q[ii] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_q[ii]) +
               static_cast<accscalar_t>(qkv_bias_q[ii])) *
              static_cast<accscalar_t>(inv_sqrt_dim_per_head));
          qkv_k[ii] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_k[ii]) +
               static_cast<accscalar_t>(qkv_bias_k[ii])));
          qkv_v[ii] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_v[ii]) +
               static_cast<accscalar_t>(qkv_bias_v[ii])));
        }
      } else if (first_item_in_bounds) {
        const auto offset = offset_for_batch + first_item_offset;
        qkv_q[0] = qkv[offset + 0 * D];
        qkv_k[0] = qkv[offset + 1 * D];
        qkv_v[0] = qkv[offset + 2 * D];
        qkv_q[0] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_q[0]) +
               static_cast<accscalar_t>(qkv_bias_q[0])) *
              static_cast<accscalar_t>(inv_sqrt_dim_per_head));
        qkv_k[0] = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_k[0]) +
               static_cast<accscalar_t>(qkv_bias_k[0])));
          qkv_v[0] = static_cast<scalar_t>(
              (static_cast<accscalar_t>(qkv_v[0]) +
               static_cast<accscalar_t>(qkv_bias_v[0])));
#pragma unroll
        for (auto ii = 1; ii < VEC; ++ii) {
          const auto loop_offset = offset + ii;
          if (loop_offset < sizes_i[0]) {
            qkv_q[ii] = qkv[loop_offset + 0 * D];
            qkv_k[ii] = qkv[loop_offset + 1 * D];
            qkv_v[ii] = qkv[loop_offset + 2 * D];
            qkv_q[ii] = static_cast<scalar_t>(
                (static_cast<accscalar_t>(qkv_q[ii]) +
                 static_cast<accscalar_t>(qkv_bias_q[ii])) *
                static_cast<accscalar_t>(inv_sqrt_dim_per_head));
            qkv_k[ii] = static_cast<scalar_t>(
                (static_cast<accscalar_t>(qkv_k[ii]) +
                 static_cast<accscalar_t>(qkv_bias_k[ii])));
            qkv_v[ii] = static_cast<scalar_t>(
                (static_cast<accscalar_t>(qkv_v[ii]) +
                 static_cast<accscalar_t>(qkv_bias_v[ii])));
          } else {
            qkv_q[ii] = 0;
            qkv_k[ii] = 0;
            qkv_v[ii] = 0;
          }
        }
      } else {
#pragma unroll
        for (auto ii = 0; ii < VEC; ++ii) {
          qkv_q[ii] = 0;
          qkv_k[ii] = 0;
          qkv_v[ii] = 0;
        }
      }

      // Here we require DH % VEC == 0 for these vectorized stores.
      *reinterpret_cast<LoadT*>(&q_k_v[0][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_q);
      *reinterpret_cast<LoadT*>(&q_k_v[1][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_k);
      *reinterpret_cast<LoadT*>(&q_k_v[2][b][nh][t][dh]) =
          *reinterpret_cast<const LoadT*>(&qkv_v);
    }
  } else {
    for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
      auto nh = d / DH;
      auto dh = d % DH;
      scalar_t qkv_bias_q = qkv_bias[d + 0 * D];
      scalar_t qkv_bias_k = qkv_bias[d + 1 * D];
      scalar_t qkv_bias_v = qkv_bias[d + 2 * D];

      const auto item_offset = t * _3D + d;
      const bool in_bounds = item_offset < sizes_i[0];
      scalar_t qkv_q, qkv_k, qkv_v;
      if (in_bounds) {
        const auto qkv_offset = offset_for_batch + item_offset;
        qkv_q = qkv[qkv_offset + 0 * D];
        qkv_k = qkv[qkv_offset + 1 * D];
        qkv_v = qkv[qkv_offset + 2 * D];
        qkv_q = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_q) +
             static_cast<accscalar_t>(qkv_bias_q)) *
            static_cast<accscalar_t>(inv_sqrt_dim_per_head));
        qkv_k = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_k) +
             static_cast<accscalar_t>(qkv_bias_k)));
        qkv_v = static_cast<scalar_t>(
            (static_cast<accscalar_t>(qkv_v) +
             static_cast<accscalar_t>(qkv_bias_v)));
      } else {
        qkv_q = 0;
        qkv_k = 0;
        qkv_v = 0;
      }

      q_k_v[0][b][nh][t][dh] = qkv_q;
      q_k_v[1][b][nh][t][dh] = qkv_k;
      q_k_v[2][b][nh][t][dh] = qkv_v;
    }
  }
}

Tensor collapse_dims_1_and_2(const Tensor& sizes) {
  auto sizes_dim1 = at::native::narrow_symint(sizes, 1, 0, 1);
  auto sizes_dim2 = at::native::narrow_symint(sizes, 1, 1, 1);

  return (sizes_dim1 * sizes_dim2).contiguous();
}

} // namespace
// compute q = (q + q_bias) / sqrt(dim_per_head), k = k + k_bias, v = v + v_bias
__host__ std::tuple<Tensor, Tensor, Tensor> transform_bias_rescale_qkv_cuda(
    const Tensor& qkv,
    const Tensor& qkv_bias,
    const int64_t num_head) {
  auto B = qkv.is_nested()
      ? get_nested_tensor_impl(qkv)->get_nested_sizes().size(0)
      : qkv.size(0);
  // TODO: calculate this without the std::vector -- NestedTensor_to_mask wants
  // this too
  auto T = qkv.is_nested()
      ? NestedTensor_get_max_size(*get_nested_tensor_impl(qkv))[0]
      : qkv.size(1);
  if (qkv.is_nested()) {
    // Don't mess with non-nested case for now since it's not set up to fiddle
    // with mask size.

    // Round T up to next multiple of 8 so as to be able to utilize Tensor
    // cores. Otherwise, sometimes with padding, *no* row will have the maximum
    // sequence length and so we'll have a non-divisible-by-8 dimension even if
    // the model author chose a multiple of 8.
    T = T + (8 - (T % 8)) % 8;
  }
  auto _3D = qkv_bias.size(0);
  auto D = _3D / 3;
  TORCH_CHECK(D % num_head == 0);
  const auto dim_per_head = D / num_head;
  auto q_k_v = at::empty({3, B, num_head, T, dim_per_head}, qkv_bias.options());
#define CALL_KERNEL(assume_aligned)                                        \
  transform_bias_rescale_qkv_kernel<scalar_t, accscalar_t, assume_aligned> \
      <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(          \
          qkv.packed_accessor64<scalar_t, 3, RestrictPtrTraits>(),         \
          qkv_bias.packed_accessor64<scalar_t, 1, RestrictPtrTraits>(),    \
          q_k_v.packed_accessor64<scalar_t, 5, RestrictPtrTraits>(),       \
          1.0 / std::sqrt(static_cast<scalar_t>(dim_per_head)))
#define CALL_ADD_PADDING_KERNEL(assume_aligned)                         \
  transform_bias_rescale_qkv_add_padding_kernel<                        \
      scalar_t,                                                         \
      accscalar_t,                                                      \
      assume_aligned>                                                   \
      <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(       \
          nt_qkv_buffer                                          \
              .packed_accessor64<scalar_t, 1, RestrictPtrTraits>(),     \
          qkv_bias.packed_accessor64<scalar_t, 1, RestrictPtrTraits>(), \
          offsets_ptr,                                                  \
          sizes_ptr,                                                    \
          q_k_v.packed_accessor64<scalar_t, 5, RestrictPtrTraits>(),    \
          1.0 / std::sqrt(static_cast<scalar_t>(dim_per_head)))

  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      qkv.scalar_type(),
      "transform_bias_rescale_qkv",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        auto threads = std::max(
            std::min<int32_t>(1024, D / TRANSFORM_BIAS_RESCALE_VEC), 1);
        auto blocks = B * T;
        const bool aligned =
            ((dim_per_head % TRANSFORM_BIAS_RESCALE_VEC) == 0) &&
            ((reinterpret_cast<intptr_t>(qkv_bias.data_ptr()) %
              TRANSFORM_BIAS_RESCALE_VEC) == 0);
        if (aligned) {
          TORCH_INTERNAL_ASSERT_DEBUG_ONLY(
              D % TRANSFORM_BIAS_RESCALE_VEC == 0,
              "D = num_heads * dim_per_head, so we should have dim_per_head % "
              "TRANSFORM_BIAS_RESCALE_VEC == 0 => "
              "D % TRANSFORM_BIAS_RESCALE_VEC == 0");
        }
        if (qkv.is_nested()) {
          auto* nt_qkv = get_nested_tensor_impl(qkv);
          const at::Tensor& nt_qkv_buffer = nt_qkv->get_buffer();
          auto sizes = collapse_dims_1_and_2(nt_qkv->get_nested_sizes());
          auto offsets =
              NestedTensor_batch_offsets_from_size_tensor(sizes, sizes.numel());
          at::native::narrow_symint(offsets, 0, sizes.numel() + 1, sizes.numel())
              .copy_(sizes.reshape({-1}));
          auto metadata = offsets.to(at::Device(kCUDA), at::kInt, true, true);
          const auto offsets_ptr = metadata.data_ptr<int>();
          const auto sizes_ptr = offsets_ptr + sizes.numel() + 1;
          const auto input_dim = sizes.sizes()[1];
          TORCH_INTERNAL_ASSERT_DEBUG_ONLY(input_dim == 1);
          if (aligned &&
              ((reinterpret_cast<intptr_t>(qkv.data_ptr()) %
                TRANSFORM_BIAS_RESCALE_VEC) == 0)) {
            CALL_ADD_PADDING_KERNEL(true);
          } else {
            CALL_ADD_PADDING_KERNEL(false);
          }
        } else if (aligned) {
          CALL_KERNEL(true);
        } else {
          CALL_KERNEL(false);
        }
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
#undef CALL_ADD_PADDING_KERNEL
#undef CALL_KERNEL
  auto q_k_v_s =
      at::native::split(q_k_v.view({3 * B, num_head, T, dim_per_head}), B, 0);
  return std::make_tuple(q_k_v_s[0], q_k_v_s[1], q_k_v_s[2]);
}

std::tuple<Tensor, Tensor> native_multi_head_attention_cuda(
    const Tensor& query,
    const Tensor& key,
    const Tensor& value,
    const int64_t embed_dim,
    const int64_t num_head,
    const Tensor& qkv_weight,
    const Tensor& qkv_bias,
    const Tensor& proj_weight,
    const Tensor& proj_bias,
    const std::optional<Tensor>& mask,
    bool need_weights,
    bool average_attn_weights,
    const std::optional<int64_t> mask_type) {
  // query shape: [B, T, D]
  // qkv_weight shape: [3 * D, D]

  TORCH_CHECK(
      !mask || !query.is_nested(),
      "NestedTensor with mask is not supported yet");
  const auto D = embed_dim;
  TORCH_CHECK(
      query.dim() == 3,
      "expected 3-D `query`, got ",
      query.dim(),
      "-D tensor");
  TORCH_CHECK(
      query.is_nested() || query.sizes()[2] == embed_dim,
      "passed-in embed_dim ",
      embed_dim,
      " didn't match last dim of query ",
      query.sizes()[2]);
  TORCH_CHECK(
      key.dim() == 3,
      "expected 3-D `key`, got ",
      key.dim(),
      "-D tensor");
  TORCH_CHECK(
      value.dim() == 3,
      "expected 3-D `value`, got ",
      value.dim(),
      "-D tensor");
  TORCH_CHECK(
      query.is_nested() || key.is_nested() || value.is_nested() ||
          (query.sizes() == key.sizes() && key.sizes() == value.sizes()),
      "expected `query`/`key`/`value` shapes to match");
  TORCH_CHECK(
      qkv_weight.dim() == 2,
      "expected 2-D `qkv_weight`, got ",
      qkv_weight.dim(),
      "-D tensor");
  TORCH_CHECK(
      D * 3 == qkv_weight.sizes()[0],
      "expected `qkv_weight` first dim to be 3x embed_dim");
  TORCH_CHECK(
      D == qkv_weight.sizes()[1],
      "expected `qkv_weight` second dim to be embed_Dim");
  TORCH_CHECK(
      qkv_bias.dim() == 1,
      "expected 1-D `qkv_bias`, got ",
      qkv_bias.dim(),
      "-D tensor");
  TORCH_CHECK(
      qkv_bias.sizes()[0] == 3 * D,
      "expected `qkv_bias` first dim and first dim of query to be equal");
  TORCH_CHECK(D % num_head == 0, "`embed_dim` must divide evenly by `num_heads`");

#ifndef NDEBUG
  const auto B = query.is_nested()
      ? get_nested_tensor_impl(query)->get_nested_sizes().size(0)
      : query.sizes()[0];
  auto T = query.is_nested() ? 0 : query.sizes()[1];

#endif
  const auto dim_per_head = D / num_head;
  if ((query.is_same(key) && key.is_same(value)) && dim_per_head % 8 == 0 && !need_weights) {

    // We have not done linear projection yet but the input for SDP
    // Is expected to be 4 dimensional. We "cheaply" create view tensors
    // That will then be used for checking hot path conditions with select_sd_backend
    auto q = query.view({query.size(0), -1, num_head, dim_per_head}).transpose(1, 2);
    auto k = key.view({key.size(0), -1, num_head, dim_per_head}).transpose(1, 2);
    auto v = value.view({value.size(0), -1, num_head, dim_per_head}).transpose(1, 2);

    sdp::sdp_params kernel_params{q, k, v, mask, 0.0, false, false};
    auto backend = select_sdp_backend(kernel_params);
    // strides from packed projection for nested tensors when seq_len is 1 will be
    // and will trigger a contiguous call in the kernel, so we prevent this
    bool no_seq_len_1_nested = query.is_nested() ? check_for_seq_len_1_nested_tensor(kernel_params, false) : true;
    // The API for transformer_encoder is a mask of shape (Batch_Size, Seq_len_q)
    // For mem-eff attention this will cause the expand call to error
    // For now I am going to turn of that path not have to deal with all the annoying
    // Mask type shape grossness
    if (!mask.has_value() && no_seq_len_1_nested &&
        (backend == sdp::SDPBackend::flash_attention || backend == sdp::SDPBackend::efficient_attention ||
         backend == sdp::SDPBackend::cudnn_attention)) {
      auto x = at::linear(query, qkv_weight, qkv_bias);
      auto chunks = x.chunk(3, -1);
      auto x_size_0 = x.size(0);

      chunks[0] = (chunks[0].view({x_size_0, -1, num_head, dim_per_head}))
                      .transpose(1, 2);
      chunks[1] = (chunks[1].view({x_size_0, -1, num_head, dim_per_head}))
                      .transpose(1, 2);
      chunks[2] = (chunks[2].view({x_size_0, -1, num_head, dim_per_head}))
                      .transpose(1, 2);
      auto y = at::scaled_dot_product_attention(
          chunks[0], chunks[1], chunks[2], mask, 0.0, false, std::nullopt);

      auto past_sdp = y.transpose(1, 2).reshape({x_size_0, -1, embed_dim});
      return std::make_tuple(
          at::linear(past_sdp, proj_weight, proj_bias), Tensor());
    }
    // Returned math or error lets not use it
  }

  // shape: [B, T, 3 x D]
  auto qkv = qkv_projection(query, key, value, embed_dim, qkv_weight);

  if (!qkv.is_nested() && qkv.numel() == 0) {
    if (query.is_nested()) {
      return std::make_tuple(Tensor(), Tensor());
    }
    return std::make_tuple(at::empty_like(query), Tensor());
  }

#ifndef NDEBUG
  if (!query.is_nested() || !qkv.is_nested()) {
    if (query.is_nested()) {
      T = qkv.size(1);
    }
    debug_assert_shape(__LINE__, qkv, {B, T, 3 * D});
  }
#endif

#ifdef DEBUG_PRINT_EACH_STEP
  if (!qkv.is_nested()) {
    std::cerr << "qkv: " << qkv << std::endl;
  }
#endif
  // shape: 3 x [B, num_head, T, dim_per_head]
  auto [q, k, v] = _transform_bias_rescale_qkv(qkv, qkv_bias, num_head);
  qkv = Tensor(); // Not used any more, allow free
#ifndef NDEBUG
  debug_assert_shape(__LINE__, q, {B, num_head, T, dim_per_head});
  debug_assert_shape(__LINE__, k, {B, num_head, T, dim_per_head});
  debug_assert_shape(__LINE__, v, {B, num_head, T, dim_per_head});
#endif
#ifdef DEBUG_PRINT_EACH_STEP
  std::cerr << "q: " << q << std::endl;
  std::cerr << "k: " << k << std::endl;
  std::cerr << "v: " << v << std::endl;
#endif

  // shape: [B, num_head, T, T]
  auto qkt = bmm_nt(q, k);
  // q & k are dead but cannot be freed because they were packed with v
#ifndef NDEBUG
  debug_assert_shape(__LINE__, qkt, {B, num_head, T, T});
#endif
#ifdef DEBUG_PRINT_EACH_STEP
  std::cerr << "qkt: " << qkt << std::endl;
#endif

  // shape: [B, num_head, T, T]
  // TODO: long-term, have a kernel that works with
  // NestedTensor directly if there is no mask passed
  qkt = masked_softmax(qkt, mask, query, mask_type);
#ifdef DEBUG_PRINT_EACH_STEP
  std::cerr << "qkt after softmax: " << qkt << std::endl;
#endif

  // shape: [B, num_head, T, dim_per_head]
  // reuse storage for q; we're done with it
  auto attn_ctx = bmm_nn(q, qkt, v);
  // qkv is not dead; we just reused storage for q!
  if (!need_weights) {
    qkt = Tensor();
  }
#ifndef NDEBUG
  debug_assert_shape(__LINE__, attn_ctx, {B, num_head, T, dim_per_head});
#endif
#ifdef DEBUG_PRINT_EACH_STEP
  std::cerr << "attn_ctx: " << attn_ctx << std::endl;
#endif

  // shape: [B, T, D]
  // Fuse transform_0213 inside
  auto proj = transform0213_gemm_nt_bias(
      attn_ctx, proj_weight, proj_bias, query);
#ifndef NDEBUG
  debug_assert_shape(__LINE__, proj, {B, T, D});
#endif
  if (need_weights && average_attn_weights) {
    // weights are not needed for full transformer, so don't worry too
    // much about performance -- we implement this just to make use
    // cases that don't disable need_weights still get some speedup.
    qkt = qkt.sum(1);
    qkt /= num_head;
  }
  return std::make_tuple(std::move(proj), std::move(qkt));
}
std::tuple<Tensor, Tensor, Tensor, Tensor, c10::SymInt, c10::SymInt, Tensor, Tensor, Tensor> _scaled_dot_product_flash_attention_cuda(
    const Tensor& query,
    const Tensor& key,
    const Tensor& value,
    double dropout_p,
    bool is_causal,
    bool return_debug_mask,
    std::optional<double> scale) {
  // Used for tracking usage statistics
  C10_LOG_API_USAGE_ONCE("torch.sdpa.flash_attention");
  // Query (Batch x Num_heads x Q_seq_len  x Dim_per_head)
  // Key   (Batch x Num_heads x KV_seq_len x Dim_per_head)
  // Value (Batch x Num_heads x KV_seq_len x Dim_per_head)

  const int64_t max_seqlen_batch_q = query.size(2);
  const int64_t max_seqlen_batch_k = key.size(2);
  const int64_t max_seqlen_batch_v = value.size(2);
  TORCH_CHECK(
      max_seqlen_batch_k == max_seqlen_batch_v,
      "Key and Value must have the same sequence length");

  // Query -> Query(Batch x Q_seq_len  x Num_heads x Dim_per_head)
  // Key   -> Key  (Batch x KV_seq_len x Num_heads x Dim_per_head)
  // Value -> Value(Batch x KV_seq_len x Num_heads x Dim_per_head)
  Tensor q_t = query.transpose(1, 2);
  Tensor k_t = key.transpose(1, 2);
  Tensor v_t = value.transpose(1, 2);

  auto
      [output,
       logsumexp,
       philox_seed,
       philox_offset,
       debug_attn_mask] =
          at::_flash_attention_forward(
              q_t,
              k_t,
              v_t,
              std::nullopt,
              std::nullopt,
              max_seqlen_batch_q,
              max_seqlen_batch_k,
              dropout_p,
              is_causal,
              return_debug_mask,
              scale,
              std::nullopt,
              std::nullopt);
  // Reshape output to convert nnz to batch_size and seq_len
  Tensor attention = output.transpose(1,2);

  return std::make_tuple(attention, logsumexp, Tensor(), Tensor(), max_seqlen_batch_q, max_seqlen_batch_k, philox_seed, philox_offset, debug_attn_mask);
}

std::tuple<Tensor, Tensor, Tensor, Tensor, c10::SymInt, c10::SymInt, Tensor, Tensor, Tensor> _cudnn_attention_forward(
    const Tensor& query,
    const Tensor& key,
    const Tensor& value,
    const std::optional<Tensor>& attn_bias,
    const std::optional<Tensor>& cumulative_sequence_length_q,
    const std::optional<Tensor>& cumulative_sequence_length_kv,
    int64_t max_seqlen_batch_q,
    int64_t max_seqlen_batch_kv,
    bool compute_logsumexp,
    double dropout_p,
    bool is_causal,
    bool return_debug_mask,
    std::optional<double> scale) {
  // TODO(eqy): debug mask support
  // Query (Batch x Num_heads x Q_seq_len  x Dim_per_head)
  // Key   (Batch x Num_heads x KV_seq_len x Dim_per_head)
  // Value (Batch x Num_heads x KV_seq_len x Dim_per_head)
  const bool is_nested = cumulative_sequence_length_q.has_value();
  if (!is_nested) {
    const int64_t batch_size = query.size(0);
    const int64_t num_heads = query.size(1);
    const int64_t head_dim_qk = query.size(3);
    const int64_t head_dim_v = value.size(3);
    auto attn_bias_ = attn_bias;
    if (attn_bias_.has_value()) {
      const auto bias_dim = attn_bias_.value().dim();
      if (bias_dim == 2) {
        attn_bias_ = attn_bias_.value().expand({batch_size, 1, max_seqlen_batch_q, max_seqlen_batch_kv});
      } else if (bias_dim == 3) {
        attn_bias_ = attn_bias_.value().expand({batch_size, 1, max_seqlen_batch_q, max_seqlen_batch_kv});
      } else {
        TORCH_CHECK(bias_dim == 4, "cuDNN SDPA expects either a 2D, 3D, or 4D attn_bias but got ", attn_bias_.value().dim(), "D");
        attn_bias_ = attn_bias_.value().expand({batch_size, attn_bias_.value().size(1), max_seqlen_batch_q, max_seqlen_batch_kv});
      }
    }

    Tensor attention, log_sumexp;
    at::Tensor cudnn_seed, cudnn_offset;
    cudnn_seed = at::empty({}, at::dtype(at::kLong).device(at::kCUDA));
    cudnn_offset = at::empty({}, at::dtype(at::kLong).device(at::kCUDA));

    const bool use_dropout = std::fpclassify(dropout_p) != FP_ZERO;

    // See Note [Seed and Offset Device] in _efficient_attention_forward
    at::PhiloxCudaState philox_state;
    const bool in_capture_stream =
        at::cuda::currentStreamCaptureStatus() != at::cuda::CaptureStatus::None;
    if (use_dropout) {
      // Device
      auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
          std::nullopt, at::cuda::detail::getDefaultCUDAGenerator());

      // See Note [Acquire lock when using random generators]
      std::lock_guard<std::mutex> lock(gen->mutex_);
      // if using dropout, we produce 1 random number for each element of the
      // attention tensor
      // TODO(eqy): should state be advanced per thread (local) amount or per call/launch (global) amount
      philox_state = gen->philox_cuda_state(batch_size * num_heads * max_seqlen_batch_q * max_seqlen_batch_kv);
      at::cuda::philox::unpack_cudnn_wrapper(
                                        philox_state, static_cast<int64_t*>(cudnn_seed.data_ptr()), static_cast<int64_t*>(cudnn_offset.data_ptr()), at::cuda::getCurrentCUDAStream());
    }

    const auto softmax_scale = sdp::calculate_scale(query, scale).expect_float();
    Tensor debugmask;

    run_cudnn_SDP_fprop(batch_size/*int64_t b*/,
                        num_heads/*int64_t h*/,
                        max_seqlen_batch_q/*int64_t s_q*/,
                        max_seqlen_batch_kv/*int64_t s_kv*/,
                        head_dim_qk/*int64_t d_qk*/,
                        head_dim_v/*int64_t d_v*/,
                        softmax_scale/*float scaling_factor*/,
                        compute_logsumexp/* bool */,
                        is_causal/* bool */,
                        dropout_p/*double dropout_probability*/,
                        query/* Tensor q*/,
                        key/* Tensor k*/,
                        value/* Tensor v*/,
                        attn_bias_ /* std::optional<Tensor> */,
                        log_sumexp/*Tensor softmaxstats*/,
                        attention/*Tensor o*/,
                        cudnn_seed/*Tensor dropoutseed*/,
                        cudnn_offset/*Tensor dropoutoffset*/);

    // TODO(eqy): support debug_attn_mask
    return std::make_tuple(std::move(attention), std::move(log_sumexp), Tensor(), Tensor(), max_seqlen_batch_q, max_seqlen_batch_kv, std::move(cudnn_seed), std::move(cudnn_offset), Tensor());
  } else {
    //auto [
    //    query_buffer_reshaped,
    //    key_buffer_reshaped,
    //    value_buffer_reshaped,
    //    cumulative_sequence_length_q,
    //    cumulative_sequence_length_kv,
    //    max_seqlen_batch_q,
    //    max_seqlen_batch_kv,
    //    output_shape] = preprocessing::sdpa_nested_preprocessing(query, key, value);
    // C10_LOG_API_USAGE_ONCE("torch.sdpa.flash_attention_cudnn");
    // TODO(eqy): debug mask support
    // BHSD ...
    const int64_t batch_size = cumulative_sequence_length_q.value().size(0) - 1;
    const int64_t num_heads_q = query.size(-2);
    const int64_t num_heads_k = key.size(-2);
    const int64_t num_heads_v = value.size(-2);
    const int64_t head_dim_qk = query.size(-1);
    const int64_t head_dim_v = value.size(-1);
    auto attn_bias_ = attn_bias;
    if (attn_bias_.has_value()) {
      const auto bias_dim = attn_bias_.value().dim();
      if (bias_dim == 2) {
        attn_bias_ = attn_bias_.value().expand({batch_size, 1, max_seqlen_batch_q, max_seqlen_batch_kv});
      } else if (bias_dim == 3) {
        attn_bias_ = attn_bias_.value().expand({batch_size, 1, max_seqlen_batch_q, max_seqlen_batch_kv});
      } else {
        attn_bias_ = attn_bias_.value().expand({batch_size, attn_bias_.value().size(1), max_seqlen_batch_q, max_seqlen_batch_kv});
        TORCH_CHECK(bias_dim == 4, "cuDNN SDPA expects either a 2D, 3D, or 4D attn_bias but got ", attn_bias_.value().dim(), "D");
      }
    }

    Tensor attention, log_sumexp;

    at::Tensor cudnn_seed, cudnn_offset;
    cudnn_seed = at::empty({}, at::dtype(at::kLong).device(at::kCUDA));
    cudnn_offset = at::empty({}, at::dtype(at::kLong).device(at::kCUDA));

    const bool use_dropout = std::fpclassify(dropout_p) != FP_ZERO;

    // See Note [Seed and Offset Device] in _efficient_attention_forward
    at::PhiloxCudaState philox_state;
    const bool in_capture_stream =
        at::cuda::currentStreamCaptureStatus() != at::cuda::CaptureStatus::None;
    if (use_dropout) {
      // Device
      auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
          std::nullopt, at::cuda::detail::getDefaultCUDAGenerator());

      // See Note [Acquire lock when using random generators]
      std::lock_guard<std::mutex> lock(gen->mutex_);
      // if using dropout, we produce 1 random number for each element of the
      // attention tensor
      // TODO(eqy): should state be advanced per thread (local) amount or per call/launch (global) amount
      philox_state = gen->philox_cuda_state(batch_size * num_heads_q * max_seqlen_batch_q * max_seqlen_batch_kv);
      at::cuda::philox::unpack_cudnn_wrapper(philox_state, static_cast<int64_t*>(cudnn_seed.data_ptr()), static_cast<int64_t*>(cudnn_offset.data_ptr()), at::cuda::getCurrentCUDAStream());
    }

    const auto softmax_scale = sdp::calculate_scale(query, scale).as_float_unchecked();

    run_cudnn_SDP_fprop_nestedtensor(batch_size/*int64_t b*/,
                                     num_heads_q/*int64_t h*/,
                                     num_heads_k,
                                     num_heads_v,
                                     max_seqlen_batch_q/*int64_t s_q*/,
                                     max_seqlen_batch_kv/*int64_t s_kv*/,
                                     head_dim_qk/*int64_t d_qk*/,
                                     head_dim_v/*int64_t d_v*/,
                                     softmax_scale/*float scaling_factor*/,
                                     compute_logsumexp/* bool */,
                                     is_causal/* bool */,
                                     dropout_p/*double dropout_probability*/,
                                     cumulative_sequence_length_q.value(),
                                     cumulative_sequence_length_kv.value(),
                                     query/* Tensor q*/,
                                     key/* Tensor k*/,
                                     value/* Tensor v*/,
                                     attn_bias_ /* std::optional<Tensor> */,
                                     log_sumexp/*Tensor softmaxstats*/,
                                     attention/*Tensor o*/,
                                     cudnn_seed/*Tensor dropoutseed*/,
                                     cudnn_offset/*Tensor dropoutoffset*/);
    //attention = wrap_buffer(attention.view(-1), output_shape).transpose(1, 2);
    return std::make_tuple(std::move(attention), std::move(log_sumexp), cumulative_sequence_length_q.value(), cumulative_sequence_length_kv.value(), max_seqlen_batch_q, max_seqlen_batch_kv, std::move(cudnn_seed), std::move(cudnn_offset), Tensor());
  }
}

std::tuple<Tensor, Tensor, Tensor, Tensor, c10::SymInt, c10::SymInt, Tensor, Tensor, Tensor> _scaled_dot_product_cudnn_attention_cuda(
    const Tensor& query,
    const Tensor& key,
    const Tensor& value,
    const std::optional<Tensor>& attn_bias,
    bool compute_logsumexp,
    double dropout_p,
    bool is_causal,
    bool return_debug_mask,
    std::optional<double> scale) {
  // Used for tracking usage statistics
  C10_LOG_API_USAGE_ONCE("torch.sdpa.flash_attention_cudnn");
  const int64_t max_seqlen_batch_q = query.size(2);
  const int64_t max_seqlen_batch_k = key.size(2);

  return at::_cudnn_attention_forward(query, key, value, attn_bias, std::nullopt, std::nullopt, max_seqlen_batch_q, max_seqlen_batch_k, compute_logsumexp, dropout_p, is_causal, return_debug_mask, scale);
}

std::tuple<Tensor, Tensor, Tensor, Tensor> _scaled_dot_product_efficient_attention_cuda(
    const Tensor& query,
    const Tensor& key,
    const Tensor& value,
    const std::optional<at::Tensor>& attn_bias,
    bool compute_log_sumexp,
    double dropout_p,
    bool is_causal,
    std::optional<double> scale) {
  // Used for tracking usage statistics
  C10_LOG_API_USAGE_ONCE("torch.sdpa.mem_efficient_attention");
  // Query -> Query(Batch x Q_seq_len x Num_heads x Dim_per_head)
  // Key   -> Key(Batch x KV_seq_len x Num_heads x Dim_per_head)
  // Value -> Value(Batch x KV_seq_len x  Num_heads x Dim_per_head)
  Tensor q_t = query.transpose(1, 2);
  Tensor k_t = key.transpose(1, 2);
  Tensor v_t = value.transpose(1, 2);

  sdp::CustomMaskType custom_mask_type = is_causal
      ? sdp::CustomMaskType::CausalFromTopLeft
      : sdp::CustomMaskType::NoCustomMask;

  auto [attention, log_sumexp, seed, offset, max_seqlen_batch_q, max_seqlen_batch_kv] = at::_efficient_attention_forward(
      q_t,
      k_t,
      v_t,
      attn_bias,
      std::nullopt,
      std::nullopt,
      std::nullopt,
      std::nullopt,
      dropout_p,
      static_cast<int64_t>(custom_mask_type),
      compute_log_sumexp,
      scale);

  attention = attention.transpose(1, 2);
  return std::make_tuple(std::move(attention), std::move(log_sumexp), std::move(seed), std::move(offset));
}

int64_t _fused_sdp_choice_cuda(const Tensor& query_, const Tensor& key, const Tensor& value,
        const std::optional<Tensor>& attn_mask_, double dropout_p, bool is_causal, std::optional<double> scale, bool enable_gqa){
  sdp::sdp_params kernel_params{query_, key, value, attn_mask_, dropout_p, is_causal, enable_gqa};
  auto backend = select_sdp_backend(kernel_params);
  if (backend == sdp::SDPBackend::error) {
    TORCH_CHECK(
        false,
        "No viable backend for scaled_dot_product_attention was found. ",
        "This is likely due to turning off both the math kernel and the fused kernels.");
  }
  return static_cast<int64_t>(backend);
}

std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor>
_flash_attention_forward(
    const Tensor& query,
    const Tensor& key,
    const Tensor& value,
    const std::optional<Tensor>& cumulative_sequence_length_q,
    const std::optional<Tensor>& cumulative_sequence_length_k,
    int64_t max_seqlen_batch_q,
    int64_t max_seqlen_batch_k,
    double dropout_p,
    bool is_causal,
    bool return_debug_mask,
    std::optional<double> scale,
    std::optional<int64_t> window_size_left,
    std::optional<int64_t> window_size_right,
    const std::optional<Tensor>& _seqused_k,
    const std::optional<Tensor>& _alibi_slopes
    ) {
#if defined(USE_FLASH_ATTENTION)
  const auto softmax_scale =
      sdp::calculate_scale(query, scale).expect_float();
  std::optional<Tensor> out = std::nullopt;

  std::optional<Tensor> seqused_k = _seqused_k;
  std::optional<at::Tensor> block_table = std::nullopt;  // we are not using the block table yet
  std::optional<Tensor> alibi_slopes = _alibi_slopes;
  const float softcap = 0.0;

  const int non_null_window_left = window_size_left.has_value() ? window_size_left.value() : -1;
  const int non_null_window_right = window_size_right.has_value() ? window_size_right.value() : -1;

  // We are going to have two paths:
  // 1. The standard MHA path for dense tensors
  // 2. The Varseqlen path
  TORCH_CHECK(
      cumulative_sequence_length_q.has_value() ==
          cumulative_sequence_length_k.has_value(),
      "cumulative_sequence_length_q and cumulative_sequence_length_k must be both set or both not set");
  Tensor output, q_padded, k_padded, v_padded, logsumexp, output_shape,
      philox_seed, philox_offset, debug_attn_mask;
  if (cumulative_sequence_length_q.has_value()) {
    std::tie(
        output,
        q_padded,
        k_padded,
        v_padded,
        logsumexp,
        philox_seed,
        philox_offset,
        debug_attn_mask) =
        FLASH_NAMESPACE::mha_varlen_fwd(
            query,
            key,
            value,
            out,
            cumulative_sequence_length_q.value(),
            cumulative_sequence_length_k.value(),
            seqused_k, /*seqused_k*/
            block_table, /*block_table*/
            alibi_slopes, /*alibi_slopes*/
            max_seqlen_batch_q,
            max_seqlen_batch_k,
            dropout_p,
            softmax_scale,
            false /*zero_tensors*/,
            is_causal,
            non_null_window_left,
            non_null_window_right,
            softcap,
            return_debug_mask,
            std::nullopt /*gen_*/);
  } else {
    std::tie(
        output,
        q_padded,
        k_padded,
        v_padded,
        logsumexp,
        philox_seed,
        philox_offset,
        debug_attn_mask) =
        FLASH_NAMESPACE::mha_fwd(
            query,
            key,
            value,
            out,
            alibi_slopes,
            dropout_p,
            softmax_scale,
            is_causal,
            non_null_window_left,
            non_null_window_right,
            softcap,
            return_debug_mask, /*return_softmax (this is used for testing)*/
            std::nullopt);
  }
  debug_attn_mask =
      return_debug_mask ? debug_attn_mask : at::empty({0}, query.options());
  return std::make_tuple(
      std::move(output),
      std::move(logsumexp),
      std::move(philox_seed),
      std::move(philox_offset),
      std::move(debug_attn_mask));

#endif
  TORCH_CHECK(false, "USE_FLASH_ATTENTION was not enabled for build.")
  return std::make_tuple(
      Tensor(),
      Tensor(),
      Tensor(),
      Tensor(),
      Tensor());
}

std::tuple<Tensor, Tensor, Tensor, Tensor, c10::SymInt, c10::SymInt> _efficient_attention_forward(
    const at::Tensor& query, // [b, seqlen, num_heads, K]
    const at::Tensor& key, // [b, seqlen, num_heads, K]
    const at::Tensor& value, // [b, seqlen, num_heads, Kv]
    const std::optional<at::Tensor>& bias, // [b, num_heads, seqlen, seqlen]
    // (Mode 1MHK only) [b+1]: cu_seqlens_q[b] contains the
    // position of the first query token for batch $b
    const std::optional<at::Tensor>& seqstart_q,
    // (Mode 1MHK only) [b+1]: cu_seqlen_k[b] contains the
    // position of the first key token for batch $b
    const std::optional<at::Tensor>& seqstart_k,
    // (Mode 1MHK only) Maximum sequence length across batches
    const std::optional<int64_t> max_seqlen_q_,
    const std::optional<int64_t> max_seqlen_k_,
    double dropout_p, // attention matrix dropout probability
    int64_t custom_mask_type,
    bool compute_logsumexp,
    std::optional<double> scale,
    const std::optional<at::Tensor>& seqlen_k,
    const std::optional<int64_t> window_size) {
#if defined(USE_MEM_EFF_ATTENTION)
// TODO In theory it is possible to compile with _CUDA_ARCH < 5.0 and run on a
// machine that is >= 5.0. In practice, this is not a problem but since
// this would avoid runtime architecture checks, we should look into it

  TORCH_CHECK(query.dim() == 4);
  TORCH_CHECK(key.dim() == 4);
  TORCH_CHECK(value.dim() == 4);

  // Batch sizes
  TORCH_CHECK(query.size(0) == key.size(0));
  TORCH_CHECK(query.size(0) == value.size(0));

  // Sequence length
  TORCH_CHECK(key.size(1) == value.size(1));

  // Num heads
  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(2) == value.size(2));

  // Embedding per head
  TORCH_CHECK(query.size(3) == key.size(3));

  int64_t max_seqlen_q = 0, max_seqlen_k = 0;
  TORCH_CHECK(seqstart_q.has_value() == seqstart_k.has_value());
  if (seqstart_q.has_value()) {
    TORCH_CHECK(seqstart_q->scalar_type() == at::ScalarType::Int);
    TORCH_CHECK(seqstart_k->scalar_type() == at::ScalarType::Int);
    TORCH_CHECK(seqstart_q->dim() == 1 && seqstart_k->dim() == 1);
    CHECK_NOSPARSE_CONTIGUOUS_CUDA((*seqstart_q));
    CHECK_NOSPARSE_CONTIGUOUS_CUDA((*seqstart_k));
    TORCH_CHECK(seqstart_q->size(0) == seqstart_k->size(0));
    TORCH_CHECK(query.size(0) == 1, "cu_seqlen only supports batch_size=1");
    TORCH_CHECK(max_seqlen_q_.has_value());
    max_seqlen_q = *max_seqlen_q_;
    max_seqlen_k = 0; // TODO: is this actually being set inside the kernel anywhere?
                      // see https://github.com/pytorch/pytorch/issues/115590s
  } else {
    max_seqlen_q = query.size(1);
    max_seqlen_k = key.size(1);
  }

  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(query);
  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(key);
  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(value);

  at::cuda::CUDAGuard device_guard(query.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t num_heads = query.size(-2);
  int64_t K = query.size(-1);
  int64_t Kv = value.size(-1);

  at::Tensor res;
  at::Tensor logsumexp;
  at::Tensor seed_t, offset_t;

  const bool use_dropout = std::fpclassify(dropout_p) != FP_ZERO;

  // Note [Seed and Offset Device]
  // If we are currently in graph capture mode, we need to create the seed and offset tensors on the device.
  // This is necessary for CUDA graph-safe random number generation, which requires the seed and offset tensors
  // to be single element tensors on device. During graph capture, when the seed and offset tensors are passed
  // the pointers act as scratch space for storing the RNG state for the backwards pass.
  // When calling backwards, we either construct a PhiloxState with the pointers or the actual values.
  // For more information on CUDA graph-safe RNG states, see Note [CUDA Graph-safe RNG states].

  at::PhiloxCudaState philox_state;
  const bool in_capture_stream =
      at::cuda::currentStreamCaptureStatus() != at::cuda::CaptureStatus::None;
  auto device = in_capture_stream ? at::kCUDA : at::kCPU;
  if (use_dropout) {
    auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
        std::nullopt, at::cuda::detail::getDefaultCUDAGenerator());

    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    // if using dropout, we produce 1 random number for each element of the
    // attention tensor
    philox_state = gen->philox_cuda_state(B * num_heads * M * N);

    if (in_capture_stream) {
      // The seed and offset will be populated by the kernel
      seed_t = at::empty({}, at::dtype(at::kLong).device(device));
      offset_t = at::empty({}, at::dtype(at::kLong).device(device));
    } else {
      auto [seed, offset] = at::cuda::philox::unpack(philox_state);
#ifdef USE_ROCM
      const auto options = at::dtype(at::kLong).device(at::kCUDA);
#else
      const auto options = at::dtype(at::kLong);
#endif
      seed_t = at::scalar_tensor(at::Scalar(static_cast<int64_t>(seed)), options);
      offset_t = at::scalar_tensor(at::Scalar(static_cast<int64_t>(offset)), options);
    }
  } else {
    // Not using dropout
    seed_t = at::empty({}, at::dtype(at::kLong).device(device));
    offset_t = at::empty({}, at::dtype(at::kLong).device(device));
  }

#ifdef USE_ROCM
  // ROCM Implementation

  // Need this in both aot and CK case
  const auto softmax_scale = sdp::calculate_scale(query, scale).expect_float();
  res = at::empty({B, M, num_heads, Kv}, query.options());

  if(at::globalContext().getROCmFAPreferredBackend() ==
    at::ROCmFABackend::Ck) {

#if defined(USE_CK_FLASH_ATTENTION)
    std::optional<Tensor> out(res);
    std::optional<Tensor> seqused_k = std::nullopt;
    std::optional<Tensor> alibi_slopes = std::nullopt;
    auto
        [out_,
         q,
         k,
         v,
         lse,
         seed_t,
         offset_t,
         p] =
            pytorch_flash::mem_eff_forward_ck(
                                    query,
                                    key,
                                    value,
                                    dropout_p,
                                    false,                                // return dropout_randval
                                    custom_mask_type == 0 ? false : true, // is_causal
                                    softmax_scale,
                                    bias,
                                    out,
                                    std::nullopt,                         // cu_seqlens_q
                                    std::nullopt,                         // cu_seqlens_k
                                    seqstart_q,
                                    seqstart_k,
                                    std::nullopt,                         // gen_
                                    seqused_k);                           // seqused_k_

    logsumexp = lse;
#else
    TORCH_CHECK(false, "Attempting to use CK mem_eff_forward backend in a build that has not built CK");
#endif
  } else { // use aotriton
#ifndef DISABLE_AOTRITON
    auto ret = aotriton::v2::flash::check_gpu(stream);
    if (hipSuccess != ret) {
        TORCH_CHECK(false,
                  "[AOTriton] Accelerated SDPA only supports MI200/MI300X/Navi31 GPUs"
                  " (gfx90a:sramecc+:xnack-/gfx942:sramecc+:xnack-/gfx1100)")
    }

    // AOTriton may accept aligned on logsumexp tensor in the future for better
    // performance, but for now it requires compact logsumexp tensor, even if
    // compute_logsumexp is false
    constexpr int kAlignLSE = 1;
    res = at::empty({B, M, num_heads, Kv}, query.options());
    at::Tensor softmax_lse;
    logsumexp = at::empty(
      { B, num_heads, compute_logsumexp ? max_seqlen_q : 0},
      query.options().dtype(at::ScalarType::Float));
    if (compute_logsumexp) {
      softmax_lse = logsumexp.view({B * num_heads, max_seqlen_q});
    }
    at::Tensor q_t = query.transpose(1, 2);
    at::Tensor k_t = key.transpose(1, 2);
    at::Tensor v_t = value.transpose(1, 2);
    at::Tensor output_t = res.transpose(1, 2);
    bool is_causal;
    if (static_cast<int64_t>(sdp::CustomMaskType::CausalFromTopLeft) == custom_mask_type) {
      is_causal = true;
    } else if (static_cast<int64_t>(sdp::CustomMaskType::NoCustomMask) == custom_mask_type) {
      is_causal = false;
    } else {
      TORCH_CHECK(false, "[_efficient_attention_forward] Unsupported mask type on ROCM, for now");
    }

    at::Tensor atomic_counter;
    if (is_causal) {
      atomic_counter = at::zeros({1}, query.options().dtype(at::kInt));
    }

    using aotriton::v2::flash::attn_fwd;
    using aotriton::v2::flash::attn_fwd_compact_varlen;
    using sdp::aotriton_adapter::mk_aotensor;
    using sdp::aotriton_adapter::mk_aoscalartensor;
    using sdp::aotriton_adapter::mk_philoxtensor;
    using sdp::aotriton_adapter::mk_atomictensor;
    aotriton::TensorView<4> empty_t4(0, {0, 0, 0, 0}, {0, 0, 0, 0}, aotriton::DType::kFloat16);
    aotriton::TensorView<2> empty_t2(0, {0, 0}, {0, 0}, aotriton::DType::kFloat32);
    at::Tensor softmax_fa_t = at::empty({ 0, 0, 0, 0 }, query.options());
    const bool use_philox_state = in_capture_stream;
    auto seed = use_philox_state ? mk_philoxtensor(philox_state.seed_.ptr) : mk_aoscalartensor(seed_t);
    auto offset1 = use_philox_state ? mk_philoxtensor(philox_state.offset_.ptr) : mk_aoscalartensor(offset_t);
    auto offset2 = use_philox_state ? philox_state.offset_intragraph_ : 0;
    auto seed_output = mk_philoxtensor(use_philox_state ? seed_t.data_ptr<int64_t>() : nullptr);
    auto offset_output = mk_philoxtensor(use_philox_state ? offset_t.data_ptr<int64_t>() : nullptr);
    auto persistent_counter = mk_atomictensor(is_causal ? atomic_counter.data_ptr<int32_t>() : nullptr);
    hipError_t err; // TODO: Error handling
    if (seqstart_q.has_value()) {
      // varlen aka nested tensor
      err = attn_fwd_compact_varlen(mk_aotensor(q_t, "q"),
                                    mk_aotensor(k_t, "k"),
                                    mk_aotensor(v_t, "v"),
                                    bias.has_value() ? mk_aotensor(bias.value(), "bias"): empty_t4,
                                    mk_aotensor<1>(seqstart_q.value(), "cu_seqlens_q"),
                                    mk_aotensor<1>(seqstart_k.value(), "cu_seqlens_k"),
                                    max_seqlen_q,
                                    max_seqlen_k,
                                    softmax_scale,
                                    compute_logsumexp ? mk_aotensor<2>(softmax_lse, "M") : empty_t2,
                                    mk_aotensor(output_t, "Out"),
                                    dropout_p,
                                    seed,
                                    offset1,
                                    offset2,
                                    seed_output,
                                    offset_output,
                                    mk_aotensor(softmax_fa_t, "encoded_softmax"),
                                    is_causal,
                                    persistent_counter,
                                    stream);
    } else {
      err = attn_fwd(mk_aotensor(q_t, "q"),
                     mk_aotensor(k_t, "k"),
                     mk_aotensor(v_t, "v"),
                     bias.has_value() ? mk_aotensor(bias.value(), "bias"): empty_t4,
                     softmax_scale,
                     compute_logsumexp ? mk_aotensor<2>(softmax_lse, "M") : empty_t2,
                     mk_aotensor(output_t, "Out"),
                     dropout_p,
                     seed,
                     offset1,
                     offset2,
                     seed_output,
                     offset_output,
                     mk_aotensor(softmax_fa_t, "encoded_softmax"),
                     is_causal,
                     persistent_counter,
                     stream);
    }
#else
    TORCH_CHECK(false, "Attempting to use AOTriton mem_eff_forward backend in a build that has not built AOTriton");
#endif
  } // CK BACKEND
#else
  // CUDA Implementation
  hipDeviceProp_t* p = at::cuda::getDeviceProperties(query.device().index());
  int computeCapability = p->major * 10 + p->minor;
  if (computeCapability == 121) {
    computeCapability = 120;
  }

  bool kernel_launched = false;
  const auto maxShmem = p->sharedMemPerBlockOptin;

  auto launchKernel = [&](auto _k, auto kernel_fn) {
    using Kernel = decltype(_k);
    using scalar_t = typename Kernel::scalar_t;
    (void)_k;

    if (kernel_launched) {
      return;
    }
    // Check if this kernel is compatible
    if (!Kernel::kSupportsDropout && use_dropout) {
      return;
    }
    if (!Kernel::kSupportsBias && bias.has_value()) {
      return;
    }

    if (value.size(3) > Kernel::kMaxK || key.size(3) > Kernel::kMaxK) {
      return;
    }
    // Alignment
    if ((query.stride(2) % Kernel::kAlignmentQ) ||
        (key.stride(2) % Kernel::kAlignmentK) ||
        (value.stride(2) % Kernel::kAlignmentV)) {
      return;
    }
    // Uses too much shmem
    size_t smem_bytes = sizeof(typename Kernel::SharedStorage);
    if (smem_bytes > maxShmem) {
      return;
    }
    kernel_launched = true;

    res = at::empty(
        {B, M, num_heads, Kv},
        query.options().dtype(
            CutlassToAtenDtype<typename Kernel::output_t>::atScalarType()));

    // NOTE: Should be aligned (by padding) in case M is
    // not a good number for loading during backward
    constexpr decltype(M) kAlignLSE = Kernel::kAlignLSE;
    logsumexp = at::empty(
        {seqstart_q.has_value() ? seqstart_q->size(0) - 1 : B,
         num_heads,
         compute_logsumexp ? ceil_div(max_seqlen_q, kAlignLSE) * kAlignLSE : 0},
        query.options().dtype(at::ScalarType::Float));
    typename Kernel::Params p;
    p.query_ptr = (const scalar_t*)query.const_data_ptr();
    p.key_ptr = (const scalar_t*)key.const_data_ptr();
    p.value_ptr = (const scalar_t*)value.const_data_ptr();
    p.logsumexp_ptr = compute_logsumexp
        ? (typename Kernel::lse_scalar_t*)logsumexp.data_ptr()
        : nullptr;
    at::Tensor output_accum;
    if (Kernel::kNeedsOutputAccumulatorBuffer) {
      output_accum = at::empty(
          {B, M, num_heads, Kv},
          query.options().dtype(
              CutlassToAtenDtype<
                  typename Kernel::output_accum_t>::atScalarType()));
      p.output_accum_ptr =
          (typename Kernel::output_accum_t*)output_accum.data_ptr();
    } else {
      p.output_accum_ptr = nullptr;
    }
    p.output_ptr = (typename Kernel::output_t*)res.data_ptr();

    if (seqstart_q.has_value()) {
      p.seqstart_q_ptr = (const int32_t*)seqstart_q->const_data_ptr();
      p.seqstart_k_ptr = (const int32_t*)seqstart_k->const_data_ptr();
    }

    p.num_heads = num_heads;
    p.head_dim = query.size(3);
    p.head_dim_value = value.size(3);
    p.num_queries = max_seqlen_q;
    p.num_keys = max_seqlen_k;
    p.num_batches = seqstart_q.has_value() ? seqstart_q->size(0) - 1 : B;
    p.custom_mask_type = custom_mask_type;

    p.seqlen_k_ptr = nullptr;
    if (seqlen_k.has_value()) {
      CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(seqlen_k.value());
      TORCH_CHECK(seqlen_k->scalar_type() == at::ScalarType::Int);
      p.seqlen_k_ptr = (const int32_t*)seqlen_k->const_data_ptr();
    }
    if (window_size.has_value()) {
      p.window_size = *window_size;
    }
    p.scale = sdp::calculate_scale(query, scale).expect_float();

    ASSIGN_CHECK_OVERFLOW(p.q_strideB, query.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.k_strideB, key.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.v_strideB, value.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.q_strideM, query.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.k_strideM, key.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.v_strideM, value.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.q_strideH, query.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.k_strideH, key.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.v_strideH, value.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.o_strideM, res.stride(1));

    if (bias.has_value()) {
      CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA((*bias));
      TORCH_CHECK(
          bias->scalar_type() == CutlassToAtenDtype<scalar_t>::atScalarType(),
          "invalid dtype for bias - should match query's dtype");
      p.attn_bias_ptr = (const scalar_t*)bias->const_data_ptr();

      TORCH_CHECK(bias->dim() == 4, "Bias expected in BMHK format");
      TORCH_CHECK(
          bias->size(0) == query.size(0),
          "attn_bias: wrong shape (batch dimension)");
      TORCH_CHECK(
          bias->size(1) == query.size(2),
          "attn_bias: wrong shape (head dimension)");
      TORCH_CHECK(
          bias->size(2) == query.size(1),
          "attn_bias: wrong shape (seqlenQ dimension)");
      TORCH_CHECK(
          bias->size(3) == key.size(1),
          "attn_bias: wrong shape (seqlenKV dimension)");
      ASSIGN_CHECK_OVERFLOW(p.bias_strideB, bias->stride(0));
      ASSIGN_CHECK_OVERFLOW(p.bias_strideH, bias->stride(1));
      ASSIGN_CHECK_OVERFLOW(p.bias_strideM, bias->stride(2));
      TORCH_CHECK(
          bias->stride(3) == 1,
          "attn_bias: wrong alignment (last dimension must be contiguous)");
    }

    p.use_dropout = use_dropout;
    if (p.use_dropout) {
      p.rng_engine_inputs = philox_state;
      p.dropout_prob = dropout_p;
      p.seed = seed_t.data_ptr<int64_t>();
      p.extragraph_offset = offset_t.data_ptr<int64_t>();
    }

    if (smem_bytes > 0xc000) {
      auto err = hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
      TORCH_CHECK(
          err != hipErrorInvalidValue,
          "This GPU does not have enough shared-memory (kernel requires ",
          smem_bytes / 1024,
          " kb)");
      AT_CUDA_CHECK(err);
    }
    auto blocks = p.getBlocksGrid();
    if (blocks.x * blocks.y * blocks.z == 0 || key.size(1) == 0) {
      res.zero_();
      return;
    }
    Kernel::check_supported(p);
    kernel_fn<<<blocks, p.getThreadsGrid(), smem_bytes, stream>>>(p);
  };

  // Dispatch to the right kernel
  DISPATCH_TYPES(query, ([&]() {
                   dispatch_cutlassF<scalar_t>(launchKernel, computeCapability);
                 }));
  TORCH_CHECK(kernel_launched, "cutlassF: no kernel found to launch!");
  AT_CUDA_CHECK(hipGetLastError());

#endif // USE_ROCM
  return std::make_tuple(
      std::move(res),
      std::move(logsumexp),
      std::move(seed_t),
      std::move(offset_t),
      max_seqlen_q,
      // TODO: why isn't this being set in the kernel?
      max_seqlen_k_.has_value() ? max_seqlen_k_.value() : max_seqlen_k);
#endif
  TORCH_CHECK(false, "USE_MEM_EFF_ATTENTION was not enabled for build.")
  return std::make_tuple(Tensor{}, Tensor{}, Tensor{}, Tensor{}, 0, 0);
}

Tensor triton_scaled_dot_attention(const Tensor& q, const Tensor& k, const Tensor& v, double dropout_p){
  TORCH_CHECK(false, "This operator should be overridden in python before use");
  return at::Tensor();
}

REGISTER_CUDA_DISPATCH(_fused_sdp_choice_stub, &_fused_sdp_choice_cuda)

#if defined(USE_MEM_EFF_ATTENTION) and !defined(USE_ROCM)
namespace {
/**
 * simple kernel that populates a tensor with rand uniform values.
 * currently only used for testing purposes, not much attention
 * is paid to performance.
 *
 * problem is partitioned as follows:
 * - (batch, head) is given by block coordinates
 * - each thread handles a row for a given (batch, head)
 */
template <typename mask_t>
__global__ void rand_uniform_kernel(
    int64_t n_heads,
    int64_t n_queries,
    int64_t n_keys,
    float dropout_prob,
    at::PhiloxCudaState rng_engine_inputs,
    mask_t* mask_out,
    int64_t mask_numel) {
  const int64_t batch_id = blockIdx.x;
  const int64_t head_id = blockIdx.y;
  const int64_t query_idx = threadIdx.x;

  const auto [seed, offset] = at::cuda::philox::unpack(rng_engine_inputs);

  const int dropout_seq_start = batch_id * (n_heads * n_queries * n_keys) +
      head_id * (n_queries * n_keys);
  const int64_t query_start_idx = query_idx * n_keys;

  hiprandStatePhilox4_32_10_t curand_state;
  hiprand_init(
      seed,
      0,
      offset + dropout_seq_start + query_start_idx,
      &curand_state);

  for (int key_start_idx = 0; key_start_idx < n_keys; key_start_idx += 4) {
    float4 rand_quad = hiprand_uniform4(&curand_state);

#pragma unroll
    for (int i = 0; i < 4; ++i) {
      const int64_t linear_idx = dropout_seq_start + query_start_idx + key_start_idx + i;
      if (linear_idx < mask_numel) {
        mask_out[linear_idx] = (&rand_quad.x)[i];
      }
    }
  }
}
} // namespace
#endif // defined(USE_MEM_EFF_ATTENTION) and !defined(USE_ROCM)
/**
 * fill tensor with random uniform values. only used for testing, not much
 * attention is paid to performance
 */
at::Tensor& _fill_mem_eff_dropout_mask_(
    Tensor& self,
    double dropout_p,
    const int64_t seed,
    const int64_t offset) {
  TORCH_CHECK(self.is_contiguous());
  TORCH_CHECK(self.dtype() == at::ScalarType::Float);
  const int64_t batch_sz = self.size(0);
  const int64_t n_heads = self.size(1);
  const int64_t n_queries = self.size(2);
  const int64_t n_keys = self.size(3);
#if defined(USE_MEM_EFF_ATTENTION)

#ifdef USE_ROCM
#ifndef DISABLE_AOTRITON
  using aotriton::v2::flash::debug_simulate_encoded_softmax;
  using sdp::aotriton_adapter::mk_aotensor;
  using sdp::aotriton_adapter::mk_aoscalartensor;
  at::cuda::CUDAGuard device_guard(self.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  at::Tensor seed_t, offset_t;
  const auto options = at::dtype(at::kLong).device(at::kCUDA);
  seed_t = at::scalar_tensor(at::Scalar(seed), options);
  offset_t = at::scalar_tensor(at::Scalar(offset), options);
  hipError_t err; // TODO: Error handling

  err = debug_simulate_encoded_softmax(mk_aotensor(self, "r"),
                                       dropout_p,
                                       mk_aoscalartensor(seed_t),
                                       mk_aoscalartensor(offset_t),
                                       0,
                                       stream);
#else
  TORCH_CHECK(false, "_fill_mem_eff_dropout_mask_ is only enabled with aotriton");
#endif
#else
  at::PhiloxCudaState rng_engine_inputs;
  rng_engine_inputs = at::PhiloxCudaState(seed, offset);
  at::cuda::CUDAGuard device_guard(self.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  rand_uniform_kernel<float><<<dim3(batch_sz, n_heads), n_queries, 0, stream>>>(
      n_heads,
      n_queries,
      n_keys,
      dropout_p,
      rng_engine_inputs,
      reinterpret_cast<float*>(self.data_ptr()),
      self.numel());
#endif

  return self;
#endif
  TORCH_CHECK(false, "USE_MEM_EFF_ATTENTION was not enabled for build.")
  return self;
}

} // namespace native
} // namespace at
