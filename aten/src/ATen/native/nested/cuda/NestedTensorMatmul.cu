#include <type_traits>

#include <ATen/ATen.h>
#include <ATen/Dispatch.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/PersistentSoftmax.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#include <c10/cuda/CUDAMathCompat.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/nested/NestedTensorTransformerFunctions.h>
#include <ATen/native/nested/NestedTensorUtils.h>

#if !defined(USE_ROCM) && !defined(_WIN32) && (defined(CUDA_VERSION) && CUDA_VERSION > 12000)
#define build_grouped_gemm
#endif


#ifdef build_grouped_gemm
#include <cutlass/gemm/device/default_gemm_configuration.h>
#include <cutlass/gemm/device/gemm_grouped.h>
#include <cutlass/gemm/kernel/default_gemm_grouped.h>
#endif

#include <ATen/NestedTensorImpl.h>

#define BLOCK_DIM 256
#define GRID_DIM_Y 16

namespace at {
namespace native {

#ifdef build_grouped_gemm
namespace {

template <
    typename scalar_t,
    unsigned int kPad,
    typename LayoutA,
    typename LayoutB,
    typename OpClass,
    typename Arch,
    typename ThreadBlockShape,
    typename WarpShape,
    typename InstructionShape>
void gemm_grouped_cuda_internal(
    const std::vector<int64_t>& lda,
    const std::vector<int64_t>& ldb,
    const std::vector<int64_t>& ldd,
    const std::vector<scalar_t*>& aptr,
    const std::vector<scalar_t*>& bptr,
    const std::vector<scalar_t*>& dptr,
    const std::vector<cutlass::gemm::GemmCoord>& gemm_sizes,
    const int64_t problem_count,
    at::Device& device) {
  using Element = scalar_t;
  using ElementAcc = float;

  using GemmConfiguration =
      typename cutlass::gemm::device::DefaultGemmConfiguration<
          OpClass,
          Arch,
          Element,
          Element,
          Element,
          ElementAcc>;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
      Element,
      LayoutA,
      cutlass::ComplexTransform::kNone,
      kPad,
      Element,
      LayoutB,
      cutlass::ComplexTransform::kNone,
      kPad,
      Element,
      cutlass::layout::RowMajor,
      ElementAcc,
      OpClass,
      Arch,
      ThreadBlockShape,
      WarpShape,
      InstructionShape,
      typename GemmConfiguration::EpilogueOutputOp,
      cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
      GemmConfiguration::kStages>::GemmKernel;

  using GemmGrouped = typename cutlass::gemm::device::GemmGrouped<GemmKernel>;
  using EpilogueOutputOp = typename GemmGrouped::GemmKernel::Epilogue::OutputOp;
  typename EpilogueOutputOp::Params epilogue_op(/*alpha*/ 1, /*beta*/ 0);

  const int64_t gemm_coord_size =
      problem_count * ((int64_t)sizeof(cutlass::gemm::GemmCoord));
  // Number of gmm args not including *problem_sizes
  at::Tensor gmm_args = at::empty(
      {problem_count * 6 + gemm_coord_size},
      at::TensorOptions().dtype(at::kLong).pinned_memory(true));

  // Obtain pointers for each argument (on host)
  int64_t* lda_data = gmm_args.data_ptr<int64_t>(); // Base pointer
  int64_t* ldb_data = lda_data + problem_count;
  int64_t* ldd_data = lda_data + 2 * problem_count;
  int64_t* ptr_a_data = lda_data + 3 * problem_count;
  int64_t* ptr_b_data = lda_data + 4 * problem_count;
  int64_t* ptr_d_data = lda_data + 5 * problem_count;
  cutlass::gemm::GemmCoord* problem_sizes_data =
      reinterpret_cast<cutlass::gemm::GemmCoord*>(lda_data + 6 * problem_count);

  // Set arguments into gmm_args from input args
  for (int i = 0; i < problem_count; ++i) {
    problem_sizes_data[i] = gemm_sizes[i];
    lda_data[i] = lda[i];
    ldb_data[i] = ldb[i];
    ldd_data[i] = ldd[i];
    ptr_a_data[i] = reinterpret_cast<int64_t>(aptr[i]);
    ptr_b_data[i] = reinterpret_cast<int64_t>(bptr[i]);
    ptr_d_data[i] = reinterpret_cast<int64_t>(dptr[i]);
  }
  const int threadblock_count =
      GemmGrouped::sufficient(problem_sizes_data, problem_count);

  // Transfer arguments to GPU
  gmm_args = gmm_args.to(device, true);

  // Obtain pointers for each of arguments (on GPU)
  lda_data = gmm_args.data_ptr<int64_t>(); // Base pointer
  ldb_data = lda_data + problem_count;
  ldd_data = lda_data + 2 * problem_count;
  ptr_a_data = lda_data + 3 * problem_count;
  ptr_b_data = lda_data + 4 * problem_count;
  ptr_d_data = lda_data + 5 * problem_count;
  problem_sizes_data =
      reinterpret_cast<cutlass::gemm::GemmCoord*>(lda_data + 6 * problem_count);

  // Create GemmGrouped::Arguments using the arguments prepared above
  typename GemmGrouped::Arguments args(
      problem_sizes_data,
      problem_count,
      threadblock_count,
      epilogue_op,
      reinterpret_cast<Element**>(ptr_a_data),
      reinterpret_cast<Element**>(ptr_b_data),
      reinterpret_cast<Element**>(ptr_d_data),
      reinterpret_cast<Element**>(ptr_d_data),
      lda_data,
      ldb_data,
      ldd_data,
      ldd_data);

  GemmGrouped gemm;
  cutlass::Status status =
      gemm.initialize(args, nullptr, at::cuda::getCurrentCUDAStream());
  TORCH_CHECK(
      status != cutlass::Status::kErrorWorkspaceNull,
      "Failed to initialize CUTLASS Grouped GEMM kernel due to workspace.");
  TORCH_CHECK(
      status != cutlass::Status::kErrorInternal,
      "Failed to initialize CUTLASS Grouped GEMM kernel due to internal error.");
  TORCH_CHECK(
      status == cutlass::Status::kSuccess,
      "Failed to initialize CUTLASS Grouped GEMM kernel.");

  // Run CUTLASS group GEMM
  status = gemm.run(at::cuda::getCurrentCUDAStream());
  TORCH_CHECK(
      status == cutlass::Status::kSuccess,
      "Failed to run CUTLASS Grouped GEMM kernel.");

  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename scalar_t>
bool group_gemm_dispatch(
    at::Device device,
    const std::vector<scalar_t*>& aptr,
    const std::vector<scalar_t*>& bptr,
    const std::vector<scalar_t*>& dptr,
    const std::vector<int64_t>& lda,
    const std::vector<int64_t>& ldb,
    const std::vector<int64_t>& ldd,
    const std::vector<cutlass::gemm::GemmCoord>& gemm_sizes,
    int64_t ntensors) {
  return false;
}

template <>
bool group_gemm_dispatch(
    at::Device device,
    const std::vector<float*>& aptr,
    const std::vector<float*>& bptr,
    const std::vector<float*>& dptr,
    const std::vector<int64_t>& lda,
    const std::vector<int64_t>& ldb,
    const std::vector<int64_t>& ldd,
    const std::vector<cutlass::gemm::GemmCoord>& gemm_sizes,
    int64_t ntensors) {

  gemm_grouped_cuda_internal<
      float,
      1,
      cutlass::layout::RowMajor,
      cutlass::layout::RowMajor,
      cutlass::arch::OpClassSimt,
      cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<128, 128, 8>,
      cutlass::gemm::GemmShape<64, 32, 8>,
      cutlass::gemm::GemmShape<1, 1, 1>>(
      lda, ldb, ldd, aptr, bptr, dptr, gemm_sizes, ntensors, device);
  return true;
}

template <>
bool group_gemm_dispatch(
    at::Device device,
    const std::vector<c10::Half*>& aptr_,
    const std::vector<c10::Half*>& bptr_,
    const std::vector<c10::Half*>& dptr_,
    const std::vector<int64_t>& lda,
    const std::vector<int64_t>& ldb,
    const std::vector<int64_t>& ldd,
    const std::vector<cutlass::gemm::GemmCoord>& gemm_sizes,
    int64_t ntensors) {

  // Check alignment
  bool all_pad_8 = true;
  for (int i = 0; i < ntensors; i++) {
    all_pad_8 = all_pad_8 && (gemm_sizes[i].n() % 8 == 0);
    all_pad_8 = all_pad_8 && (gemm_sizes[i].k() % 8 == 0);

    // Not sure if this is a requirement, on the safe side
    all_pad_8 = all_pad_8 && (lda[i] % 8 == 0);
    all_pad_8 = all_pad_8 && (ldb[i] % 8 == 0);
    all_pad_8 = all_pad_8 && (ldd[i] % 8 == 0);
  }

  std::vector<cutlass::half_t*> aptr;
  aptr.reserve(ntensors);
  std::vector<cutlass::half_t*> bptr;
  bptr.reserve(ntensors);
  std::vector<cutlass::half_t*> dptr;
  dptr.reserve(ntensors);
  for (int64_t i = 0; i < ntensors; i++) {
    aptr.push_back(reinterpret_cast<cutlass::half_t*>(aptr_[i]));
    bptr.push_back(reinterpret_cast<cutlass::half_t*>(bptr_[i]));
    dptr.push_back(reinterpret_cast<cutlass::half_t*>(dptr_[i]));
  }
  if (all_pad_8) {
    gemm_grouped_cuda_internal<
        cutlass::half_t,
        8,
        cutlass::layout::RowMajor,
        cutlass::layout::RowMajor,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm80,
        cutlass::gemm::GemmShape<128, 128, 32>,
        cutlass::gemm::GemmShape<64, 64, 32>,
        cutlass::gemm::GemmShape<16, 8, 16>>(
        lda, ldb, ldd, aptr, bptr, dptr, gemm_sizes, ntensors, device);
    return true;
  } else {
    gemm_grouped_cuda_internal<
        cutlass::half_t,
        1,
        cutlass::layout::RowMajor,
        cutlass::layout::RowMajor,
        cutlass::arch::OpClassSimt,
        cutlass::arch::Sm80,
        cutlass::gemm::GemmShape<128, 128, 8>,
        cutlass::gemm::GemmShape<64, 32, 8>,
        cutlass::gemm::GemmShape<1, 1, 1>>(
        lda, ldb, ldd, aptr, bptr, dptr, gemm_sizes, ntensors, device);
    return true;
  }
  // Did not perform GEMM
  return false;
}

template <>
bool group_gemm_dispatch(
    at::Device device,
    const std::vector<c10::BFloat16*>& aptr_,
    const std::vector<c10::BFloat16*>& bptr_,
    const std::vector<c10::BFloat16*>& dptr_,
    const std::vector<int64_t>& lda,
    const std::vector<int64_t>& ldb,
    const std::vector<int64_t>& ldd,
    const std::vector<cutlass::gemm::GemmCoord>& gemm_sizes,
    int64_t ntensors) {

  // Check alignment
  bool all_pad_8 = true;
  for (int i = 0; i < ntensors; i++) {
    all_pad_8 = all_pad_8 && (gemm_sizes[i].n() % 8 == 0);
    all_pad_8 = all_pad_8 && (gemm_sizes[i].k() % 8 == 0);

    // Not sure if this is a requirement, on the safe side
    all_pad_8 = all_pad_8 && (lda[i] % 8 == 0);
    all_pad_8 = all_pad_8 && (ldb[i] % 8 == 0);
    all_pad_8 = all_pad_8 && (ldd[i] % 8 == 0);
  }

  std::vector<cutlass::bfloat16_t*> aptr;
  aptr.reserve(ntensors);
  std::vector<cutlass::bfloat16_t*> bptr;
  bptr.reserve(ntensors);
  std::vector<cutlass::bfloat16_t*> dptr;
  dptr.reserve(ntensors);
  for (int64_t i = 0; i < ntensors; i++) {
    aptr.push_back(reinterpret_cast<cutlass::bfloat16_t*>(aptr_[i]));
    bptr.push_back(reinterpret_cast<cutlass::bfloat16_t*>(bptr_[i]));
    dptr.push_back(reinterpret_cast<cutlass::bfloat16_t*>(dptr_[i]));
  }
  if (all_pad_8) {
    gemm_grouped_cuda_internal<
        cutlass::bfloat16_t,
        8,
        cutlass::layout::RowMajor,
        cutlass::layout::RowMajor,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm80,
        cutlass::gemm::GemmShape<128, 128, 32>,
        cutlass::gemm::GemmShape<64, 64, 32>,
        cutlass::gemm::GemmShape<16, 8, 16>>(
        lda, ldb, ldd, aptr, bptr, dptr, gemm_sizes, ntensors, device);
    return true;
  } else {
    gemm_grouped_cuda_internal<
        cutlass::bfloat16_t,
        1,
        cutlass::layout::RowMajor,
        cutlass::layout::RowMajor,
        cutlass::arch::OpClassSimt,
        cutlass::arch::Sm80,
        cutlass::gemm::GemmShape<128, 128, 8>,
        cutlass::gemm::GemmShape<64, 32, 8>,
        cutlass::gemm::GemmShape<1, 1, 1>>(
        lda, ldb, ldd, aptr, bptr, dptr, gemm_sizes, ntensors, device);
    return true;
  }
  // Did not perform GEMM
  return false;
}

} // namespace

#endif

Tensor bmm_nested_cuda(const Tensor& self, const Tensor& mat2) {

  // dispatcher should have guaranteed that at least one is nested
  auto self_ptr = self.is_nested() ? get_nested_tensor_impl(self) : self.unsafeGetTensorImpl();
  auto mat2_ptr = mat2.is_nested() ? get_nested_tensor_impl(mat2) : mat2.unsafeGetTensorImpl();
  TORCH_CHECK(self_ptr->dim() == 3, "batch1 must be a 3D tensor");
  TORCH_CHECK(mat2_ptr->dim() == 3, "batch2 must be a 3D tensor");
  int64_t ntensors = self_ptr->size(0), ntensors2 = mat2_ptr->size(0);
  TORCH_CHECK(
      ntensors == ntensors2,
      "Expected size for the 1st dimension of batch2 tensor to be: ",
      ntensors,
      " but got: ",
      ntensors2,
      ".");

  // create a contiguous output
  const Tensor& self_sizemat = self.is_nested() ?
      get_nested_tensor_impl(self)->get_nested_sizes() : get_nested_tensor_impl(mat2)->get_nested_sizes();

  Tensor out_sizemat = self_sizemat.new_empty(self_sizemat.sizes());
  int64_t* out_sizemat_ptr = out_sizemat.data_ptr<int64_t>();

  int64_t out_numel = 0;
  for (int64_t i = 0; i < ntensors; i++) {
    const IntArrayRef &self_shape = get_size_for_index(self, i), &mat2_shape = get_size_for_index(mat2, i);
    const int64_t &self_size0 = self_shape[0], &self_size1 = self_shape[1],
                  &mat2_size0 = mat2_shape[0], &mat2_size1 = mat2_shape[1];
    TORCH_CHECK(
        self_size1 == mat2_size0,
        i,
        "-th nested matrices in batch cannot be multiplied (",
        self_size0,
        "x",
        self_size1,
        " and ",
        mat2_size0,
        "x",
        mat2_size1,
        ")");
    out_sizemat_ptr[0] = self_size0;
    out_sizemat_ptr[1] = mat2_size1;
    out_sizemat_ptr += 2;
    out_numel += self_size0 * mat2_size1;
  }

  const Tensor &self_buffer = self.is_nested() ? get_nested_tensor_impl(self)->get_unsafe_storage_as_tensor() : self;
  const Tensor &mat2_buffer = mat2.is_nested() ? get_nested_tensor_impl(mat2)->get_unsafe_storage_as_tensor() : mat2;

  Tensor out_buffer = self_buffer.new_empty(out_numel);
  Tensor output = wrap_buffer(out_buffer, out_sizemat);
  auto out_ptr = get_nested_tensor_impl(output);

  const int64_t *out_offsets_ptr = out_ptr->get_storage_offsets().const_data_ptr<int64_t>();

#ifdef build_grouped_gemm
  bool success = false;
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
      self.scalar_type(), "group_gemm_dispatch", [&] {
        std::vector<scalar_t*> aptr(ntensors);
        std::vector<scalar_t*> bptr(ntensors);
        std::vector<scalar_t*> dptr(ntensors);
        std::vector<int64_t> lda(ntensors);
        std::vector<int64_t> ldb(ntensors);
        std::vector<int64_t> ldd(ntensors);
        std::vector<cutlass::gemm::GemmCoord> gemm_sizes;
        gemm_sizes.reserve(ntensors);
        bool all_row_major = true;
        for (int64_t i = 0; i < ntensors; i++) {
          const IntArrayRef& self_shape = get_size_for_index(self, i);
          const IntArrayRef& mat2_shape = get_size_for_index(mat2, i);
          const int64_t &self_size0 = self_shape[0];
          const int64_t &self_size1 = self_shape[1];
          const int64_t &mat2_size0 = mat2_shape[0];
          const int64_t &mat2_size1 = mat2_shape[1];
          gemm_sizes.emplace_back(self_size0, mat2_size1, self_size1);
          aptr[i] = self_buffer.data_ptr<scalar_t>() + get_offset_for_index(self, i);
          bptr[i] = mat2_buffer.data_ptr<scalar_t>() + get_offset_for_index(mat2, i);
          dptr[i] = out_buffer.data_ptr<scalar_t>() + out_offsets_ptr[i];
          auto self_stride = get_stride_for_index(self, i);
          auto mat2_stride = get_stride_for_index(mat2, i);
          all_row_major = all_row_major && (self_stride[1] == 1);
          all_row_major = all_row_major && (mat2_stride[1] == 1);
          lda[i] = self_stride[0];
          ldb[i] = mat2_stride[0];
          ldd[i] = mat2_size1;
        }
        auto dprops = at::cuda::getCurrentDeviceProperties();
        bool is_sm8x = dprops->major == 8 && dprops->minor >= 0;
        if (all_row_major &&
            self.is_contiguous() &&
            mat2.is_contiguous() &&
            is_sm8x) {
          success = group_gemm_dispatch<scalar_t>(
              output.device(),
              aptr,
              bptr,
              dptr,
              lda,
              ldb,
              ldd,
              gemm_sizes,
              ntensors);
        }
      });
  if (success) {
    return output;
  }
#endif

  std::vector<Tensor> output_unbind = output.unbind();
  for (int64_t i = 0; i < ntensors; i++) {
    at::mm_out(output_unbind[i],
        self_buffer.as_strided(get_size_for_index(self, i), get_stride_for_index(self, i), get_offset_for_index(self, i)),
        mat2_buffer.as_strided(get_size_for_index(mat2, i), get_stride_for_index(mat2, i), get_offset_for_index(mat2, i)));
  }
  return output;
}

} // namespace native
} // namespace at
