#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <type_traits>

#include <ATen/ATen.h>
#include <ATen/Dispatch.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/PersistentSoftmax.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/nested/NestedTensorTransformerFunctions.h>
#include <ATen/native/nested/NestedTensorUtils.h>

#if !defined(USE_ROCM) && !defined(_WIN32) && (defined(CUDA_VERSION) && CUDA_VERSION > 12000)
#define build_grouped_gemm
#endif

#ifdef build_grouped_gemm
#include <cutlass/gemm/device/default_gemm_configuration.h>
#include <cutlass/gemm/device/gemm_grouped.h>
#include <cutlass/gemm/kernel/default_gemm_grouped.h>
#endif

#include <ATen/NestedTensorImpl.h>

#define BLOCK_DIM 256
#define GRID_DIM_Y 16

namespace at {
namespace native {

template <typename T>
__global__ void remove_padding_transform0213_2(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1];
  int input_offset =
      batch_id * input_sizes[1] * input_sizes[2] * input_sizes[3];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i2 = i / sizes_i[1];
    const int i13 = i % sizes_i[1];
    const int i1 = i13 / (sizes_i[1] / input_sizes[1]);
    const int i3 = i13 % (sizes_i[1] / input_sizes[1]);

    output[offset + i] = input
        [input_offset + i1 * input_sizes[2] * input_sizes[3] +
         i2 * input_sizes[3] + i3];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i2 = i / sizes_i[1];
    const int i13 = i % sizes_i[1];
    const int i1 = i13 / (sizes_i[1] / input_sizes[1]);
    const int i3 = i13 % (sizes_i[1] / input_sizes[1]);
    output[offset + i] = input
        [input_offset + i1 * input_sizes[2] * input_sizes[3] +
         i2 * input_sizes[3] + i3];
  }
}

template <typename T>
__global__ void remove_padding_2(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1];
  int input_offset = batch_id * input_sizes[1] * input_sizes[2];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / sizes_i[1];
    const int i1 = i % sizes_i[1];
    const int i0_offset = i0 * input_sizes[2];
    output[offset + i] = input[input_offset + i0_offset + i1];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i0 = i / sizes_i[1];
    const int i1 = i % sizes_i[1];
    const int i0_offset = i0 * input_sizes[2];
    output[offset + i] = input[input_offset + i0_offset + i1];
  }
}

template <typename T>
__global__ void remove_padding(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1] * sizes_i[2];
  int input_offset =
      batch_id * input_sizes[1] * input_sizes[2] * input_sizes[3];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (sizes_i[1] * sizes_i[2]);
    const int i1 = (i % (sizes_i[1] * sizes_i[2])) / sizes_i[2];
    const int i2 = i % sizes_i[2];
    const int i0_offset = i0 * input_sizes[2] * input_sizes[3];
    const int i1_offset = i1 * input_sizes[3];
    output[offset + i] = input[input_offset + i0_offset + i1_offset + i2];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i0 = i / (sizes_i[1] * sizes_i[2]);
    const int i1 = (i % (sizes_i[1] * sizes_i[2])) / sizes_i[2];
    const int i2 = i % sizes_i[2];
    const int i0_offset = i0 * input_sizes[2] * input_sizes[3];
    const int i1_offset = i1 * input_sizes[3];
    output[offset + i] = input[input_offset + i0_offset + i1_offset + i2];
  }
}

template <typename T>
void remove_padding_kernelLauncher(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int64_t output_dim,
    const int64_t batch_size) {
  dim3 grid;
  grid.x = batch_size;
  grid.y = GRID_DIM_Y;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  if (output_dim == 2) {
    remove_padding_2<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        offsets,
        input_sizes,
        output_sizes,
        output_dim,
        batch_size);
  } else {
    remove_padding<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        offsets,
        input_sizes,
        output_sizes,
        output_dim,
        batch_size);
  }
}

template <typename T>
void remove_padding_transform0213_kernelLauncher(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int64_t output_dim,
    const int64_t batch_size) {
  dim3 grid;
  grid.x = batch_size;
  grid.y = GRID_DIM_Y;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  TORCH_CHECK(
      output_dim == 2,
      "remove padding transform0213 only support output dim == 2");

  remove_padding_transform0213_2<T><<<grid, BLOCK_DIM, 0, stream>>>(
      input,
      output,
      offsets,
      input_sizes,
      output_sizes,
      output_dim,
      batch_size);
}

template void remove_padding_kernelLauncher<float>(
    const float* input,
    float* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int64_t output_dim,
    const int64_t batch_size);

template void remove_padding_kernelLauncher<c10::Half>(
    const c10::Half* input,
    c10::Half* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int64_t output_dim,
    const int64_t batch_size);

template void remove_padding_transform0213_kernelLauncher<float>(
    const float* input,
    float* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int64_t output_dim,
    const int64_t batch_size);

template void remove_padding_transform0213_kernelLauncher<c10::Half>(
    const c10::Half* input,
    c10::Half* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int64_t output_dim,
    const int64_t batch_size);

template <typename T>
__global__ void add_padding_1(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    int output_sizes_1,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int batch_output_offset = batch_id * output_sizes_1;
  for (int ii = 0; ii < (output_sizes_1 / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int output_offset = batch_output_offset + i;
    if (batch_id < batch_size && i < sizes_i[0]) {
      const int batch_input_offset = offsets[batch_id];
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
  const int i = (output_sizes_1 / grainsize) * grainsize + tid;
  if (i < output_sizes_1) {
    const int output_offset = batch_output_offset + i;
    if (batch_id < batch_size && (i < sizes_i[0])) {
      const int batch_input_offset = offsets[batch_id];
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
}

template <typename T>
__global__ void add_padding_2(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    int output_sizes_1,
    int output_sizes_2,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int output_offset = batch_id * output_sizes_1 * output_sizes_2;
  const int output_numel = output_sizes_1 * output_sizes_2;
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes_2);
    const int i1 = i - i0 * output_sizes_2;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int offset = offsets[batch_id];
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes_2);
    const int i1 = i - i0 * output_sizes_2;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int offset = offsets[batch_id];
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
}

template <typename T>
__global__ void add_padding_3(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    int output_sizes_1,
    int output_sizes_2,
    int output_sizes_3,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int output_offset =
      batch_id * output_sizes_1 * output_sizes_2 * output_sizes_3;
  const int output_numel = output_sizes_1 * output_sizes_2 * output_sizes_3;
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes_2 * output_sizes_3);
    const int i1 = (i % (output_sizes_2 * output_sizes_3)) / output_sizes_3;
    const int i2 = i % output_sizes_3;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1] &&
        i2 < sizes_i[2]) {
      const int offset = offsets[batch_id];
      const int input_offset =
          offset + i0 * (sizes_i[1] * sizes_i[2]) + i1 * sizes_i[2] + i2;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes_2 * output_sizes_3);
    const int i1 = (i % (output_sizes_2 * output_sizes_3)) / output_sizes_3;
    const int i2 = i % output_sizes_3;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1] &&
        i2 < sizes_i[2]) {
      const int offset = offsets[batch_id];
      const int input_offset =
          offset + i0 * (sizes_i[1] * sizes_i[2]) + i1 * sizes_i[2] + i2;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
}

template <typename T>
void add_padding_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size) {
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  dim3 grid;
  grid.x = output_batch_size;
  grid.y = GRID_DIM_Y;
  if (input_dim == 1) {
    add_padding_1<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes[1],
        batch_size);
  }
  if (input_dim == 2) {
    add_padding_2<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes[1],
        output_sizes[2],
        batch_size);
  }
  if (input_dim == 3) {
    add_padding_3<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes[1],
        output_sizes[2],
        output_sizes[3],
        batch_size);
  }
}

template void add_padding_kernelLauncher<double>(
    double* input,
    double* output,
    double padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size);

template void add_padding_kernelLauncher<float>(
    float* input,
    float* output,
    float padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size);

template void add_padding_kernelLauncher<c10::Half>(
    c10::Half* input,
    c10::Half* output,
    c10::Half padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size);

// NB: The following code covers jagged <-> padded dense conversions and was lifted
// from fbgemm_gpu. For more details, see
// https://github.com/pytorch/FBGEMM/tree/main/fbgemm_gpu/src/jagged_tensor_ops

// Passing lambda exp argument by value instead of by reference to avoid
// "internal compiler error: in maybe_undo_parenthesized_ref" error for specific
// compiler version.
#define JAGGED_TENSOR_DISPATCH_DIMS()                                         \
  AT_DISPATCH_INDEX_TYPES(x_offsets[0].scalar_type(), "jagged_indices", [=] { \
    switch (num_jagged_dim) {                                                 \
      case 1:                                                                 \
        INVOKE_KERNEL_WITH_DIM(1);                                            \
        break;                                                                \
      case 2:                                                                 \
        INVOKE_KERNEL_WITH_DIM(2);                                            \
        break;                                                                \
      case 3:                                                                 \
        INVOKE_KERNEL_WITH_DIM(3);                                            \
        break;                                                                \
      case 4:                                                                 \
        INVOKE_KERNEL_WITH_DIM(4);                                            \
        break;                                                                \
      case 5:                                                                 \
        INVOKE_KERNEL_WITH_DIM(5);                                            \
        break;                                                                \
      default:                                                                \
        TORCH_CHECK(                                                          \
            false, "unsupported number of jagged dim ", num_jagged_dim);      \
    }                                                                         \
  });

inline std::string torch_tensor_device_name(const at::Tensor& ten) {
  return c10::DeviceTypeName(ten.device().type());
}

inline std::string torch_tensor_device_name(
    const std::optional<at::Tensor>& ten) {
  if (ten.has_value()) {
    return torch_tensor_device_name(ten.value());
  } else {
    return "N/A";
  }
}

inline bool torch_tensor_on_cuda_gpu_check(const at::Tensor& ten) {
  return ten.is_cuda();
}

inline bool torch_tensor_on_cuda_gpu_check(
    const std::optional<at::Tensor>& ten) {
  return !ten.has_value() || torch_tensor_on_cuda_gpu_check(ten.value());
}

#define TENSOR_ON_CUDA_GPU(x)                                  \
  TORCH_CHECK(                                                 \
      torch_tensor_on_cuda_gpu_check(x),                       \
      #x " must be a CUDA tensor; it is currently on device ", \
      torch_tensor_device_name(x))

// A wrapper class for passing dynamically sized dimension information (e.g.
// tensor.dims()) from the host to device.
constexpr size_t kStackArrayMaxDims = 5;

template <typename T>
struct StackArray {
  T vals[kStackArrayMaxDims];
  size_t ndim;
};

// Warp size
#ifdef USE_ROCM
static constexpr int32_t kWarpSize = 64;
#else
static constexpr int32_t kWarpSize = 32;
#endif
// Max thread num in one thread block
static constexpr int32_t kMaxThreads = 1024;

#define DEVICE_INLINE __device__ C10_ALWAYS_INLINE

__host__ DEVICE_INLINE int32_t div_round_up(int32_t a, int32_t b) {
  return (a + b - 1) / b;
}

__host__ DEVICE_INLINE int32_t round_down(int32_t a, int32_t b) {
  return a / b * b;
}

inline std::tuple<dim3, dim3, StackArray<int64_t>> check_shape_and_partition_(
    const Tensor& values,
    const std::vector<Tensor>& offsets,
    const Tensor& dense_tensor) {
  const int outer_dense_size = dense_tensor.size(0);
  TORCH_CHECK(
      outer_dense_size == offsets[0].numel() - 1,
      "outer_dense_size, ",
      outer_dense_size,
      " != offsets[0].numel() - 1, ",
      offsets[0].numel() - 1);
  const int inner_dense_size = dense_tensor.size(-1);
  TORCH_CHECK(
      inner_dense_size == values.size(-1),
      "inner_dense_size, ",
      inner_dense_size,
      " != values.size(-1), ",
      values.size(-1));
  const int jagged_folded_size =
      dense_tensor.numel() / (outer_dense_size * inner_dense_size);

  const int threads_x =
      inner_dense_size >= kWarpSize / 2 ? kWarpSize : inner_dense_size;
  const int threads_y = kMaxThreads / kWarpSize;
  const dim3 blocks(
      div_round_up(outer_dense_size * jagged_folded_size, threads_y));

  StackArray<int64_t> jagged_dims_tensor{};
  const int num_jagged_dim = dense_tensor.dim() - 2;
  TORCH_CHECK(num_jagged_dim <= static_cast<int>(kStackArrayMaxDims));
  jagged_dims_tensor.ndim = num_jagged_dim;
  std::memcpy(
      &(jagged_dims_tensor.vals[0]),
      dense_tensor.sizes().data() + 1,
      num_jagged_dim * sizeof(int64_t));
  return {dim3(threads_x, threads_y), blocks, jagged_dims_tensor};
}

template <int NUM_JAGGED_DIM, typename index_t>
DEVICE_INLINE bool walk_down_tensor_storage_tree_(
    int& offset,
    const int flattened_jagged_idx,
    const StackArray<int64_t>& jagged_dims,
    const StackArray<index_t*>& x_offsets) {
  // compute coorindates
  int jagged_coords[NUM_JAGGED_DIM];
  int j_temp = flattened_jagged_idx;
#pragma unroll
  for (int d = NUM_JAGGED_DIM - 1; d >= 0; --d) {
    const int jagged_size = jagged_dims.vals[d];
    jagged_coords[d] = j_temp % jagged_size;
    j_temp /= jagged_size;
  }

  // walk down the tree
  bool is_zero = false;
#pragma unroll
  for (int d = 0; d < NUM_JAGGED_DIM; ++d) {
    const int begin = x_offsets.vals[d][offset];
    const int end = x_offsets.vals[d][offset + 1];
    if (jagged_coords[d] >= end - begin) {
      is_zero = true;
      break;
    }
    offset = begin + jagged_coords[d];
  }
  return is_zero;
}

// output = f(x, y) where x is jagged, y is dense, and output is dense.
// A generic elementwise operation between a jagged tensor and a dense tensor
// This kernel assumes jagged dims are clustered together, preceded by outer
// dense dimensions and followed by inner dense dimensions.
// The outer/inner dense dimensions, and jagged dimensions in between are
// assumed to be folded so physically the dense tensor is 3D and the value of
// jagged tensor is 2D.
// To support arbitrary number of jagged dimensions, we pass a vector of
// pointers to offset tensors (this is ugly and probably we can use nested
// tensor here).
// This kernel parallelizes the (folded) inner dense dimension across
// blockDim.x so the inner dense dimension should be similar to or bigger than
// warp size.
// We rely on compiler unrolling the compiler time constant NUM_JAGGED_DIM.
template <int NUM_JAGGED_DIM, typename index_t, typename scalar_t, typename F>
__global__
__launch_bounds__(kMaxThreads) void jagged_dense_elementwise_dense_output_kernel_(
    const at::PackedTensorAccessor32<scalar_t, 2, at::RestrictPtrTraits>
        x_values,
    StackArray<index_t*> x_offsets,
    const at::PackedTensorAccessor32<scalar_t, 3, at::RestrictPtrTraits> y,
    at::PackedTensorAccessor32<scalar_t, 3, at::RestrictPtrTraits> output,
    StackArray<int64_t> jagged_dims,
    F f,
    const scalar_t padding_value) {
  const int outer_dense_size = y.size(0);
  const int jagged_folded_size = y.size(1);
  const int inner_dense_size = y.size(2);

  const int outer_begin = blockIdx.x * blockDim.y + threadIdx.y;
  const int outer_stride = gridDim.x * blockDim.y;
  for (int outer = outer_begin; outer < outer_dense_size * jagged_folded_size;
       outer += outer_stride) {
    const int oidx = outer / jagged_folded_size;
    const int jidx = outer % jagged_folded_size;

    int offset = oidx;
    const bool is_zero = walk_down_tensor_storage_tree_<NUM_JAGGED_DIM>(
        offset, jidx, jagged_dims, x_offsets);

    if (is_zero) {
      int iidx;
      for (iidx = threadIdx.x; iidx * 2 + 1 < inner_dense_size;
           iidx += blockDim.x) {
        output[oidx][jidx][2 * iidx] =
            f(padding_value, y[oidx][jidx][2 * iidx]);
        output[oidx][jidx][2 * iidx + 1] =
            f(padding_value, y[oidx][jidx][2 * iidx + 1]);
      }
      if (iidx * 2 + 1 == inner_dense_size) {
        output[oidx][jidx][2 * iidx] =
            f(padding_value, y[oidx][jidx][2 * iidx]);
      }
    } else {
      int iidx;
      for (iidx = threadIdx.x; iidx * 2 + 1 < inner_dense_size;
           iidx += blockDim.x) {
        output[oidx][jidx][2 * iidx] =
            f(x_values[offset][2 * iidx], y[oidx][jidx][2 * iidx]);
        output[oidx][jidx][2 * iidx + 1] =
            f(x_values[offset][2 * iidx + 1], y[oidx][jidx][2 * iidx + 1]);
      }
      if (iidx * 2 + 1 == inner_dense_size) {
        output[oidx][jidx][2 * iidx] =
            f(x_values[offset][2 * iidx], y[oidx][jidx][2 * iidx]);
      }
    }
  }
}

template <typename scalar_t, typename F>
void jagged_dense_elementwise_dense_output_(
    const Tensor& x_values,
    const std::vector<Tensor>& x_offsets,
    const Tensor& y,
    const Tensor& output,
    F f,
    const scalar_t padding_value = static_cast<scalar_t>(0)) {
  TENSOR_ON_CUDA_GPU(x_values);
  for (auto& x_offset : x_offsets) {
    TENSOR_ON_CUDA_GPU(x_offset);
  }

  const int num_jagged_dim = y.dim() - 2;
  TORCH_CHECK(
      x_offsets.size() == static_cast<size_t>(num_jagged_dim),
      "x_offsets.size(), ",
      x_offsets.size(),
      " != num_jagged_dim ",
      num_jagged_dim);

  if (y.numel() == 0) {
    return;
  }

  dim3 threads, blocks;
  StackArray<int64_t> jagged_dims_tensor;
  std::tie(threads, blocks, jagged_dims_tensor) =
      check_shape_and_partition_(x_values, x_offsets, y);

  // Canonicalize y and output to 3D, collapsing jagged dimensions.
  const Tensor y_reshaped = y.view({y.size(0), -1, y.size(-1)});
  Tensor output_reshaped = output.view(y_reshaped.sizes());

#define INVOKE_KERNEL_WITH_DIM(NUM_JAGGED_DIM)                                \
  {                                                                           \
    std::vector<Tensor> x_offsets_contig;                                     \
    x_offsets_contig.resize(num_jagged_dim);                                  \
    StackArray<index_t*> x_offset_ptrs;                                       \
    x_offset_ptrs.ndim = num_jagged_dim;                                      \
    for (int d = 0; d < num_jagged_dim; ++d) {                                \
      x_offsets_contig[d] = x_offsets[d].contiguous();                        \
      x_offset_ptrs.vals[d] =                                                 \
          x_offsets_contig[d].template data_ptr<index_t>();                   \
    }                                                                         \
    jagged_dense_elementwise_dense_output_kernel_<NUM_JAGGED_DIM, index_t>    \
        <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(           \
            x_values.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(), \
            x_offset_ptrs,                                                    \
            y_reshaped                                                        \
                .packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),     \
            output_reshaped                                                   \
                .packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),     \
            jagged_dims_tensor,                                               \
            f,                                                                \
            padding_value);                                                   \
  }

  JAGGED_TENSOR_DISPATCH_DIMS();
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#undef INVOKE_KERNEL_WITH_DIM
}

#define INVOKE_KERNEL_WITH_DIM(NUM_JAGGED_DIM)                                 \
  {                                                                            \
    auto [threads, blocks, jagged_dims_tensor] =                               \
        check_shape_and_partition_(x_values, x_offsets, y);                    \
    blocks.x = div_round_up(x_values.size(0), threads.y);                      \
    std::vector<Tensor> x_offsets_contig;                                      \
    x_offsets_contig.resize(num_jagged_dim);                                   \
    StackArray<index_t*> x_offset_ptrs;                                        \
    x_offset_ptrs.ndim = num_jagged_dim;                                       \
    StackArray<int64_t> x_offset_sizes;                                        \
    x_offset_sizes.ndim = num_jagged_dim;                                      \
    for (int d = 0; d < num_jagged_dim; ++d) {                                 \
      x_offsets_contig[d] = x_offsets[d].contiguous();                         \
      x_offset_ptrs.vals[d] =                                                  \
          x_offsets_contig[d].template data_ptr<index_t>();                    \
      x_offset_sizes.vals[d] = x_offsets[d].numel();                           \
    }                                                                          \
    jagged_dense_dense_elementwise_jagged_output_kernel_<                      \
        NUM_JAGGED_DIM,                                                        \
        index_t><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(    \
        x_values.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),      \
        x_offset_ptrs,                                                         \
        x_offset_sizes,                                                        \
        y_reshaped.packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),    \
        y_reshaped.packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),    \
        output_values.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(), \
        jagged_dims_tensor,                                                    \
        [f] __device__(scalar_t x, scalar_t y, scalar_t /*unused*/)            \
            -> scalar_t { return f(x, y); });                                  \
  }

template <int NUM_JAGGED_DIM, typename index_t, typename scalar_t, typename F>
__global__
__launch_bounds__(kMaxThreads) void jagged_dense_dense_elementwise_jagged_output_kernel_(
    const at::PackedTensorAccessor32<scalar_t, 2, at::RestrictPtrTraits>
        x_values,
    StackArray<index_t*> x_offsets,
    StackArray<int64_t> x_offsets_sizes,
    const at::PackedTensorAccessor32<scalar_t, 3, at::RestrictPtrTraits> y_0,
    const at::PackedTensorAccessor32<scalar_t, 3, at::RestrictPtrTraits> y_1,
    at::PackedTensorAccessor32<scalar_t, 2, at::RestrictPtrTraits>
        output_values,
    StackArray<int64_t> jagged_dims,
    F f) {
  const int outer_dense_size = y_0.size(0);
  const int inner_dense_size = y_0.size(2);
  const int nnz = x_values.size(0);

  const int offset_begin = blockIdx.x * blockDim.y + threadIdx.y;
  const int offset_stride = gridDim.x * blockDim.y;
  for (int offset = offset_begin; offset < nnz; offset += offset_stride) {
    int offset_temp = offset;
    int jidx = 0;
    bool truncated = false;
    int dim_prod = 1;
#pragma unroll
    for (int d = NUM_JAGGED_DIM - 1; d >= 0; --d) {
      // Binary search the first that is bigger than offset
      int count = x_offsets_sizes.vals[d] - 1;
      int first = 1;
      while (count > 0) {
        int idx = first;
        int step = count / 2;
        idx += step;
        if (x_offsets.vals[d][idx] <= offset_temp) {
          first = ++idx;
          count -= step + 1;
        } else {
          count = step;
        }
      }

      --first;
      int coord = offset_temp - x_offsets.vals[d][first];
      if (coord >= jagged_dims.vals[d]) {
        truncated = true;
        break;
      }
      jidx += coord * dim_prod;
      dim_prod *= jagged_dims.vals[d];
      offset_temp = first;
    }

    if (offset_temp >= outer_dense_size) {
      // This can happen when values have more elements than the last element of
      // offset
      truncated = true;
    }
    if (!truncated) {
      const int oidx = offset_temp;
      int iidx = 0;
      for (iidx = threadIdx.x; iidx * 2 + 1 < inner_dense_size;
           iidx += blockDim.x) {
        output_values[offset][2 * iidx] =
            f(x_values[offset][2 * iidx],
              y_0[oidx][jidx][2 * iidx],
              y_1[oidx][jidx][2 * iidx]);
        output_values[offset][2 * iidx + 1] =
            f(x_values[offset][2 * iidx + 1],
              y_0[oidx][jidx][2 * iidx + 1],
              y_1[oidx][jidx][2 * iidx + 1]);
      }
      if (iidx * 2 + 1 == inner_dense_size) {
        output_values[offset][2 * iidx] =
            f(x_values[offset][2 * iidx],
              y_0[oidx][jidx][2 * iidx],
              y_1[oidx][jidx][2 * iidx]);
      }
    } else {
      int iidx;
      for (iidx = threadIdx.x; iidx * 2 + 1 < inner_dense_size;
           iidx += blockDim.x) {
        output_values[offset][2 * iidx] = f(x_values[offset][2 * iidx], 0, 0);
        output_values[offset][2 * iidx + 1] =
            f(x_values[offset][2 * iidx + 1], 0, 0);
      }
      if (iidx * 2 + 1 == inner_dense_size) {
        output_values[offset][2 * iidx] = f(x_values[offset][2 * iidx], 0, 0);
      }
    }
  }
}

///@addtogroup jagged-tensor-ops-cuda
template <typename scalar_t, typename F>
void jagged_dense_elementwise_jagged_output_(
    const Tensor& x_values,
    const std::vector<Tensor>& x_offsets,
    const Tensor& y,
    const Tensor& output_values,
    F f) {
  TENSOR_ON_CUDA_GPU(x_values);
  for (auto& x_offset : x_offsets) {
    TENSOR_ON_CUDA_GPU(x_offset);
  }

  const int num_jagged_dim = y.dim() - 2;
  TORCH_CHECK(
      x_offsets.size() == static_cast<size_t>(num_jagged_dim),
      "x_offsets.size(), ",
      x_offsets.size(),
      " != num_jagged_dim, ",
      num_jagged_dim);

  if (y.numel() == 0 || x_values.numel() == 0) {
    return;
  }

  // Canonicalize y to 3D, collapsing jagged dimensions.
  const Tensor y_reshaped = y.view({y.size(0), -1, y.size(-1)});

  JAGGED_TENSOR_DISPATCH_DIMS();
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

#undef INVOKE_KERNEL_WITH_DIM

template <typename T>
struct SharedMemory;

template <>
struct SharedMemory<int64_t> {
  __device__ int64_t* getPointer() {
    extern __shared__ int64_t s_int64_t[];
    return s_int64_t;
  }
};

template <>
struct SharedMemory<int32_t> {
  __device__ int32_t* getPointer() {
    extern __shared__ int32_t s_int32_t[];
    return s_int32_t;
  }
};

template <typename index_t>
__global__ void jagged_dense_dense_elementwise_jagged_output_opt_search_kernel_(
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    at::PackedTensorAccessor32<int, 1, at::RestrictPtrTraits> rows,
    at::PackedTensorAccessor32<int, 1, at::RestrictPtrTraits> cols,
    int nnz,
    int B) {
  struct SharedMemory<index_t> smem;
  index_t* offsets_sh = smem.getPointer();

  for (int i = threadIdx.x; i < B + 1; i += blockDim.x) {
    offsets_sh[i] = offsets[i];
  }
  __syncthreads();
  int row = threadIdx.x + blockIdx.x * blockDim.x;
  if (row >= nnz)
    return;
  int first = -1;
  int count = B - 1;
  first = 1;
  while (count > 0) {
    int idx = first;
    int step = count / 2;
    idx += step;
    if (offsets_sh[idx] <= row) {
      first = ++idx;
      count -= step + 1;
    } else {
      count = step;
    }
  }
  --first;

  int dense_row = first;
  int offset = offsets_sh[dense_row];
  int dense_col = row - offset;
  rows[row] = dense_row;
  cols[row] = dense_col;
}

struct VecType128 {
  typedef float4 TType; // Transaction Type
  typedef struct __align__(16) {
    __half a, b, c, d, w, x, y, z;
  }
  half8;

  union Data {
    half8 val;
    TType mask;
  } data;

  __device__ VecType128() {
    data.mask = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  }
};

struct VecType64 {
  typedef float2 TType; // Transaction Type
  typedef struct __align__(8) {
    __half a, b, c, d;
  }
  half4;

  union Data {
    half4 val;
    TType mask;
  } data;

  __device__ VecType64() {
    data.mask = make_float2(0.0f, 0.0f);
  }
};

struct VecType32 {
  typedef float TType; // Transaction Type

  union Data {
    __half2 val;
    TType mask;
  } data;

  __device__ VecType32() {
    data.mask = 0.0f;
  }
};

template <typename F>
__device__ void f128(
    VecType128& v_out,
    const VecType128& x,
    const VecType128& y0,
    const VecType128& y1,
    F f) {
  v_out.data.val.a = f(x.data.val.a, y0.data.val.a, y1.data.val.a);
  v_out.data.val.b = f(x.data.val.b, y0.data.val.b, y1.data.val.b);
  v_out.data.val.c = f(x.data.val.c, y0.data.val.c, y1.data.val.c);
  v_out.data.val.d = f(x.data.val.d, y0.data.val.d, y1.data.val.d);
  v_out.data.val.w = f(x.data.val.w, y0.data.val.w, y1.data.val.w);
  v_out.data.val.x = f(x.data.val.x, y0.data.val.x, y1.data.val.x);
  v_out.data.val.y = f(x.data.val.y, y0.data.val.y, y1.data.val.y);
  v_out.data.val.z = f(x.data.val.z, y0.data.val.z, y1.data.val.z);
}

template <typename F>
__device__ void f64(
    VecType64& v_out,
    const VecType64& x,
    const VecType64& y0,
    const VecType64& y1,
    F f) {
  v_out.data.val.a = f(x.data.val.a, y0.data.val.a, y1.data.val.a);
  v_out.data.val.b = f(x.data.val.b, y0.data.val.b, y1.data.val.b);
  v_out.data.val.c = f(x.data.val.c, y0.data.val.c, y1.data.val.c);
  v_out.data.val.d = f(x.data.val.d, y0.data.val.d, y1.data.val.d);
}

template <typename F>
__device__ void f32(
    VecType32& v_out,
    const VecType32& x,
    const VecType32& y0,
    const VecType32& y1,
    F f) {
  v_out.data.val = __halves2half2(
      f(__low2half(x.data.val),
        __low2half(y0.data.val),
        __low2half(y1.data.val)),
      f(__high2half(x.data.val),
        __high2half(y0.data.val),
        __high2half(y1.data.val)));
}

template <typename index_t, typename F>
__global__ void jagged_dense_dense_elementwise_jagged_output_opt_gather_kernel_(
    at::PackedTensorAccessor32<c10::Half, 2, at::RestrictPtrTraits> values,
    const at::PackedTensorAccessor32<c10::Half, 2, at::RestrictPtrTraits>
        x_values,
    const at::PackedTensorAccessor32<c10::Half, 3, at::RestrictPtrTraits> y0,
    const at::PackedTensorAccessor32<c10::Half, 3, at::RestrictPtrTraits> y1,
    const at::PackedTensorAccessor32<int, 1, at::RestrictPtrTraits> rows,
    const at::PackedTensorAccessor32<int, 1, at::RestrictPtrTraits> cols,
    const int nnz,
    const int E,
    F f) {
  int values_row = threadIdx.y + blockIdx.y * blockDim.y;
  if (values_row >= nnz)
    return;
  for (int real_row = values_row; real_row < nnz;
       real_row += blockDim.y * gridDim.y) {
    int dense_row = rows[real_row];
    int dense_col = cols[real_row];
    __half* values_ptr = reinterpret_cast<__half*>(&values[real_row][0]);
    const __half* x_ptr =
        reinterpret_cast<const __half*>(&x_values[real_row][0]);
    const __half* y0_ptr =
        reinterpret_cast<const __half*>(&y0[dense_row][dense_col][0]);
    const __half* y1_ptr =
        reinterpret_cast<const __half*>(&y1[dense_row][dense_col][0]);
    if ((dense_col < y0.size(1)) && (dense_row < y0.size(0)) &&
        (dense_col < y1.size(1)) && (dense_row < y1.size(0)) &&
        (dense_col >= 0) && (dense_row >= 0)) {
      for (int tid = threadIdx.x; tid < E / 8; tid += blockDim.x) {
        VecType128 v_x, v_out, v_y0, v_y1;
        v_x.data.mask =
            (reinterpret_cast<const VecType128::TType*>(x_ptr))[tid];
        v_y0.data.mask =
            (reinterpret_cast<const VecType128::TType*>(y0_ptr))[tid];
        v_y1.data.mask =
            (reinterpret_cast<const VecType128::TType*>(y1_ptr))[tid];
        f128(v_out, v_x, v_y0, v_y1, f);
        (reinterpret_cast<VecType128::TType*>(values_ptr))[tid] =
            v_out.data.mask;
      }
      for (int tid = threadIdx.x + (E / 8) * 8; tid < E / 4;
           tid += blockDim.x) {
        VecType64 v_x, v_out, v_y0, v_y1;
        v_x.data.mask = (reinterpret_cast<const VecType64::TType*>(x_ptr))[tid];
        v_y0.data.mask =
            (reinterpret_cast<const VecType64::TType*>(y0_ptr))[tid];
        v_y1.data.mask =
            (reinterpret_cast<const VecType64::TType*>(y1_ptr))[tid];
        f64(v_out, v_x, v_y0, v_y1, f);
        (reinterpret_cast<VecType64::TType*>(values_ptr))[tid] =
            v_out.data.mask;
      }
      for (int tid = threadIdx.x + (E / 4) * 4; tid < E / 2;
           tid += blockDim.x) {
        VecType32 v_x, v_out, v_y0, v_y1;
        v_x.data.mask = (reinterpret_cast<const VecType32::TType*>(x_ptr))[tid];
        v_y0.data.mask =
            (reinterpret_cast<const VecType32::TType*>(y0_ptr))[tid];
        v_y1.data.mask =
            (reinterpret_cast<const VecType32::TType*>(y1_ptr))[tid];
        f32(v_out, v_x, v_y0, v_y1, f);
        (reinterpret_cast<VecType32::TType*>(values_ptr))[tid] =
            v_out.data.mask;
      }
      for (int tid = threadIdx.x + (E / 2) * 2; tid < E; tid += blockDim.x) {
        auto v_x = static_cast<__half>(x_ptr[tid]);
        auto v_y0 = static_cast<__half>(y0_ptr[tid]);
        auto v_y1 = static_cast<__half>(y1_ptr[tid]);
        values_ptr[tid] = f(v_x, v_y0, v_y1);
      }
    } else {
      for (int tid = threadIdx.x; tid < E / 8; tid += blockDim.x) {
        VecType128 v_x, v_out, v_y0, v_y1;
        v_x.data.mask =
            (reinterpret_cast<const VecType128::TType*>(x_ptr))[tid];
        f128(v_out, v_x, v_y0, v_y1, f);
        (reinterpret_cast<VecType128::TType*>(values_ptr))[tid] =
            v_out.data.mask;
      }
      for (int tid = threadIdx.x + (E / 8) * 8; tid < E / 4;
           tid += blockDim.x) {
        VecType64 v_x, v_out, v_y0, v_y1;
        v_x.data.mask = (reinterpret_cast<const VecType64::TType*>(x_ptr))[tid];
        f64(v_out, v_x, v_y0, v_y1, f);
        (reinterpret_cast<VecType64::TType*>(values_ptr))[tid] =
            v_out.data.mask;
      }
      for (int tid = threadIdx.x + (E / 4) * 4; tid < E / 2;
           tid += blockDim.x) {
        VecType32 v_x, v_out, v_y0, v_y1;
        v_x.data.mask = (reinterpret_cast<const VecType32::TType*>(x_ptr))[tid];
        f32(v_out, v_x, v_y0, v_y1, f);
        (reinterpret_cast<VecType32::TType*>(values_ptr))[tid] =
            v_out.data.mask;
      }
      for (int tid = threadIdx.x + (E / 2) * 2; tid < E; tid += blockDim.x) {
        auto v_x = static_cast<__half>(x_ptr[tid]);
        values_ptr[tid] = f(v_x, __half{}, __half{});
      }
    }
  }
}

// Check to see if the inputs to the op are amenable to the fast path
inline bool jagged_dense_dense_elementwise_jagged_output_matches_opt(
    const int& num_jagged_dim,
    const Tensor& x_values,
    const std::vector<Tensor>& x_offsets,
    const Tensor& y_0_reshaped,
    const Tensor& y_1_reshaped,
    const Tensor& output_values) {
  bool matches = true;
  matches &= (num_jagged_dim == 1);

  // Unit stride embedding dim
  matches &= (x_values.stride(-1) == 1);
  matches &= (output_values.stride(-1) == 1);
  matches &= (y_0_reshaped.stride(-1) == 1);
  matches &= (y_1_reshaped.stride(-1) == 1);

  // Each row is aligned to 128-bit
  matches &= (x_values.stride(-2) % 8 == 0);
  matches &= (output_values.stride(-2) % 8 == 0);
  matches &= (y_0_reshaped.stride(-2) % 8 == 0);
  matches &= (y_1_reshaped.stride(-2) % 8 == 0);

  // Base addresses aligned to 128-bit
  matches &= (reinterpret_cast<uint64_t>(x_values.data_ptr()) % 16 == 0);
  matches &= (reinterpret_cast<uint64_t>(output_values.data_ptr()) % 16 == 0);
  matches &= (reinterpret_cast<uint64_t>(y_0_reshaped.data_ptr()) % 16 == 0);
  matches &= (reinterpret_cast<uint64_t>(y_1_reshaped.data_ptr()) % 16 == 0);

  // Rows and col fit into int32_t
  matches &= (y_0_reshaped.size(0) < INT_MAX);
  matches &= (y_0_reshaped.size(1) < INT_MAX);

  int max_shared_bytes = 0;
#ifndef USE_ROCM
  C10_CUDA_CHECK(hipDeviceGetAttribute(
      &max_shared_bytes,
      hipDeviceAttributeSharedMemPerBlockOptin,
      y_0_reshaped.get_device()));
#else
  // MI100 has 64 KB local memory (shared memory) per workgroup
  max_shared_bytes = 64 << 10;
#endif
  int shared_kb = max_shared_bytes >> 10;
#ifndef USE_ROCM
  // Use 2/3 of the available GPU shared mem; leave rooms for L1$.
  int used_shared_kb = round_down(shared_kb * 2 / 3, 16);
  TORCH_CHECK(used_shared_kb > 0);
#else
  // MI100 has independent shared mem and L1
  int used_shared_kb = shared_kb;
#endif
  auto used_shared_bytes = static_cast<size_t>(used_shared_kb << 10);
  AT_DISPATCH_INDEX_TYPES(
      x_offsets[0].scalar_type(), "check_shared_memory", [&] {
        auto B = y_0_reshaped.size(0);
        // the default shared memory on V100/A100/H100 is 48 KB from
        // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-8-x
        if ((B + 1) * sizeof(index_t) >= static_cast<size_t>(used_shared_bytes)) {
          matches = false;
        }
      });
  return matches;
}

#define INVOKE_KERNEL_WITH_DIM(NUM_JAGGED_DIM)                                 \
  {                                                                            \
    auto [threads, blocks, jagged_dims_tensor] =                               \
        check_shape_and_partition_(x_values, x_offsets, y);                    \
    blocks.x = div_round_up(x_values.size(0), threads.y);                      \
    std::vector<Tensor> x_offsets_contig;                                      \
    x_offsets_contig.resize(num_jagged_dim);                                   \
    StackArray<index_t*> x_offset_ptrs;                                        \
    x_offset_ptrs.ndim = num_jagged_dim;                                       \
    StackArray<int64_t> x_offset_sizes;                                        \
    x_offset_sizes.ndim = num_jagged_dim;                                      \
    for (int d = 0; d < num_jagged_dim; ++d) {                                 \
      x_offsets_contig[d] = x_offsets[d].contiguous();                         \
      x_offset_ptrs.vals[d] =                                                  \
          x_offsets_contig[d].template data_ptr<index_t>();                    \
      x_offset_sizes.vals[d] = x_offsets[d].numel();                           \
    }                                                                          \
    jagged_dense_dense_elementwise_jagged_output_kernel_<                      \
        NUM_JAGGED_DIM,                                                        \
        index_t><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(    \
        x_values.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),      \
        x_offset_ptrs,                                                         \
        x_offset_sizes,                                                        \
        y_reshaped.packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),    \
        y_reshaped.packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),    \
        output_values.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(), \
        jagged_dims_tensor,                                                    \
        [f] __device__(scalar_t x, scalar_t y, scalar_t /*unused*/)            \
            -> scalar_t { return f(x, y); });                                  \
  }

inline int calc_used_shared_bytes(const int device) {
    int max_shared_bytes;
#ifndef USE_ROCM
    C10_CUDA_CHECK(hipDeviceGetAttribute(
        &max_shared_bytes,
        hipDeviceAttributeSharedMemPerBlockOptin,
        device));
#else
    // MI100 has 64 KB local memory (shared memory) per workgroup
    max_shared_bytes = 64 << 10;
#endif
    int shared_kb = max_shared_bytes >> 10;
#ifndef USE_ROCM
    // Use 2/3 of the available GPU shared mem; leave rooms for L1$.
    int used_shared_kb = round_down(shared_kb * 2 / 3, 16);
    TORCH_CHECK(used_shared_kb > 0);
#else
    // MI100 has independent shared mem and L1
    int used_shared_kb = shared_kb;
#endif
    int used_shared_bytes = used_shared_kb << 10;
    return used_shared_bytes;
}

template <typename index_t>
inline void set_max_dynamic_shared_mem_size_for_opt_search_kernel(const int used_shared_bytes) {
#ifndef USE_ROCM
    C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
        jagged_dense_dense_elementwise_jagged_output_opt_search_kernel_<
            index_t>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        used_shared_bytes)); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
}

///@addtogroup jagged-tensor-ops-cuda
template <typename scalar_t, typename F>
void jagged_dense_elementwise_jagged_output_opt_(
    const Tensor& x_values,
    const std::vector<Tensor>& x_offsets,
    const Tensor& y,
    const Tensor& output_values,
    F f) {
  TENSOR_ON_CUDA_GPU(x_values);
  for (auto& x_offset : x_offsets) {
    TENSOR_ON_CUDA_GPU(x_offset);
  }

  const int num_jagged_dim = y.dim() - 2;
  TORCH_CHECK(
      x_offsets.size() == static_cast<size_t>(num_jagged_dim),
      "x_offsets.size(), ",
      x_offsets.size(),
      " != num_jagged_dim, ",
      num_jagged_dim);

  if (y.numel() == 0 || x_values.numel() == 0) {
    return;
  }

  // Canonicalize y to 3D, collapsing jagged dimensions.
  const Tensor y_reshaped = y.view({y.size(0), -1, y.size(-1)});
  if (jagged_dense_dense_elementwise_jagged_output_matches_opt(
          num_jagged_dim,
          x_values,
          x_offsets,
          y_reshaped,
          y_reshaped,
          output_values)) {
    AT_DISPATCH_INDEX_TYPES(
        x_offsets[0].scalar_type(), "jagged_indices_fast_path", [=] {
          auto nnz = output_values.size(0);
          auto B = y_reshaped.size(0);
          auto E = y_reshaped.size(2);
          Tensor t_rows_after_bs = at::empty(
              {nnz},
              at::TensorOptions().dtype(at::kInt).device(
                  at::kCUDA, at::cuda::current_device()));
          Tensor t_cols_after_bs = at::empty(
              {nnz},
              at::TensorOptions().dtype(at::kInt).device(
                  at::kCUDA, at::cuda::current_device()));

          // Binary search
          size_t dynamic_smem_size = (B + 1) * sizeof(index_t);
          auto cur_max_shared_bytes =
              at::cuda::getCurrentDeviceProperties()->sharedMemPerBlock;
          if (dynamic_smem_size > cur_max_shared_bytes) {
            int used_shared_bytes = calc_used_shared_bytes(y_reshaped.get_device());
            set_max_dynamic_shared_mem_size_for_opt_search_kernel<index_t>(used_shared_bytes);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            TORCH_CHECK(dynamic_smem_size <= static_cast<size_t>(used_shared_bytes));
          }
          dim3 threads_bs = dim3(1024, 1, 1);
          dim3 blocks_bs = dim3(div_round_up(nnz, threads_bs.x), 1, 1);
          jagged_dense_dense_elementwise_jagged_output_opt_search_kernel_<
              index_t>
              <<<blocks_bs,
                 threads_bs,
                 dynamic_smem_size,
                 at::cuda::getCurrentCUDAStream()>>>(
                  x_offsets[0]
                      .packed_accessor32<index_t, 1, at::RestrictPtrTraits>(),
                  t_rows_after_bs
                      .packed_accessor32<int, 1, at::RestrictPtrTraits>(),
                  t_cols_after_bs
                      .packed_accessor32<int, 1, at::RestrictPtrTraits>(),
                  nnz,
                  B);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          // Gather kernel
          dim3 threads = dim3(16, 16, 1);
          dim3 blocks = dim3(1, div_round_up(nnz, threads.y), 1);
          if (blocks.y > 65535) {
            blocks.y = 65535;
          }
          jagged_dense_dense_elementwise_jagged_output_opt_gather_kernel_<
              index_t>
              <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                  output_values
                      .packed_accessor32<c10::Half, 2, at::RestrictPtrTraits>(),
                  x_values
                      .packed_accessor32<c10::Half, 2, at::RestrictPtrTraits>(),
                  y_reshaped
                      .packed_accessor32<c10::Half, 3, at::RestrictPtrTraits>(),
                  y_reshaped
                      .packed_accessor32<c10::Half, 3, at::RestrictPtrTraits>(),
                  t_rows_after_bs
                      .packed_accessor32<int, 1, at::RestrictPtrTraits>(),
                  t_cols_after_bs
                      .packed_accessor32<int, 1, at::RestrictPtrTraits>(),
                  nnz,
                  E,
                  [f] __device__(__half x, __half y0, __half) -> __half {
                    // NB: added the static_casts here
                    return static_cast<__half>(
                        f(static_cast<scalar_t>(x), static_cast<scalar_t>(y0))
                    );
                  });
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        }); // AT_DISPATCH
  } else {
    JAGGED_TENSOR_DISPATCH_DIMS();
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

at::Tensor _fbgemm_jagged_to_padded_dense_forward(
    const Tensor& values,
    TensorList offsets,
    c10::IntArrayRef max_lengths,
    const double padding_value) {
  const size_t num_jagged_dim = offsets.size();
  TORCH_CHECK(
      max_lengths.size() == num_jagged_dim,
      "max_lengths.size(), ",
      max_lengths.size(),
      " != num_jagged_dim, ",
      num_jagged_dim);
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(values.get_device());

  const Tensor values_canonicalized = values.view(
      {values.size(0),
       std::accumulate(
           values.sizes().begin() + 1,
           values.sizes().end(),
           1,
           std::multiplies<size_t>())});
  at::SymDimVector padded_values_shape({at::SymInt(offsets[0].size(0) - 1)});
  padded_values_shape.insert(
      padded_values_shape.end(), max_lengths.begin(), max_lengths.end());

  // Canonicalize padded_values by unsqueeze the last dim if the inner dense
  // dimension is 1 and folded.
  const bool D_folded = values.dim() == 1;
  if (!D_folded) {
    padded_values_shape.push_back(values.size(-1));
  }
  Tensor padded_values =
      at::empty_symint(padded_values_shape, values.options());
  Tensor padded_values_view =
      D_folded ? padded_values.unsqueeze(-1) : padded_values;

  AT_DISPATCH_ALL_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      values.scalar_type(),
      "jagged_to_padded_dense",
      [&] {
        jagged_dense_elementwise_dense_output_<scalar_t>(
            values_canonicalized,
            offsets.vec(),
            padded_values_view, // dummy not used in the lambda function
            padded_values_view,
           [] __device__(scalar_t x, scalar_t /*unused*/) -> scalar_t {
              return x;
            },
            static_cast<scalar_t>(padding_value));
      });

  return padded_values;
}

#define DISPATCH_DENSE_TO_JAGGED_CASE(TYPE)                          \
  AT_DISPATCH_CASE(TYPE, [&] {                                       \
    jagged_dense_elementwise_jagged_output_opt_<scalar_t>(           \
        values,                                                      \
        offsets.vec(),                                               \
        dense,                                                       \
        output,                                                      \
        [] __device__(scalar_t /*unused*/, scalar_t y) -> scalar_t { \
          return y;                                                  \
        });                                                          \
  })

Tensor _fbgemm_dense_to_jagged_forward_symint(
    const Tensor& dense,
    TensorList offsets,
    std::optional<at::SymInt> total_L) {
  // D is the embedding dimension
  auto D = dense.size(-1);

  // If total_L is not given then compute it
  at::SymInt total_L_computed;
  if (total_L.has_value()) {
    total_L_computed = total_L.value();
  } else {
    total_L_computed = (int64_t)offsets.back().max().item<int64_t>();
  }
  auto values = at::empty_symint({total_L_computed, D}, dense.options());
  auto output = at::empty_like(values);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(dense.get_device());

  // clang-format off
  AT_DISPATCH_SWITCH(
      values.scalar_type(),
      "dense_to_jagged_gpu_op_forward",
      DISPATCH_DENSE_TO_JAGGED_CASE(at::ScalarType::Half)
      // NB: removed this to build
      // DISPATCH_DENSE_TO_JAGGED_CASE(at::ScalarType::Int)
      AT_DISPATCH_CASE_FLOATING_TYPES_AND2(
          at::ScalarType::Long,
          at::ScalarType::BFloat16,
          [&] {
            jagged_dense_elementwise_jagged_output_<scalar_t>(
                values,
                offsets.vec(),
                dense,
                output,
                [] __device__(scalar_t /*unused*/, scalar_t y) -> scalar_t {
                  return y;
                }); // device lambda
          } // lambda
          ) // CASE_FLOATING_TYPES_AND
  ); // SWITCH
  // clang-format on

#undef DISPATCH_DENSE_TO_JAGGED_CASE

  return output;
}

} // namespace native
} // namespace at
