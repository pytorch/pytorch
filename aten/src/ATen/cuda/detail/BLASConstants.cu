#include <ATen/Functions.h>
#include <ATen/Tensor.h>
#include <ATen/cuda/Exceptions.h>

#include <mutex>

namespace at {
namespace cuda {
namespace detail {

__device__ __constant__ float cublas_one_device;
__device__ __constant__ float cublas_zero_device;

float *get_cublas_device_one() {
  static c10::once_flag init_flag;

  c10::call_once(init_flag, []() {
    const float one = 1.f;
    AT_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cublas_one_device), &one, sizeof(float)));
  });

  float *ptr;
  AT_CUDA_CHECK(hipGetSymbolAddress(reinterpret_cast<void**>(&ptr), cublas_one_device));
  return ptr;
}

float *get_cublas_device_zero() {
  static c10::once_flag init_flag;

  c10::call_once(init_flag, []() {
    const float zero = 0.f;
    AT_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cublas_zero_device), &zero, sizeof(float)));
  });

  float *ptr;
  AT_CUDA_CHECK(hipGetSymbolAddress(reinterpret_cast<void**>(&ptr), cublas_zero_device));
  return ptr;
}

float *get_user_alpha_ptr() {
  static float *alpha_ptr;

  static c10::once_flag init_flag;

  c10::call_once(init_flag, []() {
    AT_CUDA_CHECK(hipMalloc(&alpha_ptr, sizeof(float)));
  });

  return alpha_ptr;
}

} // namespace detail
} // namespace cuda
} // namespace at
