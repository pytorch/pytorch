#include <ATen/cuda/cub-RadixSortPairs.cuh>

namespace at::cuda::hipcub::detail {

AT_INSTANTIATE_SORT_PAIRS(uint16_t, 8)

} // namespace at::cuda::hipcub::detail
