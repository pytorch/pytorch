#include <ATen/cuda/cub-RadixSortPairs.cuh>

namespace at::cuda::hipcub::detail {

AT_INSTANTIATE_SORT_PAIRS(int32_t, 2)

} // namespace at::cuda::hipcub::detail
