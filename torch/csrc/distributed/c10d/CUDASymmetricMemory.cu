#include "hip/hip_runtime.h"
#include <torch/csrc/distributed/c10d/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/CUDASymmetricMemory.hpp>
#include <torch/csrc/distributed/c10d/CUDASymmetricMemoryUtils.hpp>
#include <torch/csrc/distributed/c10d/cuda/utils.hpp>

#include <ATen/ceil_div.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/error.h>

#include <sys/socket.h>
#include <unistd.h>

#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
#include <c10/cuda/driver_api.h>
#elif defined(USE_ROCM)
#include <hip/hip_runtime_api.h>
#endif

#if defined(CUDART_VERSION) && CUDART_VERSION >= 12030
#define CUDART_SUPPORTS_MULTICAST
#endif

namespace c10d {
namespace symmetric_memory {

/* Start of CUDASymmetricMemory implementation */

// A set of exchange methods with prefix "CUDASymmetricMemory"
static StoreExchange storeExchange = StoreExchange("CUDASymmetricMemory");

AllocationRef::AllocationRef(
    void* ptr,
    HandleType handle,
    size_t block_size,
    int device_idx)
    : ptr(ptr),
      handle(handle),
      block_size(block_size),
      device_idx(device_idx) {}

AllocationRef::~AllocationRef() {
  if (is_finalizing()) {
    return;
  }
  c10::cuda::CUDAGuard guard(device_idx);
  C10_CUDA_CHECK(hipDeviceSynchronize());
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  // Leak the cuda allocations during static deinitialization
  auto driver_api = c10::cuda::DriverAPI::get();
  C10_CUDA_DRIVER_CHECK(
      driver_api->cuMemUnmap_(reinterpret_cast<hipDeviceptr_t>(ptr), block_size));
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemRelease_(handle));
#elif defined(USE_ROCM)
  C10_HIP_CHECK(hipMemUnmap(reinterpret_cast<hipDeviceptr_t>(ptr), block_size));
  C10_HIP_CHECK(hipMemRelease(handle));
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

CUDASymmetricMemory::CUDASymmetricMemory(
    std::vector<c10::intrusive_ptr<AllocationRef>> alloc_refs,
    std::vector<void*> buffers,
    std::vector<void*> signal_pads,
    HandleType mc_handle,
    void* mc_addr,
    size_t buffer_size,
    int local_device_idx,
    int rank,
    int world_size)
    : alloc_refs_(std::move(alloc_refs)),
      buffers_(std::move(buffers)),
      signal_pads_(std::move(signal_pads)),
      mc_handle_(mc_handle),
      mc_addr_(mc_addr),
      buffer_size_(buffer_size),
      local_device_idx_(local_device_idx),
      rank_(rank),
      world_size_(world_size) {
  const size_t arr_size = sizeof(void*) * world_size_;
  buffers_dev_ = reinterpret_cast<void**>(
      c10::cuda::CUDACachingAllocator::raw_alloc(arr_size));
  signal_pads_dev_ = reinterpret_cast<void**>(
      c10::cuda::CUDACachingAllocator::raw_alloc(arr_size));

  c10::cuda::CUDAGuard guard(local_device_idx);
  AT_CUDA_CHECK(hipMemcpy(
      buffers_dev_, buffers_.data(), arr_size, hipMemcpyHostToDevice));
  AT_CUDA_CHECK(hipMemcpy(
      signal_pads_dev_, signal_pads_.data(), arr_size, hipMemcpyHostToDevice));
}

std::vector<void*> CUDASymmetricMemory::get_buffer_ptrs() {
  return buffers_;
}

std::vector<void*> CUDASymmetricMemory::get_signal_pad_ptrs() {
  return signal_pads_;
}

void** CUDASymmetricMemory::get_buffer_ptrs_dev() {
  return buffers_dev_;
}

void** CUDASymmetricMemory::get_signal_pad_ptrs_dev() {
  return signal_pads_dev_;
}

size_t CUDASymmetricMemory::get_buffer_size() {
  return buffer_size_;
}

size_t CUDASymmetricMemory::get_signal_pad_size() {
  return signal_pad_size;
}

bool CUDASymmetricMemory::has_multicast_support() {
  return mc_addr_ != nullptr;
}

void* CUDASymmetricMemory::get_multicast_ptr() {
  return mc_addr_;
}

at::Tensor CUDASymmetricMemory::get_buffer(
    int rank,
    c10::IntArrayRef sizes,
    c10::ScalarType dtype,
    int64_t storage_offset) {
  const size_t numel = std::accumulate(
      sizes.begin(),
      sizes.end(),
      static_cast<size_t>(1),
      std::multiplies<size_t>());
  const auto element_size = c10::elementSize(dtype);
  const auto req_size = (numel + storage_offset) * element_size;
  TORCH_CHECK(
      req_size <= buffer_size_,
      "CUDASymmetricMemory::get_buffer: the requested size (",
      req_size,
      " bytes) exceeds the allocated size (",
      buffer_size_,
      " bytes)");
  auto data_ptr = reinterpret_cast<uint8_t*>(buffers_[rank]) +
      storage_offset * element_size;
  auto device = c10::Device(c10::DeviceType::CUDA, local_device_idx_);
  auto options = at::TensorOptions().dtype(dtype).device(device);
  return at::for_blob(data_ptr, sizes)
      .options(options)
      .target_device(device)
      .make_tensor();
}

at::Tensor CUDASymmetricMemory::get_signal_pad(
    int rank,
    c10::IntArrayRef sizes,
    std::optional<c10::ScalarType> dtype,
    int64_t storage_offset) {
  // If the dtype is unspecified, default it to UInt32, as it
  // is the most common type for signaling purposes.
  if (!dtype.has_value()) {
    dtype = c10::ScalarType::UInt32;
  }

  // If the shape is unspecified, treat the signal pad as a 1d tensor.
  const auto element_size = c10::elementSize(*dtype);
  std::vector<int64_t> shape;
  if (sizes.size() != 0) {
    shape = sizes.vec();
  } else {
    shape.push_back(signal_pad_size / element_size);
  }

  const size_t numel = std::accumulate(
      shape.begin(),
      shape.end(),
      static_cast<size_t>(1),
      std::multiplies<size_t>());
  const auto req_size = (numel + storage_offset) * element_size;
  TORCH_CHECK(
      req_size <= signal_pad_size,
      "CUDASymmetricMemory::get_signal_pad: the requested size (",
      req_size,
      " bytes) exceeds the allocated size (",
      signal_pad_size,
      " bytes)");
  auto data_ptr = reinterpret_cast<uint8_t*>(signal_pads_[rank]) +
      storage_offset * element_size;
  auto device = c10::Device(c10::DeviceType::CUDA, local_device_idx_);
  auto options = at::TensorOptions().dtype(*dtype).device(device);
  return at::for_blob(data_ptr, shape)
      .options(options)
      .target_device(device)
      .make_tensor();
}

void check_channel(int channel, int world_size) {
  TORCH_CHECK(
      channel >= 0,
      "channel for barrier(), put_signal() and wait_signal() ",
      "must be greater than 0 (got ",
      channel,
      ")");
  const size_t num_channels = signal_pad_size / sizeof(uint32_t) * world_size;
  TORCH_CHECK(
      static_cast<size_t>(channel) < num_channels,
      "The maximum supported channel for barrier(), put_signal() and wait_signal() is ",
      num_channels - 1,
      " (got ",
      channel,
      ")");
}

static __global__ void barrier_kernel(
    uint32_t** signal_pads,
    int channel,
    int rank,
    int world_size,
    size_t timeout_ms) {
  if (threadIdx.x < world_size) {
    auto target_rank = threadIdx.x;
    if (target_rank == rank) {
      return;
    }
    auto put_success = try_put_signal<std::memory_order_release>(
        signal_pads[target_rank] + world_size * channel + rank, timeout_ms);
    if (!put_success) {
      printf(
          "[FATAL] CUDASymmetricMemory::barrier: rank %d failed to send signal "
          "to rank %d on channel %d after %lu microseconds\n",
          rank,
          target_rank,
          channel,
          timeout_ms);
      trap();
    }
    auto wait_success = try_wait_signal<std::memory_order_acquire>(
        signal_pads[rank] + world_size * channel + target_rank, timeout_ms);
    if (!wait_success) {
      printf(
          "[FATAL] CUDASymmetricMemory::barrier: rank %d failed to receive signal "
          "from rank %d on channel %d after %lu microseconds\n",
          rank,
          target_rank,
          channel,
          timeout_ms);
      trap();
    }
  }
}

void CUDASymmetricMemory::barrier(int channel, size_t timeout_ms) {
  check_channel(channel, world_size_);
  c10::cuda::CUDAGuard guard(local_device_idx_);
  barrier_kernel<<<1, C10_WARP_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<uint32_t**>(signal_pads_dev_),
      channel,
      rank_,
      world_size_,
      timeout_ms);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

static __global__ void put_signal_kernel(
    uint32_t** signal_pads,
    int dst_rank,
    int channel,
    int rank,
    int world_size,
    size_t timeout_ms) {
  if (threadIdx.x == 0) {
    bool success = try_put_signal<std::memory_order_release>(
        signal_pads[dst_rank] + world_size * channel + rank, timeout_ms);
    if (!success) {
      printf(
          "[FATAL] CUDASymmetricMemory::put_signal: rank %d failed to send signal "
          "to rank %d on channel %d after %lu microseconds\n",
          rank,
          dst_rank,
          channel,
          timeout_ms);
      trap();
    }
  }
}

void CUDASymmetricMemory::put_signal(
    int dst_rank,
    int channel,
    size_t timeout_ms) {
  check_channel(channel, world_size_);
  c10::cuda::CUDAGuard guard(local_device_idx_);
  put_signal_kernel<<<1, C10_WARP_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<uint32_t**>(signal_pads_dev_),
      dst_rank,
      channel,
      rank_,
      world_size_,
      timeout_ms);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

static __global__ void wait_signal_kernel(
    uint32_t** signal_pads,
    int src_rank,
    int channel,
    int rank,
    int world_size,
    size_t timeout_ms) {
  if (threadIdx.x == 0) {
    bool success = try_wait_signal<std::memory_order_acquire>(
        signal_pads[rank] + world_size * channel + src_rank, timeout_ms);
    if (!success) {
      printf(
          "[FATAL] CUDASymmetricMemory::wait_signal rank %d failed to receive signal "
          "from rank %d on channel %d after %lu microseconds\n",
          rank,
          src_rank,
          channel,
          timeout_ms);
#if !defined(USE_ROCM)
      __trap();
#else
      assert(0);
#endif
    }
  }
  __threadfence_system();
}

void CUDASymmetricMemory::wait_signal(
    int src_rank,
    int channel,
    size_t timeout_ms) {
  check_channel(channel, world_size_);
  c10::cuda::CUDAGuard guard(local_device_idx_);
  wait_signal_kernel<<<1, C10_WARP_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<uint32_t**>(signal_pads_dev_),
      src_rank,
      channel,
      rank_,
      world_size_,
      timeout_ms);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

int CUDASymmetricMemory::get_rank() {
  return rank_;
}

int CUDASymmetricMemory::get_world_size() {
  return world_size_;
}

Block::Block(
    c10::intrusive_ptr<AllocationRef> alloc_ref,
    int device_idx,
    size_t block_size,
    size_t buffer_size,
    size_t signal_pad_offset,
    const std::optional<std::string>& group_name)
    : alloc_ref(std::move(alloc_ref)),
      device_idx(device_idx),
      block_size(block_size),
      buffer_size(buffer_size),
      signal_pad_offset(signal_pad_offset),
      default_group_name(std::move(group_name)) {}

void* CUDASymmetricMemoryAllocator::alloc(
    size_t size,
    int device_idx,
    const std::optional<std::string>& group_name) {

  size_t signal_pad_offset = at::round_up(size, 16UL);
  size_t block_size = signal_pad_offset + signal_pad_size;
  c10::cuda::CUDAGuard guard(device_idx);
  device_idx = static_cast<int>(guard.current_device().index());
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  hipMemAllocationProp prop = {};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  // NOLINTNEXTLINE(bugprone-signed-char-misuse)
  prop.location.id = device_idx;
  prop.requestedHandleTypes = hipMemHandleTypePosixFileDescriptor;


  size_t granularity;
  auto driver_api = c10::cuda::DriverAPI::get();
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemGetAllocationGranularity_(
      &granularity, &prop, hipMemAllocationGranularityRecommended));
  block_size = at::round_up(block_size, granularity);

  HandleType handle;
  C10_CUDA_DRIVER_CHECK(
      driver_api->cuMemCreate_(&handle, block_size, &prop, 0));

#elif defined(USE_ROCM)
  hipMemAllocationProp prop = {};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  // NOLINTNEXTLINE(bugprone-signed-char-misuse)
  prop.location.id = device_idx;
  prop.requestedHandleType = hipMemHandleTypePosixFileDescriptor;


  size_t granularity;
  C10_HIP_CHECK(hipMemGetAllocationGranularity(
      &granularity, &prop, hipMemAllocationGranularityRecommended));
  block_size = at::round_up(block_size, granularity);

  HandleType handle;
  C10_HIP_CHECK(hipMemCreate(reinterpret_cast<hipMemGenericAllocationHandle_t*>(&handle), block_size, &prop, 0));

#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
  void* ptr = nullptr;
  map_block(&ptr, handle, block_size, device_idx);

  AT_CUDA_CHECK(hipMemset(ptr, 0, block_size));

  auto alloc_ref =
      c10::make_intrusive<AllocationRef>(ptr, handle, block_size, device_idx);
  auto block = c10::make_intrusive<Block>(
      std::move(alloc_ref),
      device_idx,
      block_size,
      size,
      signal_pad_offset,
      group_name);
  {
    std::unique_lock lock(mutex_);
    ptr_to_block_.emplace(ptr, std::move(block));
  }
  return ptr;
}

void CUDASymmetricMemoryAllocator::free(void* ptr) {
  std::unique_lock lock(mutex_);
  ptr_to_block_.erase(ptr);
}

size_t CUDASymmetricMemoryAllocator::get_alloc_size(void* ptr) {
  auto block = find_block(ptr);
  TORCH_CHECK(
      block != nullptr,
      "CUDASymmetricMemoryAllocator::get_alloc_size: input must be allocated ",
      "via CUDASymmetricMemoryAllocator::alloc");
  return block->buffer_size;
}

struct RendezvousRequest {
  int device_idx;
  int pid;
  size_t block_size;
  size_t buffer_size;
  size_t signal_pad_offset;
  bool has_multicast_support;
};

void validate_rendezvous_requests(
    const std::vector<RendezvousRequest>& reqs,
    int world_size) {
  TORCH_CHECK(reqs.size() == (size_t)world_size);

  std::unordered_set<int> device_indices;
  device_indices.reserve(world_size);
  for (auto req : reqs) {
    device_indices.insert(req.device_idx);
  }
  if (!allow_overlapping_devices() &&
      device_indices.size() < (size_t)world_size) {
    TORCH_CHECK(
        false,
        "CUDASymmetricMemoryAllocator::rendezvous: ",
        "detected allocations from overlapping devices ",
        "from different ranks.");
  }

  for (int r = 1; r < world_size; ++r) {
    TORCH_CHECK(reqs[r].block_size == reqs[0].block_size);
    TORCH_CHECK(reqs[r].buffer_size == reqs[0].buffer_size);
    TORCH_CHECK(reqs[r].signal_pad_offset == reqs[0].signal_pad_offset);
  }
}

static bool check_group_multicast_support(
    const std::vector<RendezvousRequest>& reqs) {
  std::vector<size_t> ranks_with_multicast_support;
  for (size_t r = 0; r < reqs.size(); ++r) {
    if (reqs[r].has_multicast_support) {
      ranks_with_multicast_support.push_back(r);
    }
  }
  if (ranks_with_multicast_support.size() == reqs.size()) {
    return true;
  } else {
    // We don't expect this to happen. But we want to let the user to know if
    // this happens.
    if (ranks_with_multicast_support.size() != 0) {
      LOG(WARNING)
          << "Only a subset of ranks in the group has multicast support: "
          << ranks_with_multicast_support << " (world_size=" << reqs.size()
          << "). Skipping multicast initialization because this is unexpected.";
    }
    return false;
  }
}

static void init_multicast_for_block(
    HandleType& mc_handle,
    void*& mc_addr,
    const c10::intrusive_ptr<Block>& block,
    IpcChannel& ipc_channel,
    const std::vector<int>& pids,
    const c10::intrusive_ptr<c10d::Store>& store,
    int rank,
    int world_size) {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED) && \
    defined(CUDART_SUPPORTS_MULTICAST)
  auto driver_api = c10::cuda::DriverAPI::get();
  if (rank == 0) {
    CUmulticastObjectProp mc_prop{};
    mc_prop.numDevices = world_size;
    mc_prop.handleTypes = hipMemHandleTypePosixFileDescriptor;
    mc_prop.size = block->block_size;

    // create a multicast object, which acts as a handle that allows multiple
    // devices or processes to access the same memory allocation coherently.
    auto err = driver_api->cuMulticastCreate_(&mc_handle, &mc_prop);
    if (err != hipSuccess) {
      const char* err_str;
      hipError_t get_error_str_err = driver_api->cuGetErrorString_(err, &err_str);
      if (get_error_str_err != hipSuccess) {
        err_str = "unknown cuda driver error";
      }
      LOG(WARNING)
          << "SymmetricMemory: cuMulticastCreate failed with: \"" << err_str
          << "\". Gracefully skipping multicast initialization. "
          << "However, this is unexpected. Please report the issue on GitHub.";
      // Allow peers gracefully skip multicast initialization by sending -1
      ipc_channel.broadcast_fds(rank, 0, pids, -1);
      return;
    }

    int mc_fd;
    // using the CUDA Driver API to export a multicast object into a POSIX file descriptor.
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemExportToShareableHandle_(
        &mc_fd, mc_handle, hipMemHandleTypePosixFileDescriptor, 0));
    ipc_channel.broadcast_fds(rank, 0, pids, mc_fd);
    // Ref count is incremented as soon as SCM_RIGHTS send happens
    close(mc_fd);
  } else {
    int mc_fd = ipc_channel.broadcast_fds(rank, 0, pids, -1);
    if (mc_fd == -1) {
      return;
    }
    // Convert back to a handle from the broadcasted POSIX file descriptor.
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemImportFromShareableHandle_(
        &mc_handle,
        (void*)(uintptr_t)mc_fd,
        hipMemHandleTypePosixFileDescriptor));
    close(mc_fd);
  }

  // All rank adds their physical allocation to the multicast object
  C10_CUDA_DRIVER_CHECK(
      driver_api->cuMulticastAddDevice_(mc_handle, block->device_idx));
  C10_CUDA_DRIVER_CHECK(driver_api->cuMulticastBindMem_(
      mc_handle, 0, block->alloc_ref->handle, 0, block->block_size, 0));

  map_block(&mc_addr, mc_handle, block->block_size, block->device_idx);
  storeExchange.barrier(store, rank, world_size);
#endif
}

c10::intrusive_ptr<SymmetricMemory> CUDASymmetricMemoryAllocator::rendezvous(
    void* ptr,
    const std::optional<std::string>& group_name) {

  auto block = find_block(ptr);
  if (block == nullptr) {
    return nullptr;
  }

  // The group_name passed to rendezvous() takes precedence over
  // the default group_name specified during allocation.
  std::string group_name_;
  // Treat empty string and std::nullopt the same as empty string seems to be
  // implicitly used that way
  if (group_name.has_value() && group_name != "") {
    group_name_ = *group_name;
  } else {
    if (!block->default_group_name.has_value()) {
      TORCH_CHECK(
          false,
          "CUDASymmetricMemory::rendezvous: `group_name` is neither "
          "specified during allocation nor passed to rendezvous().");
    }
    group_name_ = *block->default_group_name;
  }

  auto it = block->symm_mems.find(group_name_);
  if (it != block->symm_mems.end()) {
    return it->second;
  }

  c10::cuda::CUDAGuard guard(block->device_idx);

  // Currently, IpcChannel is using a file based socket for inter-process communication
  IpcChannel ipc_channel;
  auto group_info = get_group_info(group_name_);
  auto store = group_info.store;
  int rank = group_info.rank;
  int world_size = group_info.world_size;
  int block_fd;

#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  auto driver_api = c10::cuda::DriverAPI::get();
  // using the CUDA Driver API to export a GPU memory block as a
  // POSIX file descriptor (FD), so it can be shared across processes via IPC.
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemExportToShareableHandle_(
      &block_fd,
      block->alloc_ref->handle,
      hipMemHandleTypePosixFileDescriptor,
      0));
#elif defined (USE_ROCM)
  C10_HIP_CHECK(hipMemExportToShareableHandle(
      &block_fd, block->alloc_ref->handle, hipMemHandleTypePosixFileDescriptor, 0));
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif

  auto local_req = RendezvousRequest{
      .device_idx = block->device_idx,
      .pid = getpid(),
      .block_size = block->block_size,
      .buffer_size = block->buffer_size,
      .signal_pad_offset = block->signal_pad_offset,
      .has_multicast_support = device_has_multicast_support(block->device_idx)};
  auto reqs = storeExchange.all_gather(store, rank, world_size, local_req);
  validate_rendezvous_requests(reqs, world_size);

  std::vector<int> pids(world_size);
  for (int r = 0; r < world_size; ++r) {
    pids[r] = reqs[r].pid;
  }
  auto imported_fds = ipc_channel.all_gather_fds(rank, pids, block_fd);

  std::vector<HandleType> handles(world_size);
  std::vector<void*> buffers(world_size, nullptr);
  std::vector<void*> signal_pads(world_size, nullptr);

  for (int r = 0; r < world_size; ++r) {
    if (r == rank) {
      handles[r] = block->alloc_ref->handle;
      buffers[r] = ptr;
      signal_pads[r] = (void*)((uintptr_t)ptr + block->signal_pad_offset);
      continue;
    }
    // This api imports a GPU memory allocation that was previously exported as a file
    // descriptor and it returns a memory handle.
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemImportFromShareableHandle_(
        &handles[r],
        (void*)(uintptr_t)imported_fds[r],
        hipMemHandleTypePosixFileDescriptor));
#elif defined (USE_ROCM)
    C10_HIP_CHECK(hipMemImportFromShareableHandle(
        &handles[r],
        (void*)(uintptr_t)&(imported_fds[r]),
        hipMemHandleTypePosixFileDescriptor));
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
    map_block(&buffers[r], handles[r], block->block_size, block->device_idx);
    signal_pads[r] = (void*)((uintptr_t)buffers[r] + block->signal_pad_offset);
    close(imported_fds[r]);
  }
  storeExchange.barrier(store, rank, world_size);
  close(block_fd);

  HandleType mc_handle{};
  void* mc_addr = nullptr;
  bool group_has_multicast_support = check_group_multicast_support(reqs);
  if (!allow_overlapping_devices() && group_has_multicast_support) {
    init_multicast_for_block(
        mc_handle, mc_addr, block, ipc_channel, pids, store, rank, world_size);
  }

  std::vector<c10::intrusive_ptr<AllocationRef>> alloc_refs;
  for (int r = 0; r < world_size; ++r) {
    if (r == rank) {
      alloc_refs.emplace_back(block->alloc_ref);
      continue;
    }
    alloc_refs.push_back(c10::make_intrusive<AllocationRef>(
        buffers[r], handles[r], block->block_size, block->device_idx));
  }

  auto symm_mem = c10::make_intrusive<CUDASymmetricMemory>(
      std::move(alloc_refs),
      std::move(buffers),
      std::move(signal_pads),
      mc_handle,
      mc_addr,
      block->buffer_size,
      block->device_idx,
      group_info.rank,
      group_info.world_size);
  block->symm_mems[group_name_] = symm_mem;
  return symm_mem;
}

bool CUDASymmetricMemoryAllocator::has_multicast_support(int device_idx) {
  return device_has_multicast_support(device_idx);
}

c10::intrusive_ptr<Block> CUDASymmetricMemoryAllocator::find_block(void* ptr) {
  std::shared_lock lock(mutex_);
  auto it = ptr_to_block_.find(ptr);
  if (it == ptr_to_block_.end()) {
    return nullptr;
  }
  return it->second;
}

struct RegisterCUDASymmetricMemoryAllocator {
  RegisterCUDASymmetricMemoryAllocator() {
    // Query backend used for CUDA tensor
    // "CUDA" backend stands for this implementation
    if (getSymmMemBackendCUDA() == "CUDA") {
      register_allocator(
          c10::DeviceType::CUDA,
          c10::make_intrusive<CUDASymmetricMemoryAllocator>());
    }
  }
};

static RegisterCUDASymmetricMemoryAllocator register_allocator_;

} // namespace symmetric_memory
} // namespace c10d
