#include "hip/hip_runtime.h"
#include <torch/csrc/distributed/c10d/nvshmem_extension.cuh>

#include <c10/cuda/CUDAGuard.h>

#include <torch/csrc/distributed/c10d/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/CUDASymmetricMemoryUtils.hpp>
#include <torch/csrc/distributed/c10d/SymmetricMemory.hpp>

#include <cuda_awbarrier_primitives.h>
// Use torch's cub wrapper instead of CUDA's <hipcub/hipcub.hpp>, see #55292
#include <ATen/cuda/cub.cuh>
#include <nvshmem.h>

namespace c10d::nvshmem_extension {

using c10d::symmetric_memory::StoreExchange;
static StoreExchange storeExchange = StoreExchange("nvshmem_ext");

#define THREADS_PER_BLOCK 512

// Bootstrap based on user's setting for NCCL
// Long term, this may be a bit unclean; short term, it improves UX
void maybe_initialize_env_vars() {
  auto nccl_socket_if_name = c10::utils::get_env("NCCL_SOCKET_IFNAME");
  auto nccl_hca_list = c10::utils::get_env("NCCL_IB_HCA");
  auto nccl_ib_gid_index = c10::utils::get_env("NCCL_IB_GID_INDEX");
  auto nvshmem_socket_if_name =
      c10::utils::get_env("NVSHMEM_BOOTSTRAP_UID_SOCK_IFNAME");
  auto nvshmem_hca_list = c10::utils::get_env("NCCL_IB_HCA");
  auto nvshmem_ib_gid_index = c10::utils::get_env("NVSHMEM_IB_GID_INDEX");

  if (!nvshmem_socket_if_name.has_value() && nccl_socket_if_name.has_value()) {
    c10::utils::set_env(
        "NVSHMEM_BOOTSTRAP_UID_SOCK_IFNAME", nccl_socket_if_name->c_str());
  }
  if (!nvshmem_hca_list.has_value() && nccl_hca_list.has_value()) {
    c10::utils::set_env("NVSHMEM_ENABLE_NIC_PE_MAPPING", "1");
    c10::utils::set_env("NVSHMEM_HCA_LIST", nccl_hca_list->c_str());
  }
  if (!nvshmem_ib_gid_index.has_value() && nccl_ib_gid_index.has_value()) {
    c10::utils::set_env("NVSHMEM_IB_GID_INDEX", nccl_ib_gid_index->c_str());
  }
}

void initialize_nvshmem_with_store(
    c10::intrusive_ptr<c10d::Store> store,
    int rank,
    int world_size) {
  static bool is_initialized = false;
  if (is_initialized) {
    return;
  }

  maybe_initialize_env_vars();

  nvshmemx_uniqueid_t unique_id;
  TORCH_CHECK(
      nvshmemx_get_uniqueid(&unique_id) == 0, "nvshmemx_get_uniqueid failed");

  // Using an existing store_all_gather due to laziness.
  // TODO(yifu): should use broadcast
  auto unique_ids = storeExchange.all_gather(store, rank, world_size, unique_id);

  nvshmemx_init_attr_t attr;
  nvshmemx_set_attr_uniqueid_args(rank, world_size, &unique_ids[0], &attr);

  TORCH_CHECK(
      nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr) == 0,
      "nvshmemx_init_attr failed");

  is_initialized = true;
}

void* nvshmem_malloc(size_t size) {
  return ::nvshmem_malloc(size);
}

void* nvshmem_ptr(const void* dest, int pe) {
  return ::nvshmem_ptr(dest, pe);
}

std::unordered_map<std::string, nvshmem_team_t> group_name_to_team_;

nvshmem_team_t group_to_team(
    const std::string& group_name,
    const std::vector<int>& global_ranks) {
  auto it = group_name_to_team_.find(group_name);
  if (it != group_name_to_team_.end()) {
    return it->second;
  }
  TORCH_CHECK(global_ranks.size() > 1);
  int stride = global_ranks[1] - global_ranks[0];
  for (size_t r = 1; r < global_ranks.size(); ++r) {
    TORCH_CHECK(global_ranks[r] - global_ranks[r - 1] == stride);
  }

  nvshmem_team_t team;
  TORCH_CHECK(
      nvshmem_team_split_strided(
          NVSHMEM_TEAM_WORLD,
          global_ranks[0],
          stride,
          global_ranks.size(),
          nullptr,
          0,
          &team) == 0);
  group_name_to_team_[group_name] = team;
  TORCH_CHECK(team != NVSHMEM_TEAM_INVALID);
  return team;
}

at::Tensor nvshmem_broadcast(at::Tensor& input, const std::string& group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto team = group_to_team(group_name, input_hdl->get_rank_to_global_rank());
  void* buffer_ptr = input_hdl->get_buffer_ptrs()[rank];

  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_broadcastmem_on_stream(team, buffer_ptr, buffer_ptr, input_hdl->get_buffer_size(), 0, stream);
  return input;
}

at::Tensor nvshmem_all_to_all(
    at::Tensor& input,
    at::Tensor& out,
    std::string group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto team = group_to_team(group_name, input_hdl->get_rank_to_global_rank());

  void* input_ptr = input_hdl->get_buffer_ptrs()[rank];
  void* output_ptr = out_hdl->get_buffer_ptrs()[rank];
  size_t bytes_per_rank = input_hdl->get_buffer_size() / world_size;

  auto stream = at::cuda::getCurrentCUDAStream(input.device().index());
  nvshmemx_alltoallmem_on_stream(team, output_ptr, input_ptr, bytes_per_rank, stream);
  return out;
}

// This is an exclusive prefix sum function that calculates read (or write) offsets for each peer.
__device__ void prefixSum(int64_t *odata, int64_t *idata, int n) {
  // Specialize BlockScan for a 1D block of threads, of type int64_t.
  // - `BLOCK_SCAN_WARP_SCANS` is a low-latency scan algorithm (instead of high
  // throughput which we don't need here).
  // - `at_cuda_detail::cub` is torch's cub wrapper, see #55292.
  using BlockScanT = at_cuda_detail::hipcub::BlockScan<int64_t, THREADS_PER_BLOCK, at_cuda_detail::hipcub::BLOCK_SCAN_WARP_SCANS>;
  // Allocate shared memory for BlockScan
  __shared__ typename BlockScanT::TempStorage temp_storage;

  // TODO: currently it is assumed that the number of PE's is smaller than
  // `THREADS_PER_BLOCK`
  CUDA_KERNEL_ASSERT(n <= THREADS_PER_BLOCK);

  // Obtain input item for each thread
  int tid = threadIdx.x;
  int64_t thread_data = (tid < n) ? idata[tid] : 0;

  // Collectively compute the block-wide exclusive prefix sum
  BlockScanT(temp_storage).ExclusiveSum(thread_data, thread_data);

  // Store the result
  if (tid < n) {
    odata[tid] = thread_data;
  }
}

// This kernel is used to exchange output splits and source offsets between peers.
// `in_out_splits` is of size (3, npes) and contains:
// - input splits (IN)
// - output splits (OUT) and
// - source offsets (OUT).
__global__ void exchangeSplitAndOffset(int64_t* in_out_splits, int mype, int npes) {
  auto input_splits = in_out_splits;
  auto output_splits = in_out_splits + npes;
  auto source_offsets = in_out_splits + npes * 2;
  int tid = threadIdx.x;

  __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];

  // Scan input splits to get the source offsets
  prefixSum(peer_offsets, input_splits, npes);
  __syncthreads();;

  // Use 1 block to do the exchange
  if (tid < npes) {
    int peer = tid;
    nvshmem_int64_p(source_offsets + mype, peer_offsets[peer], peer);
    nvshmem_int64_p(output_splits + mype, input_splits[peer], peer);
  }
  // This barrier ensures that all remote PEs see the updated values
  nvshmemx_barrier_all_block();
}

// This kernel is used to do the actual data exchange.
// `in_out_splits` has the same definition as in `exchangeSplitAndOffset`.
__global__ void allToAllV(void *send_data, void *recv_data, int64_t* in_out_splits, size_t stride, int mype, int npes) {
  auto output_splits = in_out_splits + npes;
  auto source_offsets = in_out_splits + npes * 2;
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  // Calculate the output offsets
  __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];
  prefixSum(peer_offsets, output_splits, npes);
  __syncthreads();

  // Each block targets a different peer
  size_t row_size = stride * sizeof(float);  // Assuming float (TODO)
  for (int i = bid; i < npes; i += gridDim.x) {
    int peer = (mype + i) % npes;
    auto size = output_splits[peer] * row_size;
    auto source_offset = source_offsets[peer] * row_size;
    auto write_offset = peer_offsets[peer] * row_size;
    nvshmemx_getmem_block(
      (char*)recv_data + write_offset,
      (char*)send_data + source_offset,
      size,
      peer);
  }
  // Write out the output offsets (to the scratchpad line)
  if (bid == 0 && tid < npes) {
    source_offsets[tid] = peer_offsets[tid];
  }
}

at::Tensor nvshmem_all_to_all_vdev(
    at::Tensor& input,
    at::Tensor& out,
    at::Tensor& in_out_splits,
    std::string group_name) {
  /* Perform AllToAllv operation using NVSHMEM, with split information provided on device.
   * Arguments:
   *  - `input` is the input tensor
   *  - `out` is the output tensor
   *  - `in_out_splits` is a 2D tensor of size (3, npes). The rows are (in order):
        input splits (IN)
        output splits (OUT) and
        output offsets (OUT).
  */
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  auto splits_hdl = c10d::symmetric_memory::rendezvous(in_out_splits, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();

  void* input_ptr = input_hdl->get_buffer_ptrs()[rank];
  void* output_ptr = out_hdl->get_buffer_ptrs()[rank];
  int64_t* splits_ptr = (int64_t*)(splits_hdl->get_buffer_ptrs()[rank]);

  auto stream = at::cuda::getCurrentCUDAStream(input.device().index());

  // Exchange output splits and source offsets
  // Use collective launch because kernel involves nvshmem barrier
  void* args0[] = {
      &splits_ptr,
      &rank,
      &world_size};
  nvshmemx_collective_launch(
      (const void*)exchangeSplitAndOffset,
      dim3(1),
      dim3(THREADS_PER_BLOCK),
      args0,
      0,
      stream);

  // All to all data exchange
  // Limit the number of blocks to 16
  int num_blocks = std::min(world_size, 16);
  // Stride at dim 0 (assuming input is contiguous, TODO)
  size_t stride = input.stride(0);
  void* args1[] = {
      &input_ptr,
      &output_ptr,
      &splits_ptr,
      &stride,
      &rank,
      &world_size};
  nvshmemx_collective_launch(
      (const void*)allToAllV,
      dim3(num_blocks),
      dim3(THREADS_PER_BLOCK),
      args1,
      0,
      stream);
  return out;
}

} // namespace c10d::nvshmem_extension


TORCH_LIBRARY_IMPL(symm_mem, CUDA, m) {
  m.impl("nvshmem_broadcast", c10d::nvshmem_extension::nvshmem_broadcast);
  m.impl("nvshmem_all_to_all", c10d::nvshmem_extension::nvshmem_all_to_all);
  m.impl("nvshmem_all_to_all_vdev", c10d::nvshmem_extension::nvshmem_all_to_all_vdev);
}
