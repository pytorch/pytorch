#include <torch/csrc/distributed/c10d/SymmetricMemory.hpp>
#include <torch/csrc/distributed/c10d/CUDASymmetricMemoryUtils.hpp>
#include <torch/csrc/distributed/c10d/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/cuda/utils.hpp>
#include <torch/csrc/distributed/c10d/nvshmem_extension.cuh>

#include <ATen/ceil_div.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/error.h>

namespace c10d {
namespace symmetric_memory {

/* Start of CUDASymmetricMemory implementation */

static StoreExchange storeExchange = StoreExchange("NVSHMEMSymmetricMemory");

struct NVSHMEMAllocation {
  void* ptr;
  size_t buffer_size;
  int device_idx;

  NVSHMEMAllocation(void* ptr, size_t buffer_size, int device_idx)
      : ptr(ptr), buffer_size(buffer_size), device_idx(device_idx) {}
};

class NVSHMEMSymmetricMemory : public SymmetricMemory {
 public:
  NVSHMEMSymmetricMemory(
      std::shared_ptr<NVSHMEMAllocation> allocation,
      const std::string& group_name)
      : allocation_(allocation),
        buffer_size_(allocation->buffer_size),
        device_idx_(allocation->device_idx),
        group_name_(group_name) {
    c10::cuda::CUDAGuard guard(device_idx_);

    auto global_rank = get_group_info("0").rank;
    auto group_info = get_group_info(group_name_);
    auto store = group_info.store;
    rank_ = group_info.rank;
    world_size_ = group_info.world_size;
    rank_to_global_rank_ =
        storeExchange.all_gather(store, rank_, world_size_, global_rank);
    LOG(INFO) << "[rank " << rank_ << "]"
              << "rank_to_global_rank: " << rank_to_global_rank_;

    for (int r = 0; r < world_size_; ++r) {
      buffers_.push_back(nvshmem_extension::nvshmem_ptr(
          allocation->ptr, rank_to_global_rank_[r]));
    }

    // TODO: use the same allocation for signal pad
    void* signal_pad_ptr = nvshmem_extension::nvshmem_malloc(signal_pad_size);
    AT_CUDA_CHECK(hipMemset(signal_pad_ptr, 0, signal_pad_size));

    for (int r = 0; r < world_size_; ++r) {
      signal_pads_.push_back(nvshmem_extension::nvshmem_ptr(
          signal_pad_ptr, rank_to_global_rank_[r]));
    }

    const size_t arr_size = sizeof(void*) * world_size_;
    buffers_dev_ = reinterpret_cast<void**>(
        c10::cuda::CUDACachingAllocator::raw_alloc(arr_size));
    signal_pads_dev_ = reinterpret_cast<void**>(
        c10::cuda::CUDACachingAllocator::raw_alloc(arr_size));

    AT_CUDA_CHECK(hipMemcpy(
        buffers_dev_, buffers_.data(), arr_size, hipMemcpyHostToDevice));
    AT_CUDA_CHECK(hipMemcpy(
        signal_pads_dev_,
        signal_pads_.data(),
        arr_size,
        hipMemcpyHostToDevice));

    rank_to_global_rank_dev_ = reinterpret_cast<int*>(
        c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(int) * world_size_));
    AT_CUDA_CHECK(hipMemcpy(
        rank_to_global_rank_dev_,
        rank_to_global_rank_.data(),
        sizeof(int) * world_size_,
        hipMemcpyHostToDevice));
  }

  ~NVSHMEMSymmetricMemory() override{
      // TODO
  };

  std::vector<void*> get_buffer_ptrs() override {
    return buffers_;
  }

  std::vector<void*> get_signal_pad_ptrs() override {
    return signal_pads_;
  }

  void** get_buffer_ptrs_dev() override {
    return buffers_dev_;
  }

  void** get_signal_pad_ptrs_dev() override {
    return signal_pads_dev_;
  }

  size_t get_buffer_size() override {
    return buffer_size_;
  }

  size_t get_signal_pad_size() override {
    return signal_pad_size;
  };

  bool has_multicast_support() override {
    // TODO
    return false;
  }

  void* get_multicast_ptr() override {
    // TODO
    return nullptr;
  }

  at::Tensor get_buffer(
      int rank,
      c10::IntArrayRef sizes,
      c10::ScalarType dtype,
      int64_t storage_offset) {
    // TODO: deduplicate
    const size_t numel = std::accumulate(
        sizes.begin(),
        sizes.end(),
        static_cast<size_t>(1),
        std::multiplies<size_t>());
    const auto element_size = c10::elementSize(dtype);
    const auto req_size = (numel + storage_offset) * element_size;
    TORCH_CHECK(
        req_size <= buffer_size_,
        "NVSHMEMSymmetricMemory::get_buffer: the requested size (",
        req_size,
        " bytes) exceeds the allocated size (",
        buffer_size_,
        " bytes)");
    auto data_ptr = reinterpret_cast<uint8_t*>(buffers_[rank]) +
        storage_offset * element_size;
    auto device = c10::Device(c10::DeviceType::CUDA, device_idx_);
    auto options = at::TensorOptions().dtype(dtype).device(device);
    return at::for_blob(data_ptr, sizes)
        .options(options)
        .target_device(device)
        .make_tensor();
  }

  at::Tensor get_signal_pad(
      int rank,
      c10::IntArrayRef sizes,
      std::optional<c10::ScalarType> dtype,
      int64_t storage_offset) override {
    // TODO: deduplicate
    // If the dtype is unspecified, default it to UInt32, as it
    // is the most common type for signaling purposes.
    if (!dtype.has_value()) {
      dtype = c10::ScalarType::UInt32;
    }

    // If the shape is unspecified, treat the signal pad as a 1d tensor.
    const auto element_size = c10::elementSize(*dtype);
    std::vector<int64_t> shape;
    if (sizes.size() != 0) {
      shape = sizes.vec();
    } else {
      shape.push_back(signal_pad_size / element_size);
    }

    const size_t numel = std::accumulate(
        shape.begin(),
        shape.end(),
        static_cast<size_t>(1),
        std::multiplies<size_t>());
    const auto req_size = (numel + storage_offset) * element_size;
    TORCH_CHECK(
        req_size <= signal_pad_size,
        "NVSHMEMSymmetricMemory::get_signal_pad: the requested size (",
        req_size,
        " bytes) exceeds the allocated size (",
        signal_pad_size,
        " bytes)");
    auto data_ptr = reinterpret_cast<uint8_t*>(signal_pads_[rank]) +
        storage_offset * element_size;
    auto device = c10::Device(c10::DeviceType::CUDA, device_idx_);
    auto options = at::TensorOptions().dtype(*dtype).device(device);
    return at::for_blob(data_ptr, shape)
        .options(options)
        .target_device(device)
        .make_tensor();
  }

  void barrier(int channel, size_t timeout_ms) override {
    // TODO
  }

  void put_signal(int dst_rank, int channel, size_t timeout_ms) override {
    // TODO
  }

  void wait_signal(int src_rank, int channel, size_t timeout_ms) override {
    // TODO
  }

  int get_rank() override {
    return rank_;
  }

  int get_world_size() override {
    return world_size_;
  }

  virtual std::vector<int> get_rank_to_global_rank() override {
    return rank_to_global_rank_;
  };

  int* get_rank_to_global_rank_dev() override {
    return rank_to_global_rank_dev_;
  };

 private:
  std::shared_ptr<NVSHMEMAllocation> allocation_;
  size_t buffer_size_;
  std::vector<void*> buffers_;
  std::vector<void*> signal_pads_;
  int device_idx_;
  int rank_;
  int world_size_;
  void** buffers_dev_;
  void** signal_pads_dev_;
  std::string group_name_;

  std::vector<int> rank_to_global_rank_;
  int* rank_to_global_rank_dev_;
};

class NVSHMEMSymmetricMemoryAllocator : public SymmetricMemoryAllocator {
 public:
  void* alloc(
      size_t size,
      int device_idx,
      const std::optional<std::string>& group_name) override {
    TORCH_CHECK(
        group_name == std::nullopt,
        "NVSHMEMSymmetricMemoryAllocator::alloc "
        "must not be called with a group_name");

    auto group_info = get_group_info("0");
    auto store = group_info.store;
    int rank = group_info.rank;
    int world_size = group_info.world_size;

    nvshmem_extension::initialize_nvshmem_with_store(store, rank, world_size);
    auto ptr = nvshmem_extension::nvshmem_malloc(size);
    auto allocation =
        std::make_shared<NVSHMEMAllocation>(ptr, size, device_idx);
    // TODO: thread safety
    allocations_.emplace(ptr, allocation);
    return ptr;
  }

  void free(void* ptr) override {
    // TODO: thread safety
    ptr_to_symm_mem_.erase(ptr);
  };

  size_t get_alloc_size(void* ptr) override {
    auto it = ptr_to_symm_mem_.find(ptr);
    if (it == ptr_to_symm_mem_.end()) {
      TORCH_CHECK(
          false, ptr, " is not allocated with NVSHMEMSymmetricMemoryAllocator");
    }
    return it->second->get_buffer_size();
  };

  c10::intrusive_ptr<SymmetricMemory> rendezvous(
      void* ptr,
      const std::optional<std::string>& group_name) override {
    TORCH_CHECK(group_name.has_value());
    {
      auto it = symm_mems_.find(std::make_tuple(ptr, *group_name));
      if (it != symm_mems_.end()) {
        return it->second;
      }
    }
    auto it = allocations_.find(ptr);
    TORCH_CHECK(it != allocations_.end());
    auto symm_mem =
        c10::make_intrusive<NVSHMEMSymmetricMemory>(it->second, *group_name);

    symm_mems_[std::make_tuple(ptr, *group_name)] = symm_mem;
    return symm_mem;
  };

  bool has_multicast_support(int device_idx) override {
    // TODO
    return false;
  };

 private:
  std::unordered_map<void*, c10::intrusive_ptr<SymmetricMemory>>
      ptr_to_symm_mem_;

  std::unordered_map<void*, std::shared_ptr<NVSHMEMAllocation>> allocations_;
  std::map<std::tuple<void*, std::string>, c10::intrusive_ptr<SymmetricMemory>>
      symm_mems_;
};

struct RegisterNVSHMEMSymmetricMemoryAllocator {
  RegisterNVSHMEMSymmetricMemoryAllocator() {
    // Query backend used for CUDA tensor
    if (getSymmMemBackendCUDA() == "NVSHMEM") {
      register_allocator(
          c10::DeviceType::CUDA,
          c10::make_intrusive<NVSHMEMSymmetricMemoryAllocator>());
    }
  }
};

static RegisterNVSHMEMSymmetricMemoryAllocator register_allocator_;

} // namespace symmetric_memory
} // namespace c10d
