
#include <hip/hip_runtime.h>
#ifdef USE_C10D_NCCL
#include <nccl.h>
#include <torch/csrc/cuda/nccl.h>

#if NCCL_VERSION_CODE >= NCCL_VERSION(2, 27, 1)
#define NCCL_HAS_SYMMEM_SUPPORT
#endif

#ifdef NCCL_HAS_SYMMEM_SUPPORT
#include <torch/csrc/distributed/c10d/NCCLUtils.hpp>
#include <torch/csrc/distributed/c10d/GroupRegistry.hpp>
#include <torch/csrc/distributed/c10d/ProcessGroupNCCL.hpp>
#include <torch/csrc/distributed/c10d/cuda/utils.hpp>
#include <torch/csrc/distributed/c10d/symm_mem/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/symm_mem/CUDASymmetricMemoryUtils.hpp>
#include <torch/csrc/distributed/c10d/symm_mem/SymmetricMemory.hpp>

#include <ATen/ceil_div.h>
#include <ATen/cuda/CUDAContext.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/error.h>

namespace c10d {
namespace symmetric_memory {

/* Start of NCCLAllocation implementation */

static StoreExchange storeExchange = StoreExchange("NCCLAllocation");

struct NCCLAllocation {
  void* ptr;
  size_t buffer_size;
  int device_idx;

  NCCLAllocation(void* ptr, size_t buffer_size, int device_idx)
      : ptr(ptr), buffer_size(buffer_size), device_idx(device_idx) {}
};

class NCCLSymmetricMemory : public SymmetricMemory {
 public:
 NCCLSymmetricMemory(
      std::shared_ptr<NCCLAllocation> allocation,
      const std::string& group_name,
      ncclWindow_t handle,
      ncclWindow_t signal_handle)
      : allocation_(allocation),
        buffer_size_(allocation->buffer_size),
        device_idx_(allocation->device_idx),
        group_name_(group_name),
        handle_(handle),
        signal_handle_(signal_handle) {
    c10::cuda::CUDAGuard guard(device_idx_);

    // We need some API like nvshmem_extension::nvshmem_ptr()
    // put API to get the reference of remote memory.
    // WIP
  }

  ~NCCLSymmetricMemory() override = default;

  std::vector<void*> get_buffer_ptrs() override {
    return buffers_;
  }

  std::vector<void*> get_signal_pad_ptrs() override {
    return signal_pads_;
  }

  void** get_buffer_ptrs_dev() override {
    return buffers_dev_;
  }

  void** get_signal_pad_ptrs_dev() override {
    return signal_pads_dev_;
  }

  size_t get_buffer_size() override {
    return buffer_size_;
  }

  size_t get_signal_pad_size() override {
    return signal_pad_size;
  };

  bool has_multicast_support() override {
    // TODO
    return false;
  }

  void* get_multicast_ptr() override {
    // TODO
    return nullptr;
  }

  // TODO: This is up for change.
  at::Tensor get_buffer(
      int rank,
      c10::IntArrayRef sizes,
      c10::ScalarType dtype,
      int64_t storage_offset) {
    // TODO: deduplicate
    const size_t numel = std::accumulate(
        sizes.begin(),
        sizes.end(),
        static_cast<size_t>(1),
        std::multiplies<size_t>());
    const auto element_size = c10::elementSize(dtype);
    const auto req_size = (numel + storage_offset) * element_size;
    TORCH_CHECK(
        req_size <= buffer_size_,
        "NCCLSymmetricMemory::get_buffer: the requested size (",
        req_size,
        " bytes) exceeds the allocated size (",
        buffer_size_,
        " bytes)");
    auto data_ptr = reinterpret_cast<uint8_t*>(buffers_[rank]) +
        storage_offset * element_size;
    auto device = c10::Device(c10::DeviceType::CUDA, device_idx_);
    auto options = at::TensorOptions().dtype(dtype).device(device);
    return at::for_blob(data_ptr, sizes)
        .options(options)
        .target_device(device)
        .make_tensor();
  }

  // TODO: This is up for change.
  at::Tensor get_signal_pad(
      int rank,
      c10::IntArrayRef sizes,
      std::optional<c10::ScalarType> dtype,
      int64_t storage_offset) override {
    // TODO: deduplicate
    // If the dtype is unspecified, default it to UInt32, as it
    // is the most common type for signaling purposes.
    if (!dtype.has_value()) {
      dtype = c10::ScalarType::UInt32;
    }

    // If the shape is unspecified, treat the signal pad as a 1d tensor.
    const auto element_size = c10::elementSize(*dtype);
    std::vector<int64_t> shape;
    if (!sizes.empty()) {
      shape = sizes.vec();
    } else {
      shape.push_back(signal_pad_size / element_size);
    }

    const size_t numel = std::accumulate(
        shape.begin(),
        shape.end(),
        static_cast<size_t>(1),
        std::multiplies<size_t>());
    const auto req_size = (numel + storage_offset) * element_size;
    TORCH_CHECK(
        req_size <= signal_pad_size,
        "NCCLSymmetricMemory::get_signal_pad: the requested size (",
        req_size,
        " bytes) exceeds the allocated size (",
        signal_pad_size,
        " bytes)");
    auto data_ptr = reinterpret_cast<uint8_t*>(signal_pads_[rank]) +
        storage_offset * element_size;
    auto device = c10::Device(c10::DeviceType::CUDA, device_idx_);
    auto options = at::TensorOptions().dtype(*dtype).device(device);
    return at::for_blob(data_ptr, shape)
        .options(options)
        .target_device(device)
        .make_tensor();
  }

  void barrier(int channel, size_t timeout_ms) override {
    // TODO
  }

  void put_signal(int dst_rank, int channel, size_t timeout_ms) override {
    // TODO
  }

  void wait_signal(int src_rank, int channel, size_t timeout_ms) override {
    // TODO
  }

  int get_rank() override {
    return rank_;
  }

  int get_world_size() override {
    return world_size_;
  }

  virtual std::vector<int>& get_rank_to_global_rank() override {
    return rank_to_global_rank_;
  };

  int* get_rank_to_global_rank_dev() override {
    return rank_to_global_rank_dev_;
  };

 private:
  std::shared_ptr<NCCLAllocation> allocation_;
  size_t buffer_size_;
  // TODO: We need to finalize what booking variables we need for nccl backend.
  std::vector<void*> buffers_;
  std::vector<void*> signal_pads_;
  int device_idx_;
  int rank_;
  int world_size_;
  void** buffers_dev_;
  void** signal_pads_dev_;
  std::string group_name_;
  ncclWindow_t handle_;
  ncclWindow_t signal_handle_;

  std::vector<int> rank_to_global_rank_;
  int* rank_to_global_rank_dev_;
};

class NCCLSymmetricMemoryAllocator : public SymmetricMemoryAllocator {
 public:
  void* alloc(
      size_t size,
      int device_idx,
      const std::optional<std::string>& group_name) override {
    TORCH_CHECK(
        group_name == std::nullopt,
        "NCCLSymmetricMemoryAllocator::alloc "
        "must not be called with a group_name");

    auto group_info = get_group_info("0");
    auto store = group_info.store;
    c10::cuda::CUDAGuard guard(device_idx);
    // TODO: we might need to use a roundup or mempool for mem allocation.
    void* ptr;
    C10D_NCCL_CHECK(ncclMemAlloc(&ptr, size), "ncclMemAlloc");
    auto allocation =
        std::make_shared<NCCLAllocation>(ptr, size, device_idx);
    // TODO: thread safety
    allocations_.emplace(ptr, allocation);
    return ptr;
  }

  void free(void* ptr) override {
    // TODO: thread safety
    ptr_to_symm_mem_.erase(ptr);
    allocations_.erase(ptr);
  };

  size_t get_alloc_size(void* ptr) override {
    auto it = ptr_to_symm_mem_.find(ptr);
    if (it == ptr_to_symm_mem_.end()) {
      TORCH_CHECK(
          false, ptr, " is not allocated with NCCLSymmetricMemoryAllocator");
    }
    return it->second->get_buffer_size();
  };

  c10::intrusive_ptr<SymmetricMemory> rendezvous(
      void* ptr,
      const std::optional<std::string>& group_name) override {
    TORCH_CHECK(group_name.has_value(), "group_name must be provided");
    {
      auto it = symm_mems_.find(std::make_tuple(ptr, *group_name));
      if (it != symm_mems_.end()) {
        return it->second;
      }
    }
    auto it = allocations_.find(ptr);
    TORCH_CHECK(it != allocations_.end(), "memory needs to be first allocated before calling rendezvous.");


    auto group = resolve_process_group(group_name.value());
    auto alloc = it->second;
    c10::cuda::CUDAGuard guard(alloc->device_idx);
    ncclWindow_t handle;
    ncclWindow_t signal_handle;

    auto group_info = get_group_info(group_name.value());
    auto buffer_size_map =
        storeExchange.all_gather(group_info.store, group_info.rank, group_info.world_size, it->second->buffer_size);

    LOG(INFO) << "[rank " << group_info.rank << "]"
              << "buffer_size_map: " << buffer_size_map;
    // NCCL window registration api requires all ranks to have the same buffer size
    // we have this check to make sure all ranks have the same buffer size.
    for (auto r = 0; r < group_info.world_size; ++r) {
      TORCH_CHECK(alloc->buffer_size == buffer_size_map[r], "buffer size mismatch");
    }
    auto* ncclPg = dynamic_cast<c10d::ProcessGroupNCCL*>(
        group->getBackend(c10::DeviceType::CUDA).get());
    TORCH_CHECK(ncclPg != nullptr, "backend must be a NCCL process group");
    ncclComm_t comm = reinterpret_cast<ncclComm_t>(ncclPg->getCommPtr());
    C10D_NCCL_CHECK(
      ncclCommWindowRegister(comm, ptr, alloc->buffer_size, (ncclWindow_t*)&handle, NCCL_WIN_COLL_SYMMETRIC),
      c10::str(
          "Failed to window register segment with ptr ",
          ptr,
          ", size ",
          alloc->buffer_size,
          " on ncclComm_ ",
          comm));

    void* signal_pad_ptr;
    C10D_NCCL_CHECK(ncclMemAlloc(&signal_pad_ptr, signal_pad_size), "ncclMemAlloc failed");
    C10D_NCCL_CHECK(
    ncclCommWindowRegister(comm, signal_pad_ptr, signal_pad_size, (ncclWindow_t*)&signal_handle, NCCL_WIN_COLL_SYMMETRIC),
    c10::str(
        "Failed to window register segment with ptr ",
        signal_pad_ptr,
        ", size ",
        signal_pad_size,
        " on ncclComm_ ",
        comm));

    auto symm_mem =
        c10::make_intrusive<NCCLSymmetricMemory>(alloc, *group_name, std::move(handle), std::move(signal_handle));

    symm_mems_[std::make_tuple(ptr, *group_name)] = symm_mem;
    return symm_mem;
  };

  bool has_multicast_support(int device_idx) override {
    // TODO
    return false;
  };

  c10::DeviceType supported_device_type() override {
    return c10::DeviceType::CUDA;
  }

  std::string name() override {
    return "NCCL";
  }

 private:
  std::unordered_map<void*, c10::intrusive_ptr<SymmetricMemory>>
      ptr_to_symm_mem_;

  std::unordered_map<void*, std::shared_ptr<NCCLAllocation>> allocations_;
  std::map<std::tuple<void*, std::string>, c10::intrusive_ptr<SymmetricMemory>>
      symm_mems_;
};

struct RegisterNCCLSymmetricMemoryAllocator {
    RegisterNCCLSymmetricMemoryAllocator() {
    auto allocator = c10::make_intrusive<NCCLSymmetricMemoryAllocator>();
    // Query backend used for CUDA tensor
    if (getSymmMemBackendCUDA() == "NCCL") {
      // Direct set (static registration)
      register_allocator(
          c10::DeviceType::CUDA,
          allocator);
    } else {
      // Register availability in case `set_backend` is called dynamically
      register_availability("NCCL", allocator);
    }
  }
};

static RegisterNCCLSymmetricMemoryAllocator register_allocator_;

} // namespace symmetric_memory
} // namespace c10d
#endif // NCCL_HAS_SYMMEM_SUPPORT
#endif // USE_C10D_NCCL
