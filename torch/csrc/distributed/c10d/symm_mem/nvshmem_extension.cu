#include "hip/hip_runtime.h"
#include <dlfcn.h>
#include <ATen/ceil_div.h>
#include <c10/cuda/CUDAGuard.h>

#include <torch/csrc/distributed/c10d/symm_mem/env.hpp>
#include <torch/csrc/distributed/c10d/symm_mem/nvshmem_extension.cuh>
#include <torch/csrc/distributed/c10d/symm_mem/nvshmem_team_manager.hpp>
#include <torch/csrc/distributed/c10d/symm_mem/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/symm_mem/CUDASymmetricMemoryUtils.hpp>
#include <torch/csrc/distributed/c10d/symm_mem/SymmetricMemory.hpp>

// Use torch's cub wrapper instead of CUDA's <hipcub/hipcub.hpp>, see #55292
#include <ATen/cuda/cub.cuh>

// NVSHMEM minimum SM arch
#define _NVSHMEM_MIN_SM_ARCH 700

// If CUDA_ARCH is less than sm_70, or on sm_110, skip NVSHMEM device APIs
#define _NVSHMEM_DEVICELIB_SUPPORTED 1
#if defined(__CUDA_ARCH__)
#  if (__CUDA_ARCH__ < _NVSHMEM_MIN_SM_ARCH) || (__CUDA_ARCH__ == 1100)
#    undef _NVSHMEM_DEVICELIB_SUPPORTED
#  endif
#endif

// Some NVSHMEM device APIs do not compile on older SM archs
#ifndef _NVSHMEM_DEVICELIB_SUPPORTED
// Only include host APIs. See nvshmem.h for details.
#  define NVSHMEM_HOSTLIB_ONLY
#endif  // Must be done before nvshmem.h is included

#include <nvshmem.h>
#include <nvshmemx.h>

namespace c10d::nvshmem_extension {

#define THREADS_PER_BLOCK 512
#define WARP_SIZE 32

extern "C" void nvshmem_init() __attribute__((weak));

// Check if NVSHMEM is available
bool is_nvshmem_available() {
  // Runtime check
  static std::mutex mutex;
  static int is_available = -2;
  std::lock_guard<std::mutex> lock(mutex);

  // Checked if the symbol is statically linked
  if(is_available == -2 && nvshmem_init) {
    is_available = 1;
  }

  if (is_available == -2) {
    void* handle{};
    // Open the shared library, RTLD_LAZY defers symbol resolution until needed
    handle = dlopen("libnvshmem_host.so.3", RTLD_LAZY);
    if (!handle) {
      std::cerr << dlerror() << "\n";
      is_available = 0;
    } else {
      is_available = 1;
      // Close the shared library
      dlclose(handle);
    }
  }
  return is_available == 1;
}

// Initializes the device state in hipModule_t so that it’s able to perform NVSHMEM
// operations.
void nvshmemx_cumodule_init(uintptr_t module) {
  auto cumodule = reinterpret_cast<hipModule_t>(module);
  NVSHMEM_CHECK(
    ::nvshmemx_cumodule_init(cumodule),
    "nvshmemx_cumodule_init failed");
}

at::Tensor nvshmem_broadcast(at::Tensor& input, const int64_t root, const std::string& group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  int rank = input_hdl->get_rank();
  void* buffer_ptr = input.mutable_data_ptr();
  auto buffer_size = input.numel() * input.element_size();
  auto& team_manager = TeamManager::get(input.device());
  auto team = team_manager.get_team(group_name, input_hdl->get_rank_to_global_rank());
  int team_size = nvshmem_team_n_pes(team);
  TORCH_CHECK(root < team_size, "root must be smaller than group size");

  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_broadcastmem_on_stream(team, buffer_ptr, buffer_ptr, buffer_size, root, stream);
  return input;
}

void nvshmem_put(at::Tensor& tensor, const int64_t peer) {
  // TODO: support non-contiguous tensors
  TORCH_CHECK(tensor.is_contiguous(),
      "put op currently supports contiguous tensors only");
  // TODO: rendezvous should remember the group name
  auto hdl = c10d::symmetric_memory::rendezvous(tensor, "0");
  auto rank = hdl->get_rank();
  void* buffer_ptr = hdl->get_buffer_ptrs()[rank];
  auto buffer_size = tensor.numel() * tensor.element_size();
  TORCH_CHECK(peer < hdl->get_world_size(), "peer must be smaller than world size");

  c10::cuda::CUDAGuard guard(tensor.device());
  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_putmem_on_stream(buffer_ptr, tensor.data_ptr(), buffer_size, peer, stream);
}

void nvshmem_get(at::Tensor& tensor, const int64_t peer) {
  // TODO: support non-contiguous tensors
  TORCH_CHECK(tensor.is_contiguous(),
      "get op currently supports contiguous tensors only");
  // TODO: rendezvous should remember the group name
  auto hdl = c10d::symmetric_memory::rendezvous(tensor, "0");
  auto rank = hdl->get_rank();
  void* buffer_ptr = hdl->get_buffer_ptrs()[rank];
  auto buffer_size = tensor.numel() * tensor.element_size();
  TORCH_CHECK(peer < hdl->get_world_size(), "peer must be smaller than world size");

  c10::cuda::CUDAGuard guard(tensor.device());
  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_getmem_on_stream(tensor.mutable_data_ptr(), buffer_ptr, buffer_size, peer, stream);
}

at::Tensor nvshmem_all_to_all(
    at::Tensor& input,
    at::Tensor& out,
    std::string group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto& team_manager = TeamManager::get(input.device());
  auto team = team_manager.get_team(group_name, input_hdl->get_rank_to_global_rank());

  void* input_ptr = input.data_ptr();
  void* output_ptr = out.mutable_data_ptr();
  TORCH_CHECK(input.is_contiguous() && out.is_contiguous());
  TORCH_CHECK_EQ(input.numel(), out.numel());
  TORCH_CHECK_EQ(input.dtype(), out.dtype());
  TORCH_CHECK_EQ(input.numel() % world_size, 0);
  auto buffer_size = input.numel() * input.element_size();
  size_t bytes_per_rank = buffer_size / world_size;

  auto stream = at::cuda::getCurrentCUDAStream(input.device().index());
  nvshmemx_alltoallmem_on_stream(team, output_ptr, input_ptr, bytes_per_rank, stream);
  return out;
}

// This is an exclusive prefix sum function that calculates read (or write) offsets for each peer.
__device__ int64_t prefixSum(int64_t *odata, int64_t *idata, int n) {
  // Specialize BlockScan for a 1D block of threads, of type int64_t.
  // - `BLOCK_SCAN_WARP_SCANS` is a low-latency scan algorithm (instead of high
  // throughput which we don't need here).
  // - `at_cuda_detail::cub` is torch's cub wrapper, see #55292.
  using BlockScanT = at_cuda_detail::hipcub::BlockScan<int64_t, THREADS_PER_BLOCK, at_cuda_detail::hipcub::BLOCK_SCAN_WARP_SCANS>;
  // Allocate shared memory for BlockScan
  __shared__ typename BlockScanT::TempStorage temp_storage;

  // TODO: currently it is assumed that the number of PE's is smaller than
  // `THREADS_PER_BLOCK`
  CUDA_KERNEL_ASSERT(n <= THREADS_PER_BLOCK);

  // Obtain input item for each thread
  int tid = threadIdx.x;
  int64_t thread_data = (tid < n) ? idata[tid] : 0;

  // Collectively compute the block-wide exclusive prefix sum
  int64_t block_aggregate;
  BlockScanT(temp_storage).ExclusiveSum(thread_data, thread_data, block_aggregate);

  // Store the result
  odata[tid] = thread_data;
  return block_aggregate;
}

// This kernel is used to exchange output splits and source offsets between peers.
// `in_out_splits` is of size (3, npes) and contains:
// - input splits (IN)
// - output splits (OUT) and
// - source offsets (OUT).
__global__ void exchangeSplitAndOffset(int64_t* in_out_splits, nvshmem_team_t team) {
#ifndef _NVSHMEM_DEVICELIB_SUPPORTED
  CUDA_KERNEL_ASSERT_MSG(false, "SM arch unsupported for NVSHMEM");
#else
  CUDA_KERNEL_ASSERT(team != NVSHMEM_TEAM_INVALID);
  int mype = nvshmem_team_my_pe(team);
  int npes = nvshmem_team_n_pes(team);
  auto input_splits = in_out_splits;
  auto output_splits = in_out_splits + npes;
  auto source_offsets = in_out_splits + npes * 2;
  int tid = threadIdx.x;

  CUDA_KERNEL_ASSERT(npes <= THREADS_PER_BLOCK);
  __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];

  // Scan input splits to get the source offsets
  prefixSum(peer_offsets, input_splits, npes);
  __syncthreads();;

  // Use 1 block to do the exchange
  if (tid < npes) {
    // tid is peer index within team, but put calls require global rank
    int peer_global = nvshmem_team_translate_pe(team, tid, NVSHMEM_TEAM_WORLD);
    nvshmem_int64_p(source_offsets + mype, peer_offsets[tid], peer_global);
    nvshmem_int64_p(output_splits + mype, input_splits[tid], peer_global);
  }
  // This barrier ensures that all remote PEs see the updated values
  nvshmemx_barrier_block(team);
#endif
}

// This kernel is used to do the actual data exchange.
// `in_out_splits` has the same definition as in `exchangeSplitAndOffset`.
// `stride` is the stride at dim 0, unit in byte.
__global__ void allToAllV(void *send_data, void *recv_data, int64_t* in_out_splits, size_t stride, nvshmem_team_t team) {
#ifndef _NVSHMEM_DEVICELIB_SUPPORTED
  CUDA_KERNEL_ASSERT_MSG(false, "SM arch unsupported for NVSHMEM");
#else
  CUDA_KERNEL_ASSERT(team != NVSHMEM_TEAM_INVALID);
  int mype = nvshmem_team_my_pe(team);
  int npes = nvshmem_team_n_pes(team);
  auto output_splits = in_out_splits + npes;
  auto source_offsets = in_out_splits + npes * 2;
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int blocks_per_peer = max(gridDim.x / npes, 1);

  // Calculate the output offsets
  CUDA_KERNEL_ASSERT(npes <= THREADS_PER_BLOCK);
  __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];
  prefixSum(peer_offsets, output_splits, npes);
  __syncthreads();

  // Target a different peer based on bid
  for (int i = bid / blocks_per_peer; i < npes; i += gridDim.x / blocks_per_peer) {
    int peer = (mype + i) % npes;
    auto peer_global = nvshmem_team_translate_pe(team, peer, NVSHMEM_TEAM_WORLD);
    // Total amount from `peer`
    auto peer_size = output_splits[peer] * stride;
    // Amount to get from `peer` in this block
    auto block_size = peer_size / blocks_per_peer;
    // Being lazy here, we should handle the residual if the division is not exact
    CUDA_KERNEL_ASSERT(block_size * blocks_per_peer == peer_size);
    // This block's offset in the data from `peer`
    auto block_offset = block_size * (bid % blocks_per_peer);
    auto source_offset = source_offsets[peer] * stride + block_offset;
    auto write_offset = peer_offsets[peer] * stride + block_offset;
    nvshmemx_getmem_nbi_block(
      (char*)recv_data + write_offset,
      (char*)send_data + source_offset,
      block_size,
      peer_global);
  }
  // Write out the output offsets (to the scratchpad line)
  if (bid == 0 && tid < npes) {
    source_offsets[tid] = peer_offsets[tid];
  }
  // Make sure getmem_nbi calls finish
  nvshmem_quiet();
#endif
}

static int get_a2a_nblocks(size_t size, int world_size, bool intra_node) {
  // Check user setting first
  int num_blocks = c10d::symmetric_memory::getenv_nblocks();
  if (num_blocks > 0) {  // set by user
    return num_blocks;
  }
  // 16B per thread, 8 loops
  constexpr size_t chunk_size = 16 * THREADS_PER_BLOCK * 8;
  num_blocks = at::ceil_div(size, chunk_size);
  // Allow kernel to target even number of blocks per peer
  num_blocks = at::round_up(num_blocks, world_size);
  const int max_blocks = intra_node ? 64 : 16;
  return std::min(num_blocks, max_blocks);
}

at::Tensor all_to_all_vdev(
    at::Tensor& input,
    at::Tensor& out,
    at::Tensor& in_out_splits,
    std::string group_name) {
  /* Perform AllToAllv operation using NVSHMEM, with split information provided on device.
   * Arguments:
   *  - `input` is the input tensor
   *  - `out` is the output tensor
   *  - `in_out_splits` is a 2D tensor of size (3, npes). The rows are (in order):
        input splits (IN)
        output splits (OUT) and
        output offsets (OUT).
  */
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  auto splits_hdl = c10d::symmetric_memory::rendezvous(in_out_splits, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();

  void* input_ptr = input.data_ptr();
  void* output_ptr = out.mutable_data_ptr();
  int64_t* splits_ptr = (int64_t*)(in_out_splits.mutable_data_ptr());

  TORCH_CHECK_EQ(input.device(), out.device());
  auto device = input.device();
  c10::cuda::CUDAGuard guard(device);
  auto& team_manager = TeamManager::get(device);
  auto team = team_manager.get_team(group_name, input_hdl->get_rank_to_global_rank());
  auto stream = at::cuda::getCurrentCUDAStream(device.index());

  // Exchange output splits and source offsets
  // Use collective launch because kernel involves nvshmem barrier
  void* args0[] = {
      &splits_ptr,
      &team};
  nvshmemx_collective_launch(
      (const void*)exchangeSplitAndOffset,
      dim3(1),
      dim3(THREADS_PER_BLOCK),
      args0,
      0,
      stream);

  // CTA Tuning
  auto input_size = input.numel() * input.element_size();
  int num_blocks = get_a2a_nblocks(
    input_size,
    input_hdl->get_world_size(),
    input_hdl->world_within_direct_access());

  // Stride at dim 0 (assuming input is contiguous, TODO)
  size_t stride_bytes = input.stride(0) * input.element_size();

  // All to all data exchange
  void* args1[] = {
      &input_ptr,
      &output_ptr,
      &splits_ptr,
      &stride_bytes,
      &team};
  nvshmemx_collective_launch(
      (const void*)allToAllV,
      dim3(num_blocks),
      dim3(THREADS_PER_BLOCK),
      args1,
      0,
      stream);
  return out;
}

// Start of `all_to_all_vdev_2d`

// `exchangeSplitAndOffset_2d` is used to exchange output splits and source
// offsets between peers.

/* Arguments:
 * `in_splits_offsets`: input splits and offsets (optinoal), of size (2, nsplits), or (1, nsplits) if no offsets are provided.
 * `out_splits_offsets`: output splits and offsets, of size (2, nsplits).
 * `mype`: the rank of the current PE.
 * `npes`: the number of PEs.
 * `ne`: the number of experts.
 * `input_dim0`: the size of dim 0 of the input tensor.
 * `rank_is_row_in` is a boolean flag indicating whether the input has ranks as row or experts as row.
*/

/* Template parameters:
 * `HAS_IN_OFFSETS` is a boolean flag indicating whether `in_splits_offsets` has offsets (2nd row) or not.
*/

template <bool HAS_IN_OFFSETS>
__global__ void exchangeSplitAndOffset_2d(int64_t* in_splits_offsets, int64_t* out_splits_offsets, nvshmem_team_t team, int ne, size_t input_dim0, bool rank_is_row_in) {
#ifndef _NVSHMEM_DEVICELIB_SUPPORTED
  CUDA_KERNEL_ASSERT_MSG(false, "SM arch unsupported for NVSHMEM");
#else
  CUDA_KERNEL_ASSERT(team != NVSHMEM_TEAM_INVALID);
  int mype = nvshmem_team_my_pe(team);
  int npes = nvshmem_team_n_pes(team);
  int nsplits = npes * ne;
  auto input_splits = in_splits_offsets;
  auto output_splits = out_splits_offsets;
  // Borrowing the space below as a temporary exchange pad.
  auto source_offsets = out_splits_offsets + nsplits;
  int tid = threadIdx.x;

  int64_t* input_offsets = nullptr;
  if (HAS_IN_OFFSETS) {
    // input offset are provided, so we can use them directly
    input_offsets = in_splits_offsets + nsplits;
  } else {
    // input offset are not provided, so we need to calculate them.
    // Scan input splits to get the source offsets
    __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];
    auto sum_of_splits = prefixSum(peer_offsets, input_splits, nsplits);
    __syncthreads();;
    CUDA_KERNEL_ASSERT(sum_of_splits <= input_dim0 && "sum of splits is larger than input dim\n");
    // Redirect the input splits to the calculated result
    input_offsets = peer_offsets;
  }

  // Use 1 block to do the exchange
  if (tid < nsplits) {
    int peer, e, dst_offset;
    if (rank_is_row_in) {
      peer = tid / ne;
      e = tid % ne;
      dst_offset = e * npes + mype;
    } else {  // expert is row in input
      peer = tid % npes;
      e = tid / npes;
      dst_offset = mype * ne + e;
    }
    // This does a transpose from rank-major order to expert-major order
    // (or vice versa).
    auto split_val = input_splits[tid];
    CUDA_KERNEL_ASSERT(split_val >= 0 && "split value is negative\n");
    auto peer_global = nvshmem_team_translate_pe(team, peer, NVSHMEM_TEAM_WORLD);
    nvshmem_int64_p(source_offsets + dst_offset, input_offsets[tid], peer_global);
    nvshmem_int64_p(output_splits + dst_offset, split_val, peer_global);
  }
  // This barrier ensures that all remote PEs see the updated values
  nvshmemx_barrier_block(team);
#endif
}

// This is an warp-scope, exclusive prefix sum. When called by a block of
// threads, each warp will perform an independent prefix sum, concurrently.
// Returns the sum of all elements in the warp.
// `NUM_WARPS` is the number of warps participating the concurrent prefix sum.
template <int NUM_WARPS>
__device__ int64_t prefixSum_warp(int64_t *odata, int64_t *idata, int n) {
  CUDA_KERNEL_ASSERT(n <= WARP_SIZE);

  // Specialize WarpScan for type int
  using WarpScan = at_cuda_detail::hipcub::WarpScan<int64_t>;
  // Allocate WarpScan shared memory for N warps
  __shared__ typename WarpScan::TempStorage temp_storage[NUM_WARPS];

  int warp_id = threadIdx.x / WARP_SIZE;
  if (warp_id >= NUM_WARPS) {
    return 0;
  }

  // Obtain input item for each thread
  int tid = threadIdx.x % WARP_SIZE;
  int64_t thread_data = (tid < n) ? idata[tid] : 0;

  // Total sum of all elements in the warp
  int64_t warp_aggregate;
  // Compute the warp-wide exclusive prefix sum
  WarpScan(temp_storage[warp_id]).ExclusiveSum(thread_data, thread_data, warp_aggregate);

  // Store the result
  odata[tid] = thread_data;
  return warp_aggregate;
}

// This is for abstracting a thread-group-scope, exclusive prefix sum.
// Since we use warp-scope prefix sum, the thread group size is limited to warp size.
#define A2AV_TILE_SIZE WARP_SIZE

// This kernel is used to do the actual data exchange.
// `in_out_splits` has the same definition as in `exchangeSplitAndOffset`.
// `stride` is the stride at dim 0, unit in byte.
// For meaning of `mype` and `npes`, see the docstring of `all_to_all_vdev_2d`.
// `major_align` is the alignment at dim 0, unit in element. If 0, no alignment is needed.

// `rank_is_row_out` is a boolean flag indicating whether the output has ranks as rows or experts as rows.
// In dispatch case, rank_is_row_out = false, major_size = ne, minor_size = npes.
// In combine case, rank_is_row_out = true, major_size = npes, minor_size = ne.

__global__ void allToAllV_2d(void *send_data, void *recv_data, int64_t* in_splits, int64_t* out_splits_offsets, size_t stride, int minor_size, int major_size, int64_t major_align, bool rank_is_row_out, nvshmem_team_t team) {
#ifndef _NVSHMEM_DEVICELIB_SUPPORTED
  CUDA_KERNEL_ASSERT_MSG(false, "SM arch unsupported for NVSHMEM");
#else
  int nsplits = minor_size * major_size;
  auto output_splits = out_splits_offsets;
  auto source_offsets = out_splits_offsets + nsplits;
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  // Split the thread block into tiles
  constexpr int NUM_TILES = THREADS_PER_BLOCK / A2AV_TILE_SIZE;
  int tileId = tid / A2AV_TILE_SIZE;
  int laneId = tid % A2AV_TILE_SIZE;
  // Each tile calculates its own prefix sum
  __shared__ int64_t tile_prefix_sums[NUM_TILES][A2AV_TILE_SIZE];
  // A tile takes care of minor_size worth of splits
  int nsplits_per_tile = min(minor_size, nsplits - tileId * minor_size);
  // TODO: currently it is assumed that the number of PE's is smaller than
  // `A2AV_TILE_SIZE` bc the warp-scope prefix sum can only handle up to
  // WARP_SIZE elements
  CUDA_KERNEL_ASSERT(minor_size <= A2AV_TILE_SIZE && "minor_size is too large\n");
  // Similarly, the number of experts per rank is also assumed to be smaller
  // than `NUM_TILES`
  CUDA_KERNEL_ASSERT(major_size <= NUM_TILES && "major_size is too large\n");

  // Total length of each tile
  __shared__ int64_t len_per_tile[NUM_TILES];
  // When `nsplits` is small, not every tile gets data to sum. They can skip
  // this local prefix sum.
  if (nsplits_per_tile > 0) {
    // Each tile calculates its own prefix sum, return value is the sum of all elements in the tile.
    int64_t my_tile_len = prefixSum_warp<NUM_TILES>(tile_prefix_sums[tileId], output_splits + tileId * minor_size, nsplits_per_tile);
    // Last thread in each tile does the up aligning.
    if (laneId == A2AV_TILE_SIZE - 1) {
      if (major_align != 0) {  // Needs alignment
        auto aligned_len = (my_tile_len + major_align - 1) / major_align * major_align;
        // In case `aligned_len` is 0, we set it to `major_align` to avoid an
        // empty bin, bc cutlass currently does not support it. See
        // https://github.com/pytorch/pytorch/issues/152668.
        len_per_tile[tileId] = max(aligned_len, major_align);
      } else {  // 0 means alignment not needed
        len_per_tile[tileId] = my_tile_len;
      }
    }
  }
  __syncthreads();

  // Starting offset of each tile
  __shared__ int64_t start_offset_per_tile[NUM_TILES];
  // Prefix sum again to get the tiles' start offsets.
  // `NUM_TILES` is typically not greater than 32, because 32 tiles * 32 threads
  // = 1024 threads, and this kernel is launched within 1024 threads. Thus, we
  // can use warp-scope prefix sum.
  static_assert(NUM_TILES <= WARP_SIZE);
  // Only 1 warp is needed
  prefixSum_warp<1>(start_offset_per_tile, len_per_tile, NUM_TILES);
  __syncthreads();

  // Add tile offset to every element in the tile
  tile_prefix_sums[tileId][laneId] += start_offset_per_tile[tileId];
  __syncthreads();

  // Target a different e based on bid
  for (int eid = bid; eid < nsplits; eid += gridDim.x) {
    int row = eid / minor_size;
    int col = eid % minor_size;
    // Amount from `peer` for `e`
    auto peer_size = output_splits[eid] * stride;
    auto source_offset = source_offsets[eid] * stride;
    auto e_offset = tile_prefix_sums[row][col];
    auto write_offset = e_offset * stride;
    auto peer_global = nvshmem_team_translate_pe(team, rank_is_row_out ? row : col, NVSHMEM_TEAM_WORLD);
    nvshmemx_getmem_nbi_block(
      (char*)recv_data + write_offset,
      (char*)send_data + source_offset,
      peer_size,
      peer_global);  // peer's global index
  }
  // Write out the output offsets (to the scratchpad line)
  if (bid == 0 && tid < nsplits) {
    source_offsets[tid] = tile_prefix_sums[tid / minor_size][tid % minor_size];
  }
  // Make sure getmem_nbi calls finish
  nvshmem_quiet();
#endif
}

void all_to_all_vdev_2d(
    at::Tensor& input,
    at::Tensor& out,
    at::Tensor& in_splits,
    at::Tensor& out_splits_offsets,
    std::string group_name,
    std::optional<int64_t> major_align) {
  /* Perform a 2D AllToAllv shuffle operation using NVSHMEM, with split information provided on device.
   * Arguments:
   *  - `input` is the input tensor
   *  - `out` is the output tensor
   *  - `in_out_splits` is a 2D tensor of size (3, `world_size` * `ne`). In the
        scenario of Mixture-of-Experts models, `ne` is the number of experts per
        rank. The rows of `in_out_splits` are (in order):
        input splits (IN)
        output splits (OUT) and
        output offsets (OUT).
   *  - `group_name` is the name of the group to use for the collective operation.
   *  - `major_align` is the alignment of the "major dimension" of the output
        sequence. See below for details.

   *  A 2D AllToAllv shuffle is illustrated below:
        (world_size = 2, ne = 2, total number of experts = 4)
        Source: |       Rank 0      |       Rank 1      |
                | c0 | c1 | c2 | c3 | d0 | d1 | d2 | d3 |

        Dest  : |       Rank 0      |       Rank 1      |
                | c0 | d0 | c1 | d1 | c2 | d2 | c3 | d3 |
        where each `c_i` / `d_i` are slices of the `input` tensor, targeting
        expert `i`, with length indicated by input splits (in
        `in_out_splits[0]`).  That is, the 2D AllToAllv shuffle achieves a
        transpose from rank-major order at input to expert-major order at
        output.

   *  If `major_align` is not 1, the output offsets of c1, c2, c3 will be
      up-aligned to this value. For example, if c0 has length 5 and d0 has
      length 7 (making a total of 12), and if the `major_align` is set to 16,
      the output offset of c1 will be 16. Similar for c2 and c3. This value has
      no effect on the offset of the minor dimension, i.e.  d0, d1, d2 and d3.
      Note: since cutlass does not support empty bins, we set the aligned length
      to `major_align` if it is 0. See
      https://github.com/pytorch/pytorch/issues/152668.
  */
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  auto in_splits_hdl = c10d::symmetric_memory::rendezvous(in_splits, group_name);
  auto out_splits_offsets_hdl = c10d::symmetric_memory::rendezvous(out_splits_offsets, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  // TODO: world_size is currently limited by the number of elements in a WarpScan.
  TORCH_CHECK(world_size <= A2AV_TILE_SIZE, "world_size must be smaller than A2AV_TILE_SIZE", A2AV_TILE_SIZE);

  // If `major_align` is not provided, use 1 as the default value.
  int64_t major_align_val = major_align.value_or(1);
  TORCH_CHECK(major_align_val > 0, "major_align must be positive");

  void* input_ptr = input.data_ptr();
  void* output_ptr = out.mutable_data_ptr();
  int64_t* in_splits_ptr = (int64_t*)(in_splits.data_ptr());
  int64_t* out_splits_offsets_ptr = (int64_t*)(out_splits_offsets.mutable_data_ptr());

  // Shape checks
  TORCH_CHECK(in_splits.is_contiguous()
      && out_splits_offsets.is_contiguous()
      && input.is_contiguous()
      && out.is_contiguous(),
      "input, out, in_splits and out_splits_offsets must be contiguous");
  auto in_split_shape = in_splits.sizes();
  auto out_split_shape = out_splits_offsets.sizes();
  TORCH_CHECK(out_split_shape.size() == 2
      && out_split_shape[0] == 2
      && out_split_shape[1] == in_split_shape[0]
      && in_split_shape[0] % world_size == 0,
      "out_splits_offsets must be 2D with 2 rows, "
      "each row must be a multiple of world_size");

  // Consistency checks
  TORCH_CHECK(input.dtype() == out.dtype()
      && input.stride(0) == out.stride(0),
      "input and out must have the same dtype and same stride at dim 0");
  TORCH_CHECK(in_splits.scalar_type() == at::kLong
      && out_splits_offsets.scalar_type() == at::kLong,
      "splits and offsets must be int64");

  // Number of experts per rank
  int ne = in_split_shape[0] / world_size;
  constexpr int NUM_TILES = THREADS_PER_BLOCK / A2AV_TILE_SIZE;
  TORCH_CHECK(ne <= NUM_TILES, "Number of experts must be smaller than NUM_TILES", NUM_TILES);

  // Set device context for getting the stream and launching kernels below
  auto device = input.device();
  TORCH_CHECK(device.type() == at::DeviceType::CUDA &&
      out.device() == device &&
      in_splits.device() == device &&
      out_splits_offsets.device() == device,
      "all tensor arguments must be on the same CUDA device");
  c10::cuda::CUDAGuard guard(device);
  auto stream = at::cuda::getCurrentCUDAStream();
  auto& team_manager = TeamManager::get(device);
  auto team = team_manager.get_team(group_name, input_hdl->get_rank_to_global_rank());

  // Exchange output splits and source offsets
  auto input_dim0 = input.size(0);
  bool rank_is_row_in = true;
  // Use collective launch because kernel involves nvshmem barrier
  void* args0[] = {
      &in_splits_ptr,
      &out_splits_offsets_ptr,
      &team,
      &ne,
      &input_dim0,
      &rank_is_row_in};
  nvshmemx_collective_launch(
      (const void*)exchangeSplitAndOffset_2d<false>,  // false: input offsets not provided
      dim3(1),
      dim3(THREADS_PER_BLOCK),
      args0,
      0,
      stream);

  // CTA Tuning
  // Naive for now, use 1 block per expert.
  // Total number of blocks is limited to 64 (intra-node) or 8 (inter-node).
  int num_blocks = std::min(world_size * ne, world_size > 8 ? 8 : 64);

  // Stride at dim 0
  size_t stride_bytes = input.stride(0) * input.element_size();
  bool rank_is_row_out = !rank_is_row_in;

  // All to all data exchange
  void* args1[] = {
      &input_ptr,
      &output_ptr,
      &in_splits_ptr,
      &out_splits_offsets_ptr,
      &stride_bytes,
      &world_size,
      &ne,
      &major_align_val,
      &rank_is_row_out,
      &team};
  nvshmemx_collective_launch(
      (const void*)allToAllV_2d,
      dim3(num_blocks),
      dim3(THREADS_PER_BLOCK),
      args1,
      0,
      stream);
}

void all_to_all_vdev_2d_offset(
    at::Tensor& input,
    at::Tensor& out,
    at::Tensor& in_splits_offsets,
    at::Tensor& out_splits_offsets,
    std::string group_name) {
  /* Perform a 2D AllToAllv shuffle operation, with input split and offset
   * information provided on device. The input offsets are not required to be
   * exact prefix sum of the input splits, i.e. paddings are allowed between the
   * split chunks. The paddings, however, will not be transferred to peer
   * ranks.

   * In Mixture of Experts models, this operation can be used to combine tokens
   * processed by experts on parallel ranks. This operation can be viewed as an
   * "reverse" operation to the `all_to_all_vdev_2d` operation (which shuffles
   * tokens to experts).

   * Arguments:
   *  - `input` is the input tensor
   *  - `out` is the output tensor
   *  - `in_splits_offsets` is a 2D tensor of size (2, `ne` * `world_size`). In the
        scenario of Mixture-of-Experts models, `ne` is the number of experts per
        rank. The rows of `in_splits_offsets` are (in order):
        input splits (IN) and
        input offsets (IN)
   *  - `out_splits_offsets` is a 2D tensor of size (2, `world_size` * `ne`). The
        rows are (in order):
        output splits (OUT) and
        output offsets (OUT).
   *  - `group_name` is the name of the group to use for the collective operation.
  */
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  auto out_splits_offsets_hdl = c10d::symmetric_memory::rendezvous(out_splits_offsets, group_name);
  auto in_splits_offsets_hdl = c10d::symmetric_memory::rendezvous(in_splits_offsets, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  constexpr int NUM_TILES = THREADS_PER_BLOCK / A2AV_TILE_SIZE;
  TORCH_CHECK(world_size <= NUM_TILES, "world_size must be smaller than NUM_TILES", NUM_TILES);

  int64_t major_align_val = 0;

  void* input_ptr = input.data_ptr();
  void* output_ptr = out.mutable_data_ptr();
  int64_t* out_splits_offsets_ptr = (int64_t*)(out_splits_offsets.mutable_data_ptr());
  int64_t* in_splits_offsets_ptr = (int64_t*)(in_splits_offsets.data_ptr());

  // Shape checks
  TORCH_CHECK(out_splits_offsets.is_contiguous()
      && in_splits_offsets.is_contiguous()
      && input.is_contiguous()
      && out.is_contiguous(),
      "input, out, in_splits_offsets and out_splits_offsets must be contiguous");
  auto out_split_shape = out_splits_offsets.sizes();
  auto in_split_shape = in_splits_offsets.sizes();
  TORCH_CHECK(in_split_shape.size() == 2
      && in_split_shape[0] == 2
      && in_split_shape[1] % world_size == 0,
      "in_splits_offsets must be 2D with 2 rows, "
      "each row must be a multiple of world_size");

  // Consistency checks
  TORCH_CHECK(input.dtype() == out.dtype()
      && input.stride(0) == out.stride(0),
      "input and out must have the same dtype and same stride at dim 0");
  TORCH_CHECK(out_splits_offsets.scalar_type() == at::kLong
      && in_splits_offsets.scalar_type() == at::kLong,
      "splits and offsets must be int64");

  // Number of experts per rank
  int ne = in_split_shape[1] / world_size;
  // TODO: number of experts is currently limited by the number of elements in a WarpScan.
  TORCH_CHECK(ne <= A2AV_TILE_SIZE, "Number of experts must be smaller than A2AV_TILE_SIZE", A2AV_TILE_SIZE);

  // Set device context for getting the stream and launching kernels below
  auto device = input.device();
  TORCH_CHECK(device.type() == at::DeviceType::CUDA &&
      out.device() == device &&
      in_splits_offsets.device() == device &&
      out_splits_offsets.device() == device,
      "all tensor arguments must be on the same CUDA device");
  c10::cuda::CUDAGuard guard(device);
  auto stream = at::cuda::getCurrentCUDAStream();
  auto& team_manager = TeamManager::get(device);
  auto team = team_manager.get_team(group_name, input_hdl->get_rank_to_global_rank());

  // Exchange output splits and source offsets
  auto input_dim0 = input.size(0);
  bool rank_is_row_in = false;
  // Use collective launch because kernel involves nvshmem barrier
  void* args0[] = {
      &in_splits_offsets_ptr,
      &out_splits_offsets_ptr,
      &team,
      &ne,
      &input_dim0,
      &rank_is_row_in};
  nvshmemx_collective_launch(
      (const void*)exchangeSplitAndOffset_2d<true>,  // true: input offsets provided
      dim3(1),
      dim3(THREADS_PER_BLOCK),
      args0,
      0,
      stream);

  // CTA Tuning
  // Naive for now, use 1 block per expert.
  // Total number of blocks is limited to 64 (intra-node) or 8 (inter-node).
  int num_blocks = std::min(world_size * ne, world_size > 8 ? 8 : 64);

  // Stride at dim 0
  size_t stride_bytes = input.stride(0) * input.element_size();
  bool rank_is_row_out = !rank_is_row_in;

  // All to all data exchange
  void* args1[] = {
      &input_ptr,
      &output_ptr,
      &in_splits_offsets_ptr,
      &out_splits_offsets_ptr,
      &stride_bytes,
      &ne,
      &world_size,
      &major_align_val,
      &rank_is_row_out,
      &team};
  nvshmemx_collective_launch(
      (const void*)allToAllV_2d,
      dim3(num_blocks),
      dim3(THREADS_PER_BLOCK),
      args1,
      0,
      stream);
}
} // namespace c10d::nvshmem_extension


TORCH_LIBRARY_IMPL(symm_mem, CUDA, m) {
  m.impl("nvshmem_broadcast", c10d::nvshmem_extension::nvshmem_broadcast);
  m.impl("nvshmem_put", c10d::nvshmem_extension::nvshmem_put);
  m.impl("nvshmem_get", c10d::nvshmem_extension::nvshmem_get);
  m.impl("nvshmem_all_to_all", c10d::nvshmem_extension::nvshmem_all_to_all);
  m.impl("all_to_all_vdev", c10d::nvshmem_extension::all_to_all_vdev);
  m.impl("all_to_all_vdev_2d", c10d::nvshmem_extension::all_to_all_vdev_2d);
  m.impl("all_to_all_vdev_2d_offset", c10d::nvshmem_extension::all_to_all_vdev_2d_offset);
}
