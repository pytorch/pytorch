#include "hip/hip_runtime.h"
#include <dlfcn.h>
#include <c10/cuda/CUDAGuard.h>

#include <torch/csrc/distributed/c10d/symm_mem/nvshmem_extension.cuh>
#include <torch/csrc/distributed/c10d/symm_mem/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/symm_mem/CUDASymmetricMemoryUtils.hpp>
#include <torch/csrc/distributed/c10d/symm_mem/SymmetricMemory.hpp>

// Use torch's cub wrapper instead of CUDA's <hipcub/hipcub.hpp>, see #55292
#include <ATen/cuda/cub.cuh>

// NVSHMEM minimum SM arch
#define _NVSHMEM_MIN_SM_ARCH 700

// Some NVSHMEM device APIs do not compile on older SM archs
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < _NVSHMEM_MIN_SM_ARCH)
// Only include host APIs. See nvshmem.h for details.
#define NVSHMEM_HOSTLIB_ONLY
#endif  // Must be done before nvshmem.h is included

#include <nvshmem.h>
#include <nvshmemx.h>

namespace c10d::nvshmem_extension {

#define THREADS_PER_BLOCK 512
#define WARP_SIZE 32

constexpr int MiB = 1024 * 1024;

// Check if NVSHMEM is available
bool is_nvshmem_available() {
  // Runtime check
  static std::mutex mutex;
  static int is_available = -2;
  std::lock_guard<std::mutex> lock(mutex);
  if (is_available == -2) {
    void* handle{};
    // Open the shared library, RTLD_LAZY defers symbol resolution until needed
    handle = dlopen("libnvshmem_host.so.3", RTLD_LAZY);
    if (!handle) {
      std::cerr << dlerror() << "\n";
      is_available = 0;
    } else {
      is_available = 1;
      // Close the shared library
      dlclose(handle);
    }
  }
  return is_available == 1;
}

// Initializes the device state in hipModule_t so that it’s able to perform NVSHMEM
// operations.
void nvshmemx_cumodule_init(uintptr_t module) {
  auto cumodule = reinterpret_cast<hipModule_t>(module);
  NVSHMEM_CHECK(
    ::nvshmemx_cumodule_init(cumodule),
    "nvshmemx_cumodule_init failed");
}

static std::unordered_map<std::string, nvshmem_team_t> group_name_to_team_;

nvshmem_team_t group_to_team(
    const std::string& group_name,
    const std::vector<int>& global_ranks) {
  auto it = group_name_to_team_.find(group_name);
  if (it != group_name_to_team_.end()) {
    return it->second;
  }
  TORCH_CHECK(global_ranks.size() > 1);
  int stride = global_ranks[1] - global_ranks[0];
  for (size_t r = 1; r < global_ranks.size(); ++r) {
    TORCH_CHECK(global_ranks[r] - global_ranks[r - 1] == stride);
  }

  nvshmem_team_t team;
  NVSHMEM_CHECK(
      nvshmem_team_split_strided(
          NVSHMEM_TEAM_WORLD,
          global_ranks[0],
          stride,
          global_ranks.size(),
          nullptr,
          0,
          &team),
          "nvshmem_team_split_strided failed");
  group_name_to_team_[group_name] = team;
  TORCH_CHECK(team != NVSHMEM_TEAM_INVALID);
  return team;
}

at::Tensor nvshmem_broadcast(at::Tensor& input, const std::string& group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto team = group_to_team(group_name, input_hdl->get_rank_to_global_rank());
  void* buffer_ptr = input_hdl->get_buffer_ptrs()[rank];

  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_broadcastmem_on_stream(team, buffer_ptr, buffer_ptr, input_hdl->get_buffer_size(), 0, stream);
  return input;
}

void nvshmem_put(at::Tensor& tensor, int64_t peer) {
  // TODO: support non-contiguous tensors
  TORCH_CHECK(tensor.is_contiguous(),
      "put op currently supports contiguous tensors only");
  // TODO: rendezvous should remember the group name
  auto hdl = c10d::symmetric_memory::rendezvous(tensor, "0");
  auto rank = hdl->get_rank();
  void* buffer_ptr = hdl->get_buffer_ptrs()[rank];
  auto buffer_size = tensor.numel() * tensor.element_size();

  c10::cuda::CUDAGuard guard(tensor.device());
  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_putmem_on_stream(buffer_ptr, tensor.data_ptr(), buffer_size, peer, stream);
}

void nvshmem_get(at::Tensor& tensor, int64_t peer) {
  // TODO: support non-contiguous tensors
  TORCH_CHECK(tensor.is_contiguous(),
      "get op currently supports contiguous tensors only");
  // TODO: rendezvous should remember the group name
  auto hdl = c10d::symmetric_memory::rendezvous(tensor, "0");
  auto rank = hdl->get_rank();
  void* buffer_ptr = hdl->get_buffer_ptrs()[rank];
  auto buffer_size = tensor.numel() * tensor.element_size();

  c10::cuda::CUDAGuard guard(tensor.device());
  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_getmem_on_stream(tensor.data_ptr(), buffer_ptr, buffer_size, peer, stream);
}

at::Tensor nvshmem_all_to_all(
    at::Tensor& input,
    at::Tensor& out,
    std::string group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto team = group_to_team(group_name, input_hdl->get_rank_to_global_rank());

  void* input_ptr = input_hdl->get_buffer_ptrs()[rank];
  void* output_ptr = out_hdl->get_buffer_ptrs()[rank];
  size_t bytes_per_rank = input_hdl->get_buffer_size() / world_size;

  auto stream = at::cuda::getCurrentCUDAStream(input.device().index());
  nvshmemx_alltoallmem_on_stream(team, output_ptr, input_ptr, bytes_per_rank, stream);
  return out;
}

// This is an exclusive prefix sum function that calculates read (or write) offsets for each peer.
__device__ int64_t prefixSum(int64_t *odata, int64_t *idata, int n) {
  // Specialize BlockScan for a 1D block of threads, of type int64_t.
  // - `BLOCK_SCAN_WARP_SCANS` is a low-latency scan algorithm (instead of high
  // throughput which we don't need here).
  // - `at_cuda_detail::cub` is torch's cub wrapper, see #55292.
  using BlockScanT = at_cuda_detail::hipcub::BlockScan<int64_t, THREADS_PER_BLOCK, at_cuda_detail::hipcub::BLOCK_SCAN_WARP_SCANS>;
  // Allocate shared memory for BlockScan
  __shared__ typename BlockScanT::TempStorage temp_storage;

  // TODO: currently it is assumed that the number of PE's is smaller than
  // `THREADS_PER_BLOCK`
  CUDA_KERNEL_ASSERT(n <= THREADS_PER_BLOCK);

  // Obtain input item for each thread
  int tid = threadIdx.x;
  int64_t thread_data = (tid < n) ? idata[tid] : 0;

  // Collectively compute the block-wide exclusive prefix sum
  int64_t block_aggregate;
  BlockScanT(temp_storage).ExclusiveSum(thread_data, thread_data, block_aggregate);

  // Store the result
  odata[tid] = thread_data;
  return block_aggregate;
}

// This kernel is used to exchange output splits and source offsets between peers.
// `in_out_splits` is of size (3, npes) and contains:
// - input splits (IN)
// - output splits (OUT) and
// - source offsets (OUT).
__global__ void exchangeSplitAndOffset(int64_t* in_out_splits, int mype, int npes) {
#if __CUDA_ARCH__ < _NVSHMEM_MIN_SM_ARCH
  CUDA_KERNEL_ASSERT_MSG(false, "SM arch too old for NVSHMEM");
#else
  auto input_splits = in_out_splits;
  auto output_splits = in_out_splits + npes;
  auto source_offsets = in_out_splits + npes * 2;
  int tid = threadIdx.x;

  __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];

  // Scan input splits to get the source offsets
  prefixSum(peer_offsets, input_splits, npes);
  __syncthreads();;

  // Use 1 block to do the exchange
  if (tid < npes) {
    int peer = tid;
    nvshmem_int64_p(source_offsets + mype, peer_offsets[peer], peer);
    nvshmem_int64_p(output_splits + mype, input_splits[peer], peer);
  }
  // This barrier ensures that all remote PEs see the updated values
  nvshmemx_barrier_all_block();
#endif
}

// This kernel is used to do the actual data exchange.
// `in_out_splits` has the same definition as in `exchangeSplitAndOffset`.
// `stride` is the stride at dim 0, unit in byte.
__global__ void allToAllV(void *send_data, void *recv_data, int64_t* in_out_splits, size_t stride, int mype, int npes) {
#if __CUDA_ARCH__ < _NVSHMEM_MIN_SM_ARCH
  CUDA_KERNEL_ASSERT_MSG(false, "SM arch too old for NVSHMEM");
#else
  auto output_splits = in_out_splits + npes;
  auto source_offsets = in_out_splits + npes * 2;
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int blocks_per_peer = max(gridDim.x / npes, 1);

  // Calculate the output offsets
  __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];
  prefixSum(peer_offsets, output_splits, npes);
  __syncthreads();

  // Target a different peer based on bid
  for (int i = bid / blocks_per_peer; i < npes; i += gridDim.x / blocks_per_peer) {
    int peer = (mype + i) % npes;
    // Total amount from `peer`
    auto peer_size = output_splits[peer] * stride;
    // Amount to get from `peer` in this block
    auto block_size = peer_size / blocks_per_peer;
    // Being lazy here, we should handle the residual if the division is not exact
    CUDA_KERNEL_ASSERT(block_size * blocks_per_peer == peer_size);
    // This block's offset in the data from `peer`
    auto block_offset = block_size * (bid % blocks_per_peer);
    auto source_offset = source_offsets[peer] * stride + block_offset;
    auto write_offset = peer_offsets[peer] * stride + block_offset;
    nvshmemx_getmem_block(
      (char*)recv_data + write_offset,
      (char*)send_data + source_offset,
      block_size,
      peer);
  }
  // Write out the output offsets (to the scratchpad line)
  if (bid == 0 && tid < npes) {
    source_offsets[tid] = peer_offsets[tid];
  }
#endif
}

at::Tensor all_to_all_vdev(
    at::Tensor& input,
    at::Tensor& out,
    at::Tensor& in_out_splits,
    std::string group_name) {
  /* Perform AllToAllv operation using NVSHMEM, with split information provided on device.
   * Arguments:
   *  - `input` is the input tensor
   *  - `out` is the output tensor
   *  - `in_out_splits` is a 2D tensor of size (3, npes). The rows are (in order):
        input splits (IN)
        output splits (OUT) and
        output offsets (OUT).
  */
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  auto splits_hdl = c10d::symmetric_memory::rendezvous(in_out_splits, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();

  void* input_ptr = input_hdl->get_buffer_ptrs()[rank];
  void* output_ptr = out_hdl->get_buffer_ptrs()[rank];
  int64_t* splits_ptr = (int64_t*)(splits_hdl->get_buffer_ptrs()[rank]);

  auto stream = at::cuda::getCurrentCUDAStream(input.device().index());

  // Exchange output splits and source offsets
  // Use collective launch because kernel involves nvshmem barrier
  void* args0[] = {
      &splits_ptr,
      &rank,
      &world_size};
  nvshmemx_collective_launch(
      (const void*)exchangeSplitAndOffset,
      dim3(1),
      dim3(THREADS_PER_BLOCK),
      args0,
      0,
      stream);

  // CTA Tuning
  // Intra-node: use multiple blocks per peer to increase data parallelism, up to 8.
  // Up to 1 MB -> 1 block
  // Up to 2 MB -> 2 blocks
  // Up to 4 MB -> 4 blocks
  // More -> 8 blocks
  // The tuning for `num_blocks` below multiplies these numbers by world_size
  // (e.g. 8 -> 8 * 8). If world_size is smaller, we simply shift the blocks
  // towards data parallelism. (There may be room for improvement here)
  auto input_size = input.numel() * input.element_size();
  int num_blocks = input_size < MiB ? 8 :
      (input_size < 2 * MiB ? 16 :
      (input_size < 4 * MiB ? 32 : 64));

  // Inter-node: limit the total the number of blocks:
  // = 16 for 16GPUs which is enough to max out 90 GB/s bandwidth perf
  // = 8 for more than 16 GPUs which is enough to max out approx 50 GB/s bandwidth perf
  // Above assumes 400Gb/s NIC for inter-node and 400GB/s NVLinks for intra-node comms.
  // TODO: better intra vs inter detection, currently it is based on world_size.
  int max_inter_node_blocks = world_size <= 16 ? 16 : 8;
  if (world_size > 8) {
    num_blocks = std::min(num_blocks, max_inter_node_blocks);
  }

  // Stride at dim 0 (assuming input is contiguous, TODO)
  size_t stride_bytes = input.stride(0) * input.element_size();

  // All to all data exchange
  void* args1[] = {
      &input_ptr,
      &output_ptr,
      &splits_ptr,
      &stride_bytes,
      &rank,
      &world_size};
  nvshmemx_collective_launch(
      (const void*)allToAllV,
      dim3(num_blocks),
      dim3(THREADS_PER_BLOCK),
      args1,
      0,
      stream);
  return out;
}

// Start of `all_to_all_vdev_2d`
// This kernel is used to exchange output splits and source offsets between peers.
// For meaning of `mype` and `npes`, see the docstring of `all_to_all_vdev_2d`.
// `in_out_splits` is of size (3, npes * ne) and contains:
// - input splits (IN)
// - output splits (OUT) and
// - source offsets (OUT).
__global__ void exchangeSplitAndOffset_2d(int64_t* in_out_splits, int mype, int npes, int ne, size_t input_dim0) {
#if __CUDA_ARCH__ < _NVSHMEM_MIN_SM_ARCH
  CUDA_KERNEL_ASSERT_MSG(false, "SM arch too old for NVSHMEM");
#else
  int nsplits = npes * ne;
  auto input_splits = in_out_splits;
  auto output_splits = in_out_splits + nsplits;
  auto source_offsets = in_out_splits + nsplits * 2;
  int tid = threadIdx.x;

  __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];

  // Scan input splits to get the source offsets
  auto sum_of_splits = prefixSum(peer_offsets, input_splits, nsplits);
  __syncthreads();;
  CUDA_KERNEL_ASSERT(sum_of_splits <= input_dim0);

  // Use 1 block to do the exchange
  if (tid < nsplits) {
    int peer = tid / ne;
    int e = tid % ne;
    // This does a transpose from rank-major order to expert-major order
    int dst_offset = e * npes + mype;
    auto split_val = input_splits[tid];
    CUDA_KERNEL_ASSERT(split_val >= 0);
    nvshmem_int64_p(source_offsets + dst_offset, peer_offsets[tid], peer);
    nvshmem_int64_p(output_splits + dst_offset, split_val, peer);
  }
  // This barrier ensures that all remote PEs see the updated values
  nvshmemx_barrier_all_block();
#endif
}

// This is an warp-scope, exclusive prefix sum. When called by a block of
// threads, each warp will perform an independent prefix sum, concurrently.
// Returns the sum of all elements in the warp.
// `NUM_WARPS` is the number of warps participating the concurrent prefix sum.
template <int NUM_WARPS>
__device__ int64_t prefixSum_warp(int64_t *odata, int64_t *idata, int n) {
  CUDA_KERNEL_ASSERT(n <= WARP_SIZE);

  // Specialize WarpScan for type int
  using WarpScan = at_cuda_detail::hipcub::WarpScan<int64_t>;
  // Allocate WarpScan shared memory for N warps
  __shared__ typename WarpScan::TempStorage temp_storage[NUM_WARPS];

  int warp_id = threadIdx.x / WARP_SIZE;
  if (warp_id >= NUM_WARPS) {
    return 0;
  }

  // Obtain input item for each thread
  int tid = threadIdx.x % WARP_SIZE;
  int64_t thread_data = (tid < n) ? idata[tid] : 0;

  // Total sum of all elements in the warp
  int64_t warp_aggregate;
  // Compute the warp-wide exclusive prefix sum
  WarpScan(temp_storage[warp_id]).ExclusiveSum(thread_data, thread_data, warp_aggregate);

  // Store the result
  odata[tid] = thread_data;
  return warp_aggregate;
}

// This is for abstracting a thread-group-scope, exclusive prefix sum.
// Since we use warp-scope prefix sum, the thread group size is limited to warp size.
#define A2AV_TILE_SIZE WARP_SIZE

// This kernel is used to do the actual data exchange.
// `in_out_splits` has the same definition as in `exchangeSplitAndOffset`.
// `stride` is the stride at dim 0, unit in byte.
// For meaning of `mype` and `npes`, see the docstring of `all_to_all_vdev_2d`.
__global__ void allToAllV_2d(void *send_data, void *recv_data, int64_t* in_out_splits, size_t stride, int mype, int npes, int ne, int64_t major_align) {
#if __CUDA_ARCH__ < _NVSHMEM_MIN_SM_ARCH
  CUDA_KERNEL_ASSERT_MSG(false, "SM arch too old for NVSHMEM");
#else
  int nsplits = npes * ne;
  auto output_splits = in_out_splits + nsplits;
  auto source_offsets = in_out_splits + nsplits * 2;
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  // Split the thread block into tiles
  constexpr int NUM_TILES = THREADS_PER_BLOCK / A2AV_TILE_SIZE;
  int tileId = tid / A2AV_TILE_SIZE;
  int laneId = tid % A2AV_TILE_SIZE;
  // Each tile calculates its own prefix sum
  __shared__ int64_t tile_prefix_sums[NUM_TILES][A2AV_TILE_SIZE];
  // A tile takes care of npes worth of splits
  int nsplits_per_tile = min(npes, nsplits - tileId * npes);
  // TODO: currently it is assumed that the number of PE's is smaller than
  // `A2AV_TILE_SIZE` bc the warp-scope prefix sum can only handle up to
  // WARP_SIZE elements
  CUDA_KERNEL_ASSERT(npes <= A2AV_TILE_SIZE);
  // Similarly, the number of experts per rank is also assumed to be smaller
  // than `NUM_TILES`
  CUDA_KERNEL_ASSERT(ne <= NUM_TILES);

  // Total length of each tile
  __shared__ int64_t len_per_tile[NUM_TILES];
  // When `nsplits` is small, not every tile gets data to sum. They can skip
  // this local prefix sum.
  if (nsplits_per_tile > 0) {
    // Each tile calculates its own prefix sum, return value is the sum of all elements in the tile.
    int64_t my_tile_len = prefixSum_warp<NUM_TILES>(tile_prefix_sums[tileId], output_splits + tileId * npes, nsplits_per_tile);
    // Last thread in each tile does the up aligning.
    if (laneId == A2AV_TILE_SIZE - 1) {
      auto aligned_len = (my_tile_len + major_align - 1) / major_align * major_align;
      // In case `aligned_len` is 0, we set it to `major_align` to avoid an
      // empty bin, bc cutlass currently does not support it. See
      // https://github.com/pytorch/pytorch/issues/152668.
      len_per_tile[tileId] = max(aligned_len, major_align);
    }
  }
  __syncthreads();

  // Starting offset of each tile
  __shared__ int64_t start_offset_per_tile[NUM_TILES];
  // Prefix sum again to get the tiles' start offsets.
  // `NUM_TILES` is typically not greater than 32, because 32 tiles * 32 threads
  // = 1024 threads, and this kernel is launched within 1024 threads. Thus, we
  // can use warp-scope prefix sum.
  static_assert(NUM_TILES <= WARP_SIZE);
  // Only 1 warp is needed
  prefixSum_warp<1>(start_offset_per_tile, len_per_tile, NUM_TILES);
  __syncthreads();

  // Add tile offset to every element in the tile
  tile_prefix_sums[tileId][laneId] += start_offset_per_tile[tileId];
  __syncthreads();

  // Target a different e based on bid
  for (int eid = bid; eid < nsplits; eid += gridDim.x) {
    int peer = eid % npes;
    // Amount from `peer` for `e`
    auto peer_size = output_splits[eid] * stride;
    auto source_offset = source_offsets[eid] * stride;
    auto e_offset = tile_prefix_sums[eid / npes][peer];
    auto write_offset = e_offset * stride;
    nvshmemx_getmem_block(
      (char*)recv_data + write_offset,
      (char*)send_data + source_offset,
      peer_size,
      peer);
  }
  // Write out the output offsets (to the scratchpad line)
  if (bid == 0 && tid < nsplits) {
    source_offsets[tid] = tile_prefix_sums[tid / npes][tid % npes];
  }
#endif
}

at::Tensor all_to_all_vdev_2d(
    at::Tensor& input,
    at::Tensor& out,
    at::Tensor& in_out_splits,
    std::string group_name,
    std::optional<int64_t> major_align) {
  /* Perform a 2D AllToAllv shuffle operation using NVSHMEM, with split information provided on device.
   * Arguments:
   *  - `input` is the input tensor
   *  - `out` is the output tensor
   *  - `in_out_splits` is a 2D tensor of size (3, `world_size` * `ne`). In the
        scenario of Mixture-of-Experts models, `ne` is the number of experts per
        rank. The rows of `in_out_splits` are (in order):
        input splits (IN)
        output splits (OUT) and
        output offsets (OUT).
   *  - `group_name` is the name of the group to use for the collective operation.
   *  - `major_align` is the alignment of the "major dimension" of the output
        sequence. See below for details.

   *  A 2D AllToAllv shuffle is illustrated below:
        (world_size = 2, ne = 2, total number of experts = 4)
        Source: |       Rank 0      |       Rank 1      |
                | c0 | c1 | c2 | c3 | d0 | d1 | d2 | d3 |

        Dest  : |       Rank 0      |       Rank 1      |
                | c0 | d0 | c1 | d1 | c2 | d2 | c3 | d3 |
        where each `c_i` / `d_i` are slices of the `input` tensor, targeting
        expert `i`, with length indicated by input splits (in
        `in_out_splits[0]`).  That is, the 2D AllToAllv shuffle achieves a
        transpose from rank-major order at input to expert-major order at
        output.

   *  If `major_align` is not 1, the output offsets of c1, c2, c3 will be
      up-aligned to this value. For example, if c0 has length 5 and d0 has
      length 7 (making a total of 12), and if the `major_align` is set to 16,
      the output offset of c1 will be 16. Similar for c2 and c3. This value has
      no effect on the offset of the minor dimension, i.e.  d0, d1, d2 and d3.
      Note: since cutlass does not support empty bins, we set the aligned length
      to `major_align` if it is 0. See
      https://github.com/pytorch/pytorch/issues/152668.
  */
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  auto splits_hdl = c10d::symmetric_memory::rendezvous(in_out_splits, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  // TODO: world_size is currently limited by the number of elements in a WarpScan.
  TORCH_CHECK(world_size <= A2AV_TILE_SIZE, "world_size must be smaller than A2AV_TILE_SIZE", A2AV_TILE_SIZE);

  // If `major_align` is not provided, use 1 as the default value.
  int64_t major_align_val = major_align.value_or(1);
  TORCH_CHECK(major_align_val > 0, "major_align must be positive");

  void* input_ptr = input_hdl->get_buffer_ptrs()[rank];
  void* output_ptr = out_hdl->get_buffer_ptrs()[rank];
  int64_t* splits_ptr = (int64_t*)(splits_hdl->get_buffer_ptrs()[rank]);

  // Shape checks
  auto split_shape = in_out_splits.sizes();
  TORCH_CHECK(in_out_splits.is_contiguous()
      && input.is_contiguous()
      && out.is_contiguous(),
      "input, out and in_out_splits must be contiguous");
  TORCH_CHECK(split_shape.size() == 2
      && split_shape[0] == 3
      && split_shape[1] % world_size == 0,
      "in_out_splits must be 2D with 3 rows, "
      "each row must be a multiple of world_size");

  // Consistency checks
  TORCH_CHECK(input.dtype() == out.dtype()
      && input.stride(0) == out.stride(0),
      "input and out must have the same dtype and same stride at dim 0");
  TORCH_CHECK(in_out_splits.scalar_type() == at::kLong, "in_out_splits must be int64");

  // Number of experts per rank
  int ne = split_shape[1] / world_size;
  constexpr int NUM_TILES = THREADS_PER_BLOCK / A2AV_TILE_SIZE;
  TORCH_CHECK(ne <= NUM_TILES, "Number of experts must be smaller than NUM_TILES", NUM_TILES);

  // Set device context for getting the stream and launching kernels below
  c10::cuda::CUDAGuard guard(input.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  // Exchange output splits and source offsets
  auto input_dim0 = input.size(0);
  // Use collective launch because kernel involves nvshmem barrier
  void* args0[] = {
      &splits_ptr,
      &rank,
      &world_size,
      &ne,
      &input_dim0};
  nvshmemx_collective_launch(
      (const void*)exchangeSplitAndOffset_2d,
      dim3(1),
      dim3(THREADS_PER_BLOCK),
      args0,
      0,
      stream);

  // CTA Tuning
  // Naive for now, use 1 block per expert.
  // Total number of blocks is limited to 64 (intra-node) or 8 (inter-node).
  int num_blocks = std::min(world_size * ne, world_size > 8 ? 8 : 64);

  // Stride at dim 0
  size_t stride_bytes = input.stride(0) * input.element_size();

  // All to all data exchange
  void* args1[] = {
      &input_ptr,
      &output_ptr,
      &splits_ptr,
      &stride_bytes,
      &rank,
      &world_size,
      &ne,
      &major_align_val};
  nvshmemx_collective_launch(
      (const void*)allToAllV_2d,
      dim3(num_blocks),
      dim3(THREADS_PER_BLOCK),
      args1,
      0,
      stream);
  return out;
}

} // namespace c10d::nvshmem_extension


TORCH_LIBRARY_IMPL(symm_mem, CUDA, m) {
  m.impl("nvshmem_broadcast", c10d::nvshmem_extension::nvshmem_broadcast);
  m.impl("nvshmem_put", c10d::nvshmem_extension::nvshmem_put);
  m.impl("nvshmem_get", c10d::nvshmem_extension::nvshmem_get);
  m.impl("nvshmem_all_to_all", c10d::nvshmem_extension::nvshmem_all_to_all);
  m.impl("all_to_all_vdev", c10d::nvshmem_extension::all_to_all_vdev);
  m.impl("all_to_all_vdev_2d", c10d::nvshmem_extension::all_to_all_vdev_2d);
}
