#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/nvrtc_stub/ATenNVRTC.h>
#include <c10/cuda/CUDAGuard.h>

// Two warnings in Cutlass included header files
C10_DIAGNOSTIC_PUSH_AND_IGNORED_IF_DEFINED("-Wset-but-not-used")
C10_DIAGNOSTIC_PUSH_AND_IGNORED_IF_DEFINED("-Wunused-but-set-parameter")

#if !defined(USE_ROCM) && !defined(_WIN32) && defined(CUDA_VERSION) && \
    CUDA_VERSION >= 12000
#define BUILD_ASYNC_MM_KERNEL
#endif

#if defined(BUILD_ASYNC_MM_KERNEL)

#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/half.h>
#include <cutlass/numeric_types.h>
#include <cutlass/trace.h>
#include <cutlass/util/host_tensor.h>
#include <cute/tensor.hpp>

#include <cutlass/version.h>
#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/epilogue/collective/collective_builder.hpp>

#include <cute/atom/mma_atom.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>

#include <torch/csrc/distributed/c10d/cuda/cutlass/gemm/kernel/persistent_async_input_scheduler.cuh>

C10_DIAGNOSTIC_POP()
C10_DIAGNOSTIC_POP()

namespace {

using namespace cute;

template <typename LayoutB, typename TileShape_MNK, typename ClusterShape_MNK>
at::Tensor async_input_mm_impl(
    at::Tensor a,
    at::Tensor b,
    at::Tensor a_chunk_signals,
    int64_t a_chunk_pivot,
    at::Tensor out) {
  c10::cuda::CUDAGuard guard(a.device());

  using ElementA = cutlass::bfloat16_t;
  using LayoutA = cutlass::layout::RowMajor;
  constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;

  using ElementB = cutlass::bfloat16_t;
  constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;

  using ElementC = cutlass::bfloat16_t;
  using LayoutC = cutlass::layout::RowMajor;
  constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;

  using ElementAccumulator = float;

  using KernelSchedule = cutlass::gemm::KernelTmaWarpSpecializedCooperative;
  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90,
          cutlass::arch::OpClassTensorOp,
          TileShape_MNK,
          ClusterShape_MNK,
          cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAccumulator,
          ElementAccumulator,
          ElementC,
          LayoutC,
          AlignmentC,
          ElementC,
          LayoutC,
          AlignmentC,
          EpilogueSchedule>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          cutlass::arch::Sm90,
          cutlass::arch::OpClassTensorOp,
          ElementA,
          LayoutA,
          AlignmentA,
          ElementB,
          LayoutB,
          AlignmentB,
          ElementAccumulator,
          TileShape_MNK,
          ClusterShape_MNK,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          KernelSchedule>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue,
      cutlass::gemm::PersistentAsyncInputScheduler<KernelSchedule>>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;

  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && out.dim() == 2);
  TORCH_CHECK(a.is_contiguous() && out.is_contiguous());

  if constexpr (std::is_same_v<LayoutB, cutlass::layout::RowMajor>) {
    TORCH_CHECK(b.is_contiguous());
  } else {
    TORCH_CHECK(b.stride(1) == b.size(0));
    TORCH_CHECK(b.stride(0) == 1);
  }
  TORCH_CHECK_EQ(a.scalar_type(), at::kBFloat16);
  TORCH_CHECK_EQ(b.scalar_type(), at::kBFloat16);
  TORCH_CHECK_EQ(out.scalar_type(), at::kBFloat16);

  int M = static_cast<int>(a.sizes()[0]);
  int N = static_cast<int>(b.sizes()[1]);
  int K = static_cast<int>(a.sizes()[1]);
  TORCH_CHECK_EQ(b.sizes()[0], K);
  TORCH_CHECK_EQ(out.sizes()[0], M);
  TORCH_CHECK_EQ(out.sizes()[1], N);

  auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, {M, K, 1});
  auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, {N, K, 1});
  auto stride_C = cutlass::make_cute_packed_stride(StrideC{}, {M, N, 1});

  Gemm gemm;

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K},
      {
          reinterpret_cast<ElementA*>(a.data_ptr<at::BFloat16>()),
          stride_A,
          reinterpret_cast<ElementB*>(b.data_ptr<at::BFloat16>()),
          stride_B,
      },
      {{1, 1},
       reinterpret_cast<ElementC*>(out.data_ptr<at::BFloat16>()),
       stride_C,
       reinterpret_cast<ElementC*>(out.data_ptr<at::BFloat16>()),
       stride_C},
  };

  TORCH_CHECK(
      a_chunk_signals.dim() == 1,
      "async_input_mm: `a_chunk_signals` must be a 1D tensor.");
  size_t num_chunks_M = a_chunk_signals.numel();

  TORCH_CHECK(
      M % num_chunks_M == 0,
      "async_input_mm: `a.shape(0)` must be an integer multiple of `a_chunk_signals.numel()`");
  size_t chunk_size_M = M / num_chunks_M;
  size_t tile_size_M = cute::get<0>(TileShape_MNK{});

  TORCH_CHECK(chunk_size_M % tile_size_M == 0);

  // We want to swizzle within a chunk
  arguments.scheduler.max_swizzle_size = chunk_size_M / tile_size_M;

  // PersistentAsyncInputScheduler currently only supports rastering along N
  using RasterOrderOptions = typename cutlass::gemm::kernel::detail::
      PersistentTileSchedulerSm90::RasterOrderOptions;
  arguments.scheduler.raster_order = RasterOrderOptions::AlongN;

  // Convert the number of chunks to pivot to the number of m idx to pivot
  arguments.scheduler.tile_idx_pivot_m =
      a_chunk_pivot * (chunk_size_M / tile_size_M);
  arguments.scheduler.tiles_per_chunk_m = chunk_size_M / tile_size_M;
  arguments.scheduler.chunk_signals = a_chunk_signals.data_ptr<uint32_t>();

  size_t workspace_size = Gemm::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  TORCH_CHECK(gemm.can_implement(arguments) == cutlass::Status::kSuccess);
  TORCH_CHECK(
      gemm.initialize(arguments, workspace.get()) == cutlass::Status::kSuccess);
  TORCH_CHECK(
      gemm(at::cuda::getCurrentCUDAStream()) == cutlass::Status::kSuccess);

  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return out;
}

} // namespace

#endif

namespace c10d::cuda::detail {

#define DISPATCH_LAYOUT_B(is_b_row_major, ...)    \
  if (is_b_row_major) {                           \
    using LayoutB = cutlass::layout::RowMajor;    \
    __VA_ARGS__();                                \
  } else {                                        \
    using LayoutB = cutlass::layout::ColumnMajor; \
    __VA_ARGS__();                                \
  }

at::Tensor async_input_mm_out(
    at::Tensor a,
    at::Tensor b,
    at::Tensor a_chunk_signals,
    int64_t a_chunk_pivot,
    at::Tensor out) {
  TORCH_CHECK(
      a.dim() == 2 && b.dim() == 2 && out.dim() == 2,
      "async_input_mm: `a`, `b` and `out` must be matrices")
  TORCH_CHECK(
      a.is_contiguous() && out.is_contiguous(),
      "async_input_mm: `a` and `out` must be in row-major layout");

  if (!b.is_contiguous()) {
    TORCH_CHECK(b.stride(1) == b.size(0));
    TORCH_CHECK(b.stride(0) == 1);
  }
  TORCH_CHECK_EQ(a.scalar_type(), at::kBFloat16);
  TORCH_CHECK_EQ(b.scalar_type(), at::kBFloat16);
  TORCH_CHECK_EQ(out.scalar_type(), at::kBFloat16);

  int64_t M = a.sizes()[0];
  int64_t N = b.sizes()[1];
  int64_t K = a.sizes()[1];
  TORCH_CHECK_EQ(b.sizes()[0], K);
  TORCH_CHECK_EQ(out.sizes()[0], M);
  TORCH_CHECK_EQ(out.sizes()[1], N);

#if defined(BUILD_ASYNC_MM_KERNEL)
  const bool is_b_row_major = b.is_contiguous();
  DISPATCH_LAYOUT_B(is_b_row_major, [&]() {
    // TODO(yifu): tuning
    async_input_mm_impl<LayoutB, Shape<_128, _256, _64>, Shape<_2, _1, _1>>(
        a, b, a_chunk_signals, a_chunk_pivot, out);
  });
#else
  TORCH_CHECK(
      false, "async_input_mm is not currently supported on your device");
#endif
  return out;
}

at::Tensor async_input_mm(
    at::Tensor a,
    at::Tensor b,
    at::Tensor a_chunk_signals,
    int64_t a_chunk_pivot) {
  TORCH_CHECK(
      a.dim() == 2 && b.dim() == 2,
      "async_input_mm: `a`, `b` and `out` must all be a matrix")

  int64_t M = a.sizes()[0];
  int64_t N = b.sizes()[1];
  auto out = a.new_empty({M, N});
  return async_input_mm_out(a, b, a_chunk_signals, a_chunk_pivot, out);
}

} // namespace c10d::cuda::detail
