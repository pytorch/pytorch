#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/ceil_div.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/library.h>

#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
#include <c10/cuda/driver_api.h>
#endif

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty_like.h>
#endif


#include <torch/csrc/distributed/c10d/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/CUDASymmetricMemory.hpp>
#include <torch/csrc/distributed/c10d/cuda/AsyncMM.cuh>

#if defined(USE_ROCM) || (defined(CUDART_VERSION) && CUDART_VERSION >= 12030)

#define INT_SWITCH_CASE(name, val, ...) \
  case val: {                           \
    constexpr int name = val;           \
    __VA_ARGS__();                      \
    break;                              \
  }

#define DISPATCH_WORLD_SIZES(world_size, ...)      \
  switch (world_size) {                            \
    INT_SWITCH_CASE(k_world_size, 8, __VA_ARGS__); \
    INT_SWITCH_CASE(k_world_size, 4, __VA_ARGS__); \
    INT_SWITCH_CASE(k_world_size, 2, __VA_ARGS__); \
    default: {                                     \
      constexpr int k_world_size = -1;             \
      __VA_ARGS__();                               \
    }                                              \
  }

#define DISPATCH_WORLD_SIZES_NO_DEFAULT(world_size, ...)                 \
  switch (world_size) {                                                  \
    INT_SWITCH_CASE(k_world_size, 8, __VA_ARGS__);                       \
    INT_SWITCH_CASE(k_world_size, 4, __VA_ARGS__);                       \
    INT_SWITCH_CASE(k_world_size, 2, __VA_ARGS__);                       \
    default: {                                                           \
      TORCH_CHECK(false, "Not implemented for world_size=", world_size); \
    }                                                                    \
  }

#define DISPATCH_ALIGNMENTS_16_8_4(alignment, ...)                    \
  switch (alignment) {                                                \
    INT_SWITCH_CASE(k_alignment, 16, __VA_ARGS__);                    \
    INT_SWITCH_CASE(k_alignment, 8, __VA_ARGS__);                     \
    INT_SWITCH_CASE(k_alignment, 4, __VA_ARGS__);                     \
    default: {                                                        \
      TORCH_CHECK(false, "Not implemented for aligment=", alignment); \
    }                                                                 \
  }

#define AT_DISPATCH_FLOAT_AND_BFLOAT16(scalar_type, name, ...)         \
  AT_DISPATCH_SWITCH(                                                  \
      scalar_type, name, AT_DISPATCH_CASE(at::kBFloat16, __VA_ARGS__); \
      AT_DISPATCH_CASE(at::kFloat, __VA_ARGS__));

namespace {

using namespace c10d::symmetric_memory;

size_t get_and_verify_alignment(const at::Tensor& input, const char* op_name) {
  const size_t min_alignment = std::max(4l, input.element_size());
  // Only check the offset since the multicast address is always at least
  // 128-bit aligned
  const size_t ptr_alignment = at::native::memory::get_alignment(
      static_cast<size_t>(input.storage_offset() * input.element_size()));
  TORCH_CHECK(
      ptr_alignment >= min_alignment,
      op_name,
      "<",
      input.scalar_type(),
      ">: input ptr + offset must be at least ",
      min_alignment,
      "-byte aligned.");

  const size_t size_alignment =
      at::native::memory::get_alignment(static_cast<size_t>(input.numel() * input.element_size()));
  TORCH_CHECK(
      size_alignment >= min_alignment,
      op_name,
      "<",
      input.scalar_type(),
      ">: input size must be at least ",
      min_alignment,
      "-byte aligned.");
  return std::min(ptr_alignment, size_alignment);
}

void init_elementwise_launch_config(
    size_t numel,
    size_t element_size,
    size_t alignment,
    size_t splits,
    size_t max_num_blocks,
    size_t max_num_threads,
    int& num_blocks,
    int& num_threads) {
  // Align to preserve alignment in each split
  const size_t aligned_numel = at::round_up(numel, alignment * splits);
  const size_t numel_per_split = aligned_numel / splits;
  const size_t numel_per_thread = alignment / element_size;

  if (numel_per_split <= max_num_threads * numel_per_thread) {
    num_blocks = 1;
    num_threads = at::round_up(
        at::ceil_div(numel_per_split, numel_per_thread),
        static_cast<size_t>(C10_WARP_SIZE));
  } else {
    num_blocks = std::min(
        at::ceil_div(numel_per_split, max_num_threads * numel_per_thread),
        max_num_blocks);
    num_threads = max_num_threads;
  }
}

#if !defined(USE_ROCM) //No multi-cast support on ROCm yet
template <typename T, int alignment>
static __global__ void multimem_all_reduce_kernel(
    T* input_mc_ptr,
    size_t numel,
    uint32_t** signal_pads,
    size_t rank,
    size_t world_size) {
  static_assert(alignment % sizeof(T) == 0);
  constexpr size_t numel_per_thread = alignment / sizeof(T);

  sync_remote_blocks<std::memory_order_relaxed>(signal_pads, rank, world_size);
  __syncthreads();

  const size_t numel_per_rank =
      at::round_up(numel, alignment * world_size) / world_size;
  const size_t start = numel_per_rank * rank;

  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * numel_per_thread;
  auto stride = blockDim.x * gridDim.x * numel_per_thread;
  for (size_t i = offset; i < numel_per_rank; i += stride) {
    if (start + i >= numel) {
      continue;
    }
    auto vec = multimem_ld_reduce_add<alignment>(input_mc_ptr + start + i);
    multimem_st<alignment>(input_mc_ptr + start + i, vec);
  }

  __syncthreads();
  sync_remote_blocks<std::memory_order_acq_rel>(signal_pads, rank, world_size);
}

at::Tensor multimem_all_reduce_(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name) {
  TORCH_CHECK(
      input.is_contiguous(), "multimem_all_reduce_: input must be contiguous.");
  TORCH_CHECK(
      reduce_op == "sum",
      "multimem_all_reduce_: only sum is supported for now.");

  auto symm_mem = c10d::symmetric_memory::rendezvous(input, group_name);
  TORCH_CHECK(
      symm_mem != nullptr,
      "multimem_all_reduce_: input must be allocated with empty_strided_p2p().");
  TORCH_CHECK(
      symm_mem->has_multicast_support(),
      "multimem_all_reduce_: multicast support is required.");

  const size_t alignment =
      get_and_verify_alignment(input, "multimem_all_reduce_");

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel(),
      input.element_size(),
      alignment,
      symm_mem->get_world_size(),
      8,
      1024,
      num_blocks,
      num_threads);

  AT_DISPATCH_FLOAT_AND_BFLOAT16(
      input.scalar_type(), "multimem_all_reduce_", [&]() {
        DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
          multimem_all_reduce_kernel<scalar_t, k_alignment>
              <<<num_blocks,
                 num_threads,
                 0,
                 at::cuda::getCurrentCUDAStream()>>>(
                  reinterpret_cast<scalar_t*>(symm_mem->get_multicast_ptr()) +
                      input.storage_offset(),
                  input.numel(),
                  reinterpret_cast<uint32_t**>(
                      symm_mem->get_signal_pad_ptrs_dev()),
                  symm_mem->get_rank(),
                  symm_mem->get_world_size());
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
      });
  return input;
}

template <typename T, int alignment>
static __global__ void multimem_one_shot_all_reduce_kernel(
    T* input_mc_ptr,
    T* output_ptr,
    size_t numel,
    uint32_t** signal_pads,
    size_t rank,
    size_t world_size) {
  static_assert(alignment % sizeof(T) == 0);
  constexpr size_t numel_per_thread = alignment / sizeof(T);

  sync_remote_blocks<std::memory_order_relaxed>(signal_pads, rank, world_size);
  __syncthreads();

  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * numel_per_thread;
  auto stride = blockDim.x * gridDim.x * numel_per_thread;
  for (size_t i = offset; i < numel; i += stride) {
    auto vec = multimem_ld_reduce_add<alignment>(input_mc_ptr + i);
    at::native::memory::st_vec<alignment>(output_ptr + i, vec);
  }

  __syncthreads();
  sync_remote_blocks<std::memory_order_relaxed>(signal_pads, rank, world_size);
}

at::Tensor multimem_one_shot_all_reduce_out(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor out) {
  TORCH_CHECK(
      input.is_contiguous(),
      "multimem_one_shot_all_reduce: input must be contiguous.");
  TORCH_CHECK(
      out.is_contiguous(),
      "multimem_one_shot_all_reduce: output must be contiguous.");
  TORCH_CHECK(
      out.sizes() == input.sizes(),
      "multimem_one_shot_all_reduce: input/output size mismatch.");
  TORCH_CHECK(
      reduce_op == "sum",
      "multimem_one_shot_all_reduce: only sum is supported for now.");

  auto symm_mem = c10d::symmetric_memory::rendezvous(input, group_name);
  TORCH_CHECK(
      symm_mem != nullptr,
      "multimem_one_shot_all_reduce: input must be allocated with empty_strided_p2p().");
  TORCH_CHECK(
      symm_mem->has_multicast_support(),
      "multimem_one_shot_all_reduce: requires multicast support.");

  const size_t alignment =
      get_and_verify_alignment(input, "multimem_one_shot_all_reduce");

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel(),
      input.element_size(),
      alignment,
      1,
      8,
      1024,
      num_blocks,
      num_threads);

  AT_DISPATCH_FLOAT_AND_BFLOAT16(
      input.scalar_type(), "multimem_one_shot_all_reduce", [&]() {
        DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
          multimem_one_shot_all_reduce_kernel<scalar_t, k_alignment>
              <<<num_blocks,
                 num_threads,
                 0,
                 at::cuda::getCurrentCUDAStream()>>>(
                  reinterpret_cast<scalar_t*>(symm_mem->get_multicast_ptr()) +
                      input.storage_offset(),
                  out.data_ptr<scalar_t>(),
                  input.numel(),
                  reinterpret_cast<uint32_t**>(
                      symm_mem->get_signal_pad_ptrs_dev()),
                  symm_mem->get_rank(),
                  symm_mem->get_world_size());
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
      });
  return out;
}

at::Tensor multimem_one_shot_all_reduce(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name) {
  auto out = at::empty_like(input);
  return multimem_one_shot_all_reduce_out(input, reduce_op, group_name, out);
}

template <int alignment>
static __global__ void multimem_all_gather_kernel(
    char* input_ptr,
    char* output_mc_ptr,
    size_t bytes_per_rank,
    uint32_t** signal_pads,
    size_t rank,
    size_t world_size) {
  sync_remote_blocks<std::memory_order_relaxed>(signal_pads, rank, world_size);
  __syncthreads();

  const size_t start = bytes_per_rank * rank;

  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * alignment;
  auto stride = blockDim.x * gridDim.x * alignment;
  for (size_t i = offset; i < bytes_per_rank; i += stride) {
    auto vec = at::native::memory::ld_vec<alignment>(input_ptr + i);
    multimem_st<alignment>(output_mc_ptr + start + i, vec);
  }

  __syncthreads();
  sync_remote_blocks<std::memory_order_acq_rel>(signal_pads, rank, world_size);
}

at::Tensor multimem_all_gather_out(
    const at::Tensor& input,
    std::string group_name,
    at::Tensor out) {
  auto symm_mem = c10d::symmetric_memory::rendezvous(out, group_name);
  TORCH_CHECK(
      symm_mem != nullptr,
      "multimem_all_gather_out: output must be allocated with empty_strided_p2p().");
  TORCH_CHECK(
      symm_mem->has_multicast_support(),
      "multimem_all_gather_out: output must have multicast support.");

  TORCH_CHECK(
      input.is_contiguous(),
      "multimem_all_gather_out: input must be contiguous.");
  TORCH_CHECK(
      out.is_contiguous(),
      "multimem_all_gather_out: output must be contiguous.");

  TORCH_CHECK(
      input.dim() == out.dim(),
      "multimem_all_gather_out: input/output dimension mismatch.");

  TORCH_CHECK(
      out.sizes()[0] == input.sizes()[0] * symm_mem->get_world_size(),
      "multimem_all_gather_out: out.sizes()[0] must be equal to input.sizes[0] * world_size. (out.sizes():",
      out.sizes(),
      ", input.sizes(): ",
      input.sizes(),
      ", world_size: ",
      symm_mem->get_world_size(),
      ")");

  for (auto d = 1; d < input.dim(); ++d) {
    TORCH_CHECK(
        out.sizes()[d] == input.sizes()[d],
        "multimem_all_gather_out: all non-0th dimension of input and output must match.");
  }

  const size_t alignment =
      get_and_verify_alignment(out, "multimem_all_gather_out");

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel() * input.element_size(),
      1,
      alignment,
      1,
      8,
      1024,
      num_blocks,
      num_threads);

  DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
    multimem_all_gather_kernel<k_alignment>
        <<<num_blocks, num_threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<char*>(input.data_ptr()),
            reinterpret_cast<char*>(symm_mem->get_multicast_ptr()) +
                out.storage_offset() * out.element_size(),
            input.numel() * input.element_size(),
            reinterpret_cast<uint32_t**>(symm_mem->get_signal_pad_ptrs_dev()),
            symm_mem->get_rank(),
            symm_mem->get_world_size());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
  return out;
}

#endif //no multi-cast support on ROCm

// One-shot all-reduce is register-intensive because it stages values loaded
// from peers in registers before performing reduction. Setting the thread
// count to 512 to prevent/alleviate register spill.
constexpr size_t one_shot_all_reduce_max_num_blocks = 24;
constexpr size_t one_shot_all_reduce_max_num_threads = 512;

template <typename T, int alignment, int k_world_size>
static __launch_bounds__(one_shot_all_reduce_max_num_threads) __global__
    void one_shot_all_reduce_kernel(
        T** input_ptrs,
        T* output_ptr,
        T* input_ptr,
        size_t input_offset,
        size_t numel,
        uint32_t** signal_pads,
        size_t rank,
        size_t world_size) {
  static_assert(alignment % sizeof(T) == 0);
  constexpr size_t numel_per_thread = alignment / sizeof(T);
  // copy input to shared ptr
  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * numel_per_thread;
  auto stride = blockDim.x * gridDim.x * numel_per_thread;
  if (input_ptr) {
    for (size_t i = offset; i < numel; i += stride) {
      Vec<alignment> vec_st = at::native::memory::ld_vec<alignment>(input_ptr + i);
      at::native::memory::st_vec<alignment>(input_ptrs[rank] + input_offset + i, vec_st);
    }
  }
  // TODO make it sync with one block for no-copy case
  sync_remote_blocks<std::memory_order_acq_rel>(signal_pads, rank, world_size);
  __syncthreads();

  for (size_t i = offset; i < numel; i += stride) {
    auto vec = load_and_reduce<T, alignment, k_world_size>(
        input_ptrs, rank, world_size, input_offset + i);
    at::native::memory::st_vec<alignment>(output_ptr + i, vec);
  }

  __syncthreads();
  sync_remote_blocks<std::memory_order_acq_rel>(signal_pads, rank, world_size);
}

at::Tensor one_shot_all_reduce_out_impl(
    const at::Tensor& input,
    const std::optional<at::Tensor>& local_input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor out) {
  TORCH_CHECK(
      input.is_contiguous(), "one_shot_all_reduce: input must be contiguous.");
  TORCH_CHECK(
      out.is_contiguous(), "one_shot_all_reduce: output must be contiguous.");
  TORCH_CHECK(
      out.sizes() == input.sizes(),
      "one_shot_all_reduce: input/output size mismatch, input.sizes(): ",
      input.sizes(),
      ", output.sizes(): ",
      out.sizes());
  TORCH_CHECK(
      reduce_op == "sum",
      "one_shot_all_reduce: only sum is supported for now.");
  if (local_input.has_value()) {
    TORCH_CHECK(
        local_input->is_contiguous(),
        "one_shot_all_reduce: local input must be contiguous.");
    TORCH_CHECK(
        local_input->numel() <= input.numel(),
        "one_shot_all_reduce: local input size must be smaller than symm buffer size.");
  }
  if (input.numel() == 0) {
    TORCH_CHECK(input.scalar_type() == out.scalar_type());
    return out;
  }
  auto symm_mem = c10d::symmetric_memory::rendezvous(input, group_name);
  TORCH_CHECK(
      symm_mem != nullptr,
      "one_shot_all_reduce: input must be allocated with empty_strided_p2p().");

  const size_t alignment =
      get_and_verify_alignment(input, "one_shot_all_reduce");
  if (local_input.has_value()) {
    const size_t local_alignment =
        get_and_verify_alignment(*local_input, "one_shot_all_reduce");
    TORCH_CHECK(
        alignment == local_alignment,
        "one_shot_all_reduce: local input and symm buffer must have the same alignment.");
  }

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel(),
      input.element_size(),
      alignment,
      1,
      one_shot_all_reduce_max_num_blocks,
      one_shot_all_reduce_max_num_threads,
      num_blocks,
      num_threads);

  AT_DISPATCH_FLOAT_AND_BFLOAT16(
      input.scalar_type(), "one_shot_all_reduce", [&]() {
        DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
          DISPATCH_WORLD_SIZES(symm_mem->get_world_size(), [&]() {
            one_shot_all_reduce_kernel<scalar_t, k_alignment, k_world_size>
                <<<num_blocks,
                   num_threads,
                   0,
                   at::cuda::getCurrentCUDAStream()>>>(
                    reinterpret_cast<scalar_t**>(
                        symm_mem->get_buffer_ptrs_dev()),
                    out.data_ptr<scalar_t>(),
                    local_input.has_value() ? local_input->data_ptr<scalar_t>()
                                            : nullptr,
                    input.storage_offset(),
                    input.numel(),
                    reinterpret_cast<uint32_t**>(
                        symm_mem->get_signal_pad_ptrs_dev()),
                    symm_mem->get_rank(),
                    symm_mem->get_world_size());
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
        });
      });
  return out;
}

at::Tensor one_shot_all_reduce_out(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor out) {
  return one_shot_all_reduce_out_impl(
      input, std::nullopt, reduce_op, group_name, out);
}

at::Tensor one_shot_all_reduce_copy_out(
    const at::Tensor& input,
    const at::Tensor& local_input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor out) {
  return one_shot_all_reduce_out_impl(
      input, local_input, reduce_op, group_name, out);
}

at::Tensor one_shot_all_reduce(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name) {
  auto out = at::empty_like(input);
  return one_shot_all_reduce_out_impl(
      input, std::nullopt, reduce_op, group_name, out);
}

at::Tensor one_shot_all_reduce_copy(
    const at::Tensor& input,
    const at::Tensor& local_input,
    std::string reduce_op,
    std::string group_name) {
  auto out = at::empty_like(local_input);
  return one_shot_all_reduce_out_impl(
      input, local_input, reduce_op, group_name, out);
}

constexpr size_t two_shot_all_reduce_max_num_blocks = 24;
constexpr size_t two_shot_all_reduce_max_num_threads = 1024;

template <
    typename T,
    int alignment,
    int k_world_size,
    bool reduce_scatter = false,
    bool split_last_dim = false>
static __launch_bounds__(two_shot_all_reduce_max_num_threads) __global__
    void two_shot_all_reduce_kernel(
        T** input_ptrs,
        T* output_ptr,
        size_t input_offset,
        size_t numel,
        uint32_t** signal_pads,
        size_t rank,
        size_t world_size,
        size_t last_dim_size = 0) {
  static_assert(alignment % sizeof(T) == 0);
  constexpr size_t numel_per_thread = alignment / sizeof(T);
  int32_t N_last_dim =
      last_dim_size / world_size; // used only for split_last_dim reduce_scatter
  sync_remote_blocks<std::memory_order_acq_rel>(signal_pads, rank, world_size);
  __syncthreads();

  const size_t numel_per_rank =
      at::round_up(numel, numel_per_thread * world_size) / world_size;
  const size_t start = split_last_dim ? last_dim_size / world_size * rank
                                      : numel_per_rank * rank;

  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * numel_per_thread;
  auto stride = blockDim.x * gridDim.x * numel_per_thread;
  for (size_t i = offset; i < numel_per_rank; i += stride) {
    if constexpr (!reduce_scatter) {
      // we call reduce-scatter only with evenly divisible number of elements
      if (start + i >= numel) {
        continue;
      }
    }
    size_t idx = i;
    if constexpr (split_last_dim) {
      idx = i / N_last_dim * last_dim_size + i % N_last_dim;
    }
    auto vec = load_and_reduce<T, alignment, k_world_size>(
        input_ptrs, rank, world_size, input_offset + start + idx);
    // store to local buffer or to output
    if constexpr (reduce_scatter) {
      at::native::memory::st_vec<alignment>(output_ptr + i, vec);
    } else {
      at::native::memory::st_vec<alignment>(input_ptrs[rank] + input_offset + start + i, vec);
    }
  }

  __syncthreads();
  sync_remote_blocks<std::memory_order_acq_rel>(signal_pads, rank, world_size);
  if constexpr (reduce_scatter) {
    return;
  }
  __syncthreads();
  for (size_t i = offset; i < numel_per_rank; i += stride) {
    Vec<alignment> tmp[k_world_size];
#pragma unroll k_world_size
    for (size_t step = 0; step < k_world_size; ++step) {
      size_t remote_rank = (rank + step) % k_world_size;
      size_t remote_start = numel_per_rank * remote_rank;
      if (remote_start + i >= numel) {
        continue;
      }
      tmp[step] = at::native::memory::ld_vec<alignment>(
          input_ptrs[remote_rank] + input_offset + remote_start + i);
    }
#pragma unroll k_world_size
    for (size_t step = 0; step < k_world_size; ++step) {
      size_t remote_rank = (rank + step) % k_world_size;
      size_t remote_start = numel_per_rank * remote_rank;
      if (remote_start + i >= numel) {
        continue;
      }
      at::native::memory::st_vec<alignment>(output_ptr + remote_start + i, tmp[step]);
    }
  }
  // need to make sure all blocks exit simultaneously so that the data
  // is not corrupted by the subsequent kernels
  __syncthreads();
  sync_remote_blocks<std::memory_order_acq_rel>(signal_pads, rank, world_size);
}

template <typename T, int alignment, int k_world_size>
static __launch_bounds__(two_shot_all_reduce_max_num_threads) __global__
    void two_shot_all_reduce_kernel_inplace(
        T** input_ptrs,
        size_t input_offset,
        size_t numel,
        uint32_t** signal_pads,
        size_t rank,
        size_t world_size) {
  static_assert(alignment % sizeof(T) == 0);
  constexpr size_t numel_per_thread = alignment / sizeof(T);

  sync_remote_blocks<std::memory_order_relaxed>(signal_pads, rank, world_size);
  __syncthreads();

  const size_t numel_per_rank =
      at::round_up(numel, alignment * world_size) / world_size;
  const size_t start = numel_per_rank * rank;

  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * numel_per_thread;
  auto stride = blockDim.x * gridDim.x * numel_per_thread;
  for (size_t i = offset; i < numel_per_rank; i += stride) {
    if (start + i >= numel) {
      continue;
    }
    auto vec = load_and_reduce<T, alignment, k_world_size>(
        input_ptrs, rank, world_size, input_offset + start + i);
    for (size_t step = 0; step < world_size; ++step) {
      size_t remote_rank = (rank + step) % world_size;
      at::native::memory::st_vec<alignment>(
          input_ptrs[remote_rank] + input_offset + start + i, vec);
    }
  }

  __syncthreads();
  sync_remote_blocks<std::memory_order_acq_rel>(signal_pads, rank, world_size);
}

at::Tensor two_shot_all_reduce_impl(
    at::Tensor input,
    std::optional<at::Tensor> output,
    std::string reduce_op,
    std::string group_name) {
  TORCH_CHECK(
      input.is_contiguous(), "two_shot_all_reduce: input must be contiguous.");
  TORCH_CHECK(
      reduce_op == "sum",
      "two_shot_all_reduce: only sum is supported for now.");

  auto symm_mem = c10d::symmetric_memory::rendezvous(input, group_name);
  TORCH_CHECK(
      symm_mem != nullptr,
      "two_shot_all_reduce: input must be allocated with empty_strided_p2p().");

  const size_t alignment =
      get_and_verify_alignment(input, "two_shot_all_reduce");

  if (output.has_value()) {
    TORCH_CHECK(
        output->is_contiguous(),
        "two_shot_all_reduce: output must be contiguous.");
    const size_t output_alignment =
        get_and_verify_alignment(*output, "two_shot_all_reduce");
    TORCH_CHECK(
        alignment <= output_alignment,
        "two_shot_all_reduce: output alignment must be equal to or larger than input.");
    TORCH_CHECK(
        output->sizes() == input.sizes(),
        "two_shot_all_reduce: input/output size mismatch, input.sizes(): ",
        input.sizes(),
        ", output.sizes(): ",
        output->sizes());
    if (input.numel() == 0) {
      TORCH_CHECK(output->scalar_type() == input.scalar_type());
      return *output;
    }
  } else {
    if (input.numel() == 0) {
      return input;
    }
  }

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel(),
      input.element_size(),
      alignment,
      symm_mem->get_world_size(),
      two_shot_all_reduce_max_num_blocks,
      two_shot_all_reduce_max_num_threads,
      num_blocks,
      num_threads);

  if (!output.has_value()) {
    AT_DISPATCH_FLOAT_AND_BFLOAT16(
        input.scalar_type(), "two_shot_all_reduce", [&]() {
          DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
            DISPATCH_WORLD_SIZES(symm_mem->get_world_size(), [&]() {
              two_shot_all_reduce_kernel_inplace<
                  scalar_t,
                  k_alignment,
                  k_world_size>
                  <<<num_blocks,
                     num_threads,
                     0,
                     at::cuda::getCurrentCUDAStream()>>>(
                      reinterpret_cast<scalar_t**>(
                          symm_mem->get_buffer_ptrs_dev()),
                      input.storage_offset(),
                      input.numel(),
                      reinterpret_cast<uint32_t**>(
                          symm_mem->get_signal_pad_ptrs_dev()),
                      symm_mem->get_rank(),
                      symm_mem->get_world_size());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            });
          });
        });
    return input;
  } else {
    AT_DISPATCH_FLOAT_AND_BFLOAT16(
        input.scalar_type(), "two_shot_all_reduce", [&]() {
          DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
            DISPATCH_WORLD_SIZES_NO_DEFAULT(symm_mem->get_world_size(), [&]() {
              two_shot_all_reduce_kernel<scalar_t, k_alignment, k_world_size>
                  <<<num_blocks,
                     num_threads,
                     0,
                     at::cuda::getCurrentCUDAStream()>>>(
                      reinterpret_cast<scalar_t**>(
                          symm_mem->get_buffer_ptrs_dev()),
                      output->data_ptr<scalar_t>(),
                      input.storage_offset(),
                      input.numel(),
                      reinterpret_cast<uint32_t**>(
                          symm_mem->get_signal_pad_ptrs_dev()),
                      symm_mem->get_rank(),
                      symm_mem->get_world_size());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            });
          });
        });
    return *output;
  }
}

at::Tensor two_shot_all_reduce_(
    at::Tensor input,
    std::string reduce_op,
    std::string group_name) {
  return two_shot_all_reduce_impl(input, std::nullopt, reduce_op, group_name);
}

at::Tensor two_shot_all_reduce_out(
    at::Tensor input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor output) {
  return two_shot_all_reduce_impl(input, output, reduce_op, group_name);
}

at::Tensor reduce_scatter_out(
    at::Tensor input,
    std::string group_name,
    bool split_last_dim,
    at::Tensor output) {
  TORCH_CHECK(
      input.is_contiguous(), "reduce_scatter: input must be contiguous.");
  TORCH_CHECK(
      output.is_contiguous(), "reduce_scatter: output must be contiguous.");

  auto symm_mem = c10d::symmetric_memory::rendezvous(input, group_name);
  TORCH_CHECK(
      symm_mem != nullptr,
      "reduce_scatter: input must be allocated with empty_strided_p2p().");

  const size_t alignment = get_and_verify_alignment(input, "reduce_scatter");

  const size_t output_alignment =
      get_and_verify_alignment(input, "reduce_scatter");

  TORCH_CHECK(
      input.numel() %
              (symm_mem->get_world_size() *
               (alignment / input.element_size())) ==
          0,
      "expected number of elements to be divisible by world_size * alignment, number of elements ",
      input.numel(),
      " world size ",
      symm_mem->get_world_size(),
      "alignment ",
      alignment);

  if (split_last_dim) {
    TORCH_CHECK(input.dim() == output.dim());
    bool are_equal_except_last = std::equal(
        input.sizes().begin(), input.sizes().end() - 1, output.sizes().begin());
    TORCH_CHECK(
        are_equal_except_last,
        "reduce_scatter expected input and output to have same sizes except in the last dimension");
    TORCH_CHECK(
        output.size(-1) == input.size(-1) / symm_mem->get_world_size(),
        "reduce_scatter expected output last dim size to be input last dim size / world_size");

    TORCH_CHECK(
        input.size(-1) %
                (symm_mem->get_world_size() *
                 (alignment / input.element_size())) ==
            0,
        "expected last dimension to be divisible by world_size * alignment, last dimension ",
        input.size(-1),
        " world size ",
        symm_mem->get_world_size(),
        "alignment ",
        alignment);
  } else {
    TORCH_CHECK(input.dim() == 1, "reduce_scatter expected 1D input");
    TORCH_CHECK(output.dim() == 1, "reduce_scatter expected 1D output");
    TORCH_CHECK(output.numel() == input.numel() / symm_mem->get_world_size());
  }
  if (input.numel() == 0) {
    TORCH_CHECK(input.scalar_type() == output.scalar_type());
    return output;
  }

  TORCH_CHECK(
      output_alignment >= alignment,
      "reduce_scatter: output alignment should be not smaller than input alignment");

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel(),
      input.element_size(),
      alignment,
      symm_mem->get_world_size(),
      two_shot_all_reduce_max_num_blocks,
      two_shot_all_reduce_max_num_threads,
      num_blocks,
      num_threads);
  if (split_last_dim) {
    AT_DISPATCH_FLOAT_AND_BFLOAT16(
        input.scalar_type(), "two_shot_all_reduce", [&]() {
          DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
            DISPATCH_WORLD_SIZES_NO_DEFAULT(symm_mem->get_world_size(), [&]() {
              two_shot_all_reduce_kernel<
                  scalar_t,
                  k_alignment,
                  k_world_size,
                  true,
                  true>
                  <<<num_blocks,
                     num_threads,
                     0,
                     at::cuda::getCurrentCUDAStream()>>>(
                      reinterpret_cast<scalar_t**>(
                          symm_mem->get_buffer_ptrs_dev()),
                      output.data_ptr<scalar_t>(),
                      input.storage_offset(),
                      input.numel(),
                      reinterpret_cast<uint32_t**>(
                          symm_mem->get_signal_pad_ptrs_dev()),
                      symm_mem->get_rank(),
                      symm_mem->get_world_size(),
                      input.size(-1));
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            });
          });
        });
  } else {
    AT_DISPATCH_FLOAT_AND_BFLOAT16(
        input.scalar_type(), "two_shot_all_reduce", [&]() {
          DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
            DISPATCH_WORLD_SIZES_NO_DEFAULT(symm_mem->get_world_size(), [&]() {
              two_shot_all_reduce_kernel<
                  scalar_t,
                  k_alignment,
                  k_world_size,
                  true,
                  false>
                  <<<num_blocks,
                     num_threads,
                     0,
                     at::cuda::getCurrentCUDAStream()>>>(
                      reinterpret_cast<scalar_t**>(
                          symm_mem->get_buffer_ptrs_dev()),
                      output.data_ptr<scalar_t>(),
                      input.storage_offset(),
                      input.numel(),
                      reinterpret_cast<uint32_t**>(
                          symm_mem->get_signal_pad_ptrs_dev()),
                      symm_mem->get_rank(),
                      symm_mem->get_world_size(),
                      input.size(-1));
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            });
          });
        });
  }
  return output;
}
} // namespace
#elif defined(CUDART_VERSION) && CUDART_VERSION < 12030
namespace {
at::Tensor multimem_all_reduce_(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name) {
  TORCH_CHECK(false, "multimem_all_reduce_: requires CUDA 12.3+.");
  return input;
}

at::Tensor multimem_one_shot_all_reduce_out(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor out) {
  TORCH_CHECK(false, "multimem_one_shot_all_reduce_out: requires CUDA 12.3+.");
  return out;
}

at::Tensor multimem_one_shot_all_reduce(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name) {
  TORCH_CHECK(false, "multimem_one_shot_all_reduce: requires CUDA 12.3+.");
  return input;
}

at::Tensor multimem_all_gather_out(
    const at::Tensor& input,
    std::string group_name,
    at::Tensor out) {
  TORCH_CHECK(false, "multimem_all_gather_out: requires CUDA 12.3+.");
  return out;
}

at::Tensor one_shot_all_reduce_out(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor out) {
  TORCH_CHECK(false, "one_shot_all_reduce_out: requires CUDA 12.3+.");
  return out;
}

at::Tensor one_shot_all_reduce_copy_out(
    const at::Tensor& input,
    const at::Tensor& local_input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor out) {
  TORCH_CHECK(false, "one_shot_all_reduce_copy_out: requires CUDA 12.3+.");
  return out;
}

at::Tensor one_shot_all_reduce(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name) {
  TORCH_CHECK(false, "one_shot_all_reduce: requires CUDA 12.3+.");
  return input;
}

at::Tensor one_shot_all_reduce_copy(
    const at::Tensor& input,
    const at::Tensor& local_input,
    std::string reduce_op,
    std::string group_name) {
  TORCH_CHECK(false, "one_shot_all_reduce_copy: requires CUDA 12.3+.");
  return input;
}

at::Tensor two_shot_all_reduce_(
    at::Tensor input,
    std::string reduce_op,
    std::string group_name) {
  TORCH_CHECK(false, "two_shot_all_reduce_: requires CUDA 12.3+.");
  return input;
}

at::Tensor two_shot_all_reduce_out(
    at::Tensor input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor output) {
  TORCH_CHECK(false, "two_shot_all_reduce_out: requires CUDA 12.3+.");
  return output;
}

at::Tensor reduce_scatter_out(
    at::Tensor input,
    std::string group_name,
    bool split_last_dim,
    at::Tensor output) {
  TORCH_CHECK(false, "reduce_scatter_out: requires CUDA 12.3+.");
  return output;
}
} // namespace
#endif // #if defined(CUDART_VERSION) && CUDART_VERSION < 12030

namespace {

at::Tensor memset32_(
    at::Tensor& input,
    int64_t offset,
    int64_t val,
    int64_t count) {
  TORCH_CHECK(
      input.dim() == 1 && input.is_contiguous() &&
          input.scalar_type() == c10::ScalarType::UInt32,
      "symm_mem::memset32_: input must be a flat, contiguous uint32 tensor.");

  TORCH_CHECK(
      offset >= 0,
      "symm_mem::memset32_: offset must be greater than or equal to 0 (got ",
      offset,
      ")");

  TORCH_CHECK(
      count > 0,
      "symm_mem::memset32_: count must be a positive integer (got ",
      count,
      ")");

  TORCH_CHECK(
      val >= 0 &&
          static_cast<size_t>(val) <= std::numeric_limits<uint32_t>::max(),
      "symm_mem::memset32_: val must be in the range of "
      "[0, 4294967295] (uint32_t).")

  TORCH_CHECK(
      offset + count <= input.numel(),
      "symm_mem::memset32_: offset + count (",
      offset + count,
      ") exceeded the numel of the input (",
      input.numel(),
      ")");

  auto addr = reinterpret_cast<uint32_t*>(input.data_ptr()) + offset;
  c10::cuda::CUDAGuard guard(input.device());

#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  auto driver_api = c10::cuda::DriverAPI::get();
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemsetD32Async_(
      reinterpret_cast<hipDeviceptr_t>(addr),
      val,
      count,
      at::cuda::getCurrentCUDAStream()));
#elif defined(USE_ROCM)
  C10_HIP_CHECK(hipMemsetD32Async(reinterpret_cast<hipDeviceptr_t>(addr),
                                   val,
                                   count,
                                   at::cuda::getCurrentCUDAStream()));
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
  return input;
}

at::Tensor stream_write_value32_(
    at::Tensor& input,
    int64_t offset,
    int64_t val) {
  TORCH_CHECK(
      input.dim() == 1 && input.is_contiguous() &&
          input.scalar_type() == c10::ScalarType::UInt32,
      "symm_mem::stream_write_value32_: input must be a flat, contiguous "
      "uint32 tensor.");

  TORCH_CHECK(
      offset >= 0,
      "symm_mem::stream_write_value32_: offset must be greater than or "
      "equal to 0 (got ",
      offset,
      ")");

  TORCH_CHECK(
      val >= 0 &&
          static_cast<size_t>(val) <= std::numeric_limits<uint32_t>::max(),
      "symm_mem::stream_write_value32_: "
      "val must be in the range of [0, 4294967295] (uint32_t).")

  TORCH_CHECK(
      offset < input.numel(),
      "symm_mem::stream_write_value32_: offset (",
      offset,
      ") exceeded the numel of the input (",
      input.numel(),
      ")");

  auto addr = reinterpret_cast<uint32_t*>(input.data_ptr()) + offset;
  c10::cuda::CUDAGuard guard(input.device());

#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  auto driver_api = c10::cuda::DriverAPI::get();
  // According to the documentation of CUstreamWriteValue_flags,
  // hipStreamWriteValue32 will provide a memory fence before the write, which
  // has similar semantics to __threadfence_system() but is scoped to the
  // stream rather than a CUDA thread.
  C10_CUDA_DRIVER_CHECK(driver_api->cuStreamWriteValue32_(
      at::cuda::getCurrentCUDAStream(),
      reinterpret_cast<hipDeviceptr_t>(addr),
      val,
      0));
#elif defined(USE_ROCM)
  C10_HIP_CHECK(hipStreamWriteValue32(
                                      at::cuda::getCurrentCUDAStream(),
                                      reinterpret_cast<void*>(addr),
                                      val,
                                      0));
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
  return input;
}

} // namespace

TORCH_LIBRARY_IMPL(symm_mem, CUDA, m) {
#if defined(USE_ROCM) || defined(CUDART_VERSION)
  m.impl("one_shot_all_reduce", ::one_shot_all_reduce);
  m.impl("one_shot_all_reduce_out", ::one_shot_all_reduce_out);
  m.impl("one_shot_all_reduce_copy", ::one_shot_all_reduce_copy);
  m.impl("one_shot_all_reduce_copy_out", ::one_shot_all_reduce_copy_out);
  m.impl("two_shot_all_reduce_", ::two_shot_all_reduce_);
  m.impl("two_shot_all_reduce_out", ::two_shot_all_reduce_out);
  m.impl("reduce_scatter_out", ::reduce_scatter_out);

  m.impl("_async_input_mm", c10d::cuda::detail::async_input_mm);
#endif
#if defined(CUDART_VERSION)
  m.impl("multimem_all_reduce_", ::multimem_all_reduce_);

  // NOTE: [multimem_one_shot_all_reduce]
  // multimem.ld_reduce does not guarantee a fixed accumulation order. This
  // means that while multimem_one_shot_all_reduce is faster and has higher
  // numerical accuracy than one_shot_all_reduce, it doesn't guarantee
  // identical results across ranks. There may be use cases that can take
  // advantage of this property, but it should not be used without
  // understanding the caveats.
  m.impl("multimem_one_shot_all_reduce", ::multimem_one_shot_all_reduce);
  m.impl(
      "multimem_one_shot_all_reduce_out", ::multimem_one_shot_all_reduce_out);
  m.impl("multimem_all_gather_out", ::multimem_all_gather_out);
#endif
  m.impl("stream_write_value32_", ::stream_write_value32_);
  m.impl("memset32_", ::memset32_);
}
