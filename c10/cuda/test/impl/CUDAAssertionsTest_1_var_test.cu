#include "hip/hip_runtime.h"
#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <c10/cuda/CUDADeviceAssertion.h>
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAFunctions.h>
#include <c10/cuda/CUDAStream.h>

#include <chrono>
#include <iostream>
#include <string>
#include <thread>

using ::testing::HasSubstr;

void did_not_fail_diagnostics() {
  std::cerr
      << "c10::cuda::CUDAKernelLaunchRegistry::get_singleton_ref().enabled_at_runtime = "
      << c10::cuda::CUDAKernelLaunchRegistry::get_singleton_ref().enabled_at_runtime
      << std::endl;
  std::cerr
      << "c10::cuda::CUDAKernelLaunchRegistry::get_singleton_ref().enabled_at_compile_time = "
      << c10::cuda::CUDAKernelLaunchRegistry::get_singleton_ref().enabled_at_compile_time
      << std::endl;
  std::cerr
      << "c10::cuda::CUDAKernelLaunchRegistry::get_singleton_ref().do_all_devices_support_managed_memory = "
      << c10::cuda::CUDAKernelLaunchRegistry::get_singleton_ref()
             .do_all_devices_support_managed_memory
      << std::endl;
}

/**
 * Device kernel that takes a single integer parameter as argument and
 * will always trigger a device side assertion.
 */
__global__ void cuda_always_fail_assertion_kernel(
    const int a,
    TORCH_DSA_KERNEL_ARGS) {
  CUDA_KERNEL_ASSERT2(a != a);
}

/**
 * TEST: Triggering device side assertion on a simple <<<1,1>>> config.
 * kernel used takes only 1 variable as parameter function.
 */
void cuda_device_assertions_1_var_test() {
  const auto stream = c10::cuda::getStreamFromPool();
  TORCH_DSA_KERNEL_LAUNCH(
      cuda_always_fail_assertion_kernel,
      1, /* Blocks */
      1, /* Threads */
      0, /* Shared mem */
      stream, /* Stream */
      1);

  try {
    c10::cuda::device_synchronize();
    did_not_fail_diagnostics();
    throw std::runtime_error("Test didn't fail, but should have.");
  } catch (const c10::Error& err) {
    const auto err_str = std::string(err.what());
    ASSERT_THAT(
        err_str,
        HasSubstr("CUDA device-side assertion failures were found on GPU #0!"));
    ASSERT_THAT(
        err_str, HasSubstr("Thread ID that failed assertion = [0,0,0]"));
    ASSERT_THAT(err_str, HasSubstr("Block ID that failed assertion = [0,0,0]"));
    ASSERT_THAT(err_str, HasSubstr("Device that launched kernel = 0"));
    ASSERT_THAT(
        err_str,
        HasSubstr(
            "Name of kernel launched that led to failure = cuda_always_fail_assertion_kernel"));
    ASSERT_THAT(
        err_str, HasSubstr("File containing kernel launch = " __FILE__));
    ASSERT_THAT(
        err_str,
        HasSubstr(
            "Function containing kernel launch = " +
            std::string(__FUNCTION__)));
    ASSERT_THAT(
        err_str,
        HasSubstr(
            "Stream kernel was launched on = " + std::to_string(stream.id())));
  }
}

TEST(CUDATest, cuda_device_assertions_1_var_test) {
  GTEST_SKIP() << "CUDA device-side assertions (DSA) was not enabled at compile time.";
}
