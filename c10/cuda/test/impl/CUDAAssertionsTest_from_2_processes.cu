#include "hip/hip_runtime.h"
#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <c10/cuda/CUDADeviceAssertion.h>
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAFunctions.h>
#include <c10/cuda/CUDAStream.h>

#include <chrono>
#include <iostream>
#include <string>
#include <thread>

using ::testing::HasSubstr;

const auto max_assertions_failure_str =
    "Assertion failure " + std::to_string(C10_CUDA_DSA_ASSERTION_COUNT - 1);

/**
 * Device kernel that takes a single integer parameter as argument and
 * will always trigger a device side assertion.
 */
__global__ void cuda_always_fail_assertion_kernel(
    const int a,
    TORCH_DSA_KERNEL_ARGS) {
  CUDA_KERNEL_ASSERT2(a != a);
}

/**
 * Device kernel that takes a single integer parameter as argument and
 * will never trigger a device side assertion.
 */
__global__ void cuda_always_succeed_assertion_kernel(
    const int a,
    TORCH_DSA_KERNEL_ARGS) {
  CUDA_KERNEL_ASSERT2(a == a);
}

// Windows doesn't like `fork`
#ifndef _MSC_VER
/**
 * TEST: Triggering device side assertion from 2 different processes from CPU.
 * The following code is testing if two processes from CPU that are running
 * GPU kernels (not necessarily simultaneously) and are asserting & writing
 * to the respective UVMs, mess up anything for each other.
 * Once parent process's kernel launch fails and causes a device-side assertion
 * and is still alive when the second process is interacting with the GPU,
 * trying to launch another kernel.
 */
void cuda_device_assertions_from_2_processes() {
  const auto n1 = fork();
  if (n1 == 0) {
    // This is the parent process, that will call an assertion failure.
    // This should execute before the child process.
    // We are achieving this by putting the child process to sleep.
    TORCH_DSA_KERNEL_LAUNCH(
        cuda_always_fail_assertion_kernel,
        1, /* Blocks */
        1, /* Threads */
        0, /* Shared mem */
        c10::cuda::getStreamFromPool(), /* Stream */
        1);
    try {
      c10::cuda::device_synchronize();
      throw std::runtime_error("Test didn't fail, but should have.");
    } catch (const c10::Error& err) {
      const auto err_str = std::string(err.what());
      ASSERT_THAT(
          err_str,
          HasSubstr(
              "1 CUDA device-side assertion failures were found on GPU #0!"));
    }
    // Keep this alive so we can see what happened to the other process
    std::this_thread::sleep_for(std::chrono::milliseconds(3000));
  } else {
    // This is the child process
    // We put it to sleep for next 2 seconds, to make sure that the parent has
    // asserted a failure already.
    std::this_thread::sleep_for(std::chrono::milliseconds(2000));
    TORCH_DSA_KERNEL_LAUNCH(
        cuda_always_succeed_assertion_kernel,
        1, /* Blocks */
        1, /* Threads */
        0, /* Shared mem */
        c10::cuda::getStreamFromPool(), /* Stream */
        1);
    try {
      c10::cuda::device_synchronize();
    } catch (const c10::Error& err) {
      ASSERT_TRUE(false); // This kernel should not have failed, but did.
    }
    // End the child process
    exit(0);
  }
}

TEST(CUDATest, cuda_device_assertions_from_2_processes) {
  GTEST_SKIP() << "CUDA device-side assertions (DSA) was not enabled at compile time.";
}

#else

#endif
